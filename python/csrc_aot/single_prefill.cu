/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <c10/cuda/CUDAGuard.h>
#include <torch/extension.h>

#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/prefill_params.cuh>
#include <flashinfer/attention/variants.cuh>
#include <optional>

#include "pytorch_extension_utils.h"

namespace flashinfer {

template <uint32_t HEAD_DIM, PosEncodingMode POS_ENCODING_MODE, bool ALLOW_FP16_QK_REDUCTION,
          MaskMode MASK_MODE, typename AttentionVariant>
hipError_t SinglePrefillWithKVCacheDispatched(typename AttentionVariant::ParamsT params,
                                               typename AttentionVariant::DTypeO* tmp,
                                               hipStream_t stream);

}  // namespace flashinfer

torch::Tensor single_prefill_with_kv_cache(
    unsigned int mask_mode_code, torch::Tensor q, torch::Tensor k, torch::Tensor v,
    std::optional<torch::Tensor> maybe_packed_custom_mask, torch::Tensor tmp,
    std::optional<torch::Tensor> maybe_alibi_slopes, unsigned int layout, int32_t window_left,
    float logits_soft_cap, float sm_scale, float rope_scale, float rope_theta,
    std::optional<torch::Tensor> maybe_lse) {
  auto device = q.device();
  unsigned int head_dim = q.size(2);
  unsigned int kv_len, qo_len, num_kv_heads, num_qo_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  qo_len = q.size(0);
  num_qo_heads = q.size(1);
  uint32_t q_stride_n = q.stride(0), q_stride_h = q.stride(1), kv_stride_n, kv_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    kv_len = k.size(0);
    num_kv_heads = k.size(1);
    kv_stride_n = k.stride(0);
    kv_stride_h = k.stride(1);
  } else {
    kv_len = k.size(1);
    num_kv_heads = k.size(0);
    kv_stride_h = k.stride(0);
    kv_stride_n = k.stride(1);
  }
  const at::cuda::CUDAGuard device_guard(device);
  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto o = torch::empty_like(q, q.options());
  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == qo_len, lse.size(0), q.size(0));
    TORCH_CHECK(lse.size(1) == num_qo_heads, lse.size(1), q.size(1));
    TORCH_CHECK(lse.dtype() == torch::kFloat32, "lse must be float32");
  }

  constexpr auto POS_ENCODING_MODE = PosEncodingMode::kNone;
  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  bool use_logits_soft_cap = logits_soft_cap > 0.f;

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();

  DISPATCH_PYTORCH_QKV_DTYPE_TO_CTYPE(q_scalar_type, kv_scalar_type, q_type, kv_type, [&] {
    using DTypeQ = q_type;
    using DTypeKV = kv_type;
    using DTypeO = DTypeQ;
    return DISPATCH_mask_mode(mask_mode, MASK_MODE, [&] {
      return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
        return DISPATCH_LOGITS_SOFT_CAP(use_logits_soft_cap, USE_LOGITS_SOFT_CAP, [&] {
          using ParamsT = SinglePrefillParams<DTypeQ, DTypeKV, DTypeO>;
          using AttentionVariant =
              ComposedAttention<ParamsT, get_variant_code(
                                             /*use_custom_mask=*/MASK_MODE == MaskMode::kCustom,
                                             /*use_sliding_window=*/true, USE_LOGITS_SOFT_CAP,
                                             /*use_alibi_slopes=*/false)>;

          ParamsT params(static_cast<DTypeQ*>(q.data_ptr()), static_cast<DTypeKV*>(k.data_ptr()),
                         static_cast<DTypeKV*>(v.data_ptr()),
                         maybe_packed_custom_mask.has_value()
                             ? static_cast<uint8_t*>(maybe_packed_custom_mask->data_ptr())
                             : nullptr,
                         static_cast<DTypeO*>(o.data_ptr()),
                         /*lse=*/(maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr),
                         /*alibi_slopes=*/nullptr, num_qo_heads, num_kv_heads, qo_len, kv_len,
                         q_stride_n, q_stride_h, kv_stride_n, kv_stride_h, head_dim, window_left,
                         logits_soft_cap, sm_scale, rope_scale, rope_theta);

          hipError_t status =
              flashinfer::SinglePrefillWithKVCacheDispatched<HEAD_DIM, POS_ENCODING_MODE,
                                                             /*use_fp16_qk_reduction=*/false,
                                                             MASK_MODE, AttentionVariant>(
                  params, static_cast<DTypeO*>(tmp.data_ptr()), torch_current_stream);
          TORCH_CHECK(status == hipSuccess,
                      "SinglePrefillWithKVCache kernel launch failed, error: " +
                          std::string(hipGetErrorString(status)));
          return true;
        });
      });
    });
  });

  return o;
}
