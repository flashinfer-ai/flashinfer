/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/group_gemm/wrapper.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer::group_gemm;

void CutlassSegmentGEMMPyTorchWrapper::RegisterWorkspaceBuffer(torch::Tensor workspace_buffer) {
  handler_->RegisterWorkspace(static_cast<void*>(workspace_buffer.data_ptr()),
                              workspace_buffer.size(0) * workspace_buffer.element_size());
}

torch::Tensor CutlassSegmentGEMMPyTorchWrapper::Forward(torch::Tensor seg_indptr,
                                                        torch::Tensor weight_indices,
                                                        torch::Tensor x, torch::Tensor weight,
                                                        unsigned int batch_size,
                                                        bool weight_column_major) {
  // TODO(Zihao): Add more checks here
  CHECK_CUDA(seg_indptr);
  CHECK_CUDA(x);
  CHECK_CUDA(weight);
  CHECK_DIM(2, x);       // x: [sum(m_i), d_in]
  CHECK_DIM(3, weight);  // weight: [num_weights, d_out, d_in] if weight_column_major, [num_weights,
                         // d_in, d_out] otherwise
  int64_t cumulative_batch_size = x.size(0);
  int64_t d_out = weight_column_major ? weight.size(1) : weight.size(2);
  int64_t d_in = weight_column_major ? weight.size(2) : weight.size(1);
  CHECK_EQ(x.size(1), d_in);
  auto y = torch::zeros({cumulative_batch_size, d_out}, x.options());
  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  seg_indptr = seg_indptr.to(torch::kInt64);

  bool weight_indices_defined = weight_indices.numel() > 0;
  if (weight_indices_defined) {
    CHECK_CUDA(weight_indices);
    weight_indices = weight_indices.to(torch::kInt64);
  }

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(x.scalar_type(), c_type, [&] {
    using cutlass_t = typename cutlass_dtype<c_type>::type;
    auto status = CutlassSegmentGEMMWrapper<cutlass_t>(
        handler_.get(), static_cast<cutlass_t*>(x.data_ptr()),
        static_cast<cutlass_t*>(weight.data_ptr()), static_cast<cutlass_t*>(y.data_ptr()),
        static_cast<int64_t*>(seg_indptr.data_ptr()),
        weight_indices_defined ? static_cast<int64_t*>(weight_indices.data_ptr()) : nullptr,
        batch_size, d_in, d_out, weight_column_major, torch_current_stream);
    TORCH_CHECK(status == hipSuccess,
                "Failed to run CutlassSegmentGEMM: ", hipGetErrorString(status));
    return true;
  });

  return y;
}