/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/sampling.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

torch::Tensor sampling_from_probs(torch::Tensor probs, torch::Tensor uniform_samples) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(1, uniform_samples);  // uniform_samples: (batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(0));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(probs.device()));

  hipError_t status = sampling::SamplingFromProb(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), batch_size, vocab_size, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "SamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));
  return samples;
}

std::vector<torch::Tensor> top_p_sampling_from_probs(torch::Tensor probs,
                                                     torch::Tensor uniform_samples, double top_p) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_p_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_p_rounds = uniform_samples.size(0);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(probs.device()));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(probs.device()));

  hipError_t status = sampling::TopPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()), top_p,
      batch_size, vocab_size, max_top_p_rounds, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopPSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

std::vector<torch::Tensor> top_k_sampling_from_probs(torch::Tensor probs,
                                                     torch::Tensor uniform_samples,
                                                     unsigned int top_k) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_k_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_k_rounds = uniform_samples.size(0);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(probs.device()));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(probs.device()));

  hipError_t status = sampling::TopKSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()), top_k,
      batch_size, vocab_size, max_top_k_rounds, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopKSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

torch::Tensor top_p_renorm_prob(torch::Tensor probs, double top_p, double eps) {
  CHECK_INPUT(probs);
  CHECK_DIM(2, probs);  // probs: (batch_size, vocab_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  probs = probs.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto renorm_probs =
      torch::empty({batch_size, vocab_size}, torch::dtype(torch::kFloat32).device(probs.device()));

  hipError_t status = sampling::TopPRenormProb<float>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(renorm_probs.data_ptr()), top_p,
      eps, batch_size, vocab_size, torch_current_stream);
  TORCH_CHECK(status == hipSuccess,
              "TopPRenormProb failed with error code " + std::string(hipGetErrorString(status)));
  return renorm_probs;
}

torch::Tensor top_k_renorm_prob(torch::Tensor probs, unsigned int top_k, double eps) {
  CHECK_INPUT(probs);
  CHECK_DIM(2, probs);  // probs: (batch_size, vocab_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  probs = probs.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto renorm_probs =
      torch::empty({batch_size, vocab_size}, torch::dtype(torch::kFloat32).device(probs.device()));

  hipError_t status = sampling::TopKRenormProb<float>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(renorm_probs.data_ptr()), top_k,
      eps, batch_size, vocab_size, torch_current_stream);

  TORCH_CHECK(status == hipSuccess,
              "TopKRenormProb failed with error code " + std::string(hipGetErrorString(status)));
  return renorm_probs;
}

torch::Tensor chain_speculative_sampling(torch::Tensor draft_probs, torch::Tensor draft_token_ids,
                                         torch::Tensor uniform_samples,
                                         torch::Tensor target_probs) {
  CHECK_INPUT(draft_probs);
  CHECK_INPUT(draft_token_ids);
  CHECK_INPUT(uniform_samples);
  CHECK_INPUT(target_probs);
  CHECK_DIM(3, draft_probs);      // draft_probs: (batch_size, num_speculate_tokens, vocab_size)
  CHECK_DIM(2, draft_token_ids);  // draft_token_ids: (batch_size, num_speculate_tokens)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (batch_size, num_speculate_tokens + 1)
  CHECK_DIM(3, target_probs);  // target_probs: (batch_size, num_speculate_tokens + 1, vocab_size)
  unsigned int batch_size = draft_probs.size(0);
  unsigned int num_speculate_tokens = draft_probs.size(1);
  unsigned int vocab_size = draft_probs.size(2);
  CHECK_EQ(batch_size, draft_token_ids.size(0));
  CHECK_EQ(batch_size, uniform_samples.size(0));
  CHECK_EQ(batch_size, target_probs.size(0));
  CHECK_EQ(num_speculate_tokens + 1, uniform_samples.size(1));
  CHECK_EQ(num_speculate_tokens + 1, target_probs.size(1));
  CHECK_EQ(vocab_size, target_probs.size(2));

  draft_probs = draft_probs.to(torch::kFloat32);
  draft_token_ids = draft_token_ids.to(torch::kInt32);
  uniform_samples = uniform_samples.to(torch::kFloat32);
  target_probs = target_probs.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto output_token_ids =
      torch::empty({batch_size, num_speculate_tokens + 1},
                   torch::dtype(torch::kInt32).device(draft_token_ids.device()));

  hipError_t status = sampling::ChainSpeculativeSampling<float, int>(
      static_cast<float*>(draft_probs.data_ptr()), static_cast<int*>(draft_token_ids.data_ptr()),
      static_cast<float*>(uniform_samples.data_ptr()), static_cast<float*>(target_probs.data_ptr()),
      static_cast<int*>(output_token_ids.data_ptr()), batch_size, num_speculate_tokens, vocab_size,
      torch_current_stream);

  TORCH_CHECK(status == hipSuccess, "ChainSpeculativeSampling failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return output_token_ids;
}
