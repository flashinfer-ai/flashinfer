/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/sampling.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

torch::Tensor sampling_from_probs(torch::Tensor probs, torch::Tensor uniform_samples,
                                  bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(1, uniform_samples);  // uniform_samples: (batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(0));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));

  hipError_t status = sampling::SamplingFromProb(static_cast<float*>(probs.data_ptr()),
                                                  static_cast<float*>(uniform_samples.data_ptr()),
                                                  static_cast<int*>(samples.data_ptr()), batch_size,
                                                  vocab_size, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "SamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));
  return samples;
}

std::vector<torch::Tensor> top_p_sampling_from_probs(torch::Tensor probs,
                                                     torch::Tensor uniform_samples,
                                                     std::optional<torch::Tensor> maybe_top_p_arr,
                                                     double top_p_val, bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_p_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_p_rounds = uniform_samples.size(0);
  bool has_top_p_arr = maybe_top_p_arr.has_value();
  auto top_p_arr = maybe_top_p_arr.value_or(torch::empty({0}, torch::dtype(torch::kFloat32)));
  if (has_top_p_arr) {
    CHECK_INPUT(top_p_arr);
    CHECK_DIM(1, top_p_arr);  // top_p_arr: (batch_size,)
    CHECK_EQ(top_p_arr.size(0), batch_size);
    CHECK_EQ(top_p_arr.device(), device);
  }
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);
  top_p_arr = top_p_arr.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(device));

  hipError_t status = sampling::TopPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()),
      has_top_p_arr ? static_cast<float*>(top_p_arr.data_ptr()) : nullptr, batch_size, top_p_val,
      vocab_size, max_top_p_rounds, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopPSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

std::vector<torch::Tensor> top_k_sampling_from_probs(torch::Tensor probs,
                                                     torch::Tensor uniform_samples,
                                                     std::optional<torch::Tensor> maybe_top_k_arr,
                                                     unsigned int top_k_val, bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_k_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_k_rounds = uniform_samples.size(0);
  bool has_top_k_arr = maybe_top_k_arr.has_value();
  auto top_k_arr = maybe_top_k_arr.value_or(torch::empty({0}, torch::dtype(torch::kInt32)));
  if (has_top_k_arr) {
    CHECK_INPUT(top_k_arr);
    CHECK_DIM(1, top_k_arr);  // top_k_arr: (batch_size,)
    CHECK_EQ(top_k_arr.size(0), batch_size);
    CHECK_EQ(top_k_arr.device(), device);
  }
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);
  top_k_arr = top_k_arr.to(torch::kInt32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(device));

  hipError_t status = sampling::TopKSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()),
      has_top_k_arr ? static_cast<float*>(top_k_arr.data_ptr()) : nullptr, batch_size, top_k_val,
      vocab_size, max_top_k_rounds, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopKSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

std::vector<torch::Tensor> min_p_sampling_from_probs(torch::Tensor probs,
                                                     torch::Tensor uniform_samples,
                                                     std::optional<torch::Tensor> maybe_min_p_arr,
                                                     double min_p_val, bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_rounds, batch_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_rounds = uniform_samples.size(0);
  CHECK_EQ(uniform_samples.size(1), batch_size);
  bool has_min_p_arr = maybe_min_p_arr.has_value();
  auto min_p_arr = maybe_min_p_arr.value_or(torch::empty({0}, torch::dtype(torch::kFloat32)));
  if (has_min_p_arr) {
    CHECK_INPUT(min_p_arr);
    CHECK_DIM(1, min_p_arr);  // min_p_arr: (batch_size,)
    CHECK_EQ(min_p_arr.size(0), batch_size);
    CHECK_EQ(min_p_arr.device(), device);
  }
  min_p_arr = min_p_arr.to(torch::kFloat32);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(device));

  hipError_t status = sampling::MinPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      has_min_p_arr ? static_cast<float*>(min_p_arr.data_ptr()) : nullptr,
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()), batch_size,
      min_p_val, vocab_size, max_rounds, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "MinPSamplingFromProb failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

std::vector<torch::Tensor> top_k_top_p_sampling_from_probs(
    torch::Tensor probs, torch::Tensor uniform_samples,
    std::optional<torch::Tensor> maybe_top_k_arr, double top_k_val,
    std::optional<torch::Tensor> maybe_top_p_arr, double top_p_val, bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_rounds, batch_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_rounds = uniform_samples.size(0);
  CHECK_EQ(uniform_samples.size(1), batch_size);
  bool has_top_k_arr = maybe_top_k_arr.has_value();
  auto top_k_arr = maybe_top_k_arr.value_or(torch::empty({0}, torch::dtype(torch::kInt32)));
  if (has_top_k_arr) {
    CHECK_INPUT(top_k_arr);
    CHECK_DIM(1, top_k_arr);  // top_k_arr: (batch_size,)
    CHECK_EQ(top_k_arr.size(0), batch_size);
    CHECK_EQ(top_k_arr.device(), device);
  }
  top_k_arr = top_k_arr.to(torch::kInt32);
  bool has_top_p_arr = maybe_top_p_arr.has_value();
  auto top_p_arr = maybe_top_p_arr.value_or(torch::empty({0}, torch::dtype(torch::kFloat32)));
  if (has_top_p_arr) {
    CHECK_INPUT(top_p_arr);
    CHECK_DIM(1, top_p_arr);  // top_p_arr: (batch_size,)
    CHECK_EQ(top_p_arr.size(0), batch_size);
    CHECK_EQ(top_p_arr.device(), device);
  }
  top_p_arr = top_p_arr.to(torch::kFloat32);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(device));

  hipError_t status = sampling::TopKTopPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      has_top_k_arr ? static_cast<int*>(top_k_arr.data_ptr()) : nullptr,
      has_top_p_arr ? static_cast<float*>(top_p_arr.data_ptr()) : nullptr,
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()), batch_size,
      top_k_val, top_p_val, vocab_size, max_rounds, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopKTopPSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

torch::Tensor top_p_renorm_probs(torch::Tensor probs, std::optional<torch::Tensor> maybe_top_p_arr,
                                 double top_p_val) {
  CHECK_INPUT(probs);
  auto device = probs.device();
  CHECK_DIM(2, probs);  // probs: (batch_size, vocab_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  bool has_top_p_arr = maybe_top_p_arr.has_value();
  auto top_p_arr = maybe_top_p_arr.value_or(torch::empty({0}, torch::dtype(torch::kFloat32)));
  if (has_top_p_arr) {
    CHECK_INPUT(top_p_arr);
    CHECK_DIM(1, top_p_arr);  // top_p_arr: (batch_size,)
    CHECK_EQ(top_p_arr.size(0), batch_size);
    CHECK_EQ(top_p_arr.device(), device);
  }
  top_p_arr = top_p_arr.to(torch::kFloat32);
  probs = probs.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto renorm_probs =
      torch::empty({batch_size, vocab_size}, torch::dtype(torch::kFloat32).device(device));

  hipError_t status = sampling::TopPRenormProb<float>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(renorm_probs.data_ptr()),
      has_top_p_arr ? static_cast<float*>(top_p_arr.data_ptr()) : nullptr, batch_size, top_p_val,
      vocab_size, torch_current_stream);
  TORCH_CHECK(status == hipSuccess,
              "TopPRenormProb failed with error code " + std::string(hipGetErrorString(status)));
  return renorm_probs;
}

torch::Tensor top_k_renorm_probs(torch::Tensor probs, std::optional<torch::Tensor> maybe_top_k_arr,
                                 unsigned int top_k_val) {
  CHECK_INPUT(probs);
  auto device = probs.device();
  CHECK_DIM(2, probs);  // probs: (batch_size, vocab_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  bool has_top_k_arr = maybe_top_k_arr.has_value();
  auto top_k_arr = maybe_top_k_arr.value_or(torch::empty({0}, torch::dtype(torch::kInt32)));
  if (has_top_k_arr) {
    CHECK_INPUT(top_k_arr);
    CHECK_DIM(1, top_k_arr);  // top_k_arr: (batch_size,)
    CHECK_EQ(top_k_arr.size(0), batch_size);
    CHECK_EQ(top_k_arr.device(), device);
  }
  top_k_arr = top_k_arr.to(torch::kInt32);
  probs = probs.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto renorm_probs =
      torch::empty({batch_size, vocab_size}, torch::dtype(torch::kFloat32).device(device));

  hipError_t status = sampling::TopKRenormProb<float>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(renorm_probs.data_ptr()),
      has_top_k_arr ? static_cast<int*>(top_k_arr.data_ptr()) : nullptr, batch_size, top_k_val,
      vocab_size, torch_current_stream);

  TORCH_CHECK(status == hipSuccess,
              "TopKRenormProb failed with error code " + std::string(hipGetErrorString(status)));
  return renorm_probs;
}

torch::Tensor top_k_mask_logits(torch::Tensor logits, std::optional<torch::Tensor> maybe_top_k_arr,
                                unsigned int top_k_val) {
  CHECK_INPUT(logits);
  auto device = logits.device();
  CHECK_DIM(2, logits);  // logits: (batch_size, vocab_size)
  unsigned int batch_size = logits.size(0);
  unsigned int vocab_size = logits.size(1);
  bool has_top_k_arr = maybe_top_k_arr.has_value();
  auto top_k_arr = maybe_top_k_arr.value_or(torch::empty({0}, torch::dtype(torch::kInt32)));
  if (has_top_k_arr) {
    CHECK_INPUT(top_k_arr);
    CHECK_DIM(1, top_k_arr);  // top_k_arr: (batch_size,)
    CHECK_EQ(top_k_arr.size(0), batch_size);
    CHECK_EQ(top_k_arr.device(), device);
  }
  top_k_arr = top_k_arr.to(torch::kInt32);
  logits = logits.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto mask_logits =
      torch::empty({batch_size, vocab_size}, torch::dtype(torch::kFloat32).device(device));

  hipError_t status = sampling::TopKMaskLogits<float>(
      static_cast<float*>(logits.data_ptr()), static_cast<float*>(mask_logits.data_ptr()),
      has_top_k_arr ? static_cast<int*>(top_k_arr.data_ptr()) : nullptr, batch_size, top_k_val,
      vocab_size, torch_current_stream);

  TORCH_CHECK(status == hipSuccess,
              "TopKMaskLogits failed with error code " + std::string(hipGetErrorString(status)));
  return mask_logits;
}

std::vector<torch::Tensor> chain_speculative_sampling(
    torch::Tensor draft_probs, torch::Tensor draft_token_ids, torch::Tensor uniform_samples,
    torch::Tensor target_probs, std::optional<torch::Tensor> maybe_output_accepted_token_num,
    std::optional<torch::Tensor> maybe_output_emitted_token_num, bool deterministic) {
  CHECK_INPUT(draft_probs);
  CHECK_INPUT(draft_token_ids);
  CHECK_INPUT(uniform_samples);
  CHECK_INPUT(target_probs);
  auto device = draft_probs.device();
  CHECK_EQ(draft_token_ids.device(), device);
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_EQ(target_probs.device(), device);
  CHECK_DIM(3, draft_probs);      // draft_probs: (batch_size, num_speculate_tokens, vocab_size)
  CHECK_DIM(2, draft_token_ids);  // draft_token_ids: (batch_size, num_speculate_tokens)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (batch_size, num_speculate_tokens + 1)
  CHECK_DIM(3, target_probs);  // target_probs: (batch_size, num_speculate_tokens + 1, vocab_size)
  unsigned int batch_size = draft_probs.size(0);
  unsigned int num_speculate_tokens = draft_probs.size(1);
  unsigned int vocab_size = draft_probs.size(2);
  CHECK_EQ(batch_size, draft_token_ids.size(0));
  CHECK_EQ(batch_size, uniform_samples.size(0));
  CHECK_EQ(batch_size, target_probs.size(0));
  CHECK_EQ(num_speculate_tokens + 1, uniform_samples.size(1));
  CHECK_EQ(num_speculate_tokens + 1, target_probs.size(1));
  CHECK_EQ(vocab_size, target_probs.size(2));

  draft_probs = draft_probs.to(torch::kFloat32);
  draft_token_ids = draft_token_ids.to(torch::kInt32);
  uniform_samples = uniform_samples.to(torch::kFloat32);
  target_probs = target_probs.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto output_token_ids = torch::empty({batch_size, num_speculate_tokens + 1},
                                       torch::dtype(torch::kInt32).device(device));

  bool has_output_accepted_token_num = maybe_output_accepted_token_num.has_value();
  bool has_output_emitted_token_num = maybe_output_emitted_token_num.has_value();
  auto output_accepted_token_num = maybe_output_accepted_token_num.value_or(
      torch::zeros({batch_size}, torch::dtype(torch::kInt32).device(device)));
  auto output_emitted_token_num = maybe_output_emitted_token_num.value_or(
      torch::zeros({batch_size}, torch::dtype(torch::kInt32).device(device)));
  if (has_output_accepted_token_num) {
    CHECK_EQ(has_output_emitted_token_num, true);
    CHECK_EQ(batch_size, output_accepted_token_num.size(0));
    CHECK_EQ(batch_size, output_emitted_token_num.size(0));
  }

  hipError_t status = sampling::ChainSpeculativeSampling<float, int>(
      static_cast<float*>(draft_probs.data_ptr()), static_cast<int*>(draft_token_ids.data_ptr()),
      static_cast<float*>(uniform_samples.data_ptr()), static_cast<float*>(target_probs.data_ptr()),
      static_cast<int*>(output_token_ids.data_ptr()),
      static_cast<int*>(output_accepted_token_num.data_ptr()),
      static_cast<int*>(output_emitted_token_num.data_ptr()), batch_size, num_speculate_tokens,
      vocab_size, deterministic, torch_current_stream);

  TORCH_CHECK(status == hipSuccess, "ChainSpeculativeSampling failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {output_token_ids, output_accepted_token_num, output_emitted_token_num};
}
