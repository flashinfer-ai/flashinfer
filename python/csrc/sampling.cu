/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/sampling.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

torch::Tensor sampling_from_probs(torch::Tensor probs, torch::Tensor uniform_samples,
                                  bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(1, uniform_samples);  // uniform_samples: (batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(0));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));

  hipError_t status = sampling::SamplingFromProb(static_cast<float*>(probs.data_ptr()),
                                                  static_cast<float*>(uniform_samples.data_ptr()),
                                                  static_cast<int*>(samples.data_ptr()), batch_size,
                                                  vocab_size, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "SamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));
  return samples;
}

std::vector<torch::Tensor> top_p_sampling_from_probs(torch::Tensor probs,
                                                     torch::Tensor uniform_samples, double top_p,
                                                     bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_p_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_p_rounds = uniform_samples.size(0);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(device));

  hipError_t status = sampling::TopPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()), top_p,
      batch_size, vocab_size, max_top_p_rounds, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopPSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

std::vector<torch::Tensor> top_k_sampling_from_probs(torch::Tensor probs,
                                                     torch::Tensor uniform_samples,
                                                     unsigned int top_k, bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_k_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_k_rounds = uniform_samples.size(0);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(device));

  hipError_t status = sampling::TopKSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()), top_k,
      batch_size, vocab_size, max_top_k_rounds, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopKSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

std::vector<torch::Tensor> top_k_top_p_sampling_from_probs(torch::Tensor probs,
                                                           torch::Tensor uniform_samples,
                                                           torch::Tensor top_k, torch::Tensor top_p,
                                                           bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  CHECK_INPUT(top_k);
  CHECK_INPUT(top_p);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_EQ(top_k.device(), device);
  CHECK_EQ(top_p.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_rounds, batch_size)
  CHECK_DIM(1, top_k);            // top_k: (batch_size,)
  CHECK_DIM(1, top_p);            // top_p: (batch_size,)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_rounds = uniform_samples.size(0);
  CHECK_EQ(uniform_samples.size(1), batch_size);
  CHECK_EQ(top_k.size(0), batch_size);
  CHECK_EQ(top_p.size(0), batch_size);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);
  top_k = top_k.to(torch::kInt32);
  top_p = top_p.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(device));

  hipError_t status = sampling::TopKTopPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(top_k.data_ptr()), static_cast<float*>(top_p.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()), batch_size,
      vocab_size, max_rounds, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopKTopPSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

torch::Tensor top_p_renorm_prob(torch::Tensor probs, double top_p, double eps) {
  CHECK_INPUT(probs);
  auto device = probs.device();
  CHECK_DIM(2, probs);  // probs: (batch_size, vocab_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  probs = probs.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto renorm_probs =
      torch::empty({batch_size, vocab_size}, torch::dtype(torch::kFloat32).device(device));

  hipError_t status = sampling::TopPRenormProb<float>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(renorm_probs.data_ptr()), top_p,
      eps, batch_size, vocab_size, torch_current_stream);
  TORCH_CHECK(status == hipSuccess,
              "TopPRenormProb failed with error code " + std::string(hipGetErrorString(status)));
  return renorm_probs;
}

torch::Tensor top_k_renorm_prob(torch::Tensor probs, unsigned int top_k, double eps) {
  CHECK_INPUT(probs);
  auto device = probs.device();
  CHECK_DIM(2, probs);  // probs: (batch_size, vocab_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  probs = probs.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto renorm_probs =
      torch::empty({batch_size, vocab_size}, torch::dtype(torch::kFloat32).device(device));

  hipError_t status = sampling::TopKRenormProb<float>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(renorm_probs.data_ptr()), top_k,
      eps, batch_size, vocab_size, torch_current_stream);

  TORCH_CHECK(status == hipSuccess,
              "TopKRenormProb failed with error code " + std::string(hipGetErrorString(status)));
  return renorm_probs;
}

torch::Tensor chain_speculative_sampling(torch::Tensor draft_probs, torch::Tensor draft_token_ids,
                                         torch::Tensor uniform_samples, torch::Tensor target_probs,
                                         bool deterministic) {
  CHECK_INPUT(draft_probs);
  CHECK_INPUT(draft_token_ids);
  CHECK_INPUT(uniform_samples);
  CHECK_INPUT(target_probs);
  auto device = draft_probs.device();
  CHECK_EQ(draft_token_ids.device(), device);
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_EQ(target_probs.device(), device);
  CHECK_DIM(3, draft_probs);      // draft_probs: (batch_size, num_speculate_tokens, vocab_size)
  CHECK_DIM(2, draft_token_ids);  // draft_token_ids: (batch_size, num_speculate_tokens)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (batch_size, num_speculate_tokens + 1)
  CHECK_DIM(3, target_probs);  // target_probs: (batch_size, num_speculate_tokens + 1, vocab_size)
  unsigned int batch_size = draft_probs.size(0);
  unsigned int num_speculate_tokens = draft_probs.size(1);
  unsigned int vocab_size = draft_probs.size(2);
  CHECK_EQ(batch_size, draft_token_ids.size(0));
  CHECK_EQ(batch_size, uniform_samples.size(0));
  CHECK_EQ(batch_size, target_probs.size(0));
  CHECK_EQ(num_speculate_tokens + 1, uniform_samples.size(1));
  CHECK_EQ(num_speculate_tokens + 1, target_probs.size(1));
  CHECK_EQ(vocab_size, target_probs.size(2));

  draft_probs = draft_probs.to(torch::kFloat32);
  draft_token_ids = draft_token_ids.to(torch::kInt32);
  uniform_samples = uniform_samples.to(torch::kFloat32);
  target_probs = target_probs.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto output_token_ids = torch::empty({batch_size, num_speculate_tokens + 1},
                                       torch::dtype(torch::kInt32).device(device));

  hipError_t status = sampling::ChainSpeculativeSampling<float, int>(
      static_cast<float*>(draft_probs.data_ptr()), static_cast<int*>(draft_token_ids.data_ptr()),
      static_cast<float*>(uniform_samples.data_ptr()), static_cast<float*>(target_probs.data_ptr()),
      static_cast<int*>(output_token_ids.data_ptr()), batch_size, num_speculate_tokens, vocab_size,
      deterministic, torch_current_stream);

  TORCH_CHECK(status == hipSuccess, "ChainSpeculativeSampling failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return output_token_ids;
}
