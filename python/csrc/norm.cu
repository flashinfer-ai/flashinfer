#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cstdint>
#include <flashinfer/norm.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;

void rmsnorm(at::Tensor& output, at::Tensor& input, at::Tensor& weight, double eps,
             int64_t hip_stream) {
  CHECK_INPUT(input);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);   // input: (batch_size, hidden_size)
  CHECK_DIM(1, weight);  // weight: (hidden_size)
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);
  CHECK_EQ(output.size(0), batch_size);
  CHECK_EQ(output.size(1), hidden_size);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status = norm::RMSNorm(
        static_cast<c_type*>(input.data_ptr()), static_cast<c_type*>(weight.data_ptr()),
        static_cast<c_type*>(output.data_ptr()), batch_size, hidden_size, eps, stream);
    TORCH_CHECK(status == hipSuccess,
                "RMSNorm failed with error code " + std::string(hipGetErrorString(status)));
    return true;
  });
}

void fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight, double eps,
                       int64_t hip_stream) {
  CHECK_INPUT(input);
  CHECK_INPUT(residual);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(residual.device(), device);
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);     // input: (batch_size, hidden_size)
  CHECK_DIM(2, residual);  // residual: (batch_size, hidden_size)
  CHECK_DIM(1, weight);    // weight: (hidden_size)
  CHECK_EQ(input.size(0), residual.size(0));
  CHECK_EQ(input.size(1), residual.size(1));
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status = norm::FusedAddRMSNorm(
        static_cast<c_type*>(input.data_ptr()), static_cast<c_type*>(residual.data_ptr()),
        static_cast<c_type*>(weight.data_ptr()), batch_size, hidden_size, eps, stream);
    TORCH_CHECK(status == hipSuccess, "FusedAddRMSNorm failed with error code " +
                                           std::string(hipGetErrorString(status)));
    return true;
  });
}

void gemma_rmsnorm(at::Tensor& output, at::Tensor& input, at::Tensor& weight, double eps,
                   int64_t hip_stream) {
  CHECK_INPUT(input);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);   // input: (batch_size, hidden_size)
  CHECK_DIM(1, weight);  // weight: (hidden_size)
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);
  CHECK_EQ(output.size(0), batch_size);
  CHECK_EQ(output.size(1), hidden_size);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status = norm::GemmaRMSNorm(
        static_cast<c_type*>(input.data_ptr()), static_cast<c_type*>(weight.data_ptr()),
        static_cast<c_type*>(output.data_ptr()), batch_size, hidden_size, eps, stream);
    TORCH_CHECK(status == hipSuccess,
                "GemmaRMSNorm failed with error code " + std::string(hipGetErrorString(status)));
    return true;
  });
}

void gemma_fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight,
                             double eps, int64_t hip_stream) {
  CHECK_INPUT(input);
  CHECK_INPUT(residual);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(residual.device(), device);
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);     // input: (batch_size, hidden_size)
  CHECK_DIM(2, residual);  // residual: (batch_size, hidden_size)
  CHECK_DIM(1, weight);    // weight: (hidden_size)
  CHECK_EQ(input.size(0), residual.size(0));
  CHECK_EQ(input.size(1), residual.size(1));
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status = norm::GemmaFusedAddRMSNorm(
        static_cast<c_type*>(input.data_ptr()), static_cast<c_type*>(residual.data_ptr()),
        static_cast<c_type*>(weight.data_ptr()), batch_size, hidden_size, eps, stream);
    TORCH_CHECK(status == hipSuccess, "GemmaFusedAddRMSNorm failed with error code " +
                                           std::string(hipGetErrorString(status)));
    return true;
  });
}
