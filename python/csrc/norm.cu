#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/norm.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

torch::Tensor rmsnorm(torch::Tensor input, torch::Tensor weight, double eps) {
  CHECK_INPUT(input);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);   // input: (batch_size, hidden_size)
  CHECK_DIM(1, weight);  // weight: (hidden_size)
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto output = torch::empty_like(input);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status = norm::RMSNorm(static_cast<c_type*>(input.data_ptr()),
                                       static_cast<c_type*>(weight.data_ptr()),
                                       static_cast<c_type*>(output.data_ptr()), batch_size,
                                       hidden_size, eps, torch_current_stream);
    TORCH_CHECK(status == hipSuccess,
                "RMSNorm failed with error code " + std::string(hipGetErrorString(status)));
    return true;
  });
  return output;
}

void fused_add_rmsnorm(torch::Tensor input, torch::Tensor residual, torch::Tensor weight,
                       double eps) {
  CHECK_INPUT(input);
  CHECK_INPUT(residual);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(residual.device(), device);
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);     // input: (batch_size, hidden_size)
  CHECK_DIM(2, residual);  // residual: (batch_size, hidden_size)
  CHECK_DIM(1, weight);    // weight: (hidden_size)
  CHECK_EQ(input.size(0), residual.size(0));
  CHECK_EQ(input.size(1), residual.size(1));
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status = norm::FusedAddRMSNorm(static_cast<c_type*>(input.data_ptr()),
                                               static_cast<c_type*>(residual.data_ptr()),
                                               static_cast<c_type*>(weight.data_ptr()), batch_size,
                                               hidden_size, eps, torch_current_stream);
    TORCH_CHECK(status == hipSuccess, "FusedAddRMSNorm failed with error code " +
                                           std::string(hipGetErrorString(status)));
    return true;
  });
}

torch::Tensor gemma_rmsnorm(torch::Tensor input, torch::Tensor weight, double eps) {
  CHECK_INPUT(input);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);   // input: (batch_size, hidden_size)
  CHECK_DIM(1, weight);  // weight: (hidden_size)
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto output = torch::empty_like(input);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status = norm::GemmaRMSNorm(static_cast<c_type*>(input.data_ptr()),
                                            static_cast<c_type*>(weight.data_ptr()),
                                            static_cast<c_type*>(output.data_ptr()), batch_size,
                                            hidden_size, eps, torch_current_stream);
    TORCH_CHECK(status == hipSuccess,
                "GemmaRMSNorm failed with error code " + std::string(hipGetErrorString(status)));
    return true;
  });
  return output;
}

void gemma_fused_add_rmsnorm(torch::Tensor input, torch::Tensor residual, torch::Tensor weight,
                             double eps) {
  CHECK_INPUT(input);
  CHECK_INPUT(residual);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(residual.device(), device);
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);     // input: (batch_size, hidden_size)
  CHECK_DIM(2, residual);  // residual: (batch_size, hidden_size)
  CHECK_DIM(1, weight);    // weight: (hidden_size)
  CHECK_EQ(input.size(0), residual.size(0));
  CHECK_EQ(input.size(1), residual.size(1));
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status = norm::GemmaFusedAddRMSNorm(
        static_cast<c_type*>(input.data_ptr()), static_cast<c_type*>(residual.data_ptr()),
        static_cast<c_type*>(weight.data_ptr()), batch_size, hidden_size, eps,
        torch_current_stream);
    TORCH_CHECK(status == hipSuccess, "GemmaFusedAddRMSNorm failed with error code " +
                                           std::string(hipGetErrorString(status)));
    return true;
  });
}
