#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <flashinfer/activation.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

void silu_and_mul(torch::Tensor& out, torch::Tensor& input) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
    flashinfer::activation::act_and_mul_kernel<c_type, flashinfer::activation::silu_kernel>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()),
                                     static_cast<c_type*>(input.data_ptr()), d);

    return true;
  });
}

void gelu_tanh_and_mul(torch::Tensor& out, torch::Tensor& input) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
    flashinfer::activation::act_and_mul_kernel<c_type, flashinfer::activation::gelu_tanh_kernel>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()),
                                     static_cast<c_type*>(input.data_ptr()), d);

    return true;
  });
}

void gelu_and_mul(torch::Tensor& out, torch::Tensor& input) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
    flashinfer::activation::act_and_mul_kernel<c_type, flashinfer::activation::gelu_kernel>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()),
                                     static_cast<c_type*>(input.data_ptr()), d);

    return true;
  });
}
