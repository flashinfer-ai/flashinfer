/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

std::vector<torch::Tensor> merge_state(torch::Tensor v_a, torch::Tensor s_a, torch::Tensor v_b,
                                       torch::Tensor s_b) {
  CHECK_INPUT(v_a);
  CHECK_INPUT(s_a);
  CHECK_INPUT(v_b);
  CHECK_INPUT(s_b);
  CHECK_DIM(3, v_a);
  CHECK_DIM(2, s_a);
  CHECK_DIM(3, v_b);
  CHECK_DIM(2, s_b);
  CHECK_SHAPE(v_a, v_b);
  CHECK_SHAPE(s_a, s_b);
  CHECK_EQ(v_a.size(0), s_a.size(0));
  CHECK_EQ(v_a.size(1), s_b.size(1));
  s_a = s_a.to(torch::kFloat32);
  s_b = s_b.to(torch::kFloat32);
  unsigned int seq_len = v_a.size(0);
  unsigned int num_heads = v_a.size(1);
  unsigned int head_dim = v_a.size(2);
  auto v_merged = torch::empty_like(v_a, v_a.options());
  auto s_merged = torch::empty({seq_len, num_heads}, s_a.options());

  bool success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(v_a.scalar_type(), c_type, [&] {
    hipError_t status =
        MergeState(static_cast<c_type*>(v_a.data_ptr()), static_cast<float*>(s_a.data_ptr()),
                   static_cast<c_type*>(v_b.data_ptr()), static_cast<float*>(s_b.data_ptr()),
                   static_cast<c_type*>(v_merged.data_ptr()),
                   static_cast<float*>(s_merged.data_ptr()), seq_len, num_heads, head_dim);
    TORCH_CHECK(status == hipSuccess,
                "MergeState kernel launch failed: ", hipGetErrorString(status));
    return true;
  });

  TORCH_CHECK(success, "MergeState kernel launch failed: unsupported data type");
  return {v_merged, s_merged};
}

std::vector<torch::Tensor> merge_states(torch::Tensor v, torch::Tensor s) {
  CHECK_INPUT(v);
  CHECK_INPUT(s);
  CHECK_DIM(4, v);
  CHECK_DIM(3, s);
  CHECK_EQ(v.size(0), s.size(0));
  CHECK_EQ(v.size(1), s.size(1));
  CHECK_EQ(v.size(2), s.size(2));
  unsigned int seq_len = v.size(0);
  unsigned int num_index_sets = v.size(1);
  unsigned int num_heads = v.size(2);
  unsigned int head_dim = v.size(3);
  s = s.to(torch::kFloat32);
  auto v_merged = torch::empty({seq_len, num_heads, head_dim}, v.options());
  auto s_merged = torch::empty({seq_len, num_heads}, s.options());

  bool success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(v.scalar_type(), c_type, [&] {
    hipError_t status = MergeStates(
        static_cast<c_type*>(v.data_ptr()), static_cast<float*>(s.data_ptr()),
        static_cast<c_type*>(v_merged.data_ptr()), static_cast<float*>(s_merged.data_ptr()),
        num_index_sets, seq_len, num_heads, head_dim);
    TORCH_CHECK(status == hipSuccess,
                "MergeStates kernel launch failed: ", hipGetErrorString(status));
    return true;
  });

  TORCH_CHECK(success, "MergeStates kernel launch failed: unsupported data type");
  return {v_merged, s_merged};
}
