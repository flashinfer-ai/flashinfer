/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/prefill_attention_decl.cuh>

#include "flashinfer_ops_prefill.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

std::vector<torch::Tensor> single_prefill_with_kv_cache(
    torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor tmp, bool causal,
    unsigned int layout, unsigned int pos_encoding_mode, bool allow_fp16_qk_reduction,
    int32_t window_left, float logits_soft_cap, float sm_scale, float rope_scale, float rope_theta,
    bool return_lse) {
  CHECK_CUDA(q);
  CHECK_CUDA(k);
  CHECK_CUDA(v);
  CHECK_INPUT(tmp);
  auto device = q.device();
  CHECK_EQ(k.device(), device);
  CHECK_EQ(v.device(), device);
  CHECK_EQ(tmp.device(), device);
  CHECK_DIM(3, q);
  CHECK_DIM(3, k);
  CHECK_DIM(3, v);
  CHECK_SHAPE(k, v);
  CHECK_EQ(q.stride(2), 1);
  CHECK_EQ(k.stride(2), 1);
  CHECK_EQ(v.stride(2), 1);
  CHECK_EQ(q.size(2), k.size(2));
  CHECK_EQ(q.scalar_type(), k.scalar_type());
  CHECK_EQ(q.scalar_type(), v.scalar_type());
  unsigned int head_dim = q.size(2);
  unsigned int kv_len, qo_len, num_kv_heads, num_qo_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  qo_len = q.size(0);
  num_qo_heads = q.size(1);
  uint32_t q_stride_n = q.stride(0), q_stride_h = q.stride(1), kv_stride_n, kv_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    kv_len = k.size(0);
    num_kv_heads = k.size(1);
    kv_stride_n = k.stride(0);
    kv_stride_h = k.stride(1);
  } else {  // QKVLayout::kHND
    kv_len = k.size(1);
    num_kv_heads = k.size(0);
    kv_stride_h = k.stride(0);
    kv_stride_n = k.stride(1);
  }
  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);
  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto o = torch::empty_like(q, q.options());
  torch::Tensor lse = torch::empty({0});
  if (return_lse) {
    lse = torch::empty({qo_len, num_qo_heads}, q.options().dtype(torch::kFloat32));
  }

  const MaskMode mask_mode = causal ? MaskMode::kCausal : MaskMode::kNone;
  TORCH_CHECK(logits_soft_cap >= 0.f, "logits_soft_cap must be non-negative");
  const LogitsPostHook logits_post_hook =
      logits_soft_cap > 0.f ? LogitsPostHook::kSoftCap : LogitsPostHook::kNone;

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();
  TORCH_CHECK(q_scalar_type == kv_scalar_type,
              "q and k must have the same scalar type, but got q: ", q_scalar_type,
              " and k: ", kv_scalar_type);

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(q_scalar_type, c_type, [&] {
    return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
      return DISPATCH_mask_mode(mask_mode, MASK_MODE, [&] {
        return DISPATCH_logits_post_hook(logits_post_hook, LOGITS_POST_HOOK, [&] {
          return DISPATCH_allow_fp16_qk_reduction(
              allow_fp16_qk_reduction, ALLOW_FP16_QK_REDUCTION, [&] {
                return DISPATCH_pos_encoding_mode(
                    PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                      hipError_t status =
                          SinglePrefillWithKVCacheDispatched<HEAD_DIM, LOGITS_POST_HOOK,
                                                             POS_ENCODING_MODE,
                                                             ALLOW_FP16_QK_REDUCTION, MASK_MODE>(
                              static_cast<c_type*>(q.data_ptr()),
                              static_cast<c_type*>(k.data_ptr()),
                              static_cast<c_type*>(v.data_ptr()),
                              /*custom_mask=*/nullptr, static_cast<c_type*>(o.data_ptr()),
                              static_cast<c_type*>(tmp.data_ptr()),
                              /*lse=*/return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr,
                              num_qo_heads, num_kv_heads, qo_len, kv_len, q_stride_n, q_stride_h,
                              kv_stride_n, kv_stride_h, window_left, logits_soft_cap, sm_scale,
                              rope_scale, rope_theta, torch_current_stream);
                      TORCH_CHECK(status == hipSuccess,
                                  "SinglePrefillWithKVCache kernel launch failed, error: " +
                                      std::string(hipGetErrorString(status)));
                      return true;
                    });
              });
        });
      });
    });
  });

  if (return_lse) {
    return {o, lse};
  } else {
    return {o};
  }
}

std::vector<torch::Tensor> single_prefill_with_kv_cache_custom_mask(
    torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor packed_custom_mask,
    torch::Tensor tmp, unsigned int layout, unsigned int pos_encoding_mode,
    bool allow_fp16_qk_reduction, int32_t window_left, float logits_soft_cap, float sm_scale,
    float rope_scale, float rope_theta, bool return_lse) {
  CHECK_CUDA(q);
  CHECK_CUDA(k);
  CHECK_CUDA(v);
  CHECK_INPUT(packed_custom_mask);
  auto device = q.device();
  CHECK_EQ(k.device(), device);
  CHECK_EQ(v.device(), device);
  CHECK_EQ(packed_custom_mask.device(), device);
  CHECK_DIM(3, q);
  CHECK_DIM(3, k);
  CHECK_DIM(3, v);
  CHECK_DIM(1, packed_custom_mask);
  CHECK_SHAPE(k, v);
  CHECK_EQ(q.stride(2), 1);
  CHECK_EQ(k.stride(2), 1);
  CHECK_EQ(v.stride(2), 1);
  CHECK_EQ(q.size(2), k.size(2));
  // packed_custom_mask must be uint8
  TORCH_CHECK(packed_custom_mask.scalar_type() == torch::kUInt8,
              "packed_custom_mask must be uint8");
  unsigned int head_dim = q.size(2);
  unsigned int kv_len, qo_len, num_kv_heads, num_qo_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  qo_len = q.size(0);
  num_qo_heads = q.size(1);
  uint32_t q_stride_n = q.stride(0), q_stride_h = q.stride(1), kv_stride_n, kv_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    kv_len = k.size(0);
    num_kv_heads = k.size(1);
    kv_stride_n = k.stride(0);
    kv_stride_h = k.stride(1);
  } else {
    kv_len = k.size(1);
    num_kv_heads = k.size(0);
    kv_stride_h = k.stride(0);
    kv_stride_n = k.stride(1);
  }
  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);
  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto o = torch::empty_like(q, q.options());
  torch::Tensor lse = torch::empty({0});
  if (return_lse) {
    lse = torch::empty({qo_len, num_qo_heads}, q.options().dtype(torch::kFloat32));
  }

  constexpr MaskMode MASK_MODE = MaskMode::kCustom;
  TORCH_CHECK(logits_soft_cap >= 0.f, "logits_soft_cap must be non-negative");
  const LogitsPostHook logits_post_hook =
      logits_soft_cap > 0.f ? LogitsPostHook::kSoftCap : LogitsPostHook::kNone;

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();
  TORCH_CHECK(q_scalar_type == kv_scalar_type,
              "q and k must have the same scalar type, but got q: ", q_scalar_type,
              " and k: ", kv_scalar_type);

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(q.scalar_type(), c_type, [&] {
    return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
      return DISPATCH_logits_post_hook(logits_post_hook, LOGITS_POST_HOOK, [&] {
        return DISPATCH_allow_fp16_qk_reduction(
            allow_fp16_qk_reduction, ALLOW_FP16_QK_REDUCTION, [&] {
              return DISPATCH_pos_encoding_mode(
                  PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                    hipError_t status =
                        SinglePrefillWithKVCacheDispatched<HEAD_DIM, LOGITS_POST_HOOK,
                                                           POS_ENCODING_MODE,
                                                           ALLOW_FP16_QK_REDUCTION, MASK_MODE>(
                            static_cast<c_type*>(q.data_ptr()), static_cast<c_type*>(k.data_ptr()),
                            static_cast<c_type*>(v.data_ptr()),
                            static_cast<uint8_t*>(packed_custom_mask.data_ptr()),
                            static_cast<c_type*>(o.data_ptr()),
                            static_cast<c_type*>(tmp.data_ptr()),
                            /*lse=*/return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr,
                            num_qo_heads, num_kv_heads, qo_len, kv_len, q_stride_n, q_stride_h,
                            kv_stride_n, kv_stride_h, window_left, logits_soft_cap, sm_scale,
                            rope_scale, rope_theta, torch_current_stream);
                    TORCH_CHECK(status == hipSuccess,
                                "SinglePrefillWithKVCache kernel launch failed, error: " +
                                    std::string(hipGetErrorString(status)));
                    return true;
                  });
            });
      });
    });
  });

  if (return_lse) {
    return {o, lse};
  } else {
    return {o};
  }
}
