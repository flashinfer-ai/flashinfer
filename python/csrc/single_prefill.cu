/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "flashinfer_decl.h"
#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

std::vector<torch::Tensor> single_prefill_with_kv_cache(
    torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor tmp, bool causal,
    unsigned int layout, unsigned int pos_encoding_mode, bool allow_fp16_qk_reduction,
    float sm_scale, float rope_scale, float rope_theta, bool return_lse) {
  CHECK_INPUT(q);
  CHECK_INPUT(k);
  CHECK_INPUT(v);
  CHECK_DIM(3, q);
  CHECK_DIM(3, k);
  CHECK_DIM(3, v);
  CHECK_SHAPE(k, v);
  CHECK_EQ(q.size(2), k.size(2));
  unsigned int head_dim = q.size(2);
  unsigned int kv_len, qo_len, num_kv_heads, num_qo_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  if (kv_layout == QKVLayout::kNHD) {
    kv_len = k.size(0);
    qo_len = q.size(0);
    num_kv_heads = k.size(1);
    num_qo_heads = q.size(1);
  } else {
    kv_len = k.size(1);
    qo_len = q.size(1);
    num_kv_heads = k.size(0);
    num_qo_heads = q.size(0);
  }
  CHECK(num_qo_heads % num_kv_heads == 0);
  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto o = torch::empty_like(q, q.options());
  torch::Tensor lse = torch::empty({0});
  if (return_lse) {
    lse = torch::empty({qo_len, num_qo_heads}, q.options().dtype(torch::kFloat32));
  }

  bool success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q.scalar_type(), c_type, [&] {
    return DISPATCH_group_size(num_qo_heads / num_kv_heads, [&] {
      return DISPATCH_head_dim(head_dim, [&] {
        DISPATCH_CAUSAL(causal, CAUSAL, {
          DISPATCH_LAYOUT(kv_layout, KV_LAYOUT, {
            DISPATCH_ALLOW_FP16_QK_REDUCTION(allow_fp16_qk_reduction, ALLOW_FP16_QK_REDUCTION, {
              DISPATCH_POS_ENCODING_MODE(PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, {
                hipError_t status =
                    SinglePrefillWithKVCacheDispatched<GROUP_SIZE, HEAD_DIM, KV_LAYOUT,
                                                       POS_ENCODING_MODE, ALLOW_FP16_QK_REDUCTION,
                                                       CAUSAL>(
                        static_cast<c_type*>(q.data_ptr()), static_cast<c_type*>(k.data_ptr()),
                        static_cast<c_type*>(v.data_ptr()), static_cast<c_type*>(o.data_ptr()),
                        static_cast<float*>(tmp.data_ptr()),
                        /*lse=*/return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr,
                        num_kv_heads, qo_len, kv_len, sm_scale, rope_scale, rope_theta,
                        torch_current_stream);
                TORCH_CHECK(status == hipSuccess,
                            "SinglePrefillWithKVCache kernel launch failed, error: " +
                                std::string(hipGetErrorString(status)));
              });
            });
          });
        });
        return true;
      });
    });
  });

  TORCH_CHECK(success, "SinglePrefillWithKVCache kernel launch failed, error: unknown dtype");
  if (return_lse) {
    return {o, lse};
  } else {
    return {o};
  }
}
