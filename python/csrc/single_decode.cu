/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/decode_attention_decl.cuh>

#include "flashinfer_ops_decode.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

torch::Tensor single_decode_with_kv_cache(torch::Tensor q, torch::Tensor k, torch::Tensor v,
                                          torch::Tensor tmp, unsigned int pos_encoding_mode,
                                          unsigned int layout, int window_left,
                                          float logits_soft_cap, float sm_scale, float rope_scale,
                                          float rope_theta) {
  CHECK_INPUT(q);
  CHECK_INPUT(k);
  CHECK_INPUT(v);
  CHECK_INPUT(tmp);
  auto device = q.device();
  CHECK_EQ(k.device(), device);
  CHECK_EQ(v.device(), device);
  CHECK_EQ(tmp.device(), device);
  CHECK_DIM(2, q);
  CHECK_DIM(3, k);
  CHECK_DIM(3, v);
  CHECK_SHAPE(k, v);
  CHECK_EQ(q.size(1), k.size(2));
  CHECK_EQ(v.scalar_type(), k.scalar_type());
  unsigned int num_qo_heads = q.size(0);
  unsigned int head_dim = q.size(1);
  unsigned int kv_len, num_kv_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  if (kv_layout == QKVLayout::kNHD) {
    kv_len = k.size(0);
    num_kv_heads = k.size(1);
  } else {
    num_kv_heads = k.size(0);
    kv_len = k.size(1);
  }
  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);
  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto o = torch::empty_like(q);

  TORCH_CHECK(logits_soft_cap >= 0.f, "logits_soft_cap must be non-negative");
  const LogitsPostHook logits_post_hook =
      logits_soft_cap > 0.f ? LogitsPostHook::kSoftCap : LogitsPostHook::kNone;

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();

  if (q_scalar_type == kv_scalar_type) {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q_scalar_type, qkv_type, [&] {
      return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
        return DISPATCH_logits_post_hook(logits_post_hook, LOGITS_POST_HOOK, [&] {
          return DISPATCH_pos_encoding_mode(
              PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                hipError_t status = SingleDecodeWithKVCacheDispatched<HEAD_DIM, LOGITS_POST_HOOK,
                                                                       POS_ENCODING_MODE>(
                    static_cast<qkv_type*>(q.data_ptr()), static_cast<qkv_type*>(k.data_ptr()),
                    static_cast<qkv_type*>(v.data_ptr()), static_cast<qkv_type*>(o.data_ptr()),
                    static_cast<qkv_type*>(tmp.data_ptr()), num_qo_heads, num_kv_heads, kv_len,
                    kv_layout, window_left, logits_soft_cap, sm_scale, rope_scale, rope_theta,
                    torch_current_stream);
                TORCH_CHECK(status == hipSuccess,
                            "SingleDecodeWithKVCache kernel launch failed, error: " +
                                std::string(hipGetErrorString(status)));
                return true;
              });
        });
      });
    });
  } else {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(q_scalar_type, q_type, [&] {
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(kv_scalar_type, kv_type, [&] {
        return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
          return DISPATCH_logits_post_hook(logits_post_hook, LOGITS_POST_HOOK, [&] {
            return DISPATCH_pos_encoding_mode(
                PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                  hipError_t status = SingleDecodeWithKVCacheDispatched<HEAD_DIM, LOGITS_POST_HOOK,
                                                                         POS_ENCODING_MODE>(
                      static_cast<q_type*>(q.data_ptr()), static_cast<kv_type*>(k.data_ptr()),
                      static_cast<kv_type*>(v.data_ptr()), static_cast<q_type*>(o.data_ptr()),
                      static_cast<q_type*>(tmp.data_ptr()), num_qo_heads, num_kv_heads, kv_len,
                      kv_layout, window_left, logits_soft_cap, sm_scale, rope_scale, rope_theta,
                      torch_current_stream);
                  TORCH_CHECK(status == hipSuccess,
                              "SingleDecodeWithKVCache kernel launch failed, error: " +
                                  std::string(hipGetErrorString(status)));
                  return true;
                });
          });
        });
      });
    });
  }

  return o;
}
