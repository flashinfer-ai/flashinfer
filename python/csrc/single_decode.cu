/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/decode_attention_decl.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

torch::Tensor single_decode_with_kv_cache(torch::Tensor q, torch::Tensor k, torch::Tensor v,
                                          torch::Tensor tmp, unsigned int pos_encoding_mode,
                                          unsigned int layout, float sm_scale, float rope_scale,
                                          float rope_theta) {
  CHECK_INPUT(q);
  CHECK_INPUT(k);
  CHECK_INPUT(v);
  CHECK_DIM(2, q);
  CHECK_DIM(3, k);
  CHECK_DIM(3, v);
  CHECK_SHAPE(k, v);
  CHECK_EQ(q.size(1), k.size(2));
  unsigned int num_qo_heads = q.size(0);
  unsigned int head_dim = q.size(1);
  unsigned int kv_len, num_kv_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  if (kv_layout == QKVLayout::kNHD) {
    kv_len = k.size(0);
    num_kv_heads = k.size(1);
  } else {
    num_kv_heads = k.size(0);
    kv_len = k.size(1);
  }
  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);
  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto o = torch::empty_like(
      q, q.options().dtype(is_float8_tensor(q) ? torch::kFloat16 : q.scalar_type()));

  if (is_float8_tensor(q)) {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(q.scalar_type(), c_type, [&] {
      return DISPATCH_group_size(num_qo_heads / num_kv_heads, GROUP_SIZE, [&] {
        return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
          return DISPATCH_kv_layout(kv_layout, KV_LAYOUT, [&] {
            return DISPATCH_pos_encoding_mode(
                PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                  hipError_t status =
                      SingleDecodeWithKVCacheDispatched<GROUP_SIZE, HEAD_DIM, KV_LAYOUT,
                                                        POS_ENCODING_MODE>(
                          static_cast<c_type*>(q.data_ptr()), static_cast<c_type*>(k.data_ptr()),
                          static_cast<c_type*>(v.data_ptr()), static_cast<nv_half*>(o.data_ptr()),
                          static_cast<nv_half*>(tmp.data_ptr()), num_kv_heads, kv_len, sm_scale,
                          rope_scale, rope_theta, torch_current_stream);
                  TORCH_CHECK(status == hipSuccess,
                              "SingleDecodeWithKVCache kernel launch failed, error: " +
                                  std::string(hipGetErrorString(status)));
                  return true;
                });
          });
        });
      });
    });
  } else {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q.scalar_type(), c_type, [&] {
      return DISPATCH_group_size(num_qo_heads / num_kv_heads, GROUP_SIZE, [&] {
        return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
          return DISPATCH_kv_layout(kv_layout, KV_LAYOUT, [&] {
            return DISPATCH_pos_encoding_mode(
                PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                  hipError_t status =
                      SingleDecodeWithKVCacheDispatched<GROUP_SIZE, HEAD_DIM, KV_LAYOUT,
                                                        POS_ENCODING_MODE>(
                          static_cast<c_type*>(q.data_ptr()), static_cast<c_type*>(k.data_ptr()),
                          static_cast<c_type*>(v.data_ptr()), static_cast<c_type*>(o.data_ptr()),
                          static_cast<c_type*>(tmp.data_ptr()), num_kv_heads, kv_len, sm_scale,
                          rope_scale, rope_theta, torch_current_stream);
                  TORCH_CHECK(status == hipSuccess,
                              "SingleDecodeWithKVCache kernel launch failed, error: " +
                                  std::string(hipGetErrorString(status)));
                  return true;
                });
          });
        });
      });
    });
  }

  return o;
}
