/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/decode_attention_decl.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

std::vector<torch::Tensor> batch_decode_with_padded_kv_cache(
    torch::Tensor q, torch::Tensor k_padded, torch::Tensor v_padded, unsigned int layout,
    unsigned int pos_encoding_mode, float sm_scale, float rope_scale, float rope_theta,
    bool return_lse) {
  CHECK_INPUT(q);
  CHECK_INPUT(k_padded);
  CHECK_INPUT(v_padded);
  CHECK_DIM(3, q);
  CHECK_DIM(4, k_padded);
  CHECK_DIM(4, v_padded);
  CHECK_SHAPE(k_padded, v_padded);
  CHECK_EQ(q.size(0), k_padded.size(0));
  CHECK_EQ(q.size(2), k_padded.size(3));
  unsigned int batch_size = q.size(0);
  unsigned int num_qo_heads = q.size(1);
  unsigned int head_dim = q.size(2);
  unsigned int padded_kv_len, num_kv_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  if (kv_layout == QKVLayout::kNHD) {
    padded_kv_len = k_padded.size(1);
    num_kv_heads = k_padded.size(2);
  } else {
    padded_kv_len = k_padded.size(2);
    num_kv_heads = k_padded.size(1);
  }
  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto o = torch::empty_like(
      q, q.options().dtype(is_float8_tensor(q) ? torch::kFloat16 : q.scalar_type()));
  torch::Tensor lse = torch::empty({0});
  if (return_lse) {
    lse = torch::empty({batch_size, num_qo_heads}, q.options()).to(torch::kFloat32);
  }

  if (is_float8_tensor(q)) {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(q.scalar_type(), c_type, [&] {
      return DISPATCH_group_size(num_qo_heads / num_kv_heads, GROUP_SIZE, [&] {
        return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
          return DISPATCH_pos_encoding_mode(
              PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                return DISPATCH_kv_layout(kv_layout, KV_LAYOUT, [&] {
                  nv_half* tmp = nullptr;
                  hipError_t status =
                      BatchDecodeWithPaddedKVCacheDispatched<GROUP_SIZE, HEAD_DIM, KV_LAYOUT,
                                                             POS_ENCODING_MODE, c_type, nv_half>(
                          static_cast<c_type*>(q.data_ptr()),
                          static_cast<c_type*>(k_padded.data_ptr()),
                          static_cast<c_type*>(v_padded.data_ptr()),
                          static_cast<nv_half*>(o.data_ptr()),
                          /*tmp=*/tmp,
                          /*lse=*/return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr,
                          batch_size, padded_kv_len, num_qo_heads, sm_scale, rope_scale, rope_theta,
                          torch_current_stream);
                  TORCH_CHECK(status == hipSuccess,
                              "BatchDecodeWithPaddedKVCache failed with error code ", status);
                  return true;
                });
              });
        });
      });
    });
  } else {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q.scalar_type(), c_type, [&] {
      c_type* tmp = nullptr;
      return DISPATCH_group_size(num_qo_heads / num_kv_heads, GROUP_SIZE, [&] {
        return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
          return DISPATCH_pos_encoding_mode(
              PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                return DISPATCH_kv_layout(kv_layout, KV_LAYOUT, [&] {
                  hipError_t status = BatchDecodeWithPaddedKVCacheDispatched<
                      GROUP_SIZE, HEAD_DIM, KV_LAYOUT, POS_ENCODING_MODE, c_type, c_type>(
                      static_cast<c_type*>(q.data_ptr()), static_cast<c_type*>(k_padded.data_ptr()),
                      static_cast<c_type*>(v_padded.data_ptr()), static_cast<c_type*>(o.data_ptr()),
                      /*tmp=*/tmp,
                      /*lse=*/return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr,
                      batch_size, padded_kv_len, num_qo_heads, sm_scale, rope_scale, rope_theta,
                      torch_current_stream);
                  TORCH_CHECK(status == hipSuccess,
                              "BatchDecodeWithPaddedKVCache failed with error code ", status);
                  return true;
                });
              });
        });
      });
    });
  }

  if (return_lse) {
    return {o, lse};
  } else {
    return {o};
  }
}

void BatchDecodeWithPagedKVCachePyTorchWrapper::BeginForward(
    torch::Tensor workspace_buffer, torch::Tensor indptr, torch::Tensor last_page_len,
    unsigned int batch_size, unsigned int num_qo_heads, unsigned int num_kv_heads,
    unsigned int head_dim, unsigned int page_size, unsigned int pos_encoding_mode,
    torch::Tensor empty_data) {
  // NOTE(zihao): not necessary to be CUDA tensor
  CHECK_CONTIGUOUS(indptr);
  CHECK_CONTIGUOUS(last_page_len);
  CHECK_CONTIGUOUS(workspace_buffer);
  CHECK_DIM(1, indptr);
  CHECK_DIM(1, last_page_len);
  CHECK_DIM(1, workspace_buffer);
  CHECK_EQ(indptr.scalar_type(), torch::kInt32);
  CHECK_EQ(indptr.scalar_type(), torch::kInt32);
  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);
  size_t workspace_size_in_bytes = workspace_buffer.size(0) * workspace_buffer.element_size();
  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  handler_->SetCUDAStream(torch_current_stream);

  if (is_float8_tensor(empty_data)) {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(empty_data.scalar_type(), c_type, [&] {
      return DISPATCH_group_size(num_qo_heads / num_kv_heads, GROUP_SIZE, [&] {
        return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
          return DISPATCH_kv_layout(kv_layout_, KV_LAYOUT, [&] {
            return DISPATCH_pos_encoding_mode(
                PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                  if (handler_->IsCUDAGraphMode()) {
                    // NOTE(Zihao): use runtime dispatch because template function is not virtual
                    auto cuda_graph_handler_ =
                        dynamic_cast<CUDAGraphBatchDecodeHandler*>(handler_.get());
                    hipError_t status = cuda_graph_handler_->CUDAGraphBeginForwardDispatched<
                        GROUP_SIZE, HEAD_DIM, PageStorage::kIndices, KV_LAYOUT, POS_ENCODING_MODE,
                        c_type, nv_half, int32_t>(static_cast<void*>(workspace_buffer.data_ptr()),
                                                  workspace_size_in_bytes,
                                                  static_cast<int32_t*>(indptr.data_ptr()),
                                                  static_cast<int32_t*>(last_page_len.data_ptr()),
                                                  batch_size, num_qo_heads, page_size);
                    TORCH_CHECK(status == hipSuccess,
                                "BatchDecodeWithPagedKVCache (CUDAGraph Mode) failed with error ",
                                hipGetErrorString(status));
                  } else {
                    hipError_t status = handler_->BeginForwardDispatched<
                        GROUP_SIZE, HEAD_DIM, PageStorage::kIndices, KV_LAYOUT, POS_ENCODING_MODE,
                        c_type, nv_half, int32_t>(static_cast<void*>(workspace_buffer.data_ptr()),
                                                  workspace_size_in_bytes,
                                                  static_cast<int32_t*>(indptr.data_ptr()),
                                                  static_cast<int32_t*>(last_page_len.data_ptr()),
                                                  batch_size, num_qo_heads, page_size);
                    TORCH_CHECK(status == hipSuccess,
                                "BatchDecodeWithPagedKVCache failed with error ",
                                hipGetErrorString(status));
                  }
                  return true;
                });
          });
        });
      });
    });
  } else {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE(empty_data.scalar_type(), c_type, [&] {
      return DISPATCH_group_size(num_qo_heads / num_kv_heads, GROUP_SIZE, [&] {
        return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
          return DISPATCH_kv_layout(kv_layout_, KV_LAYOUT, [&] {
            return DISPATCH_pos_encoding_mode(
                PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                  if (handler_->IsCUDAGraphMode()) {
                    // NOTE(Zihao): use runtime dispatch because template function is not virtual
                    auto cuda_graph_handler_ =
                        dynamic_cast<CUDAGraphBatchDecodeHandler*>(handler_.get());
                    auto status = cuda_graph_handler_->CUDAGraphBeginForwardDispatched<
                        GROUP_SIZE, HEAD_DIM, PageStorage::kIndices, KV_LAYOUT, POS_ENCODING_MODE,
                        c_type, c_type, int32_t>(static_cast<void*>(workspace_buffer.data_ptr()),
                                                 workspace_size_in_bytes,
                                                 static_cast<int32_t*>(indptr.data_ptr()),
                                                 static_cast<int32_t*>(last_page_len.data_ptr()),
                                                 batch_size, num_qo_heads, page_size);
                    TORCH_CHECK(status == hipSuccess,
                                "BatchDecodeWithPagedKVCache (CUDAGraph Mode) failed with error ",
                                hipGetErrorString(status));
                  } else {
                    hipError_t status = handler_->BeginForwardDispatched<
                        GROUP_SIZE, HEAD_DIM, PageStorage::kIndices, KV_LAYOUT, POS_ENCODING_MODE,
                        c_type, c_type, int32_t>(static_cast<void*>(workspace_buffer.data_ptr()),
                                                 workspace_size_in_bytes,
                                                 static_cast<int32_t*>(indptr.data_ptr()),
                                                 static_cast<int32_t*>(last_page_len.data_ptr()),
                                                 batch_size, num_qo_heads, page_size);
                    TORCH_CHECK(status == hipSuccess,
                                "BatchDecodeWithPagedKVCache failed with error ",
                                hipGetErrorString(status));
                  }
                  return true;
                });
          });
        });
      });
    });
  }
}

void BatchDecodeWithPagedKVCachePyTorchWrapper::EndForward() { handler_->EndForward(); }

void BatchDecodeWithPagedKVCachePyTorchWrapper::UpdatePageLockedBufferSize(
    unsigned int max_workspace_size_in_bytes) {
  handler_->UpdatePageLockedBufferSize(max_workspace_size_in_bytes);
}

std::vector<torch::Tensor> BatchDecodeWithPagedKVCachePyTorchWrapper::Forward(
    torch::Tensor q, torch::Tensor paged_kv_data, torch::Tensor paged_kv_indptr,
    torch::Tensor paged_kv_indices, torch::Tensor paged_kv_last_page_len,
    unsigned int pos_encoding_mode, float sm_scale, float rope_scale, float rope_theta,
    bool return_lse) {
  CHECK_INPUT(q);
  CHECK_INPUT(paged_kv_data);
  CHECK_INPUT(paged_kv_indptr);
  CHECK_INPUT(paged_kv_indices);
  CHECK_INPUT(paged_kv_last_page_len);
  CHECK_DIM(3, q);                       // (B, H_qo, D)
  CHECK_DIM(1, paged_kv_last_page_len);  // (B,)
  CHECK_DIM(1, paged_kv_indptr);         // (B+1,)
  CHECK_DIM(1, paged_kv_indices);        // (nnz,)
  // (num_max_pages, 2, H_kv, page_size, head_dim) for HND
  // (num_max_pages, 2, page_size, H_kv, head_dim) for NHD
  CHECK_DIM(5, paged_kv_data);
  int64_t batch_size = q.size(0);
  int64_t num_qo_heads = q.size(1);
  int64_t head_dim = q.size(2);
  int64_t num_kv_heads, page_size;
  if (kv_layout_ == QKVLayout::kHND) {
    num_kv_heads = paged_kv_data.size(2);
    page_size = paged_kv_data.size(3);
  } else {
    page_size = paged_kv_data.size(2);
    num_kv_heads = paged_kv_data.size(3);
  }
  CHECK_EQ(paged_kv_data.size(1), 2);
  CHECK_EQ(paged_kv_data.size(4), head_dim);
  CHECK_EQ(paged_kv_indptr.size(0), batch_size + 1);
  CHECK_EQ(paged_kv_last_page_len.size(0), batch_size);
  // TODO(Zihao): support dispatching to different data types
  CHECK_EQ(paged_kv_indptr.scalar_type(), torch::kInt32);
  CHECK_EQ(paged_kv_indices.scalar_type(), torch::kInt32);
  CHECK_EQ(paged_kv_last_page_len.scalar_type(), torch::kInt32);
  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  torch::Tensor o = torch::empty_like(
      q, q.options().dtype(is_float8_tensor(q) ? torch::kFloat16 : q.scalar_type()));
  torch::Tensor lse;
  if (return_lse) {
    lse = torch::empty({batch_size, num_qo_heads}, q.options()).to(torch::kFloat32);
  }

  if (is_float8_tensor(q)) {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(q.scalar_type(), c_type, [&] {
      return DISPATCH_kv_layout(kv_layout_, KV_LAYOUT, [&] {
        return DISPATCH_group_size(num_qo_heads / num_kv_heads, GROUP_SIZE, [&] {
          return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
            return DISPATCH_pos_encoding_mode(
                PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                  paged_kv_t<PageStorage::kIndices, KV_LAYOUT, c_type, int32_t> paged_kv(
                      num_kv_heads, page_size, head_dim, batch_size,
                      static_cast<c_type*>(paged_kv_data.data_ptr()),
                      static_cast<int32_t*>(paged_kv_indices.data_ptr()),
                      static_cast<int32_t*>(paged_kv_indptr.data_ptr()),
                      static_cast<int32_t*>(paged_kv_last_page_len.data_ptr()));
                  hipError_t status = BatchDecodeWithPagedKVCacheWrapperDispatched<
                      PageStorage::kIndices, KV_LAYOUT, GROUP_SIZE, HEAD_DIM, POS_ENCODING_MODE,
                      c_type, nv_half, int32_t>(
                      handler_.get(), static_cast<c_type*>(q.data_ptr()), /*q_offset=*/nullptr,
                      paged_kv, static_cast<nv_half*>(o.data_ptr()),
                      /*lse=*/(return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr),
                      sm_scale, rope_scale, rope_theta,
                      /*stream=*/torch_current_stream);
                  TORCH_CHECK(status == hipSuccess,
                              "BatchDecodeWithPagedKVCache failed with error ",
                              hipGetErrorString(status));
                  return true;
                });
          });
        });
      });
    });
  } else {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q.scalar_type(), c_type, [&] {
      return DISPATCH_kv_layout(kv_layout_, KV_LAYOUT, [&] {
        return DISPATCH_group_size(num_qo_heads / num_kv_heads, GROUP_SIZE, [&] {
          return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
            return DISPATCH_pos_encoding_mode(
                PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                  paged_kv_t<PageStorage::kIndices, KV_LAYOUT, c_type, int32_t> paged_kv(
                      num_kv_heads, page_size, head_dim, batch_size,
                      static_cast<c_type*>(paged_kv_data.data_ptr()),
                      static_cast<int32_t*>(paged_kv_indices.data_ptr()),
                      static_cast<int32_t*>(paged_kv_indptr.data_ptr()),
                      static_cast<int32_t*>(paged_kv_last_page_len.data_ptr()));
                  hipError_t status = BatchDecodeWithPagedKVCacheWrapperDispatched<
                      PageStorage::kIndices, KV_LAYOUT, GROUP_SIZE, HEAD_DIM, POS_ENCODING_MODE,
                      c_type, c_type, int32_t>(
                      handler_.get(), static_cast<c_type*>(q.data_ptr()), /*q_offset=*/nullptr,
                      paged_kv, static_cast<c_type*>(o.data_ptr()),
                      /*lse=*/(return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr),
                      sm_scale, rope_scale, rope_theta,
                      /*stream=*/torch_current_stream);
                  TORCH_CHECK(status == hipSuccess,
                              "BatchDecodeWithPagedKVCache failed with error ",
                              hipGetErrorString(status));
                  return true;
                });
          });
        });
      });
    });
  }

  if (return_lse) {
    return {o, lse};
  } else {
    return {o};
  }
}
