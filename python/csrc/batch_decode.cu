/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/decode_attention_decl.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

std::vector<torch::Tensor> batch_decode_with_padded_kv_cache(
    torch::Tensor q, torch::Tensor k_padded, torch::Tensor v_padded, unsigned int layout,
    unsigned int pos_encoding_mode, float sm_scale, float rope_scale, float rope_theta,
    bool return_lse) {
  CHECK_INPUT(q);
  CHECK_INPUT(k_padded);
  CHECK_INPUT(v_padded);
  CHECK_DIM(3, q);
  CHECK_DIM(4, k_padded);
  CHECK_DIM(4, v_padded);
  CHECK_SHAPE(k_padded, v_padded);
  CHECK_EQ(q.size(0), k_padded.size(0));
  CHECK_EQ(q.size(2), k_padded.size(3));
  unsigned int batch_size = q.size(0);
  unsigned int num_qo_heads = q.size(1);
  unsigned int head_dim = q.size(2);
  unsigned int padded_kv_len, num_kv_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  if (kv_layout == QKVLayout::kNHD) {
    padded_kv_len = k_padded.size(1);
    num_kv_heads = k_padded.size(2);
  } else {
    padded_kv_len = k_padded.size(2);
    num_kv_heads = k_padded.size(1);
  }

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  auto o = torch::empty_like(
      q, q.options().dtype(is_float8_tensor(q) ? torch::kFloat16 : q.scalar_type()));
  torch::Tensor lse = torch::empty({0});
  if (return_lse) {
    lse = torch::empty({batch_size, num_qo_heads}, q.options()).to(torch::kFloat32);
  }

  bool success;
  if (is_float8_tensor(q)) {
    success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(q.scalar_type(), c_type, [&] {
      nv_half* tmp = nullptr;
      hipError_t status = BatchDecodeWithPaddedKVCache<c_type, nv_half>(
          static_cast<c_type*>(q.data_ptr()), static_cast<c_type*>(k_padded.data_ptr()),
          static_cast<c_type*>(v_padded.data_ptr()), static_cast<nv_half*>(o.data_ptr()),
          /*tmp=*/tmp,
          /*lse=*/return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr, batch_size,
          padded_kv_len, num_qo_heads, num_kv_heads, head_dim, kv_layout,
          PosEncodingMode(pos_encoding_mode), sm_scale, rope_scale, rope_theta,
          torch_current_stream);
      TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPaddedKVCache failed with error code ",
                  status);
      return true;
    });
  } else {
    success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q.scalar_type(), c_type, [&] {
      c_type* tmp = nullptr;
      hipError_t status = BatchDecodeWithPaddedKVCache<c_type, c_type>(
          static_cast<c_type*>(q.data_ptr()), static_cast<c_type*>(k_padded.data_ptr()),
          static_cast<c_type*>(v_padded.data_ptr()), static_cast<c_type*>(o.data_ptr()),
          /*tmp=*/tmp,
          /*lse=*/return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr, batch_size,
          padded_kv_len, num_qo_heads, num_kv_heads, head_dim, kv_layout,
          PosEncodingMode(pos_encoding_mode), sm_scale, rope_scale, rope_theta,
          torch_current_stream);
      TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPaddedKVCache failed with error code ",
                  status);
      return true;
    });
  }
  TORCH_CHECK(success, "BatchDecodeWithPaddedKVCache kernel launch failed: supported data type");

  if (return_lse) {
    return {o, lse};
  } else {
    return {o};
  }
}

void BatchDecodeWithPagedKVCachePyTorchWrapper::BeginForward(
    torch::Tensor workspace_buffer, torch::Tensor indptr, torch::Tensor last_page_len,
    unsigned int batch_size, unsigned int num_qo_heads, unsigned int num_kv_heads,
    unsigned int head_dim, unsigned int page_size, unsigned int pos_encoding_mode,
    torch::Tensor empty_data) {
  // NOTE(zihao): not necessary to be CUDA tensor
  CHECK_CONTIGUOUS(indptr);
  CHECK_CONTIGUOUS(last_page_len);
  CHECK_CONTIGUOUS(workspace_buffer);
  CHECK_DIM(1, indptr);
  CHECK_DIM(1, last_page_len);
  CHECK_DIM(1, workspace_buffer);
  CHECK_EQ(indptr.scalar_type(), torch::kInt32);
  CHECK_EQ(indptr.scalar_type(), torch::kInt32);
  size_t workspace_size_in_bytes = workspace_buffer.size(0) * workspace_buffer.element_size();
  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  handler_.SetCUDAStream(torch_current_stream);

  bool success;
  if (is_float8_tensor(empty_data)) {
    success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(empty_data.scalar_type(), c_type, [&] {
      DISPATCH_LAYOUT(kv_layout_, KV_LAYOUT, {
        hipError_t status =
            handler_.BeginForward<PageStorage::kIndices, KV_LAYOUT, c_type, nv_half, int32_t>(
                static_cast<void*>(workspace_buffer.data_ptr()), workspace_size_in_bytes,
                static_cast<int32_t*>(indptr.data_ptr()),
                static_cast<int32_t*>(last_page_len.data_ptr()), batch_size, num_qo_heads,
                num_kv_heads, head_dim, page_size, PosEncodingMode(pos_encoding_mode));
        TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCache failed with error ",
                    hipGetErrorString(status));
        return true;
      })
    });
  } else {
    success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(empty_data.scalar_type(), c_type, [&] {
      DISPATCH_LAYOUT(kv_layout_, KV_LAYOUT, {
        hipError_t status =
            handler_.BeginForward<PageStorage::kIndices, KV_LAYOUT, c_type, c_type, int32_t>(
                static_cast<void*>(workspace_buffer.data_ptr()), workspace_size_in_bytes,
                static_cast<int32_t*>(indptr.data_ptr()),
                static_cast<int32_t*>(last_page_len.data_ptr()), batch_size, num_qo_heads,
                num_kv_heads, head_dim, page_size, PosEncodingMode(pos_encoding_mode));
        TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCache failed with error ",
                    hipGetErrorString(status));
        return true;
      })
    });
  }

  TORCH_CHECK(success, "BatchDecodeWithPagedKVCache failed to dispatch with dtype ",
              empty_data.scalar_type());
}

void BatchDecodeWithPagedKVCachePyTorchWrapper::EndForward() { handler_.EndForward(); }

std::vector<torch::Tensor> BatchDecodeWithPagedKVCachePyTorchWrapper::Forward(
    torch::Tensor q, torch::Tensor paged_kv_data, torch::Tensor paged_kv_indptr,
    torch::Tensor paged_kv_indices, torch::Tensor paged_kv_last_page_len,
    unsigned int pos_encoding_mode, float sm_scale, float rope_scale, float rope_theta,
    bool return_lse) {
  CHECK_INPUT(q);
  CHECK_INPUT(paged_kv_data);
  CHECK_INPUT(paged_kv_indptr);
  CHECK_INPUT(paged_kv_indices);
  CHECK_INPUT(paged_kv_last_page_len);
  CHECK_DIM(3, q);                       // (B, H_qo, D)
  CHECK_DIM(1, paged_kv_last_page_len);  // (B,)
  CHECK_DIM(1, paged_kv_indptr);         // (B+1,)
  CHECK_DIM(1, paged_kv_indices);        // (nnz,)
  // (num_max_pages, 2, H_kv, page_size, head_dim) for HND
  // (num_max_pages, 2, page_size, H_kv, head_dim) for NHD
  CHECK_DIM(5, paged_kv_data);
  int64_t batch_size = q.size(0);
  int64_t num_qo_heads = q.size(1);
  int64_t head_dim = q.size(2);
  int64_t num_kv_heads, page_size;
  if (kv_layout_ == QKVLayout::kHND) {
    num_kv_heads = paged_kv_data.size(2);
    page_size = paged_kv_data.size(3);
  } else {
    page_size = paged_kv_data.size(2);
    num_kv_heads = paged_kv_data.size(3);
  }
  CHECK_EQ(paged_kv_data.size(1), 2);
  CHECK_EQ(paged_kv_data.size(4), head_dim);
  CHECK_EQ(paged_kv_indptr.size(0), batch_size + 1);
  CHECK_EQ(paged_kv_last_page_len.size(0), batch_size);
  // TODO(Zihao): support dispatching to different data types
  CHECK_EQ(paged_kv_indptr.scalar_type(), torch::kInt32);
  CHECK_EQ(paged_kv_indices.scalar_type(), torch::kInt32);
  CHECK_EQ(paged_kv_last_page_len.scalar_type(), torch::kInt32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream();
  torch::Tensor o = torch::empty_like(
      q, q.options().dtype(is_float8_tensor(q) ? torch::kFloat16 : q.scalar_type()));
  torch::Tensor lse;
  if (return_lse) {
    lse = torch::empty({batch_size, num_qo_heads}, q.options()).to(torch::kFloat32);
  }

  bool success;
  if (is_float8_tensor(q)) {
    success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(q.scalar_type(), c_type, [&] {
      DISPATCH_LAYOUT(kv_layout_, KV_LAYOUT, {
        paged_kv_t<PageStorage::kIndices, KV_LAYOUT, c_type, int32_t> paged_kv(
            num_kv_heads, page_size, head_dim, batch_size,
            static_cast<c_type*>(paged_kv_data.data_ptr()),
            static_cast<int32_t*>(paged_kv_indices.data_ptr()),
            static_cast<int32_t*>(paged_kv_indptr.data_ptr()),
            static_cast<int32_t*>(paged_kv_last_page_len.data_ptr()));
        hipError_t status = BatchDecodeWithPagedKVCacheWrapper<PageStorage::kIndices, KV_LAYOUT,
                                                                c_type, nv_half, int32_t>(
            &handler_, static_cast<c_type*>(q.data_ptr()), /*q_offset=*/nullptr, paged_kv,
            static_cast<nv_half*>(o.data_ptr()),
            /*lse=*/(return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr), num_qo_heads,
            PosEncodingMode(pos_encoding_mode), sm_scale, rope_scale, rope_theta,
            /*stream=*/torch_current_stream);
        TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCache failed with error ",
                    hipGetErrorString(status));
      });
      return true;
    });
  } else {
    success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q.scalar_type(), c_type, [&] {
      DISPATCH_LAYOUT(kv_layout_, KV_LAYOUT, {
        paged_kv_t<PageStorage::kIndices, KV_LAYOUT, c_type, int32_t> paged_kv(
            num_kv_heads, page_size, head_dim, batch_size,
            static_cast<c_type*>(paged_kv_data.data_ptr()),
            static_cast<int32_t*>(paged_kv_indices.data_ptr()),
            static_cast<int32_t*>(paged_kv_indptr.data_ptr()),
            static_cast<int32_t*>(paged_kv_last_page_len.data_ptr()));
        hipError_t status = BatchDecodeWithPagedKVCacheWrapper<PageStorage::kIndices, KV_LAYOUT,
                                                                c_type, c_type, int32_t>(
            &handler_, static_cast<c_type*>(q.data_ptr()), /*q_offset=*/nullptr, paged_kv,
            static_cast<c_type*>(o.data_ptr()),
            /*lse=*/(return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr), num_qo_heads,
            PosEncodingMode(pos_encoding_mode), sm_scale, rope_scale, rope_theta,
            /*stream=*/torch_current_stream);
        TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCache failed with error ",
                    hipGetErrorString(status));
      });
      return true;
    });
  }

  TORCH_CHECK(success, "BatchDecodeWithPagedKVCache failed to dispatch with dtype ",
              q.scalar_type());

  if (return_lse) {
    return {o, lse};
  } else {
    return {o};
  }
}
