/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer.cuh>

#include "flashinfer_ops.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

torch::Tensor batch_decode_with_padded_kv_cache(torch::Tensor q, torch::Tensor k_padded,
                                                torch::Tensor v_padded, unsigned int layout,
                                                unsigned int rotary_mode, float sm_scale,
                                                float rope_scale, float rope_theta) {
  CHECK_INPUT(q);
  CHECK_INPUT(k_padded);
  CHECK_INPUT(v_padded);
  CHECK_DIM(3, q);
  CHECK_DIM(4, k_padded);
  CHECK_DIM(4, v_padded);
  CHECK_SHAPE(k_padded, v_padded);
  CHECK_EQ(q.size(0), k_padded.size(0));
  CHECK_EQ(q.size(2), k_padded.size(3));
  unsigned int batch_size = q.size(0);
  unsigned int num_qo_heads = q.size(1);
  unsigned int head_dim = q.size(2);
  unsigned int padded_kv_len, num_kv_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  if (kv_layout == QKVLayout::kNHD) {
    padded_kv_len = k_padded.size(1);
    num_kv_heads = k_padded.size(2);
  } else {
    padded_kv_len = k_padded.size(2);
    num_kv_heads = k_padded.size(1);
  }

  auto o = torch::empty_like(q, q.options());

  bool success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q.scalar_type(), c_type, [&] {
    c_type* tmp = nullptr;
    float* lse = nullptr;
    hipError_t status = BatchDecodeWithPaddedKVCache<c_type, c_type>(
        static_cast<c_type*>(q.data_ptr()), static_cast<c_type*>(k_padded.data_ptr()),
        static_cast<c_type*>(v_padded.data_ptr()), static_cast<c_type*>(o.data_ptr()),
        /*tmp=*/tmp,
        /*lse=*/lse, batch_size, padded_kv_len, num_qo_heads, num_kv_heads, head_dim, kv_layout,
        RotaryMode(rotary_mode), rope_scale, rope_theta);
    TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPaddedKVCache failed with error code ",
                status);
    return true;
  });
  TORCH_CHECK(success, "BatchDecodeWithPaddedKVCache kernel launch failed: supported data type");
  return o;
}

std::vector<torch::Tensor> batch_decode_with_padded_kv_cache_return_lse(
    torch::Tensor q, torch::Tensor k_padded, torch::Tensor v_padded, unsigned int layout,
    unsigned int rotary_mode, float sm_scale, float rope_scale, float rope_theta) {
  CHECK_INPUT(q);
  CHECK_INPUT(k_padded);
  CHECK_INPUT(v_padded);
  CHECK_DIM(3, q);
  CHECK_DIM(4, k_padded);
  CHECK_DIM(4, v_padded);
  CHECK_SHAPE(k_padded, v_padded);
  CHECK_EQ(q.size(0), k_padded.size(0));
  CHECK_EQ(q.size(2), k_padded.size(3));
  unsigned int batch_size = q.size(0);
  unsigned int num_qo_heads = q.size(1);
  unsigned int head_dim = q.size(2);
  unsigned int padded_kv_len, num_kv_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  if (kv_layout == QKVLayout::kNHD) {
    padded_kv_len = k_padded.size(1);
    num_kv_heads = k_padded.size(2);
  } else {
    padded_kv_len = k_padded.size(2);
    num_kv_heads = k_padded.size(1);
  }

  auto o = torch::empty_like(q, q.options());
  auto lse = torch::empty({batch_size, num_qo_heads}, q.options().dtype(torch::kFloat32));

  bool success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q.scalar_type(), c_type, [&] {
    hipError_t status = BatchDecodeWithPaddedKVCache<c_type, c_type>(
        static_cast<c_type*>(q.data_ptr()), static_cast<c_type*>(k_padded.data_ptr()),
        static_cast<c_type*>(v_padded.data_ptr()), static_cast<c_type*>(o.data_ptr()),
        /*tmp=*/nullptr,
        /*lse=*/static_cast<float*>(lse.data_ptr()), batch_size, padded_kv_len, num_qo_heads,
        num_kv_heads, head_dim, kv_layout, RotaryMode(rotary_mode), rope_scale, rope_theta);
    TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPaddedKVCache failed with error code ",
                status);
    return true;
  });
  TORCH_CHECK(success, "BatchDecodeWithPaddedKVCache kernel launch failed: supported data type");
  return {o, lse};
}

void BatchDecodeWithPagedKVCachePyTorchWrapper::BeginForward(
    torch::Tensor indptr, torch::Tensor last_page_len, unsigned int batch_size,
    unsigned int num_qo_heads, unsigned int num_kv_heads, unsigned int head_dim,
    unsigned int page_size, unsigned int rotary_mode, torch::Tensor empty_data) {
  // NOTE(zihao): not necessary to be CUDA tensor
  CHECK_CONTIGUOUS(indptr);
  CHECK_CONTIGUOUS(last_page_len);
  CHECK_DIM(1, indptr);
  CHECK_DIM(1, last_page_len);
  CHECK_EQ(indptr.scalar_type(), torch::kInt32);
  CHECK_EQ(indptr.scalar_type(), torch::kInt32);

  bool success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(empty_data.scalar_type(), c_type, [&] {
    SWITCH_LAYOUT(kv_layout_, KV_LAYOUT, {
      hipError_t status =
          handler_.BeginForward<PageStorage::kIndices, KV_LAYOUT, c_type, c_type, int32_t>(
              static_cast<int32_t*>(indptr.data_ptr()),
              static_cast<int32_t*>(last_page_len.data_ptr()), batch_size, num_qo_heads,
              num_kv_heads, head_dim, page_size, RotaryMode(rotary_mode));
      TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCache failed with error ",
                  hipGetErrorString(status));
      return true;
    })
  });

  TORCH_CHECK(success, "BatchDecodeWithPagedKVCache failed to dispatch with dtype ",
              empty_data.scalar_type());
}

void BatchDecodeWithPagedKVCachePyTorchWrapper::EndForward() { handler_.EndForward(); }

std::vector<torch::Tensor> BatchDecodeWithPagedKVCachePyTorchWrapper::Forward(
    torch::Tensor q, torch::Tensor paged_kv_data, torch::Tensor paged_kv_indptr,
    torch::Tensor paged_kv_indices, torch::Tensor paged_kv_last_page_len, unsigned int rotary_mode,
    float rope_scale, float rope_theta, bool return_lse) {
  CHECK_INPUT(q);
  CHECK_INPUT(paged_kv_data);
  CHECK_INPUT(paged_kv_indptr);
  CHECK_INPUT(paged_kv_indices);
  CHECK_INPUT(paged_kv_last_page_len);
  CHECK_DIM(3, q);                       // (B, H_qo, D)
  CHECK_DIM(1, paged_kv_last_page_len);  // (B,)
  CHECK_DIM(1, paged_kv_indptr);         // (B+1,)
  CHECK_DIM(1, paged_kv_indices);        // (nnz,)
  // (num_max_pages, 2, H_kv, page_size, head_dim) for HND
  // (num_max_pages, 2, page_size, H_kv, head_dim) for NHD
  CHECK_DIM(5, paged_kv_data);
  int64_t batch_size = q.size(0);
  int64_t num_qo_heads = q.size(1);
  int64_t head_dim = q.size(2);
  int64_t num_kv_heads, page_size;
  if (kv_layout_ == QKVLayout::kHND) {
    num_kv_heads = paged_kv_data.size(2);
    page_size = paged_kv_data.size(3);
  } else {
    page_size = paged_kv_data.size(2);
    num_kv_heads = paged_kv_data.size(3);
  }
  CHECK_EQ(paged_kv_data.size(1), 2);
  CHECK_EQ(paged_kv_data.size(4), head_dim);
  CHECK_EQ(paged_kv_indptr.size(0), batch_size + 1);
  CHECK_EQ(paged_kv_last_page_len.size(0), batch_size);
  // TODO(Zihao): support dispatching to different data types
  CHECK_EQ(paged_kv_indptr.scalar_type(), torch::kInt32);
  CHECK_EQ(paged_kv_indices.scalar_type(), torch::kInt32);
  CHECK_EQ(paged_kv_last_page_len.scalar_type(), torch::kInt32);

  torch::Tensor o = torch::empty_like(q, q.options());
  torch::Tensor lse;
  if (return_lse) {
    lse = torch::empty({batch_size, num_qo_heads}, q.options()).to(torch::kFloat32);
  }
  bool success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(q.scalar_type(), c_type, [&] {
    SWITCH_LAYOUT(kv_layout_, KV_LAYOUT, {
      paged_kv_t<PageStorage::kIndices, KV_LAYOUT, c_type, int32_t> paged_kv(
          num_kv_heads, page_size, head_dim, batch_size,
          static_cast<c_type*>(paged_kv_data.data_ptr()),
          static_cast<int32_t*>(paged_kv_indices.data_ptr()),
          static_cast<int32_t*>(paged_kv_indptr.data_ptr()),
          static_cast<int32_t*>(paged_kv_last_page_len.data_ptr()));
      hipError_t status = BatchDecodeWithPagedKVCacheWrapper<PageStorage::kIndices, KV_LAYOUT,
                                                              c_type, c_type, int32_t>(
          &handler_, static_cast<c_type*>(q.data_ptr()), paged_kv,
          static_cast<c_type*>(o.data_ptr()),
          /*lse=*/(return_lse ? static_cast<float*>(lse.data_ptr()) : nullptr), num_qo_heads,
          RotaryMode(rotary_mode), rope_scale, rope_theta, /*stream=*/nullptr);
      TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCache failed with error ",
                  hipGetErrorString(status));
    });
    return true;
  });

  TORCH_CHECK(success, "BatchDecodeWithPagedKVCache failed to dispatch with dtype ",
              q.scalar_type());

  if (return_lse) {
    return {o, lse};
  } else {
    return {o};
  }
}
