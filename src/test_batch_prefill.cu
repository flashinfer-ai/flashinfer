#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <gtest/gtest.h>

#include <cstdint>
#include <flashinfer/attention/prefill.cuh>

#include "cpu_reference.h"
#include "utils.h"

using namespace flashinfer;
constexpr QKVLayout kv_layout = QKVLayout::kNHD;

template <typename T>
void _TestBatchPrefillKernelOneHotCorrectness(size_t num_kv_heads, size_t num_qo_heads,
                                              size_t page_size, size_t head_dim, bool causal,
                                              PosEncodingMode pos_encoding_mode,
                                              bool allow_fp16_qk_reduction) {
  uint32_t batch_size = 9;
  std::vector<int32_t> q_lens(batch_size), kv_lens(batch_size);
  utils::vec_randint_(q_lens, 1, 15);
  utils::vec_randint_(kv_lens, 15, 257);
  std::vector<int32_t> append_indptr{0};
  for (size_t request_idx = 0; request_idx < batch_size; ++request_idx) {
    append_indptr.push_back(append_indptr.back() + kv_lens[request_idx]);
  }
  std::vector<T> kv_data;
  std::vector<int32_t> kv_indptr{0};
  std::vector<int32_t> kv_indices;
  std::vector<int32_t> kv_last_page_len;
  size_t page_counter = 0;

  std::vector<std::vector<T>> key, value;
  for (uint32_t request_idx = 0; request_idx < batch_size; ++request_idx) {
    size_t kv_len = kv_lens[request_idx];
    size_t num_pages = (kv_len + page_size - 1) / page_size;
    size_t last_page_len = (kv_len - 1) % page_size + 1;
    std::vector<T> k(kv_len * num_kv_heads * head_dim), v(kv_len * num_kv_heads * head_dim);
    utils::vec_normal_(k);
    utils::vec_normal_(v);
    key.push_back(k);
    value.push_back(v);
    kv_last_page_len.push_back(last_page_len);
    kv_indptr.push_back(kv_indptr.back() + num_pages);
    for (size_t j = 0; j < num_pages; ++j) {
      kv_indices.push_back(page_counter++);
    }
  }

  kv_data.resize(page_counter * 2 * num_kv_heads * page_size * head_dim);
  flashinfer::paged_kv_t<PageStorage::kIndices, kv_layout, T, int32_t> paged_kv_cpu(
      num_kv_heads, page_size, head_dim, batch_size, kv_data.data(), kv_indices.data(),
      kv_indptr.data(), kv_last_page_len.data());
  cpu_reference::append_paged_kv_cache<kv_layout, T, int32_t>(paged_kv_cpu, key, value,
                                                              append_indptr);

  // copy data to device
  thrust::device_vector<T> kv_data_device(kv_data);
  thrust::device_vector<int32_t> kv_indptr_device(kv_indptr);
  thrust::device_vector<int32_t> kv_indices_device(kv_indices);
  thrust::device_vector<int32_t> kv_last_page_len_device(kv_last_page_len);

  // create paged_kv object
  flashinfer::paged_kv_t<PageStorage::kIndices, kv_layout, T, int32_t> paged_kv = paged_kv_cpu;
  paged_kv.data = thrust::raw_pointer_cast(kv_data_device.data());
  paged_kv.indices = thrust::raw_pointer_cast(kv_indices_device.data());
  paged_kv.indptr = thrust::raw_pointer_cast(kv_indptr_device.data());
  paged_kv.last_page_len = thrust::raw_pointer_cast(kv_last_page_len_device.data());

  for (uint32_t request_idx = 0; request_idx < batch_size; ++request_idx) {
    // create one-hot queries
    int32_t q_len = q_lens[request_idx], kv_len = kv_lens[request_idx];
    std::vector<int32_t> q_indptr{0};
    for (uint32_t i = 0; i < batch_size; ++i) {
      q_indptr.push_back(i >= request_idx ? q_len : 0);
    }
    std::vector<T> q(q_len * num_qo_heads * head_dim);
    utils::vec_normal_(q);

    std::vector<T> o_ref = cpu_reference::single_mha<T, T>(
        q, key[request_idx], value[request_idx], q_len, kv_len, num_qo_heads, num_kv_heads,
        head_dim, causal, QKVLayout::kNHD, pos_encoding_mode);

    thrust::device_vector<int32_t> q_indptr_device(q_indptr);
    thrust::device_vector<T> q_device(q);
    thrust::device_vector<T> o_device(q_len * num_qo_heads * head_dim);

    for (uint32_t num_runs = 0; num_runs < 10; ++num_runs) {
      auto status = BatchPrefillWithPagedKVCache<PageStorage::kIndices, kv_layout, T, T, int32_t>(
          thrust::raw_pointer_cast(q_device.data()),
          thrust::raw_pointer_cast(q_indptr_device.data()), /*q_offset=*/nullptr, paged_kv,
          thrust::raw_pointer_cast(o_device.data()), /*tmp=*/nullptr,
          /*lse=*/nullptr, num_qo_heads, causal, pos_encoding_mode, allow_fp16_qk_reduction);
      EXPECT_EQ(status, hipSuccess) << "CUDA error: " + std::string(hipGetErrorString(status));
    }

    thrust::host_vector<T> o_host(o_device);
    size_t num_result_errors_atol_1e_3_rtol_1e_3 = 0;
    bool nan_detected = false;
    for (size_t i = 0; i < q_len * num_qo_heads * head_dim; ++i) {
      if (std::isnan(float(o_host[i]))) {
        nan_detected = true;
      }
      num_result_errors_atol_1e_3_rtol_1e_3 +=
          (!utils::isclose(float(o_host[i]), float(o_ref[i]), 1e-3, 1e-3));
    }
    float result_accuracy = 1. - float(num_result_errors_atol_1e_3_rtol_1e_3) /
                                     max(float(q_len * num_qo_heads * head_dim), 1.f);
    std::cout << "request_idx=" << request_idx << ", page_size=" << page_size
              << ", num_qo_heads=" << num_qo_heads << ", num_kv_heads=" << num_kv_heads
              << ", q_len=" << q_len << ", kv_len=" << kv_len << ", head_dim=" << head_dim
              << ", causal=" << causal
              << ", pos_encoding_mode=" << PosEncodingModeToString(pos_encoding_mode)
              << ", result_accuracy=" << result_accuracy << std::endl;
    EXPECT_GT(result_accuracy, 0.99) << "Result correctness test failed.";
    EXPECT_EQ(nan_detected, false) << "NaN detected in output.";
  }
}

template <typename T>
void _TestBatchPrefillKernelShortContextCorrectness(size_t num_kv_heads, size_t num_qo_heads,
                                                    size_t page_size, size_t head_dim, bool causal,
                                                    PosEncodingMode pos_encoding_mode,
                                                    bool allow_fp16_qk_reduction) {
  uint32_t batch_size = 7;
  std::vector<int32_t> q_lens(batch_size);
  utils::vec_randint_(q_lens, 1, 64);
  std::vector<int32_t> kv_lens(q_lens);
  std::vector<int32_t> q_indptr{0};
  for (uint32_t i = 0; i < batch_size; ++i) {
    q_indptr.push_back(q_indptr.back() + q_lens[i]);
  }
  std::vector<int32_t> append_indptr{0};
  for (size_t request_idx = 0; request_idx < batch_size; ++request_idx) {
    append_indptr.push_back(append_indptr.back() + kv_lens[request_idx]);
  }
  std::vector<T> kv_data;
  std::vector<int32_t> kv_indptr{0};
  std::vector<int32_t> kv_indices;
  std::vector<int32_t> kv_last_page_len;
  size_t page_counter = 0;
  std::vector<std::vector<T>> key, value;
  for (uint32_t request_idx = 0; request_idx < batch_size; ++request_idx) {
    size_t kv_len = kv_lens[request_idx];
    size_t num_pages = (kv_len + page_size - 1) / page_size;
    size_t last_page_len = (kv_len - 1) % page_size + 1;
    std::vector<T> k(kv_len * num_kv_heads * head_dim), v(kv_len * num_kv_heads * head_dim);
    utils::vec_normal_(k);
    utils::vec_normal_(v);
    key.push_back(k);
    value.push_back(v);
    kv_last_page_len.push_back(last_page_len);
    kv_indptr.push_back(kv_indptr.back() + num_pages);
    for (size_t j = 0; j < num_pages; ++j) {
      kv_indices.push_back(page_counter++);
    }
  }

  kv_data.resize(page_counter * 2 * num_kv_heads * page_size * head_dim);
  flashinfer::paged_kv_t<PageStorage::kIndices, kv_layout, T, int32_t> paged_kv_cpu(
      num_kv_heads, page_size, head_dim, batch_size, kv_data.data(), kv_indices.data(),
      kv_indptr.data(), kv_last_page_len.data());
  cpu_reference::append_paged_kv_cache<kv_layout, T, int32_t>(paged_kv_cpu, key, value,
                                                              append_indptr);

  // copy data to device
  thrust::device_vector<T> kv_data_device(kv_data);
  thrust::device_vector<int32_t> kv_indptr_device(kv_indptr);
  thrust::device_vector<int32_t> kv_indices_device(kv_indices);
  thrust::device_vector<int32_t> kv_last_page_len_device(kv_last_page_len);

  // create paged_kv object
  flashinfer::paged_kv_t<PageStorage::kIndices, kv_layout, T, int32_t> paged_kv = paged_kv_cpu;
  paged_kv.data = thrust::raw_pointer_cast(kv_data_device.data());
  paged_kv.indices = thrust::raw_pointer_cast(kv_indices_device.data());
  paged_kv.indptr = thrust::raw_pointer_cast(kv_indptr_device.data());
  paged_kv.last_page_len = thrust::raw_pointer_cast(kv_last_page_len_device.data());

  std::vector<std::vector<T>> q, o_ref;
  for (uint32_t request_idx = 0; request_idx < batch_size; ++request_idx) {
    int32_t q_len = q_lens[request_idx];
    std::vector<T> qi(q_len * num_qo_heads * head_dim);
    utils::vec_normal_(qi);
    q.push_back(qi);
  }
  for (uint32_t request_idx = 0; request_idx < batch_size; ++request_idx) {
    // create one-hot queries
    int32_t q_len = q_lens[request_idx], kv_len = kv_lens[request_idx];
    std::vector<T> o_ref_i = cpu_reference::single_mha<T, T>(
        q[request_idx], key[request_idx], value[request_idx], q_len, kv_len, num_qo_heads,
        num_kv_heads, head_dim, causal, QKVLayout::kNHD, pos_encoding_mode);
    o_ref.push_back(o_ref_i);
  }

  std::vector<T> q_concat, o_concat_ref;
  for (uint32_t request_idx = 0; request_idx < batch_size; ++request_idx) {
    q_concat.insert(q_concat.end(), q[request_idx].begin(), q[request_idx].end());
    o_concat_ref.insert(o_concat_ref.end(), o_ref[request_idx].begin(), o_ref[request_idx].end());
  }
  thrust::device_vector<T> q_device(q_concat);

  thrust::device_vector<int32_t> q_indptr_device(q_indptr);
  thrust::device_vector<T> o_device(o_concat_ref.size());

  auto status = BatchPrefillWithPagedKVCache<PageStorage::kIndices, kv_layout, T, T, int32_t>(
      thrust::raw_pointer_cast(q_device.data()), thrust::raw_pointer_cast(q_indptr_device.data()),
      /*q_offset=*/nullptr, paged_kv, thrust::raw_pointer_cast(o_device.data()),
      /*tmp=*/nullptr,
      /*lse=*/nullptr, num_qo_heads, causal, pos_encoding_mode, allow_fp16_qk_reduction);
  EXPECT_EQ(status, hipSuccess) << "CUDA error: " + std::string(hipGetErrorString(status));

  thrust::host_vector<T> o_host(o_device);
  size_t num_result_errors_atol_1e_3_rtol_1e_3 = 0;
  bool nan_detected = false;
  for (size_t i = 0; i < o_concat_ref.size(); ++i) {
    if (std::isnan(float(o_host[i]))) {
      nan_detected = true;
    }
    num_result_errors_atol_1e_3_rtol_1e_3 +=
        (!utils::isclose(float(o_host[i]), float(o_concat_ref[i]), 1e-3, 1e-3));
  }
  float result_accuracy =
      1. - float(num_result_errors_atol_1e_3_rtol_1e_3) / max(float(o_concat_ref.size()), 1.f);
  std::cout << "page_size=" << page_size << ", num_qo_heads=" << num_qo_heads
            << ", num_kv_heads=" << num_kv_heads << ", head_dim=" << head_dim
            << ", causal=" << causal
            << ", pos_encoding_mode=" << PosEncodingModeToString(pos_encoding_mode)
            << ", result_accuracy=" << result_accuracy << std::endl;
  EXPECT_GT(result_accuracy, 0.99) << "Result correctness test failed.";
  EXPECT_EQ(nan_detected, false) << "NaN detected in output.";
}

template <typename T>
void _TestBatchPrefillKernelLongContextCorrectness(size_t num_kv_heads, size_t num_qo_heads,
                                                   size_t page_size, size_t head_dim, bool causal,
                                                   PosEncodingMode pos_encoding_mode,
                                                   bool allow_fp16_qk_reduction) {
  std::vector<std::vector<std::vector<T>>> keys, values;
  std::vector<int32_t> q_lens{63}, kv_lens{2047};
  std::vector<int32_t> q_indptr{0, 63};
  std::vector<int32_t> append_indptr{0, 2047};
  std::vector<T> kv_data;
  std::vector<int32_t> kv_indptr{0};
  std::vector<int32_t> kv_indices;
  std::vector<int32_t> kv_last_page_len;
  size_t page_counter = 0;

  size_t num_pages = (kv_lens[0] + page_size - 1) / page_size;
  size_t last_page_len = (kv_lens[0] - 1) % page_size + 1;
  std::vector<T> k(kv_lens[0] * num_kv_heads * head_dim), v(kv_lens[0] * num_kv_heads * head_dim);
  utils::vec_normal_(k);
  utils::vec_normal_(v);
  kv_last_page_len.push_back(last_page_len);
  kv_indptr.push_back(kv_indptr.back() + num_pages);
  for (size_t j = 0; j < num_pages; ++j) {
    kv_indices.push_back(page_counter++);
  }

  kv_data.resize(page_counter * 1 * 2 * num_kv_heads * page_size * head_dim);
  flashinfer::paged_kv_t<PageStorage::kIndices, kv_layout, T, int32_t> paged_kv_cpu(
      num_kv_heads, page_size, head_dim, 1, kv_data.data(), kv_indices.data(), kv_indptr.data(),
      kv_last_page_len.data());
  cpu_reference::append_paged_kv_cache<kv_layout, T, int32_t>(paged_kv_cpu, {k}, {v},
                                                              append_indptr);

  // copy data to device
  thrust::device_vector<T> kv_data_device(kv_data);
  thrust::device_vector<int32_t> kv_indptr_device(kv_indptr);
  thrust::device_vector<int32_t> kv_indices_device(kv_indices);
  thrust::device_vector<int32_t> kv_last_page_len_device(kv_last_page_len);

  // create paged_kv object
  flashinfer::paged_kv_t<PageStorage::kIndices, kv_layout, T, int32_t> paged_kv = paged_kv_cpu;
  paged_kv.data = thrust::raw_pointer_cast(kv_data_device.data());
  paged_kv.indices = thrust::raw_pointer_cast(kv_indices_device.data());
  paged_kv.indptr = thrust::raw_pointer_cast(kv_indptr_device.data());
  paged_kv.last_page_len = thrust::raw_pointer_cast(kv_last_page_len_device.data());

  // create one-hot queries
  std::vector<T> q(q_lens[0] * num_qo_heads * head_dim);
  utils::vec_normal_(q);

  std::vector<T> o_ref =
      cpu_reference::single_mha<T, T>(q, k, v, q_lens[0], kv_lens[0], num_qo_heads, num_kv_heads,
                                      head_dim, causal, QKVLayout::kNHD, pos_encoding_mode);

  thrust::device_vector<int32_t> q_indptr_device(q_indptr);
  thrust::device_vector<T> q_device(q);
  thrust::device_vector<T> o_device(q_lens[0] * num_qo_heads * head_dim);

  auto status = BatchPrefillWithPagedKVCache<PageStorage::kIndices, kv_layout, T, T, int32_t>(
      thrust::raw_pointer_cast(q_device.data()), thrust::raw_pointer_cast(q_indptr_device.data()),
      /*q_offset=*/nullptr, paged_kv, thrust::raw_pointer_cast(o_device.data()),
      /*tmp=*/nullptr, /*lse=*/nullptr, num_qo_heads, causal, pos_encoding_mode,
      allow_fp16_qk_reduction);
  EXPECT_EQ(status, hipSuccess) << "CUDA error: " + std::string(hipGetErrorString(status));

  thrust::host_vector<T> o_host(o_device);
  size_t num_result_errors_atol_1e_3_rtol_1e_3 = 0;
  bool nan_detected = false;
  for (size_t i = 0; i < q_lens[0] * num_qo_heads * head_dim; ++i) {
    if (std::isnan(float(o_host[i]))) {
      nan_detected = true;
    }
    num_result_errors_atol_1e_3_rtol_1e_3 +=
        (!utils::isclose(float(o_host[i]), float(o_ref[i]), 1e-3, 1e-3));
  }
  float result_accuracy = 1. - float(num_result_errors_atol_1e_3_rtol_1e_3) /
                                   max(float(q_lens[0] * num_qo_heads * head_dim), 1.f);
  std::cout << ", page_size=" << page_size << ", num_qo_heads=" << num_qo_heads
            << ", num_kv_heads=" << num_kv_heads << ", q_len=" << q_lens[0]
            << ", kv_len=" << kv_lens[0] << ", head_dim=" << head_dim << ", causal=" << causal
            << ", pos_encoding_mode=" << PosEncodingModeToString(pos_encoding_mode)
            << ", result_accuracy=" << result_accuracy << std::endl;
  EXPECT_GT(result_accuracy, 0.99) << "Result correctness test failed.";
  EXPECT_EQ(nan_detected, false) << "NaN detected in output.";
}

template <typename T>
void TestBatchPrefillKernelOneHotCorrectness(bool allow_fp16_qk_reduction) {
  for (size_t num_kv_heads : {4, 8, 32}) {
    for (size_t num_qo_heads : {32}) {
      for (size_t page_size : {1, 7, 16}) {
        for (size_t head_dim : {64, 128, 256}) {
          for (size_t causal : {false, true}) {
            for (size_t pos_encoding_mode : {0, 1}) {
              _TestBatchPrefillKernelOneHotCorrectness<T>(
                  num_kv_heads, num_qo_heads, page_size, head_dim, causal,
                  PosEncodingMode(pos_encoding_mode), allow_fp16_qk_reduction);
            }
          }
        }
      }
    }
  }
}

template <typename T>
void TestBatchPrefillKernelShortContextCorrectness(bool allow_fp16_qk_reduction) {
  for (size_t num_kv_heads : {4, 8, 32}) {
    for (size_t num_qo_heads : {32}) {
      for (size_t page_size : {1, 7, 16}) {
        for (size_t head_dim : {64, 128, 256}) {
          for (size_t causal : {false, true}) {
            for (size_t pos_encoding_mode : {0, 1}) {
              _TestBatchPrefillKernelShortContextCorrectness<T>(
                  num_kv_heads, num_qo_heads, page_size, head_dim, causal,
                  PosEncodingMode(pos_encoding_mode), allow_fp16_qk_reduction);
            }
          }
        }
      }
    }
  }
}

template <typename T>
void TestBatchPrefillKernelLongContextCorrectness(bool allow_fp16_qk_reduction) {
  for (size_t num_kv_heads : {1, 2, 8}) {
    for (size_t num_qo_heads : {8}) {
      for (size_t page_size : {1, 7, 16}) {
        for (size_t head_dim : {64, 128, 256}) {
          for (size_t causal : {false, true}) {
            for (size_t pos_encoding_mode : {0, 1}) {
              _TestBatchPrefillKernelLongContextCorrectness<T>(
                  num_kv_heads, num_qo_heads, page_size, head_dim, causal,
                  PosEncodingMode(pos_encoding_mode), allow_fp16_qk_reduction);
            }
          }
        }
      }
    }
  }
}

TEST(FlashInferCorrectnessTest, BatchPrefillShortContextTestFP16) {
  TestBatchPrefillKernelShortContextCorrectness<half>(false);
}

TEST(FlashInferCorrectnessTest, BatchPrefillShortContextTestFP16QKHalfAccum) {
  TestBatchPrefillKernelShortContextCorrectness<half>(false);
}

TEST(FlashInferCorrectnessTest, BatchPrefillLongContextTestFP16) {
  TestBatchPrefillKernelLongContextCorrectness<half>(false);
}

TEST(FlashInferCorrectnessTest, BatchPrefillLongContextTestFP16QKHalfAccum) {
  TestBatchPrefillKernelLongContextCorrectness<half>(true);
}

TEST(FlashInferCorrectnessTest, BatchPrefillKernelCorrectnessTestOneHotFP16) {
  TestBatchPrefillKernelOneHotCorrectness<half>(false);
}

TEST(FlashInferCorrectnessTest, BatchPrefillKernelCorrectnessTestOneHotFP16QKHalfAccum) {
  TestBatchPrefillKernelOneHotCorrectness<half>(true);
}
