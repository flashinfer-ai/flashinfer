#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <gtest/gtest.h>

#include <cstdint>
#include <flashinfer/sampling.cuh>
#include <random>

#include "cpu_reference.h"
#include "utils.h"

using namespace flashinfer;

template <typename T, typename IdType>
void _TestTopKSamplingFromProb(size_t batch_size, uint32_t k, size_t vocab_size) {
  std::vector<T> probs_h(batch_size * vocab_size);
  float p = float(k) * 0.1;
  utils::vec_fill_<T>(probs_h, (1 - p) / float((vocab_size - k)));
  std::vector<int32_t> all_token_ids(vocab_size);
  std::iota(all_token_ids.begin(), all_token_ids.end(), 0);
  std::vector<std::set<int32_t>> high_prob_token_ids_sets;
  for (uint32_t i = 0; i < batch_size; ++i) {
    std::vector<int32_t> high_prob_token_ids;
    std::set<int32_t> high_prob_token_ids_set;
    std::sample(all_token_ids.begin(), all_token_ids.end(), std::back_inserter(high_prob_token_ids),
                k, std::mt19937{std::random_device{}()});
    high_prob_token_ids_set.insert(high_prob_token_ids.begin(), high_prob_token_ids.end());
    high_prob_token_ids_sets.emplace_back(high_prob_token_ids_set);
    for (uint32_t j = 0; j < k; ++j) {
      probs_h[i * vocab_size + high_prob_token_ids[j]] = 0.1;
    }
  }

  thrust::device_vector<T> probs_d(probs_h);
  thrust::device_vector<bool> success_d(batch_size);
  thrust::device_vector<IdType> sampled_ids_d(batch_size);
  const int32_t num_samples = 1000;
  const uint32_t max_top_p_rounds = 32;
  std::vector<int32_t> counter(batch_size * vocab_size);
  utils::vec_fill_<int32_t>(counter, 0);
  for (uint32_t draw = 0; draw < num_samples; ++draw) {
    std::vector<T> uniform_samples_h(batch_size * max_top_p_rounds);
    utils::vec_uniform_<T>(uniform_samples_h, 0, 1);
    thrust::device_vector<T> uniform_samples_d(uniform_samples_h);

    auto status = sampling::TopKSamplingFromProb<max_top_p_rounds, T, IdType>(
        thrust::raw_pointer_cast(probs_d.data()),
        thrust::raw_pointer_cast(uniform_samples_d.data()),
        thrust::raw_pointer_cast(sampled_ids_d.data()), thrust::raw_pointer_cast(success_d.data()),
        k, batch_size, vocab_size);

    EXPECT_EQ(status, hipSuccess) << "TopKSamplingFromProb kernel launch failed, error message: "
                                   << hipGetErrorString(status);

    thrust::host_vector<bool> success_h(success_d);
    for (uint32_t i = 0; i < batch_size; ++i) {
      EXPECT_TRUE(success_h[i]) << "TopKSamplingFromProb failed for batch " << i;
    }

    thrust::host_vector<IdType> sampled_ids_h(sampled_ids_d);
    for (uint32_t i = 0; i < batch_size; ++i) {
      counter[i * vocab_size + sampled_ids_h[i]]++;
    }
  }

  for (uint32_t i = 0; i < batch_size; ++i) {
    for (uint32_t j = 0; j < vocab_size; ++j) {
      if (counter[i * vocab_size + j] > 0) {
        EXPECT_TRUE(high_prob_token_ids_sets[i].find(j) != high_prob_token_ids_sets[i].end())
            << "high_prob_token_ids_sets[" << i << "] does not contain " << j << std::endl;
      }
    }
  }

  std::cout << "batch_size: " << batch_size << ", k: " << k << ", vocab_size: " << vocab_size
            << ", accuracy test passed." << std::endl;
}

template <typename T, typename IdType>
void _TestTopPSamplingFromProb(size_t batch_size, uint32_t k, size_t vocab_size) {
  std::vector<T> probs_h(batch_size * vocab_size);
  float p = float(k) * 0.1;
  utils::vec_fill_<T>(probs_h, (1 - p) / float((vocab_size - k)));
  std::vector<int32_t> all_token_ids(vocab_size);
  std::iota(all_token_ids.begin(), all_token_ids.end(), 0);
  std::vector<std::set<int32_t>> high_prob_token_ids_sets;
  for (uint32_t i = 0; i < batch_size; ++i) {
    std::vector<int32_t> high_prob_token_ids;
    std::set<int32_t> high_prob_token_ids_set;
    std::sample(all_token_ids.begin(), all_token_ids.end(), std::back_inserter(high_prob_token_ids),
                k, std::mt19937{std::random_device{}()});
    high_prob_token_ids_set.insert(high_prob_token_ids.begin(), high_prob_token_ids.end());
    high_prob_token_ids_sets.emplace_back(high_prob_token_ids_set);
    for (uint32_t j = 0; j < k; ++j) {
      probs_h[i * vocab_size + high_prob_token_ids[j]] = 0.1;
    }
  }

  thrust::device_vector<T> probs_d(probs_h);
  thrust::device_vector<bool> success_d(batch_size);
  thrust::device_vector<IdType> sampled_ids_d(batch_size);
  const int32_t num_samples = 1000;
  const uint32_t max_top_p_rounds = 32;
  std::vector<int32_t> counter(batch_size * vocab_size);
  utils::vec_fill_<int32_t>(counter, 0);
  for (uint32_t draw = 0; draw < num_samples; ++draw) {
    std::vector<T> uniform_samples_h(batch_size * max_top_p_rounds);
    utils::vec_uniform_<T>(uniform_samples_h, 0, 1);
    thrust::device_vector<T> uniform_samples_d(uniform_samples_h);

    auto status = sampling::TopPSamplingFromProb<max_top_p_rounds, T, IdType>(
        thrust::raw_pointer_cast(probs_d.data()),
        thrust::raw_pointer_cast(uniform_samples_d.data()),
        thrust::raw_pointer_cast(sampled_ids_d.data()), thrust::raw_pointer_cast(success_d.data()),
        p, batch_size, vocab_size);

    EXPECT_EQ(status, hipSuccess) << "TopPSamplingFromProb kernel launch failed, error message: "
                                   << hipGetErrorString(status);

    thrust::host_vector<bool> success_h(success_d);
    for (uint32_t i = 0; i < batch_size; ++i) {
      EXPECT_TRUE(success_h[i]) << "TopPSamplingFromProb failed for batch " << i;
    }

    thrust::host_vector<IdType> sampled_ids_h(sampled_ids_d);
    for (uint32_t i = 0; i < batch_size; ++i) {
      counter[i * vocab_size + sampled_ids_h[i]]++;
    }
  }

  for (uint32_t i = 0; i < batch_size; ++i) {
    for (uint32_t j = 0; j < vocab_size; ++j) {
      if (counter[i * vocab_size + j] > 0) {
        EXPECT_TRUE(high_prob_token_ids_sets[i].find(j) != high_prob_token_ids_sets[i].end())
            << "high_prob_token_ids_sets[" << i << "] does not contain " << j << std::endl;
      }
    }
  }

  std::cout << "batch_size: " << batch_size << ", p: " << p << ", vocab_size: " << vocab_size
            << ", accuracy test passed." << std::endl;
}

template <typename T, typename IdType>
void _TestSamplingFromProb(size_t batch_size, size_t vocab_size) {
  std::vector<IdType> sampled_ids_ref_h(batch_size);
  std::vector<T> probs_h(batch_size * vocab_size);
  std::vector<T> uniform_samples_h(batch_size);
  utils::vec_randint_<int32_t>(sampled_ids_ref_h, 0, vocab_size - 1);
  utils::vec_uniform_<T>(uniform_samples_h, 0, 1);
  utils::vec_zero_<T>(probs_h);

  for (uint32_t i = 0; i < batch_size; ++i) {
    probs_h[i * vocab_size + sampled_ids_ref_h[i]] = 1;
  }

  thrust::device_vector<T> probs_d(probs_h);
  thrust::device_vector<T> uniform_samples_d(uniform_samples_h);
  thrust::device_vector<IdType> sampled_ids_d(batch_size);

  auto status = sampling::SamplingFromProb<T>(
      thrust::raw_pointer_cast(probs_d.data()), thrust::raw_pointer_cast(uniform_samples_d.data()),
      thrust::raw_pointer_cast(sampled_ids_d.data()), batch_size, vocab_size);
  EXPECT_EQ(status, hipSuccess) << "SamplingFromProb kernel launch failed, error message: "
                                 << hipGetErrorString(status);

  thrust::host_vector<IdType> sampled_ids_h(sampled_ids_d);

  for (uint32_t i = 0; i < batch_size; ++i) {
    EXPECT_EQ(sampled_ids_h[i], sampled_ids_ref_h[i])
        << "sampled_ids_h[" << i << "] != sampled_ids_ref_h[" << i << "]";
  }

  std::cout << "batch_size: " << batch_size << ", vocab_size: " << vocab_size
            << ", accuracy test passed." << std::endl;
}

template <typename T>
void _TestInclusiveExclusiveParallelScan(size_t batch_size, size_t d, bool pin_smem) {
  std::vector<T> probs_h(batch_size * d);
  utils::vec_uniform_<T>(probs_h, 0, 1);

  // normalize the probs_h
  for (size_t i = 0; i < batch_size; ++i) {
    T sum = 0;
    for (size_t j = 0; j < d; ++j) {
      sum += probs_h[i * d + j];
    }
    for (size_t j = 0; j < d; ++j) {
      probs_h[i * d + j] /= sum;
    }
  }

  thrust::device_vector<T> probs_d(probs_h);
  thrust::device_vector<T> exclusive_cdf_d(batch_size * d);

  if (pin_smem) {
    auto status = sampling::DebugThreadBlockSMEMPrefixSum<T>(
        thrust::raw_pointer_cast(probs_d.data()), thrust::raw_pointer_cast(exclusive_cdf_d.data()),
        batch_size, d);
    EXPECT_EQ(status, hipSuccess)
        << "DebugThreadBlockSMEMPrefixSum kernel launch failed, error message: "
        << hipGetErrorString(status);
  } else {
    auto status = sampling::DebugThreadBlockPrefixSum<T>(
        thrust::raw_pointer_cast(probs_d.data()), thrust::raw_pointer_cast(exclusive_cdf_d.data()),
        batch_size, d);
    EXPECT_EQ(status, hipSuccess)
        << "DebugThreadBlockPrefixSum kernel launch failed, error message: "
        << hipGetErrorString(status);
  }

  thrust::host_vector<T> exclusive_cdf_h(exclusive_cdf_d);
  std::vector<T> exclusive_cdf_ref_h =
      cpu_reference::exclusive_prefix_sum(probs_h.data(), batch_size, d);
  size_t num_result_errors_atol_1e_3_rtol_1e_3 = 0;
  bool nan_detected = false;
  for (uint32_t i = 0; i < batch_size; ++i) {
    for (uint32_t j = 0; j < d; ++j) {
      if (isnan(float(exclusive_cdf_h[i * d + j]))) {
        nan_detected = true;
      }
      if (!utils::isclose(exclusive_cdf_h[i * d + j], exclusive_cdf_ref_h[i * d + j], 1e-3, 1e-3)) {
        std::cout << "i: " << i << ", j: " << j
                  << ", exclusive_cdf_h: " << exclusive_cdf_h[i * d + j]
                  << ", exclusive_cdf_ref_h: " << exclusive_cdf_ref_h[i * d + j] << std::endl;
      }
      num_result_errors_atol_1e_3_rtol_1e_3 +=
          !utils::isclose(exclusive_cdf_h[i * d + j], exclusive_cdf_ref_h[i * d + j], 1e-3, 1e-3);
    }
  }
  float result_accuracy =
      1.0f - float(num_result_errors_atol_1e_3_rtol_1e_3) / float(batch_size * d);
  std::cout << "batch_size: " << batch_size << ", d: " << d << ", pin_smem: " << pin_smem
            << ", result_accuracy: " << result_accuracy << ", nan_detected: " << nan_detected
            << std::endl;
  EXPECT_GT(result_accuracy, 0.99) << "Result accuracy test failed.";
  EXPECT_FALSE(nan_detected) << "NaN detected in the output.";
}

template <typename T>
void TestInclusiveExclusiveParallelScan() {
  for (size_t batch_size : {1, 17, 333}) {
    for (size_t d : {24, 4096, 32000}) {
      for (bool pin_smem : {true, false}) {
        _TestInclusiveExclusiveParallelScan<T>(batch_size, d, pin_smem);
      }
    }
  }
}

template <typename T, typename IdType>
void TestSamplingFromProb() {
  for (size_t batch_size : {1, 7, 333}) {
    for (size_t d : {24, 4096, 32000, 128000}) {
      _TestSamplingFromProb<T, IdType>(batch_size, d);
    }
  }
}

template <typename T, typename IdType>
void TestTopKSamplingFromProb() {
  for (size_t batch_size : {1, 7, 333}) {
    for (size_t k : {1, 2, 3, 4, 5, 6, 7, 8, 9, 10}) {
      for (size_t d : {24, 4096, 32000, 128000}) {
        _TestTopKSamplingFromProb<T, IdType>(batch_size, k, d);
      }
    }
  }
}

template <typename T, typename IdType>
void TestTopPSamplingFromProb() {
  for (size_t batch_size : {1, 7, 333}) {
    for (size_t k : {1, 2, 3, 4, 5, 6, 7, 8, 9, 10}) {
      for (size_t d : {24, 4096, 32000, 128000}) {
        _TestTopPSamplingFromProb<T, IdType>(batch_size, k, d);
      }
    }
  }
}

TEST(FlashInferCorrectnessTests, TestTopKSamplingFromProbFP32) {
  TestTopKSamplingFromProb<float, int32_t>();
}

TEST(FlashInferCorrectnessTests, TestTopPSamplingFromProbFP32) {
  TestTopPSamplingFromProb<float, int32_t>();
}

TEST(FlashInferCorrectnessTests, TestSamplingFromProbFP32) {
  TestSamplingFromProb<float, int32_t>();
}

TEST(FlashInferCorrectnessTests, TestInclusiveExclusiveParallelScanFP32) {
  TestInclusiveExclusiveParallelScan<float>();
}
