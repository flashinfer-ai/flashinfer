#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <dlpack/dlpack.h>
#include <tvm/runtime/logging.h>
#include <tvm/runtime/module.h>
#include <tvm/runtime/ndarray.h>
#include <tvm/runtime/packed_func.h>
#include <tvm/runtime/registry.h>

#include <flashinfer/attention/cascade.cuh>
#include <flashinfer/sampling.cuh>
#include <optional>

#include "flashinfer_ops.cuh"

using tvm::runtime::Array;
using tvm::runtime::DataType;
using tvm::runtime::NDArray;
using tvm::runtime::ShapeTuple;
using namespace flashinfer;

#define DISPATCH_TVM_CUDA_DTYPE(dl_dtype, cuda_dtype, ...)   \
  if (dl_dtype.code == kDLFloat && dl_dtype.bits == 16) {    \
    using cuda_dtype = half;                                 \
    __VA_ARGS__                                              \
  } else {                                                   \
    LOG(FATAL) << "Unsupported data type " << dl_dtype.code; \
  }

#define DISPATCH_TVM_CUDA_IDTYPE(dl_dtype, cuda_dtype, ...)  \
  if (dl_dtype.code == kDLInt && dl_dtype.bits == 32) {      \
    using cuda_dtype = int32_t;                              \
    __VA_ARGS__                                              \
  } else {                                                   \
    LOG(FATAL) << "Unsupported data type " << dl_dtype.code; \
  }

int _FlashInferSinglePrefillWithKVCache(DLTensor* q, DLTensor* k, DLTensor* v, DLTensor* tmp,
                                        bool causal, int64_t kv_layout, int64_t pos_encoding_mode,
                                        bool allow_fp16_qk_reduction, double rope_scale,
                                        double rope_theta, DLTensor* o) {
  // `tmp` is user-provided scratch space of at least 16MB, e.g. 4 * 1024 * 1024 float32.
  CHECK_EQ(q->device.device_type, kDLCUDA) << "The device of q matrix must be CUDA.";
  CHECK_EQ(k->device.device_type, kDLCUDA) << "The device of k matrix must be CUDA.";
  CHECK_EQ(v->device.device_type, kDLCUDA) << "The device of v matrix must be CUDA.";
  CHECK_EQ(o->device.device_type, kDLCUDA) << "The device of o matrix must be CUDA.";

  size_t dev_id = q->device.device_id;
  CHECK_EQ(k->device.device_id, dev_id) << "The device id of q and k matrix doesn't match.";
  CHECK_EQ(v->device.device_id, dev_id) << "The device id of q and v matrix doesn't match.";
  CHECK_EQ(o->device.device_id, dev_id) << "The device id of q and o matrix doesn't match.";

  CHECK_GE(q->ndim, 3);
  size_t qo_len = q->shape[q->ndim - 3];
  size_t num_qo_heads = q->shape[q->ndim - 2];
  size_t head_dim = q->shape[q->ndim - 1];

  CHECK_GE(k->ndim, 3);
  size_t kv_len = k->shape[k->ndim - 3];
  size_t num_kv_heads = k->shape[k->ndim - 2];
  CHECK_EQ(head_dim, k->shape[k->ndim - 1]);

  CHECK_GE(v->ndim, 3);
  CHECK_EQ(kv_len, v->shape[v->ndim - 3]);
  CHECK_EQ(num_kv_heads, v->shape[v->ndim - 2]);
  CHECK_EQ(head_dim, v->shape[v->ndim - 1]);

  CHECK_GE(o->ndim, 2);
  CHECK_EQ(qo_len, o->shape[o->ndim - 2]);
  CHECK_EQ(num_qo_heads * head_dim, o->shape[o->ndim - 1]);

  CHECK(q->dtype.lanes == 1 && k->dtype.lanes == 1 && v->dtype.lanes == 1);
  CHECK(k->dtype.bits == v->dtype.bits && k->dtype.code == v->dtype.code);

  DISPATCH_TVM_CUDA_DTYPE(q->dtype, dtype_q, {
    DISPATCH_TVM_CUDA_DTYPE(k->dtype, dtype_kv, {
      DISPATCH_TVM_CUDA_DTYPE(o->dtype, dtype_out, {
        hipError_t status = SinglePrefillWithKVCache(
            (dtype_q*)q->data, (dtype_kv*)k->data, (dtype_kv*)v->data, (dtype_out*)o->data,
            (dtype_out*)tmp->data, /*lse=*/nullptr, num_qo_heads, num_kv_heads, qo_len, kv_len,
            head_dim, causal, QKVLayout(kv_layout), PosEncodingMode(pos_encoding_mode),
            allow_fp16_qk_reduction, std::nullopt, rope_scale, rope_theta, 0);
        if (status != hipSuccess) {
          LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
        }
    })})});
  return 0;
}

int _FlashInferSingleDecodeWithKVCache(DLTensor* q, DLTensor* k, DLTensor* v, DLTensor* tmp,
                                       int64_t kv_layout, int64_t pos_encoding_mode,
                                       double rope_scale, double rope_theta, DLTensor* o) {
  // `tmp` is user-provided scratch space of at least 16MB, e.g. 4 * 1024 * 1024 float32.
  CHECK_EQ(q->device.device_type, kDLCUDA) << "The device of q matrix must be CUDA.";
  CHECK_EQ(k->device.device_type, kDLCUDA) << "The device of k matrix must be CUDA.";
  CHECK_EQ(v->device.device_type, kDLCUDA) << "The device of v matrix must be CUDA.";
  CHECK_EQ(o->device.device_type, kDLCUDA) << "The device of o matrix must be CUDA.";

  size_t dev_id = q->device.device_id;
  CHECK_EQ(k->device.device_id, dev_id) << "The device id of q and k matrix doesn't match.";
  CHECK_EQ(v->device.device_id, dev_id) << "The device id of q and v matrix doesn't match.";
  CHECK_EQ(o->device.device_id, dev_id) << "The device id of q and o matrix doesn't match.";

  CHECK_GE(q->ndim, 2);
  size_t num_qo_heads = q->shape[q->ndim - 2];
  size_t head_dim = q->shape[q->ndim - 1];

  CHECK_GE(k->ndim, 3);
  size_t seq_len = k->shape[k->ndim - 3];
  size_t num_kv_heads = k->shape[k->ndim - 2];
  CHECK_EQ(head_dim, k->shape[k->ndim - 1]);

  CHECK_GE(v->ndim, 3);
  CHECK_EQ(seq_len, v->shape[v->ndim - 3]);
  CHECK_EQ(num_kv_heads, v->shape[v->ndim - 2]);
  CHECK_EQ(head_dim, v->shape[v->ndim - 1]);

  CHECK_GE(o->ndim, 1);
  CHECK_EQ(num_qo_heads * head_dim, o->shape[o->ndim - 1]);

  CHECK(q->dtype.lanes == 1 && k->dtype.lanes == 1 && v->dtype.lanes == 1);
  CHECK(q->dtype.bits == k->dtype.bits && q->dtype.code == k->dtype.code);
  CHECK(q->dtype.bits == v->dtype.bits && q->dtype.code == v->dtype.code);

  DISPATCH_TVM_CUDA_DTYPE(
      q->dtype, dtype_in, {DISPATCH_TVM_CUDA_DTYPE(o->dtype, dtype_out, {
        hipError_t status = SingleDecodeWithKVCache(
            (dtype_in*)q->data, (dtype_in*)k->data, (dtype_in*)v->data, (dtype_out*)o->data,
            (dtype_out*)tmp->data, num_qo_heads, num_kv_heads, seq_len, head_dim,
            QKVLayout(kv_layout), PosEncodingMode(pos_encoding_mode), rope_scale, rope_theta, 0);
        if (status != hipSuccess) {
          LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
        }
      })});
  return 0;
}

constexpr uint32_t max_num_handlers = 8;
thread_local BatchPrefillHandler batch_prefill_paged_kv_handlers[max_num_handlers];
thread_local BatchPrefillHandler batch_prefill_ragged_kv_handler;

void _FlashInferAttentionPrefillWithPagedKVCache(int64_t handler_id, DLTensor* q_data,
                                                 DLTensor* qo_indptr,          //
                                                 DLTensor* pages,              //
                                                 DLTensor* page_table_indptr,  //
                                                 DLTensor* page_table_values,  //
                                                 DLTensor* last_page_len,      //
                                                 DLTensor* k_rope_pos_offset,  //
                                                 DLTensor* q_offset,           //
                                                 DLTensor* output,             //
                                                 DLTensor* lse,                //
                                                 int64_t causal,               //
                                                 int64_t pos_encoding_mode,    //
                                                 double rope_scale,            //
                                                 double rope_theta,
                                                 double attn_score_scaling_factor = 1.0f) {
  CHECK(handler_id < max_num_handlers) << "The handler id must be less than " << max_num_handlers;
  CHECK_EQ(q_data->device.device_type, kDLCUDA) << "The device of q_data must be CUDA.";
  CHECK_EQ(pages->device.device_type, kDLCUDA) << "The device of kv pages must be CUDA.";
  CHECK_EQ(page_table_indptr->device.device_type, kDLCUDA)
      << "The device of page_table_indptr matrix must be CUDA.";
  CHECK_EQ(page_table_values->device.device_type, kDLCUDA)
      << "The device of page_table_values matrix must be CUDA.";
  CHECK_EQ(last_page_len->device.device_type, kDLCUDA)
      << "The device of last_page_len matrix must be CUDA.";
  CHECK_EQ(q_offset->device.device_type, kDLCUDA) << "The device of q_offset matrix must be CUDA.";
  CHECK_EQ(k_rope_pos_offset->device.device_type, kDLCUDA)
      << "The device of k_rope_pos_offset matrix must be CUDA.";
  CHECK_EQ(qo_indptr->device.device_type, kDLCUDA)
      << "The device of qo_indptr matrix must be CUDA.";
  CHECK_EQ(output->device.device_type, kDLCUDA) << "The device of output must be CUDA.";

  int32_t dev_id = q_data->device.device_id;
  CHECK_EQ(pages->device.device_id, dev_id);
  CHECK_EQ(page_table_indptr->device.device_id, dev_id);
  CHECK_EQ(page_table_values->device.device_id, dev_id);
  CHECK_EQ(last_page_len->device.device_id, dev_id);
  CHECK_EQ(q_offset->device.device_id, dev_id);
  CHECK_EQ(k_rope_pos_offset->device.device_id, dev_id);
  CHECK_EQ(qo_indptr->device.device_id, dev_id);
  CHECK_EQ(output->device.device_id, dev_id);

  CHECK(q_data->dtype.lanes == 1 && pages->dtype.lanes == 1 && output->dtype.lanes == 1);
  CHECK(page_table_indptr->dtype.lanes == 1 && page_table_values->dtype.lanes == 1 &&
        last_page_len->dtype.lanes == 1 && q_offset->dtype.lanes == 1 &&
        k_rope_pos_offset->dtype.lanes == 1 && qo_indptr->dtype.lanes == 1);
  CHECK(page_table_indptr->dtype.bits == page_table_values->dtype.bits &&
        page_table_indptr->dtype.bits == last_page_len->dtype.bits &&
        page_table_indptr->dtype.bits == qo_indptr->dtype.bits &&
        page_table_indptr->dtype.code == page_table_values->dtype.code &&
        page_table_indptr->dtype.code == last_page_len->dtype.code &&
        page_table_indptr->dtype.code == q_offset->dtype.code &&
        page_table_indptr->dtype.code == k_rope_pos_offset->dtype.code &&
        page_table_indptr->dtype.code == qo_indptr->dtype.code);

  CHECK_EQ(pages->ndim, 5);
  CHECK_EQ(pages->shape[1], 2);
  int64_t nhead_kv = pages->shape[2];
  int64_t nhead_qo = q_data->shape[1];
  int64_t nfeat = pages->shape[4];
  int64_t page_size = pages->shape[3];

  CHECK_EQ(last_page_len->ndim, 1);
  int64_t num_total_seqs = last_page_len->shape[0];

  CHECK_EQ(qo_indptr->ndim, 1);
  CHECK_EQ(qo_indptr->shape[0], num_total_seqs + 1);

  CHECK_EQ(page_table_indptr->ndim, 1);
  CHECK_EQ(page_table_indptr->shape[0], num_total_seqs + 1);
  CHECK_EQ(page_table_values->ndim, 1);

  CHECK_EQ(q_data->ndim, 3);
  CHECK_EQ(output->ndim, 3);
  CHECK_EQ(q_data->shape[2], nfeat);
  CHECK_EQ(output->shape[1], nhead_qo);
  CHECK_EQ(output->shape[2], nfeat);
  CHECK_EQ(q_offset->ndim, 1);
  CHECK_EQ(q_offset->shape[0], q_data->shape[0]);

  CHECK_EQ(k_rope_pos_offset->ndim, 1);
  CHECK_EQ(k_rope_pos_offset->shape[0], num_total_seqs);

  constexpr PageStorage page_storage = PageStorage::kIndices;
  constexpr QKVLayout kv_layout = QKVLayout::kHND;
  const float sm_scale = attn_score_scaling_factor / std::sqrt(static_cast<float>(nfeat));

  DISPATCH_TVM_CUDA_DTYPE(q_data->dtype, dtype_q,
    {DISPATCH_TVM_CUDA_DTYPE(pages->dtype, dtype_kv,
      {DISPATCH_TVM_CUDA_DTYPE(
          output->dtype, dtype_out, {DISPATCH_TVM_CUDA_IDTYPE(page_table_values->dtype, dtype_idx, {
            paged_kv_t<page_storage, kv_layout, dtype_kv, dtype_idx> cache(
                nhead_kv, page_size, nfeat, num_total_seqs, static_cast<dtype_kv*>(pages->data),
                static_cast<dtype_idx*>(page_table_values->data) +
                    page_table_values->byte_offset / sizeof(dtype_idx),
                static_cast<dtype_idx*>(page_table_indptr->data) +
                    page_table_indptr->byte_offset / sizeof(dtype_idx),
                static_cast<dtype_idx*>(last_page_len->data) +
                    last_page_len->byte_offset / sizeof(dtype_idx),
                static_cast<dtype_idx*>(k_rope_pos_offset->data) +
                    k_rope_pos_offset->byte_offset / sizeof(dtype_idx));
            hipError_t status = BatchPrefillWithPagedKVCacheWrapper<
                page_storage, kv_layout, dtype_q, dtype_kv, dtype_out, dtype_idx>(
                &batch_prefill_paged_kv_handlers[handler_id], static_cast<dtype_q*>(q_data->data),
                static_cast<dtype_idx*>(qo_indptr->data) +
                    qo_indptr->byte_offset / sizeof(dtype_idx),
                static_cast<dtype_idx*>(q_offset->data) + q_offset->byte_offset / sizeof(dtype_idx),
                cache, static_cast<dtype_out*>(output->data),
                /*lse=*/static_cast<float*>(lse->data), nhead_qo,
                /*causal=*/causal, PosEncodingMode(pos_encoding_mode),
                /*allow_fp16_qk_reduction=*/false, sm_scale, rope_scale, rope_theta,
                /*stream=*/0);
            if (status != hipSuccess) {
              LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
            }
          })})})});
}

void _FlashInferAttentionPrefillWithPagedKVCacheBeginForward(
    int64_t handler_idx, DLTensor* workspace_buffer, DLTensor* qo_indptr, DLTensor* kv_indptr,
    int64_t batch_size, int64_t num_qo_heads, int64_t num_kv_heads, int64_t head_dim,
    int64_t page_size, TVMStreamHandle copy_stream) {
  CHECK_EQ(workspace_buffer->ndim, 1) << "The workspace buffer must be a 1-D tensor";
  size_t workspace_size_in_bytes = workspace_buffer->shape[0] * workspace_buffer->dtype.bits / 8;
  CHECK(handler_idx < max_num_handlers) << "The handler id must be less than " << max_num_handlers;

  // NOTE(Zihao): here we presume the input data type is half, in the future we should
  //   leave a parameter for the input data type.
  using dtype_in = half;
  hipStream_t original_stream = batch_prefill_paged_kv_handlers[handler_idx].GetCUDAStream();
  batch_prefill_paged_kv_handlers[handler_idx].SetCUDAStream(
      static_cast<hipStream_t>(copy_stream));
  DISPATCH_TVM_CUDA_IDTYPE(qo_indptr->dtype, dtype_idx, {
    hipError_t status =
        batch_prefill_paged_kv_handlers[handler_idx].BeginForward<dtype_in, dtype_idx>(
            static_cast<void*>(workspace_buffer->data), workspace_size_in_bytes,
            static_cast<dtype_idx*>(qo_indptr->data) + qo_indptr->byte_offset / sizeof(dtype_idx),
            static_cast<dtype_idx*>(kv_indptr->data) + kv_indptr->byte_offset / sizeof(dtype_idx),
            batch_size, num_qo_heads, num_kv_heads, head_dim, page_size);
    if (status != hipSuccess) {
      LOG(FATAL) << "FlashInfer prefill BeginForward error " << hipGetErrorString(status);
    }
  });
  batch_prefill_paged_kv_handlers[handler_idx].SetCUDAStream(original_stream);
}

void _FlashInferAttentionPrefillWithPagedKVCacheEndForward(int64_t handler_idx) {
  CHECK(handler_idx < max_num_handlers) << "The handler id must be less than " << max_num_handlers;
  batch_prefill_paged_kv_handlers[handler_idx].EndForward();
}

// Creates a pool of handlers with a fixed size to independently handle decoding forward passes.
thread_local BatchDecodeHandler batch_decode_handlers[max_num_handlers];

void _FlashInferAttentionDecodeWithPagedKVCache(int64_t handler_id, DLTensor* q_data,
                                                DLTensor* pages,
                                                DLTensor* page_table_indptr,    //
                                                DLTensor* page_table_values,    //
                                                DLTensor* last_page_len,        //
                                                DLTensor* k_rope_pos_offset,    //
                                                DLTensor* q_offset,             //
                                                DLTensor* output,               //
                                                DLTensor* lse,                  //
                                                int64_t pos_encoding_mode = 0,  //
                                                double rope_scale = 1.0f,       //
                                                double rope_theta = 1e4,
                                                double attn_score_scaling_factor = 1.0f) {
  CHECK_LT(handler_id, max_num_handlers) << "The handler id must be less than " << max_num_handlers;
  CHECK_EQ(q_data->device.device_type, kDLCUDA) << "The device of q_data must be CUDA.";
  CHECK_EQ(pages->device.device_type, kDLCUDA) << "The device of kv pages must be CUDA.";
  CHECK_EQ(page_table_indptr->device.device_type, kDLCUDA)
      << "The device of page_table_indptr matrix must be CUDA.";
  CHECK_EQ(page_table_values->device.device_type, kDLCUDA)
      << "The device of page_table_values matrix must be CUDA.";
  CHECK_EQ(last_page_len->device.device_type, kDLCUDA)
      << "The device of last_page_len matrix must be CUDA.";
  CHECK_EQ(q_offset->device.device_type, kDLCUDA) << "The device of q_offset matrix must be CUDA.";
  CHECK_EQ(k_rope_pos_offset->device.device_type, kDLCUDA)
      << "The device of k_rope_pos_offset matrix must be CUDA.";
  CHECK_EQ(output->device.device_type, kDLCUDA) << "The device of output must be CUDA.";

  int32_t dev_id = q_data->device.device_id;
  CHECK_EQ(pages->device.device_id, dev_id);
  CHECK_EQ(page_table_indptr->device.device_id, dev_id);
  CHECK_EQ(page_table_values->device.device_id, dev_id);
  CHECK_EQ(last_page_len->device.device_id, dev_id);
  CHECK_EQ(q_offset->device.device_id, dev_id);
  CHECK_EQ(k_rope_pos_offset->device.device_id, dev_id);
  CHECK_EQ(output->device.device_id, dev_id);

  CHECK(q_data->dtype.lanes == 1 && pages->dtype.lanes == 1 && output->dtype.lanes == 1);
  CHECK(q_data->dtype.bits == pages->dtype.bits && q_data->dtype.code == pages->dtype.code);
  CHECK(page_table_indptr->dtype.lanes == 1 && page_table_values->dtype.lanes == 1 &&
        last_page_len->dtype.lanes == 1 && q_offset->dtype.lanes == 1 &&
        k_rope_pos_offset->dtype.lanes == 1);
  CHECK(page_table_indptr->dtype.bits == page_table_values->dtype.bits &&
        page_table_indptr->dtype.bits == last_page_len->dtype.bits &&
        page_table_indptr->dtype.code == page_table_values->dtype.code &&
        page_table_indptr->dtype.code == last_page_len->dtype.code &&
        page_table_indptr->dtype.code == q_offset->dtype.code &&
        page_table_indptr->dtype.code == k_rope_pos_offset->dtype.code);

  CHECK_EQ(pages->ndim, 5);
  CHECK_EQ(pages->shape[1], 2);
  int64_t nhead_kv = pages->shape[2];
  int64_t nfeat = pages->shape[4];
  int64_t page_size = pages->shape[3];

  CHECK_EQ(last_page_len->ndim, 1);
  int64_t num_total_seqs = last_page_len->shape[0];

  CHECK_EQ(page_table_indptr->ndim, 1);
  CHECK_EQ(page_table_indptr->shape[0], num_total_seqs + 1);
  CHECK_EQ(page_table_values->ndim, 1);

  CHECK_EQ(q_data->ndim, 3);
  CHECK_EQ(output->ndim, 3);
  CHECK_GE(q_data->shape[0], 1);
  CHECK_EQ(q_data->shape[0], output->shape[0]);
  CHECK_EQ(q_data->shape[2], nfeat);
  int64_t nhead_qo = q_data->shape[1];
  CHECK_EQ(output->shape[1], nhead_qo);
  CHECK_EQ(output->shape[2], nfeat);
  CHECK_EQ(q_offset->ndim, 1);
  CHECK_EQ(q_offset->shape[0], num_total_seqs);

  CHECK_EQ(k_rope_pos_offset->ndim, 1);
  CHECK_EQ(k_rope_pos_offset->shape[0], num_total_seqs);

  constexpr PageStorage page_storage = PageStorage::kIndices;
  constexpr QKVLayout kv_layout = QKVLayout::kHND;
  const float sm_scale = attn_score_scaling_factor / std::sqrt(static_cast<float>(nfeat));

  DISPATCH_TVM_CUDA_DTYPE(
      pages->dtype, dtype_in,
      {DISPATCH_TVM_CUDA_DTYPE(
          output->dtype, dtype_out, {DISPATCH_TVM_CUDA_IDTYPE(page_table_values->dtype, dtype_idx, {
            paged_kv_t<page_storage, kv_layout, dtype_in, dtype_idx> cache(
                nhead_kv, page_size, nfeat, num_total_seqs, static_cast<dtype_in*>(pages->data),
                static_cast<dtype_idx*>(page_table_values->data) +
                    page_table_values->byte_offset / sizeof(dtype_idx),
                static_cast<dtype_idx*>(page_table_indptr->data) +
                    page_table_indptr->byte_offset / sizeof(dtype_idx),
                static_cast<dtype_idx*>(last_page_len->data) +
                    last_page_len->byte_offset / sizeof(dtype_idx),
                static_cast<dtype_idx*>(k_rope_pos_offset->data) +
                    k_rope_pos_offset->byte_offset / sizeof(dtype_idx));
            hipError_t status = BatchDecodeWithPagedKVCacheWrapper<
                page_storage, kv_layout, dtype_in, dtype_in, dtype_out, dtype_idx>(
                &batch_decode_handlers[handler_id], static_cast<dtype_in*>(q_data->data),
                static_cast<dtype_idx*>(q_offset->data) + q_offset->byte_offset / sizeof(dtype_idx),
                cache, static_cast<dtype_out*>(output->data),
                /*lse=*/static_cast<float*>(lse->data), nhead_qo,
                PosEncodingMode(pos_encoding_mode), sm_scale, rope_scale, rope_theta,
                /*stream=*/0);
            if (status != hipSuccess) {
              LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
            }
          })})});
}

void _FlashInferAttentionDecodeWithPagedKVCacheBeginForward(
    int64_t handler_idx, DLTensor* workspace_buffer, DLTensor* page_table_indptr,
    DLTensor* last_page_len, int64_t num_qo_heads, int64_t num_kv_heads, int64_t head_dim,
    int64_t page_size, int64_t pos_encoding_mode, TVMStreamHandle copy_stream) {
  CHECK_EQ(workspace_buffer->ndim, 1) << "The workspace buffer must be a 1-D tensor";
  size_t workspace_size_in_bytes = workspace_buffer->shape[0] * workspace_buffer->dtype.bits / 8;
  CHECK_LT(handler_idx, max_num_handlers)
      << "The handler id must be less than " << max_num_handlers;
  constexpr PageStorage page_storage = PageStorage::kIndices;
  constexpr QKVLayout kv_layout = QKVLayout::kHND;
  // NOTE(Zihao): here we presume the input data type is half, in the future we should
  //   leave a parameter for the input data type.
  using dtype_in = half;
  const uint32_t batch_size = page_table_indptr->shape[0] - 1;
  hipStream_t original_stream = batch_decode_handlers[handler_idx].GetCUDAStream();
  batch_decode_handlers[handler_idx].SetCUDAStream(static_cast<hipStream_t>(copy_stream));
  DISPATCH_TVM_CUDA_IDTYPE(page_table_indptr->dtype, dtype_idx, {
    hipError_t status = BatchDecodeHandlerBeginForward<page_storage, kv_layout, dtype_in, dtype_in,
                                                        dtype_in, dtype_idx>(
        batch_decode_handlers + handler_idx, static_cast<void*>(workspace_buffer->data),
        workspace_size_in_bytes,
        static_cast<dtype_idx*>(page_table_indptr->data) +
            page_table_indptr->byte_offset / sizeof(dtype_idx),
        static_cast<dtype_idx*>(last_page_len->data) +
            last_page_len->byte_offset / sizeof(dtype_idx),
        batch_size, num_qo_heads, num_kv_heads, head_dim, page_size,
        PosEncodingMode(pos_encoding_mode));
    if (status != hipSuccess) {
      LOG(FATAL) << "FlashInfer decode BeginForward error " << hipGetErrorString(status);
    }
  });
  batch_decode_handlers[handler_idx].SetCUDAStream(original_stream);
}

void _FlashInferAttentionDecodeWithPagedKVCacheEndForward(int64_t handler_id) {
  CHECK_LT(handler_id, max_num_handlers) << "The handler id must be less than " << max_num_handlers;
  batch_decode_handlers[handler_id].EndForward();
}

void _FlashInferAttentionPrefillWithRaggedKVCache(
    DLTensor* q_data, DLTensor* qo_indptr, DLTensor* k_data, DLTensor* v_data, DLTensor* kv_indptr,
    DLTensor* q_offset_map, DLTensor* k_rope_pos_offset, DLTensor* output, DLTensor* lse,
    int64_t causal = 1, int64_t pos_encoding_mode = 0, double rope_scale = 1.0f,
    double rope_theta = 1e4, double attn_score_scaling_factor = 1.0f) {
  CHECK_EQ(q_data->device.device_type, kDLCUDA) << "The device of q_data must be CUDA.";
  CHECK_EQ(qo_indptr->device.device_type, kDLCUDA) << "The device of qo_indptr must be CUDA.";
  CHECK_EQ(k_data->device.device_type, kDLCUDA) << "The device of k_data must be CUDA.";
  CHECK_EQ(v_data->device.device_type, kDLCUDA) << "The device of v_data must be CUDA.";
  CHECK_EQ(kv_indptr->device.device_type, kDLCUDA) << "The device of kv_indptr must be CUDA.";
  CHECK_EQ(output->device.device_type, kDLCUDA) << "The device of output must be CUDA.";
  CHECK_EQ(lse->device.device_type, kDLCUDA) << "The lse of output must be CUDA.";
  CHECK_EQ(q_offset_map->device.device_type, kDLCUDA) << "The device of q_offset_map must be CUDA.";
  CHECK_EQ(k_rope_pos_offset->device.device_type, kDLCUDA)
      << "The device of k_rope_pos_offset must be CUDA.";

  int dev_id = q_data->device.device_id;
  CHECK_EQ(qo_indptr->device.device_id, dev_id);
  CHECK_EQ(k_data->device.device_id, dev_id);
  CHECK_EQ(v_data->device.device_id, dev_id);
  CHECK_EQ(kv_indptr->device.device_id, dev_id);
  CHECK_EQ(output->device.device_id, dev_id);
  CHECK_EQ(lse->device.device_id, dev_id);
  CHECK_EQ(q_offset_map->device.device_id, dev_id);
  CHECK_EQ(k_rope_pos_offset->device.device_id, dev_id);

  CHECK(q_data->dtype.lanes == 1 && qo_indptr->dtype.lanes == 1 && k_data->dtype.lanes == 1 &&
        v_data->dtype.lanes == 1 && kv_indptr->dtype.lanes == 1 && output->dtype.lanes == 1 &&
        lse->dtype.lanes == 1 && q_offset_map->dtype.lanes == 1 &&
        k_rope_pos_offset->dtype.lanes == 1);
  CHECK(q_data->dtype.bits == k_data->dtype.bits && q_data->dtype.code == v_data->dtype.code);
  CHECK(qo_indptr->dtype.bits == kv_indptr->dtype.bits);
  CHECK(lse->dtype.bits == 32);
  CHECK(q_data->dtype.code == k_data->dtype.code && q_data->dtype.code == v_data->dtype.code);
  CHECK(qo_indptr->dtype.code == kv_indptr->dtype.code);
  CHECK(q_offset_map->dtype.code == kv_indptr->dtype.code);
  CHECK(k_rope_pos_offset->dtype.code == kv_indptr->dtype.code);
  CHECK(lse->dtype.code == kDLFloat);

  CHECK_EQ(q_data->ndim, 3);  // qo_nnz, nhead_qo, nfeat
  CHECK_EQ(output->ndim, 3);  // qo_nnz, nhead_qo, nfeat
  CHECK_EQ(lse->ndim, 2);     // qo_nnz, nhead_qo
  CHECK_EQ(k_data->ndim, 3);  // kv_nnz, nhead_kv, nfeat
  CHECK_EQ(v_data->ndim, 3);  // kv_nnz, nhead_kv, nfeat
  int64_t nhead_qo = q_data->shape[1];
  int64_t nfeat = q_data->shape[2];
  int64_t nhead_kv = k_data->shape[1];
  CHECK_EQ(output->shape[0], q_data->shape[0]);
  CHECK_EQ(output->shape[1], nhead_qo);
  CHECK_EQ(output->shape[2], nfeat);
  CHECK_EQ(lse->shape[0], q_data->shape[0]);
  CHECK_EQ(lse->shape[1], nhead_qo);
  CHECK_EQ(k_data->shape[2], nfeat);
  CHECK_EQ(v_data->shape[0], k_data->shape[0]);
  CHECK_EQ(v_data->shape[1], nhead_kv);
  CHECK_EQ(v_data->shape[2], nfeat);

  CHECK_EQ(qo_indptr->ndim, 1);
  CHECK_EQ(kv_indptr->ndim, 1);
  int64_t batch_size = qo_indptr->shape[0] - 1;
  CHECK_EQ(kv_indptr->shape[0], batch_size + 1);

  CHECK_EQ(q_offset_map->ndim, 1);
  CHECK_EQ(q_offset_map->shape[0], q_data->shape[0]);
  CHECK_EQ(k_rope_pos_offset->ndim, 1);
  CHECK_EQ(k_rope_pos_offset->shape[0], batch_size);

  const float sm_scale = attn_score_scaling_factor / std::sqrt(static_cast<float>(nfeat));

  DISPATCH_TVM_CUDA_DTYPE(q_data->dtype, dtype_q,
    {DISPATCH_TVM_CUDA_DTYPE(k_data->dtype, dtype_kv,
      {DISPATCH_TVM_CUDA_DTYPE(
          output->dtype, dtype_out, {DISPATCH_TVM_CUDA_IDTYPE(qo_indptr->dtype, dtype_idx, {
            hipError_t status =
                BatchPrefillWithRaggedKVCacheWrapper<dtype_q, dtype_kv, dtype_out, dtype_idx>(
                    &batch_prefill_ragged_kv_handler, static_cast<dtype_q*>(q_data->data),
                    static_cast<dtype_idx*>(qo_indptr->data) +
                        qo_indptr->byte_offset / sizeof(dtype_idx),
                    static_cast<dtype_kv*>(k_data->data), static_cast<dtype_kv*>(v_data->data),
                    static_cast<dtype_idx*>(kv_indptr->data) +
                        kv_indptr->byte_offset / sizeof(dtype_idx),
                    static_cast<dtype_idx*>(q_offset_map->data) +
                        q_offset_map->byte_offset / sizeof(dtype_idx),
                    static_cast<dtype_idx*>(k_rope_pos_offset->data) +
                        k_rope_pos_offset->byte_offset / sizeof(dtype_idx),
                    static_cast<dtype_out*>(output->data),
                    /*lse=*/static_cast<float*>(lse->data), batch_size, nhead_qo, nhead_kv, nfeat,
                    /*causal=*/bool(causal), QKVLayout::kNHD, PosEncodingMode(pos_encoding_mode),
                    /*allow_fp16_qk_reduction=*/false, sm_scale, rope_scale, rope_theta,
                    /*sm_scale=*/0);
            if (status != hipSuccess) {
              LOG(FATAL) << "FlashInfer AttentionPrefillWithRaggedKVCache error "
                         << hipGetErrorString(status);
            }
          })})})})
}

void _FlashInferAttentionPrefillWithRaggedKVCacheBeginForward(
    DLTensor* workspace_buffer, DLTensor* qo_indptr, DLTensor* kv_indptr, int64_t batch_size,
    int64_t num_qo_heads, int64_t num_kv_heads, int64_t head_dim, TVMStreamHandle copy_stream) {
  CHECK_EQ(workspace_buffer->ndim, 1) << "The workspace buffer must be a 1-D tensor";
  size_t workspace_size_in_bytes = workspace_buffer->shape[0] * workspace_buffer->dtype.bits / 8;
  hipStream_t original_stream = batch_prefill_ragged_kv_handler.GetCUDAStream();
  batch_prefill_ragged_kv_handler.SetCUDAStream(static_cast<hipStream_t>(copy_stream));

  // NOTE(Zihao): here we presume the input data type is half, in the future we should
  //  leave a parameter for the input data type.
  using dtype_in = half;

  DISPATCH_TVM_CUDA_IDTYPE(qo_indptr->dtype, dtype_idx, {
    hipError_t status = batch_prefill_ragged_kv_handler.BeginForward<dtype_in, dtype_idx>(
        static_cast<void*>(workspace_buffer->data), workspace_size_in_bytes,
        static_cast<dtype_idx*>(qo_indptr->data) + qo_indptr->byte_offset / sizeof(dtype_idx),
        static_cast<dtype_idx*>(kv_indptr->data) + kv_indptr->byte_offset / sizeof(dtype_idx),
        batch_size, num_qo_heads, num_kv_heads, head_dim,
        /*page_size=*/1);
    if (status != hipSuccess) {
      LOG(FATAL) << "FlashInfer PrefillWithRaggedKVCache BeginForward error "
                 << hipGetErrorString(status);
    }
  });
  batch_prefill_ragged_kv_handler.SetCUDAStream(original_stream);
}

void _FlashInferAttentionPrefillWithRaggedKVCacheEndForward() {
  batch_prefill_ragged_kv_handler.EndForward();
}

void _FlashInferMergeState(DLTensor* v_a, DLTensor* s_a, DLTensor* v_b, DLTensor* s_b,
                           DLTensor* v_merged, DLTensor* s_merged) {
  CHECK_EQ(v_a->device.device_type, kDLCUDA) << "The device of v_a must be CUDA.";
  CHECK_EQ(s_a->device.device_type, kDLCUDA) << "The device of s_a must be CUDA.";
  CHECK_EQ(v_b->device.device_type, kDLCUDA) << "The device of v_b must be CUDA.";
  CHECK_EQ(s_b->device.device_type, kDLCUDA) << "The device of s_b must be CUDA.";
  CHECK_EQ(v_merged->device.device_type, kDLCUDA) << "The device of v_merged must be CUDA.";
  CHECK_EQ(s_merged->device.device_type, kDLCUDA) << "The device of s_merged must be CUDA.";
  int32_t dev_id = v_a->device.device_id;
  CHECK_EQ(s_a->device.device_id, dev_id);
  CHECK_EQ(v_b->device.device_id, dev_id);
  CHECK_EQ(s_b->device.device_id, dev_id);
  CHECK_EQ(v_merged->device.device_id, dev_id);
  CHECK_EQ(s_merged->device.device_id, dev_id);

  CHECK(v_a->dtype.lanes == 1 && s_a->dtype.lanes == 1 && v_b->dtype.lanes == 1 &&
        s_b->dtype.lanes == 1 && v_merged->dtype.lanes == 1 && s_merged->dtype.lanes == 1);
  CHECK(v_a->dtype.bits == v_b->dtype.bits && v_a->dtype.code == v_b->dtype.code);
  CHECK(s_a->dtype.bits == 32 && s_a->dtype.code == kDLFloat);
  CHECK(s_b->dtype.bits == 32 && s_b->dtype.code == kDLFloat);
  CHECK(s_merged->dtype.bits == 32 && s_merged->dtype.code == kDLFloat);

  CHECK_EQ(v_a->ndim, 3);
  int64_t batch_size = v_a->shape[0];
  int64_t num_heads = v_a->shape[1];
  int64_t head_dim = v_a->shape[2];
  CHECK_EQ(s_a->shape[0], batch_size);
  CHECK_EQ(s_a->shape[1], num_heads);
  CHECK_EQ(v_b->shape[0], batch_size);
  CHECK_EQ(v_b->shape[1], num_heads);
  CHECK_EQ(v_b->shape[2], head_dim);
  CHECK_EQ(s_b->shape[0], batch_size);
  CHECK_EQ(s_b->shape[1], num_heads);
  CHECK_EQ(v_merged->shape[0], batch_size);
  CHECK_EQ(v_merged->shape[1], num_heads);
  CHECK_EQ(v_merged->shape[2], head_dim);
  CHECK_EQ(s_merged->shape[0], batch_size);
  CHECK_EQ(s_merged->shape[1], num_heads);

  DISPATCH_TVM_CUDA_DTYPE(
      v_a->dtype, dtype_in, {DISPATCH_TVM_CUDA_DTYPE(v_merged->dtype, dtype_out, {
        hipError_t status =
            MergeState(static_cast<dtype_in*>(v_a->data), static_cast<float*>(s_a->data),
                       static_cast<dtype_in*>(v_b->data), static_cast<float*>(s_b->data),
                       static_cast<dtype_out*>(v_merged->data), static_cast<float*>(s_merged->data),
                       batch_size, num_heads, head_dim);
        if (status != hipSuccess) {
          LOG(FATAL) << "FlashInfer CUDA MergeState error " << hipGetErrorString(status);
        }
      })});
}

void _FlashInferMergeStateInPlace(DLTensor* v, DLTensor* s, DLTensor* v_other, DLTensor* s_other) {
  CHECK_EQ(v->device.device_type, kDLCUDA) << "The device of v must be CUDA.";
  CHECK_EQ(s->device.device_type, kDLCUDA) << "The device of s must be CUDA.";
  CHECK_EQ(v_other->device.device_type, kDLCUDA) << "The device of v_other must be CUDA.";
  CHECK_EQ(s_other->device.device_type, kDLCUDA) << "The device of s_other must be CUDA.";
  int32_t dev_id = v->device.device_id;
  CHECK_EQ(s->device.device_id, dev_id);
  CHECK_EQ(v_other->device.device_id, dev_id);
  CHECK_EQ(s_other->device.device_id, dev_id);

  CHECK(v->dtype.lanes == 1 && s->dtype.lanes == 1 && v_other->dtype.lanes == 1 &&
        s_other->dtype.lanes == 1);
  CHECK(v->dtype.bits == v_other->dtype.bits && v->dtype.code == v_other->dtype.code);
  CHECK(s->dtype.bits == 32 && s->dtype.code == kDLFloat);
  CHECK(s_other->dtype.bits == 32 && s_other->dtype.code == kDLFloat);

  CHECK_EQ(v->ndim, 3);
  CHECK_EQ(v_other->ndim, 3);
  CHECK_EQ(s->ndim, 2);        // qo_nnz, nhead_qo
  CHECK_EQ(s_other->ndim, 2);  // qo_nnz, nhead_qo
  int64_t batch_size = v->shape[0];
  int64_t num_heads = v->shape[1];
  int64_t head_dim = v->shape[2];
  CHECK_EQ(s->shape[0], batch_size);
  CHECK_EQ(s->shape[1], num_heads);
  CHECK_EQ(v_other->shape[0], batch_size);
  CHECK_EQ(v_other->shape[1], num_heads);
  CHECK_EQ(v_other->shape[2], head_dim);
  CHECK_EQ(s_other->shape[0], batch_size);
  CHECK_EQ(s_other->shape[1], num_heads);

  DISPATCH_TVM_CUDA_DTYPE(v->dtype, dtype, {
    hipError_t status =
        MergeStateInPlace(static_cast<dtype*>(v->data), static_cast<float*>(s->data),
                          static_cast<dtype*>(v_other->data), static_cast<float*>(s_other->data),
                          batch_size, num_heads, head_dim);
    if (status != hipSuccess) {
      LOG(FATAL) << "FlashInfer CUDA MergeStateInPlace error " << hipGetErrorString(status);
    }
  });
}

void _FlashInferBatchQKApplyRotaryInPlace(DLTensor* q, DLTensor* k, DLTensor* indptr,
                                          DLTensor* offsets, int64_t batch_size,
                                          int64_t num_qo_heads, int64_t num_kv_heads,
                                          int64_t head_dim, double rope_scale, double rope_theta) {
  DISPATCH_TVM_CUDA_DTYPE(
      q->dtype, dtype, {DISPATCH_TVM_CUDA_IDTYPE(indptr->dtype, idtype, {
        hipError_t status = BatchQKApplyRotaryInPlace(
            static_cast<dtype*>(q->data), static_cast<dtype*>(k->data),
            static_cast<idtype*>(indptr->data), static_cast<idtype*>(offsets->data), batch_size,
            num_qo_heads, num_kv_heads, head_dim, rope_scale, rope_theta);
        if (status != hipSuccess) {
          LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
        }
      })});
}

void _FlashInferParallelSamplingFromProb(DLTensor* probs, DLTensor* uniform_samples,
                                         DLTensor* row_indices, DLTensor* sampled_token_ids) {
  CHECK_EQ(probs->device.device_type, kDLCUDA) << "The device of probs must be CUDA.";
  CHECK_EQ(uniform_samples->device.device_type, kDLCUDA)
      << "The device of uniform_samples must be CUDA.";
  CHECK_EQ(row_indices->device.device_type, kDLCUDA) << "The device of row_indices must be CUDA.";
  CHECK_EQ(sampled_token_ids->device.device_type, kDLCUDA)
      << "The device of sampled_token_ids must be CUDA.";

  int dev_id = probs->device.device_id;
  CHECK_EQ(uniform_samples->device.device_id, dev_id);
  CHECK_EQ(row_indices->device.device_id, dev_id);
  CHECK_EQ(sampled_token_ids->device.device_id, dev_id);

  CHECK(probs->dtype.lanes == 1 && uniform_samples->dtype.lanes == 1 &&
        row_indices->dtype.lanes == 1 && sampled_token_ids->dtype.lanes == 1);
  CHECK(probs->dtype.code == kDLFloat && probs->dtype.bits == 32);
  CHECK(uniform_samples->dtype.code == kDLFloat && uniform_samples->dtype.bits == 32);
  CHECK(row_indices->dtype.code == kDLInt && row_indices->dtype.bits == 32);
  CHECK(sampled_token_ids->dtype.code == kDLInt && sampled_token_ids->dtype.bits == 32);

  CHECK_EQ(probs->ndim, 2);              // num_probs, vocab_size
  CHECK_EQ(uniform_samples->ndim, 1);    // batch_size,
  CHECK_EQ(row_indices->ndim, 1);        // batch_size,
  CHECK_EQ(sampled_token_ids->ndim, 1);  // batch_size,
  int64_t num_probs = probs->shape[0];
  int64_t vocab_size = probs->shape[1];
  int64_t batch_size = row_indices->shape[0];
  CHECK_EQ(uniform_samples->shape[0], batch_size);
  CHECK_EQ(sampled_token_ids->shape[0], batch_size);

  hipError_t status = sampling::ParallelSamplingFromProb<float, int32_t>(
      static_cast<float*>(probs->data), static_cast<float*>(uniform_samples->data),
      static_cast<int32_t*>(sampled_token_ids->data), static_cast<int32_t*>(row_indices->data),
      batch_size, vocab_size);
  if (status != hipSuccess) {
    LOG(FATAL) << "FlashInfer ParallelTopPSamplingFromProb error " << hipGetErrorString(status);
  }
}

void _FlashInferParallelTopPSamplingFromProb(DLTensor* probs, DLTensor* uniform_samples,
                                             DLTensor* row_indices, DLTensor* top_p,
                                             DLTensor* sampled_token_ids) {
  CHECK_EQ(probs->device.device_type, kDLCUDA) << "The device of probs must be CUDA.";
  CHECK_EQ(uniform_samples->device.device_type, kDLCUDA)
      << "The device of uniform_samples must be CUDA.";
  CHECK_EQ(row_indices->device.device_type, kDLCUDA) << "The device of row_indices must be CUDA.";
  CHECK_EQ(top_p->device.device_type, kDLCUDA) << "The device of top_p must be CUDA.";
  CHECK_EQ(sampled_token_ids->device.device_type, kDLCUDA)
      << "The device of sampled_token_ids must be CUDA.";

  int dev_id = probs->device.device_id;
  CHECK_EQ(uniform_samples->device.device_id, dev_id);
  CHECK_EQ(row_indices->device.device_id, dev_id);
  CHECK_EQ(top_p->device.device_id, dev_id);
  CHECK_EQ(sampled_token_ids->device.device_id, dev_id);

  CHECK(probs->dtype.lanes == 1 && uniform_samples->dtype.lanes == 1 &&
        row_indices->dtype.lanes == 1 && top_p->dtype.lanes == 1 &&
        sampled_token_ids->dtype.lanes == 1);
  CHECK(probs->dtype.code == kDLFloat && probs->dtype.bits == 32);
  CHECK(uniform_samples->dtype.code == kDLFloat && uniform_samples->dtype.bits == 32);
  CHECK(top_p->dtype.code == kDLFloat && top_p->dtype.bits == 32);
  CHECK(row_indices->dtype.code == kDLInt && row_indices->dtype.bits == 32);
  CHECK(sampled_token_ids->dtype.code == kDLInt && sampled_token_ids->dtype.bits == 32);

  CHECK_EQ(probs->ndim, 2);              // num_probs, vocab_size
  CHECK_EQ(uniform_samples->ndim, 2);    // num_rounds, batch_size
  CHECK_EQ(row_indices->ndim, 1);        // batch_size,
  CHECK_EQ(top_p->ndim, 1);              // num_probs,
  CHECK_EQ(sampled_token_ids->ndim, 1);  // batch_size,
  int64_t num_probs = probs->shape[0];
  int64_t vocab_size = probs->shape[1];
  int64_t batch_size = row_indices->shape[0];
  int64_t num_rounds = uniform_samples->shape[0];
  CHECK_EQ(uniform_samples->shape[1], batch_size);
  CHECK_EQ(top_p->shape[0], num_probs);
  CHECK_EQ(sampled_token_ids->shape[0], batch_size);

  hipError_t status = sampling::ParallelTopPSamplingFromProb<float, int32_t>(
      static_cast<float*>(probs->data), static_cast<float*>(uniform_samples->data),
      static_cast<int32_t*>(sampled_token_ids->data), /*success=*/nullptr,
      static_cast<int32_t*>(row_indices->data), static_cast<float*>(top_p->data), batch_size,
      vocab_size, num_rounds);
  if (status != hipSuccess) {
    LOG(FATAL) << "FlashInfer ParallelTopPSamplingFromProb error " << hipGetErrorString(status);
  }
}

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_prefill_with_paged_kv_cache")
    .set_body_typed(_FlashInferAttentionPrefillWithPagedKVCache);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_prefill_with_paged_kv_cache_begin_forward")
    .set_body_typed(_FlashInferAttentionPrefillWithPagedKVCacheBeginForward);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_prefill_with_paged_kv_cache_end_forward")
    .set_body_typed(_FlashInferAttentionPrefillWithPagedKVCacheEndForward);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_decode_with_paged_kv_cache")
    .set_body_typed(_FlashInferAttentionDecodeWithPagedKVCache);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_decode_with_paged_kv_cache_begin_forward")
    .set_body_typed(_FlashInferAttentionDecodeWithPagedKVCacheBeginForward);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_decode_with_paged_kv_cache_end_forward")
    .set_body_typed(_FlashInferAttentionDecodeWithPagedKVCacheEndForward);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_prefill_with_ragged_kv_cache")
    .set_body_typed(_FlashInferAttentionPrefillWithRaggedKVCache);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_prefill_with_ragged_kv_cache_begin_forward")
    .set_body_typed(_FlashInferAttentionPrefillWithRaggedKVCacheBeginForward);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_prefill_with_ragged_kv_cache_end_forward")
    .set_body_typed(_FlashInferAttentionPrefillWithRaggedKVCacheEndForward);

TVM_REGISTER_GLOBAL("flashinfer.merge_state").set_body_typed(_FlashInferMergeState);

TVM_REGISTER_GLOBAL("flashinfer.merge_state_in_place").set_body_typed(_FlashInferMergeStateInPlace);

TVM_REGISTER_GLOBAL("flashinfer.batch_qk_apply_rotary_in_place")
    .set_body_typed(_FlashInferBatchQKApplyRotaryInPlace);

TVM_REGISTER_GLOBAL("flashinfer.single_prefill")
    .set_body_typed(_FlashInferSinglePrefillWithKVCache);

TVM_REGISTER_GLOBAL("flashinfer.single_decode").set_body_typed(_FlashInferSingleDecodeWithKVCache);

TVM_REGISTER_GLOBAL("flashinfer.sampling.parallel_sampling_from_prob")
    .set_body_typed(_FlashInferParallelSamplingFromProb);

TVM_REGISTER_GLOBAL("flashinfer.sampling.parallel_top_p_sampling_from_prob")
    .set_body_typed(_FlashInferParallelTopPSamplingFromProb);
