/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <dlpack/dlpack.h>
#include <tvm/runtime/logging.h>
#include <tvm/runtime/module.h>
#include <tvm/runtime/ndarray.h>
#include <tvm/runtime/packed_func.h>
#include <tvm/runtime/registry.h>

#include <flashinfer.cuh>

using tvm::runtime::Array;
using tvm::runtime::DataType;
using tvm::runtime::NDArray;
using tvm::runtime::ShapeTuple;
using namespace flashinfer;

#define SWITCH_TVM_CUDA_DTYPE(dl_dtype, cuda_dtype, ...)     \
  if (dl_dtype.code == kDLFloat && dl_dtype.bits == 16) {    \
    using cuda_dtype = half;                                 \
    __VA_ARGS__                                              \
  } else {                                                   \
    LOG(FATAL) << "Unsupported data type " << dl_dtype.code; \
  }

#define SWITCH_TVM_CUDA_IDTYPE(dl_dtype, cuda_dtype, ...)    \
  if (dl_dtype.code == kDLInt && dl_dtype.bits == 32) {      \
    using cuda_dtype = int32_t;                              \
    __VA_ARGS__                                              \
  } else {                                                   \
    LOG(FATAL) << "Unsupported data type " << dl_dtype.code; \
  }

/*!
 * \brief The SinglePrefillWithKVCache function with some parameters fixed at compile time
 *   to accelerate the dispatching.
 */
template <typename DTypeIn, typename DTypeOut>
hipError_t _SinglePrefillWithKVCacheNoLSE(
    DTypeIn* q, DTypeIn* k, DTypeIn* v, DTypeOut* o, float* tmp, uint32_t num_qo_heads,
    uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len, uint32_t head_dim, bool causal = true,
    QKVLayout kv_layout = QKVLayout::kNHD, RotaryMode rotary_mode = RotaryMode::kNone,
    bool allow_fp16_qk_reduction = false, float rope_scale = 1.f, float rope_theta = 1e4,
    hipStream_t stream = nullptr) {
  CHECK(head_dim == 128) << "The head dimension must be 128";
  CHECK(kv_layout == QKVLayout::kNHD) << "The KV layout must be NHD";
  const uint32_t group_size = num_qo_heads / num_kv_heads;

  SWITCH_ALLOW_FP16_QK_REDUCTION(
      allow_fp16_qk_reduction, ALLOW_FP16_QK_REDUCTION,
      {SWITCH_GQA_GROUP_SIZE(
          group_size, GROUP_SIZE,
          {SWITCH_CAUSAL(causal, CAUSAL, {SWITCH_ROTARY_MODE(rotary_mode, ROTARY_MODE, {
                           SinglePrefillWithKVCacheDispatched<
                               GROUP_SIZE, /*head_dim=*/128, /*layout=*/QKVLayout::kNHD,
                               ROTARY_MODE, ALLOW_FP16_QK_REDUCTION, CAUSAL>(
                               q, k, v, o, tmp, /*lse=*/nullptr, num_kv_heads, qo_len, kv_len,
                               rope_scale, rope_theta, stream);
                         })})})});
  return hipSuccess;
}

int _FlashInferSinglePrefillWithKVCache(DLTensor* q, DLTensor* k, DLTensor* v, DLTensor* tmp,
                                        bool causal, int64_t kv_layout, int64_t rotary_mode,
                                        bool allow_fp16_qk_reduction, double rope_scale,
                                        double rope_theta, DLTensor* o) {
  // `tmp` is user-provided scratch space of at least 16MB, e.g. 4 * 1024 * 1024 float32.
  CHECK_EQ(q->device.device_type, kDLCUDA) << "The device of q matrix must be CUDA.";
  CHECK_EQ(k->device.device_type, kDLCUDA) << "The device of k matrix must be CUDA.";
  CHECK_EQ(v->device.device_type, kDLCUDA) << "The device of v matrix must be CUDA.";
  CHECK_EQ(o->device.device_type, kDLCUDA) << "The device of o matrix must be CUDA.";

  size_t dev_id = q->device.device_id;
  CHECK_EQ(k->device.device_id, dev_id) << "The device id of q and k matrix doesn't match.";
  CHECK_EQ(v->device.device_id, dev_id) << "The device id of q and v matrix doesn't match.";
  CHECK_EQ(o->device.device_id, dev_id) << "The device id of q and o matrix doesn't match.";

  CHECK_GE(q->ndim, 3);
  size_t qo_len = q->shape[q->ndim - 3];
  size_t num_qo_heads = q->shape[q->ndim - 2];
  size_t head_dim = q->shape[q->ndim - 1];

  CHECK_GE(k->ndim, 3);
  size_t kv_len = k->shape[k->ndim - 3];
  size_t num_kv_heads = k->shape[k->ndim - 2];
  CHECK_EQ(head_dim, k->shape[k->ndim - 1]);

  CHECK_GE(v->ndim, 3);
  CHECK_EQ(kv_len, v->shape[v->ndim - 3]);
  CHECK_EQ(num_kv_heads, v->shape[v->ndim - 2]);
  CHECK_EQ(head_dim, v->shape[v->ndim - 1]);

  CHECK_GE(o->ndim, 2);
  CHECK_EQ(qo_len, o->shape[o->ndim - 2]);
  CHECK_EQ(num_qo_heads * head_dim, o->shape[o->ndim - 1]);

  CHECK(q->dtype.lanes == 1 && k->dtype.lanes == 1 && v->dtype.lanes == 1);
  CHECK(q->dtype.bits == k->dtype.bits && q->dtype.code == k->dtype.code);
  CHECK(q->dtype.bits == v->dtype.bits && q->dtype.code == v->dtype.code);

  SWITCH_TVM_CUDA_DTYPE(
      q->dtype, dtype_in, {SWITCH_TVM_CUDA_DTYPE(o->dtype, dtype_out, {
        hipError_t status = _SinglePrefillWithKVCacheNoLSE(
            (dtype_in*)q->data, (dtype_in*)k->data, (dtype_in*)v->data, (dtype_out*)o->data,
            (float*)tmp->data, num_qo_heads, num_kv_heads, qo_len, kv_len, head_dim, causal,
            QKVLayout(kv_layout), RotaryMode(rotary_mode), allow_fp16_qk_reduction, rope_scale,
            rope_theta, 0);
        if (status != hipSuccess) {
          LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
        }
      })});
  return 0;
}

int _FlashInferSingleDecodeWithKVCache(DLTensor* q, DLTensor* k, DLTensor* v, DLTensor* tmp,
                                       int64_t kv_layout, int64_t rotary_mode, double rope_scale,
                                       double rope_theta, DLTensor* o) {
  // `tmp` is user-provided scratch space of at least 16MB, e.g. 4 * 1024 * 1024 float32.
  CHECK_EQ(q->device.device_type, kDLCUDA) << "The device of q matrix must be CUDA.";
  CHECK_EQ(k->device.device_type, kDLCUDA) << "The device of k matrix must be CUDA.";
  CHECK_EQ(v->device.device_type, kDLCUDA) << "The device of v matrix must be CUDA.";
  CHECK_EQ(o->device.device_type, kDLCUDA) << "The device of o matrix must be CUDA.";

  size_t dev_id = q->device.device_id;
  CHECK_EQ(k->device.device_id, dev_id) << "The device id of q and k matrix doesn't match.";
  CHECK_EQ(v->device.device_id, dev_id) << "The device id of q and v matrix doesn't match.";
  CHECK_EQ(o->device.device_id, dev_id) << "The device id of q and o matrix doesn't match.";

  CHECK_GE(q->ndim, 2);
  size_t num_qo_heads = q->shape[q->ndim - 2];
  size_t head_dim = q->shape[q->ndim - 1];

  CHECK_GE(k->ndim, 3);
  size_t seq_len = k->shape[k->ndim - 3];
  size_t num_kv_heads = k->shape[k->ndim - 2];
  CHECK_EQ(head_dim, k->shape[k->ndim - 1]);

  CHECK_GE(v->ndim, 3);
  CHECK_EQ(seq_len, v->shape[v->ndim - 3]);
  CHECK_EQ(num_kv_heads, v->shape[v->ndim - 2]);
  CHECK_EQ(head_dim, v->shape[v->ndim - 1]);

  CHECK_GE(o->ndim, 1);
  CHECK_EQ(num_qo_heads * head_dim, o->shape[o->ndim - 1]);

  CHECK(q->dtype.lanes == 1 && k->dtype.lanes == 1 && v->dtype.lanes == 1);
  CHECK(q->dtype.bits == k->dtype.bits && q->dtype.code == k->dtype.code);
  CHECK(q->dtype.bits == v->dtype.bits && q->dtype.code == v->dtype.code);

  SWITCH_TVM_CUDA_DTYPE(
      q->dtype, dtype_in, {SWITCH_TVM_CUDA_DTYPE(o->dtype, dtype_out, {
        hipError_t status = SingleDecodeWithKVCache(
            (dtype_in*)q->data, (dtype_in*)k->data, (dtype_in*)v->data, (dtype_out*)o->data,
            (dtype_out*)tmp->data, num_qo_heads, num_kv_heads, seq_len, head_dim,
            QKVLayout(kv_layout), RotaryMode(rotary_mode), rope_scale, rope_theta, 0);
        if (status != hipSuccess) {
          LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
        }
      })});
  return 0;
}

constexpr uint32_t max_num_handlers = 8;
thread_local BatchPrefillHandler batch_prefill_paged_kv_handlers[max_num_handlers];
thread_local BatchPrefillHandler batch_prefill_ragged_kv_handler;

/*!
 * \brief The BatchPrefillWithKVCacheWrapper function with some parameters fixed at compile time
 *    to accelerate the dispatching.
 */
template <PageStorage page_storage, QKVLayout kv_layout, typename DTypeIn, typename DTypeOut,
          typename IdType>
hipError_t _BatchPrefillWithPagedKVCacheWrapper(
    BatchPrefillHandler* handler, DTypeIn* q, IdType* qo_indptr,
    paged_kv_t<page_storage, kv_layout, DTypeIn, IdType> paged_kv, DTypeOut* o, float* lse,
    uint32_t num_qo_heads, bool causal = true, RotaryMode rotary_mode = RotaryMode::kNone,
    bool allow_fp16_qk_reduction = false, float rope_scale = 1.f, float rope_theta = 1e4,
    hipStream_t stream = nullptr) {
  CHECK(lse != nullptr) << "The lse buffer must be provided";
  CHECK(allow_fp16_qk_reduction == false) << "The fp16 qk reduction is not supported";
  CHECK(paged_kv.head_dim == 128) << "The head dimension must be 128";
  const uint32_t num_kv_heads = paged_kv.num_heads;
  const uint32_t head_dim = paged_kv.head_dim;
  const uint32_t batch_size = paged_kv.batch_size;
  const uint32_t group_size = num_qo_heads / num_kv_heads;
  SWITCH_GQA_GROUP_SIZE(
      group_size, GROUP_SIZE,
      {SWITCH_CAUSAL(causal, CAUSAL, {SWITCH_ROTARY_MODE(rotary_mode, ROTARY_MODE, {
                       return BatchPrefillWithPagedKVCacheWrapperDispatched<
                           page_storage, kv_layout, GROUP_SIZE, /*head_dim=*/128, ROTARY_MODE,
                           /*allow_fp16_qk_reduction=*/false, CAUSAL, DTypeIn, DTypeOut, IdType>(
                           handler, q, qo_indptr, paged_kv, o, lse, num_qo_heads, rope_scale,
                           rope_theta, stream);
                     })})});
  return hipSuccess;
}

void _FlashInferAttentionPrefillWithPagedKVCache(int64_t handler_id, DLTensor* q_data,
                                                 DLTensor* qo_indptr,          //
                                                 DLTensor* pages,              //
                                                 DLTensor* page_table_indptr,  //
                                                 DLTensor* page_table_values,  //
                                                 DLTensor* last_page_len,      //
                                                 DLTensor* output,             //
                                                 DLTensor* lse,                //
                                                 int64_t causal = 1,           //
                                                 int64_t rotary_mode = 0,      //
                                                 double rope_scale = 1.0f,     //
                                                 double rope_theta = 1e4) {
  CHECK(handler_id < max_num_handlers) << "The handler id must be less than " << max_num_handlers;
  CHECK_EQ(q_data->device.device_type, kDLCUDA) << "The device of q_data must be CUDA.";
  CHECK_EQ(pages->device.device_type, kDLCUDA) << "The device of kv pages must be CUDA.";
  CHECK_EQ(page_table_indptr->device.device_type, kDLCUDA)
      << "The device of page_table_indptr matrix must be CUDA.";
  CHECK_EQ(page_table_values->device.device_type, kDLCUDA)
      << "The device of page_table_values matrix must be CUDA.";
  CHECK_EQ(last_page_len->device.device_type, kDLCUDA)
      << "The device of last_page_len matrix must be CUDA.";
  CHECK_EQ(qo_indptr->device.device_type, kDLCUDA)
      << "The device of qo_indptr matrix must be CUDA.";
  CHECK_EQ(output->device.device_type, kDLCUDA) << "The device of output must be CUDA.";

  int32_t dev_id = q_data->device.device_id;
  CHECK_EQ(pages->device.device_id, dev_id);
  CHECK_EQ(page_table_indptr->device.device_id, dev_id);
  CHECK_EQ(page_table_values->device.device_id, dev_id);
  CHECK_EQ(last_page_len->device.device_id, dev_id);
  CHECK_EQ(qo_indptr->device.device_id, dev_id);
  CHECK_EQ(output->device.device_id, dev_id);

  CHECK(q_data->dtype.lanes == 1 && pages->dtype.lanes == 1 && output->dtype.lanes == 1);
  CHECK(q_data->dtype.bits == pages->dtype.bits && q_data->dtype.code == pages->dtype.code);
  CHECK(page_table_indptr->dtype.lanes == 1 && page_table_values->dtype.lanes == 1 &&
        last_page_len->dtype.lanes == 1 && qo_indptr->dtype.lanes == 1);
  CHECK(page_table_indptr->dtype.bits == page_table_values->dtype.bits &&
        page_table_indptr->dtype.bits == last_page_len->dtype.bits &&
        page_table_indptr->dtype.bits == qo_indptr->dtype.bits &&
        page_table_indptr->dtype.code == page_table_values->dtype.code &&
        page_table_indptr->dtype.code == last_page_len->dtype.code &&
        page_table_indptr->dtype.code == qo_indptr->dtype.code);

  CHECK_EQ(pages->ndim, 5);
  CHECK_EQ(pages->shape[1], 2);
  int64_t nhead_kv = pages->shape[2];
  int64_t nhead_qo = q_data->shape[1];
  int64_t nfeat = pages->shape[4];
  int64_t page_size = pages->shape[3];

  CHECK_EQ(last_page_len->ndim, 1);
  int64_t num_total_seqs = last_page_len->shape[0];

  CHECK_EQ(qo_indptr->ndim, 1);
  CHECK_EQ(qo_indptr->shape[0], num_total_seqs + 1);

  CHECK_EQ(page_table_indptr->ndim, 1);
  CHECK_EQ(page_table_indptr->shape[0], num_total_seqs + 1);
  CHECK_EQ(page_table_values->ndim, 1);

  CHECK_EQ(q_data->ndim, 3);
  CHECK_EQ(output->ndim, 3);
  CHECK_EQ(q_data->shape[2], nfeat);
  CHECK_EQ(output->shape[1], nhead_qo);
  CHECK_EQ(output->shape[2], nfeat);

  constexpr PageStorage page_storage = PageStorage::kIndices;
  constexpr QKVLayout kv_layout = QKVLayout::kHND;

  SWITCH_TVM_CUDA_DTYPE(
      pages->dtype, dtype_in,
      {SWITCH_TVM_CUDA_DTYPE(
          output->dtype, dtype_out, {SWITCH_TVM_CUDA_IDTYPE(page_table_values->dtype, dtype_idx, {
            paged_kv_t<page_storage, kv_layout, dtype_in, dtype_idx> cache(
                nhead_kv, page_size, nfeat, num_total_seqs, static_cast<dtype_in*>(pages->data),
                static_cast<dtype_idx*>(page_table_values->data),
                static_cast<dtype_idx*>(page_table_indptr->data),
                static_cast<dtype_idx*>(last_page_len->data));
            hipError_t status =
                _BatchPrefillWithPagedKVCacheWrapper<page_storage, kv_layout, dtype_in, dtype_out,
                                                     dtype_idx>(
                    &batch_prefill_paged_kv_handlers[handler_id],
                    static_cast<dtype_in*>(q_data->data), static_cast<dtype_idx*>(qo_indptr->data),
                    cache, static_cast<dtype_out*>(output->data),
                    /*lse=*/static_cast<float*>(lse->data), nhead_qo,
                    /*causal=*/causal, RotaryMode(rotary_mode), /*allow_fp16_qk_reduction=*/false,
                    rope_scale, rope_theta, 0);
            if (status != hipSuccess) {
              LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
            }
          })})});
}

void _FlashInferAttentionPrefillWithPagedKVCacheBeginForward(int64_t handler_idx,
                                                             DLTensor* qo_indptr,
                                                             int64_t batch_size,
                                                             int64_t num_qo_heads,
                                                             int64_t num_kv_heads) {
  CHECK(handler_idx < max_num_handlers) << "The handler id must be less than " << max_num_handlers;
  SWITCH_TVM_CUDA_IDTYPE(qo_indptr->dtype, dtype_idx, {
    hipError_t status = batch_prefill_paged_kv_handlers[handler_idx].BeginForward(
        static_cast<dtype_idx*>(qo_indptr->data), batch_size, num_qo_heads, num_kv_heads);
    if (status != hipSuccess) {
      LOG(FATAL) << "FlashInfer prefill BeginForward error " << hipGetErrorString(status);
    }
  });
}

void _FlashInferAttentionPrefillWithPagedKVCacheEndForward(int64_t handler_idx) {
  CHECK(handler_idx < max_num_handlers) << "The handler id must be less than " << max_num_handlers;
  batch_prefill_paged_kv_handlers[handler_idx].EndForward();
}

// Creates a pool of handlers with a fixed size to independently handle decoding forward passes.
thread_local BatchDecodeHandler batch_decode_handlers[max_num_handlers];

void _FlashInferAttentionDecodeWithPagedKVCache(int64_t handler_id, DLTensor* q_data,
                                                DLTensor* pages,
                                                DLTensor* page_table_indptr,  //
                                                DLTensor* page_table_values,  //
                                                DLTensor* last_page_len,      //
                                                DLTensor* output,             //
                                                DLTensor* lse,                //
                                                int64_t rotary_mode = 0,      //
                                                double rope_scale = 1.0f,     //
                                                double rope_theta = 1e4) {
  CHECK_LT(handler_id, max_num_handlers) << "The handler id must be less than " << max_num_handlers;
  CHECK_EQ(q_data->device.device_type, kDLCUDA) << "The device of q_data must be CUDA.";
  CHECK_EQ(pages->device.device_type, kDLCUDA) << "The device of kv pages must be CUDA.";
  CHECK_EQ(page_table_indptr->device.device_type, kDLCUDA)
      << "The device of page_table_indptr matrix must be CUDA.";
  CHECK_EQ(page_table_values->device.device_type, kDLCUDA)
      << "The device of page_table_values matrix must be CUDA.";
  CHECK_EQ(last_page_len->device.device_type, kDLCUDA)
      << "The device of last_page_len matrix must be CUDA.";
  CHECK_EQ(output->device.device_type, kDLCUDA) << "The device of output must be CUDA.";

  int32_t dev_id = q_data->device.device_id;
  CHECK_EQ(pages->device.device_id, dev_id);
  CHECK_EQ(page_table_indptr->device.device_id, dev_id);
  CHECK_EQ(page_table_values->device.device_id, dev_id);
  CHECK_EQ(last_page_len->device.device_id, dev_id);
  CHECK_EQ(output->device.device_id, dev_id);

  CHECK(q_data->dtype.lanes == 1 && pages->dtype.lanes == 1 && output->dtype.lanes == 1);
  CHECK(q_data->dtype.bits == pages->dtype.bits && q_data->dtype.code == pages->dtype.code);
  CHECK(page_table_indptr->dtype.lanes == 1 && page_table_values->dtype.lanes == 1 &&
        last_page_len->dtype.lanes == 1);
  CHECK(page_table_indptr->dtype.bits == page_table_values->dtype.bits &&
        page_table_indptr->dtype.bits == last_page_len->dtype.bits &&
        page_table_indptr->dtype.code == page_table_values->dtype.code &&
        page_table_indptr->dtype.code == last_page_len->dtype.code);

  CHECK_EQ(pages->ndim, 5);
  CHECK_EQ(pages->shape[1], 2);
  int64_t nhead_kv = pages->shape[2];
  int64_t nfeat = pages->shape[4];
  int64_t page_size = pages->shape[3];

  CHECK_EQ(last_page_len->ndim, 1);
  int64_t num_total_seqs = last_page_len->shape[0];

  CHECK_EQ(page_table_indptr->ndim, 1);
  CHECK_EQ(page_table_indptr->shape[0], num_total_seqs + 1);
  CHECK_EQ(page_table_values->ndim, 1);

  CHECK_EQ(q_data->ndim, 3);
  CHECK_EQ(output->ndim, 3);
  CHECK_GE(q_data->shape[0], 1);
  CHECK_EQ(q_data->shape[0], output->shape[0]);
  CHECK_EQ(q_data->shape[2], nfeat);
  int64_t nhead_qo = q_data->shape[1];
  CHECK_EQ(output->shape[1], nhead_qo);
  CHECK_EQ(output->shape[2], nfeat);

  constexpr PageStorage page_storage = PageStorage::kIndices;
  constexpr QKVLayout kv_layout = QKVLayout::kHND;

  SWITCH_TVM_CUDA_DTYPE(
      pages->dtype, dtype_in,
      {SWITCH_TVM_CUDA_DTYPE(
          output->dtype, dtype_out, {SWITCH_TVM_CUDA_IDTYPE(page_table_values->dtype, dtype_idx, {
            paged_kv_t<page_storage, kv_layout, dtype_in, dtype_idx> cache(
                nhead_kv, page_size, nfeat, num_total_seqs, static_cast<dtype_in*>(pages->data),
                static_cast<dtype_idx*>(page_table_values->data),
                static_cast<dtype_idx*>(page_table_indptr->data),
                static_cast<dtype_idx*>(last_page_len->data));
            hipError_t status =
                BatchDecodeWithPagedKVCacheWrapper<page_storage, dtype_in, dtype_out, dtype_idx>(
                    &batch_decode_handlers[handler_id], static_cast<dtype_in*>(q_data->data), cache,
                    static_cast<dtype_out*>(output->data),
                    /*lse=*/static_cast<float*>(lse->data), nhead_qo, RotaryMode(rotary_mode),
                    rope_scale, rope_theta, 0);
            if (status != hipSuccess) {
              LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
            }
          })})});
}

void _FlashInferAttentionDecodeWithPagedKVCacheBeginForward(
    int64_t handler_idx, DLTensor* page_table_indptr, DLTensor* last_page_len, int64_t num_qo_heads,
    int64_t num_kv_heads, int64_t head_dim, int64_t page_size, int64_t rotary_mode) {
  CHECK_LT(handler_idx, max_num_handlers)
      << "The handler id must be less than " << max_num_handlers;
  constexpr PageStorage page_storage = PageStorage::kIndices;
  // NOTE(Zihao): here we presume the input data type is half, in the future we should
  //   leave a parameter for the input data type.
  using dtype_in = half;
  const uint32_t batch_size = page_table_indptr->shape[0] - 1;
  SWITCH_TVM_CUDA_IDTYPE(page_table_indptr->dtype, dtype_idx, {
    hipError_t status =
        batch_decode_handlers[handler_idx]
            .BeginForward<page_storage, dtype_in, dtype_in, dtype_idx>(
                static_cast<dtype_idx*>(page_table_indptr->data),
                static_cast<dtype_idx*>(last_page_len->data), batch_size, num_qo_heads,
                num_kv_heads, head_dim, page_size, RotaryMode(rotary_mode));
    if (status != hipSuccess) {
      LOG(FATAL) << "FlashInfer decode BeginForward error " << hipGetErrorString(status);
    }
  });
}

void _FlashInferAttentionDecodeWithPagedKVCacheEndForward(int64_t handler_id) {
  CHECK_LT(handler_id, max_num_handlers) << "The handler id must be less than " << max_num_handlers;
  batch_decode_handlers[handler_id].EndForward();
}

/*!
 * \brief The BatchPrefillWithRaggedKVCacheWrapper function with some parameters fixed at compile
 * time to accelerate the dispatching.
 */
template <typename DTypeIn, typename DTypeOut, typename IdType>
hipError_t _BatchPrefillWithRaggedKVCacheWrapper(
    BatchPrefillHandler* handler, DTypeIn* q, IdType* qo_indptr, DTypeIn* k, DTypeIn* v,
    IdType* kv_indptr, DTypeOut* o, float* lse, const uint32_t batch_size,
    const uint32_t num_qo_heads, const uint32_t num_kv_heads, const uint32_t head_dim,
    bool causal = true, QKVLayout kv_layout = QKVLayout::kNHD,
    RotaryMode rotary_mode = RotaryMode::kNone, bool allow_fp16_qk_reduction = false,
    const float rope_scale = 1.f, const float rope_theta = 1e4, hipStream_t stream = nullptr) {
  CHECK(lse != nullptr) << "The lse buffer must be provided";
  CHECK(head_dim == 128) << "The head dimension must be 128";
  CHECK(kv_layout == QKVLayout::kNHD) << "The layout must be NHD";
  CHECK(allow_fp16_qk_reduction == false) << "The fp16 qk reduction is not supported";
  SWITCH_GQA_GROUP_SIZE(
      num_qo_heads / num_kv_heads, GROUP_SIZE,
      {SWITCH_CAUSAL(causal, CAUSAL, {SWITCH_ROTARY_MODE(rotary_mode, ROTARY_MODE, {
                       return BatchPrefillWithRaggedKVCacheWrapperDispatched<
                           GROUP_SIZE, /*head_dim=*/128, /*layout=*/QKVLayout::kNHD, ROTARY_MODE,
                           /*allow_fp16_qk_reduction=*/false, CAUSAL, DTypeIn, DTypeOut, IdType>(
                           handler, q, qo_indptr, k, v, kv_indptr, o, lse, batch_size, num_kv_heads,
                           rope_scale, rope_theta, stream);
                     })})});
  return hipSuccess;
}

void _FlashInferAttentionPrefillWithRaggedKVCache(DLTensor* q_data, DLTensor* qo_indptr,
                                                  DLTensor* k_data, DLTensor* v_data,
                                                  DLTensor* kv_indptr, DLTensor* output,
                                                  DLTensor* lse, int64_t causal = 1,
                                                  int64_t rotary_mode = 0, double rope_scale = 1.0f,
                                                  double rope_theta = 1e4) {
  CHECK_EQ(q_data->device.device_type, kDLCUDA) << "The device of q_data must be CUDA.";
  CHECK_EQ(qo_indptr->device.device_type, kDLCUDA) << "The device of qo_indptr must be CUDA.";
  CHECK_EQ(k_data->device.device_type, kDLCUDA) << "The device of k_data must be CUDA.";
  CHECK_EQ(v_data->device.device_type, kDLCUDA) << "The device of v_data must be CUDA.";
  CHECK_EQ(kv_indptr->device.device_type, kDLCUDA) << "The device of kv_indptr must be CUDA.";
  CHECK_EQ(output->device.device_type, kDLCUDA) << "The device of output must be CUDA.";

  int dev_id = q_data->device.device_id;
  CHECK_EQ(qo_indptr->device.device_id, dev_id);
  CHECK_EQ(k_data->device.device_id, dev_id);
  CHECK_EQ(v_data->device.device_id, dev_id);
  CHECK_EQ(kv_indptr->device.device_id, dev_id);
  CHECK_EQ(output->device.device_id, dev_id);
  CHECK_EQ(lse->device.device_id, dev_id);

  CHECK(q_data->dtype.lanes == 1 && qo_indptr->dtype.lanes == 1 && k_data->dtype.lanes == 1 &&
        v_data->dtype.lanes == 1 && kv_indptr->dtype.lanes == 1 && output->dtype.lanes == 1 &&
        lse->dtype.lanes == 1);
  CHECK(q_data->dtype.bits == k_data->dtype.bits && q_data->dtype.code == v_data->dtype.code);
  CHECK(qo_indptr->dtype.bits == kv_indptr->dtype.bits);
  CHECK(lse->dtype.bits == 32);
  CHECK(q_data->dtype.code == k_data->dtype.code && q_data->dtype.code == v_data->dtype.code);
  CHECK(qo_indptr->dtype.code == kv_indptr->dtype.code);
  CHECK(lse->dtype.code == kDLFloat);

  CHECK_EQ(q_data->ndim, 3);  // qo_nnz, nhead_qo, nfeat
  CHECK_EQ(output->ndim, 3);  // qo_nnz, nhead_qo, nfeat
  CHECK_EQ(lse->ndim, 2);     // qo_nnz, nhead_qo
  CHECK_EQ(k_data->ndim, 3);  // kv_nnz, nhead_kv, nfeat
  CHECK_EQ(v_data->ndim, 3);  // kv_nnz, nhead_kv, nfeat
  int64_t nhead_qo = q_data->shape[1];
  int64_t nfeat = q_data->shape[2];
  int64_t nhead_kv = k_data->shape[1];
  CHECK_EQ(output->shape[0], q_data->shape[0]);
  CHECK_EQ(output->shape[1], nhead_qo);
  CHECK_EQ(output->shape[2], nfeat);
  CHECK_EQ(lse->shape[0], q_data->shape[0]);
  CHECK_EQ(lse->shape[1], nhead_qo);
  CHECK_EQ(k_data->shape[2], nfeat);
  CHECK_EQ(v_data->shape[0], k_data->shape[0]);
  CHECK_EQ(v_data->shape[1], nhead_kv);
  CHECK_EQ(v_data->shape[2], nfeat);

  CHECK_EQ(qo_indptr->ndim, 1);
  CHECK_EQ(kv_indptr->ndim, 1);
  int64_t batch_size = qo_indptr->shape[0] - 1;
  CHECK_EQ(kv_indptr->shape[0], batch_size + 1);

  SWITCH_TVM_CUDA_DTYPE(
      q_data->dtype, dtype_in,
      {SWITCH_TVM_CUDA_DTYPE(
          output->dtype, dtype_out, {SWITCH_TVM_CUDA_IDTYPE(qo_indptr->dtype, dtype_idx, {
            hipError_t status =
                _BatchPrefillWithRaggedKVCacheWrapper<dtype_in, dtype_out, dtype_idx>(
                    &batch_prefill_ragged_kv_handler, static_cast<dtype_in*>(q_data->data),
                    static_cast<dtype_idx*>(qo_indptr->data), static_cast<dtype_in*>(k_data->data),
                    static_cast<dtype_in*>(v_data->data), static_cast<dtype_idx*>(kv_indptr->data),
                    static_cast<dtype_out*>(output->data),
                    /*lse=*/static_cast<float*>(lse->data), batch_size, nhead_qo, nhead_kv, nfeat,
                    /*causal=*/bool(causal), QKVLayout::kNHD, RotaryMode(rotary_mode),
                    /*allow_fp16_qk_reduction=*/false, rope_scale, rope_theta, 0);
          })})})
}

void _FlashInferAttentionPrefillWithRaggedKVCacheBeginForward(DLTensor* qo_indptr,
                                                              int64_t batch_size,
                                                              int64_t num_qo_heads,
                                                              int64_t num_kv_heads) {
  SWITCH_TVM_CUDA_IDTYPE(qo_indptr->dtype, dtype_idx, {
    hipError_t status = batch_prefill_ragged_kv_handler.BeginForward(
        static_cast<dtype_idx*>(qo_indptr->data), batch_size, num_qo_heads, num_kv_heads);
    if (status != hipSuccess) {
      LOG(FATAL) << "FlashInfer PrefillWithRaggedKVCache BeginForward error "
                 << hipGetErrorString(status);
    }
  });
}

void _FlashInferAttentionPrefillWithRaggedKVCacheEndForward() {
  batch_prefill_ragged_kv_handler.EndForward();
}

void _FlashInferMergeState(DLTensor* v_a, DLTensor* s_a, DLTensor* v_b, DLTensor* s_b,
                           DLTensor* v_merged, DLTensor* s_merged) {
  CHECK_EQ(v_a->device.device_type, kDLCUDA) << "The device of v_a must be CUDA.";
  CHECK_EQ(s_a->device.device_type, kDLCUDA) << "The device of s_a must be CUDA.";
  CHECK_EQ(v_b->device.device_type, kDLCUDA) << "The device of v_b must be CUDA.";
  CHECK_EQ(s_b->device.device_type, kDLCUDA) << "The device of s_b must be CUDA.";
  CHECK_EQ(v_merged->device.device_type, kDLCUDA) << "The device of v_merged must be CUDA.";
  CHECK_EQ(s_merged->device.device_type, kDLCUDA) << "The device of s_merged must be CUDA.";
  int32_t dev_id = v_a->device.device_id;
  CHECK_EQ(s_a->device.device_id, dev_id);
  CHECK_EQ(v_b->device.device_id, dev_id);
  CHECK_EQ(s_b->device.device_id, dev_id);
  CHECK_EQ(v_merged->device.device_id, dev_id);
  CHECK_EQ(s_merged->device.device_id, dev_id);

  CHECK(v_a->dtype.lanes == 1 && s_a->dtype.lanes == 1 && v_b->dtype.lanes == 1 &&
        s_b->dtype.lanes == 1 && v_merged->dtype.lanes == 1 && s_merged->dtype.lanes == 1);
  CHECK(v_a->dtype.bits == v_b->dtype.bits && v_a->dtype.code == v_b->dtype.code);
  CHECK(s_a->dtype.bits == 32 && s_a->dtype.code == kDLFloat);
  CHECK(s_b->dtype.bits == 32 && s_b->dtype.code == kDLFloat);
  CHECK(s_merged->dtype.bits == 32 && s_merged->dtype.code == kDLFloat);

  CHECK_EQ(v_a->ndim, 3);
  int64_t batch_size = v_a->shape[0];
  int64_t num_heads = v_a->shape[1];
  int64_t head_dim = v_a->shape[2];
  CHECK_EQ(s_a->shape[0], batch_size);
  CHECK_EQ(s_a->shape[1], num_heads);
  CHECK_EQ(v_b->shape[0], batch_size);
  CHECK_EQ(v_b->shape[1], num_heads);
  CHECK_EQ(v_b->shape[2], head_dim);
  CHECK_EQ(s_b->shape[0], batch_size);
  CHECK_EQ(s_b->shape[1], num_heads);
  CHECK_EQ(v_merged->shape[0], batch_size);
  CHECK_EQ(v_merged->shape[1], num_heads);
  CHECK_EQ(v_merged->shape[2], head_dim);
  CHECK_EQ(s_merged->shape[0], batch_size);
  CHECK_EQ(s_merged->shape[1], num_heads);

  SWITCH_TVM_CUDA_DTYPE(v_a->dtype, dtype_in, {SWITCH_TVM_CUDA_DTYPE(v_merged->dtype, dtype_out, {
                          hipError_t status = MergeState(
                              static_cast<dtype_in*>(v_a->data), static_cast<float*>(s_a->data),
                              static_cast<dtype_in*>(v_b->data), static_cast<float*>(s_b->data),
                              static_cast<dtype_out*>(v_merged->data),
                              static_cast<float*>(s_merged->data), batch_size, num_heads, head_dim);
                          if (status != hipSuccess) {
                            LOG(FATAL)
                                << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
                          }
                        })});
}

void _FlashInferMergeStateInPlace(DLTensor* v, DLTensor* s, DLTensor* v_other, DLTensor* s_other) {
  CHECK_EQ(v->device.device_type, kDLCUDA) << "The device of v must be CUDA.";
  CHECK_EQ(s->device.device_type, kDLCUDA) << "The device of s must be CUDA.";
  CHECK_EQ(v_other->device.device_type, kDLCUDA) << "The device of v_other must be CUDA.";
  CHECK_EQ(s_other->device.device_type, kDLCUDA) << "The device of s_other must be CUDA.";
  int32_t dev_id = v->device.device_id;
  CHECK_EQ(s->device.device_id, dev_id);
  CHECK_EQ(v_other->device.device_id, dev_id);
  CHECK_EQ(s_other->device.device_id, dev_id);

  CHECK(v->dtype.lanes == 1 && s->dtype.lanes == 1 && v_other->dtype.lanes == 1 &&
        s_other->dtype.lanes == 1);
  CHECK(v->dtype.bits == v_other->dtype.bits && v->dtype.code == v_other->dtype.code);
  CHECK(s->dtype.bits == 32 && s->dtype.code == kDLFloat);
  CHECK(s_other->dtype.bits == 32 && s_other->dtype.code == kDLFloat);

  CHECK_EQ(v->ndim, 3);
  CHECK_EQ(v_other->ndim, 3);
  CHECK_EQ(s->ndim, 2);        // qo_nnz, nhead_qo
  CHECK_EQ(s_other->ndim, 2);  // qo_nnz, nhead_qo
  int64_t batch_size = v->shape[0];
  int64_t num_heads = v->shape[1];
  int64_t head_dim = v->shape[2];
  CHECK_EQ(s->shape[0], batch_size);
  CHECK_EQ(s->shape[1], num_heads);
  CHECK_EQ(v_other->shape[0], batch_size);
  CHECK_EQ(v_other->shape[1], num_heads);
  CHECK_EQ(v_other->shape[2], head_dim);
  CHECK_EQ(s_other->shape[0], batch_size);
  CHECK_EQ(s_other->shape[1], num_heads);

  SWITCH_TVM_CUDA_DTYPE(v->dtype, dtype, {
    hipError_t status =
        MergeStateInPlace(static_cast<dtype*>(v->data), static_cast<float*>(s->data),
                          static_cast<dtype*>(v_other->data), static_cast<float*>(s_other->data),
                          batch_size, num_heads, head_dim);
    if (status != hipSuccess) {
      LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
    }
  });
}

void _FlashInferBatchQKApplyRotaryInPlace(DLTensor* q, DLTensor* k, DLTensor* indptr,
                                          DLTensor* offsets, int64_t batch_size,
                                          int64_t num_qo_heads, int64_t num_kv_heads,
                                          int64_t head_dim, double rope_scale, double rope_theta) {
  SWITCH_TVM_CUDA_DTYPE(
      q->dtype, dtype, {SWITCH_TVM_CUDA_IDTYPE(indptr->dtype, idtype, {
        hipError_t status = BatchQKApplyRotaryInPlace(
            static_cast<dtype*>(q->data), static_cast<dtype*>(k->data),
            static_cast<idtype*>(indptr->data), static_cast<idtype*>(offsets->data), batch_size,
            num_qo_heads, num_kv_heads, head_dim, rope_scale, rope_theta);
        if (status != hipSuccess) {
          LOG(FATAL) << "FlashInfer CUDA kernel error " << hipGetErrorString(status);
        }
      })});
}

// TODO(Zihao): Unify the symbol names
TVM_REGISTER_GLOBAL("paged_kv_cache.attention_kernel_prefill")
    .set_body_typed(_FlashInferAttentionPrefillWithPagedKVCache);

TVM_REGISTER_GLOBAL("paged_kv_cache.attention_kernel_prefill_begin_forward")
    .set_body_typed(_FlashInferAttentionPrefillWithPagedKVCacheBeginForward);

TVM_REGISTER_GLOBAL("paged_kv_cache.attention_kernel_prefill_end_forward")
    .set_body_typed(_FlashInferAttentionPrefillWithPagedKVCacheEndForward);

TVM_REGISTER_GLOBAL("paged_kv_cache.attention_kernel_decode")
    .set_body_typed(_FlashInferAttentionDecodeWithPagedKVCache);

TVM_REGISTER_GLOBAL("paged_kv_cache.attention_kernel_decode_begin_forward")
    .set_body_typed(_FlashInferAttentionDecodeWithPagedKVCacheBeginForward);

TVM_REGISTER_GLOBAL("paged_kv_cache.attention_kernel_decode_end_forward")
    .set_body_typed(_FlashInferAttentionDecodeWithPagedKVCacheEndForward);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_prefill_with_ragged_kv_cache")
    .set_body_typed(_FlashInferAttentionPrefillWithRaggedKVCache);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_prefill_with_ragged_kv_cache_begin_forward")
    .set_body_typed(_FlashInferAttentionPrefillWithRaggedKVCacheBeginForward);

TVM_REGISTER_GLOBAL("flashinfer.attention_kernel_prefill_with_ragged_kv_cache_end_forward")
    .set_body_typed(_FlashInferAttentionPrefillWithRaggedKVCacheEndForward);

TVM_REGISTER_GLOBAL("flashinfer.merge_state").set_body_typed(_FlashInferMergeState);

TVM_REGISTER_GLOBAL("flashinfer.merge_state_in_place").set_body_typed(_FlashInferMergeStateInPlace);

TVM_REGISTER_GLOBAL("flashinfer.batch_qk_apply_rotary_in_place")
    .set_body_typed(_FlashInferBatchQKApplyRotaryInPlace);

TVM_REGISTER_GLOBAL("flashinfer.single_prefill")
    .set_body_typed(_FlashInferSinglePrefillWithKVCache);
TVM_REGISTER_GLOBAL("flashinfer.single_decode").set_body_typed(_FlashInferSingleDecodeWithKVCache);
