/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/device_vector.h>

#include <flashinfer/cascade.cuh>
#include <nvbench/nvbench.cuh>

#include "utils.h"

template <typename T>
void bench_merge_states(nvbench::state& state) {
  const auto num_index_sets = state.get_int64("num_index_sets");
  const auto batch_size = state.get_int64("batch_size");
  const auto num_heads = state.get_int64("num_heads");
  const auto head_dim = state.get_int64("head_dim");

  std::vector<T> V_host(num_index_sets * batch_size * num_heads * head_dim);
  std::vector<float> S_host(num_index_sets * batch_size * num_heads);

  utils::vec_normal_(V_host);
  utils::vec_uniform_(S_host, 5, 10);

  thrust::device_vector<T> V_device(V_host);
  thrust::device_vector<float> S_device(S_host);
  thrust::device_vector<T> V_merged(batch_size * num_heads * head_dim);
  thrust::device_vector<float> S_merged(batch_size * num_heads);

  state.add_global_memory_reads<T>(V_host.size(), "Read");
  state.add_global_memory_writes<T>(V_merged.size(), "Write");

  state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
    timer.start();
    hipError_t status = flashinfer::MergeStates(
        thrust::raw_pointer_cast(V_device.data()), thrust::raw_pointer_cast(S_device.data()),
        thrust::raw_pointer_cast(V_merged.data()), thrust::raw_pointer_cast(S_merged.data()),
        num_index_sets, batch_size, num_heads, head_dim);
    timer.stop();
  });
}

#define STR_HELPER(x) #x
#define STR(x) STR_HELPER(x)
#define BENCH_FLASHINFER_MERGE_KERNELS(T)                               \
  auto bench_flashinfer_merge_states_##T##_ = bench_merge_states<T>;    \
  NVBENCH_BENCH(bench_flashinfer_merge_states_##T##_)                   \
      .set_name("flashinfer_merge_states_" STR(T))                      \
      .add_int64_axis("num_index_sets", {2, 16, 64, 128, 256})          \
      .add_int64_axis("batch_size", {1, 2, 4, 8, 16, 32, 64, 128, 256}) \
      .add_int64_axis("num_heads", {32})                                \
      .add_int64_axis("head_dim", {128})

BENCH_FLASHINFER_MERGE_KERNELS(half);
