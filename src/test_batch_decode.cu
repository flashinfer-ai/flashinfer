#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <gtest/gtest.h>

#include <flashinfer/attention/wrapper.cuh>
#include <type_traits>

#include "cpu_reference.h"
#include "utils.h"

using namespace flashinfer;

constexpr QKVLayout kv_layout = QKVLayout::kNHD;

template <typename T>
void _TestBatchDecodingKernelCorrectness(size_t page_size, size_t batch_size, size_t num_qo_heads,
                                         size_t num_kv_heads, size_t head_dim,
                                         flashinfer::PosEncodingMode pos_encoding_mode,
                                         bool cooperative) {
  std::vector<int32_t> seq_lens(batch_size);
  utils::vec_randint_(seq_lens, 1, 1024);
  std::vector<int32_t> append_indptr{0};
  for (size_t i = 0; i < batch_size; ++i) {
    append_indptr.push_back(append_indptr.back() + seq_lens[i]);
  }
  std::vector<T> q;
  std::vector<T> o_ref;
  std::vector<T> kv_data;
  std::vector<int32_t> kv_indptr{0};
  std::vector<int32_t> kv_indices;
  std::vector<int32_t> kv_last_page_len;
  size_t page_counter = 0;

  std::vector<std::vector<T>> keys, values;
  for (size_t i = 0; i < batch_size; ++i) {
    size_t seq_len = seq_lens[i];
    size_t num_pages = (seq_len + page_size - 1) / page_size;
    size_t last_page_len = (seq_len - 1) % page_size + 1;
    std::vector<T> qi(num_qo_heads * head_dim), ki(seq_len * num_kv_heads * head_dim),
        vi(seq_len * num_kv_heads * head_dim);
    utils::vec_normal_(qi);
    utils::vec_normal_(ki);
    utils::vec_normal_(vi);

    // compute reference output
    std::vector<T> o_ref_i =
        cpu_reference::single_mha<T, T>(qi, ki, vi, 1, seq_len, num_qo_heads, num_kv_heads,
                                        head_dim, false, QKVLayout::kNHD, pos_encoding_mode);
    keys.push_back(ki);
    values.push_back(vi);
    // append new q and o_ref
    q.insert(q.end(), qi.begin(), qi.end());
    o_ref.insert(o_ref.end(), o_ref_i.begin(), o_ref_i.end());
    // append new kv_indptr, kv_indices and kv_last_page_len
    kv_last_page_len.push_back(last_page_len);
    kv_indptr.push_back(kv_indptr.back() + num_pages);
    for (size_t j = 0; j < num_pages; ++j) {
      kv_indices.push_back(page_counter++);
    }
  }
  kv_data.resize(page_counter * 2 * num_kv_heads * page_size * head_dim);
  utils::vec_zero_(kv_data);
  assert(q.size() == batch_size * num_qo_heads * head_dim);
  assert(o_ref.size() == batch_size * num_qo_heads * head_dim);

  flashinfer::paged_kv_t<PageStorage::kIndices, kv_layout, T, int32_t> paged_kv_cpu(
      num_kv_heads, page_size, head_dim, batch_size, kv_data.data(), kv_indices.data(),
      kv_indptr.data(), kv_last_page_len.data());
  cpu_reference::append_paged_kv_cache<kv_layout, T, int32_t>(paged_kv_cpu, keys, values,
                                                              append_indptr);

  // copy data to device
  thrust::device_vector<T> kv_data_device(kv_data);
  thrust::device_vector<int32_t> kv_indptr_device(kv_indptr);
  thrust::device_vector<int32_t> kv_indices_device(kv_indices);
  thrust::device_vector<int32_t> kv_last_page_len_device(kv_last_page_len);
  thrust::device_vector<T> q_device(q);
  thrust::device_vector<T> o_device(o_ref.size());

  // create paged_kv object
  flashinfer::paged_kv_t<PageStorage::kIndices, kv_layout, T, int32_t> paged_kv(
      num_kv_heads, page_size, head_dim, batch_size,
      thrust::raw_pointer_cast(kv_data_device.data()),
      thrust::raw_pointer_cast(kv_indices_device.data()),
      thrust::raw_pointer_cast(kv_indptr_device.data()),
      thrust::raw_pointer_cast(kv_last_page_len_device.data()));
  flashinfer::BatchDecodeHandler handler;
  size_t workspace_size_in_bytes = 32 * 1024 * 1024;
  thrust::device_vector<char> buffer(workspace_size_in_bytes);
  handler.BeginForward<PageStorage::kIndices, kv_layout, T, T, int32_t>(
      (void*)thrust::raw_pointer_cast(buffer.data()), workspace_size_in_bytes, kv_indptr.data(),
      kv_last_page_len.data(), batch_size, num_qo_heads, num_kv_heads, head_dim, page_size,
      pos_encoding_mode);

  if (!cooperative) {
    // use non-cooperative kernel
    hipError_t status =
        flashinfer::BatchDecodeWithPagedKVCache<PageStorage::kIndices, kv_layout, T, T, int32_t>(
            thrust::raw_pointer_cast(q_device.data()), /*q_offset=*/nullptr, paged_kv,
            kv_partition_info_t<int32_t>(), thrust::raw_pointer_cast(o_device.data()),
            /*tmp=*/nullptr, /*lse=*/nullptr, num_qo_heads, pos_encoding_mode);
    EXPECT_EQ(status, hipSuccess) << "CUDA error: " + std::string(hipGetErrorString(status));
  } else {
    hipError_t status = flashinfer::BatchDecodeWithPagedKVCacheWrapper<PageStorage::kIndices,
                                                                        kv_layout, T, T, int32_t>(
        &handler, thrust::raw_pointer_cast(q_device.data()), /*q_offset=*/nullptr, paged_kv,
        thrust::raw_pointer_cast(o_device.data()), /*lse=*/nullptr, num_qo_heads,
        pos_encoding_mode);
    EXPECT_EQ(status, hipSuccess) << "CUDA error: " + std::string(hipGetErrorString(status));
  }
  // compare result
  thrust::host_vector<T> o_host = o_device;
  size_t num_result_errors_atol_1e_3_rtol_1e_3 = 0;
  bool nan_detected = false;
  for (size_t i = 0; i < batch_size * num_qo_heads * head_dim; ++i) {
    if (std::isnan(float(o_host[i]))) {
      nan_detected = true;
    }
    num_result_errors_atol_1e_3_rtol_1e_3 +=
        (!utils::isclose(float(o_host[i]), float(o_ref[i]), 1e-3, 1e-3));
  }
  float result_accuracy = 1. - float(num_result_errors_atol_1e_3_rtol_1e_3) /
                                   float(batch_size * num_qo_heads * head_dim);
  std::cout << "page_size=" << page_size << ", num_qo_heads=" << num_qo_heads
            << ", num_kv_heads=" << num_kv_heads << ", batch_size=" << batch_size
            << ", head_dim=" << head_dim
            << ", pos_encoding_mode=" << flashinfer::PosEncodingModeToString(pos_encoding_mode)
            << ", result accuracy (atol=1e-3, rtol=1e-3): " << result_accuracy << std::endl;
  EXPECT_GT(result_accuracy, 0.90) << "Result correctness test failed.";
  EXPECT_EQ(nan_detected, false) << "NaN detected.";
}

template <typename T>
void TestBatchDecodeKernelCorrectness() {
  for (size_t page_size : {1, 3, 7, 16}) {
    for (size_t batch_size : {1, 7, 37, 61}) {
      for (size_t num_qo_heads : {32}) {
        for (size_t num_kv_heads : {32, 8, 4}) {
          for (size_t head_dim : {64, 128, 256}) {
            for (size_t pos_encoding_mode : {0U, 1U}) {
              _TestBatchDecodingKernelCorrectness<T>(
                  page_size, batch_size, num_qo_heads, num_kv_heads, head_dim,
                  flashinfer::PosEncodingMode(pos_encoding_mode), false);
            }
          }
        }
      }
    }
  }
}

template <typename T>
void TestCooperativeBatchDecodeKernelCorrectness() {
  for (size_t page_size : {1, 3, 7, 16}) {
    for (size_t batch_size : {1, 2, 4, 8}) {
      for (size_t num_qo_heads : {32}) {
        for (size_t num_kv_heads : {32, 8, 4}) {
          for (size_t head_dim : {64, 128, 256}) {
            for (size_t pos_encoding_mode : {0U, 1U}) {
              _TestBatchDecodingKernelCorrectness<T>(
                  page_size, batch_size, num_qo_heads, num_kv_heads, head_dim,
                  flashinfer::PosEncodingMode(pos_encoding_mode), true);
            }
          }
        }
      }
    }
  }
}

TEST(FlashInferCorrectnessTest, BatchDecodeKernelCorrectnessTestFP16) {
  TestBatchDecodeKernelCorrectness<half>();
}

#ifdef FLASHINFER_ENABLE_BF16
TEST(FlashInferCorrectnessTest, TestBatchDecodeKernelCorrectnessBF16) {
  TestBatchDecodeKernelCorrectness<__hip_bfloat16>();
}
#endif

TEST(FlashInferCorrectnessTest, TestBatchDecodeKernelCorrectnessFP32) {
  TestBatchDecodeKernelCorrectness<float>();
}

#ifdef FLASHINFER_ENABLE_FP8
TEST(FlashInferCorrectnessTest, TestBatchDecodeKernelCorrectnessE4M3) {
  TestBatchDecodeKernelCorrectness<__hip_fp8_e4m3_fnuz>();
}

TEST(FlashInferCorrectnessTest, TestBatchDecodeKernelCorrectnessE5M2) {
  TestBatchDecodeKernelCorrectness<__hip_fp8_e5m2_fnuz>();
}
#endif

TEST(FlashInferCorrectnessTest, TestCooperativeBatchDecodeKernelCorrectnessTestFP16) {
  TestCooperativeBatchDecodeKernelCorrectness<half>();
}
