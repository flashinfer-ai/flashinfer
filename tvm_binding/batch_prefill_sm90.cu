/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <flashinfer/attention/hopper/utils.cuh>
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/layout.cuh>
#include <flashinfer/math.cuh>
#include <optional>

#include "batch_prefill_sm90_config.inc"
#include "tvm_binding_utils.h"

namespace flashinfer {

template <uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO, MaskMode MASK_MODE, bool LEFT_SLIDING_WINDOW,
          bool SAME_SCHEDULE_FOR_ALL_HEADS, typename AttentionVariant, typename Params>
hipError_t BatchPrefillWithRaggedKVCacheDispatched(Params& params, hipStream_t stream);

template <uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO, MaskMode MASK_MODE, bool LEFT_SLIDING_WINDOW,
          bool SAME_SCHEDULE_FOR_ALL_HEADS, typename AttentionVariant, typename Params>
hipError_t BatchPrefillWithPagedKVCacheDispatched(Params& params, hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

IntTuple BatchPrefillWithKVCacheSM90Plan(
    DLTensor* float_workspace_buffer, DLTensor* int_workspace_buffer,
    DLTensor* page_locked_int_workspace_buffer, DLTensor* qo_indptr, DLTensor* kv_indptr,
    IntTuple kv_len_arr, int64_t total_num_rows, int64_t batch_size, int64_t num_qo_heads,
    int64_t num_kv_heads, int64_t page_size, bool enable_cuda_graph, int64_t head_dim_qk,
    int64_t head_dim_vo, bool causal, TVMStreamHandle hip_stream) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer->shape[0] * DataType(float_workspace_buffer->dtype).bytes();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer->shape[0] * DataType(int_workspace_buffer->dtype).bytes();
  std::vector<IdType> kv_len_vec{kv_len_arr->data, kv_len_arr->data + kv_len_arr->size};

  flashinfer::PrefillPlanSM90Info plan_info;

  hipStream_t stream = static_cast<hipStream_t>(hip_stream);

  hipError_t status = PrefillSM90Plan(
      static_cast<char*>(float_workspace_buffer->data) + float_workspace_buffer->byte_offset,
      float_workspace_size_in_bytes,
      static_cast<char*>(int_workspace_buffer->data) + int_workspace_buffer->byte_offset,
      static_cast<char*>(page_locked_int_workspace_buffer->data) +
          page_locked_int_workspace_buffer->byte_offset,
      int_workspace_size_in_bytes, plan_info,
      static_cast<IdType*>(qo_indptr->data) + qo_indptr->byte_offset / sizeof(IdType),
      static_cast<IdType*>(kv_indptr->data) + kv_indptr->byte_offset / sizeof(IdType),
      kv_len_vec.data(), total_num_rows, batch_size, num_qo_heads, num_kv_heads, head_dim_qk,
      head_dim_vo, page_size, causal, enable_cuda_graph,
      /*sizeof_dtype_o=*/2, stream);

  CHECK(status == hipSuccess) << "PrefillSM90Plan failed with error: "
                               << hipGetErrorString(status);

  std::vector<int64_t> plan_info_vec = plan_info.ToVector();
  return IntTuple{plan_info_vec.begin(), plan_info_vec.end()};
}

void BatchPrefillWithRaggedKVCacheSM90Run(
    DLTensor* float_workspace_buffer, DLTensor* int_workspace_buffer, IntTuple plan_info_vec,
    DLTensor* q, DLTensor* k, DLTensor* v, DLTensor* qo_indptr, DLTensor* kv_indptr,
    DLTensor* q_rope_offset, DLTensor* k_rope_offset, DLTensor* o, DLTensor* lse,
    int64_t mask_mode_code, int64_t pos_encoding_mode_code, int64_t layout,
    int64_t window_left ADDITIONAL_FUNC_PARAMS, TVMStreamHandle hip_stream) {
  PrefillPlanSM90Info plan_info;
  std::vector<int64_t> plan_info_vec_(plan_info_vec->data,
                                      plan_info_vec->data + plan_info_vec->size);
  plan_info.FromVector(plan_info_vec_);

  CHECK(lse->shape[0] == q->shape[0]) << "LSE shape mismatch on dim 0";
  CHECK(lse->shape[1] == q->shape[1]) << "LSE shape mismatch on dim 1";

  void* float_buffer_ptr =
      static_cast<char*>(float_workspace_buffer->data) + float_workspace_buffer->byte_offset;
  void* int_buffer_ptr =
      static_cast<char*>(int_workspace_buffer->data) + int_workspace_buffer->byte_offset;

  int64_t head_dim_qk = q->shape[2];
  int64_t head_dim_vo = v->shape[2];

  DataType q_scalar_type(q->dtype);
  DataType kv_scalar_type(k->dtype);

  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  hipStream_t stream = static_cast<hipStream_t>(hip_stream);
  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  const PosEncodingMode pos_encoding_mode = static_cast<PosEncodingMode>(pos_encoding_mode_code);
  bool use_swa = window_left != -1;

  int64_t q_strides[3] = {q->strides ? q->strides[0] : q->shape[1] * q->shape[2],  //
                          q->strides ? q->strides[1] : q->shape[2],                //
                          q->strides ? q->strides[2] : 1};
  int64_t k_strides[3] = {k->strides ? k->strides[0] : k->shape[1] * k->shape[2],  //
                          k->strides ? k->strides[1] : k->shape[2],                //
                          k->strides ? k->strides[2] : 1};
  int64_t v_strides[3] = {v->strides ? v->strides[0] : v->shape[1] * v->shape[2],  //
                          v->strides ? v->strides[1] : v->shape[2],                //
                          v->strides ? v->strides[2] : 1};
  int64_t o_strides[3] = {o->strides ? o->strides[0] : o->shape[1] * o->shape[2],  //
                          o->strides ? o->strides[1] : o->shape[2],                //
                          o->strides ? o->strides[2] : 1};
  uint32_t q_stride_n = q_strides[0], q_stride_h = q_strides[1];
  uint32_t o_stride_n = o_strides[0], o_stride_h = o_strides[1];
  uint32_t k_stride_n, k_stride_h, v_stride_n, v_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    k_stride_n = k_strides[0];
    k_stride_h = k_strides[1];
    v_stride_n = v_strides[0];
    v_stride_h = v_strides[1];
  } else {
    k_stride_h = k_strides[0];
    k_stride_n = k_strides[1];
    v_stride_h = v_strides[0];
    v_stride_n = v_strides[1];
  }

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, USE_SLIDING_WINDOW,
      USE_LOGITS_SOFT_CAP, AttentionVariant, RaggedParams, PagedParams, [&] {
        RaggedParams params;

        params.q_ptr = static_cast<DTypeQ*>(q->data) + q->byte_offset / sizeof(DTypeQ);
        params.k_ptr = static_cast<DTypeKV*>(k->data) + k->byte_offset / sizeof(DTypeKV);
        params.v_ptr = static_cast<DTypeKV*>(v->data) + v->byte_offset / sizeof(DTypeKV);
        params.o_ptr = static_cast<DTypeO*>(o->data) + o->byte_offset / sizeof(DTypeO);
        params.lse_ptr = static_cast<float*>(lse->data) + lse->byte_offset / sizeof(float);
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.o_stride_n = o_stride_n;
        params.o_stride_h = o_stride_h;
        params.k_stride_n = k_stride_n;
        params.k_stride_h = k_stride_h;
        params.v_stride_n = v_stride_n;
        params.v_stride_h = v_stride_h;
        params.nnz_qo = q->shape[0];
        params.nnz_kv = k->shape[0];
        params.num_qo_heads = q->shape[1];
        params.num_kv_heads = k->shape[1];
        params.group_size = params.num_qo_heads / params.num_kv_heads;
        params.maybe_q_rope_offset = q_rope_offset != nullptr
                                         ? static_cast<IdType*>(q_rope_offset->data) +
                                               q_rope_offset->byte_offset / sizeof(IdType)
                                         : nullptr;
        params.maybe_k_rope_offset = k_rope_offset != nullptr
                                         ? static_cast<IdType*>(k_rope_offset->data) +
                                               k_rope_offset->byte_offset / sizeof(IdType)
                                         : nullptr;
        params.window_left = window_left;
        params.causal = mask_mode_code == 1;
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.qo_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_indptr_offset);
        params.kv_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_indptr_offset);
        params.qo_lens = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_len_offset);
        params.kv_lens = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_len_offset);
        params.head_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.head_indices_offset);
        params.work_indptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.work_indptr_offset);

        ADDITIONAL_PARAMS_SETTER

        bool same_schedule_for_all_heads = plan_info.same_schedule_for_all_heads;
        DISPATCH_BOOL(same_schedule_for_all_heads, SAME_SCHEDULER_FOR_ALL_HEADS, [&] {
          hipError_t status = BatchPrefillWithRaggedKVCacheDispatched<
              HEAD_DIM_QK, HEAD_DIM_VO, MASK_MODE, USE_SLIDING_WINDOW, SAME_SCHEDULER_FOR_ALL_HEADS,
              AttentionVariant>(params, stream);
          CHECK(status == hipSuccess) << "BatchPrefillWithRaggedKVCacheSM90Run failed with error: "
                                       << hipGetErrorString(status);
          return true;
        });
      });
}

void BatchPrefillWithPagedKVCacheSM90Run(
    DLTensor* float_workspace_buffer, DLTensor* int_workspace_buffer, IntTuple plan_info_vec,
    DLTensor* q, DLTensor* paged_kv_cache, DLTensor* qo_indptr, DLTensor* paged_kv_indptr,
    DLTensor* paged_kv_indices, DLTensor* paged_kv_last_page_len, DLTensor* q_rope_offset,
    DLTensor* paged_kv_rope_pos_offset, DLTensor* o, DLTensor* lse, int64_t mask_mode_code,
    int64_t pos_encoding_mode_code, int64_t layout, int64_t window_left ADDITIONAL_FUNC_PARAMS,
    TVMStreamHandle hip_stream) {
  PrefillPlanSM90Info plan_info;
  std::vector<int64_t> plan_info_vec_(plan_info_vec->data,
                                      plan_info_vec->data + plan_info_vec->size);
  plan_info.FromVector(plan_info_vec_);

  CHECK(lse->shape[0] == q->shape[0]) << "LSE shape mismatch on dim 0";
  CHECK(lse->shape[1] == q->shape[1]) << "LSE shape mismatch on dim 1";

  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  int64_t num_kv_heads, page_size;
  int64_t head_dim_qk = q->shape[2];
  int64_t head_dim_vo = paged_kv_cache->shape[3];
  if (kv_layout == QKVLayout::kHND) {
    num_kv_heads = paged_kv_cache->shape[2];
    page_size = paged_kv_cache->shape[3];
  } else {
    page_size = paged_kv_cache->shape[2];
    num_kv_heads = paged_kv_cache->shape[3];
  }

  void* float_buffer_ptr =
      static_cast<char*>(float_workspace_buffer->data) + float_workspace_buffer->byte_offset;
  void* int_buffer_ptr =
      static_cast<char*>(int_workspace_buffer->data) + int_workspace_buffer->byte_offset;

  DataType q_scalar_type(q->dtype);
  DataType kv_scalar_type(paged_kv_cache->dtype);

  hipStream_t stream = static_cast<hipStream_t>(hip_stream);
  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  const PosEncodingMode pos_encoding_mode = static_cast<PosEncodingMode>(pos_encoding_mode_code);
  bool use_swa = window_left != -1;

  // get q_stride_n and q_stride_h
  int64_t q_strides[3] = {q->strides ? q->strides[0] : q->shape[1] * q->shape[2],  //
                          q->strides ? q->strides[1] : q->shape[2],                //
                          q->strides ? q->strides[2] : 1};
  int64_t o_strides[3] = {o->strides ? o->strides[0] : o->shape[1] * o->shape[2],  //
                          o->strides ? o->strides[1] : o->shape[2],                //
                          o->strides ? o->strides[2] : 1};
  const auto q_stride_n = q_strides[0];
  const auto q_stride_h = q_strides[1];
  const auto o_stride_n = o_strides[0];
  const auto o_stride_h = o_strides[1];

  // get kv_cache_strides
  int64_t kv_cache_strides[4] = {
      paged_kv_cache->strides ? paged_kv_cache->strides[0]
                              : paged_kv_cache->shape[1] * paged_kv_cache->shape[2] *
                                    paged_kv_cache->shape[3] * paged_kv_cache->shape[4],
      paged_kv_cache->strides ? paged_kv_cache->strides[2]
                              : paged_kv_cache->shape[3] * paged_kv_cache->shape[4],    //
      paged_kv_cache->strides ? paged_kv_cache->strides[3] : paged_kv_cache->shape[4],  //
      paged_kv_cache->strides ? paged_kv_cache->strides[4] : 1};
  int64_t v_offset = paged_kv_cache->strides ? paged_kv_cache->strides[1]
                                             : paged_kv_cache->shape[2] * paged_kv_cache->shape[3] *
                                                   paged_kv_cache->shape[4];

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, USE_SLIDING_WINDOW,
      USE_LOGITS_SOFT_CAP, AttentionVariant, RaggedParams, PagedParams, [&] {
        PagedParams params;

        params.q_ptr = static_cast<DTypeQ*>(q->data) + q->byte_offset / sizeof(DTypeQ);
        params.k_ptr = static_cast<DTypeKV*>(paged_kv_cache->data) +
                       paged_kv_cache->byte_offset / sizeof(DTypeKV);
        params.v_ptr = static_cast<DTypeKV*>(paged_kv_cache->data) +
                       paged_kv_cache->byte_offset / sizeof(DTypeKV) + v_offset;
        params.o_ptr = static_cast<DTypeO*>(o->data) + o->byte_offset / sizeof(DTypeO);
        params.lse_ptr = static_cast<float*>(lse->data) + lse->byte_offset / sizeof(float);
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.o_stride_n = o_stride_n;
        params.o_stride_h = o_stride_h;
        if (kv_layout == QKVLayout::kNHD) {
          // (num_pages, page_size, num_heads, head_dim)
          params.k_stride_n = kv_cache_strides[1];
          params.k_stride_h = kv_cache_strides[2];
          params.v_stride_n = kv_cache_strides[1];
          params.v_stride_h = kv_cache_strides[2];
        } else {
          // (num_pages, num_heads, page_size, head_dim)
          params.k_stride_h = kv_cache_strides[1];
          params.k_stride_n = kv_cache_strides[2];
          params.v_stride_h = kv_cache_strides[1];
          params.v_stride_n = kv_cache_strides[2];
        }
        params.nnz_qo = q->shape[0];
        params.num_qo_heads = q->shape[1];
        params.num_kv_heads = num_kv_heads;
        params.group_size = params.num_qo_heads / num_kv_heads;
        params.maybe_q_rope_offset = q_rope_offset != nullptr
                                         ? static_cast<IdType*>(q_rope_offset->data) +
                                               q_rope_offset->byte_offset / sizeof(IdType)
                                         : nullptr;
        params.page_size = page_size;
        params.window_left = window_left;
        params.causal = mask_mode_code == 1;
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.qo_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_indptr_offset);
        params.kv_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_indptr_offset);
        params.qo_lens = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_len_offset);
        params.kv_lens = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_len_offset);
        params.head_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.head_indices_offset);
        params.work_indptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.work_indptr_offset);
        params.kv_indices = static_cast<IdType*>(paged_kv_indices->data) +
                            paged_kv_indices->byte_offset / sizeof(IdType);

        ADDITIONAL_PARAMS_SETTER

        bool same_schedule_for_all_heads = plan_info.same_schedule_for_all_heads;
        DISPATCH_BOOL(same_schedule_for_all_heads, SAME_SCHEDULER_FOR_ALL_HEADS, [&] {
          hipError_t status = BatchPrefillWithPagedKVCacheDispatched<
              HEAD_DIM_QK, HEAD_DIM_VO, MASK_MODE, USE_SLIDING_WINDOW, SAME_SCHEDULER_FOR_ALL_HEADS,
              AttentionVariant>(params, stream);
          CHECK(status == hipSuccess) << "BatchPrefillWithPagedKVCacheSM90Run failed with error: "
                                       << hipGetErrorString(status);
          return true;
        });
      });
}
