/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/pos_enc.cuh>
#include <optional>

#include "batch_prefill_config.inc"
#include "tvm_binding_utils.h"

namespace flashinfer {

template <uint32_t CTA_TILE_Q, uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO,
          PosEncodingMode POS_ENCODING_MODE, bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE,
          typename AttentionVariant, typename Params>
hipError_t BatchPrefillWithPagedKVCacheDispatched(Params params, typename Params::DTypeO* tmp_v,
                                                   float* tmp_s, hipStream_t stream);

template <uint32_t CTA_TILE_Q, uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO,
          PosEncodingMode POS_ENCODING_MODE, bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE,
          typename AttentionVariant, typename Params>
hipError_t BatchPrefillWithRaggedKVCacheDispatched(Params params, typename Params::DTypeO* tmp_v,
                                                    float* tmp_s, hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

IntTuple BatchPrefillWithKVCachePlan(
    DLTensor* float_workspace_buffer, DLTensor* int_workspace_buffer,
    DLTensor* page_locked_int_workspace_buffer, DLTensor* qo_indptr, DLTensor* kv_indptr,
    IntTuple kv_len_arr, int64_t total_num_rows, int64_t batch_size, int64_t num_qo_heads,
    int64_t num_kv_heads, int64_t page_size, bool enable_cuda_graph, int64_t head_dim_qk,
    int64_t head_dim_vo, bool causal, TVMStreamHandle hip_stream) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer->shape[0] * DataType(float_workspace_buffer->dtype).bytes();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer->shape[0] * DataType(int_workspace_buffer->dtype).bytes();

  PrefillPlanInfo plan_info;

  hipStream_t stream = static_cast<hipStream_t>(hip_stream);
  hipError_t status = PrefillPlan<IdType>(
      static_cast<char*>(float_workspace_buffer->data) + float_workspace_buffer->byte_offset,
      float_workspace_size_in_bytes,
      static_cast<char*>(int_workspace_buffer->data) + int_workspace_buffer->byte_offset,
      static_cast<char*>(page_locked_int_workspace_buffer->data) +
          page_locked_int_workspace_buffer->byte_offset,
      int_workspace_size_in_bytes, plan_info,
      static_cast<IdType*>(qo_indptr->data) + qo_indptr->byte_offset / sizeof(IdType),
      static_cast<IdType*>(kv_indptr->data) + kv_indptr->byte_offset / sizeof(IdType),
      total_num_rows, batch_size, num_qo_heads, num_kv_heads, head_dim_qk, head_dim_vo, page_size,
      enable_cuda_graph,
      /*sizeof_dtype_o=*/2, stream);

  CHECK(status == hipSuccess) << "Failed to plan prefill with error: "
                               << hipGetErrorString(status);

  std::vector<int64_t> plan_info_vec = plan_info.ToVector();
  return IntTuple{plan_info_vec.begin(), plan_info_vec.end()};
}

void BatchPrefillWithRaggedKVCacheRun(DLTensor* float_workspace_buffer,
                                      DLTensor* int_workspace_buffer, IntTuple plan_info_vec,
                                      DLTensor* q, DLTensor* k, DLTensor* v, DLTensor* qo_indptr,
                                      DLTensor* kv_indptr, DLTensor* q_rope_offset,
                                      DLTensor* k_rope_offset, DLTensor* o, DLTensor* lse,
                                      int64_t mask_mode_code, int64_t pos_encoding_mode_code,
                                      int64_t layout, int64_t window_left ADDITIONAL_FUNC_PARAMS,
                                      TVMStreamHandle hip_stream) {
  PrefillPlanInfo plan_info;
  std::vector<int64_t> plan_info_vec_(plan_info_vec->data,
                                      plan_info_vec->data + plan_info_vec->size);
  plan_info.FromVector(plan_info_vec_);
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);

  int64_t num_qo_heads = q->shape[1];
  int64_t head_dim_qk = q->shape[2];
  int64_t num_kv_heads = (kv_layout == QKVLayout::kNHD) ? k->shape[1] : k->shape[0];
  int64_t q_strides[3] = {q->strides ? q->strides[0] : q->shape[1] * q->shape[2],  //
                          q->strides ? q->strides[1] : q->shape[2],                //
                          q->strides ? q->strides[2] : 1};
  int64_t k_strides[3] = {k->strides ? k->strides[0] : k->shape[1] * k->shape[2],  //
                          k->strides ? k->strides[1] : k->shape[2],                //
                          k->strides ? k->strides[2] : 1};
  int64_t v_strides[3] = {v->strides ? v->strides[0] : v->shape[1] * v->shape[2],  //
                          v->strides ? v->strides[1] : v->shape[2],                //
                          v->strides ? v->strides[2] : 1};
  uint32_t q_stride_n = q_strides[0], q_stride_h = q_strides[1];
  uint32_t k_stride_n, k_stride_h, v_stride_n, v_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    k_stride_n = k_strides[0];
    k_stride_h = k_strides[1];
    v_stride_n = v_strides[0];
    v_stride_h = v_strides[1];
  } else {
    k_stride_h = k_strides[0];
    k_stride_n = k_strides[1];
    v_stride_h = v_strides[0];
    v_stride_n = v_strides[1];
  }

  CHECK(lse->shape[0] == q->shape[0]) << "LSE shape mismatch on dim 0";
  CHECK(lse->shape[1] == q->shape[1]) << "LSE shape mismatch on dim 1";

  void* float_buffer_ptr =
      static_cast<char*>(float_workspace_buffer->data) + float_workspace_buffer->byte_offset;
  void* int_buffer_ptr =
      static_cast<char*>(int_workspace_buffer->data) + int_workspace_buffer->byte_offset;

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  const PosEncodingMode pos_encoding_mode = static_cast<PosEncodingMode>(pos_encoding_mode_code);

  DataType q_scalar_type(q->dtype);
  DataType kv_scalar_type(k->dtype);

  hipStream_t stream = static_cast<hipStream_t>(hip_stream);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
      USE_SLIDING_WINDOW, USE_LOGITS_SOFT_CAP, USE_FP16_QK_REDUCTION, AttentionVariant,
      RaggedParams, PagedParams, [&] {
        RaggedParams params;

        params.q = static_cast<DTypeQ*>(q->data) + q->byte_offset / sizeof(DTypeQ);
        params.k = static_cast<DTypeKV*>(k->data) + k->byte_offset / sizeof(DTypeKV);
        params.v = static_cast<DTypeKV*>(v->data) + v->byte_offset / sizeof(DTypeKV);
        params.o = static_cast<DTypeO*>(o->data) + o->byte_offset / sizeof(DTypeO);
        params.lse = static_cast<float*>(lse->data) + lse->byte_offset / sizeof(float);
        params.q_indptr =
            static_cast<IdType*>(qo_indptr->data) + qo_indptr->byte_offset / sizeof(IdType);
        params.kv_indptr =
            static_cast<IdType*>(kv_indptr->data) + kv_indptr->byte_offset / sizeof(IdType);
        params.num_qo_heads = num_qo_heads;
        params.num_kv_heads = num_kv_heads;
        params.group_size = uint_fastdiv(num_qo_heads / num_kv_heads);
        params.maybe_q_rope_offset = q_rope_offset != nullptr
                                         ? static_cast<IdType*>(q_rope_offset->data) +
                                               q_rope_offset->byte_offset / sizeof(IdType)
                                         : nullptr;
        params.maybe_k_rope_offset = k_rope_offset != nullptr
                                         ? static_cast<IdType*>(k_rope_offset->data) +
                                               k_rope_offset->byte_offset / sizeof(IdType)
                                         : nullptr;
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.k_stride_n = k_stride_n;
        params.k_stride_h = k_stride_h;
        params.v_stride_n = v_stride_n;
        params.v_stride_h = v_stride_h;
        params.window_left = window_left;

        params.request_indices = nullptr;
        params.qo_tile_indices = nullptr;
        params.kv_tile_indices = nullptr;
        params.merge_indptr = nullptr;
        params.o_indptr = nullptr;
        params.kv_chunk_size_ptr = nullptr;
        params.block_valid_mask = nullptr;
        params.total_num_rows = nullptr;
        params.max_total_num_rows = 0;
        params.padded_batch_size = 0;
        params.partition_kv = false;

        ADDITIONAL_PARAMS_SETTER

        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;

        params.request_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.kv_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
        params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
        params.kv_chunk_size_ptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset);
        if (plan_info.split_kv) {
          params.merge_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
          tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
          tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
          if (plan_info.enable_cuda_graph) {
            params.block_valid_mask =
                GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
          }
        }
        params.padded_batch_size = plan_info.padded_batch_size;
        params.max_total_num_rows = plan_info.total_num_rows;
        if (plan_info.enable_cuda_graph) {
          params.total_num_rows =
              GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
        }

        hipError_t status = hipSuccess;

        DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q, CTA_TILE_Q, {
          status = flashinfer::BatchPrefillWithRaggedKVCacheDispatched<
              CTA_TILE_Q, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
              /*use_fp16_qk_reduction=*/USE_FP16_QK_REDUCTION, MASK_MODE, AttentionVariant,
              RaggedParams>(params, tmp_v, tmp_s, stream);
        });

        CHECK(status == hipSuccess)
            << "BatchPrefillWithRaggedKVCache failed with error " << hipGetErrorString(status);
        return true;
      });
}

void BatchPrefillWithPagedKVCacheRun(DLTensor* float_workspace_buffer,
                                     DLTensor* int_workspace_buffer, IntTuple plan_info_vec,
                                     DLTensor* q, DLTensor* paged_kv_cache, DLTensor* qo_indptr,
                                     DLTensor* paged_kv_indptr, DLTensor* paged_kv_indices,
                                     DLTensor* paged_kv_last_page_len, DLTensor* q_rope_offset,
                                     DLTensor* paged_kv_rope_pos_offset, DLTensor* o, DLTensor* lse,
                                     int64_t mask_mode_code, int64_t pos_encoding_mode_code,
                                     int64_t layout, int64_t window_left ADDITIONAL_FUNC_PARAMS,
                                     TVMStreamHandle hip_stream) {
  PrefillPlanInfo plan_info;
  std::vector<int64_t> plan_info_vec_(plan_info_vec->data,
                                      plan_info_vec->data + plan_info_vec->size);
  plan_info.FromVector(plan_info_vec_);
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  int64_t batch_size = paged_kv_indptr->shape[0] - 1;
  int64_t num_qo_heads = q->shape[1];
  int64_t num_kv_heads, page_size;
  uint32_t head_dim_qk = q->shape[2];
  if (kv_layout == QKVLayout::kHND) {
    num_kv_heads = paged_kv_cache->shape[2];
    page_size = paged_kv_cache->shape[3];
  } else {
    page_size = paged_kv_cache->shape[2];
    num_kv_heads = paged_kv_cache->shape[3];
  }

  CHECK(lse->shape[0] == q->shape[0]) << "LSE shape mismatch on dim 0";
  CHECK(lse->shape[1] == q->shape[1]) << "LSE shape mismatch on dim 1";

  void* float_buffer_ptr =
      static_cast<char*>(float_workspace_buffer->data) + float_workspace_buffer->byte_offset;
  void* int_buffer_ptr =
      static_cast<char*>(int_workspace_buffer->data) + int_workspace_buffer->byte_offset;

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  const PosEncodingMode pos_encoding_mode = static_cast<PosEncodingMode>(pos_encoding_mode_code);
  DataType q_scalar_type(q->dtype);
  DataType kv_scalar_type(paged_kv_cache->dtype);

  // get q_stride_n and q_stride_h
  int64_t q_strides[3] = {q->strides ? q->strides[0] : q->shape[1] * q->shape[2],  //
                          q->strides ? q->strides[1] : q->shape[2],                //
                          q->strides ? q->strides[2] : 1};
  const auto q_stride_n = q_strides[0];
  const auto q_stride_h = q_strides[1];

  // get kv_cache_strides
  int64_t kv_cache_strides[4] = {
      paged_kv_cache->strides ? paged_kv_cache->strides[0]
                              : paged_kv_cache->shape[1] * paged_kv_cache->shape[2] *
                                    paged_kv_cache->shape[3] * paged_kv_cache->shape[4],
      paged_kv_cache->strides ? paged_kv_cache->strides[2]
                              : paged_kv_cache->shape[3] * paged_kv_cache->shape[4],    //
      paged_kv_cache->strides ? paged_kv_cache->strides[3] : paged_kv_cache->shape[4],  //
      paged_kv_cache->strides ? paged_kv_cache->strides[4] : 1};
  int64_t v_offset = paged_kv_cache->strides ? paged_kv_cache->strides[1]
                                             : paged_kv_cache->shape[2] * paged_kv_cache->shape[3] *
                                                   paged_kv_cache->shape[4];

  hipStream_t stream = static_cast<hipStream_t>(hip_stream);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
      USE_SLIDING_WINDOW, USE_LOGITS_SOFT_CAP, USE_FP16_QK_REDUCTION, AttentionVariant,
      RaggedParams, PagedParams, [&] {
        PagedParams params;

        params.q = static_cast<DTypeQ*>(q->data) + q->byte_offset / sizeof(DTypeQ);
        paged_kv_t<DTypeKV, IdType> paged_kv(
            num_kv_heads, page_size, HEAD_DIM_VO, batch_size, kv_layout,
            static_cast<DTypeKV*>(paged_kv_cache->data) +
                paged_kv_cache->byte_offset / sizeof(DTypeKV),
            static_cast<DTypeKV*>(paged_kv_cache->data) +
                paged_kv_cache->byte_offset / sizeof(DTypeKV) + v_offset,
            kv_cache_strides,
            static_cast<IdType*>(paged_kv_indices->data) +
                paged_kv_indices->byte_offset / sizeof(IdType),
            static_cast<IdType*>(paged_kv_indptr->data) +
                paged_kv_indptr->byte_offset / sizeof(IdType),
            static_cast<IdType*>(paged_kv_last_page_len->data) +
                paged_kv_last_page_len->byte_offset / sizeof(IdType),
            paged_kv_rope_pos_offset != nullptr
                ? static_cast<IdType*>(paged_kv_rope_pos_offset->data) +
                      paged_kv_rope_pos_offset->byte_offset / sizeof(IdType)
                : nullptr);
        params.paged_kv = paged_kv;
        params.q_indptr =
            static_cast<IdType*>(qo_indptr->data) + qo_indptr->byte_offset / sizeof(IdType);
        params.o = static_cast<DTypeO*>(o->data) + o->byte_offset / sizeof(DTypeO);

        params.lse = static_cast<float*>(lse->data) + lse->byte_offset / sizeof(float);
        params.num_qo_heads = num_qo_heads;
        params.group_size = uint_fastdiv(num_qo_heads / paged_kv.num_heads);
        params.maybe_q_rope_offset = q_rope_offset != nullptr
                                         ? static_cast<IdType*>(q_rope_offset->data) +
                                               q_rope_offset->byte_offset / sizeof(IdType)
                                         : nullptr;
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.window_left = window_left;

        params.request_indices = nullptr;
        params.qo_tile_indices = nullptr;
        params.kv_tile_indices = nullptr;
        params.merge_indptr = nullptr;
        params.o_indptr = nullptr;
        params.kv_chunk_size_ptr = nullptr;
        params.block_valid_mask = nullptr;
        params.total_num_rows = nullptr;
        params.max_total_num_rows = 0;
        params.padded_batch_size = 0;
        params.partition_kv = false;

        ADDITIONAL_PARAMS_SETTER

        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;

        params.request_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.kv_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
        params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
        params.kv_chunk_size_ptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset);
        if (plan_info.split_kv) {
          params.merge_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
          tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
          tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
          if (plan_info.enable_cuda_graph) {
            params.block_valid_mask =
                GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
          }
        }
        params.padded_batch_size = plan_info.padded_batch_size;
        params.max_total_num_rows = plan_info.total_num_rows;
        if (plan_info.enable_cuda_graph) {
          params.total_num_rows =
              GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
        }

        hipError_t status = hipSuccess;

        DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q, CTA_TILE_Q, {
          status = flashinfer::BatchPrefillWithPagedKVCacheDispatched<
              CTA_TILE_Q, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
              /*use_fp16_qk_reduction=*/USE_FP16_QK_REDUCTION, MASK_MODE, AttentionVariant,
              PagedParams>(params, tmp_v, tmp_s, stream);
        });

        CHECK(status == hipSuccess)
            << "BatchPrefillWithPagedKVCache failed with error " << hipGetErrorString(status);
        return true;
      });
}
