#include "hip/hip_runtime.h"
/*
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/EmptyTensor.h>
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/util/Exception.h>
#include <hip/hip_runtime.h>
#include <flashinfer/exception.h>
#include <hip/hiprtc.h>
#include <torch/library.h>

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include "flashinfer/trtllm/batched_gemm/trtllmGen_bmm_export/trtllm/gen/DtypeDecl.h"
#include "flashinfer/trtllm/fused_moe/DevKernel.h"
#include "flashinfer/trtllm/fused_moe/RoutingKernel.h"
#include "flashinfer/trtllm/fused_moe/runner.h"
#include "nv_internal/tensorrt_llm/thop/thUtils.h"

namespace tensorrt_llm {
// HACK from: cpp/tensorrt_llm/kernels/quantization.h
inline int computeFP4LinearLayoutSFSize(int totalRow, int totalColumn) {
  return totalRow * totalColumn;
}
}  // namespace tensorrt_llm

namespace flashinfer {

using tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::RoutingMethodType;

at::Tensor trtllm_fp8_per_tensor_scale_moe_launcher(
    at::Tensor const& routing_logits, at::Tensor const& routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& gemm1_weights,
    at::Tensor const& output1_scales_scalar, at::Tensor const& output1_scales_gate_scalar,
    at::Tensor const& gemm2_weights, at::Tensor const& output2_scales_scalar,
    int64_t const num_experts, int64_t const top_k, int64_t const n_group, int64_t const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, double const routed_scaling_factor,
    bool const use_routing_scales_on_input, int64_t const tile_tokens_dim,
    int64_t const routing_method_type) {
  auto device = hidden_states.device();
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device.index());
  TORCH_CHECK(prop.major == 10 && prop.minor == 0,
              "This kernel requires SM 100 architecture. Current device has SM ", prop.major,
              prop.minor, " (", prop.name, ")");

  if (use_routing_scales_on_input) {
    TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::BFloat16,
                "routing_logits must be bfloat16.");
  } else {
    TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::Float,
                "routing_logits must be float.");
  }
  TORCH_CHECK(routing_logits.dim() == 2, "routing_logits must be 2D.");
  TORCH_CHECK(routing_logits.sizes()[1] == num_experts, "routing_logits has incorrect shape.");
  TORCH_CHECK(routing_bias.scalar_type() == at::ScalarType::BFloat16,
              "routing_bias must be bfloat16.");
  TORCH_CHECK(routing_bias.dim() == 1, "routing_bias must be 1D.");
  TORCH_CHECK(routing_bias.sizes()[0] == num_experts, "routing_bias has incorrect shape.");

  if (n_group <= 0 || topk_group <= 0) {
    TORCH_CHECK(top_k == 1, "Current routing kernel (no groups) only supports top_k=1.");
  } else {
    TORCH_CHECK(top_k <= 8, "Current routing kernel (with groups) only supports top_k<=8.");
    TORCH_CHECK(topk_group <= 4,
                "Current routing kernel (with groups) only supports topk_group<=4.");
    TORCH_CHECK(topk_group <= n_group, "n_group must not be smaller than topk_group.");
    TORCH_CHECK(num_experts % n_group == 0, "num_experts must be divisible by n_group");
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TORCH_CHECK(top_k < (topk_group * num_experts / n_group),
                "top_k must be less than total number of experts in selected groups");
  }
  TORCH_CHECK(num_experts % 4 == 0,
              "Routing kernel expects that num_experts must be divisible by 4");
  TORCH_CHECK(num_experts > top_k, "num_experts must be greater than top_k");

  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::MoEWorkspace workspace;

  // Convert PyTorch dtype to TensorRT-LLM dtype
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half) {
    args.mDtypeElt = batchedGemm::trtllm::gen::Dtype::Fp16;
  } else if (dtype == at::ScalarType::BFloat16) {
    args.mDtypeElt = batchedGemm::trtllm::gen::Dtype::Bfloat16;
  } else if (dtype == at::ScalarType::Float8_e4m3fn) {
    args.mDtypeElt = batchedGemm::trtllm::gen::Dtype::E4m3;
  } else {
    TORCH_CHECK(false, "Unsupported input dtype for MoE: ", dtype);
  }

  args.routing_logits = routing_logits.data_ptr();
  args.routing_bias = routing_bias.data_ptr();
  args.hidden_states = hidden_states.data_ptr();
  args.gemm1_weights = gemm1_weights.data_ptr();
  args.output1_scales_scalar = output1_scales_scalar.data_ptr<float>();
  args.output1_scales_gate_scalar = output1_scales_gate_scalar.data_ptr<float>();
  args.gemm2_weights = gemm2_weights.data_ptr();
  args.output2_scales_scalar = output2_scales_scalar.data_ptr<float>();
  args.num_tokens = hidden_states.sizes()[0];
  args.num_experts = num_experts;
  args.hidden_size = hidden_states.sizes()[1];
  args.top_k = top_k;
  args.n_group = n_group;
  args.topk_group = topk_group;
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor;
  args.intermediate_size = intermediate_size;
  args.mUseRoutingScalesOnInput = use_routing_scales_on_input;

  // allocate workspace for routing kernel
  at::Tensor num_tokens_per_expert = at::detail::empty_cuda({num_experts}, at::ScalarType::Int,
                                                            routing_logits.device(), std::nullopt);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  at::Tensor total_num_padded_tokens =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));
  at::Tensor expanded_idx_to_permuted_idx = at::detail::empty_cuda(
      {args.num_tokens * args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor permuted_idx_to_token_idx = at::detail::empty_cuda(
      {max_num_padded_tokens}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_weights =
      at::detail::empty_cuda({args.num_tokens, args.top_k}, at::ScalarType::BFloat16,
                             routing_logits.device(), std::nullopt);
  at::Tensor expert_indexes = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_count_histogram = at::detail::empty_cuda(
      {2 * 256},
      at::ScalarType::Int,  // 256 is the max number of threads per block and max number of experts
      routing_logits.device(), std::nullopt);

  // allocate workspace for activation/gemm/finalize kernels
  at::Tensor gemm1_output =
      at::detail::empty_cuda({max_num_padded_tokens, 2 * intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor gemm1_output_scale =
      at::detail::empty_cuda({2 * intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor activation_output =
      at::detail::empty_cuda({max_num_padded_tokens, intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor activation_output_scale =
      at::detail::empty_cuda({intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor gemm2_output =
      at::detail::empty_cuda({max_num_padded_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  int32_t max_num_ctas =
      tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::getMaxNumCtasInBatchDim(
          args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  at::Tensor cta_idx_xy_to_batch_idx = at::detail::empty_cuda(
      {max_num_ctas}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                             routing_logits.device(), std::nullopt);
  at::Tensor num_non_exiting_ctas =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));

  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::Runner routing_runner(tile_tokens_dim);
  auto const& stream = at::cuda::getCurrentCUDAStream(routing_logits.get_device());
  routing_runner.run(routing_logits.data_ptr(), routing_bias.data_ptr(), args.num_tokens,
                     args.num_experts, args.top_k, args.n_group, args.topk_group,
                     args.local_expert_offset, args.local_num_experts, args.routed_scaling_factor,
                     expert_indexes.data_ptr<int>(), expert_count_histogram.data_ptr<int>(),
                     total_num_padded_tokens.data_ptr<int>(),
                     expanded_idx_to_permuted_idx.data_ptr<int>(),
                     nullptr /*permuted_idx_to_expanded_idx.data_ptr<int>()*/,
                     permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
                     num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
                     cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(),
                     args.mDtypeElt, use_routing_scales_on_input, false /* use_deep_seek_fp8 */,
                     static_cast<RoutingMethodType>(routing_method_type), stream);

  // MoE kernel except routing
  TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "hidden_states must be fp8.");
  TORCH_CHECK(gemm1_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm1_weights must be fp8.");
  TORCH_CHECK(gemm1_weights.dim() == 3, "gemm1_weights must be 3D.");
  TORCH_CHECK(gemm1_weights.sizes()[1] % 2 == 0, "the second dimension of weights must be even.");
  TORCH_CHECK(intermediate_size == gemm1_weights.sizes()[1] / 2,
              "intermediate_size has incorrect shape.");
  TORCH_CHECK(gemm1_weights.sizes()[2] == hidden_states.sizes()[1],
              "the third dimension of weights must be equal to hidden_size.");
  TORCH_CHECK(intermediate_size % 128 == 0,
              "the second dimension of weights must be a multiple of 128.");

  TORCH_CHECK(output1_scales_scalar.scalar_type() == at::ScalarType::Float,
              "output1_scales_scalar must be float.");
  TORCH_CHECK(output1_scales_scalar.dim() == 1, "output1_scales_scalar must be 1D.");
  TORCH_CHECK(output1_scales_scalar.sizes()[0] == local_num_experts,
              "output1_scales_scalar has incorrect dim 0.");
  TORCH_CHECK(output1_scales_gate_scalar.scalar_type() == at::ScalarType::Float,
              "output1_scales_gate_scalar must be float.");
  TORCH_CHECK(output1_scales_gate_scalar.dim() == 1, "output1_scales_gate_scalar must be 1D.");
  TORCH_CHECK(output1_scales_gate_scalar.sizes()[0] == local_num_experts,
              "output1_scales_gate_scalar has incorrect dim 0.");

  TORCH_CHECK(gemm2_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm2_weights must be fp8.");
  TORCH_CHECK(gemm2_weights.dim() == 3, "gemm2_weights must be 3D.");
  TORCH_CHECK(gemm2_weights.sizes()[2] == intermediate_size,
              "the third dimension of weights must be equal to intermediate_size.");

  TORCH_CHECK(output2_scales_scalar.scalar_type() == at::ScalarType::Float,
              "output2_scales_scalar must be float.");
  TORCH_CHECK(output2_scales_scalar.dim() == 1, "output2_scales_scalar must be 1D.");
  TORCH_CHECK(output2_scales_scalar.sizes()[0] == local_num_experts,
              "output2_scales_scalar has incorrect dim 0.");

  // allocate output
  at::Tensor output =
      at::detail::empty_cuda({args.num_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  // setup workspace
  workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = expert_indexes.data_ptr<int>();
  workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
  workspace.expanded_idx_to_permuted_idx =
      expanded_idx_to_permuted_idx.data_ptr<int>();  // Needed by activation/finalize kernels
  workspace.permuted_idx_to_token_idx =
      permuted_idx_to_token_idx.data_ptr<int>();         // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights.data_ptr();  // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
  workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
  workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output.data_ptr();
  workspace.gemm1_output_scale = gemm1_output_scale.data_ptr<float>();
  // activation intermediate ws
  workspace.activation_output = activation_output.data_ptr();
  workspace.activation_output_scale = activation_output_scale.data_ptr<float>();
  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output.data_ptr();
  workspace.gemm2_output_scale = nullptr;
  args.output = output.data_ptr();
  args.output_scale = nullptr;

  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::Runner moe_runner(
      args.mDtypeElt, args.mUseDeepSeekFp8, tile_tokens_dim);

  auto const moeConfigIndex =
      moe_runner.getDefaultValidConfigIndex(args.top_k, args.hidden_size, args.intermediate_size,
                                            args.local_num_experts, args.num_tokens);

  auto workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);
  at::Tensor workspace_fc1 = at::detail::empty_cuda(
      {std::get<0>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  at::Tensor workspace_fc2 = at::detail::empty_cuda(
      {std::get<1>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  workspace.bmm1_workspace = workspace_fc1.data_ptr();
  workspace.bmm2_workspace = workspace_fc2.data_ptr();
  auto const& moe_stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  moe_runner.run(args, workspace, hidden_states.get_device(), moe_stream, moeConfigIndex);
  return output;
}

at::Tensor trtllm_fp8_per_tensor_scale_moe(
    at::Tensor routing_logits, at::Tensor routing_bias, at::Tensor hidden_states,
    at::Tensor gemm1_weights, at::Tensor output1_scales_scalar,
    at::Tensor output1_scales_gate_scalar, at::Tensor gemm2_weights,
    at::Tensor output2_scales_scalar, int64_t num_experts, int64_t top_k, int64_t n_group,
    int64_t topk_group, int64_t intermediate_size, int64_t local_expert_offset,
    int64_t local_num_experts, double routed_scaling_factor, bool use_routing_scales_on_input,
    int64_t tile_tokens_dim, int64_t routing_method_type) {
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half || dtype == at::ScalarType::BFloat16 ||
      dtype == at::ScalarType::Float8_e4m3fn) {
    return trtllm_fp8_per_tensor_scale_moe_launcher(
        routing_logits, routing_bias, hidden_states, gemm1_weights, output1_scales_scalar,
        output1_scales_gate_scalar, gemm2_weights, output2_scales_scalar, num_experts, top_k,
        n_group, topk_group, intermediate_size, local_expert_offset, local_num_experts,
        routed_scaling_factor, use_routing_scales_on_input, tile_tokens_dim, routing_method_type);
  } else {
    TORCH_CHECK(false, "Unsupported input type: ", dtype);
  }
}

at::Tensor trtllm_fp8_block_scale_moe_launcher(
    at::Tensor const& routing_logits, at::Tensor const& routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale,
    int64_t const num_experts, int64_t const top_k, int64_t const n_group, int64_t const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, double const routed_scaling_factor,
    int64_t const tile_tokens_dim, int64_t const routing_method_type,
    tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::Runner& moe_runner,
    int64_t moeConfigIndex) {
  auto device = hidden_states.device();
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device.index());
  TORCH_CHECK(prop.major == 10 && prop.minor == 0,
              "This kernel requires SM 100 architecture. Current device has SM ", prop.major,
              prop.minor, " (", prop.name, ")");

  TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::Float,
              "routing_logits must be float.");
  TORCH_CHECK(routing_logits.dim() == 2, "routing_logits must be 2D.");
  TORCH_CHECK(routing_logits.sizes()[0] == hidden_states.sizes()[0],
              "routing_logits and hidden_states must have the same number of tokens.");
  TORCH_CHECK(routing_logits.sizes()[1] == num_experts,
              "routing_logits dim1 must match num_experts.");
  TORCH_CHECK(routing_bias.scalar_type() == at::ScalarType::BFloat16 ||
                  routing_bias.scalar_type() == at::ScalarType::Float,
              "routing_bias must be bfloat16 or float.");
  TORCH_CHECK(routing_bias.dim() == 1, "routing_bias must be 1D.");
  TORCH_CHECK(routing_bias.sizes()[0] == num_experts, "routing_bias has incorrect shape.");

  if (n_group <= 0 || topk_group <= 0) {
    TORCH_CHECK(top_k == 1, "Current routing kernel (no groups) only supports top_k=1.");
  } else {
    TORCH_CHECK(top_k <= 8, "Current routing kernel (with groups) only supports top_k<=8.");
    TORCH_CHECK(topk_group <= 4,
                "Current routing kernel (with groups) only supports topk_group<=4.");
    TORCH_CHECK(topk_group <= n_group, "n_group must not be smaller than topk_group.");
    TORCH_CHECK(num_experts % n_group == 0, "num_experts must be divisible by n_group");
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TORCH_CHECK(top_k < (topk_group * num_experts / n_group),
                "top_k must be less than total number of experts in selected groups");
  }
  TORCH_CHECK(num_experts % 4 == 0,
              "Routing kernel expects that num_experts must be divisible by 4");
  TORCH_CHECK(num_experts > top_k, "num_experts must be greater than top_k");

  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::MoEWorkspace workspace;

  // Convert PyTorch dtype to TensorRT-LLM dtype
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half) {
    args.mDtypeElt = batchedGemm::trtllm::gen::Dtype::Fp16;
  } else if (dtype == at::ScalarType::BFloat16) {
    args.mDtypeElt = batchedGemm::trtllm::gen::Dtype::Bfloat16;
  } else if (dtype == at::ScalarType::Float8_e4m3fn) {
    args.mDtypeElt = batchedGemm::trtllm::gen::Dtype::E4m3;
  } else {
    TORCH_CHECK(false, "Unsupported input dtype for MoE: ", dtype);
  }

  args.mDtypeExpW = routing_bias.scalar_type() == at::ScalarType::BFloat16
                        ? batchedGemm::trtllm::gen::Dtype::Bfloat16
                        : batchedGemm::trtllm::gen::Dtype::Fp32;
  args.routing_logits = routing_logits.data_ptr<float>();
  args.routing_bias = routing_bias.data_ptr();
  args.hidden_states = hidden_states.data_ptr();
  args.hidden_states_scale = hidden_states_scale.data_ptr<float>();
  args.gemm1_weights = gemm1_weights.data_ptr();
  args.gemm1_weights_scale = gemm1_weights_scale.data_ptr<float>();
  args.gemm2_weights = gemm2_weights.data_ptr();
  args.gemm2_weights_scale = gemm2_weights_scale.data_ptr<float>();
  args.num_tokens = hidden_states.sizes()[0];
  args.num_experts = num_experts;
  args.hidden_size = hidden_states.sizes()[1];
  args.top_k = top_k;
  args.n_group = n_group;
  args.topk_group = topk_group;
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor;
  args.intermediate_size = intermediate_size;
  args.mUseDeepSeekFp8 = true;

  // allocate workspace for routing kernel
  at::Tensor num_tokens_per_expert = at::detail::empty_cuda({num_experts}, at::ScalarType::Int,
                                                            routing_logits.device(), std::nullopt);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  at::Tensor total_num_padded_tokens =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));
  at::Tensor expanded_idx_to_permuted_idx = at::detail::empty_cuda(
      {args.num_tokens * args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor permuted_idx_to_token_idx = at::detail::empty_cuda(
      {max_num_padded_tokens}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_weights =
      at::detail::empty_cuda({args.num_tokens, args.top_k}, routing_bias.scalar_type(),
                             routing_logits.device(), std::nullopt);
  at::Tensor expert_indexes = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  int64_t const size_of_expert_count_histogram = std::max(num_experts * 2, int64_t(256 * 2));
  at::Tensor expert_count_histogram = at::detail::empty_cuda(
      {size_of_expert_count_histogram},
      at::ScalarType::Int,  // 256 is the max number of threads per block and max number of experts
      routing_logits.device(), std::nullopt);

  // allocate workspace for activation/gemm/finalize kernels
  at::Tensor gemm1_output =
      at::detail::empty_cuda({max_num_padded_tokens, 2 * intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor gemm1_output_scale =
      at::detail::empty_cuda({2 * intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor activation_output =
      at::detail::empty_cuda({max_num_padded_tokens, intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor activation_output_scale =
      at::detail::empty_cuda({intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor gemm2_output =
      at::detail::empty_cuda({max_num_padded_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  int32_t max_num_ctas =
      tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::getMaxNumCtasInBatchDim(
          args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  at::Tensor cta_idx_xy_to_batch_idx = at::detail::empty_cuda(
      {max_num_ctas}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                             routing_logits.device(), std::nullopt);
  at::Tensor num_non_exiting_ctas =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));

  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::Runner routing_runner(tile_tokens_dim);
  auto const& stream = at::cuda::getCurrentCUDAStream(routing_logits.get_device());
  routing_runner.run(
      routing_logits.data_ptr<float>(), routing_bias.data_ptr(), args.num_tokens, args.num_experts,
      args.top_k, args.n_group, args.topk_group, args.local_expert_offset, args.local_num_experts,
      args.routed_scaling_factor, expert_indexes.data_ptr<int>(),
      expert_count_histogram.data_ptr<int>(), total_num_padded_tokens.data_ptr<int>(),
      expanded_idx_to_permuted_idx.data_ptr<int>(),
      nullptr /*permuted_idx_to_expanded_idx.data_ptr<int>()*/,
      permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
      num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
      cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(), args.mDtypeElt,
      false, true, static_cast<RoutingMethodType>(routing_method_type), stream);

  // MoE kernel except routing
  TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "hidden_states must be fp8.");
  TORCH_CHECK(hidden_states_scale.scalar_type() == at::ScalarType::Float,
              "hidden_states_scale must be float.");
  TORCH_CHECK(hidden_states_scale.dim() == 2, "hidden_states_scale must be 2D.");
  TORCH_CHECK(hidden_states_scale.sizes()[0] == hidden_states.sizes()[1] / 128,
              "hidden_states_scale dim0 must match hidden_states dim1 / 128.");
  TORCH_CHECK(hidden_states_scale.sizes()[1] == args.num_tokens,
              "hidden_states_scale dim1 must match num_tokens.");
  TORCH_CHECK(gemm1_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm1_weights must be fp8.");
  TORCH_CHECK(gemm1_weights.dim() == 3, "gemm1_weights must be 3D.");
  TORCH_CHECK(gemm1_weights.sizes()[1] % 2 == 0, "the second dimension of weights must be even.");
  TORCH_CHECK(intermediate_size == gemm1_weights.sizes()[1] / 2,
              "intermediate_size has incorrect shape.");
  TORCH_CHECK(gemm1_weights.sizes()[2] == hidden_states.sizes()[1],
              "the third dimension of weights must be equal to hidden_size.");
  TORCH_CHECK(gemm1_weights_scale.scalar_type() == at::ScalarType::Float,
              "gemm1_weights_scale must be float.");
  TORCH_CHECK(gemm1_weights_scale.dim() == 3, "gemm1_weights_scale must be 3D.");

  TORCH_CHECK(gemm1_weights_scale.sizes()[0] == local_num_experts,
              "gemm1_weights_scale has incorrect shape.");
  TORCH_CHECK(intermediate_size % 128 == 0,
              "the second dimension of weights must be a multiple of 128.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[1] == 2 * intermediate_size / 128,
              "gemm1_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[2] == args.hidden_size / 128,
              "gemm1_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm2_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm2_weights must be fp8.");
  TORCH_CHECK(gemm2_weights.dim() == 3, "gemm2_weights must be 3D.");
  TORCH_CHECK(gemm2_weights.sizes()[2] == intermediate_size,
              "the third dimension of weights must be equal to intermediate_size.");
  TORCH_CHECK(gemm2_weights_scale.scalar_type() == at::ScalarType::Float,
              "gemm2_weights_scale must be float.");
  TORCH_CHECK(gemm2_weights_scale.dim() == 3, "gemm2_weights_scale must be 3D.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[0] == local_num_experts,
              "gemm2_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[1] == args.hidden_size / 128,
              "gemm2_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[2] == intermediate_size / 128,
              "gemm2_weights_scale has incorrect shape.");

  // allocate output
  at::Tensor output =
      at::detail::empty_cuda({args.num_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  // setup workspace
  workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = expert_indexes.data_ptr<int>();
  workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
  workspace.expanded_idx_to_permuted_idx =
      expanded_idx_to_permuted_idx.data_ptr<int>();  // Needed by activation/finalize kernels
  workspace.permuted_idx_to_token_idx =
      permuted_idx_to_token_idx.data_ptr<int>();         // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights.data_ptr();  // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
  workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
  workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output.data_ptr();
  workspace.gemm1_output_scale = gemm1_output_scale.data_ptr<float>();
  // activation intermediate ws
  workspace.activation_output = activation_output.data_ptr();
  workspace.activation_output_scale = activation_output_scale.data_ptr<float>();
  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output.data_ptr();
  workspace.gemm2_output_scale = nullptr;
  args.output = output.data_ptr();
  args.output_scale = nullptr;

  auto workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);
  at::Tensor workspace_fc1 = at::detail::empty_cuda(
      {std::get<0>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  at::Tensor workspace_fc2 = at::detail::empty_cuda(
      {std::get<1>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  workspace.bmm1_workspace = workspace_fc1.data_ptr();
  workspace.bmm2_workspace = workspace_fc2.data_ptr();

  auto const& moe_stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  moe_runner.run(args, workspace, hidden_states.get_device(), moe_stream, moeConfigIndex);
  return output;
}

at::Tensor trtllm_fp8_block_scale_moe(
    at::Tensor const& routing_logits, at::Tensor const& routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale, int64_t num_experts,
    int64_t top_k, int64_t n_group, int64_t topk_group, int64_t intermediate_size,
    int64_t local_expert_offset, int64_t local_num_experts, double routed_scaling_factor,
    int64_t tile_tokens_dim, int64_t routing_method_type) {
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half || dtype == at::ScalarType::BFloat16 ||
      dtype == at::ScalarType::Float8_e4m3fn) {
    using RunnerType = tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::Runner;

    batchedGemm::trtllm::gen::Dtype mDtypeElt{
        batchedGemm::trtllm::gen::Dtype::E4m3};  // FP8 runner so hard-coded
    bool mUseDeepSeekFp8{true};                  // Always true for BlockScaleMoe

    // Properly initialize the runner using make_unique like in the original code
    auto mRunner = std::make_unique<RunnerType>(mDtypeElt, mUseDeepSeekFp8, tile_tokens_dim);

    // Always use fallback config (equivalent to moeConfigIndex == -1 case from original code)
    auto const num_tokens = hidden_states.sizes()[0];
    auto const hidden_size = hidden_states.sizes()[1];

    int64_t moeConfigIndex = mRunner->getDefaultValidConfigIndex(
        top_k, hidden_size, intermediate_size, local_num_experts, num_tokens);

    return trtllm_fp8_block_scale_moe_launcher(
        routing_logits, routing_bias, hidden_states, hidden_states_scale, gemm1_weights,
        gemm1_weights_scale, gemm2_weights, gemm2_weights_scale, num_experts, top_k, n_group,
        topk_group, intermediate_size, local_expert_offset, local_num_experts,
        routed_scaling_factor, tile_tokens_dim, routing_method_type, *mRunner, moeConfigIndex);
  } else {
    TORCH_CHECK(false, "Unsupported input type: ", dtype);
  }
}

std::vector<at::Tensor> trtllm_fp4_block_scale_moe_launcher(
    at::Tensor const& routing_logits, at::optional<at::Tensor> const& routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale,
    at::Tensor const& output1_scales_scalar, at::Tensor const& output1_scales_gate_scalar,
    at::Tensor const& output2_scales_scalar, int64_t const num_experts, int64_t const top_k,
    std::optional<int64_t> const n_group, std::optional<int64_t> const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, std::optional<double> const routed_scaling_factor,
    int64_t const tile_tokens_dim, int64_t const routing_method_type, bool const do_finalize,
    tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::Runner& moe_runner,
    int64_t const moeConfigIndex) {
  auto device = hidden_states.device();
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device.index());
  TORCH_CHECK(prop.major == 10 && prop.minor == 0,
              "This kernel requires SM 100 architecture. Current device has SM ", prop.major,
              prop.minor, " (", prop.name, ")");

  TORCH_CHECK(tile_tokens_dim == 8 || tile_tokens_dim == 16 || tile_tokens_dim == 32 ||
                  tile_tokens_dim == 64,
              "tile_tokens_dim must be 8, 16, 32, 64");
  if (static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::DeepSeekV3) {
    TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::Float,
                "routing_logits must be float");
  } else {
    TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::BFloat16,
                "routing_logits must be bfloat16");
  }
  TORCH_CHECK(routing_logits.dim() == 2, "routing_logits must be 2D.");
  TORCH_CHECK(routing_logits.sizes()[0] == hidden_states.sizes()[0],
              "routing_logits and hidden_states must have the same number of tokens.");
  TORCH_CHECK(routing_logits.sizes()[1] == num_experts, "routing_logits has incorrect shape.");
  if (routing_bias.has_value()) {
    TORCH_CHECK(routing_bias.value().scalar_type() == at::ScalarType::BFloat16,
                "routing_bias must be bfloat16.");
    TORCH_CHECK(routing_bias.value().dim() == 1, "routing_bias must be 1D.");
    TORCH_CHECK(routing_bias.value().sizes()[0] == num_experts,
                "routing_bias has incorrect shape.");
  }

  if (n_group.has_value()) {
    TORCH_CHECK(
        static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::DeepSeekV3,
        "Routing kernel with groups implies DeepSeekV3 routing method.");
    TORCH_CHECK(topk_group.has_value(), "if n_group is given, topk_group must be given");
    TORCH_CHECK(num_experts % n_group.value() == 0, "num_experts must be divisible by n_group");
    TORCH_CHECK(top_k <= 8, "Current routing kernel (with groups) only supports top_k<=8.");
    TORCH_CHECK(topk_group.value() <= 4,
                "Current routing kernel only (with groups) supports topk_group<=4.");
    TORCH_CHECK(topk_group.value() <= n_group.value(),
                "n_group must not be smaller than topk_group.");
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TORCH_CHECK(top_k < (topk_group.value() * num_experts / n_group.value()),
                "top_k must be less than total number of experts in selected groups");
  } else if (static_cast<RoutingMethodType>(routing_method_type) ==
                 RoutingMethodType::Renormalize ||
             static_cast<RoutingMethodType>(routing_method_type) ==
                 RoutingMethodType::RenormalizeNaive) {
    TORCH_CHECK(top_k == 8,
                "Current routing kernel (no groups, renormalize) only supports top_k=8.");
  } else if (static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::Llama4) {
    TORCH_CHECK(top_k == 1, "Current routing kernel (no groups, Llama4) only supports top_k=1.");
  }

  TORCH_CHECK(num_experts % 4 == 0,
              "Routing kernel expects that num_experts must be divisible by 4");
  TORCH_CHECK(num_experts > top_k, "num_experts must be greater than top_k");
  TORCH_CHECK(num_experts <= 256, "num_experts must be less than or equal to 256");

  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::MoEWorkspace workspace;

  // setup args
  // note: the assumption is that output data type is always Bfloat16 (the default)
  auto const routing_bias_dtype =
      routing_bias.has_value() ? routing_bias.value().scalar_type() : at::ScalarType::BFloat16;
  args.mDtypeElt = batchedGemm::trtllm::gen::Dtype::E2m1;
  args.mDtypeExpW = routing_bias_dtype == at::ScalarType::Float
                        ? batchedGemm::trtllm::gen::Dtype::Fp32
                        : batchedGemm::trtllm::gen::Dtype::Bfloat16;
  args.routing_logits = routing_logits.data_ptr();
  args.routing_bias = routing_bias.has_value() ? routing_bias.value().data_ptr() : nullptr;
  args.hidden_states = hidden_states.data_ptr();
  args.hidden_states_scale = hidden_states_scale.data_ptr();
  args.gemm1_weights = gemm1_weights.data_ptr();
  args.gemm1_weights_scale = gemm1_weights_scale.data_ptr();
  args.gemm2_weights = gemm2_weights.data_ptr();
  args.gemm2_weights_scale = gemm2_weights_scale.data_ptr();
  args.num_tokens = hidden_states.sizes()[0];
  args.num_experts = num_experts;
  // * 2 to compensate for the fact that sizeof(hidden_states.dtype) is 1 because we pack 2 e2m1
  // into 1 byte.
  args.hidden_size = hidden_states.sizes()[1] * 2;
  args.top_k = top_k;
  args.n_group = n_group.value_or(1);
  args.topk_group = topk_group.value_or(top_k);
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor.value_or(1.0);
  args.intermediate_size = intermediate_size;

  // allocate workspace for routing kernel
  at::Tensor num_tokens_per_expert = at::detail::empty_cuda({num_experts}, at::ScalarType::Int,
                                                            routing_logits.device(), std::nullopt);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  at::Tensor total_num_padded_tokens =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));
  at::Tensor expanded_idx_to_permuted_idx = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);

  at::Tensor permuted_idx_to_token_idx = at::detail::empty_cuda(
      {max_num_padded_tokens}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_weights = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, routing_bias_dtype, routing_logits.device(), std::nullopt);
  at::Tensor expert_indexes = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  int64_t const size_of_expert_count_histogram = std::max(num_experts * 2, int64_t(256 * 2));
  at::Tensor expert_count_histogram = at::detail::empty_cuda(
      {size_of_expert_count_histogram},
      at::ScalarType::Int,  // 256 is the max number of threads per block and max number of experts
      routing_logits.device(), std::nullopt);

  // allocate workspace for activation/gemm/finalize kernels
  at::Tensor gemm1_output =
      at::detail::empty_cuda({max_num_padded_tokens, intermediate_size / 2},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);

  at::Tensor gemm1_output_scale =
      at::detail::empty_cuda({max_num_padded_tokens, intermediate_size / 16},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);

  at::Tensor gemm2_output =
      at::detail::empty_cuda({max_num_padded_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  int32_t max_num_ctas =
      tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::getMaxNumCtasInBatchDim(
          args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  at::Tensor cta_idx_xy_to_batch_idx = at::detail::empty_cuda(
      {max_num_ctas}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                             routing_logits.device(), std::nullopt);
  at::Tensor num_non_exiting_ctas =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));

  // FIXME: check shape
  auto const hidden_states_scale_linear_size =
      tensorrt_llm::computeFP4LinearLayoutSFSize(args.num_tokens, args.hidden_size / 16);
  at::Tensor hidden_states_scale_linear =
      at::detail::empty_cuda(hidden_states_scale_linear_size, at::ScalarType::Float8_e4m3fn,
                             hidden_states.device(), std::nullopt);

  //
  // TopK routing
  //

  tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::Routing::Runner routing_runner(tile_tokens_dim);
  auto const& stream = at::cuda::getCurrentCUDAStream(routing_logits.get_device());
  routing_runner.run(
      args.routing_logits, args.routing_bias, args.num_tokens, args.num_experts, args.top_k,
      args.n_group, args.topk_group, args.local_expert_offset, args.local_num_experts,
      args.routed_scaling_factor, expert_indexes.data_ptr<int>(),
      expert_count_histogram.data_ptr<int>(), total_num_padded_tokens.data_ptr<int>(),
      expanded_idx_to_permuted_idx.data_ptr<int>(),
      nullptr, /*permuted_idx_to_expanded_idx.data_ptr<int>(),*/
      permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
      num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
      cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(), args.mDtypeElt,
      false /* use_routing_scales_on_input */, false /* use_deep_seek_fp8 */,
      static_cast<RoutingMethodType>(routing_method_type), stream);

  //
  // FC13 (gemm1) + FC2 (gemm2)
  //

  TORCH_CHECK(hidden_states.scalar_type() == torch_ext::FLOAT4_E2M1X2,
              "hidden_states must be byte.");
  TORCH_CHECK(hidden_states_scale.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "hidden_states_scale must be fp8.");

  TORCH_CHECK(hidden_states_scale.dim() == 1, "hidden_states_scale must be 1D.");
  TORCH_CHECK(hidden_states_scale.sizes()[0] == tensorrt_llm::computeFP4LinearLayoutSFSize(
                                                    args.num_tokens, args.hidden_size / 16),
              "hidden_states_scale has incorrect size");

  TORCH_CHECK(gemm1_weights.scalar_type() == torch_ext::FLOAT4_E2M1X2,
              "gemm1_weights must be byte.");

  TORCH_CHECK(gemm1_weights.dim() == 3, "gemm1_weights must be 3D.");
  TORCH_CHECK(gemm1_weights.sizes()[1] % 2 == 0, "the second dimension of weights must be even.");
  TORCH_CHECK(intermediate_size == gemm1_weights.sizes()[1] / 2,
              "intermediate_size has incorrect dim 1.");
  // This check passes even though the actual shape of the weights[2] and hidden_states[1] is
  // 2 times larger due to the fact that 2 e2m1 are packed into 1 byte.
  TORCH_CHECK(gemm1_weights.sizes()[2] == hidden_states.sizes()[1],
              "the third dimension of weights must be equal to hidden_size.");

  TORCH_CHECK(gemm1_weights_scale.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm1_weights_scale must be fp8.");

  TORCH_CHECK(gemm1_weights_scale.dim() == 3, "gemm1_weights_scale must be 3D.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[0] == local_num_experts,
              "gemm1_weights_scale has incorrect dim 0.");
  TORCH_CHECK(intermediate_size % 16 == 0,
              "the second dimension of weights must be a multiple of 16.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[1] == 2 * intermediate_size,
              "gemm1_weights_scale has incorrect dim 1.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[2] == args.hidden_size / 16,
              "gemm1_weights_scale has incorrect dim 2.");

  TORCH_CHECK(gemm2_weights.scalar_type() == torch_ext::FLOAT4_E2M1X2,
              "gemm2_weights must be byte.");

  TORCH_CHECK(gemm2_weights.dim() == 3, "gemm2_weights must be 3D.");
  // / 2 to compensate for the fact that we pack 2 e2m1 into 1 byte.
  TORCH_CHECK(gemm2_weights.sizes()[2] == intermediate_size / 2,
              "the third dimension of weights must be equal to intermediate_size.");

  TORCH_CHECK(gemm2_weights_scale.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm2_weights_scale must be fp8.");

  TORCH_CHECK(gemm2_weights_scale.dim() == 3, "gemm2_weights_scale must be 3D.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[0] == local_num_experts,
              "gemm2_weights_scale has incorrect dim 0.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[1] == args.hidden_size,
              "gemm2_weights_scale has incorrect dim 1.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[2] == intermediate_size / 16,
              "gemm2_weights_scale has incorrect dim 2.");

  TORCH_CHECK(output1_scales_scalar.scalar_type() == at::ScalarType::Float,
              "output1_scales_scalar must be float.");
  TORCH_CHECK(output1_scales_scalar.dim() == 1, "output1_scales_scalar must be 1D.");
  TORCH_CHECK(output1_scales_scalar.sizes()[0] == local_num_experts,
              "output1_scales_scalar has incorrect dim 0.");

  TORCH_CHECK(output1_scales_gate_scalar.scalar_type() == at::ScalarType::Float,
              "output1_scales_gate_scalar must be float.");
  TORCH_CHECK(output1_scales_gate_scalar.dim() == 1, "output1_scales_gate_scalar must be 1D.");
  TORCH_CHECK(output1_scales_gate_scalar.sizes()[0] == local_num_experts,
              "output1_scales_gate_scalar has incorrect dim 0.");

  TORCH_CHECK(output2_scales_scalar.scalar_type() == at::ScalarType::Float,
              "output2_scales_scalar must be float.");
  TORCH_CHECK(output2_scales_scalar.dim() == 1, "output2_scales_scalar must be 1D.");
  TORCH_CHECK(output2_scales_scalar.sizes()[0] == local_num_experts,
              "output2_scales_scalar has incorrect dim 0.");

  // allocate output
  at::Tensor output =
      at::detail::empty_cuda({args.num_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  // setup workspace
  workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = expert_indexes.data_ptr<int>();
  workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
  workspace.expanded_idx_to_permuted_idx =
      expanded_idx_to_permuted_idx.data_ptr<int>();  // Needed by permute/finalize kernels
  workspace.permuted_idx_to_token_idx =
      permuted_idx_to_token_idx.data_ptr<int>();         // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights.data_ptr();  // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
  workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
  workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();

  workspace.hidden_states_scale_linear = hidden_states_scale_linear.data_ptr();

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output.data_ptr();
  workspace.gemm1_output_scale = reinterpret_cast<float*>(gemm1_output_scale.data_ptr());

  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output.data_ptr();
  workspace.gemm2_output_scale = nullptr;
  args.output = output.data_ptr();
  args.output_scale = nullptr;
  args.output1_scales_scalar = output1_scales_scalar.data_ptr<float>();
  args.output1_scales_gate_scalar = output1_scales_gate_scalar.data_ptr<float>();
  args.output2_scales_scalar = output2_scales_scalar.data_ptr<float>();
  args.do_finalize = do_finalize;

  auto const workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);

  at::Tensor workspace_fc1 = at::detail::empty_cuda(
      {std::get<0>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  at::Tensor workspace_fc2 = at::detail::empty_cuda(
      {std::get<1>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  workspace.bmm1_workspace = workspace_fc1.data_ptr();
  workspace.bmm2_workspace = workspace_fc2.data_ptr();
  auto const& moe_stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  moe_runner.run(args, workspace, hidden_states.get_device(), moe_stream, moeConfigIndex);

  if (!do_finalize) {
    return {gemm2_output, expert_weights, expanded_idx_to_permuted_idx};
  }

  return {output};
}

std::vector<at::Tensor> trtllm_fp4_block_scale_moe(
    at::Tensor const& routing_logits, at::optional<at::Tensor> const& routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale,
    at::Tensor const& output1_scales_scalar, at::Tensor const& output1_scales_gate_scalar,
    at::Tensor const& output2_scales_scalar, int64_t num_experts, int64_t top_k,
    std::optional<int64_t> n_group, std::optional<int64_t> topk_group, int64_t intermediate_size,
    int64_t local_expert_offset, int64_t local_num_experts,
    std::optional<double> routed_scaling_factor, int64_t tile_tokens_dim,
    int64_t routing_method_type, bool do_finalize) {
  using RunnerType = tensorrt_llm::kernels::trtllmGenFp8BlockScaleMoe::MoE::Runner;

  batchedGemm::trtllm::gen::Dtype mDtypeElt{batchedGemm::trtllm::gen::Dtype::E2m1};  // FP4 runner
  bool mUseDeepSeekFp8{false};  // FP4 doesn't use DeepSeek FP8

  // Properly initialize the runner using make_unique like in the original code
  auto mRunner = std::make_unique<RunnerType>(mDtypeElt, mUseDeepSeekFp8, tile_tokens_dim);

  auto const num_tokens = hidden_states.sizes()[0];

  // 2x FP4 per byte element
  auto const hidden_size = 2 * hidden_states.sizes()[1];

  auto const moeConfigIndex = mRunner->getDefaultValidConfigIndex(
      top_k, hidden_size, intermediate_size, local_num_experts, num_tokens);

  return trtllm_fp4_block_scale_moe_launcher(
      routing_logits, routing_bias, hidden_states, hidden_states_scale, gemm1_weights,
      gemm1_weights_scale, gemm2_weights, gemm2_weights_scale, output1_scales_scalar,
      output1_scales_gate_scalar, output2_scales_scalar, num_experts, top_k, n_group, topk_group,
      intermediate_size, local_expert_offset, local_num_experts, routed_scaling_factor,
      tile_tokens_dim, routing_method_type, do_finalize, *mRunner, moeConfigIndex);
}

namespace trtllm_cubin_loader {
#include <flashinfer/cubin_loader.h>
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_fp8_per_tensor_scale_moe", trtllm_fp8_per_tensor_scale_moe);
  m.def("trtllm_fp8_block_scale_moe", trtllm_fp8_block_scale_moe);
  m.def("trtllm_fp4_block_scale_moe", trtllm_fp4_block_scale_moe);
}

}  // namespace flashinfer
