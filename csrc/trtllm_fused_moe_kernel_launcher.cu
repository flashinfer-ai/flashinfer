#include "hip/hip_runtime.h"
/*
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/EmptyTensor.h>
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/util/Exception.h>
#include <hip/hip_runtime.h>
#include <flashinfer/exception.h>
#include <hip/hiprtc.h>
#include <torch/library.h>

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include "flashinfer/trtllm/batched_gemm/trtllmGen_bmm_export/GemmGatedActOptions.h"
#include "flashinfer/trtllm/batched_gemm/trtllmGen_bmm_export/trtllm/gen/DtypeDecl.h"
#include "flashinfer/trtllm/fused_moe/DevKernel.h"
#include "flashinfer/trtllm/fused_moe/RoutingKernel.h"
#include "flashinfer/trtllm/fused_moe/runner.h"
#include "nv_internal/tensorrt_llm/kernels/quantization.h"
#include "nv_internal/tensorrt_llm/thop/thUtils.h"

namespace flashinfer {

namespace btg = batchedGemm::trtllm::gen;
using tensorrt_llm::kernels::trtllmgen_moe::Routing::RoutingMethodType;

at::Tensor trtllm_fp8_per_tensor_scale_moe_launcher(
    at::Tensor const& routing_logits, std::optional<at::Tensor> routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& gemm1_weights,
    at::Tensor const& output1_scales_scalar, at::Tensor const& output1_scales_gate_scalar,
    at::Tensor const& gemm2_weights, at::Tensor const& output2_scales_scalar,
    int64_t const num_experts, int64_t const top_k, int64_t const n_group, int64_t const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, double const routed_scaling_factor,
    bool const use_routing_scales_on_input, int64_t const tile_tokens_dim,
    int64_t const routing_method_type) {
  auto device = hidden_states.device();

  static const std::tuple<int, int> device_props = [&device] {
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device.index());
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device.index());
    return std::make_tuple(major, minor);
  }();

  TORCH_CHECK(std::get<0>(device_props) == 10 && std::get<1>(device_props) == 0,
              "This kernel requires SM 100 architecture. Current device has SM ",
              std::get<0>(device_props), std::get<1>(device_props));

  if (use_routing_scales_on_input) {
    TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::BFloat16,
                "routing_logits must be bfloat16.");
  } else {
    TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::Float,
                "routing_logits must be float.");
  }
  TORCH_CHECK(routing_logits.dim() == 2, "routing_logits must be 2D.");
  TORCH_CHECK(routing_logits.sizes()[1] == num_experts, "routing_logits has incorrect shape.");
  if (routing_bias.has_value()) {
    TORCH_CHECK(routing_bias.value().scalar_type() == at::ScalarType::BFloat16,
                "routing_bias must be bfloat16.");
    TORCH_CHECK(routing_bias.value().dim() == 1, "routing_bias must be 1D.");
    TORCH_CHECK(routing_bias.value().sizes()[0] == num_experts,
                "routing_bias has incorrect shape.");
  }

  if (n_group <= 0 || topk_group <= 0) {
    TORCH_CHECK(top_k == 1, "Current routing kernel (no groups) only supports top_k=1.");
  } else {
    TORCH_CHECK(top_k <= 8, "Current routing kernel (with groups) only supports top_k<=8.");
    TORCH_CHECK(topk_group <= 4,
                "Current routing kernel (with groups) only supports topk_group<=4.");
    TORCH_CHECK(topk_group <= n_group, "n_group must not be smaller than topk_group.");
    TORCH_CHECK(num_experts % n_group == 0, "num_experts must be divisible by n_group");
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TORCH_CHECK(top_k < (topk_group * num_experts / n_group),
                "top_k must be less than total number of experts in selected groups");
  }
  TORCH_CHECK(num_experts % 4 == 0,
              "Routing kernel expects that num_experts must be divisible by 4");
  TORCH_CHECK(num_experts > top_k, "num_experts must be greater than top_k");

  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  // Convert PyTorch dtype to TensorRT-LLM dtype
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half) {
    args.mDtypeElt = btg::Dtype::Fp16;
  } else if (dtype == at::ScalarType::BFloat16) {
    args.mDtypeElt = btg::Dtype::Bfloat16;
  } else if (dtype == at::ScalarType::Float8_e4m3fn) {
    args.mDtypeElt = btg::Dtype::E4m3;
  } else {
    TORCH_CHECK(false, "Unsupported input dtype for MoE: ", dtype);
  }

  args.routing_logits = routing_logits.data_ptr();
  auto const routing_bias_dtype =
      routing_bias.has_value() ? routing_bias.value().scalar_type() : at::ScalarType::BFloat16;
  args.routing_bias = routing_bias.has_value() ? routing_bias.value().data_ptr() : nullptr;
  args.hidden_states = hidden_states.data_ptr();
  args.gemm1_weights = gemm1_weights.data_ptr();
  args.output1_scales_scalar = output1_scales_scalar.data_ptr<float>();
  args.output1_scales_gate_scalar = output1_scales_gate_scalar.data_ptr<float>();
  args.gemm2_weights = gemm2_weights.data_ptr();
  args.output2_scales_scalar = output2_scales_scalar.data_ptr<float>();
  args.num_tokens = hidden_states.sizes()[0];
  args.num_experts = num_experts;
  args.hidden_size = hidden_states.sizes()[1];
  args.hidden_size_output = args.hidden_size;
  args.top_k = top_k;
  args.n_group = n_group;
  args.topk_group = topk_group;
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor;
  args.intermediate_size = intermediate_size;
  args.mUseRoutingScalesOnInput = use_routing_scales_on_input;

  // allocate workspace for routing kernel
  at::Tensor num_tokens_per_expert = at::detail::empty_cuda({num_experts}, at::ScalarType::Int,
                                                            routing_logits.device(), std::nullopt);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  at::Tensor total_num_padded_tokens =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));
  at::Tensor expanded_idx_to_permuted_idx = at::detail::empty_cuda(
      {args.num_tokens * args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor permuted_idx_to_token_idx = at::detail::empty_cuda(
      {max_num_padded_tokens}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_weights = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, routing_bias_dtype, routing_logits.device(), std::nullopt);
  at::Tensor expert_indexes = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_count_histogram = at::detail::empty_cuda(
      {2 * 256},
      at::ScalarType::Int,  // 256 is the max number of threads per block and max number of experts
      routing_logits.device(), std::nullopt);

  // allocate workspace for activation/gemm/finalize kernels
  at::Tensor gemm1_output =
      at::detail::empty_cuda({max_num_padded_tokens, 2 * intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor gemm1_output_scale =
      at::detail::empty_cuda({2 * intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor activation_output =
      at::detail::empty_cuda({max_num_padded_tokens, intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor activation_output_scale =
      at::detail::empty_cuda({intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor gemm2_output =
      at::detail::empty_cuda({max_num_padded_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
      args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  at::Tensor cta_idx_xy_to_batch_idx = at::detail::empty_cuda(
      {max_num_ctas}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                             routing_logits.device(), std::nullopt);
  at::Tensor num_non_exiting_ctas =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));

  tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
  auto const& stream = at::cuda::getCurrentCUDAStream(routing_logits.get_device());
  routing_runner.run(routing_logits.data_ptr(), args.routing_bias, args.num_tokens,
                     args.num_experts, args.top_k, args.n_group, args.topk_group,
                     args.local_expert_offset, args.local_num_experts, args.routed_scaling_factor,
                     expert_indexes.data_ptr<int>(), expert_count_histogram.data_ptr<int>(),
                     total_num_padded_tokens.data_ptr<int>(),
                     expanded_idx_to_permuted_idx.data_ptr<int>(),
                     nullptr /*permuted_idx_to_expanded_idx.data_ptr<int>()*/,
                     permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
                     num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
                     cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(),
                     args.mDtypeElt, use_routing_scales_on_input, false /* use_deep_seek_fp8 */,
                     static_cast<RoutingMethodType>(routing_method_type), stream);

  // MoE kernel except routing
  TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "hidden_states must be fp8.");
  TORCH_CHECK(gemm1_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm1_weights must be fp8.");
  TORCH_CHECK(gemm1_weights.dim() == 3, "gemm1_weights must be 3D.");
  TORCH_CHECK(gemm1_weights.sizes()[1] % 2 == 0, "the second dimension of weights must be even.");
  TORCH_CHECK(intermediate_size == gemm1_weights.sizes()[1] / 2,
              "intermediate_size has incorrect shape.");
  TORCH_CHECK(gemm1_weights.sizes()[2] == hidden_states.sizes()[1],
              "the third dimension of weights must be equal to hidden_size.");
  TORCH_CHECK(intermediate_size % 128 == 0,
              "the second dimension of weights must be a multiple of 128.");

  TORCH_CHECK(output1_scales_scalar.scalar_type() == at::ScalarType::Float,
              "output1_scales_scalar must be float.");
  TORCH_CHECK(output1_scales_scalar.dim() == 1, "output1_scales_scalar must be 1D.");
  TORCH_CHECK(output1_scales_scalar.sizes()[0] == local_num_experts,
              "output1_scales_scalar has incorrect dim 0.");
  TORCH_CHECK(output1_scales_gate_scalar.scalar_type() == at::ScalarType::Float,
              "output1_scales_gate_scalar must be float.");
  TORCH_CHECK(output1_scales_gate_scalar.dim() == 1, "output1_scales_gate_scalar must be 1D.");
  TORCH_CHECK(output1_scales_gate_scalar.sizes()[0] == local_num_experts,
              "output1_scales_gate_scalar has incorrect dim 0.");

  TORCH_CHECK(gemm2_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm2_weights must be fp8.");
  TORCH_CHECK(gemm2_weights.dim() == 3, "gemm2_weights must be 3D.");
  TORCH_CHECK(gemm2_weights.sizes()[2] == intermediate_size,
              "the third dimension of weights must be equal to intermediate_size.");

  TORCH_CHECK(output2_scales_scalar.scalar_type() == at::ScalarType::Float,
              "output2_scales_scalar must be float.");
  TORCH_CHECK(output2_scales_scalar.dim() == 1, "output2_scales_scalar must be 1D.");
  TORCH_CHECK(output2_scales_scalar.sizes()[0] == local_num_experts,
              "output2_scales_scalar has incorrect dim 0.");

  // allocate output
  at::Tensor output =
      at::detail::empty_cuda({args.num_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  // setup workspace
  workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = expert_indexes.data_ptr<int>();
  workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
  workspace.expanded_idx_to_permuted_idx =
      expanded_idx_to_permuted_idx.data_ptr<int>();  // Needed by activation/finalize kernels
  workspace.permuted_idx_to_token_idx =
      permuted_idx_to_token_idx.data_ptr<int>();         // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights.data_ptr();  // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
  workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
  workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output.data_ptr();
  workspace.gemm1_output_scale = gemm1_output_scale.data_ptr<float>();
  // activation intermediate ws
  workspace.activation_output = activation_output.data_ptr();
  workspace.activation_output_scale = activation_output_scale.data_ptr<float>();
  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output.data_ptr();
  workspace.gemm2_output_scale = nullptr;
  args.output = output.data_ptr();
  args.output_scale = nullptr;

  tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner moe_runner(
      args.mDtypeElt, args.mUseDeepSeekFp8, tile_tokens_dim, /*useShuffledMatrixA*/ true);

  auto const moeConfigIndex =
      moe_runner.getDefaultValidConfigIndex(args.top_k, args.hidden_size, args.intermediate_size,
                                            args.local_num_experts, args.num_tokens);

  auto workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);
  at::Tensor workspace_fc1 = at::detail::empty_cuda(
      {std::get<0>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  at::Tensor workspace_fc2 = at::detail::empty_cuda(
      {std::get<1>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  workspace.bmm1_workspace = workspace_fc1.data_ptr();
  workspace.bmm2_workspace = workspace_fc2.data_ptr();
  auto const& moe_stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  moe_runner.run(args, workspace, hidden_states.get_device(), moe_stream, moeConfigIndex);
  return output;
}

at::Tensor trtllm_fp8_per_tensor_scale_moe(
    at::Tensor routing_logits, std::optional<at::Tensor> routing_bias, at::Tensor hidden_states,
    at::Tensor gemm1_weights, at::Tensor output1_scales_scalar,
    at::Tensor output1_scales_gate_scalar, at::Tensor gemm2_weights,
    at::Tensor output2_scales_scalar, int64_t num_experts, int64_t top_k, int64_t n_group,
    int64_t topk_group, int64_t intermediate_size, int64_t local_expert_offset,
    int64_t local_num_experts, double routed_scaling_factor, bool use_routing_scales_on_input,
    int64_t tile_tokens_dim, int64_t routing_method_type) {
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half || dtype == at::ScalarType::BFloat16 ||
      dtype == at::ScalarType::Float8_e4m3fn) {
    return trtllm_fp8_per_tensor_scale_moe_launcher(
        routing_logits, routing_bias, hidden_states, gemm1_weights, output1_scales_scalar,
        output1_scales_gate_scalar, gemm2_weights, output2_scales_scalar, num_experts, top_k,
        n_group, topk_group, intermediate_size, local_expert_offset, local_num_experts,
        routed_scaling_factor, use_routing_scales_on_input, tile_tokens_dim, routing_method_type);
  } else {
    TORCH_CHECK(false, "Unsupported input type: ", dtype);
  }
}

at::Tensor trtllm_fp8_block_scale_moe_launcher(
    at::Tensor const& routing_logits, std::optional<at::Tensor> routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale,
    int64_t const num_experts, int64_t const top_k, int64_t const n_group, int64_t const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, double const routed_scaling_factor,
    int64_t const tile_tokens_dim, int64_t const routing_method_type,
    tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner& moe_runner, int64_t moeConfigIndex) {
  auto device = hidden_states.device();

  static const std::tuple<int, int> device_props = [&device] {
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device.index());
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device.index());
    return std::make_tuple(major, minor);
  }();

  TORCH_CHECK(std::get<0>(device_props) == 10 && std::get<1>(device_props) == 0,
              "This kernel requires SM 100 architecture. Current device has SM ",
              std::get<0>(device_props), std::get<1>(device_props));

  TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::Float,
              "routing_logits must be float.");
  TORCH_CHECK(routing_logits.dim() == 2, "routing_logits must be 2D.");
  TORCH_CHECK(routing_logits.sizes()[0] == hidden_states.sizes()[0],
              "routing_logits and hidden_states must have the same number of tokens.");
  TORCH_CHECK(routing_logits.sizes()[1] == num_experts,
              "routing_logits dim1 must match num_experts.");
  if (routing_bias.has_value()) {
    TORCH_CHECK(routing_bias.value().scalar_type() == at::ScalarType::BFloat16 ||
                    routing_bias.value().scalar_type() == at::ScalarType::Float,
                "routing_bias must be bfloat16 or float.");
    TORCH_CHECK(routing_bias.value().dim() == 1, "routing_bias must be 1D.");
    TORCH_CHECK(routing_bias.value().sizes()[0] == num_experts,
                "routing_bias has incorrect shape.");
  }

  if (n_group <= 0 || topk_group <= 0) {
    TORCH_CHECK(top_k == 1, "Current routing kernel (no groups) only supports top_k=1.");
  } else {
    TORCH_CHECK(top_k <= 8, "Current routing kernel (with groups) only supports top_k<=8.");
    TORCH_CHECK(topk_group <= 4,
                "Current routing kernel (with groups) only supports topk_group<=4.");
    TORCH_CHECK(topk_group <= n_group, "n_group must not be smaller than topk_group.");
    TORCH_CHECK(num_experts % n_group == 0, "num_experts must be divisible by n_group");
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TORCH_CHECK(top_k < (topk_group * num_experts / n_group),
                "top_k must be less than total number of experts in selected groups");
  }
  TORCH_CHECK(num_experts % 4 == 0,
              "Routing kernel expects that num_experts must be divisible by 4");
  TORCH_CHECK(num_experts > top_k, "num_experts must be greater than top_k");

  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  // Convert PyTorch dtype to TensorRT-LLM dtype
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half) {
    args.mDtypeElt = btg::Dtype::Fp16;
  } else if (dtype == at::ScalarType::BFloat16) {
    args.mDtypeElt = btg::Dtype::Bfloat16;
  } else if (dtype == at::ScalarType::Float8_e4m3fn) {
    args.mDtypeElt = btg::Dtype::E4m3;
  } else {
    TORCH_CHECK(false, "Unsupported input dtype for MoE: ", dtype);
  }

  auto const routing_bias_dtype =
      routing_bias.has_value() ? routing_bias.value().scalar_type() : at::ScalarType::BFloat16;
  args.mDtypeExpW =
      routing_bias_dtype == at::ScalarType::BFloat16 ? btg::Dtype::Bfloat16 : btg::Dtype::Fp32;
  args.routing_logits = routing_logits.data_ptr<float>();
  args.routing_bias = routing_bias.has_value() ? routing_bias.value().data_ptr() : nullptr;
  args.hidden_states = hidden_states.data_ptr();
  args.hidden_states_scale = hidden_states_scale.data_ptr<float>();
  args.gemm1_weights = gemm1_weights.data_ptr();
  args.gemm1_weights_scale = gemm1_weights_scale.data_ptr<float>();
  args.gemm2_weights = gemm2_weights.data_ptr();
  args.gemm2_weights_scale = gemm2_weights_scale.data_ptr<float>();
  args.num_tokens = hidden_states.sizes()[0];
  args.num_experts = num_experts;
  args.hidden_size = hidden_states.sizes()[1];
  args.hidden_size_output = args.hidden_size;
  args.top_k = top_k;
  args.n_group = n_group;
  args.topk_group = topk_group;
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor;
  args.intermediate_size = intermediate_size;
  args.mUseDeepSeekFp8 = true;

  // allocate workspace for routing kernel
  at::Tensor num_tokens_per_expert = at::detail::empty_cuda({num_experts}, at::ScalarType::Int,
                                                            routing_logits.device(), std::nullopt);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  at::Tensor total_num_padded_tokens =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));
  at::Tensor expanded_idx_to_permuted_idx = at::detail::empty_cuda(
      {args.num_tokens * args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor permuted_idx_to_token_idx = at::detail::empty_cuda(
      {max_num_padded_tokens}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_weights = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, routing_bias_dtype, routing_logits.device(), std::nullopt);
  at::Tensor expert_indexes = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  int64_t const size_of_expert_count_histogram = std::max(num_experts * 2, int64_t(256 * 2));
  at::Tensor expert_count_histogram = at::detail::empty_cuda(
      {size_of_expert_count_histogram},
      at::ScalarType::Int,  // 256 is the max number of threads per block and max number of experts
      routing_logits.device(), std::nullopt);

  // allocate workspace for activation/gemm/finalize kernels
  at::Tensor gemm1_output =
      at::detail::empty_cuda({max_num_padded_tokens, 2 * intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor gemm1_output_scale =
      at::detail::empty_cuda({2 * intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor activation_output =
      at::detail::empty_cuda({max_num_padded_tokens, intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor activation_output_scale =
      at::detail::empty_cuda({intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor gemm2_output =
      at::detail::empty_cuda({max_num_padded_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
      args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  at::Tensor cta_idx_xy_to_batch_idx = at::detail::empty_cuda(
      {max_num_ctas}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                             routing_logits.device(), std::nullopt);
  at::Tensor num_non_exiting_ctas =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));

  tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
  auto const& stream = at::cuda::getCurrentCUDAStream(routing_logits.get_device());
  routing_runner.run(
      routing_logits.data_ptr<float>(), args.routing_bias, args.num_tokens, args.num_experts,
      args.top_k, args.n_group, args.topk_group, args.local_expert_offset, args.local_num_experts,
      args.routed_scaling_factor, expert_indexes.data_ptr<int>(),
      expert_count_histogram.data_ptr<int>(), total_num_padded_tokens.data_ptr<int>(),
      expanded_idx_to_permuted_idx.data_ptr<int>(),
      nullptr /*permuted_idx_to_expanded_idx.data_ptr<int>()*/,
      permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
      num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
      cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(), args.mDtypeElt,
      false, true, static_cast<RoutingMethodType>(routing_method_type), stream);

  // MoE kernel except routing
  TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "hidden_states must be fp8.");
  TORCH_CHECK(hidden_states_scale.scalar_type() == at::ScalarType::Float,
              "hidden_states_scale must be float.");
  TORCH_CHECK(hidden_states_scale.dim() == 2, "hidden_states_scale must be 2D.");
  TORCH_CHECK(hidden_states_scale.sizes()[0] == hidden_states.sizes()[1] / 128,
              "hidden_states_scale dim0 must match hidden_states dim1 / 128.");
  TORCH_CHECK(hidden_states_scale.sizes()[1] == args.num_tokens,
              "hidden_states_scale dim1 must match num_tokens.");
  TORCH_CHECK(gemm1_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm1_weights must be fp8.");

  TORCH_CHECK(gemm1_weights.dim() == 3 || gemm1_weights.dim() == 4,
              "gemm1_weights must be 3D or 4D.");
  {
    int64_t Mn = 0, K = 0;
    if (gemm1_weights.dim() == 3) {
      // MajorK [num_experts, M, K]
      Mn = gemm1_weights.sizes()[1];
      K = gemm1_weights.sizes()[2];
    } else if (gemm1_weights.dim() == 4) {
      // BlockMajorK [num_experts, K/block_k, M, block_k]
      Mn = gemm1_weights.sizes()[2];
      int64_t block_k = gemm1_weights.sizes()[3];
      K = gemm1_weights.sizes()[1] * block_k;
    }
    TORCH_CHECK(Mn % 2 == 0, "the second dimension of weights must be even.");
    TORCH_CHECK(intermediate_size == Mn / 2, "intermediate_size has incorrect shape.");
    TORCH_CHECK(K == hidden_states.sizes()[1],
                "the third dimension of weights must be equal to hidden_size.");
  }
  TORCH_CHECK(gemm1_weights_scale.scalar_type() == at::ScalarType::Float,
              "gemm1_weights_scale must be float.");
  TORCH_CHECK(gemm1_weights_scale.dim() == 3, "gemm1_weights_scale must be 3D.");

  TORCH_CHECK(gemm1_weights_scale.sizes()[0] == local_num_experts,
              "gemm1_weights_scale has incorrect shape.");
  TORCH_CHECK(intermediate_size % 128 == 0,
              "the second dimension of weights must be a multiple of 128.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[1] == 2 * intermediate_size / 128,
              "gemm1_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[2] == args.hidden_size / 128,
              "gemm1_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm2_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm2_weights must be fp8.");

  TORCH_CHECK(gemm2_weights.dim() == 3 || gemm2_weights.dim() == 4,
              "gemm2_weights must be 3D or 4D.");
  {
    int64_t K = 0;
    if (gemm2_weights.dim() == 3) {
      // MajorK [num_experts, M, K]
      K = gemm2_weights.sizes()[2];
    } else if (gemm2_weights.dim() == 4) {
      // BlockMajorK [num_experts, K/block_k, M, block_k]
      int64_t block_k = gemm2_weights.sizes()[3];
      K = gemm2_weights.sizes()[1] * block_k;
    }
    TORCH_CHECK(K == intermediate_size,
                "the third dimension of weights must be equal to intermediate_size.");
  }
  TORCH_CHECK(gemm2_weights_scale.scalar_type() == at::ScalarType::Float,
              "gemm2_weights_scale must be float.");
  TORCH_CHECK(gemm2_weights_scale.dim() == 3, "gemm2_weights_scale must be 3D.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[0] == local_num_experts,
              "gemm2_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[1] == args.hidden_size / 128,
              "gemm2_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[2] == intermediate_size / 128,
              "gemm2_weights_scale has incorrect shape.");

  // allocate output
  at::Tensor output =
      at::detail::empty_cuda({args.num_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  // setup workspace
  workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = expert_indexes.data_ptr<int>();
  workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
  workspace.expanded_idx_to_permuted_idx =
      expanded_idx_to_permuted_idx.data_ptr<int>();  // Needed by activation/finalize kernels
  workspace.permuted_idx_to_token_idx =
      permuted_idx_to_token_idx.data_ptr<int>();         // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights.data_ptr();  // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
  workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
  workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output.data_ptr();
  workspace.gemm1_output_scale = gemm1_output_scale.data_ptr<float>();
  // activation intermediate ws
  workspace.activation_output = activation_output.data_ptr();
  workspace.activation_output_scale = activation_output_scale.data_ptr<float>();
  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output.data_ptr();
  workspace.gemm2_output_scale = nullptr;
  args.output = output.data_ptr();
  args.output_scale = nullptr;

  auto workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);
  at::Tensor workspace_fc1 = at::detail::empty_cuda(
      {std::get<0>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  at::Tensor workspace_fc2 = at::detail::empty_cuda(
      {std::get<1>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  workspace.bmm1_workspace = workspace_fc1.data_ptr();
  workspace.bmm2_workspace = workspace_fc2.data_ptr();

  auto const& moe_stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  moe_runner.run(args, workspace, hidden_states.get_device(), moe_stream, moeConfigIndex);
  return output;
}

at::Tensor trtllm_fp8_block_scale_moe(
    at::Tensor const& routing_logits, std::optional<at::Tensor> routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale, int64_t num_experts,
    int64_t top_k, int64_t n_group, int64_t topk_group, int64_t intermediate_size,
    int64_t local_expert_offset, int64_t local_num_experts, double routed_scaling_factor,
    int64_t tile_tokens_dim, int64_t routing_method_type, bool use_shuffled_weight,
    int64_t weight_layout) {
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half || dtype == at::ScalarType::BFloat16 ||
      dtype == at::ScalarType::Float8_e4m3fn) {
    using RunnerType = tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner;

    btg::Dtype mDtypeElt{btg::Dtype::E4m3};  // FP8 runner so hard-coded
    bool mUseDeepSeekFp8{true};              // Always true for BlockScaleMoe

    TORCH_CHECK(0 <= weight_layout && weight_layout <= 2,
                "the value of weight_layout is not recognized");

    // Properly initialize the runner using make_unique like in the original code
    auto mRunner = std::make_unique<RunnerType>(
        mDtypeElt, mUseDeepSeekFp8, tile_tokens_dim, use_shuffled_weight,
        static_cast<batchedGemm::gemm::MatrixLayout>(weight_layout));

    // Always use fallback config (equivalent to moeConfigIndex == -1 case from original code)
    auto const num_tokens = hidden_states.sizes()[0];
    auto const hidden_size = hidden_states.sizes()[1];

    int64_t moeConfigIndex = mRunner->getDefaultValidConfigIndex(
        top_k, hidden_size, intermediate_size, local_num_experts, num_tokens);

    return trtllm_fp8_block_scale_moe_launcher(
        routing_logits, routing_bias, hidden_states, hidden_states_scale, gemm1_weights,
        gemm1_weights_scale, gemm2_weights, gemm2_weights_scale, num_experts, top_k, n_group,
        topk_group, intermediate_size, local_expert_offset, local_num_experts,
        routed_scaling_factor, tile_tokens_dim, routing_method_type, *mRunner, moeConfigIndex);
  } else {
    TORCH_CHECK(false, "Unsupported input type: ", dtype);
  }
}

// TODO(siyuan): This launcher supports flexible weight and activation types.
// We should cleanup other launchers and only use this one in the future.
std::vector<at::Tensor> trtllm_fp4_block_scale_moe_launcher(
    std::optional<at::Tensor> const& routing_logits, at::Tensor& expert_indices,
    at::Tensor& expert_weights, std::optional<at::Tensor> const& routing_bias,
    at::Tensor const& hidden_states, std::optional<at::Tensor> const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    std::optional<at::Tensor> const& gemm1_bias, std::optional<at::Tensor> const& gemm1_alpha,
    std::optional<at::Tensor> const& gemm1_beta, std::optional<at::Tensor> const& gemm1_clamp_limit,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale,
    std::optional<at::Tensor> const& gemm2_bias,
    std::optional<at::Tensor> const& output1_scales_scalar,
    std::optional<at::Tensor> const& output1_scales_gate_scalar,
    std::optional<at::Tensor> const& output2_scales_scalar, int64_t const num_experts,
    int64_t const top_k, std::optional<int64_t> const n_group,
    std::optional<int64_t> const topk_group, int64_t const intermediate_size,
    int64_t const local_expert_offset, int64_t const local_num_experts,
    std::optional<double> const routed_scaling_factor, int64_t const tile_tokens_dim,
    int64_t const routing_method_type, bool const do_finalize,
    tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner& moe_runner, btg::Dtype dtype_act,
    btg::Dtype dtype_weights, int64_t const moeConfigIndex, at::Tensor& output) {
  auto device = hidden_states.device();

  static const std::tuple<int, int> device_props = [&device] {
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device.index());
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device.index());
    return std::make_tuple(major, minor);
  }();

  TORCH_CHECK(std::get<0>(device_props) == 10 && std::get<1>(device_props) == 0,
              "This kernel requires SM 100 architecture. Current device has SM ",
              std::get<0>(device_props), std::get<1>(device_props));

  TORCH_CHECK(dtype_act == btg::Dtype::E2m1 || dtype_act == btg::Dtype::Bfloat16 ||
                  dtype_act == btg::Dtype::E4m3 || dtype_act == btg::Dtype::MxE4m3,
              "Only E2m1, Bfloat16, MxE4m3 and E4m3 are supported by block scale MoE");
  if (dtype_act == btg::Dtype::E2m1) {
    TORCH_CHECK(dtype_weights == btg::Dtype::E2m1,
                "Only E2m1 and MxE2m1 are supported by block scale MoE with E2m1 activation");
    TORCH_CHECK(hidden_states_scale.has_value(),
                "hidden_states_scale is required for E2m1 activation");
    TORCH_CHECK(output1_scales_scalar.has_value(),
                "output1_scales_scalar is required for E2m1 activation");
    TORCH_CHECK(output1_scales_gate_scalar.has_value(),
                "output1_scales_gate_scalar is required for E2m1 activation");
    TORCH_CHECK(output2_scales_scalar.has_value(),
                "output2_scales_scalar is required for E2m1 activation");
  } else if (dtype_act == btg::Dtype::Bfloat16 || dtype_act == btg::Dtype::E4m3 ||
             dtype_act == btg::Dtype::MxE4m3) {
    TORCH_CHECK(dtype_weights == btg::Dtype::MxE2m1,
                "Only MxE2m1 weights are supported by block scale MoE with Bfloat16, E4m3 or "
                "MxE4m3 activation");
  } else {
    TORCH_CHECK(false, "Invalid dtype_act");
  }

  if (dtype_act == btg::Dtype::E4m3) {
    TORCH_CHECK(output1_scales_scalar.has_value(),
                "output1_scales_scalar is required for E4m3 activation");
    TORCH_CHECK(output1_scales_gate_scalar.has_value(),
                "output1_scales_gate_scalar is required for E4m3 activation");
    TORCH_CHECK(output2_scales_scalar.has_value(),
                "output2_scales_scalar is required for E4m3 activation");
  }

  if (routing_logits.has_value()) {
    TORCH_CHECK(routing_logits.value().scalar_type() == at::ScalarType::Float ||
                    routing_logits.value().scalar_type() == at::ScalarType::BFloat16,
                "routing_logits must be float or bfloat16.");
    TORCH_CHECK(routing_logits.value().dim() == 2, "routing_logits must be 2D.");
    TORCH_CHECK(routing_logits.value().sizes()[1] == num_experts,
                "routing_logits has incorrect shape.");
  }
  if (routing_bias.has_value()) {
    TORCH_CHECK(routing_bias.value().scalar_type() == at::ScalarType::BFloat16,
                "routing_bias must be bfloat16.");
    TORCH_CHECK(routing_bias.value().dim() == 1, "routing_bias must be 1D.");
    TORCH_CHECK(routing_bias.value().sizes()[0] == num_experts,
                "routing_bias has incorrect shape.");
  }

  if (n_group.value_or(0) != 0) {
    TORCH_CHECK(
        static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::DeepSeekV3,
        "Routing kernel with groups implies DeepSeekV3 routing method.");
    TORCH_CHECK(topk_group.has_value(), "if n_group is given, topk_group must be given");
    TORCH_CHECK(num_experts % n_group.value() == 0, "num_experts must be divisible by n_group");
    TORCH_CHECK(top_k <= 8 && top_k > 0,
                "Current routing kernel (with groups) only supports top_k<=8 && top_k>0.");
    TORCH_CHECK(
        topk_group.value() <= 4 && topk_group.value() > 0,
        "Current routing kernel only (with groups) supports topk_group<=4 && topk_group > 0.");
    TORCH_CHECK(topk_group.value() <= n_group.value(),
                "n_group must not be smaller than topk_group.");
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TORCH_CHECK(top_k < (topk_group.value() * num_experts / n_group.value()),
                "top_k must be less than total number of experts in selected groups");
  } else if (static_cast<RoutingMethodType>(routing_method_type) ==
                 RoutingMethodType::Renormalize ||
             static_cast<RoutingMethodType>(routing_method_type) ==
                 RoutingMethodType::RenormalizeNaive) {
    TORCH_CHECK(
        top_k <= 8 && top_k > 0,
        "Current routing kernel (no groups, renormalize) only supports top_k<=8 && top_k>0.");
  } else if (static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::Llama4) {
    TORCH_CHECK(top_k == 1, "Current routing kernel (no groups, Llama4) only supports top_k=1.");
  }

  TORCH_CHECK(num_experts % 4 == 0,
              "Routing kernel expects that num_experts must be divisible by 4");
  TORCH_CHECK(num_experts > top_k, "num_experts must be greater than top_k");

  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  // setup args
  // note: the assumption is that output data type is always Bfloat16 (the default)
  auto routing_bias_dtype = at::ScalarType::BFloat16;
  if (routing_bias.has_value()) {
    routing_bias_dtype = routing_bias.value().scalar_type();
  } else if (routing_logits.has_value()) {
    routing_bias_dtype = routing_logits.value().scalar_type();
  }
  args.mDtypeElt = dtype_act;
  args.mDtypeExpW =
      routing_bias_dtype == at::ScalarType::Float ? btg::Dtype::Fp32 : btg::Dtype::Bfloat16;
  args.routing_logits = routing_logits.has_value() ? routing_logits.value().data_ptr() : nullptr;
  args.routing_bias = routing_bias.has_value() ? routing_bias.value().data_ptr() : nullptr;
  args.hidden_states = hidden_states.data_ptr();
  args.hidden_states_scale =
      hidden_states_scale.has_value() ? hidden_states_scale.value().data_ptr() : nullptr;
  args.gemm1_weights = gemm1_weights.data_ptr();
  args.gemm1_weights_scale = gemm1_weights_scale.data_ptr();
  args.gemm1_bias = gemm1_bias.has_value() ? gemm1_bias.value().data_ptr<float>() : nullptr;
  args.gemm1_alpha = gemm1_alpha.has_value() ? gemm1_alpha.value().data_ptr<float>() : nullptr;
  args.gemm1_beta = gemm1_beta.has_value() ? gemm1_beta.value().data_ptr<float>() : nullptr;
  args.gemm1_clamp_limit =
      gemm1_clamp_limit.has_value() ? gemm1_clamp_limit.value().data_ptr<float>() : nullptr;
  args.gemm2_weights = gemm2_weights.data_ptr();
  args.gemm2_weights_scale = gemm2_weights_scale.data_ptr();
  args.gemm2_bias = gemm2_bias.has_value() ? gemm2_bias.value().data_ptr<float>() : nullptr;
  args.num_tokens = hidden_states.sizes()[0];
  args.num_experts = num_experts;
  // * 2 to compensate for the fact that sizeof(hidden_states.dtype) is 1 because we pack 2 e2m1
  // into 1 byte.
  auto const hidden_states_hidden_size =
      dtype_act == btg::Dtype::E2m1 ? hidden_states.sizes()[1] * 2 : hidden_states.sizes()[1];
  args.hidden_size = hidden_states_hidden_size;
  args.hidden_size_output = args.hidden_size;
  args.top_k = top_k;
  args.n_group = n_group.value_or(0);
  args.topk_group = topk_group.value_or(0);
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor.value_or(1.0);
  args.intermediate_size = intermediate_size;

  // allocate workspace for routing kernel
  at::Tensor num_tokens_per_expert = at::detail::empty_cuda({num_experts}, at::ScalarType::Int,
                                                            hidden_states.device(), std::nullopt);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  at::Tensor total_num_padded_tokens =
      at::empty({1}, at::TensorOptions().device(hidden_states.device()).dtype(at::ScalarType::Int));
  at::Tensor expanded_idx_to_permuted_idx = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, hidden_states.device(), std::nullopt);

  at::Tensor permuted_idx_to_token_idx = at::detail::empty_cuda(
      {max_num_padded_tokens}, at::ScalarType::Int, hidden_states.device(), std::nullopt);
  // at::Tensor expert_weights = at::detail::empty_cuda(
  //     {args.num_tokens, args.top_k}, routing_bias_dtype, hidden_states.device(), std::nullopt);
  // at::Tensor expert_indexes = at::detail::empty_cuda(
  //     {args.num_tokens, args.top_k}, at::ScalarType::Int, hidden_states.device(), std::nullopt);
  at::Tensor expert_count_histogram = at::detail::empty_cuda(
      {2 * 256},
      at::ScalarType::Int,  // 256 is the max number of threads per block and max number of experts
      hidden_states.device(), std::nullopt);

  auto const sf_vec_size = dtype_weights == btg::Dtype::MxE2m1 ? 32 : 16;

  // allocate workspace for activation/gemm/finalize kernels
  auto const gemm1_output_hidden =
      dtype_act == btg::Dtype::E2m1 ? intermediate_size / 2 : intermediate_size;
  at::Tensor gemm1_output = at::detail::empty_cuda(
      {max_num_padded_tokens, gemm1_output_hidden},
      dtype_act == btg::Dtype::Bfloat16 ? at::ScalarType::BFloat16 : at::ScalarType::Float8_e4m3fn,
      hidden_states.device(), std::nullopt);

  std::optional<at::Tensor> gemm1_output_scale = std::nullopt;
  if (dtype_act == btg::Dtype::E2m1 || dtype_act == btg::Dtype::MxE4m3) {
    int64_t sf_size = tensorrt_llm::computeFP4SwizzledLayoutSFSize(max_num_padded_tokens,
                                                                   intermediate_size / sf_vec_size);
    gemm1_output_scale = at::detail::empty_cuda({sf_size}, at::ScalarType::Float8_e4m3fn,
                                                hidden_states.device(), std::nullopt);
  }

  at::Tensor gemm2_output =
      at::detail::empty_cuda({max_num_padded_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
      args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  at::Tensor cta_idx_xy_to_batch_idx = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                              hidden_states.device(), std::nullopt);
  at::Tensor cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                             hidden_states.device(), std::nullopt);
  at::Tensor num_non_exiting_ctas =
      at::empty({1}, at::TensorOptions().device(hidden_states.device()).dtype(at::ScalarType::Int));

  //
  // TopK routing
  //

  tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
  auto const& stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  routing_runner.run(
      args.routing_logits, args.routing_bias, args.num_tokens, args.num_experts, args.top_k,
      args.n_group, args.topk_group, args.local_expert_offset, args.local_num_experts,
      args.routed_scaling_factor, expert_indices.data_ptr<int>(),
      expert_count_histogram.data_ptr<int>(), total_num_padded_tokens.data_ptr<int>(),
      expanded_idx_to_permuted_idx.data_ptr<int>(),
      nullptr, /*permuted_idx_to_expanded_idx.data_ptr<int>(),*/
      permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
      num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
      cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(), args.mDtypeElt,
      false /* use_routing_scales_on_input */, false /* use_deep_seek_fp8 */,
      static_cast<RoutingMethodType>(routing_method_type), stream);

  //
  // FC13 (gemm1) + FC2 (gemm2)
  //

  if (dtype_act == btg::Dtype::E2m1) {
    TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Byte, "hidden_states must be byte.");
  } else if (dtype_act == btg::Dtype::E4m3 || dtype_act == btg::Dtype::MxE4m3) {
    TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
                "hidden_states must be fp8.");
  } else if (dtype_act == btg::Dtype::Bfloat16) {
    TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::BFloat16,
                "hidden_states must be bfloat16.");
  } else {
    TORCH_CHECK(false, "Invalid dtype_act");
  }

  if (hidden_states_scale.has_value()) {
    TORCH_CHECK(hidden_states_scale.value().scalar_type() == at::ScalarType::Float8_e4m3fn,
                "hidden_states_scale must be fp8.");

    TORCH_CHECK(hidden_states_scale.value().dim() == 1, "hidden_states_scale must be 1D.");
    TORCH_CHECK(hidden_states_scale.value().sizes()[0] ==
                    tensorrt_llm::computeFP4LinearLayoutSFSize(args.num_tokens,
                                                               args.hidden_size / sf_vec_size),
                "hidden_states_scale has incorrect size");
  }

  TORCH_CHECK(gemm1_weights.scalar_type() == torch_ext::FLOAT4_E2M1X2,
              "gemm1_weights must be byte.");

  TORCH_CHECK(gemm1_weights.dim() == 3, "gemm1_weights must be 3D.");
  TORCH_CHECK(gemm1_weights.sizes()[1] % 2 == 0, "the second dimension of weights must be even.");
  TORCH_CHECK(intermediate_size == gemm1_weights.sizes()[1] / 2,
              "intermediate_size has incorrect dim 1.");
  // This check passes even though the actual shape of the weights[2] and hidden_states[1] is
  // 2 times larger due to the fact that 2 e2m1 are packed into 1 byte.
  TORCH_CHECK(
      gemm1_weights.sizes()[2] ==
          (dtype_act == btg::Dtype::E2m1 ? hidden_states.sizes()[1] : hidden_states.sizes()[1] / 2),
      "the third dimension of weights must be equal to hidden_size.");

  TORCH_CHECK(gemm1_weights_scale.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm1_weights_scale must be fp8.");

  TORCH_CHECK(gemm1_weights_scale.dim() == 3, "gemm1_weights_scale must be 3D.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[0] == local_num_experts,
              "gemm1_weights_scale has incorrect dim 0.");
  TORCH_CHECK(intermediate_size % sf_vec_size == 0,
              "the second dimension of weights must be a multiple of %d.", sf_vec_size);
  TORCH_CHECK(gemm1_weights_scale.sizes()[1] == 2 * intermediate_size,
              "gemm1_weights_scale has incorrect dim 1.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[2] == args.hidden_size / sf_vec_size,
              "gemm1_weights_scale has incorrect dim 2.");

  if (gemm1_bias.has_value()) {
    TORCH_CHECK(gemm1_bias.value().scalar_type() == at::ScalarType::Float,
                "gemm1_bias must be float, got %s.",
                c10::toString(gemm1_bias.value().scalar_type()));
    TORCH_CHECK(gemm1_bias.value().dim() == 2, "gemm1_bias must be 2D.");
    TORCH_CHECK(gemm1_bias.value().sizes()[0] == local_num_experts,
                "gemm1_bias has incorrect dim 0.");
    TORCH_CHECK(gemm1_bias.value().sizes()[1] == 2 * intermediate_size,
                "gemm1_bias has incorrect dim 1.");
  }

  if (gemm1_alpha.has_value()) {
    TORCH_CHECK(gemm1_alpha.value().scalar_type() == at::ScalarType::Float,
                "gemm1_alpha must be float, got %s.",
                c10::toString(gemm1_alpha.value().scalar_type()));
    TORCH_CHECK(gemm1_alpha.value().dim() == 1, "gemm1_alpha must be 1D.");
    TORCH_CHECK(gemm1_alpha.value().sizes()[0] == local_num_experts,
                "gemm1_alpha has incorrect dim 0.");
  }
  if (gemm1_beta.has_value()) {
    TORCH_CHECK(gemm1_beta.value().scalar_type() == at::ScalarType::Float,
                "gemm1_beta must be float, got %s.",
                c10::toString(gemm1_beta.value().scalar_type()));
    TORCH_CHECK(gemm1_beta.value().dim() == 1, "gemm1_beta must be 1D.");
    TORCH_CHECK(gemm1_beta.value().sizes()[0] == local_num_experts,
                "gemm1_beta has incorrect dim 0.");
  }

  TORCH_CHECK(gemm2_weights.scalar_type() == torch_ext::FLOAT4_E2M1X2,
              "gemm2_weights must be byte.");

  TORCH_CHECK(gemm2_weights.dim() == 3, "gemm2_weights must be 3D.");
  // / 2 to compensate for the fact that we pack 2 e2m1 into 1 byte.
  TORCH_CHECK(gemm2_weights.sizes()[2] == intermediate_size / 2,
              "the third dimension of weights must be equal to intermediate_size.");

  TORCH_CHECK(gemm2_weights_scale.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm2_weights_scale must be fp8.");

  TORCH_CHECK(gemm2_weights_scale.dim() == 3, "gemm2_weights_scale must be 3D.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[0] == local_num_experts,
              "gemm2_weights_scale has incorrect dim 0.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[1] == args.hidden_size,
              "gemm2_weights_scale has incorrect dim 1.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[2] == intermediate_size / sf_vec_size,
              "gemm2_weights_scale has incorrect dim 2.");

  if (output1_scales_scalar.has_value()) {
    TORCH_CHECK(output1_scales_scalar.value().scalar_type() == at::ScalarType::Float,
                "output1_scales_scalar must be float.");
    TORCH_CHECK(output1_scales_scalar.value().dim() == 1, "output1_scales_scalar must be 1D.");
    TORCH_CHECK(output1_scales_scalar.value().sizes()[0] == local_num_experts,
                "output1_scales_scalar has incorrect dim 0.");
  }

  if (output1_scales_gate_scalar.has_value()) {
    TORCH_CHECK(output1_scales_gate_scalar.value().scalar_type() == at::ScalarType::Float,
                "output1_scales_gate_scalar must be float.");
    TORCH_CHECK(output1_scales_gate_scalar.value().dim() == 1,
                "output1_scales_gate_scalar must be 1D.");
    TORCH_CHECK(output1_scales_gate_scalar.value().sizes()[0] == local_num_experts,
                "output1_scales_gate_scalar has incorrect dim 0.");
  }

  if (output2_scales_scalar.has_value()) {
    TORCH_CHECK(output2_scales_scalar.value().scalar_type() == at::ScalarType::Float,
                "output2_scales_scalar must be float.");
    TORCH_CHECK(output2_scales_scalar.value().dim() == 1, "output2_scales_scalar must be 1D.");
    TORCH_CHECK(output2_scales_scalar.value().sizes()[0] == local_num_experts,
                "output2_scales_scalar has incorrect dim 0.");
  }

  // setup workspace
  workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = expert_indices.data_ptr<int>();
  workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
  workspace.expanded_idx_to_permuted_idx =
      expanded_idx_to_permuted_idx.data_ptr<int>();  // Needed by permute/finalize kernels
  workspace.permuted_idx_to_token_idx =
      permuted_idx_to_token_idx.data_ptr<int>();         // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights.data_ptr();  // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
  workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
  workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();

  workspace.hidden_states_scale_linear = nullptr;

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output.data_ptr();
  workspace.gemm1_output_scale =
      gemm1_output_scale.has_value()
          ? reinterpret_cast<float*>(gemm1_output_scale.value().data_ptr())
          : nullptr;

  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output.data_ptr();
  workspace.gemm2_output_scale = nullptr;
  args.output = output.data_ptr();
  args.output_scale = nullptr;
  args.output1_scales_scalar =
      output1_scales_scalar.has_value() ? output1_scales_scalar.value().data_ptr<float>() : nullptr;
  args.output1_scales_gate_scalar = output1_scales_gate_scalar.has_value()
                                        ? output1_scales_gate_scalar.value().data_ptr<float>()
                                        : nullptr;
  args.output2_scales_scalar =
      output2_scales_scalar.has_value() ? output2_scales_scalar.value().data_ptr<float>() : nullptr;
  args.do_finalize = do_finalize;

  auto const workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);

  at::Tensor workspace_fc1 = at::detail::empty_cuda(
      {std::get<0>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  at::Tensor workspace_fc2 = at::detail::empty_cuda(
      {std::get<1>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  workspace.bmm1_workspace = workspace_fc1.data_ptr();
  workspace.bmm2_workspace = workspace_fc2.data_ptr();
  auto const& moe_stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  moe_runner.run(args, workspace, hidden_states.get_device(), moe_stream, moeConfigIndex);

  if (!do_finalize) {
    return {gemm2_output, expert_weights, expanded_idx_to_permuted_idx};
  }
  return {output};
}

std::vector<at::Tensor> trtllm_fp4_block_scale_moe(
    std::optional<at::Tensor> const& routing_logits, at::Tensor& topk_ids,
    at::Tensor& expert_weights, std::optional<at::Tensor> const& routing_bias,
    at::Tensor const& hidden_states, std::optional<at::Tensor> const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    std::optional<at::Tensor> const& gemm1_bias, std::optional<at::Tensor> const& gemm1_alpha,
    std::optional<at::Tensor> const& gemm1_beta, std::optional<at::Tensor> const& gemm1_clamp_limit,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale,
    std::optional<at::Tensor> const& gemm2_bias,
    std::optional<at::Tensor> const& output1_scales_scalar,
    std::optional<at::Tensor> const& output1_scales_gate_scalar,
    std::optional<at::Tensor> const& output2_scales_scalar, int64_t num_experts, int64_t top_k,
    std::optional<int64_t> n_group, std::optional<int64_t> topk_group, int64_t intermediate_size,
    int64_t local_expert_offset, int64_t local_num_experts,
    std::optional<double> routed_scaling_factor, int64_t tile_tokens_dim,
    int64_t routing_method_type, bool do_finalize, at::Tensor& output) {
  using RunnerType = tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner;

  int const num_tokens = hidden_states.sizes()[0];
  int hidden_size = hidden_states.sizes()[1];
  if (hidden_states.scalar_type() == torch_ext::FLOAT4_E2M1X2) hidden_size *= 2;
  int hidden_states_scale_vec_size = -1;
  if (hidden_states_scale.has_value()) {
    hidden_states_scale_vec_size = (num_tokens * hidden_size) / hidden_states_scale.value().numel();
  }
  int weight_scale_vec_size =
      (num_experts * intermediate_size * 2 * hidden_size) / gemm1_weights_scale.numel();
  TORCH_CHECK(weight_scale_vec_size == 16 || weight_scale_vec_size == 32,
              "unsupported weight_scale_vec_size.");
  auto mDtypeWeights = weight_scale_vec_size == 16 ? btg::Dtype::E2m1 : btg::Dtype::MxE2m1;

  TORCH_CHECK(gemm1_weights.scalar_type() == at::ScalarType::Byte &&
                  gemm2_weights.scalar_type() == at::ScalarType::Byte,
              "weights must be fp4 packed in uint8.");
  TORCH_CHECK(hidden_states.scalar_type() == torch_ext::FLOAT4_E2M1X2 ||
                  hidden_states.scalar_type() == at::ScalarType::BFloat16 ||
                  hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "hidden_states must be bf16, fp8 or uint8 (packed fp4).");
  auto mDtypeAct = btg::Dtype::Bfloat16;
  if (hidden_states.scalar_type() == torch_ext::FLOAT4_E2M1X2) {
    TORCH_CHECK(hidden_states_scale.has_value() &&
                    hidden_states_scale.value().scalar_type() == at::ScalarType::Float8_e4m3fn,
                "hidden_states_scale must be provided for fp4 activation.");
    if (hidden_states_scale_vec_size == 16) {
      mDtypeAct = btg::Dtype::E2m1;
    } else if (hidden_states_scale_vec_size == 32) {
      mDtypeAct = btg::Dtype::MxE2m1;
    } else {
      TORCH_CHECK(false, "unsupported hidden_states_scale shape.");
    }
  } else if (hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn) {
    if (hidden_states_scale.has_value()) {
      if (hidden_states_scale_vec_size == 32) {
        mDtypeAct = btg::Dtype::MxE4m3;
      } else {
        TORCH_CHECK(false, "unsupported hidden_states_scale shape.");
      }
    } else {
      mDtypeAct = btg::Dtype::E4m3;
    }
  }
  bool mUseDeepSeekFp8{false};  // FP4 doesn't use DeepSeek FP8

  // Properly initialize the runner using make_unique like in the original code
  auto mRunner = std::make_unique<RunnerType>(
      mDtypeAct, mDtypeWeights, mUseDeepSeekFp8, (int32_t)tile_tokens_dim,
      tensorrt_llm::kernels::ActType::SwiGlu, /*useShuffledMatrixA*/ true);

  auto const moeConfigIndex = mRunner->getDefaultValidConfigIndex(
      top_k, hidden_size, intermediate_size, local_num_experts, num_tokens);

  return trtllm_fp4_block_scale_moe_launcher(
      routing_logits, topk_ids, expert_weights, routing_bias, hidden_states, hidden_states_scale,
      gemm1_weights, gemm1_weights_scale, gemm1_bias, gemm1_alpha, gemm1_beta, gemm1_clamp_limit,
      gemm2_weights, gemm2_weights_scale, gemm2_bias, output1_scales_scalar,
      output1_scales_gate_scalar, output2_scales_scalar, num_experts, top_k, n_group, topk_group,
      intermediate_size, local_expert_offset, local_num_experts, routed_scaling_factor,
      tile_tokens_dim, routing_method_type, do_finalize, *mRunner, mDtypeAct, mDtypeWeights,
      moeConfigIndex, output);
}

namespace trtllm_cubin_loader {
#include <flashinfer/cubin_loader.h>
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_fp8_per_tensor_scale_moe", trtllm_fp8_per_tensor_scale_moe);
  m.def("trtllm_fp8_block_scale_moe", trtllm_fp8_block_scale_moe);
  m.def("trtllm_fp4_block_scale_moe", trtllm_fp4_block_scale_moe);
}

}  // namespace flashinfer
