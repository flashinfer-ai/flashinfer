#include "hip/hip_runtime.h"
/*
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/EmptyTensor.h>
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/util/Exception.h>
#include <hip/hip_runtime.h>
#include <flashinfer/exception.h>
#include <hip/hiprtc.h>
#include <torch/library.h>

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include "flashinfer/trtllm/batched_gemm/trtllmGen_bmm_export/GemmGatedActOptions.h"
#include "flashinfer/trtllm/batched_gemm/trtllmGen_bmm_export/trtllm/gen/DtypeDecl.h"
#include "flashinfer/trtllm/fused_moe/DevKernel.h"
#include "flashinfer/trtllm/fused_moe/RoutingKernel.h"
#include "flashinfer/trtllm/fused_moe/runner.h"
#include "nv_internal/tensorrt_llm/kernels/quantization.h"
#include "nv_internal/tensorrt_llm/thop/thUtils.h"

namespace flashinfer {

namespace btg = batchedGemm::trtllm::gen;
using batchedGemm::gemm::MatrixLayout;
using tensorrt_llm::kernels::trtllmgen_moe::MoE::GatedActType;
using tensorrt_llm::kernels::trtllmgen_moe::Routing::RoutingMethodType;

/*

Abstraction layers:

1. TORCH_LIBRARY_FRAGMENT bindings
These are currently the same signature as the public python APIs.
We strive to make the python interface relatively stable
and the naming of parameters meaningful to the users.

2. FusedMoeLauncher
This performs checks and preparations for the execution,
organized in several stages, see FusedMoeLauncher::run().

3. MoE::Runner
Orchestrate and dispatch all the kernels executions to fulfill the requested operation.
This includes PermuteGemm1, Gemm2, activation (if not fused), and finalize.

4. TrtllmGenBatchedGemmRunner
This provides tactic selection if not determined yet at the public API (or auto-tuning)

5. BatchedGemm Runner
The low-level gemm kernel executor which is updated together with the kernels.

6. BatchedGemmInterface
Driver calls take place to carry out the gemm operations.
*/

class FusedMoeLauncher {
 protected:
  at::Tensor const* routing_logits{};
  at::Tensor const* routing_bias{};
  at::Tensor const* hidden_states{};
  at::Tensor const* gemm1_weights{};
  at::Tensor const* output1_scales_scalar{};
  at::Tensor const* output1_scales_gate_scalar{};
  at::Tensor const* gemm2_weights{};
  at::Tensor const* output2_scales_scalar{};

  int64_t tile_tokens_dim{};
  int64_t routing_method_type{};
  bool use_shuffled_weight{};
  MatrixLayout weight_layout{MatrixLayout::MajorK};

  std::tuple<int, int> device_version;
  std::unique_ptr<tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs> args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  btg::Dtype mDtypeAct{btg::Dtype::Bfloat16};
  btg::Dtype mDtypeWeights{btg::Dtype::Bfloat16};
  GatedActType gated_act_type{GatedActType::SwiGlu};

  // Initialize common data necessary for later.
  // May throw exception from TORCH_CHECK.
  void init_common(at::Tensor const* routing_logits, at::Tensor const* routing_bias,
                   at::Tensor const* hidden_states, at::Tensor const* gemm1_weights,
                   at::Tensor const* gemm2_weights,
                   std::unique_ptr<tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs>&& args,
                   int64_t tile_tokens_dim, int64_t routing_method_type, bool use_shuffled_weight,
                   int64_t weight_layout, int64_t gated_act_type);

  // Routing logits [num_tokens, num_experts]
  void check_routing_logits_shape() const {
    TORCH_CHECK(routing_logits->dim() == 2, "routing_logits must be 2D.");
    TORCH_CHECK(routing_logits->sizes()[0] == hidden_states->sizes()[0],
                "routing_logits and hidden_states must have the same number of tokens.");
    TORCH_CHECK(routing_logits->sizes()[1] == args->num_experts,
                "routing_logits dim1 must match num_experts.");
  }

  // Routing bias [num_experts]
  void check_routing_bias_shape() const {
    if (routing_bias != nullptr) {
      TORCH_CHECK(routing_bias->dim() == 1, "routing_bias must be 1D.");
      TORCH_CHECK(routing_bias->sizes()[0] == args->num_experts,
                  "routing_bias has incorrect shape.");
    }
  }

  // Hidden states [num_tokens, hidden_size]
  void check_hidden_states_shape() const {
    TORCH_CHECK(hidden_states->dim() == 2, "hidden_states must be 2D.");
    TORCH_CHECK(hidden_states->sizes()[1] == args->intermediate_size,
                "hidden_states has incorrect shape.");
  }

  // GEMM1 or GEMM2 weights [num_experts, M, K] or [num_experts, K/block_k, M, block_k]
  void check_weights_shape(std::string which_weights) const {
    at::Tensor const* weights{};
    if (which_weights == "gemm1") {
      weights = gemm1_weights;
    } else if (which_weights == "gemm2") {
      weights = gemm2_weights;
    } else {
      TORCH_CHECK(false, "Internal error: which_weights = ", which_weights);
    }

    int64_t Mn = 0, K = 0;
    if (weight_layout == MatrixLayout::MajorK) {
      // MajorK [num_experts, M, K]
      Mn = weights->sizes()[1];
      K = weights->sizes()[2];
    } else if (weight_layout == MatrixLayout::BlockMajorK) {
      // BlockMajorK [num_experts, K/block_k, M, block_k]
      Mn = weights->sizes()[2];
      int64_t block_k = weights->sizes()[3];
      K = weights->sizes()[1] * block_k;
    } else {
      TORCH_CHECK(false, "Unsupported weight_layout: ", weight_layout);
    }
    TORCH_CHECK(weights->sizes()[0] == args->num_experts,
                which_weights + " weights expert dimension must match num_experts");
    if (which_weights == "gemm1") {
      TORCH_CHECK(Mn % 2 == 0, which_weights + " weights Mn dimension must be even.");
      TORCH_CHECK(args->intermediate_size == Mn / 2, "intermediate_size has incorrect shape.");
      TORCH_CHECK(K == hidden_states->sizes()[1],
                  which_weights + " weights K dimension must be equal to hidden_size.");
    } else if (which_weights == "gemm2") {
      TORCH_CHECK(K == args->intermediate_size,
                  which_weights + " weights K dimension must be equal to intermediate_size.");
    }
  }

  void check_routing_common() const {
    TORCH_CHECK(args->top_k > 0 && args->top_k <= args->num_experts,
                "top_k must be between 1 and num_experts");
    TORCH_CHECK(args->local_num_experts > 0 && args->local_num_experts <= args->num_experts,
                "local_num_experts must be between 1 and num_experts");
    TORCH_CHECK(args->local_expert_offset >= 0 &&
                    args->local_expert_offset + args->local_num_experts <= args->num_experts,
                "expert offset and count must be within valid range");

    check_routing_logits_shape();

    if (routing_bias) {
      check_routing_bias_shape();
    }
  }

  // Routing phase workspace tensors (allocated in prepare_routing() or prepare_routing_common())
  at::Tensor num_tokens_per_expert;
  at::Tensor total_num_padded_tokens;
  at::Tensor expanded_idx_to_permuted_idx;
  at::Tensor permuted_idx_to_token_idx;
  at::Tensor expert_weights;
  at::Tensor expert_indexes;
  at::Tensor expert_count_histogram;
  at::Tensor cta_idx_xy_to_batch_idx;
  at::Tensor cta_idx_xy_to_mn_limit;
  at::Tensor num_non_exiting_ctas;

  void prepare_routing_common() {
    // Allocate routing phase workspace tensors
    int32_t max_num_padded_tokens =
        tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
            args->num_tokens, args->top_k, args->num_experts, tile_tokens_dim);

    // Common routing workspace tensors allocation
    num_tokens_per_expert = at::detail::empty_cuda({args->num_experts}, at::ScalarType::Int,
                                                   routing_logits->device(), std::nullopt);

    total_num_padded_tokens = at::empty(
        {}, at::TensorOptions().device(routing_logits->device()).dtype(at::ScalarType::Int));

    expanded_idx_to_permuted_idx =
        at::detail::empty_cuda({args->num_tokens * args->top_k}, at::ScalarType::Int,
                               routing_logits->device(), std::nullopt);

    permuted_idx_to_token_idx = at::detail::empty_cuda({max_num_padded_tokens}, at::ScalarType::Int,
                                                       routing_logits->device(), std::nullopt);

    expert_indexes = at::detail::empty_cuda({args->num_tokens, args->top_k}, at::ScalarType::Int,
                                            routing_logits->device(), std::nullopt);

    // expert_weights allocation should be done by derived class since data type could vary

    int64_t const size_of_expert_count_histogram = std::max(args->num_experts * 2, 256 * 2);
    expert_count_histogram =
        at::detail::empty_cuda({size_of_expert_count_histogram},
                               at::ScalarType::Int,  // 256 is the max number of threads per block
                                                     // and max number of experts
                               routing_logits->device(), std::nullopt);

    int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
        args->num_tokens, args->top_k, args->num_experts, tile_tokens_dim);

    cta_idx_xy_to_batch_idx = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                     routing_logits->device(), std::nullopt);

    cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                    routing_logits->device(), std::nullopt);

    num_non_exiting_ctas = at::empty(
        {}, at::TensorOptions().device(routing_logits->device()).dtype(at::ScalarType::Int));

    workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
    workspace.total_max_padded_tokens = max_num_padded_tokens;
    workspace.ProjUpTileN = tile_tokens_dim;
    workspace.routing_expert_indexes = expert_indexes.data_ptr<int>();
    workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
    workspace.expanded_idx_to_permuted_idx = expanded_idx_to_permuted_idx.data_ptr<int>();
    workspace.permuted_idx_to_token_idx = permuted_idx_to_token_idx.data_ptr<int>();
    // workspace.expert_weights will be set by derived class after expert_weights allocation
    workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
    workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
    workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();
  }

  void check_moe_common() const {
    // Hidden states [num_tokens, hidden_size]
    TORCH_CHECK(hidden_states->dim() == 2, "hidden_states must be 2D.");
  }

  // MoE computation phase workspace tensors (allocated in prepare_moe() or prepare_moe_common())
  at::Tensor gemm1_output;
  at::Tensor activation_output;
  at::Tensor gemm2_output;
  at::Tensor workspace_fc1;
  at::Tensor workspace_fc2;
  at::Tensor output;
  int64_t moe_tactic{-1};
  std::unique_ptr<tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner> moe_runner;

  void prepare_moe_common(int64_t& moe_tactic) {
    using RunnerType = tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner;
    moe_runner = std::make_unique<RunnerType>(
        this->mDtypeAct, this->mDtypeWeights, args->mUseDeepSeekFp8, (int32_t)tile_tokens_dim,
        static_cast<GatedActType>(this->gated_act_type), this->use_shuffled_weight, this->weight_layout);

    if (moe_tactic == -1) {
      moe_tactic = moe_runner->getDefaultValidConfigIndex(
          args->top_k, args->hidden_size, args->intermediate_size, args->local_num_experts,
          args->num_tokens);
    }
    this->moe_tactic = moe_tactic;

    auto workspace_sizes = moe_runner->getWorkspaceSizeInBytes(*args, moe_tactic);
    workspace_fc1 = at::detail::empty_cuda({std::get<0>(workspace_sizes)}, at::ScalarType::Char,
                                           hidden_states->device(), std::nullopt);
    workspace_fc2 = at::detail::empty_cuda({std::get<1>(workspace_sizes)}, at::ScalarType::Char,
                                           hidden_states->device(), std::nullopt);
    workspace.bmm1_workspace = workspace_fc1.data_ptr();
    workspace.bmm2_workspace = workspace_fc2.data_ptr();
  }

 public:
  virtual void check_routing() const = 0;
  virtual void prepare_routing() = 0;
  virtual void check_moe() const = 0;
  virtual void prepare_moe(int64_t& moe_tactic) = 0;

  // Main entry point for all the executions.
  // Do initializations prior to calling this as the initializations are different for bf16, fp8 and
  // fp4. The executions are non-blocking by default.
  std::vector<at::Tensor> run(int64_t moe_tactic, bool enable_pdl = true) {
    check_routing();
    prepare_routing();

    // Execute routing
    tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
    int routing_device = routing_logits->get_device();
    auto const& routing_stream = at::cuda::getCurrentCUDAStream(routing_device);
    routing_runner.run(
        routing_logits->data_ptr<float>(), args->routing_bias, args->num_tokens, args->num_experts,
        args->top_k, args->n_group, args->topk_group, args->local_expert_offset,
        args->local_num_experts, args->routed_scaling_factor, expert_indexes.data_ptr<int>(),
        expert_count_histogram.data_ptr<int>(), total_num_padded_tokens.data_ptr<int>(),
        expanded_idx_to_permuted_idx.data_ptr<int>(),
        nullptr /*permuted_idx_to_expanded_idx.data_ptr<int>()*/,
        permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
        num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
        cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(),
        args->mDtypeElt, false, true, static_cast<RoutingMethodType>(routing_method_type),
        routing_stream);

    check_moe();
    // if moe_tactic is -1, it will be set to the default valid config index
    prepare_moe(moe_tactic);

    // Execute MoE
    int moe_device = hidden_states->get_device();
    auto const& moe_stream = at::cuda::getCurrentCUDAStream(moe_device);
    moe_runner->run(*args, workspace, moe_device, moe_stream, moe_tactic, enable_pdl);

    if (args->do_finalize) {
      return {output};
    }
    return {gemm2_output, expert_weights, expanded_idx_to_permuted_idx};
  }
};

void FusedMoeLauncher::init_common(
    at::Tensor const* routing_logits, at::Tensor const* routing_bias,
    at::Tensor const* hidden_states, at::Tensor const* gemm1_weights,
    at::Tensor const* gemm2_weights,
    std::unique_ptr<tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs>&& args,
    int64_t tile_tokens_dim, int64_t routing_method_type, bool use_shuffled_weight,
    int64_t weight_layout, int64_t gated_act_type) {
  // Check devicearchitecture: Blackwell (SM 10.x) required
  TORCH_CHECK(hidden_states != nullptr, "hidden_states is required");
  auto device = hidden_states->device().index();
  int major = 0, minor = 0;
  hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
  hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
  TORCH_CHECK(major == 10, "BF16 MoE requires 10.x architecture. Current device has SM ", major,
              minor);
  this->device_version = std::make_tuple(major, minor);

  this->routing_logits = routing_logits;
  this->routing_bias = routing_bias;
  this->hidden_states = hidden_states;
  this->gemm1_weights = gemm1_weights;
  this->gemm2_weights = gemm2_weights;

  args->routing_logits = routing_logits->data_ptr<float>();
  args->routing_bias = routing_bias ? routing_bias->data_ptr() : nullptr;
  args->hidden_states = hidden_states->data_ptr();
  args->gemm1_weights = gemm1_weights->data_ptr();
  args->gemm2_weights = gemm2_weights->data_ptr();

  this->args = std::move(args);
  this->tile_tokens_dim = tile_tokens_dim;
  this->routing_method_type = routing_method_type;
  this->use_shuffled_weight = use_shuffled_weight;
  TORCH_CHECK(0 <= weight_layout && weight_layout <= 2,
              "the value of weight_layout is not recognized");
  this->weight_layout = static_cast<MatrixLayout>(weight_layout);
  TORCH_CHECK(0 <= gated_act_type && gated_act_type <= 1,
              "the value of gated_act_type is not recognized");
  this->gated_act_type = static_cast<GatedActType>(gated_act_type);
}

class Bf16MoeLauncher : public FusedMoeLauncher {
 public:
  Bf16MoeLauncher() = default;

  void init(at::Tensor const& routing_logits, std::optional<at::Tensor> const& routing_bias,
            at::Tensor const& hidden_states, at::Tensor const& gemm1_weights,
            at::Tensor const& gemm2_weights,
            std::unique_ptr<tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs>&& args,
            int64_t tile_tokens_dim, int64_t routing_method_type, bool use_shuffled_weight,
            int64_t weight_layout) {
    constexpr int64_t gated_act_type =
        static_cast<int64_t>(GatedActType::SwiGlu);  // not exposed in api for now

    // Do base class init and perform common checks
    FusedMoeLauncher::init_common(
        &routing_logits, routing_bias.has_value() ? &routing_bias.value() : nullptr, &hidden_states,
        &gemm1_weights, &gemm2_weights, std::move(args), tile_tokens_dim, routing_method_type,
        use_shuffled_weight, weight_layout, gated_act_type);
  }

  void check_routing() const override {
    FusedMoeLauncher::check_routing_common();

    // TODO n_group, topk_group validation?
  }

  void prepare_routing() override {
    FusedMoeLauncher::prepare_routing_common();

    args->mDtypeElt = btg::Dtype::Bfloat16;
    args->mDtypeExpW = btg::Dtype::Bfloat16;
    args->mUseDeepSeekFp8 = false;

    auto const routing_bias_dtype = at::ScalarType::BFloat16;
    expert_weights = at::detail::empty_cuda({args->num_tokens, args->top_k}, routing_bias_dtype,
                                            routing_logits->device(), std::nullopt);

    workspace.expert_weights = expert_weights.data_ptr();
  }

  void check_moe() const override {
    FusedMoeLauncher::check_moe_common();

    TORCH_CHECK(weight_layout == MatrixLayout::BlockMajorK,
                "BF16 Moe: weight_layout must be BlockMajorK");
    check_weights_shape("gemm1");
    check_weights_shape("gemm2");

    TORCH_CHECK(args->intermediate_size % 128 == 0,
                "the second dimension of weights must be a multiple of 128.");
  }

  void prepare_moe(int64_t& moe_tactic) override {
    // in the next line moe_tactic is passed by reference so modification will be propagated back
    // here
    FusedMoeLauncher::prepare_moe_common(moe_tactic);

    int32_t max_num_padded_tokens = workspace.total_max_padded_tokens;
    gemm1_output =
        at::detail::empty_cuda({max_num_padded_tokens, 2 * args->intermediate_size},
                               at::ScalarType::BFloat16, hidden_states->device(), std::nullopt);
    activation_output =
        at::detail::empty_cuda({max_num_padded_tokens, args->intermediate_size},
                               at::ScalarType::BFloat16, hidden_states->device(), std::nullopt);
    gemm2_output =
        at::detail::empty_cuda({max_num_padded_tokens, args->hidden_size}, at::ScalarType::BFloat16,
                               hidden_states->device(), std::nullopt);

    workspace.hidden_states_scale_linear = nullptr;
    workspace.gemm1_output = gemm1_output.data_ptr();
    workspace.gemm1_output_scale = nullptr;  // BF16 doesn't use scale tensors
    workspace.activation_output = activation_output.data_ptr();
    workspace.activation_output_scale = nullptr;  // BF16 doesn't use scale tensors
    workspace.gemm2_output = gemm2_output.data_ptr();
    workspace.gemm2_output_scale = nullptr;

    output = at::detail::empty_cuda({args->num_tokens, args->hidden_size}, at::ScalarType::BFloat16,
                                    hidden_states->device(), std::nullopt);
    args->output = output.data_ptr();
    args->output_scale = nullptr;
  }
};

at::Tensor trtllm_bf16_moe(at::Tensor const& routing_logits,
                           std::optional<at::Tensor> const& routing_bias,
                           at::Tensor const& hidden_states, at::Tensor const& gemm1_weights,
                           at::Tensor const& gemm2_weights, int64_t num_experts, int64_t top_k,
                           int64_t n_group, int64_t topk_group, int64_t intermediate_size,
                           int64_t local_expert_offset, int64_t local_num_experts,
                           int64_t tile_tokens_dim, int64_t routing_method_type,
                           bool use_shuffled_weight, int64_t weight_layout, int64_t moe_tactic,
                           bool enable_pdl) {
  // Just some basic type validation first and leave more checks to the launcher
  TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::Float ||
                  routing_logits.scalar_type() == at::ScalarType::BFloat16,
              "BF16 MoE: routing_logits must be bfoat16 or float.");
  if (routing_bias.has_value()) {
    TORCH_CHECK(routing_bias.value().scalar_type() == at::ScalarType::BFloat16,
                "BF16 MoE: routing_bias must be bfloat16.");
  }
  TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::BFloat16,
              "BF16 MoE: hidden_states must be bfloat16.");
  TORCH_CHECK(gemm1_weights.scalar_type() == at::ScalarType::BFloat16,
              "BF16 MoE: gemm1_weights must be bfloat16.");
  TORCH_CHECK(gemm2_weights.scalar_type() == at::ScalarType::BFloat16,
              "BF16 MoE: gemm2_weights must be bfloat16.");

  // Save params to MoE arguments
  auto args = std::make_unique<tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs>();
  args->num_tokens = hidden_states.sizes()[0];
  args->num_experts = num_experts;
  args->hidden_size = hidden_states.sizes()[1];
  args->hidden_size_output = args->hidden_size;
  args->top_k = top_k;
  args->n_group = n_group;
  args->topk_group = topk_group;
  args->local_expert_offset = local_expert_offset;
  args->local_num_experts = local_num_experts;
  args->intermediate_size = intermediate_size;

  Bf16MoeLauncher launcher;
  launcher.init(routing_logits, routing_bias, hidden_states, gemm1_weights, gemm2_weights,
                std::move(args), tile_tokens_dim, routing_method_type, use_shuffled_weight,
                weight_layout);
  return launcher.run(moe_tactic, enable_pdl)[0];
}

at::Tensor trtllm_fp8_per_tensor_scale_moe_launcher(
    at::Tensor const& routing_logits, std::optional<at::Tensor> routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& gemm1_weights,
    at::Tensor const& output1_scales_scalar, at::Tensor const& output1_scales_gate_scalar,
    at::Tensor const& gemm2_weights, at::Tensor const& output2_scales_scalar,
    int64_t const num_experts, int64_t const top_k, int64_t const n_group, int64_t const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, double const routed_scaling_factor,
    bool const use_routing_scales_on_input, int64_t const tile_tokens_dim,
    int64_t const routing_method_type, bool enable_pdl) {
  auto device = hidden_states.device();

  static const std::tuple<int, int> device_props = [&device] {
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device.index());
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device.index());
    return std::make_tuple(major, minor);
  }();

  TORCH_CHECK(std::get<0>(device_props) == 10,
              "This kernel requires 10.x architecture. Current device has SM ",
              std::get<0>(device_props), std::get<1>(device_props));

  if (use_routing_scales_on_input) {
    TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::BFloat16,
                "routing_logits must be bfloat16.");
  } else {
    TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::Float,
                "routing_logits must be float.");
  }
  TORCH_CHECK(routing_logits.dim() == 2, "routing_logits must be 2D.");
  TORCH_CHECK(routing_logits.sizes()[1] == num_experts, "routing_logits has incorrect shape.");
  if (routing_bias.has_value()) {
    TORCH_CHECK(routing_bias.value().scalar_type() == at::ScalarType::BFloat16,
                "routing_bias must be bfloat16.");
    TORCH_CHECK(routing_bias.value().dim() == 1, "routing_bias must be 1D.");
    TORCH_CHECK(routing_bias.value().sizes()[0] == num_experts,
                "routing_bias has incorrect shape.");
  }

  if (n_group <= 0 || topk_group <= 0) {
    TORCH_CHECK(top_k == 1, "Current routing kernel (no groups) only supports top_k=1.");
  } else {
    TORCH_CHECK(top_k <= 8, "Current routing kernel (with groups) only supports top_k<=8.");
    TORCH_CHECK(topk_group <= 4,
                "Current routing kernel (with groups) only supports topk_group<=4.");
    TORCH_CHECK(topk_group <= n_group, "n_group must not be smaller than topk_group.");
    TORCH_CHECK(num_experts % n_group == 0, "num_experts must be divisible by n_group");
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TORCH_CHECK(top_k < (topk_group * num_experts / n_group),
                "top_k must be less than total number of experts in selected groups");
  }
  TORCH_CHECK(num_experts % 4 == 0,
              "Routing kernel expects that num_experts must be divisible by 4");
  TORCH_CHECK(num_experts > top_k, "num_experts must be greater than top_k");
  TORCH_CHECK(local_num_experts + local_expert_offset <= num_experts,
              "num_experts must be greater or equal to local_num_experts + local_expert_offset");

  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  // Convert PyTorch dtype to TensorRT-LLM dtype
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half) {
    args.mDtypeElt = btg::Dtype::Fp16;
  } else if (dtype == at::ScalarType::BFloat16) {
    args.mDtypeElt = btg::Dtype::Bfloat16;
  } else if (dtype == at::ScalarType::Float8_e4m3fn) {
    args.mDtypeElt = btg::Dtype::E4m3;
  } else {
    TORCH_CHECK(false, "Unsupported input dtype for MoE: ", dtype);
  }

  args.routing_logits = routing_logits.data_ptr();
  auto const routing_bias_dtype =
      routing_bias.has_value() ? routing_bias.value().scalar_type() : at::ScalarType::BFloat16;
  args.routing_bias = routing_bias.has_value() ? routing_bias.value().data_ptr() : nullptr;
  args.hidden_states = hidden_states.data_ptr();
  args.gemm1_weights = gemm1_weights.data_ptr();
  args.output1_scales_scalar = output1_scales_scalar.data_ptr<float>();
  args.output1_scales_gate_scalar = output1_scales_gate_scalar.data_ptr<float>();
  args.gemm2_weights = gemm2_weights.data_ptr();
  args.output2_scales_scalar = output2_scales_scalar.data_ptr<float>();
  args.num_tokens = hidden_states.sizes()[0];
  args.num_experts = num_experts;
  args.hidden_size = hidden_states.sizes()[1];
  args.hidden_size_output = args.hidden_size;
  args.top_k = top_k;
  args.n_group = n_group;
  args.topk_group = topk_group;
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor;
  args.intermediate_size = intermediate_size;
  args.mUseRoutingScalesOnInput = use_routing_scales_on_input;

  // allocate workspace for routing kernel
  at::Tensor num_tokens_per_expert = at::detail::empty_cuda({num_experts}, at::ScalarType::Int,
                                                            routing_logits.device(), std::nullopt);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  at::Tensor total_num_padded_tokens =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));
  at::Tensor expanded_idx_to_permuted_idx = at::detail::empty_cuda(
      {args.num_tokens * args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor permuted_idx_to_token_idx = at::detail::empty_cuda(
      {max_num_padded_tokens}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_weights = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, routing_bias_dtype, routing_logits.device(), std::nullopt);
  at::Tensor expert_indexes = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_count_histogram = at::detail::empty_cuda(
      {2 * 256},
      at::ScalarType::Int,  // 256 is the max number of threads per block and max number of experts
      routing_logits.device(), std::nullopt);

  // allocate workspace for activation/gemm/finalize kernels
  at::Tensor gemm1_output =
      at::detail::empty_cuda({max_num_padded_tokens, 2 * intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor gemm1_output_scale =
      at::detail::empty_cuda({2 * intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor activation_output =
      at::detail::empty_cuda({max_num_padded_tokens, intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor activation_output_scale =
      at::detail::empty_cuda({intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor gemm2_output =
      at::detail::empty_cuda({max_num_padded_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
      args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  at::Tensor cta_idx_xy_to_batch_idx = at::detail::empty_cuda(
      {max_num_ctas}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                             routing_logits.device(), std::nullopt);
  at::Tensor num_non_exiting_ctas =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));

  tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
  auto const& stream = at::cuda::getCurrentCUDAStream(routing_logits.get_device());
  routing_runner.run(routing_logits.data_ptr(), args.routing_bias, args.num_tokens,
                     args.num_experts, args.top_k, args.n_group, args.topk_group,
                     args.local_expert_offset, args.local_num_experts, args.routed_scaling_factor,
                     expert_indexes.data_ptr<int>(), expert_count_histogram.data_ptr<int>(),
                     total_num_padded_tokens.data_ptr<int>(),
                     expanded_idx_to_permuted_idx.data_ptr<int>(),
                     nullptr /*permuted_idx_to_expanded_idx.data_ptr<int>()*/,
                     permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
                     num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
                     cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(),
                     args.mDtypeElt, use_routing_scales_on_input, false /* use_deep_seek_fp8 */,
                     static_cast<RoutingMethodType>(routing_method_type), stream);

  // MoE kernel except routing
  TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "hidden_states must be fp8.");
  TORCH_CHECK(gemm1_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm1_weights must be fp8.");
  TORCH_CHECK(gemm1_weights.dim() == 3, "gemm1_weights must be 3D.");
  TORCH_CHECK(gemm1_weights.sizes()[1] % 2 == 0, "the second dimension of weights must be even.");
  TORCH_CHECK(intermediate_size == gemm1_weights.sizes()[1] / 2,
              "intermediate_size has incorrect shape.");
  TORCH_CHECK(gemm1_weights.sizes()[2] == hidden_states.sizes()[1],
              "the third dimension of weights must be equal to hidden_size.");
  TORCH_CHECK(intermediate_size % 128 == 0,
              "the second dimension of weights must be a multiple of 128.");

  TORCH_CHECK(output1_scales_scalar.scalar_type() == at::ScalarType::Float,
              "output1_scales_scalar must be float.");
  TORCH_CHECK(output1_scales_scalar.dim() == 1, "output1_scales_scalar must be 1D.");
  TORCH_CHECK(output1_scales_scalar.sizes()[0] == local_num_experts,
              "output1_scales_scalar has incorrect dim 0.");
  TORCH_CHECK(output1_scales_gate_scalar.scalar_type() == at::ScalarType::Float,
              "output1_scales_gate_scalar must be float.");
  TORCH_CHECK(output1_scales_gate_scalar.dim() == 1, "output1_scales_gate_scalar must be 1D.");
  TORCH_CHECK(output1_scales_gate_scalar.sizes()[0] == local_num_experts,
              "output1_scales_gate_scalar has incorrect dim 0.");

  TORCH_CHECK(gemm2_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm2_weights must be fp8.");
  TORCH_CHECK(gemm2_weights.dim() == 3, "gemm2_weights must be 3D.");
  TORCH_CHECK(gemm2_weights.sizes()[2] == intermediate_size,
              "the third dimension of weights must be equal to intermediate_size.");

  TORCH_CHECK(output2_scales_scalar.scalar_type() == at::ScalarType::Float,
              "output2_scales_scalar must be float.");
  TORCH_CHECK(output2_scales_scalar.dim() == 1, "output2_scales_scalar must be 1D.");
  TORCH_CHECK(output2_scales_scalar.sizes()[0] == local_num_experts,
              "output2_scales_scalar has incorrect dim 0.");

  // allocate output
  at::Tensor output =
      at::detail::empty_cuda({args.num_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  // setup workspace
  workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = expert_indexes.data_ptr<int>();
  workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
  workspace.expanded_idx_to_permuted_idx =
      expanded_idx_to_permuted_idx.data_ptr<int>();  // Needed by activation/finalize kernels
  workspace.permuted_idx_to_token_idx =
      permuted_idx_to_token_idx.data_ptr<int>();         // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights.data_ptr();  // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
  workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
  workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output.data_ptr();
  workspace.gemm1_output_scale = gemm1_output_scale.data_ptr<float>();
  // activation intermediate ws
  workspace.activation_output = activation_output.data_ptr();
  workspace.activation_output_scale = activation_output_scale.data_ptr<float>();
  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output.data_ptr();
  workspace.gemm2_output_scale = nullptr;
  args.output = output.data_ptr();
  args.output_scale = nullptr;

  tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner moe_runner(
      args.mDtypeElt, args.mUseDeepSeekFp8, tile_tokens_dim, /*useShuffledMatrixA*/ true);

  auto const moeConfigIndex =
      moe_runner.getDefaultValidConfigIndex(args.top_k, args.hidden_size, args.intermediate_size,
                                            args.local_num_experts, args.num_tokens);

  auto workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);
  at::Tensor workspace_fc1 = at::detail::empty_cuda(
      {std::get<0>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  at::Tensor workspace_fc2 = at::detail::empty_cuda(
      {std::get<1>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  workspace.bmm1_workspace = workspace_fc1.data_ptr();
  workspace.bmm2_workspace = workspace_fc2.data_ptr();
  auto const& moe_stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  moe_runner.run(args, workspace, hidden_states.get_device(), moe_stream, moeConfigIndex,
                 enable_pdl);
  return output;
}

at::Tensor trtllm_fp8_per_tensor_scale_moe(
    at::Tensor routing_logits, std::optional<at::Tensor> routing_bias, at::Tensor hidden_states,
    at::Tensor gemm1_weights, at::Tensor output1_scales_scalar,
    at::Tensor output1_scales_gate_scalar, at::Tensor gemm2_weights,
    at::Tensor output2_scales_scalar, int64_t num_experts, int64_t top_k, int64_t n_group,
    int64_t topk_group, int64_t intermediate_size, int64_t local_expert_offset,
    int64_t local_num_experts, double routed_scaling_factor, bool use_routing_scales_on_input,
    int64_t tile_tokens_dim, int64_t routing_method_type, bool enable_pdl) {
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half || dtype == at::ScalarType::BFloat16 ||
      dtype == at::ScalarType::Float8_e4m3fn) {
    //        // Create unified runner for FP8 per-tensor mode
    // using RunnerType = tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner;
    // auto mRunner = std::make_unique<RunnerType>(
    //     btg::Dtype::E4m3, false, tile_tokens_dim, /*useShuffledMatrixA*/ true);

    // auto const moeConfigIndex = mRunner->getDefaultValidConfigIndex(
    //     top_k, hidden_states.sizes()[1], intermediate_size, local_num_experts,
    //     hidden_states.sizes()[0]);

    // // Call unified launcher with nullopt for expert_indices, expert_weights, and output (will be
    // created internally) auto results = trtllm_fp4_block_scale_moe_launcher(
    //     routing_logits, std::nullopt, std::nullopt, routing_bias, hidden_states, std::nullopt,
    //     gemm1_weights, std::nullopt, std::nullopt, std::nullopt, std::nullopt, std::nullopt,
    //     gemm2_weights, std::nullopt, std::nullopt,
    //     output1_scales_scalar, output1_scales_gate_scalar, output2_scales_scalar,
    //     num_experts, top_k, n_group, topk_group, intermediate_size, local_expert_offset,
    //     local_num_experts, routed_scaling_factor, tile_tokens_dim, routing_method_type, true, //
    //     do_finalize = true *mRunner, btg::Dtype::E4m3, btg::Dtype::E4m3, moeConfigIndex,
    //     enable_pdl);

    // return results[0];  // Return the first tensor from the vector
    return trtllm_fp8_per_tensor_scale_moe_launcher(
        routing_logits, routing_bias, hidden_states, gemm1_weights, output1_scales_scalar,
        output1_scales_gate_scalar, gemm2_weights, output2_scales_scalar, num_experts, top_k,
        n_group, topk_group, intermediate_size, local_expert_offset, local_num_experts,
        routed_scaling_factor, use_routing_scales_on_input, tile_tokens_dim, routing_method_type,
        enable_pdl);
  } else {
    TORCH_CHECK(false, "Unsupported input type: ", dtype);
  }
}

at::Tensor trtllm_fp8_block_scale_moe_launcher(
    at::Tensor const& routing_logits, std::optional<at::Tensor> routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale,
    int64_t const num_experts, int64_t const top_k, int64_t const n_group, int64_t const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, double const routed_scaling_factor,
    int64_t const tile_tokens_dim, int64_t const routing_method_type,
    tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner& moe_runner, int64_t moeConfigIndex,
    bool enable_pdl) {
  auto device = hidden_states.device();

  static const std::tuple<int, int> device_props = [&device] {
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device.index());
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device.index());
    return std::make_tuple(major, minor);
  }();

  TORCH_CHECK(std::get<0>(device_props) == 10,
              "This kernel requires 10.x architecture. Current device has SM ",
              std::get<0>(device_props), std::get<1>(device_props));

  TORCH_CHECK(routing_logits.scalar_type() == at::ScalarType::Float,
              "routing_logits must be float.");
  TORCH_CHECK(routing_logits.dim() == 2, "routing_logits must be 2D.");
  TORCH_CHECK(routing_logits.sizes()[0] == hidden_states.sizes()[0],
              "routing_logits and hidden_states must have the same number of tokens.");
  TORCH_CHECK(routing_logits.sizes()[1] == num_experts,
              "routing_logits dim1 must match num_experts.");
  if (routing_bias.has_value()) {
    TORCH_CHECK(routing_bias.value().scalar_type() == at::ScalarType::BFloat16 ||
                    routing_bias.value().scalar_type() == at::ScalarType::Float,
                "routing_bias must be bfloat16 or float.");
    TORCH_CHECK(routing_bias.value().dim() == 1, "routing_bias must be 1D.");
    TORCH_CHECK(routing_bias.value().sizes()[0] == num_experts,
                "routing_bias has incorrect shape.");
  }

  if (n_group <= 0 || topk_group <= 0) {
    TORCH_CHECK(top_k == 1, "Current routing kernel (no groups) only supports top_k=1.");
  } else {
    TORCH_CHECK(top_k <= 8, "Current routing kernel (with groups) only supports top_k<=8.");
    TORCH_CHECK(topk_group <= 4,
                "Current routing kernel (with groups) only supports topk_group<=4.");
    TORCH_CHECK(topk_group <= n_group, "n_group must not be smaller than topk_group.");
    TORCH_CHECK(num_experts % n_group == 0, "num_experts must be divisible by n_group");
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TORCH_CHECK(top_k < (topk_group * num_experts / n_group),
                "top_k must be less than total number of experts in selected groups");
  }
  TORCH_CHECK(num_experts % 4 == 0,
              "Routing kernel expects that num_experts must be divisible by 4");
  TORCH_CHECK(num_experts > top_k, "num_experts must be greater than top_k");

  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  // Convert PyTorch dtype to TensorRT-LLM dtype
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half) {
    args.mDtypeElt = btg::Dtype::Fp16;
  } else if (dtype == at::ScalarType::BFloat16) {
    args.mDtypeElt = btg::Dtype::Bfloat16;
  } else if (dtype == at::ScalarType::Float8_e4m3fn) {
    args.mDtypeElt = btg::Dtype::E4m3;
  } else {
    TORCH_CHECK(false, "Unsupported input dtype for MoE: ", dtype);
  }

  auto const routing_bias_dtype =
      routing_bias.has_value() ? routing_bias.value().scalar_type() : at::ScalarType::BFloat16;
  args.mDtypeExpW =
      routing_bias_dtype == at::ScalarType::BFloat16 ? btg::Dtype::Bfloat16 : btg::Dtype::Fp32;
  args.routing_logits = routing_logits.data_ptr<float>();
  args.routing_bias = routing_bias.has_value() ? routing_bias.value().data_ptr() : nullptr;
  args.hidden_states = hidden_states.data_ptr();
  args.hidden_states_scale = hidden_states_scale.data_ptr<float>();
  args.gemm1_weights = gemm1_weights.data_ptr();
  args.gemm1_weights_scale = gemm1_weights_scale.data_ptr<float>();
  args.gemm2_weights = gemm2_weights.data_ptr();
  args.gemm2_weights_scale = gemm2_weights_scale.data_ptr<float>();
  args.num_tokens = hidden_states.sizes()[0];
  args.num_experts = num_experts;
  args.hidden_size = hidden_states.sizes()[1];
  args.hidden_size_output = args.hidden_size;
  args.top_k = top_k;
  args.n_group = n_group;
  args.topk_group = topk_group;
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor;
  args.intermediate_size = intermediate_size;
  args.mUseDeepSeekFp8 = true;

  // allocate workspace for routing kernel
  at::Tensor num_tokens_per_expert = at::detail::empty_cuda({num_experts}, at::ScalarType::Int,
                                                            routing_logits.device(), std::nullopt);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  at::Tensor total_num_padded_tokens =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));
  at::Tensor expanded_idx_to_permuted_idx = at::detail::empty_cuda(
      {args.num_tokens * args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor permuted_idx_to_token_idx = at::detail::empty_cuda(
      {max_num_padded_tokens}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor expert_weights = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, routing_bias_dtype, routing_logits.device(), std::nullopt);
  at::Tensor expert_indexes = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  int64_t const size_of_expert_count_histogram = std::max(num_experts * 2, int64_t(256 * 2));
  at::Tensor expert_count_histogram = at::detail::empty_cuda(
      {size_of_expert_count_histogram},
      at::ScalarType::Int,  // 256 is the max number of threads per block and max number of experts
      routing_logits.device(), std::nullopt);

  // allocate workspace for activation/gemm/finalize kernels
  at::Tensor gemm1_output =
      at::detail::empty_cuda({max_num_padded_tokens, 2 * intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor gemm1_output_scale =
      at::detail::empty_cuda({2 * intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor activation_output =
      at::detail::empty_cuda({max_num_padded_tokens, intermediate_size},
                             at::ScalarType::Float8_e4m3fn, hidden_states.device(), std::nullopt);
  at::Tensor activation_output_scale =
      at::detail::empty_cuda({intermediate_size / 128, max_num_padded_tokens},
                             at::ScalarType::Float, hidden_states.device(), std::nullopt);
  at::Tensor gemm2_output =
      at::detail::empty_cuda({max_num_padded_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
      args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  at::Tensor cta_idx_xy_to_batch_idx = at::detail::empty_cuda(
      {max_num_ctas}, at::ScalarType::Int, routing_logits.device(), std::nullopt);
  at::Tensor cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                             routing_logits.device(), std::nullopt);
  at::Tensor num_non_exiting_ctas =
      at::empty({}, at::TensorOptions().device(routing_logits.device()).dtype(at::ScalarType::Int));

  tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
  auto const& stream = at::cuda::getCurrentCUDAStream(routing_logits.get_device());
  routing_runner.run(
      routing_logits.data_ptr<float>(), args.routing_bias, args.num_tokens, args.num_experts,
      args.top_k, args.n_group, args.topk_group, args.local_expert_offset, args.local_num_experts,
      args.routed_scaling_factor, expert_indexes.data_ptr<int>(),
      expert_count_histogram.data_ptr<int>(), total_num_padded_tokens.data_ptr<int>(),
      expanded_idx_to_permuted_idx.data_ptr<int>(),
      nullptr /*permuted_idx_to_expanded_idx.data_ptr<int>()*/,
      permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
      num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
      cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(), args.mDtypeElt,
      false, true, static_cast<RoutingMethodType>(routing_method_type), stream);

  // MoE kernel except routing
  TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "hidden_states must be fp8.");
  TORCH_CHECK(hidden_states_scale.scalar_type() == at::ScalarType::Float,
              "hidden_states_scale must be float.");
  TORCH_CHECK(hidden_states_scale.dim() == 2, "hidden_states_scale must be 2D.");
  TORCH_CHECK(hidden_states_scale.sizes()[0] == hidden_states.sizes()[1] / 128,
              "hidden_states_scale dim0 must match hidden_states dim1 / 128.");
  TORCH_CHECK(hidden_states_scale.sizes()[1] == args.num_tokens,
              "hidden_states_scale dim1 must match num_tokens.");
  TORCH_CHECK(gemm1_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm1_weights must be fp8.");

  TORCH_CHECK(gemm1_weights.dim() == 3 || gemm1_weights.dim() == 4,
              "gemm1_weights must be 3D or 4D.");
  {
    int64_t Mn = 0, K = 0;
    if (gemm1_weights.dim() == 3) {
      // MajorK [num_experts, M, K]
      Mn = gemm1_weights.sizes()[1];
      K = gemm1_weights.sizes()[2];
    } else if (gemm1_weights.dim() == 4) {
      // BlockMajorK [num_experts, K/block_k, M, block_k]
      Mn = gemm1_weights.sizes()[2];
      int64_t block_k = gemm1_weights.sizes()[3];
      K = gemm1_weights.sizes()[1] * block_k;
    }
    TORCH_CHECK(Mn % 2 == 0, "the second dimension of weights must be even.");
    TORCH_CHECK(intermediate_size == Mn / 2, "intermediate_size has incorrect shape.");
    TORCH_CHECK(K == hidden_states.sizes()[1],
                "the third dimension of weights must be equal to hidden_size.");
  }
  TORCH_CHECK(gemm1_weights_scale.scalar_type() == at::ScalarType::Float,
              "gemm1_weights_scale must be float.");
  TORCH_CHECK(gemm1_weights_scale.dim() == 3, "gemm1_weights_scale must be 3D.");

  TORCH_CHECK(gemm1_weights_scale.sizes()[0] == local_num_experts,
              "gemm1_weights_scale has incorrect shape.");
  TORCH_CHECK(intermediate_size % 128 == 0,
              "the second dimension of weights must be a multiple of 128.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[1] == 2 * intermediate_size / 128,
              "gemm1_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[2] == args.hidden_size / 128,
              "gemm1_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm2_weights.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm2_weights must be fp8.");

  TORCH_CHECK(gemm2_weights.dim() == 3 || gemm2_weights.dim() == 4,
              "gemm2_weights must be 3D or 4D.");
  {
    int64_t K = 0;
    if (gemm2_weights.dim() == 3) {
      // MajorK [num_experts, M, K]
      K = gemm2_weights.sizes()[2];
    } else if (gemm2_weights.dim() == 4) {
      // BlockMajorK [num_experts, K/block_k, M, block_k]
      int64_t block_k = gemm2_weights.sizes()[3];
      K = gemm2_weights.sizes()[1] * block_k;
    }
    TORCH_CHECK(K == intermediate_size,
                "the third dimension of weights must be equal to intermediate_size.");
  }
  TORCH_CHECK(gemm2_weights_scale.scalar_type() == at::ScalarType::Float,
              "gemm2_weights_scale must be float.");
  TORCH_CHECK(gemm2_weights_scale.dim() == 3, "gemm2_weights_scale must be 3D.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[0] == local_num_experts,
              "gemm2_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[1] == args.hidden_size / 128,
              "gemm2_weights_scale has incorrect shape.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[2] == intermediate_size / 128,
              "gemm2_weights_scale has incorrect shape.");

  // allocate output
  at::Tensor output =
      at::detail::empty_cuda({args.num_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  // setup workspace
  workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = expert_indexes.data_ptr<int>();
  workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
  workspace.expanded_idx_to_permuted_idx =
      expanded_idx_to_permuted_idx.data_ptr<int>();  // Needed by activation/finalize kernels
  workspace.permuted_idx_to_token_idx =
      permuted_idx_to_token_idx.data_ptr<int>();         // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights.data_ptr();  // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
  workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
  workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output.data_ptr();
  workspace.gemm1_output_scale = gemm1_output_scale.data_ptr<float>();
  // activation intermediate ws
  workspace.activation_output = activation_output.data_ptr();
  workspace.activation_output_scale = activation_output_scale.data_ptr<float>();
  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output.data_ptr();
  workspace.gemm2_output_scale = nullptr;
  args.output = output.data_ptr();
  args.output_scale = nullptr;

  auto workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);
  at::Tensor workspace_fc1 = at::detail::empty_cuda(
      {std::get<0>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  at::Tensor workspace_fc2 = at::detail::empty_cuda(
      {std::get<1>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  workspace.bmm1_workspace = workspace_fc1.data_ptr();
  workspace.bmm2_workspace = workspace_fc2.data_ptr();

  auto const& moe_stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  moe_runner.run(args, workspace, hidden_states.get_device(), moe_stream, moeConfigIndex,
                 enable_pdl);
  return output;
}

at::Tensor trtllm_fp8_block_scale_moe(
    at::Tensor const& routing_logits, std::optional<at::Tensor> routing_bias,
    at::Tensor const& hidden_states, at::Tensor const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale, int64_t num_experts,
    int64_t top_k, int64_t n_group, int64_t topk_group, int64_t intermediate_size,
    int64_t local_expert_offset, int64_t local_num_experts, double routed_scaling_factor,
    int64_t tile_tokens_dim, int64_t routing_method_type, bool use_shuffled_weight,
    int64_t weight_layout, bool enable_pdl) {
  auto dtype = hidden_states.dtype();
  if (dtype == at::ScalarType::Half || dtype == at::ScalarType::BFloat16 ||
      dtype == at::ScalarType::Float8_e4m3fn) {
    using RunnerType = tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner;

    btg::Dtype mDtypeElt{btg::Dtype::E4m3};  // FP8 runner so hard-coded
    bool mUseDeepSeekFp8{true};              // Always true for BlockScaleMoe

    TORCH_CHECK(0 <= weight_layout && weight_layout <= 2,
                "the value of weight_layout is not recognized");

    // Properly initialize the runner using make_unique like in the original code
    auto mRunner = std::make_unique<RunnerType>(
        mDtypeElt, mUseDeepSeekFp8, tile_tokens_dim, use_shuffled_weight,
        static_cast<batchedGemm::gemm::MatrixLayout>(weight_layout));

    // Always use fallback config (equivalent to moeConfigIndex == -1 case from original code)
    auto const num_tokens = hidden_states.sizes()[0];
    auto const hidden_size = hidden_states.sizes()[1];

    int64_t moeConfigIndex = mRunner->getDefaultValidConfigIndex(
        top_k, hidden_size, intermediate_size, local_num_experts, num_tokens);

    return trtllm_fp8_block_scale_moe_launcher(
        routing_logits, routing_bias, hidden_states, hidden_states_scale, gemm1_weights,
        gemm1_weights_scale, gemm2_weights, gemm2_weights_scale, num_experts, top_k, n_group,
        topk_group, intermediate_size, local_expert_offset, local_num_experts,
        routed_scaling_factor, tile_tokens_dim, routing_method_type, *mRunner, moeConfigIndex,
        enable_pdl);
  } else {
    TORCH_CHECK(false, "Unsupported input type: ", dtype);
  }
}

// TODO(siyuan): This launcher supports flexible weight and activation types.
// We should cleanup other launchers and only use this one in the future.
std::vector<at::Tensor> trtllm_fp4_block_scale_moe_launcher(
    std::optional<at::Tensor> const& routing_logits, at::Tensor& expert_indices,
    at::Tensor& expert_weights, std::optional<at::Tensor> const& routing_bias,
    at::Tensor const& hidden_states, std::optional<at::Tensor> const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    std::optional<at::Tensor> const& gemm1_bias, std::optional<at::Tensor> const& gemm1_alpha,
    std::optional<at::Tensor> const& gemm1_beta, std::optional<at::Tensor> const& gemm1_clamp_limit,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale,
    std::optional<at::Tensor> const& gemm2_bias,
    std::optional<at::Tensor> const& output1_scales_scalar,
    std::optional<at::Tensor> const& output1_scales_gate_scalar,
    std::optional<at::Tensor> const& output2_scales_scalar, int64_t const num_experts,
    int64_t const top_k, std::optional<int64_t> const n_group,
    std::optional<int64_t> const topk_group, int64_t const intermediate_size,
    int64_t const local_expert_offset, int64_t const local_num_experts,
    std::optional<double> const routed_scaling_factor, int64_t const tile_tokens_dim,
    int64_t const routing_method_type, bool const do_finalize,
    tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner& moe_runner, btg::Dtype dtype_act,
    btg::Dtype dtype_weights, int64_t const moeConfigIndex, bool enable_pdl, at::Tensor& output) {
  auto device = hidden_states.device();

  static const std::tuple<int, int> device_props = [&device] {
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device.index());
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device.index());
    return std::make_tuple(major, minor);
  }();

  TORCH_CHECK(std::get<0>(device_props) == 10,
              "This kernel requires 10.x architecture. Current device has SM ",
              std::get<0>(device_props), std::get<1>(device_props));

  TORCH_CHECK(dtype_act == btg::Dtype::E2m1 || dtype_act == btg::Dtype::Bfloat16 ||
                  dtype_act == btg::Dtype::E4m3 || dtype_act == btg::Dtype::MxE4m3,
              "Only E2m1, Bfloat16, MxE4m3 and E4m3 are supported by block scale MoE");
  if (dtype_act == btg::Dtype::E2m1) {
    TORCH_CHECK(dtype_weights == btg::Dtype::E2m1,
                "Only E2m1 and MxE2m1 are supported by block scale MoE with E2m1 activation");
    TORCH_CHECK(hidden_states_scale.has_value(),
                "hidden_states_scale is required for E2m1 activation");
    TORCH_CHECK(output1_scales_scalar.has_value(),
                "output1_scales_scalar is required for E2m1 activation");
    TORCH_CHECK(output1_scales_gate_scalar.has_value(),
                "output1_scales_gate_scalar is required for E2m1 activation");
    TORCH_CHECK(output2_scales_scalar.has_value(),
                "output2_scales_scalar is required for E2m1 activation");
  } else if (dtype_act == btg::Dtype::Bfloat16 || dtype_act == btg::Dtype::E4m3 ||
             dtype_act == btg::Dtype::MxE4m3) {
    TORCH_CHECK(dtype_weights == btg::Dtype::MxE2m1,
                "Only MxE2m1 weights are supported by block scale MoE with Bfloat16, E4m3 or "
                "MxE4m3 activation");
  } else {
    TORCH_CHECK(false, "Invalid dtype_act");
  }

  if (dtype_act == btg::Dtype::E4m3) {
    TORCH_CHECK(output1_scales_scalar.has_value(),
                "output1_scales_scalar is required for E4m3 activation");
    TORCH_CHECK(output1_scales_gate_scalar.has_value(),
                "output1_scales_gate_scalar is required for E4m3 activation");
    TORCH_CHECK(output2_scales_scalar.has_value(),
                "output2_scales_scalar is required for E4m3 activation");
  }

  if (routing_logits.has_value()) {
    TORCH_CHECK(routing_logits.value().scalar_type() == at::ScalarType::Float ||
                    routing_logits.value().scalar_type() == at::ScalarType::BFloat16,
                "routing_logits must be float or bfloat16.");
    TORCH_CHECK(routing_logits.value().dim() == 2, "routing_logits must be 2D.");
    TORCH_CHECK(routing_logits.value().sizes()[1] == num_experts,
                "routing_logits has incorrect shape.");
  }
  if (routing_bias.has_value()) {
    TORCH_CHECK(routing_bias.value().scalar_type() == at::ScalarType::BFloat16,
                "routing_bias must be bfloat16.");
    TORCH_CHECK(routing_bias.value().dim() == 1, "routing_bias must be 1D.");
    TORCH_CHECK(routing_bias.value().sizes()[0] == num_experts,
                "routing_bias has incorrect shape.");
  }

  if (n_group.value_or(0) != 0) {
    TORCH_CHECK(
        static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::DeepSeekV3,
        "Routing kernel with groups implies DeepSeekV3 routing method.");
    TORCH_CHECK(topk_group.has_value(), "if n_group is given, topk_group must be given");
    TORCH_CHECK(num_experts % n_group.value() == 0, "num_experts must be divisible by n_group");
    TORCH_CHECK(top_k <= 8 && top_k > 0,
                "Current routing kernel (with groups) only supports top_k<=8 && top_k>0.");
    TORCH_CHECK(
        topk_group.value() <= 4 && topk_group.value() > 0,
        "Current routing kernel only (with groups) supports topk_group<=4 && topk_group > 0.");
    TORCH_CHECK(topk_group.value() <= n_group.value(),
                "n_group must not be smaller than topk_group.");
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TORCH_CHECK(top_k < (topk_group.value() * num_experts / n_group.value()),
                "top_k must be less than total number of experts in selected groups");
  } else if (static_cast<RoutingMethodType>(routing_method_type) ==
                 RoutingMethodType::Renormalize ||
             static_cast<RoutingMethodType>(routing_method_type) ==
                 RoutingMethodType::RenormalizeNaive ||
             static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::TopK) {
    TORCH_CHECK(
        top_k <= 8 && top_k > 0,
        "Current routing kernel (no groups, renormalize/topk) only supports top_k<=8 && top_k>0.");
  } else if (static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::Llama4) {
    TORCH_CHECK(top_k == 1, "Current routing kernel (no groups, Llama4) only supports top_k=1.");
  }

  TORCH_CHECK(num_experts % 4 == 0,
              "Routing kernel expects that num_experts must be divisible by 4");
  TORCH_CHECK(num_experts > top_k, "num_experts must be greater than top_k");

  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  // setup args
  // note: the assumption is that output data type is always Bfloat16 (the default)
  auto routing_bias_dtype = at::ScalarType::BFloat16;
  if (routing_bias.has_value()) {
    routing_bias_dtype = routing_bias.value().scalar_type();
  } else if (routing_logits.has_value()) {
    routing_bias_dtype = routing_logits.value().scalar_type();
  }
  args.mDtypeElt = dtype_act;
  args.mDtypeExpW =
      routing_bias_dtype == at::ScalarType::Float ? btg::Dtype::Fp32 : btg::Dtype::Bfloat16;
  args.routing_logits = routing_logits.has_value() ? routing_logits.value().data_ptr() : nullptr;
  args.routing_bias = routing_bias.has_value() ? routing_bias.value().data_ptr() : nullptr;
  args.hidden_states = hidden_states.data_ptr();
  args.hidden_states_scale =
      hidden_states_scale.has_value() ? hidden_states_scale.value().data_ptr() : nullptr;
  args.gemm1_weights = gemm1_weights.data_ptr();
  args.gemm1_weights_scale = gemm1_weights_scale.data_ptr();
  args.gemm1_bias = gemm1_bias.has_value() ? gemm1_bias.value().data_ptr<float>() : nullptr;
  args.gemm1_alpha = gemm1_alpha.has_value() ? gemm1_alpha.value().data_ptr<float>() : nullptr;
  args.gemm1_beta = gemm1_beta.has_value() ? gemm1_beta.value().data_ptr<float>() : nullptr;
  args.gemm1_clamp_limit =
      gemm1_clamp_limit.has_value() ? gemm1_clamp_limit.value().data_ptr<float>() : nullptr;
  args.gemm2_weights = gemm2_weights.data_ptr();
  args.gemm2_weights_scale = gemm2_weights_scale.data_ptr();
  args.gemm2_bias = gemm2_bias.has_value() ? gemm2_bias.value().data_ptr<float>() : nullptr;
  args.num_tokens = hidden_states.sizes()[0];
  args.num_experts = num_experts;
  // * 2 to compensate for the fact that sizeof(hidden_states.dtype) is 1 because we pack 2 e2m1
  // into 1 byte.
  auto const hidden_states_hidden_size =
      dtype_act == btg::Dtype::E2m1 ? hidden_states.sizes()[1] * 2 : hidden_states.sizes()[1];
  args.hidden_size = hidden_states_hidden_size;
  args.hidden_size_output = args.hidden_size;
  args.top_k = top_k;
  args.n_group = n_group.value_or(0);
  args.topk_group = topk_group.value_or(0);
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor.value_or(1.0);
  args.intermediate_size = intermediate_size;

  // allocate workspace for routing kernel
  at::Tensor num_tokens_per_expert = at::detail::empty_cuda({num_experts}, at::ScalarType::Int,
                                                            hidden_states.device(), std::nullopt);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  at::Tensor total_num_padded_tokens =
      at::empty({1}, at::TensorOptions().device(hidden_states.device()).dtype(at::ScalarType::Int));
  at::Tensor expanded_idx_to_permuted_idx = at::detail::empty_cuda(
      {args.num_tokens, args.top_k}, at::ScalarType::Int, hidden_states.device(), std::nullopt);

  at::Tensor permuted_idx_to_token_idx = at::detail::empty_cuda(
      {max_num_padded_tokens}, at::ScalarType::Int, hidden_states.device(), std::nullopt);
  // at::Tensor expert_weights = at::detail::empty_cuda(
  //     {args.num_tokens, args.top_k}, routing_bias_dtype, hidden_states.device(), std::nullopt);
  // at::Tensor expert_indexes = at::detail::empty_cuda(
  //     {args.num_tokens, args.top_k}, at::ScalarType::Int, hidden_states.device(), std::nullopt);
  at::Tensor expert_count_histogram = at::detail::empty_cuda(
      {2 * 256},
      at::ScalarType::Int,  // 256 is the max number of threads per block and max number of experts
      hidden_states.device(), std::nullopt);

  auto const sf_vec_size = dtype_weights == btg::Dtype::MxE2m1 ? 32 : 16;

  // allocate workspace for activation/gemm/finalize kernels
  auto const gemm1_output_hidden =
      dtype_act == btg::Dtype::E2m1 ? intermediate_size / 2 : intermediate_size;
  at::Tensor gemm1_output = at::detail::empty_cuda(
      {max_num_padded_tokens, gemm1_output_hidden},
      dtype_act == btg::Dtype::Bfloat16 ? at::ScalarType::BFloat16 : at::ScalarType::Float8_e4m3fn,
      hidden_states.device(), std::nullopt);

  std::optional<at::Tensor> gemm1_output_scale = std::nullopt;
  if (dtype_act == btg::Dtype::E2m1 || dtype_act == btg::Dtype::MxE4m3) {
    int64_t sf_size = tensorrt_llm::computeSwizzledLayoutSFSize(max_num_padded_tokens,
                                                                intermediate_size / sf_vec_size);
    gemm1_output_scale = at::detail::empty_cuda({sf_size}, at::ScalarType::Float8_e4m3fn,
                                                hidden_states.device(), std::nullopt);
  }

  at::Tensor gemm2_output =
      at::detail::empty_cuda({max_num_padded_tokens, args.hidden_size}, at::ScalarType::BFloat16,
                             hidden_states.device(), std::nullopt);

  int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
      args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  at::Tensor cta_idx_xy_to_batch_idx = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                              hidden_states.device(), std::nullopt);
  at::Tensor cta_idx_xy_to_mn_limit = at::detail::empty_cuda({max_num_ctas}, at::ScalarType::Int,
                                                             hidden_states.device(), std::nullopt);
  at::Tensor num_non_exiting_ctas =
      at::empty({1}, at::TensorOptions().device(hidden_states.device()).dtype(at::ScalarType::Int));

  //
  // TopK routing
  //

  tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
  auto const& stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  routing_runner.run(
      args.routing_logits, args.routing_bias, args.num_tokens, args.num_experts, args.top_k,
      args.n_group, args.topk_group, args.local_expert_offset, args.local_num_experts,
      args.routed_scaling_factor, expert_indices.data_ptr<int>(),
      expert_count_histogram.data_ptr<int>(), total_num_padded_tokens.data_ptr<int>(),
      expanded_idx_to_permuted_idx.data_ptr<int>(),
      nullptr, /*permuted_idx_to_expanded_idx.data_ptr<int>(),*/
      permuted_idx_to_token_idx.data_ptr<int>(), expert_weights.data_ptr(),
      num_tokens_per_expert.data_ptr<int>(), cta_idx_xy_to_batch_idx.data_ptr<int>(),
      cta_idx_xy_to_mn_limit.data_ptr<int>(), num_non_exiting_ctas.data_ptr<int>(), args.mDtypeElt,
      false /* use_routing_scales_on_input */, false /* use_deep_seek_fp8 */,
      static_cast<RoutingMethodType>(routing_method_type), stream);

  //
  // FC13 (gemm1) + FC2 (gemm2)
  //

  if (dtype_act == btg::Dtype::E2m1) {
    TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Byte, "hidden_states must be byte.");
  } else if (dtype_act == btg::Dtype::E4m3 || dtype_act == btg::Dtype::MxE4m3) {
    TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
                "hidden_states must be fp8.");
  } else if (dtype_act == btg::Dtype::Bfloat16) {
    TORCH_CHECK(hidden_states.scalar_type() == at::ScalarType::BFloat16,
                "hidden_states must be bfloat16.");
  } else {
    TORCH_CHECK(false, "Invalid dtype_act");
  }

  if (hidden_states_scale.has_value()) {
    TORCH_CHECK(hidden_states_scale.value().scalar_type() == at::ScalarType::Float8_e4m3fn,
                "hidden_states_scale must be fp8.");

    TORCH_CHECK(
        hidden_states_scale.value().numel() == tensorrt_llm::computeLinearLayoutSFSize(
                                                   args.num_tokens, args.hidden_size / sf_vec_size),
        "hidden_states_scale has incorrect size");
  }

  TORCH_CHECK(gemm1_weights.scalar_type() == torch_ext::FLOAT4_E2M1X2,
              "gemm1_weights must be byte.");

  TORCH_CHECK(gemm1_weights.dim() == 3, "gemm1_weights must be 3D.");
  TORCH_CHECK(gemm1_weights.sizes()[1] % 2 == 0, "the second dimension of weights must be even.");
  TORCH_CHECK(intermediate_size == gemm1_weights.sizes()[1] / 2,
              "intermediate_size has incorrect dim 1.");
  // This check passes even though the actual shape of the weights[2] and hidden_states[1] is
  // 2 times larger due to the fact that 2 e2m1 are packed into 1 byte.
  TORCH_CHECK(
      gemm1_weights.sizes()[2] ==
          (dtype_act == btg::Dtype::E2m1 ? hidden_states.sizes()[1] : hidden_states.sizes()[1] / 2),
      "the third dimension of weights must be equal to hidden_size.");

  TORCH_CHECK(gemm1_weights_scale.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm1_weights_scale must be fp8.");

  TORCH_CHECK(gemm1_weights_scale.dim() == 3, "gemm1_weights_scale must be 3D.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[0] == local_num_experts,
              "gemm1_weights_scale has incorrect dim 0.");
  TORCH_CHECK(intermediate_size % sf_vec_size == 0,
              "the second dimension of weights must be a multiple of ", sf_vec_size);
  TORCH_CHECK(gemm1_weights_scale.sizes()[1] == 2 * intermediate_size,
              "gemm1_weights_scale has incorrect dim 1.");
  TORCH_CHECK(gemm1_weights_scale.sizes()[2] == args.hidden_size / sf_vec_size,
              "gemm1_weights_scale has incorrect dim 2.");

  if (gemm1_bias.has_value()) {
    TORCH_CHECK(gemm1_bias.value().scalar_type() == at::ScalarType::Float,
                "gemm1_bias must be float, got ", c10::toString(gemm1_bias.value().scalar_type()));
    TORCH_CHECK(gemm1_bias.value().dim() == 2, "gemm1_bias must be 2D.");
    TORCH_CHECK(gemm1_bias.value().sizes()[0] == local_num_experts,
                "gemm1_bias has incorrect dim 0.");
    TORCH_CHECK(gemm1_bias.value().sizes()[1] == 2 * intermediate_size,
                "gemm1_bias has incorrect dim 1.");
  }

  if (gemm1_alpha.has_value()) {
    TORCH_CHECK(gemm1_alpha.value().scalar_type() == at::ScalarType::Float,
                "gemm1_alpha must be float, got ",
                c10::toString(gemm1_alpha.value().scalar_type()));
    TORCH_CHECK(gemm1_alpha.value().dim() == 1, "gemm1_alpha must be 1D.");
    TORCH_CHECK(gemm1_alpha.value().sizes()[0] == local_num_experts,
                "gemm1_alpha has incorrect dim 0.");
  }
  if (gemm1_beta.has_value()) {
    TORCH_CHECK(gemm1_beta.value().scalar_type() == at::ScalarType::Float,
                "gemm1_beta must be float, got ", c10::toString(gemm1_beta.value().scalar_type()));
    TORCH_CHECK(gemm1_beta.value().dim() == 1, "gemm1_beta must be 1D.");
    TORCH_CHECK(gemm1_beta.value().sizes()[0] == local_num_experts,
                "gemm1_beta has incorrect dim 0.");
  }

  TORCH_CHECK(gemm2_weights.scalar_type() == torch_ext::FLOAT4_E2M1X2,
              "gemm2_weights must be byte.");

  TORCH_CHECK(gemm2_weights.dim() == 3, "gemm2_weights must be 3D.");
  // / 2 to compensate for the fact that we pack 2 e2m1 into 1 byte.
  TORCH_CHECK(gemm2_weights.sizes()[2] == intermediate_size / 2,
              "the third dimension of weights must be equal to intermediate_size.");

  TORCH_CHECK(gemm2_weights_scale.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "gemm2_weights_scale must be fp8.");

  TORCH_CHECK(gemm2_weights_scale.dim() == 3, "gemm2_weights_scale must be 3D.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[0] == local_num_experts,
              "gemm2_weights_scale has incorrect dim 0.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[1] == args.hidden_size,
              "gemm2_weights_scale has incorrect dim 1.");
  TORCH_CHECK(gemm2_weights_scale.sizes()[2] == intermediate_size / sf_vec_size,
              "gemm2_weights_scale has incorrect dim 2.");

  if (output1_scales_scalar.has_value()) {
    TORCH_CHECK(output1_scales_scalar.value().scalar_type() == at::ScalarType::Float,
                "output1_scales_scalar must be float.");
    TORCH_CHECK(output1_scales_scalar.value().dim() == 1, "output1_scales_scalar must be 1D.");
    TORCH_CHECK(output1_scales_scalar.value().sizes()[0] == local_num_experts,
                "output1_scales_scalar has incorrect dim 0.");
  }

  if (output1_scales_gate_scalar.has_value()) {
    TORCH_CHECK(output1_scales_gate_scalar.value().scalar_type() == at::ScalarType::Float,
                "output1_scales_gate_scalar must be float.");
    TORCH_CHECK(output1_scales_gate_scalar.value().dim() == 1,
                "output1_scales_gate_scalar must be 1D.");
    TORCH_CHECK(output1_scales_gate_scalar.value().sizes()[0] == local_num_experts,
                "output1_scales_gate_scalar has incorrect dim 0.");
  }

  if (output2_scales_scalar.has_value()) {
    TORCH_CHECK(output2_scales_scalar.value().scalar_type() == at::ScalarType::Float,
                "output2_scales_scalar must be float.");
    TORCH_CHECK(output2_scales_scalar.value().dim() == 1, "output2_scales_scalar must be 1D.");
    TORCH_CHECK(output2_scales_scalar.value().sizes()[0] == local_num_experts,
                "output2_scales_scalar has incorrect dim 0.");
  }

  // setup workspace
  workspace.total_num_padded_tokens = total_num_padded_tokens.data_ptr<int>();
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = expert_indices.data_ptr<int>();
  workspace.permuted_idx_size = total_num_padded_tokens.data_ptr<int>();
  workspace.expanded_idx_to_permuted_idx =
      expanded_idx_to_permuted_idx.data_ptr<int>();  // Needed by permute/finalize kernels
  workspace.permuted_idx_to_token_idx =
      permuted_idx_to_token_idx.data_ptr<int>();         // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights.data_ptr();  // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = cta_idx_xy_to_batch_idx.data_ptr<int>();
  workspace.cta_idx_xy_to_mn_limit = cta_idx_xy_to_mn_limit.data_ptr<int>();
  workspace.num_non_exiting_ctas = num_non_exiting_ctas.data_ptr<int>();

  workspace.hidden_states_scale_linear = nullptr;

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output.data_ptr();
  workspace.gemm1_output_scale =
      gemm1_output_scale.has_value()
          ? reinterpret_cast<float*>(gemm1_output_scale.value().data_ptr())
          : nullptr;

  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output.data_ptr();
  workspace.gemm2_output_scale = nullptr;
  args.output = output.data_ptr();
  args.output_scale = nullptr;
  args.output1_scales_scalar =
      output1_scales_scalar.has_value() ? output1_scales_scalar.value().data_ptr<float>() : nullptr;
  args.output1_scales_gate_scalar = output1_scales_gate_scalar.has_value()
                                        ? output1_scales_gate_scalar.value().data_ptr<float>()
                                        : nullptr;
  args.output2_scales_scalar =
      output2_scales_scalar.has_value() ? output2_scales_scalar.value().data_ptr<float>() : nullptr;
  args.do_finalize = do_finalize;

  auto const workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);

  at::Tensor workspace_fc1 = at::detail::empty_cuda(
      {std::get<0>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  at::Tensor workspace_fc2 = at::detail::empty_cuda(
      {std::get<1>(workspace_sizes)}, at::ScalarType::Char, hidden_states.device(), std::nullopt);
  workspace.bmm1_workspace = workspace_fc1.data_ptr();
  workspace.bmm2_workspace = workspace_fc2.data_ptr();
  auto const& moe_stream = at::cuda::getCurrentCUDAStream(hidden_states.get_device());
  moe_runner.run(args, workspace, hidden_states.get_device(), moe_stream, moeConfigIndex,
                 enable_pdl);

  if (!do_finalize) {
    return {gemm2_output, expert_weights, expanded_idx_to_permuted_idx};
  }
  return {output};
}

std::vector<at::Tensor> trtllm_fp4_block_scale_moe(
    std::optional<at::Tensor> const& routing_logits, at::Tensor& topk_ids,
    at::Tensor& expert_weights, std::optional<at::Tensor> const& routing_bias,
    at::Tensor const& hidden_states, std::optional<at::Tensor> const& hidden_states_scale,
    at::Tensor const& gemm1_weights, at::Tensor const& gemm1_weights_scale,
    std::optional<at::Tensor> const& gemm1_bias, std::optional<at::Tensor> const& gemm1_alpha,
    std::optional<at::Tensor> const& gemm1_beta, std::optional<at::Tensor> const& gemm1_clamp_limit,
    at::Tensor const& gemm2_weights, at::Tensor const& gemm2_weights_scale,
    std::optional<at::Tensor> const& gemm2_bias,
    std::optional<at::Tensor> const& output1_scales_scalar,
    std::optional<at::Tensor> const& output1_scales_gate_scalar,
    std::optional<at::Tensor> const& output2_scales_scalar, int64_t num_experts, int64_t top_k,
    std::optional<int64_t> n_group, std::optional<int64_t> topk_group, int64_t intermediate_size,
    int64_t local_expert_offset, int64_t local_num_experts,
    std::optional<double> routed_scaling_factor, int64_t tile_tokens_dim,
    int64_t routing_method_type, bool do_finalize, bool enable_pdl, int64_t gated_act_type,
    at::Tensor& output, int64_t config_index) {
  using RunnerType = tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner;

  int const num_tokens = hidden_states.sizes()[0];
  int hidden_size = hidden_states.sizes()[1];
  if (hidden_states.scalar_type() == torch_ext::FLOAT4_E2M1X2) hidden_size *= 2;
  int hidden_states_scale_vec_size = -1;
  if (hidden_states_scale.has_value()) {
    hidden_states_scale_vec_size = (num_tokens * hidden_size) / hidden_states_scale.value().numel();
  }
  int weight_scale_vec_size =
      (local_num_experts * intermediate_size * 2 * hidden_size) / gemm1_weights_scale.numel();
  TORCH_CHECK(weight_scale_vec_size == 16 || weight_scale_vec_size == 32,
              "unsupported weight_scale_vec_size.");
  auto mDtypeWeights = weight_scale_vec_size == 16 ? btg::Dtype::E2m1 : btg::Dtype::MxE2m1;

  TORCH_CHECK(gemm1_weights.scalar_type() == at::ScalarType::Byte &&
                  gemm2_weights.scalar_type() == at::ScalarType::Byte,
              "weights must be fp4 packed in uint8.");
  TORCH_CHECK(hidden_states.scalar_type() == torch_ext::FLOAT4_E2M1X2 ||
                  hidden_states.scalar_type() == at::ScalarType::BFloat16 ||
                  hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn,
              "hidden_states must be bf16, fp8 or uint8 (packed fp4).");
  auto mDtypeAct = btg::Dtype::Bfloat16;
  if (hidden_states.scalar_type() == torch_ext::FLOAT4_E2M1X2) {
    TORCH_CHECK(hidden_states_scale.has_value() &&
                    hidden_states_scale.value().scalar_type() == at::ScalarType::Float8_e4m3fn,
                "hidden_states_scale must be provided for fp4 activation.");
    if (hidden_states_scale_vec_size == 16) {
      mDtypeAct = btg::Dtype::E2m1;
    } else if (hidden_states_scale_vec_size == 32) {
      mDtypeAct = btg::Dtype::MxE2m1;
    } else {
      TORCH_CHECK(false, "unsupported hidden_states_scale shape.");
    }
  } else if (hidden_states.scalar_type() == at::ScalarType::Float8_e4m3fn) {
    if (hidden_states_scale.has_value()) {
      if (hidden_states_scale_vec_size == 32) {
        mDtypeAct = btg::Dtype::MxE4m3;
      } else {
        TORCH_CHECK(false, "unsupported hidden_states_scale shape.");
      }
    } else {
      mDtypeAct = btg::Dtype::E4m3;
    }
  }
  bool mUseDeepSeekFp8{false};  // FP4 doesn't use DeepSeek FP8

  // Properly initialize the runner using make_unique like in the original code
  auto mRunner = std::make_unique<RunnerType>(
      mDtypeAct, mDtypeWeights, mUseDeepSeekFp8, (int32_t)tile_tokens_dim,
      static_cast<GatedActType>(gated_act_type), /*useShuffledMatrixA*/ true);

  if (config_index == -1) {
    config_index = mRunner->getDefaultValidConfigIndex(top_k, hidden_size, intermediate_size,
                                                       local_num_experts, num_tokens);
  }

  return trtllm_fp4_block_scale_moe_launcher(
      routing_logits, topk_ids, expert_weights, routing_bias, hidden_states, hidden_states_scale,
      gemm1_weights, gemm1_weights_scale, gemm1_bias, gemm1_alpha, gemm1_beta, gemm1_clamp_limit,
      gemm2_weights, gemm2_weights_scale, gemm2_bias, output1_scales_scalar,
      output1_scales_gate_scalar, output2_scales_scalar, num_experts, top_k, n_group, topk_group,
      intermediate_size, local_expert_offset, local_num_experts, routed_scaling_factor,
      tile_tokens_dim, routing_method_type, do_finalize, *mRunner, mDtypeAct, mDtypeWeights,
      config_index, enable_pdl, output);
}

int64_t trtllm_get_default_moe_configs(int64_t const tile_tokens_dim, int64_t const dtype_act_,
                                       int64_t const dtype_weights_, bool const useDeepSeekFp8,
                                       int64_t const top_k, int64_t const hidden_size,
                                       int64_t const intermediate_size,
                                       int64_t const num_local_experts,
                                       int64_t const gated_act_type, int64_t const num_tokens) {
  auto dtype_act = static_cast<btg::Dtype>(dtype_act_);
  auto dtype_weights = static_cast<btg::Dtype>(dtype_weights_);
  tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner moe_runner(
      dtype_act, dtype_weights, useDeepSeekFp8, (int32_t)tile_tokens_dim,
      static_cast<GatedActType>(gated_act_type), /*useShuffledMatrixA*/ true);
  return moe_runner.getDefaultValidConfigIndex(top_k, hidden_size, intermediate_size,
                                               num_local_experts, num_tokens);
}

std::vector<int64_t> trtllm_get_valid_moe_configs(
    int64_t const tile_tokens_dim, int64_t const dtype_act_, int64_t const dtype_weights_,
    bool const useDeepSeekFp8, int64_t const top_k, int64_t const hidden_size,
    int64_t const intermediate_size, int64_t const num_local_experts, int64_t const gated_act_type,
    int64_t const num_tokens) {
  auto dtype_act = static_cast<btg::Dtype>(dtype_act_);
  auto dtype_weights = static_cast<btg::Dtype>(dtype_weights_);
  tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner moe_runner(
      dtype_act, dtype_weights, useDeepSeekFp8, (int32_t)tile_tokens_dim,
      static_cast<GatedActType>(gated_act_type), /*useShuffledMatrixA*/ true);
  return moe_runner.getValidConfigIndices(top_k, hidden_size, intermediate_size, num_local_experts,
                                          num_tokens);
}

namespace trtllm_cubin_loader {
#include <flashinfer/cubin_loader.h>
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_bf16_moe", trtllm_bf16_moe);
  m.def("trtllm_fp8_per_tensor_scale_moe", trtllm_fp8_per_tensor_scale_moe);
  m.def("trtllm_fp8_block_scale_moe", trtllm_fp8_block_scale_moe);
  m.def("trtllm_fp4_block_scale_moe", trtllm_fp4_block_scale_moe);
  m.def("trtllm_get_default_moe_configs", trtllm_get_default_moe_configs);
  m.def("trtllm_get_valid_moe_configs", trtllm_get_valid_moe_configs);
}

}  // namespace flashinfer
