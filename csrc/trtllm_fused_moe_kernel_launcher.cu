#include "hip/hip_runtime.h"
/*
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hip/hip_runtime.h>
#include <flashinfer/exception.h>
#include <hip/hiprtc.h>

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include "flashinfer/trtllm/batched_gemm/trtllmGen_bmm_export/GemmGatedActOptions.h"
#include "flashinfer/trtllm/batched_gemm/trtllmGen_bmm_export/trtllm/gen/DtypeDecl.h"
#include "flashinfer/trtllm/fused_moe/DevKernel.h"
#include "flashinfer/trtllm/fused_moe/RoutingKernel.h"
#include "flashinfer/trtllm/fused_moe/runner.h"
#include "nv_internal/tensorrt_llm/kernels/quantization.h"
#include "nv_internal/tensorrt_llm/thop/utils.h"
#include "tvm_ffi_utils.h"

namespace flashinfer {

namespace btg = batchedGemm::trtllm::gen;
using tensorrt_llm::kernels::trtllmgen_moe::MoE::GatedActType;
using tensorrt_llm::kernels::trtllmgen_moe::Routing::RoutingMethodType;
using tvm::ffi::Array;
using tvm::ffi::Optional;

Tensor trtllm_fp8_per_tensor_scale_moe_launcher(
    Tensor routing_logits, Optional<Tensor> routing_bias, Tensor hidden_states,
    Tensor gemm1_weights, Tensor output1_scales_scalar, Tensor output1_scales_gate_scalar,
    Tensor gemm2_weights, Tensor output2_scales_scalar, int64_t const num_experts,
    int64_t const top_k, int64_t const n_group, int64_t const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, double const routed_scaling_factor,
    bool const use_routing_scales_on_input, int64_t const tile_tokens_dim,
    int64_t const routing_method_type, bool enable_pdl) {
  static const std::tuple<int, int> device_props = [hidden_states] {
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor,
                           hidden_states->device.device_id);
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor,
                           hidden_states->device.device_id);
    return std::make_tuple(major, minor);
  }();

  TVM_FFI_ICHECK_EQ(std::get<0>(device_props), 10)
      << "This kernel requires 10.x architecture. Current device has SM "
      << std::get<0>(device_props) << std::get<1>(device_props);

  if (use_routing_scales_on_input) {
    TVM_FFI_ICHECK_EQ(routing_logits->dtype, dl_bfloat16) << "routing_logits must be bfloat16.";
  } else {
    TVM_FFI_ICHECK_EQ(routing_logits->dtype, dl_float32) << "routing_logits must be float.";
  }
  TVM_FFI_ICHECK_EQ(routing_logits->ndim, 2) << "routing_logits must be 2D.";
  TVM_FFI_ICHECK_EQ(routing_logits->shape[1], num_experts) << "routing_logits has incorrect shape.";
  if (routing_bias.has_value()) {
    TVM_FFI_ICHECK_EQ(routing_bias.value()->dtype, dl_bfloat16) << "routing_bias must be bfloat16.";
    TVM_FFI_ICHECK_EQ(routing_bias.value()->ndim, 1) << "routing_bias must be 1D.";
    TVM_FFI_ICHECK_EQ(routing_bias.value()->shape[0], num_experts)
        << "routing_bias has incorrect shape.";
  }

  if (n_group <= 0 || topk_group <= 0) {
    TVM_FFI_ICHECK_EQ(top_k, 1) << "Current routing kernel (no groups) only supports top_k=1.";
  } else {
    TVM_FFI_ICHECK_LE(top_k, 8) << "Current routing kernel (with groups) only supports top_k<=8.";
    TVM_FFI_ICHECK_LE(topk_group, 4)
        << "Current routing kernel (with groups) only supports topk_group<=4.";
    TVM_FFI_ICHECK_LE(topk_group, n_group) << "n_group must not be smaller than topk_group.";
    TVM_FFI_ICHECK_EQ(num_experts % n_group, 0) << "num_experts must be divisible by n_group";
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TVM_FFI_ICHECK_LT(top_k, (topk_group * num_experts / n_group))
        << "top_k must be less than total number of experts in selected groups";
  }
  TVM_FFI_ICHECK_EQ(num_experts % 4, 0)
      << "Routing kernel expects that num_experts must be divisible by 4";
  TVM_FFI_ICHECK_GT(num_experts, top_k) << "num_experts must be greater than top_k";
  TVM_FFI_ICHECK_LE(local_num_experts + local_expert_offset, num_experts)
      << "num_experts must be greater or equal to local_num_experts + local_expert_offset";

  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  // Convert PyTorch dtype to TensorRT-LLM dtype
  auto dtype = hidden_states->dtype;
  if (dtype == dl_float16) {
    args.mDtypeElt = btg::Dtype::Fp16;
  } else if (dtype == dl_bfloat16) {
    args.mDtypeElt = btg::Dtype::Bfloat16;
  } else if (dtype == dl_float8_e4m3fn) {
    args.mDtypeElt = btg::Dtype::E4m3;
  } else {
    TVM_FFI_LOG_AND_THROW(NotImplementedError) << "Unsupported input dtype for MoE.";
  }

  args.routing_logits = routing_logits->data;
  auto const routing_bias_dtype =
      routing_bias.has_value() ? routing_bias.value()->dtype : dl_bfloat16;
  args.routing_bias = routing_bias.has_value() ? routing_bias.value()->data : nullptr;
  args.hidden_states = hidden_states->data;
  args.gemm1_weights = gemm1_weights->data;
  args.output1_scales_scalar = static_cast<float*>(output1_scales_scalar->data);
  args.output1_scales_gate_scalar = static_cast<float*>(output1_scales_gate_scalar->data);
  args.gemm2_weights = gemm2_weights->data;
  args.output2_scales_scalar = static_cast<float*>(output2_scales_scalar->data);
  args.num_tokens = hidden_states->shape[0];
  args.num_experts = num_experts;
  args.hidden_size = hidden_states->shape[1];
  args.hidden_size_output = args.hidden_size;
  args.top_k = top_k;
  args.n_group = n_group;
  args.topk_group = topk_group;
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor;
  args.intermediate_size = intermediate_size;
  args.mUseRoutingScalesOnInput = use_routing_scales_on_input;

  // allocate workspace for routing kernel
  Tensor num_tokens_per_expert = alloc_tensor({num_experts}, dl_int32, routing_logits->device);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  Tensor total_num_padded_tokens = alloc_tensor({1}, dl_int32, routing_logits->device);
  Tensor expanded_idx_to_permuted_idx =
      alloc_tensor({args.num_tokens * args.top_k}, dl_int32, routing_logits->device);
  Tensor permuted_idx_to_token_idx =
      alloc_tensor({max_num_padded_tokens}, dl_int32, routing_logits->device);
  Tensor expert_weights =
      alloc_tensor({args.num_tokens, args.top_k}, routing_bias_dtype, routing_logits->device);
  Tensor expert_indexes =
      alloc_tensor({args.num_tokens, args.top_k}, dl_int32, routing_logits->device);
  Tensor expert_count_histogram = alloc_tensor(
      {2 * 256},
      dl_int32,  // 256 is the max number of threads per block and max number of experts
      routing_logits->device);

  // allocate workspace for activation/gemm/finalize kernels
  // Tensor gemm1_output = alloc_tensor({max_num_padded_tokens, 2 * intermediate_size},
  //                                    dl_float8_e4m3fn, hidden_states->device);
  // Tensor activation_output = alloc_tensor({max_num_padded_tokens, intermediate_size},
  //                                         dl_float8_e4m3fn, hidden_states->device);
  Tensor gemm1_output =
      alloc_tensor({max_num_padded_tokens, 2 * intermediate_size}, dl_uint8, hidden_states->device);
  Tensor gemm1_output_scale = alloc_tensor({2 * intermediate_size / 128, max_num_padded_tokens},
                                           dl_float32, hidden_states->device);
  Tensor activation_output =
      alloc_tensor({max_num_padded_tokens, intermediate_size}, dl_uint8, hidden_states->device);
  Tensor activation_output_scale = alloc_tensor({intermediate_size / 128, max_num_padded_tokens},
                                                dl_float32, hidden_states->device);
  Tensor gemm2_output =
      alloc_tensor({max_num_padded_tokens, args.hidden_size}, dl_bfloat16, hidden_states->device);

  int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
      args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  Tensor cta_idx_xy_to_batch_idx = alloc_tensor({max_num_ctas}, dl_int32, routing_logits->device);
  Tensor cta_idx_xy_to_mn_limit = alloc_tensor({max_num_ctas}, dl_int32, routing_logits->device);
  Tensor num_non_exiting_ctas = alloc_tensor({1}, dl_int32, routing_logits->device);

  tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
  hipStream_t stream = get_stream(routing_logits->device);
  routing_runner.run(
      routing_logits->data, args.routing_bias, args.num_tokens, args.num_experts, args.top_k,
      args.n_group, args.topk_group, args.local_expert_offset, args.local_num_experts,
      args.routed_scaling_factor, static_cast<int*>(expert_indexes->data),
      static_cast<int*>(expert_count_histogram->data),
      static_cast<int*>(total_num_padded_tokens->data),
      static_cast<int*>(expanded_idx_to_permuted_idx->data),
      nullptr /*static_cast<int*>(permuted_idx_to_expanded_idx->data)*/,
      static_cast<int*>(permuted_idx_to_token_idx->data), expert_weights->data,
      static_cast<int*>(num_tokens_per_expert->data),
      static_cast<int*>(cta_idx_xy_to_batch_idx->data),
      static_cast<int*>(cta_idx_xy_to_mn_limit->data),
      static_cast<int*>(num_non_exiting_ctas->data), args.mDtypeElt, use_routing_scales_on_input,
      false /* use_deep_seek_fp8 */, static_cast<RoutingMethodType>(routing_method_type), stream);

  // MoE kernel except routing
  TVM_FFI_ICHECK_EQ(hidden_states->dtype, dl_float8_e4m3fn) << "hidden_states must be fp8.";
  TVM_FFI_ICHECK_EQ(gemm1_weights->dtype, dl_float8_e4m3fn) << "gemm1_weights must be fp8.";
  TVM_FFI_ICHECK_EQ(gemm1_weights->ndim, 3) << "gemm1_weights must be 3D.";
  TVM_FFI_ICHECK_EQ(gemm1_weights->shape[1] % 2, 0)
      << "the second dimension of weights must be even.";
  TVM_FFI_ICHECK_EQ(intermediate_size, gemm1_weights->shape[1] / 2)
      << "intermediate_size has incorrect shape.";
  TVM_FFI_ICHECK_EQ(gemm1_weights->shape[2], hidden_states->shape[1])
      << "the third dimension of weights must be equal to hidden_size.";
  TVM_FFI_ICHECK_EQ(intermediate_size % 128, 0)
      << "the second dimension of weights must be a multiple of 128.";

  TVM_FFI_ICHECK_EQ(output1_scales_scalar->dtype, dl_float32)
      << "output1_scales_scalar must be float.";
  TVM_FFI_ICHECK_EQ(output1_scales_scalar->ndim, 1) << "output1_scales_scalar must be 1D.";
  TVM_FFI_ICHECK_EQ(output1_scales_scalar->shape[0], local_num_experts)
      << "output1_scales_scalar has incorrect dim 0.";
  TVM_FFI_ICHECK_EQ(output1_scales_gate_scalar->dtype, dl_float32)
      << "output1_scales_gate_scalar must be float.";
  TVM_FFI_ICHECK_EQ(output1_scales_gate_scalar->ndim, 1)
      << "output1_scales_gate_scalar must be 1D.";
  TVM_FFI_ICHECK_EQ(output1_scales_gate_scalar->shape[0], local_num_experts)
      << "output1_scales_gate_scalar has incorrect dim 0.";

  TVM_FFI_ICHECK_EQ(gemm2_weights->dtype, dl_float8_e4m3fn) << "gemm2_weights must be fp8.";
  TVM_FFI_ICHECK_EQ(gemm2_weights->ndim, 3) << "gemm2_weights must be 3D.";
  TVM_FFI_ICHECK_EQ(gemm2_weights->shape[2], intermediate_size)
      << "the third dimension of weights must be equal to intermediate_size.";

  TVM_FFI_ICHECK_EQ(output2_scales_scalar->dtype, dl_float32)
      << "output2_scales_scalar must be float.";
  TVM_FFI_ICHECK_EQ(output2_scales_scalar->ndim, 1) << "output2_scales_scalar must be 1D.";
  TVM_FFI_ICHECK_EQ(output2_scales_scalar->shape[0], local_num_experts)
      << "output2_scales_scalar has incorrect dim 0.";

  // allocate output
  Tensor output =
      alloc_tensor({args.num_tokens, args.hidden_size}, dl_bfloat16, hidden_states->device);

  // setup workspace
  workspace.total_num_padded_tokens = static_cast<int*>(total_num_padded_tokens->data);
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = static_cast<int*>(expert_indexes->data);
  workspace.permuted_idx_size = static_cast<int*>(total_num_padded_tokens->data);
  workspace.expanded_idx_to_permuted_idx = static_cast<int*>(
      expanded_idx_to_permuted_idx->data);  // Needed by activation/finalize kernels
  workspace.permuted_idx_to_token_idx =
      static_cast<int*>(permuted_idx_to_token_idx->data);  // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights->data;         // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = static_cast<int*>(cta_idx_xy_to_batch_idx->data);
  workspace.cta_idx_xy_to_mn_limit = static_cast<int*>(cta_idx_xy_to_mn_limit->data);
  workspace.num_non_exiting_ctas = static_cast<int*>(num_non_exiting_ctas->data);

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output->data;
  workspace.gemm1_output_scale = static_cast<float*>(gemm1_output_scale->data);
  // activation intermediate ws
  workspace.activation_output = activation_output->data;
  workspace.activation_output_scale = static_cast<float*>(activation_output_scale->data);
  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output->data;
  workspace.gemm2_output_scale = nullptr;
  args.output = output->data;
  args.output_scale = nullptr;

  tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner moe_runner(
      args.mDtypeElt, args.mUseDeepSeekFp8, tile_tokens_dim, /*useShuffledMatrixA*/ true);

  auto const moeConfigIndex =
      moe_runner.getDefaultValidConfigIndex(args.top_k, args.hidden_size, args.intermediate_size,
                                            args.local_num_experts, args.num_tokens);

  auto workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);
  Tensor workspace_fc1 =
      alloc_tensor({std::get<0>(workspace_sizes)}, dl_int8, hidden_states->device);
  Tensor workspace_fc2 =
      alloc_tensor({std::get<1>(workspace_sizes)}, dl_int8, hidden_states->device);
  workspace.bmm1_workspace = workspace_fc1->data;
  workspace.bmm2_workspace = workspace_fc2->data;
  hipStream_t moe_stream = get_stream(hidden_states->device);
  moe_runner.run(args, workspace, hidden_states->device.device_id, moe_stream, moeConfigIndex,
                 enable_pdl);
  return output;
}

Tensor trtllm_fp8_per_tensor_scale_moe(
    Tensor routing_logits, Optional<Tensor> routing_bias, Tensor hidden_states,
    Tensor gemm1_weights, Tensor output1_scales_scalar, Tensor output1_scales_gate_scalar,
    Tensor gemm2_weights, Tensor output2_scales_scalar, int64_t num_experts, int64_t top_k,
    int64_t n_group, int64_t topk_group, int64_t intermediate_size, int64_t local_expert_offset,
    int64_t local_num_experts, double routed_scaling_factor, bool use_routing_scales_on_input,
    int64_t tile_tokens_dim, int64_t routing_method_type, bool enable_pdl) {
  auto dtype = hidden_states->dtype;
  if (dtype == dl_float16 || dtype == dl_bfloat16 || dtype == dl_float8_e4m3fn) {
    return trtllm_fp8_per_tensor_scale_moe_launcher(
        routing_logits, routing_bias, hidden_states, gemm1_weights, output1_scales_scalar,
        output1_scales_gate_scalar, gemm2_weights, output2_scales_scalar, num_experts, top_k,
        n_group, topk_group, intermediate_size, local_expert_offset, local_num_experts,
        routed_scaling_factor, use_routing_scales_on_input, tile_tokens_dim, routing_method_type,
        enable_pdl);
  } else {
    TVM_FFI_LOG_AND_THROW(NotImplementedError) << "Unsupported input dtype.";
  }
}

void trtllm_fp8_block_scale_moe_launcher(
    Tensor routing_logits, Optional<Tensor> routing_bias, Tensor hidden_states,
    Tensor hidden_states_scale, Tensor gemm1_weights, Tensor gemm1_weights_scale,
    Tensor gemm2_weights, Tensor gemm2_weights_scale, Tensor output, int64_t const num_experts,
    int64_t const top_k, int64_t const n_group, int64_t const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, double const routed_scaling_factor,
    int64_t const tile_tokens_dim, int64_t const routing_method_type,
    tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner& moe_runner, int64_t moeConfigIndex,
    bool enable_pdl) {
  static const std::tuple<int, int> device_props = [hidden_states] {
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor,
                           hidden_states->device.device_id);
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor,
                           hidden_states->device.device_id);
    return std::make_tuple(major, minor);
  }();

  TVM_FFI_ICHECK_EQ(std::get<0>(device_props), 10)
      << "This kernel requires 10.x architecture. Current device has SM "
      << std::get<0>(device_props) << std::get<1>(device_props);

  TVM_FFI_ICHECK_EQ(routing_logits->dtype, dl_float32) << "routing_logits must be float.";
  TVM_FFI_ICHECK_EQ(routing_logits->ndim, 2) << "routing_logits must be 2D.";
  TVM_FFI_ICHECK_EQ(routing_logits->shape[0], hidden_states->shape[0])
      << "routing_logits and hidden_states must have the same number of tokens.";
  TVM_FFI_ICHECK_EQ(routing_logits->shape[1], num_experts)
      << "routing_logits dim1 must match num_experts.";
  if (routing_bias.has_value()) {
    TVM_FFI_ICHECK(routing_bias.value()->dtype == dl_bfloat16 ||
                   routing_bias.value()->dtype == dl_float32)
        << "routing_bias must be bfloat16 or float.";
    TVM_FFI_ICHECK_EQ(routing_bias.value()->ndim, 1) << "routing_bias must be 1D.";
    TVM_FFI_ICHECK_EQ(routing_bias.value()->shape[0], num_experts)
        << "routing_bias has incorrect shape.";
  }

  if (n_group <= 0 || topk_group <= 0) {
    TVM_FFI_ICHECK_EQ(top_k, 1) << "Current routing kernel (no groups) only supports top_k=1.";
  } else {
    TVM_FFI_ICHECK_LE(top_k, 8) << "Current routing kernel (with groups) only supports top_k<=8.";
    TVM_FFI_ICHECK_LE(topk_group, 4)
        << "Current routing kernel (with groups) only supports topk_group<=4.";
    TVM_FFI_ICHECK_LE(topk_group, n_group) << "n_group must not be smaller than topk_group.";
    TVM_FFI_ICHECK_EQ(num_experts % n_group, 0) << "num_experts must be divisible by n_group";
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TVM_FFI_ICHECK_LT(top_k, (topk_group * num_experts / n_group))
        << "top_k must be less than total number of experts in selected groups";
  }
  TVM_FFI_ICHECK_EQ(num_experts % 4, 0)
      << "Routing kernel expects that num_experts must be divisible by 4";
  TVM_FFI_ICHECK_GT(num_experts, top_k) << "num_experts must be greater than top_k";

  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  // Convert PyTorch dtype to TensorRT-LLM dtype
  auto dtype = hidden_states->dtype;
  if (dtype == dl_float16) {
    args.mDtypeElt = btg::Dtype::Fp16;
  } else if (dtype == dl_bfloat16) {
    args.mDtypeElt = btg::Dtype::Bfloat16;
  } else if (dtype == dl_float8_e4m3fn) {
    args.mDtypeElt = btg::Dtype::E4m3;
  } else {
    TVM_FFI_LOG_AND_THROW(NotImplementedError) << "Unsupported input dtype for MoE.";
  }

  auto const routing_bias_dtype =
      routing_bias.has_value() ? routing_bias.value()->dtype : dl_bfloat16;
  args.mDtypeExpW = routing_bias_dtype == dl_bfloat16 ? btg::Dtype::Bfloat16 : btg::Dtype::Fp32;
  args.routing_logits = static_cast<float*>(routing_logits->data);
  args.routing_bias = routing_bias.has_value() ? routing_bias.value()->data : nullptr;
  args.hidden_states = hidden_states->data;
  args.hidden_states_scale = static_cast<float*>(hidden_states_scale->data);
  args.gemm1_weights = gemm1_weights->data;
  args.gemm1_weights_scale = static_cast<float*>(gemm1_weights_scale->data);
  args.gemm2_weights = gemm2_weights->data;
  args.gemm2_weights_scale = static_cast<float*>(gemm2_weights_scale->data);
  args.num_tokens = hidden_states->shape[0];
  args.num_experts = num_experts;
  args.hidden_size = hidden_states->shape[1];
  args.hidden_size_output = args.hidden_size;
  args.top_k = top_k;
  args.n_group = n_group;
  args.topk_group = topk_group;
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor;
  args.intermediate_size = intermediate_size;
  args.mUseDeepSeekFp8 = true;

  // allocate workspace for routing kernel
  Tensor num_tokens_per_expert = alloc_tensor({num_experts}, dl_int32, routing_logits->device);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  Tensor total_num_padded_tokens = alloc_tensor({1}, dl_int32, routing_logits->device);
  Tensor expanded_idx_to_permuted_idx =
      alloc_tensor({args.num_tokens * args.top_k}, dl_int32, routing_logits->device);
  Tensor permuted_idx_to_token_idx =
      alloc_tensor({max_num_padded_tokens}, dl_int32, routing_logits->device);
  Tensor expert_weights =
      alloc_tensor({args.num_tokens, args.top_k}, routing_bias_dtype, routing_logits->device);
  Tensor expert_indexes =
      alloc_tensor({args.num_tokens, args.top_k}, dl_int32, routing_logits->device);
  int64_t const size_of_expert_count_histogram = std::max(num_experts * 2, int64_t(256 * 2));
  Tensor expert_count_histogram = alloc_tensor(
      {size_of_expert_count_histogram},
      dl_int32,  // 256 is the max number of threads per block and max number of experts
      routing_logits->device);

  // allocate workspace for activation/gemm/finalize kernels
  // Tensor gemm1_output = alloc_tensor({max_num_padded_tokens, 2 * intermediate_size},
  //                                    dl_float8_e4m3fn, hidden_states->device);
  // Tensor activation_output = alloc_tensor({max_num_padded_tokens, intermediate_size},
  //                                         dl_float8_e4m3fn, hidden_states->device);
  Tensor gemm1_output =
      alloc_tensor({max_num_padded_tokens, 2 * intermediate_size}, dl_uint8, hidden_states->device);
  Tensor gemm1_output_scale = alloc_tensor({2 * intermediate_size / 128, max_num_padded_tokens},
                                           dl_float32, hidden_states->device);
  Tensor activation_output =
      alloc_tensor({max_num_padded_tokens, intermediate_size}, dl_uint8, hidden_states->device);
  Tensor activation_output_scale = alloc_tensor({intermediate_size / 128, max_num_padded_tokens},
                                                dl_float32, hidden_states->device);
  Tensor gemm2_output =
      alloc_tensor({max_num_padded_tokens, args.hidden_size}, dl_bfloat16, hidden_states->device);

  int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
      args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  Tensor cta_idx_xy_to_batch_idx = alloc_tensor({max_num_ctas}, dl_int32, routing_logits->device);
  Tensor cta_idx_xy_to_mn_limit = alloc_tensor({max_num_ctas}, dl_int32, routing_logits->device);
  Tensor num_non_exiting_ctas = alloc_tensor({1}, dl_int32, routing_logits->device);

  tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
  hipStream_t stream = get_stream(routing_logits->device);
  routing_runner.run(static_cast<float*>(routing_logits->data), args.routing_bias, args.num_tokens,
                     args.num_experts, args.top_k, args.n_group, args.topk_group,
                     args.local_expert_offset, args.local_num_experts, args.routed_scaling_factor,
                     static_cast<int*>(expert_indexes->data),
                     static_cast<int*>(expert_count_histogram->data),
                     static_cast<int*>(total_num_padded_tokens->data),
                     static_cast<int*>(expanded_idx_to_permuted_idx->data),
                     nullptr /*static_cast<int*>(permuted_idx_to_expanded_idx->data)*/,
                     static_cast<int*>(permuted_idx_to_token_idx->data), expert_weights->data,
                     static_cast<int*>(num_tokens_per_expert->data),
                     static_cast<int*>(cta_idx_xy_to_batch_idx->data),
                     static_cast<int*>(cta_idx_xy_to_mn_limit->data),
                     static_cast<int*>(num_non_exiting_ctas->data), args.mDtypeElt, false, true,
                     static_cast<RoutingMethodType>(routing_method_type), stream);

  // MoE kernel except routing
  TVM_FFI_ICHECK_EQ(hidden_states->dtype, dl_float8_e4m3fn) << "hidden_states must be fp8.";
  TVM_FFI_ICHECK_EQ(hidden_states_scale->dtype, dl_float32) << "hidden_states_scale must be float.";
  TVM_FFI_ICHECK_EQ(hidden_states_scale->ndim, 2) << "hidden_states_scale must be 2D.";
  TVM_FFI_ICHECK_EQ(hidden_states_scale->shape[0], hidden_states->shape[1] / 128)
      << "hidden_states_scale dim0 must match hidden_states dim1 / 128.";
  TVM_FFI_ICHECK_EQ(hidden_states_scale->shape[1], args.num_tokens)
      << "hidden_states_scale dim1 must match num_tokens.";
  TVM_FFI_ICHECK_EQ(gemm1_weights->dtype, dl_float8_e4m3fn) << "gemm1_weights must be fp8.";

  TVM_FFI_ICHECK(gemm1_weights->ndim == 3 || gemm1_weights->ndim == 4)
      << "gemm1_weights must be 3D or 4D.";
  {
    int64_t Mn = 0, K = 0;
    if (gemm1_weights->ndim == 3) {
      // MajorK [num_experts, M, K]
      Mn = gemm1_weights->shape[1];
      K = gemm1_weights->shape[2];
    } else if (gemm1_weights->ndim == 4) {
      // BlockMajorK [num_experts, K/block_k, M, block_k]
      Mn = gemm1_weights->shape[2];
      int64_t block_k = gemm1_weights->shape[3];
      K = gemm1_weights->shape[1] * block_k;
    }
    TVM_FFI_ICHECK_EQ(Mn % 2, 0) << "the second dimension of weights must be even.";
    TVM_FFI_ICHECK_EQ(intermediate_size, Mn / 2) << "intermediate_size has incorrect shape.";
    TVM_FFI_ICHECK_EQ(K, hidden_states->shape[1])
        << "the third dimension of weights must be equal to hidden_size.";
  }
  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->dtype, dl_float32) << "gemm1_weights_scale must be float.";
  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->ndim, 3) << "gemm1_weights_scale must be 3D.";

  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->shape[0], local_num_experts)
      << "gemm1_weights_scale has incorrect shape.";
  TVM_FFI_ICHECK_EQ(intermediate_size % 128, 0)
      << "the second dimension of weights must be a multiple of 128.";
  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->shape[1], 2 * intermediate_size / 128)
      << "gemm1_weights_scale has incorrect shape.";
  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->shape[2], args.hidden_size / 128)
      << "gemm1_weights_scale has incorrect shape.";
  TVM_FFI_ICHECK_EQ(gemm2_weights->dtype, dl_float8_e4m3fn) << "gemm2_weights must be fp8.";

  TVM_FFI_ICHECK(gemm2_weights->ndim == 3 || gemm2_weights->ndim == 4)
      << "gemm2_weights must be 3D or 4D.";
  {
    int64_t K = 0;
    if (gemm2_weights->ndim == 3) {
      // MajorK [num_experts, M, K]
      K = gemm2_weights->shape[2];
    } else if (gemm2_weights->ndim == 4) {
      // BlockMajorK [num_experts, K/block_k, M, block_k]
      int64_t block_k = gemm2_weights->shape[3];
      K = gemm2_weights->shape[1] * block_k;
    }
    TVM_FFI_ICHECK_EQ(K, intermediate_size)
        << "the third dimension of weights must be equal to intermediate_size.";
  }
  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->dtype, dl_float32) << "gemm2_weights_scale must be float.";
  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->ndim, 3) << "gemm2_weights_scale must be 3D.";
  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->shape[0], local_num_experts)
      << "gemm2_weights_scale has incorrect shape.";
  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->shape[1], args.hidden_size / 128)
      << "gemm2_weights_scale has incorrect shape.";
  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->shape[2], intermediate_size / 128)
      << "gemm2_weights_scale has incorrect shape.";

  TVM_FFI_ICHECK_EQ(output->shape[0], args.num_tokens) << "output has incorrect shape.";
  TVM_FFI_ICHECK_EQ(output->shape[1], args.hidden_size) << "output has incorrect shape.";
  TVM_FFI_ICHECK_EQ(output->dtype, dl_bfloat16) << "output must be bf16.";

  // setup workspace
  workspace.total_num_padded_tokens = static_cast<int*>(total_num_padded_tokens->data);
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = static_cast<int*>(expert_indexes->data);
  workspace.permuted_idx_size = static_cast<int*>(total_num_padded_tokens->data);
  workspace.expanded_idx_to_permuted_idx = static_cast<int*>(
      expanded_idx_to_permuted_idx->data);  // Needed by activation/finalize kernels
  workspace.permuted_idx_to_token_idx =
      static_cast<int*>(permuted_idx_to_token_idx->data);  // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights->data;         // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = static_cast<int*>(cta_idx_xy_to_batch_idx->data);
  workspace.cta_idx_xy_to_mn_limit = static_cast<int*>(cta_idx_xy_to_mn_limit->data);
  workspace.num_non_exiting_ctas = static_cast<int*>(num_non_exiting_ctas->data);

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output->data;
  workspace.gemm1_output_scale = static_cast<float*>(gemm1_output_scale->data);
  // activation intermediate ws
  workspace.activation_output = activation_output->data;
  workspace.activation_output_scale = static_cast<float*>(activation_output_scale->data);
  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output->data;
  workspace.gemm2_output_scale = nullptr;
  args.output = output->data;
  args.output_scale = nullptr;

  auto workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);
  Tensor workspace_fc1 =
      alloc_tensor({std::get<0>(workspace_sizes)}, dl_int8, hidden_states->device);
  Tensor workspace_fc2 =
      alloc_tensor({std::get<1>(workspace_sizes)}, dl_int8, hidden_states->device);
  workspace.bmm1_workspace = workspace_fc1->data;
  workspace.bmm2_workspace = workspace_fc2->data;

  hipStream_t moe_stream = get_stream(hidden_states->device);
  moe_runner.run(args, workspace, hidden_states->device.device_id, moe_stream, moeConfigIndex,
                 enable_pdl);
}

void trtllm_fp8_block_scale_moe(Tensor routing_logits, Optional<Tensor> routing_bias,
                                Tensor hidden_states, Tensor hidden_states_scale,
                                Tensor gemm1_weights, Tensor gemm1_weights_scale,
                                Tensor gemm2_weights, Tensor gemm2_weights_scale, Tensor output,
                                int64_t num_experts, int64_t top_k, int64_t n_group,
                                int64_t topk_group, int64_t intermediate_size,
                                int64_t local_expert_offset, int64_t local_num_experts,
                                double routed_scaling_factor, int64_t tile_tokens_dim,
                                int64_t routing_method_type, bool use_shuffled_weight,
                                int64_t weight_layout, bool enable_pdl) {
  auto dtype = hidden_states->dtype;
  if (dtype == dl_float16 || dtype == dl_bfloat16 || dtype == dl_float8_e4m3fn) {
    using RunnerType = tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner;

    btg::Dtype mDtypeElt{btg::Dtype::E4m3};  // FP8 runner so hard-coded
    bool mUseDeepSeekFp8{true};              // Always true for BlockScaleMoe

    TVM_FFI_ICHECK(0 <= weight_layout && weight_layout <= 2)
        << "the value of weight_layout is not recognized";

    // Properly initialize the runner using make_unique like in the original code
    auto mRunner = std::make_unique<RunnerType>(
        mDtypeElt, mUseDeepSeekFp8, tile_tokens_dim, use_shuffled_weight,
        static_cast<batchedGemm::gemm::MatrixLayout>(weight_layout));

    // Always use fallback config (equivalent to moeConfigIndex == -1 case from original code)
    auto const num_tokens = hidden_states->shape[0];
    auto const hidden_size = hidden_states->shape[1];

    int64_t moeConfigIndex = mRunner->getDefaultValidConfigIndex(
        top_k, hidden_size, intermediate_size, local_num_experts, num_tokens);

    return trtllm_fp8_block_scale_moe_launcher(
        routing_logits, routing_bias, hidden_states, hidden_states_scale, gemm1_weights,
        gemm1_weights_scale, gemm2_weights, gemm2_weights_scale, output, num_experts, top_k,
        n_group, topk_group, intermediate_size, local_expert_offset, local_num_experts,
        routed_scaling_factor, tile_tokens_dim, routing_method_type, *mRunner, moeConfigIndex,
        enable_pdl);
  } else {
    TVM_FFI_LOG_AND_THROW(NotImplementedError) << "Unsupported hidden state dtype.";
  }
}

// TODO(siyuan): This launcher supports flexible weight and activation types.
// We should cleanup other launchers and only use this one in the future.
Array<Tensor> trtllm_fp4_block_scale_moe_launcher(
    Optional<Tensor> routing_logits, Tensor expert_indices, Tensor expert_weights,
    Optional<Tensor> routing_bias, Tensor hidden_states, Optional<Tensor> hidden_states_scale,
    Tensor gemm1_weights, Tensor gemm1_weights_scale, Optional<Tensor> gemm1_bias,
    Optional<Tensor> gemm1_alpha, Optional<Tensor> gemm1_beta, Optional<Tensor> gemm1_clamp_limit,
    Tensor gemm2_weights, Tensor gemm2_weights_scale, Optional<Tensor> gemm2_bias,
    Optional<Tensor> output1_scales_scalar, Optional<Tensor> output1_scales_gate_scalar,
    Optional<Tensor> output2_scales_scalar, int64_t const num_experts, int64_t const top_k,
    Optional<int64_t> const n_group, Optional<int64_t> const topk_group,
    int64_t const intermediate_size, int64_t const local_expert_offset,
    int64_t const local_num_experts, Optional<double> const routed_scaling_factor,
    int64_t const tile_tokens_dim, int64_t const routing_method_type, bool const do_finalize,
    tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner& moe_runner, btg::Dtype dtype_act,
    btg::Dtype dtype_weights, int64_t const moeConfigIndex, bool enable_pdl, Tensor output) {
  static const std::tuple<int, int> device_props = [hidden_states] {
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor,
                           hidden_states->device.device_id);
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor,
                           hidden_states->device.device_id);
    return std::make_tuple(major, minor);
  }();

  TVM_FFI_ICHECK_EQ(std::get<0>(device_props), 10)
      << "This kernel requires 10.x architecture. Current device has SM "
      << std::get<0>(device_props) << std::get<1>(device_props);

  TVM_FFI_ICHECK(dtype_act == btg::Dtype::E2m1 || dtype_act == btg::Dtype::Bfloat16 ||
                 dtype_act == btg::Dtype::E4m3 || dtype_act == btg::Dtype::MxE4m3)
      << "Only E2m1, Bfloat16, MxE4m3 and E4m3 are supported by block scale MoE";
  if (dtype_act == btg::Dtype::E2m1) {
    TVM_FFI_ICHECK(dtype_weights == btg::Dtype::E2m1)
        << "Only E2m1 and MxE2m1 are supported by block scale MoE with E2m1 activation";
    TVM_FFI_ICHECK(hidden_states_scale.has_value())
        << "hidden_states_scale is required for E2m1 activation";
    TVM_FFI_ICHECK(output1_scales_scalar.has_value())
        << "output1_scales_scalar is required for E2m1 activation";
    TVM_FFI_ICHECK(output1_scales_gate_scalar.has_value())
        << "output1_scales_gate_scalar is required for E2m1 activation";
    TVM_FFI_ICHECK(output2_scales_scalar.has_value())
        << "output2_scales_scalar is required for E2m1 activation";
  } else if (dtype_act == btg::Dtype::Bfloat16 || dtype_act == btg::Dtype::E4m3 ||
             dtype_act == btg::Dtype::MxE4m3) {
    TVM_FFI_ICHECK(dtype_weights == btg::Dtype::MxE2m1)
        << "Only MxE2m1 weights are supported by block scale MoE with Bfloat16, E4m3 or "
           "MxE4m3 activation";
  } else {
    TVM_FFI_LOG_AND_THROW(NotImplementedError) << "Unsupported act dtype.";
  }

  if (dtype_act == btg::Dtype::E4m3) {
    TVM_FFI_ICHECK(output1_scales_scalar.has_value())
        << "output1_scales_scalar is required for E4m3 activation";
    TVM_FFI_ICHECK(output1_scales_gate_scalar.has_value())
        << "output1_scales_gate_scalar is required for E4m3 activation";
    TVM_FFI_ICHECK(output2_scales_scalar.has_value())
        << "output2_scales_scalar is required for E4m3 activation";
  }

  if (routing_logits.has_value()) {
    TVM_FFI_ICHECK(routing_logits.value()->dtype == dl_float32 ||
                   routing_logits.value()->dtype == dl_bfloat16)
        << "routing_logits must be float or bfloat16.";
    TVM_FFI_ICHECK_EQ(routing_logits.value()->ndim, 2) << "routing_logits must be 2D.";
    TVM_FFI_ICHECK_EQ(routing_logits.value()->shape[1], num_experts)
        << "routing_logits has incorrect shape.";
  }
  if (routing_bias.has_value()) {
    TVM_FFI_ICHECK_EQ(routing_bias.value()->dtype, dl_bfloat16) << "routing_bias must be bfloat16.";
    TVM_FFI_ICHECK_EQ(routing_bias.value()->ndim, 1) << "routing_bias must be 1D.";
    TVM_FFI_ICHECK_EQ(routing_bias.value()->shape[0], num_experts)
        << "routing_bias has incorrect shape.";
  }

  if (n_group.value_or(0) != 0) {
    TVM_FFI_ICHECK(static_cast<RoutingMethodType>(routing_method_type) ==
                   RoutingMethodType::DeepSeekV3)
        << "Routing kernel with groups implies DeepSeekV3 routing method.";
    TVM_FFI_ICHECK(topk_group.has_value()) << "if n_group is given, topk_group must be given";
    TVM_FFI_ICHECK_EQ(num_experts % n_group.value(), 0)
        << "num_experts must be divisible by n_group";
    TVM_FFI_ICHECK(top_k <= 8 && top_k > 0)
        << "Current routing kernel (with groups) only supports top_k<=8 && top_k>0.";
    TVM_FFI_ICHECK(topk_group.value() <= 4 && topk_group.value() > 0)
        << "Current routing kernel only (with groups) supports topk_group<=4 && topk_group > 0.";
    TVM_FFI_ICHECK_LE(topk_group.value(), n_group.value())
        << "n_group must not be smaller than topk_group.";
    // This check ensures we have enough experts in the selected groups to handle the top_k routing
    TVM_FFI_ICHECK_LT(top_k, (topk_group.value() * num_experts / n_group.value()))
        << "top_k must be less than total number of experts in selected groups";
  } else if (static_cast<RoutingMethodType>(routing_method_type) ==
                 RoutingMethodType::Renormalize ||
             static_cast<RoutingMethodType>(routing_method_type) ==
                 RoutingMethodType::RenormalizeNaive ||
             static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::TopK) {
    TVM_FFI_ICHECK(top_k <= 8 && top_k > 0)
        << "Current routing kernel (no groups, renormalize/topk) only supports top_k<=8 && "
           "top_k>0.";
  } else if (static_cast<RoutingMethodType>(routing_method_type) == RoutingMethodType::Llama4) {
    TVM_FFI_ICHECK_EQ(top_k, 1)
        << "Current routing kernel (no groups, Llama4) only supports top_k=1.";
  }

  TVM_FFI_ICHECK_EQ(num_experts % 4, 0)
      << "Routing kernel expects that num_experts must be divisible by 4";
  TVM_FFI_ICHECK_GT(num_experts, top_k) << "num_experts must be greater than top_k";

  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoERunnerArgs args;
  tensorrt_llm::kernels::trtllmgen_moe::MoE::MoEWorkspace workspace;

  // setup args
  // note: the assumption is that output data type is always Bfloat16 (the default)
  auto routing_bias_dtype = dl_bfloat16;
  if (routing_bias.has_value()) {
    routing_bias_dtype = routing_bias.value()->dtype;
  } else if (routing_logits.has_value()) {
    routing_bias_dtype = routing_logits.value()->dtype;
  }
  args.mDtypeElt = dtype_act;
  args.mDtypeExpW = routing_bias_dtype == dl_float32 ? btg::Dtype::Fp32 : btg::Dtype::Bfloat16;
  args.routing_logits = routing_logits.has_value() ? routing_logits.value()->data : nullptr;
  args.routing_bias = routing_bias.has_value() ? routing_bias.value()->data : nullptr;
  args.hidden_states = hidden_states->data;
  args.hidden_states_scale =
      hidden_states_scale.has_value() ? hidden_states_scale.value()->data : nullptr;
  args.gemm1_weights = gemm1_weights->data;
  args.gemm1_weights_scale = gemm1_weights_scale->data;
  args.gemm1_bias =
      gemm1_bias.has_value() ? static_cast<float*>(gemm1_bias.value()->data) : nullptr;
  args.gemm1_alpha =
      gemm1_alpha.has_value() ? static_cast<float*>(gemm1_alpha.value()->data) : nullptr;
  args.gemm1_beta =
      gemm1_beta.has_value() ? static_cast<float*>(gemm1_beta.value()->data) : nullptr;
  args.gemm1_clamp_limit = gemm1_clamp_limit.has_value()
                               ? static_cast<float*>(gemm1_clamp_limit.value()->data)
                               : nullptr;
  args.gemm2_weights = gemm2_weights->data;
  args.gemm2_weights_scale = gemm2_weights_scale->data;
  args.gemm2_bias =
      gemm2_bias.has_value() ? static_cast<float*>(gemm2_bias.value()->data) : nullptr;
  args.num_tokens = hidden_states->shape[0];
  args.num_experts = num_experts;
  // * 2 to compensate for the fact that sizeof(hidden_states.dtype) is 1 because we pack 2 e2m1
  // into 1 byte.
  auto const hidden_states_hidden_size =
      dtype_act == btg::Dtype::E2m1 ? hidden_states->shape[1] * 2 : hidden_states->shape[1];
  args.hidden_size = hidden_states_hidden_size;
  args.hidden_size_output = args.hidden_size;
  args.top_k = top_k;
  args.n_group = n_group.value_or(0);
  args.topk_group = topk_group.value_or(0);
  args.local_expert_offset = local_expert_offset;
  args.local_num_experts = local_num_experts;
  args.routed_scaling_factor = routed_scaling_factor.value_or(1.0);
  args.intermediate_size = intermediate_size;

  // allocate workspace for routing kernel
  Tensor num_tokens_per_expert = alloc_tensor({num_experts}, dl_int32, hidden_states->device);
  int32_t max_num_padded_tokens =
      tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxPermutedPaddedCount(
          args.num_tokens, top_k, num_experts, tile_tokens_dim);
  Tensor total_num_padded_tokens = alloc_tensor({1}, dl_int32, hidden_states->device);
  Tensor expanded_idx_to_permuted_idx =
      alloc_tensor({args.num_tokens, args.top_k}, dl_int32, hidden_states->device);

  Tensor permuted_idx_to_token_idx =
      alloc_tensor({max_num_padded_tokens}, dl_int32, hidden_states->device);
  // Tensor expert_weights = alloc_tensor(
  //     {args.num_tokens, args.top_k}, routing_bias_dtype, hidden_states->device);
  // Tensor expert_indexes = alloc_tensor(
  //     {args.num_tokens, args.top_k}, dl_int32, hidden_states->device);
  int constexpr MAX_NUM_EXPERTS = 384;
  Tensor expert_count_histogram = alloc_tensor(
      {2 * MAX_NUM_EXPERTS},
      dl_int32,  // 256 is the max number of threads per block and max number of experts
      hidden_states->device);

  auto const sf_vec_size = dtype_weights == btg::Dtype::MxE2m1 ? 32 : 16;

  // allocate workspace for activation/gemm/finalize kernels
  auto const gemm1_output_hidden =
      dtype_act == btg::Dtype::E2m1 ? intermediate_size / 2 : intermediate_size;
  // Tensor gemm1_output = alloc_tensor(
  //     {max_num_padded_tokens, gemm1_output_hidden},
  //     dtype_act == btg::Dtype::Bfloat16 ? dl_bfloat16 : dl_float8_e4m3fn, hidden_states->device);
  Tensor gemm1_output = alloc_tensor({max_num_padded_tokens, gemm1_output_hidden},
                                     dtype_act == btg::Dtype::Bfloat16 ? dl_bfloat16 : dl_uint8,
                                     hidden_states->device);

  Optional<Tensor> gemm1_output_scale = std::nullopt;
  if (dtype_act == btg::Dtype::E2m1 || dtype_act == btg::Dtype::MxE4m3) {
    int64_t sf_size = tensorrt_llm::computeSwizzledLayoutSFSize(max_num_padded_tokens,
                                                                intermediate_size / sf_vec_size);
    // gemm1_output_scale = alloc_tensor({sf_size}, dl_float8_e4m3fn, hidden_states->device);
    gemm1_output_scale = alloc_tensor({sf_size}, dl_uint8, hidden_states->device);
  }

  Tensor gemm2_output =
      alloc_tensor({max_num_padded_tokens, args.hidden_size}, dl_bfloat16, hidden_states->device);

  int32_t max_num_ctas = tensorrt_llm::kernels::trtllmgen_moe::Routing::getMaxNumCtasInBatchDim(
      args.num_tokens, args.top_k, args.num_experts, tile_tokens_dim);
  Tensor cta_idx_xy_to_batch_idx = alloc_tensor({max_num_ctas}, dl_int32, hidden_states->device);
  Tensor cta_idx_xy_to_mn_limit = alloc_tensor({max_num_ctas}, dl_int32, hidden_states->device);
  Tensor num_non_exiting_ctas = alloc_tensor({1}, dl_int32, hidden_states->device);

  //
  // TopK routing
  //

  tensorrt_llm::kernels::trtllmgen_moe::Routing::Runner routing_runner(tile_tokens_dim);
  hipStream_t stream = get_stream(hidden_states->device);
  routing_runner.run(args.routing_logits, args.routing_bias, args.num_tokens, args.num_experts,
                     args.top_k, args.n_group, args.topk_group, args.local_expert_offset,
                     args.local_num_experts, args.routed_scaling_factor,
                     static_cast<int*>(expert_indices->data),
                     static_cast<int*>(expert_count_histogram->data),
                     static_cast<int*>(total_num_padded_tokens->data),
                     static_cast<int*>(expanded_idx_to_permuted_idx->data),
                     nullptr, /*static_cast<int*>(permuted_idx_to_expanded_idx->data),*/
                     static_cast<int*>(permuted_idx_to_token_idx->data), expert_weights->data,
                     static_cast<int*>(num_tokens_per_expert->data),
                     static_cast<int*>(cta_idx_xy_to_batch_idx->data),
                     static_cast<int*>(cta_idx_xy_to_mn_limit->data),
                     static_cast<int*>(num_non_exiting_ctas->data), args.mDtypeElt,
                     false /* use_routing_scales_on_input */, false /* use_deep_seek_fp8 */,
                     static_cast<RoutingMethodType>(routing_method_type), stream);

  //
  // FC13 (gemm1) + FC2 (gemm2)
  //

  if (dtype_act == btg::Dtype::E2m1) {
    TVM_FFI_ICHECK_EQ(hidden_states->dtype, dl_uint8) << "hidden_states must be byte.";
  } else if (dtype_act == btg::Dtype::E4m3 || dtype_act == btg::Dtype::MxE4m3) {
    TVM_FFI_ICHECK_EQ(hidden_states->dtype, dl_float8_e4m3fn) << "hidden_states must be fp8.";
  } else if (dtype_act == btg::Dtype::Bfloat16) {
    TVM_FFI_ICHECK_EQ(hidden_states->dtype, dl_bfloat16) << "hidden_states must be bfloat16.";
  } else {
    TVM_FFI_LOG_AND_THROW(NotImplementedError) << "Unsupported act dtype.";
  }

  if (hidden_states_scale.has_value()) {
    TVM_FFI_ICHECK_EQ(hidden_states_scale.value()->dtype, dl_float8_e4m3fn)
        << "hidden_states_scale must be fp8.";

    TVM_FFI_ICHECK_EQ(
        get_numel(hidden_states_scale.value()),
        tensorrt_llm::computeLinearLayoutSFSize(args.num_tokens, args.hidden_size / sf_vec_size))
        << "hidden_states_scale has incorrect size";
  }

  TVM_FFI_ICHECK_EQ(gemm1_weights->dtype, dl_uint8) << "gemm1_weights must be byte.";

  TVM_FFI_ICHECK_EQ(gemm1_weights->ndim, 3) << "gemm1_weights must be 3D.";
  TVM_FFI_ICHECK_EQ(gemm1_weights->shape[1] % 2, 0)
      << "the second dimension of weights must be even.";
  TVM_FFI_ICHECK_EQ(intermediate_size, gemm1_weights->shape[1] / 2)
      << "intermediate_size has incorrect dim 1.";
  // This check passes even though the actual shape of the weights[2] and hidden_states[1] is
  // 2 times larger due to the fact that 2 e2m1 are packed into 1 byte.
  TVM_FFI_ICHECK_EQ(
      gemm1_weights->shape[2],
      (dtype_act == btg::Dtype::E2m1 ? hidden_states->shape[1] : hidden_states->shape[1] / 2))
      << "the third dimension of weights must be equal to hidden_size.";

  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->dtype, dl_float8_e4m3fn)
      << "gemm1_weights_scale must be fp8.";

  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->ndim, 3) << "gemm1_weights_scale must be 3D.";
  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->shape[0], local_num_experts)
      << "gemm1_weights_scale has incorrect dim 0.";
  TVM_FFI_ICHECK_EQ(intermediate_size % sf_vec_size, 0)
      << "the second dimension of weights must be a multiple of ",
      sf_vec_size;
  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->shape[1], 2 * intermediate_size)
      << "gemm1_weights_scale has incorrect dim 1.";
  TVM_FFI_ICHECK_EQ(gemm1_weights_scale->shape[2], args.hidden_size / sf_vec_size)
      << "gemm1_weights_scale has incorrect dim 2.";

  if (gemm1_bias.has_value()) {
    TVM_FFI_ICHECK_EQ(gemm1_bias.value()->dtype, dl_float32)
        << "gemm1_bias must be float, got "
        << tvm::ffi::DLDataTypeToString(gemm1_bias.value()->dtype);
    TVM_FFI_ICHECK_EQ(gemm1_bias.value()->ndim, 2) << "gemm1_bias must be 2D.";
    TVM_FFI_ICHECK_EQ(gemm1_bias.value()->shape[0], local_num_experts)
        << "gemm1_bias has incorrect dim 0.";
    TVM_FFI_ICHECK_EQ(gemm1_bias.value()->shape[1], 2 * intermediate_size)
        << "gemm1_bias has incorrect dim 1.";
  }

  if (gemm1_alpha.has_value()) {
    TVM_FFI_ICHECK_EQ(gemm1_alpha.value()->dtype, dl_float32)
        << "gemm1_alpha must be float, got "
        << tvm::ffi::DLDataTypeToString(gemm1_alpha.value()->dtype);
    TVM_FFI_ICHECK_EQ(gemm1_alpha.value()->ndim, 1) << "gemm1_alpha must be 1D.";
    TVM_FFI_ICHECK_EQ(gemm1_alpha.value()->shape[0], local_num_experts)
        << "gemm1_alpha has incorrect dim 0.";
  }
  if (gemm1_beta.has_value()) {
    TVM_FFI_ICHECK_EQ(gemm1_beta.value()->dtype, dl_float32)
        << "gemm1_beta must be float, got "
        << tvm::ffi::DLDataTypeToString(gemm1_beta.value()->dtype);
    TVM_FFI_ICHECK_EQ(gemm1_beta.value()->ndim, 1) << "gemm1_beta must be 1D.";
    TVM_FFI_ICHECK_EQ(gemm1_beta.value()->shape[0], local_num_experts)
        << "gemm1_beta has incorrect dim 0.";
  }

  TVM_FFI_ICHECK_EQ(gemm2_weights->dtype, dl_uint8) << "gemm2_weights must be byte.";

  TVM_FFI_ICHECK_EQ(gemm2_weights->ndim, 3) << "gemm2_weights must be 3D.";
  // / 2 to compensate for the fact that we pack 2 e2m1 into 1 byte.
  TVM_FFI_ICHECK_EQ(gemm2_weights->shape[2], intermediate_size / 2)
      << "the third dimension of weights must be equal to intermediate_size.";

  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->dtype, dl_float8_e4m3fn)
      << "gemm2_weights_scale must be fp8.";

  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->ndim, 3) << "gemm2_weights_scale must be 3D.";
  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->shape[0], local_num_experts)
      << "gemm2_weights_scale has incorrect dim 0.";
  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->shape[1], args.hidden_size)
      << "gemm2_weights_scale has incorrect dim 1.";
  TVM_FFI_ICHECK_EQ(gemm2_weights_scale->shape[2], intermediate_size / sf_vec_size)
      << "gemm2_weights_scale has incorrect dim 2.";

  if (output1_scales_scalar.has_value()) {
    TVM_FFI_ICHECK_EQ(output1_scales_scalar.value()->dtype, dl_float32)
        << "output1_scales_scalar must be float.";
    TVM_FFI_ICHECK_EQ(output1_scales_scalar.value()->ndim, 1)
        << "output1_scales_scalar must be 1D.";
    TVM_FFI_ICHECK_EQ(output1_scales_scalar.value()->shape[0], local_num_experts)
        << "output1_scales_scalar has incorrect dim 0.";
  }

  if (output1_scales_gate_scalar.has_value()) {
    TVM_FFI_ICHECK_EQ(output1_scales_gate_scalar.value()->dtype, dl_float32)
        << "output1_scales_gate_scalar must be float.";
    TVM_FFI_ICHECK_EQ(output1_scales_gate_scalar.value()->ndim, 1)
        << "output1_scales_gate_scalar must be 1D.";
    TVM_FFI_ICHECK_EQ(output1_scales_gate_scalar.value()->shape[0], local_num_experts)
        << "output1_scales_gate_scalar has incorrect dim 0.";
  }

  if (output2_scales_scalar.has_value()) {
    TVM_FFI_ICHECK_EQ(output2_scales_scalar.value()->dtype, dl_float32)
        << "output2_scales_scalar must be float.";
    TVM_FFI_ICHECK_EQ(output2_scales_scalar.value()->ndim, 1)
        << "output2_scales_scalar must be 1D.";
    TVM_FFI_ICHECK_EQ(output2_scales_scalar.value()->shape[0], local_num_experts)
        << "output2_scales_scalar has incorrect dim 0.";
  }

  // setup workspace
  workspace.total_num_padded_tokens = static_cast<int*>(total_num_padded_tokens->data);
  workspace.total_max_padded_tokens = max_num_padded_tokens;
  workspace.ProjUpTileN = tile_tokens_dim;
  workspace.routing_expert_indexes = static_cast<int*>(expert_indices->data);
  workspace.permuted_idx_size = static_cast<int*>(total_num_padded_tokens->data);
  workspace.expanded_idx_to_permuted_idx =
      static_cast<int*>(expanded_idx_to_permuted_idx->data);  // Needed by permute/finalize kernels
  workspace.permuted_idx_to_token_idx =
      static_cast<int*>(permuted_idx_to_token_idx->data);  // Needed by permuteGemm1 kernel
  workspace.expert_weights = expert_weights->data;         // Consumed by finalize kernel

  workspace.cta_idx_xy_to_batch_idx = static_cast<int*>(cta_idx_xy_to_batch_idx->data);
  workspace.cta_idx_xy_to_mn_limit = static_cast<int*>(cta_idx_xy_to_mn_limit->data);
  workspace.num_non_exiting_ctas = static_cast<int*>(num_non_exiting_ctas->data);

  workspace.hidden_states_scale_linear = nullptr;

  // gemm1 intermediate ws
  workspace.gemm1_output = gemm1_output->data;
  workspace.gemm1_output_scale = gemm1_output_scale.has_value()
                                     ? static_cast<float*>(gemm1_output_scale.value()->data)
                                     : nullptr;

  // gemm2 intermediate ws
  workspace.gemm2_output = gemm2_output->data;
  workspace.gemm2_output_scale = nullptr;
  args.output = output->data;
  args.output_scale = nullptr;
  args.output1_scales_scalar = output1_scales_scalar.has_value()
                                   ? static_cast<float*>(output1_scales_scalar.value()->data)
                                   : nullptr;
  args.output1_scales_gate_scalar =
      output1_scales_gate_scalar.has_value()
          ? static_cast<float*>(output1_scales_gate_scalar.value()->data)
          : nullptr;
  args.output2_scales_scalar = output2_scales_scalar.has_value()
                                   ? static_cast<float*>(output2_scales_scalar.value()->data)
                                   : nullptr;
  args.do_finalize = do_finalize;

  auto const workspace_sizes = moe_runner.getWorkspaceSizeInBytes(args, moeConfigIndex);

  Tensor workspace_fc1 =
      alloc_tensor({std::get<0>(workspace_sizes)}, dl_int8, hidden_states->device);
  Tensor workspace_fc2 =
      alloc_tensor({std::get<1>(workspace_sizes)}, dl_int8, hidden_states->device);
  workspace.bmm1_workspace = workspace_fc1->data;
  workspace.bmm2_workspace = workspace_fc2->data;
  hipStream_t moe_stream = get_stream(hidden_states->device);
  moe_runner.run(args, workspace, hidden_states->device.device_id, moe_stream, moeConfigIndex,
                 enable_pdl);

  if (!do_finalize) {
    return {gemm2_output, expert_weights, expanded_idx_to_permuted_idx};
  }
  return {output};
}

Array<Tensor> trtllm_fp4_block_scale_moe(
    Optional<Tensor> routing_logits, Tensor topk_ids, Tensor expert_weights,
    Optional<Tensor> routing_bias, Tensor hidden_states, Optional<Tensor> hidden_states_scale,
    Tensor gemm1_weights, Tensor gemm1_weights_scale, Optional<Tensor> gemm1_bias,
    Optional<Tensor> gemm1_alpha, Optional<Tensor> gemm1_beta, Optional<Tensor> gemm1_clamp_limit,
    Tensor gemm2_weights, Tensor gemm2_weights_scale, Optional<Tensor> gemm2_bias,
    Optional<Tensor> output1_scales_scalar, Optional<Tensor> output1_scales_gate_scalar,
    Optional<Tensor> output2_scales_scalar, int64_t num_experts, int64_t top_k,
    Optional<int64_t> n_group, Optional<int64_t> topk_group, int64_t intermediate_size,
    int64_t local_expert_offset, int64_t local_num_experts, Optional<double> routed_scaling_factor,
    int64_t tile_tokens_dim, int64_t routing_method_type, bool do_finalize, bool enable_pdl,
    int64_t gated_act_type, Tensor output, int64_t config_index) {
  using RunnerType = tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner;

  int const num_tokens = hidden_states->shape[0];
  int hidden_size = hidden_states->shape[1];
  if (hidden_states->dtype == dl_uint8) hidden_size *= 2;
  int hidden_states_scale_vec_size = -1;
  if (hidden_states_scale.has_value()) {
    hidden_states_scale_vec_size =
        (num_tokens * hidden_size) / get_numel(hidden_states_scale.value());
  }
  int weight_scale_vec_size =
      (local_num_experts * intermediate_size * 2 * hidden_size) / get_numel(gemm1_weights_scale);
  TVM_FFI_ICHECK(weight_scale_vec_size == 16 || weight_scale_vec_size == 32)
      << "unsupported weight_scale_vec_size.";
  auto mDtypeWeights = weight_scale_vec_size == 16 ? btg::Dtype::E2m1 : btg::Dtype::MxE2m1;

  TVM_FFI_ICHECK(gemm1_weights->dtype == dl_uint8 && gemm2_weights->dtype == dl_uint8)
      << "weights must be fp4 packed in uint8.";
  TVM_FFI_ICHECK(hidden_states->dtype == dl_uint8 || hidden_states->dtype == dl_bfloat16 ||
                 hidden_states->dtype == dl_float8_e4m3fn)
      << "hidden_states must be bf16, fp8 or uint8 (packed fp4).";
  auto mDtypeAct = btg::Dtype::Bfloat16;
  if (hidden_states->dtype == dl_uint8) {
    TVM_FFI_ICHECK(hidden_states_scale.has_value() &&
                   hidden_states_scale.value()->dtype == dl_float8_e4m3fn)
        << "hidden_states_scale must be provided for fp4 activation.";
    if (hidden_states_scale_vec_size == 16) {
      mDtypeAct = btg::Dtype::E2m1;
    } else if (hidden_states_scale_vec_size == 32) {
      mDtypeAct = btg::Dtype::MxE2m1;
    } else {
      TVM_FFI_LOG_AND_THROW(NotImplementedError) << "Unsupported hidden state scale shape.";
    }
  } else if (hidden_states->dtype == dl_float8_e4m3fn) {
    if (hidden_states_scale.has_value()) {
      if (hidden_states_scale_vec_size == 32) {
        mDtypeAct = btg::Dtype::MxE4m3;
      } else {
        TVM_FFI_LOG_AND_THROW(NotImplementedError) << "Unsupported hidden state scale shape.";
      }
    } else {
      mDtypeAct = btg::Dtype::E4m3;
    }
  }
  bool mUseDeepSeekFp8{false};  // FP4 doesn't use DeepSeek FP8

  // Properly initialize the runner using make_unique like in the original code
  auto mRunner = std::make_unique<RunnerType>(
      mDtypeAct, mDtypeWeights, mUseDeepSeekFp8, (int32_t)tile_tokens_dim,
      static_cast<GatedActType>(gated_act_type), /*useShuffledMatrixA*/ true);

  if (config_index == -1) {
    config_index = mRunner->getDefaultValidConfigIndex(top_k, hidden_size, intermediate_size,
                                                       local_num_experts, num_tokens);
  }

  return trtllm_fp4_block_scale_moe_launcher(
      routing_logits, topk_ids, expert_weights, routing_bias, hidden_states, hidden_states_scale,
      gemm1_weights, gemm1_weights_scale, gemm1_bias, gemm1_alpha, gemm1_beta, gemm1_clamp_limit,
      gemm2_weights, gemm2_weights_scale, gemm2_bias, output1_scales_scalar,
      output1_scales_gate_scalar, output2_scales_scalar, num_experts, top_k, n_group, topk_group,
      intermediate_size, local_expert_offset, local_num_experts, routed_scaling_factor,
      tile_tokens_dim, routing_method_type, do_finalize, *mRunner, mDtypeAct, mDtypeWeights,
      config_index, enable_pdl, output);
}

int64_t trtllm_get_default_moe_configs(int64_t const tile_tokens_dim, int64_t const dtype_act_,
                                       int64_t const dtype_weights_, bool const useDeepSeekFp8,
                                       int64_t const top_k, int64_t const hidden_size,
                                       int64_t const intermediate_size,
                                       int64_t const num_local_experts,
                                       int64_t const gated_act_type, int64_t const num_tokens) {
  auto dtype_act = static_cast<btg::Dtype>(dtype_act_);
  auto dtype_weights = static_cast<btg::Dtype>(dtype_weights_);
  tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner moe_runner(
      dtype_act, dtype_weights, useDeepSeekFp8, (int32_t)tile_tokens_dim,
      static_cast<GatedActType>(gated_act_type), /*useShuffledMatrixA*/ true);
  return moe_runner.getDefaultValidConfigIndex(top_k, hidden_size, intermediate_size,
                                               num_local_experts, num_tokens);
}

Array<int64_t> trtllm_get_valid_moe_configs(int64_t const tile_tokens_dim, int64_t const dtype_act_,
                                            int64_t const dtype_weights_, bool const useDeepSeekFp8,
                                            int64_t const top_k, int64_t const hidden_size,
                                            int64_t const intermediate_size,
                                            int64_t const num_local_experts,
                                            int64_t const gated_act_type,
                                            int64_t const num_tokens) {
  auto dtype_act = static_cast<btg::Dtype>(dtype_act_);
  auto dtype_weights = static_cast<btg::Dtype>(dtype_weights_);
  tensorrt_llm::kernels::trtllmgen_moe::MoE::Runner moe_runner(
      dtype_act, dtype_weights, useDeepSeekFp8, (int32_t)tile_tokens_dim,
      static_cast<GatedActType>(gated_act_type), /*useShuffledMatrixA*/ true);
  return moe_runner.getValidConfigIndices(top_k, hidden_size, intermediate_size, num_local_experts,
                                          num_tokens);
}

namespace trtllm_cubin_loader {
#include <flashinfer/cubin_loader.h>
}

TVM_FFI_DLL_EXPORT_TYPED_FUNC(trtllm_fp8_per_tensor_scale_moe, trtllm_fp8_per_tensor_scale_moe);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(trtllm_fp8_block_scale_moe, trtllm_fp8_block_scale_moe);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(trtllm_fp4_block_scale_moe, trtllm_fp4_block_scale_moe);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(trtllm_get_default_moe_configs, trtllm_get_default_moe_configs);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(trtllm_get_valid_moe_configs, trtllm_get_valid_moe_configs);

}  // namespace flashinfer
