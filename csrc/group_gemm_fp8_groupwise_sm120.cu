/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/cutlass_utils.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;

#define DISPATCH_PYTORCH_INPUT_OUTPUT_DTYPE(input_dtype, output_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                                                  \
    return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(output_dtype, c_type_out, [&] {                    \
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(input_dtype, c_type_in,                           \
                                                 [&] { return __VA_ARGS__(); });                   \
    });                                                                                            \
  }()

#define DISPATCH_SCALE_GRANULARITY(scale_granularity_m, scale_granularity_n, scale_granularity_k,  \
                                   SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K,  \
                                   ...)                                                            \
  [&]() -> bool {                                                                                  \
    constexpr int SCALE_GRANULARITY_K = 128;                                                       \
    if (scale_granularity_k != 128) {                                                              \
      TORCH_CHECK(                                                                                 \
          false,                                                                                   \
          "SM120 requires scale_granularity_k=128. CUTLASS enforces ScaleGranularityK must equal " \
          "tile shape K dimension (128 for both Cooperative and PingPong schedules).");            \
      return false;                                                                                \
    }                                                                                              \
    /* Match SM100's approach: support only (1,128,128) and (128,128,128) */                       \
    if (scale_granularity_m == 1 && scale_granularity_n == 128) {                                  \
      constexpr int SCALE_GRANULARITY_M = 1;                                                       \
      constexpr int SCALE_GRANULARITY_N = 128;                                                     \
      return __VA_ARGS__();                                                                        \
    } else if (scale_granularity_m == 128 && scale_granularity_n == 128) {                         \
      constexpr int SCALE_GRANULARITY_M = 128;                                                     \
      constexpr int SCALE_GRANULARITY_N = 128;                                                     \
      return __VA_ARGS__();                                                                        \
    }                                                                                              \
    TORCH_CHECK(false, "SM120: Unsupported scale granularity combination (", scale_granularity_m,  \
                ",", scale_granularity_n, ",", scale_granularity_k, ")");                          \
    return false;                                                                                  \
  }()

#define DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, ...) \
  [&]() -> bool {                                                    \
    if (scale_major_mode == "K") {                                   \
      constexpr bool SCALE_MAJOR_K = true;                           \
      return __VA_ARGS__();                                          \
    } else if (scale_major_mode == "MN") {                           \
      constexpr bool SCALE_MAJOR_K = false;                          \
      return __VA_ARGS__();                                          \
    }                                                                \
    TORCH_CHECK(false, "Unsupported Scale Major Mode");              \
    return false;                                                    \
  }()

namespace flashinfer {
namespace group_gemm {

template <int ScaleGranularityM, int ScaleGranularityN, int ScaleGranularityK, bool ScaleMajorK,
          typename DTypeIn, typename DTypeOut>
hipError_t CutlassFP8GroupwiseScaledGroupGEMMSM120(
    void* int_buffer, size_t int_buffer_size_in_bytes, void* float_buffer,
    size_t float_buffer_size_in_bytes, DTypeIn* A, DTypeIn* B, float* SFA, float* SFB, DTypeOut* D,
    int* m_indptr, int max_m, int n, int k, int num_groups, hipStream_t stream);

}  // namespace group_gemm
}  // namespace flashinfer

void CutlassGroupGemmFP8GroupwiseScaledSM120(
    at::Tensor int_workspace_buffer, at::Tensor float_workspace_buffer, at::Tensor A, at::Tensor B,
    at::Tensor SFA, at::Tensor SFB, at::Tensor D, at::Tensor m_indptr, int64_t n, int64_t k,
    int64_t scale_granularity_m, int64_t scale_granularity_n, int64_t scale_granularity_k,
    std::string scale_major_mode) {
  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer.device());
  auto stream = at::cuda::getCurrentCUDAStream();
  int num_groups = m_indptr.size(0) - 1;

  // Ensure scales are contiguous
  // Note: We keep the original shape and let the kernel's layout handle interpretation
  at::Tensor SFA_contig = SFA.is_contiguous() ? SFA : SFA.contiguous();
  at::Tensor SFB_contig = SFB.is_contiguous() ? SFB : SFB.contiguous();

  // Get max_m from SFA shape
  int max_m = SFA.size(SFA.dim() > 1 ? 1 : 0);

  DISPATCH_PYTORCH_INPUT_OUTPUT_DTYPE(A.scalar_type(), D.scalar_type(), c_type_in, c_type_out, [&] {
    return DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, [&] {
      return DISPATCH_SCALE_GRANULARITY(
          scale_granularity_m, scale_granularity_n, scale_granularity_k, SCALE_GRANULARITY_M,
          SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, [&] {
            using cutlass_t_in = cutlass_dtype_t<c_type_in>;
            using cutlass_t_out = cutlass_dtype_t<c_type_out>;
            auto status = flashinfer::group_gemm::CutlassFP8GroupwiseScaledGroupGEMMSM120<
                SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, SCALE_MAJOR_K,
                cutlass_t_in, cutlass_t_out>(
                static_cast<int*>(int_workspace_buffer.data_ptr()),
                int_workspace_buffer.element_size() * int_workspace_buffer.size(0),
                static_cast<float*>(float_workspace_buffer.data_ptr()),
                float_workspace_buffer.element_size() * float_workspace_buffer.size(0),
                static_cast<cutlass_t_in*>(A.data_ptr()), static_cast<cutlass_t_in*>(B.data_ptr()),
                static_cast<float*>(SFA_contig.data_ptr()),
                static_cast<float*>(SFB_contig.data_ptr()),
                static_cast<cutlass_t_out*>(D.data_ptr()), static_cast<int*>(m_indptr.data_ptr()),
                max_m, n, k, num_groups, stream);
            return status == hipSuccess;
          });
    });
  });
}
