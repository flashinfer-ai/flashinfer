/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/cutlass_utils.cuh>

#include "tvm_ffi_utils.h"

using namespace flashinfer;

#define DISPATCH_DLPACK_INPUT_OUTPUT_DTYPE(input_dtype, output_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                                                 \
    return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(output_dtype, c_type_out, [&] {                    \
      return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP8(input_dtype, c_type_in,                           \
                                                [&] { return __VA_ARGS__(); });                   \
    });                                                                                           \
  }()

#define DISPATCH_SCALE_GRANULARITY(scale_granularity_m, scale_granularity_n, scale_granularity_k, \
                                   SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, \
                                   ...)                                                           \
  [&]() -> bool {                                                                                 \
    constexpr int SCALE_GRANULARITY_K = 128;                                                      \
    if (scale_granularity_k != 128) {                                                             \
      TVM_FFI_ICHECK(false)                                                                       \
          << "SM120 requires scale_granularity_k=128. CUTLASS enforces ScaleGranularityK must "   \
             "equal tile shape K dimension (128 for both Cooperative and PingPong schedules).";   \
      return false;                                                                               \
    }                                                                                             \
    /* Match SM100's approach: support only (1,128,128) and (128,128,128) */                      \
    if (scale_granularity_m == 1 && scale_granularity_n == 128) {                                 \
      constexpr int SCALE_GRANULARITY_M = 1;                                                      \
      constexpr int SCALE_GRANULARITY_N = 128;                                                    \
      return __VA_ARGS__();                                                                       \
    } else if (scale_granularity_m == 128 && scale_granularity_n == 128) {                        \
      constexpr int SCALE_GRANULARITY_M = 128;                                                    \
      constexpr int SCALE_GRANULARITY_N = 128;                                                    \
      return __VA_ARGS__();                                                                       \
    }                                                                                             \
    TVM_FFI_ICHECK(false) << "SM120: Unsupported scale granularity combination ("                 \
                          << scale_granularity_m << "," << scale_granularity_n << ","             \
                          << scale_granularity_k << ")";                                          \
    return false;                                                                                 \
  }()

#define DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, ...) \
  [&]() -> bool {                                                    \
    if (scale_major_mode == "K") {                                   \
      constexpr bool SCALE_MAJOR_K = true;                           \
      return __VA_ARGS__();                                          \
    } else if (scale_major_mode == "MN") {                           \
      constexpr bool SCALE_MAJOR_K = false;                          \
      return __VA_ARGS__();                                          \
    }                                                                \
    TVM_FFI_ICHECK(false) << "Unsupported Scale Major Mode";         \
    return false;                                                    \
  }()

namespace flashinfer {
namespace group_gemm {

template <int ScaleGranularityM, int ScaleGranularityN, int ScaleGranularityK, bool ScaleMajorK,
          typename DTypeIn, typename DTypeOut>
hipError_t CutlassFP8GroupwiseScaledGroupGEMMSM120(
    void* int_buffer, size_t int_buffer_size_in_bytes, void* float_buffer,
    size_t float_buffer_size_in_bytes, DTypeIn* A, DTypeIn* B, float* SFA, float* SFB, DTypeOut* D,
    int* m_indptr, int max_m, int n, int k, int num_groups, hipStream_t stream);

}  // namespace group_gemm
}  // namespace flashinfer

void CutlassGroupGemmFP8GroupwiseScaledSM120(
    TensorView int_workspace_buffer, TensorView float_workspace_buffer, TensorView A, TensorView B,
    TensorView SFA, TensorView SFB, TensorView D, TensorView m_indptr, int64_t n, int64_t k,
    int64_t scale_granularity_m, int64_t scale_granularity_n, int64_t scale_granularity_k,
    std::string scale_major_mode) {
  hipSetDevice(float_workspace_buffer->device.device_id);
  auto stream = get_stream(D->device);
  int num_groups = m_indptr->shape[0] - 1;

  // Ensure scales are contiguous
  // Note: We keep the original shape and let the kernel's layout handle interpretation
  CHECK_CONTIGUOUS(SFA);
  CHECK_CONTIGUOUS(SFB);

  // Get max_m from SFA shape
  int max_m = SFA->shape[SFA->ndim > 1 ? 1 : 0];

  DISPATCH_DLPACK_INPUT_OUTPUT_DTYPE(A->dtype, D->dtype, c_type_in, c_type_out, [&] {
    return DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, [&] {
      return DISPATCH_SCALE_GRANULARITY(
          scale_granularity_m, scale_granularity_n, scale_granularity_k, SCALE_GRANULARITY_M,
          SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, [&] {
            using cutlass_t_in = cutlass_dtype_t<c_type_in>;
            using cutlass_t_out = cutlass_dtype_t<c_type_out>;
            auto status = flashinfer::group_gemm::CutlassFP8GroupwiseScaledGroupGEMMSM120<
                SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, SCALE_MAJOR_K,
                cutlass_t_in, cutlass_t_out>(
                static_cast<int*>(int_workspace_buffer->data),
                get_element_size(int_workspace_buffer) * int_workspace_buffer->shape[0],
                static_cast<float*>(float_workspace_buffer->data),
                get_element_size(float_workspace_buffer) * float_workspace_buffer->shape[0],
                static_cast<cutlass_t_in*>(A->data), static_cast<cutlass_t_in*>(B->data),
                static_cast<float*>(SFA->data), static_cast<float*>(SFB->data),
                static_cast<cutlass_t_out*>(D->data), static_cast<int*>(m_indptr->data), max_m, n,
                k, num_groups, stream);
            return status == hipSuccess;
          });
    });
  });
}
