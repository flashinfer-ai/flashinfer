/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/cutlass_utils.cuh>

#include "tvm_ffi_utils.h"

using namespace flashinfer;

#define DISPATCH_DLPACK_INPUT_OUTPUT_DTYPE(input_dtype, output_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                                                 \
    return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(output_dtype, c_type_out, [&] {                    \
      return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP8(input_dtype, c_type_in,                           \
                                                [&] { return __VA_ARGS__(); });                   \
    });                                                                                           \
  }()

#define DISPATCH_SCALE_GRANULARITY(scale_granularity_m, scale_granularity_n, scale_granularity_k, \
                                   SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, \
                                   ...)                                                           \
  [&]() -> bool {                                                                                 \
    /* SM120 Cooperative schedule uses 128x128x128 tile shape */                                  \
    /* TODO (yongwww): PingPong schedule (64x128x128) will need additional dispatch logic */      \
    constexpr int SCALE_GRANULARITY_K = 128;                                                      \
    if (scale_granularity_k != 128) {                                                             \
      TVM_FFI_ICHECK(false)                                                                       \
          << "SM120 requires scale_granularity_k=128. CUTLASS enforces ScaleGranularityK must "   \
             "equal tile shape K dimension (128 for both Cooperative and PingPong schedules).";   \
      return false;                                                                               \
    }                                                                                             \
    /* Support (1,128,128) and (128,128,128) as per SM100's approach */                           \
    if (scale_granularity_m == 1 && scale_granularity_n == 128) {                                 \
      constexpr int SCALE_GRANULARITY_M = 1;                                                      \
      constexpr int SCALE_GRANULARITY_N = 128;                                                    \
      return __VA_ARGS__();                                                                       \
    } else if (scale_granularity_m == 128 && scale_granularity_n == 128) {                        \
      constexpr int SCALE_GRANULARITY_M = 128;                                                    \
      constexpr int SCALE_GRANULARITY_N = 128;                                                    \
      return __VA_ARGS__();                                                                       \
    }                                                                                             \
    TVM_FFI_ICHECK(false) << "SM120: Unsupported scale granularity combination ("                 \
                          << scale_granularity_m << "," << scale_granularity_n << ","             \
                          << scale_granularity_k << ")";                                          \
    return false;                                                                                 \
  }()

#define DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, ...) \
  [&]() -> bool {                                                    \
    if (scale_major_mode == "K") {                                   \
      constexpr bool SCALE_MAJOR_K = true;                           \
      return __VA_ARGS__();                                          \
    } else if (scale_major_mode == "MN") {                           \
      constexpr bool SCALE_MAJOR_K = false;                          \
      return __VA_ARGS__();                                          \
    }                                                                \
    TVM_FFI_ICHECK(false) << "Unsupported Scale Major Mode";         \
    return false;                                                    \
  }()

namespace flashinfer {
namespace gemm {

template <int ScaleGranularityM, int ScaleGranularityN, int ScaleGranularityK, bool ScaleMajorK,
          typename DTypeIn, typename DTypeOut>
hipError_t CutlassGroupwiseScaledGEMMSM120(void* float_buffer, size_t float_buffer_size_in_bytes,
                                            DTypeIn* A_ptr, DTypeIn* B_ptr, float* SFA_ptr,
                                            float* SFB_ptr, DTypeOut* C_ptr, int m, int n, int k,
                                            int l, hipStream_t stream);

}  // namespace gemm
}  // namespace flashinfer

void CutlassGemmGroupwiseScaledSM120(Tensor float_workspace_buffer, Tensor A, Tensor B, Tensor SFA,
                                     Tensor SFB, Tensor C, int64_t scale_granularity_m,
                                     int64_t scale_granularity_n, int64_t scale_granularity_k,
                                     std::string scale_major_mode) {
  hipSetDevice(float_workspace_buffer->device.device_id);
  auto stream = get_stream(C->device);

  // Ensure scales are contiguous
  // Note: We keep the original shape and let the kernel's layout handle interpretation
  CHECK_CONTIGUOUS(SFA);
  CHECK_CONTIGUOUS(SFB);

  DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, [&] {
    return DISPATCH_DLPACK_INPUT_OUTPUT_DTYPE(A->dtype, C->dtype, c_type_in, c_type_out, [&] {
      return DISPATCH_SCALE_GRANULARITY(
          scale_granularity_m, scale_granularity_n, scale_granularity_k, SCALE_GRANULARITY_M,
          SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, [&] {
            using cutlass_t_in = cutlass_dtype_t<c_type_in>;
            using cutlass_t_out = cutlass_dtype_t<c_type_out>;

            // Handle both 2D and 3D tensors (BMM)
            int m, n, k, l;
            if (A->ndim == 2) {
              // 2D case: simple matrix multiplication
              m = A->shape[0];
              k = A->shape[1];
              n = B->shape[0];
              l = 1;  // no batch dimension
            } else if (A->ndim == 3) {
              // 3D case: batch matrix multiplication
              l = A->shape[0];  // batch size
              m = A->shape[1];  // per-batch m dimension
              k = A->shape[2];  // per-batch k dimension
              n = B->shape[2];  // per-batch n dimension (B is [batch, k, n] column-major)
            } else {
              return false;  // Unsupported tensor dimension
            }

            auto status = flashinfer::gemm::CutlassGroupwiseScaledGEMMSM120<
                SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, SCALE_MAJOR_K>(
                static_cast<void*>(float_workspace_buffer->data),
                get_element_size(float_workspace_buffer) * get_numel(float_workspace_buffer),
                static_cast<cutlass_t_in*>(A->data), static_cast<cutlass_t_in*>(B->data),
                static_cast<float*>(SFA->data), static_cast<float*>(SFB->data),
                static_cast<cutlass_t_out*>(C->data), m, n, k, l,
                stream);  // C is the output (D)
            return status == hipSuccess;
          });
    });
  });
}
