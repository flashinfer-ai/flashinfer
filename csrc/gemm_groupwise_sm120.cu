/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/cutlass_utils.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;

#define DISPATCH_PYTORCH_INPUT_OUTPUT_DTYPE(input_dtype, output_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                                                  \
    return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(output_dtype, c_type_out, [&] {                    \
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(input_dtype, c_type_in,                           \
                                                 [&] { return __VA_ARGS__(); });                   \
    });                                                                                            \
  }()

#define DISPATCH_SCALE_GRANULARITY(scale_granularity_m, scale_granularity_n, scale_granularity_k,  \
                                   SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K,  \
                                   ...)                                                            \
  [&]() -> bool {                                                                                  \
    /* SM120 Cooperative schedule uses 128x128x128 tile shape */                                   \
    /* TODO (yongwww): PingPong schedule (64x128x128) will need additional dispatch logic */       \
    constexpr int SCALE_GRANULARITY_K = 128;                                                       \
    if (scale_granularity_k != 128) {                                                              \
      TORCH_CHECK(                                                                                 \
          false,                                                                                   \
          "SM120 requires scale_granularity_k=128. CUTLASS enforces ScaleGranularityK must equal " \
          "tile shape K dimension (128 for both Cooperative and PingPong schedules).");            \
      return false;                                                                                \
    }                                                                                              \
    /* Support (1,128,128) and (128,128,128) as per SM100's approach */                            \
    if (scale_granularity_m == 1 && scale_granularity_n == 128) {                                  \
      constexpr int SCALE_GRANULARITY_M = 1;                                                       \
      constexpr int SCALE_GRANULARITY_N = 128;                                                     \
      return __VA_ARGS__();                                                                        \
    } else if (scale_granularity_m == 128 && scale_granularity_n == 128) {                         \
      constexpr int SCALE_GRANULARITY_M = 128;                                                     \
      constexpr int SCALE_GRANULARITY_N = 128;                                                     \
      return __VA_ARGS__();                                                                        \
    }                                                                                              \
    TORCH_CHECK(false, "SM120: Unsupported scale granularity combination (", scale_granularity_m,  \
                ",", scale_granularity_n, ",", scale_granularity_k, ")");                          \
    return false;                                                                                  \
  }()

#define DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, ...) \
  [&]() -> bool {                                                    \
    if (scale_major_mode == "K") {                                   \
      constexpr bool SCALE_MAJOR_K = true;                           \
      return __VA_ARGS__();                                          \
    } else if (scale_major_mode == "MN") {                           \
      constexpr bool SCALE_MAJOR_K = false;                          \
      return __VA_ARGS__();                                          \
    }                                                                \
    TORCH_CHECK(false, "Unsupported Scale Major Mode");              \
    return false;                                                    \
  }()

namespace flashinfer {
namespace gemm {

template <int ScaleGranularityM, int ScaleGranularityN, int ScaleGranularityK, bool ScaleMajorK,
          typename DTypeIn, typename DTypeOut>
hipError_t CutlassGroupwiseScaledGEMMSM120(void* float_buffer, size_t float_buffer_size_in_bytes,
                                            DTypeIn* A_ptr, DTypeIn* B_ptr, float* SFA_ptr,
                                            float* SFB_ptr, DTypeOut* C_ptr, int m, int n, int k,
                                            int l, hipStream_t stream);

}  // namespace gemm
}  // namespace flashinfer

void CutlassGemmGroupwiseScaledSM120(at::Tensor float_workspace_buffer, at::Tensor A, at::Tensor B,
                                     at::Tensor SFA, at::Tensor SFB, at::Tensor C,
                                     int64_t scale_granularity_m, int64_t scale_granularity_n,
                                     int64_t scale_granularity_k, std::string scale_major_mode) {
  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  // Ensure scales are contiguous
  // Note: We keep the original shape and let the kernel's layout handle interpretation
  at::Tensor SFA_contig = SFA.is_contiguous() ? SFA : SFA.contiguous();
  at::Tensor SFB_contig = SFB.is_contiguous() ? SFB : SFB.contiguous();

  DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, [&] {
    return DISPATCH_PYTORCH_INPUT_OUTPUT_DTYPE(
        A.scalar_type(), C.scalar_type(), c_type_in, c_type_out, [&] {
          return DISPATCH_SCALE_GRANULARITY(
              scale_granularity_m, scale_granularity_n, scale_granularity_k, SCALE_GRANULARITY_M,
              SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, [&] {
                using cutlass_t_in = cutlass_dtype_t<c_type_in>;
                using cutlass_t_out = cutlass_dtype_t<c_type_out>;

                // Handle both 2D and 3D tensors (BMM)
                int m, n, k, l;
                if (A.dim() == 2) {
                  // 2D case: simple matrix multiplication
                  m = A.size(0);
                  k = A.size(1);
                  n = B.size(0);
                  l = 1;  // no batch dimension
                } else if (A.dim() == 3) {
                  // 3D case: batch matrix multiplication
                  l = A.size(0);  // batch size
                  m = A.size(1);  // per-batch m dimension
                  k = A.size(2);  // per-batch k dimension
                  n = B.size(2);  // per-batch n dimension (B is [batch, k, n] column-major)
                } else {
                  return false;  // Unsupported tensor dimension
                }

                auto status = flashinfer::gemm::CutlassGroupwiseScaledGEMMSM120<
                    SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, SCALE_MAJOR_K>(
                    static_cast<void*>(float_workspace_buffer.data_ptr()),
                    float_workspace_buffer.element_size() * float_workspace_buffer.numel(),
                    static_cast<cutlass_t_in*>(A.data_ptr()),
                    static_cast<cutlass_t_in*>(B.data_ptr()),
                    static_cast<float*>(SFA_contig.data_ptr()),
                    static_cast<float*>(SFB_contig.data_ptr()),
                    static_cast<cutlass_t_out*>(C.data_ptr()), m, n, k, l,
                    stream);  // C is the output (D)
                return status == hipSuccess;
              });
        });
  });
}
