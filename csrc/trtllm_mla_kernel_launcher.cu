#include "hip/hip_runtime.h"
/*
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <flashinfer/semaphore_utils.cuh>
#include <flashinfer/trtllm/fmha/fmhaRunner.cuh>
#include <flashinfer/trtllm/fmha/gen_kernel_launcher.cuh>
#include <iostream>

// NOTE(Yingyi):
// dummy sliding window attention
// quantization not supported
namespace flashinfer {
template <Data_type CACHE_T>
void trtllm_paged_attention_mla_launcher(
    at::Tensor& out, at::Tensor& query, at::Tensor& key_value_cache, at::Tensor& workspace_buffer,
    double scale, at::Tensor& block_tables, at::Tensor& seq_lens, int64_t block_size,
    int64_t max_seq_len, int64_t qk_nope_head_dim, int64_t kv_lora_rank, int64_t qk_rope_head_dim,
    double bmm1_scale, double bmm2_scale, std::optional<int64_t> acc_q_len,
    std::optional<int64_t> max_attention_window_size,
    std::optional<int64_t> cyclic_attention_window_size) {
  int const num_seqs = query.size(0);
  int const batch_size = num_seqs;
  int const num_q_heads = query.size(1);
  int const num_kv_heads = 1;
  int head_size = query.size(2);
  int const beam_width = 1;                        // NOTE: beam_width always 1
  int const batch_beam = beam_width * batch_size;  // NOTE: batch_beam = batch_size
  int const max_num_blocks_per_seq = block_tables.size(-1);

  auto device = query.device();
  const auto stream = at::cuda::getCurrentCUDAStream(device.index());

  uint32_t tokens_per_page = block_size;

  // todo(Yingyi): use multi_block mode always true??
  bool use_multi_block = true;
  static auto fmha_runner = TllmGenFmhaRunner(CACHE_T, CACHE_T, DATA_TYPE_BF16);

  TllmGenFmhaRunnerParams runner_params;
  memset(&runner_params, 0, sizeof(runner_params));

  // Parameters to select kernels.
  runner_params.mMaskType = TrtllmGenAttentionMaskType::Dense;
  runner_params.mKernelType = FmhaKernelType::Generation;
  // Note that the tileScheduler and multiCtasKvMode will be automatically tuned when using
  // multi_block mode. Otherwise, always enable the persistent scheduler for better performance.
  runner_params.mTileScheduler =
      use_multi_block ? TileScheduler::Static : TileScheduler::Persistent;
  runner_params.mMultiCtasKvMode = use_multi_block;

  // Q buffer.
  // NOTE(Yingyi): no additional quantization input data field here
  runner_params.qPtr = query.data_ptr();

  // KV buffer
  // Paged KV
  runner_params.mQkvLayout = QkvLayout::PagedKv;
  runner_params.kvPtr = key_value_cache.data_ptr();
  runner_params.kvPageIdxPtr = block_tables.data_ptr<KVCachePageIndex>();
  runner_params.mMaxNumPagesPerSeqKv = max_num_blocks_per_seq;
  runner_params.mNumTokensPerPage = tokens_per_page;

  // num_kv_heads should be enough, but num_heads for safty at long seq len.
  size_t num_semaphores = batch_size * num_q_heads;

  // The partial buffers' pointers when the multiCtasKv mode is enabled.
  runner_params.multiCtasKvScratchPtr = reinterpret_cast<void*>(
      static_cast<char*>(workspace_buffer.data_ptr()) + num_semaphores * sizeof(uint32_t));
  runner_params.multiCtasKvCounterPtr = reinterpret_cast<int32_t*>(workspace_buffer.data_ptr());

  // The sequence lengths for K/V.
  runner_params.seqLensKvPtr = reinterpret_cast<int const*>(seq_lens.data_ptr<int>());

  runner_params.oPtr = out.data_ptr();
  // NOTE(yingyi): quantization is not supported for now
  runner_params.oSfPtr = nullptr;

  runner_params.mHeadDimQk = head_size;
  runner_params.mHeadDimV = kv_lora_rank;

  // NOTE: MLA use kv_heads = 1
  runner_params.mNumHeadsQ = num_q_heads;
  runner_params.mNumHeadsKv = num_kv_heads;
  runner_params.mNumHeadsQPerKv = num_q_heads / num_kv_heads;

  // NOTE: beam_width = 1
  runner_params.mBatchSize = batch_size;

  // It is used to construct contiguous kv cache TMA descriptors.
  auto const max_attention_window_size_opt = max_attention_window_size.value_or(max_seq_len);
  runner_params.mMaxSeqLenCacheKv = max_attention_window_size_opt;

  // This should be set to numDraftTokens + 1.
  auto const acc_q_len_opt = acc_q_len.value_or(batch_beam);
  runner_params.mMaxSeqLenQ = acc_q_len_opt / batch_beam;  // should be 1 if acc_q_len not provided
  runner_params.mMaxSeqLenKv = max_seq_len;
  runner_params.mSumOfSeqLensQ = int(batch_beam * runner_params.mMaxSeqLenQ);
  // Not used in the generation kernels as contiguous_kv or paged_kv layouts are used.
  runner_params.mSumOfSeqLensKv = int(batch_beam * runner_params.mMaxSeqLenKv);

  // The attention window size.
  // NOTE(Yingyi): for sliding window attention, temp to the fixed INT_MAX
  runner_params.mAttentionWindowSize = INT_MAX;
  // The chunked attention size.
  runner_params.mChunkedAttentionSize = INT_MAX;

  // The scaleQ that will be applied to the BMM1 output.
  // NOTE(Yingyi): set scale to be a api param, default 1.0
  // Q_SCALE & KV_SCALE not supported for now
  // runner_params.mScaleQ = scale * sqrt((float)(qk_nope_head_dim + qk_rope_head_dim)) /
  //                         sqrtf((float)(kv_lora_rank + qk_rope_head_dim));
  runner_params.mScaleQ = scale;

  // runner_params.mNumPagesInMemPool = INT_MAX;
  auto const [free_memory, total_memory] = getDeviceMemoryInfo(false);
  int max_head_dim_kv = head_size;
  // runner_params.mNumPagesInMemPool =
  //     total_memory / (runner_params.mNumHeadsKv * runner_params.mNumTokensPerPage *
  //                     max_head_dim_kv * get_size_in_bytes(CACHE_T));
  runner_params.mNumPagesInMemPool = 0;

  runner_params.mMultiProcessorCount = getMultiProcessorCount();
  runner_params.stream = stream;
  // NOTE (Yingyi): quantization, not supported for now
  runner_params.mSfStartTokenIdx = 0;

  runner_params.outputScale = bmm2_scale;
  runner_params.scaleSoftmaxLog2 = bmm1_scale;
  // if (CACHE_T == Data_type::DATA_TYPE_E4M3) {
  //   // NOTE(Yingyi): bmm1_scale and bmm2_scale are 1.0 could work already
  //   runner_params.outputScale = bmm2_scale;
  //   runner_params.scaleSoftmaxLog2 = bmm1_scale;

  //   // NOTE(Yingyi): if loadsScalesFromGmem enabled, the scales will be loaded from gmem
  //   // runner_params.outputScalePtr = bmm2_scale_tensor.has_value()
  //   //                                    ? bmm2_scale_tensor.value().data_ptr<float>()
  //   //                                    : nullptr;
  //   // runner_params.scaleSoftmaxLog2Ptr = bmm1_scale_tensor.has_value()
  //   //                                         ? bmm1_scale_tensor.value().data_ptr<float>()
  //   //                                         : nullptr;
  // }

  zero_gmem_semaphore_launcher(runner_params.multiCtasKvCounterPtr, num_semaphores,
                               /*enable_pdl=*/true, stream);

  fmha_runner.run(runner_params);
}

#define CALL_GEN_LAUNCHER(CACHE_T_ENUM)                                                         \
  trtllm_paged_attention_mla_launcher<CACHE_T_ENUM>(                                            \
      out, query, key_value_cache, workspace_buffer, scale, block_tables, seq_lens, block_size, \
      max_seq_len, qk_nope_head_dim, kv_lora_rank, qk_rope_head_dim, bmm1_scale, bmm2_scale,    \
      acc_q_len, max_attention_window_size, cyclic_attention_window_size);

// The following macro is used to dispatch the conversion function based on
// the data type of the key and value cache. The FN is a macro that calls a
// function with template<typename scalar_t, typename cache_t>
#define DISPATCH_BY_QKV_DTYPE(Q_DTYPE, KV_DTYPE, FN)                                               \
  FLASHINFER_CHECK(Q_DTYPE == KV_DTYPE,                                                            \
                   "Q_DTYPE must be the same as KV_DTYPE. Hybrid type is not supported for now."); \
  if (Q_DTYPE == at::ScalarType::Float8_e4m3fn) {                                                  \
    FN(Data_type::DATA_TYPE_E4M3);                                                                 \
  } else if (Q_DTYPE == at::ScalarType::BFloat16) {                                                \
    FN(Data_type::DATA_TYPE_BF16);                                                                 \
  } else {                                                                                         \
    TORCH_CHECK(false, "Unsupported input type of QKV type: ", Q_DTYPE);                           \
  }

void trtllm_paged_attention_mla(at::Tensor& out, at::Tensor& query, at::Tensor& key_value_cache,
                                at::Tensor& workspace_buffer, double scale,
                                at::Tensor& block_tables, at::Tensor& seq_lens, int64_t block_size,
                                int64_t max_seq_len, int64_t qk_nope_head_dim, int64_t kv_lora_rank,
                                int64_t qk_rope_head_dim, double bmm1_scale, double bmm2_scale,
                                std::optional<int64_t> acc_q_len,
                                std::optional<int64_t> max_attention_window_size,
                                std::optional<int64_t> cyclic_attention_window_size) {
  DISPATCH_BY_QKV_DTYPE(query.dtype(), key_value_cache.dtype(),
                        CALL_GEN_LAUNCHER);  // hybrid attention is not supported for now
}

namespace trtllm_cubin_loader {
#include <flashinfer/cubin_loader.h>
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_paged_attention_mla", trtllm_paged_attention_mla);
}

}  // namespace flashinfer
