/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/cutlass_utils.cuh>
#include <flashinfer/gemm/gemm_groupwise_sm100.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;

#define DISPATCH_PYTORCH_INPUT_OUTPUT_DTYPE(input_dtype, output_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                                                  \
    return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(output_dtype, c_type_out, [&] {                    \
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(input_dtype, c_type_in,                           \
                                                 [&] { return __VA_ARGS__(); });                   \
    });                                                                                            \
  }()

#define DISPATCH_SCALE_GRANULARITY(scale_granularity_m, scale_granularity_n, scale_granularity_k, \
                                   SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, \
                                   ...)                                                           \
  [&]() -> bool {                                                                                 \
    if (scale_granularity_m == 1 && scale_granularity_n == 128 && scale_granularity_k == 128) {   \
      constexpr int SCALE_GRANULARITY_M = 1;                                                      \
      constexpr int SCALE_GRANULARITY_N = 128;                                                    \
      constexpr int SCALE_GRANULARITY_K = 128;                                                    \
      return __VA_ARGS__();                                                                       \
    } else if (scale_granularity_m == 128 && scale_granularity_n == 128 &&                        \
               scale_granularity_k == 128) {                                                      \
      constexpr int SCALE_GRANULARITY_M = 128;                                                    \
      constexpr int SCALE_GRANULARITY_N = 128;                                                    \
      constexpr int SCALE_GRANULARITY_K = 128;                                                    \
      return __VA_ARGS__();                                                                       \
    }                                                                                             \
    TORCH_CHECK(false, "Unsupported scale granularity");                                          \
    return false;                                                                                 \
  }()

#define DISPATCH_MMA_SM(mma_sm, MMA_SM, ...)  \
  [&]() -> bool {                             \
    if (mma_sm == 1) {                        \
      constexpr int MMA_SM = 1;               \
      return __VA_ARGS__();                   \
    } else if (mma_sm == 2) {                 \
      constexpr int MMA_SM = 2;               \
      return __VA_ARGS__();                   \
    }                                         \
    TORCH_CHECK(false, "Unsupported MMA SM"); \
    return false;                             \
  }()

#define DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, ...) \
  [&]() -> bool {                                                    \
    if (scale_major_mode == "K") {                                   \
      constexpr bool SCALE_MAJOR_K = true;                           \
      return __VA_ARGS__();                                          \
    } else if (scale_major_mode == "MN") {                           \
      constexpr bool SCALE_MAJOR_K = false;                          \
      return __VA_ARGS__();                                          \
    }                                                                \
    TORCH_CHECK(false, "Unsupported Scale Major Mode");              \
    return false;                                                    \
  }()

void CutlassGemmGroupwiseScaledSM100(at::Tensor float_workspace_buffer, at::Tensor A, at::Tensor B,
                                     at::Tensor SFA, at::Tensor SFB, at::Tensor C,
                                     int64_t scale_granularity_m, int64_t scale_granularity_n,
                                     int64_t scale_granularity_k, std::string scale_major_mode,
                                     int64_t mma_sm) {
  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer.device());
  auto stream = at::cuda::getCurrentCUDAStream();
  DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, [&] {
    return DISPATCH_MMA_SM(mma_sm, MMA_SM, [&] {
      return DISPATCH_PYTORCH_INPUT_OUTPUT_DTYPE(
          A.scalar_type(), C.scalar_type(), c_type_in, c_type_out, [&] {
            return DISPATCH_SCALE_GRANULARITY(
                scale_granularity_m, scale_granularity_n, scale_granularity_k, SCALE_GRANULARITY_M,
                SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, [&] {
                  using cutlass_t_in = cutlass_dtype_t<c_type_in>;
                  using cutlass_t_out = cutlass_dtype_t<c_type_out>;
                  auto status = flashinfer::gemm::CutlassGroupwiseScaledGEMMSM100<
                      SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, SCALE_MAJOR_K,
                      MMA_SM>(
                      static_cast<float*>(float_workspace_buffer.data_ptr()),
                      float_workspace_buffer.element_size() * float_workspace_buffer.size(0),
                      static_cast<cutlass_t_in*>(A.data_ptr()),
                      static_cast<cutlass_t_in*>(B.data_ptr()), static_cast<float*>(SFA.data_ptr()),
                      static_cast<float*>(SFB.data_ptr()),
                      static_cast<cutlass_t_out*>(C.data_ptr()), A.size(0), B.size(0), A.size(1), 1,
                      stream);
                  TORCH_CHECK(status == hipSuccess,
                              "Failed to run cutlass gemm groupwise scaled sm100",
                              hipGetErrorString(status));
                  return true;
                });
          });
    });
  });
}
