/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/cutlass_utils.cuh>

#include "tvm_ffi_utils.h"

using namespace flashinfer;

#define DISPATCH_DLPACK_INPUT_OUTPUT_DTYPE(input_dtype, output_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                                                 \
    return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(output_dtype, c_type_out, [&] {                    \
      return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP8(input_dtype, c_type_in,                           \
                                                [&] { return __VA_ARGS__(); });                   \
    });                                                                                           \
  }()

#define DISPATCH_SCALE_GRANULARITY(scale_granularity_m, scale_granularity_n, scale_granularity_k, \
                                   SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, \
                                   ...)                                                           \
  [&]() -> bool {                                                                                 \
    if (scale_granularity_m == 1 && scale_granularity_n == 128 && scale_granularity_k == 128) {   \
      constexpr int SCALE_GRANULARITY_M = 1;                                                      \
      constexpr int SCALE_GRANULARITY_N = 128;                                                    \
      constexpr int SCALE_GRANULARITY_K = 128;                                                    \
      return __VA_ARGS__();                                                                       \
    } else if (scale_granularity_m == 128 && scale_granularity_n == 128 &&                        \
               scale_granularity_k == 128) {                                                      \
      constexpr int SCALE_GRANULARITY_M = 128;                                                    \
      constexpr int SCALE_GRANULARITY_N = 128;                                                    \
      constexpr int SCALE_GRANULARITY_K = 128;                                                    \
      return __VA_ARGS__();                                                                       \
    }                                                                                             \
    TVM_FFI_ICHECK(false) << "Unsupported scale granularity";                                     \
    return false;                                                                                 \
  }()

#define DISPATCH_MMA_SM(mma_sm, MMA_SM, ...)       \
  [&]() -> bool {                                  \
    if (mma_sm == 1) {                             \
      constexpr int MMA_SM = 1;                    \
      return __VA_ARGS__();                        \
    } else if (mma_sm == 2) {                      \
      constexpr int MMA_SM = 2;                    \
      return __VA_ARGS__();                        \
    }                                              \
    TVM_FFI_ICHECK(false) << "Unsupported MMA SM"; \
    return false;                                  \
  }()

#define DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, ...) \
  [&]() -> bool {                                                    \
    if (scale_major_mode == "K") {                                   \
      constexpr bool SCALE_MAJOR_K = true;                           \
      return __VA_ARGS__();                                          \
    } else if (scale_major_mode == "MN") {                           \
      constexpr bool SCALE_MAJOR_K = false;                          \
      return __VA_ARGS__();                                          \
    }                                                                \
    TVM_FFI_ICHECK(false) << "Unsupported Scale Major Mode";         \
    return false;                                                    \
  }()

namespace flashinfer {
namespace gemm {

template <int ScaleGranularityM, int ScaleGranularityN, int ScaleGranularityK, bool ScaleMajorK,
          int MmaSM, typename DTypeIn, typename DTypeOut>
hipError_t CutlassGroupwiseScaledGEMMSM100(void* float_buffer, size_t float_buffer_size_in_bytes,
                                            DTypeIn* A_ptr, DTypeIn* B_ptr, float* SFA_ptr,
                                            float* SFB_ptr, DTypeOut* C_ptr, int m, int n, int k,
                                            int l, hipStream_t stream);

}  // namespace gemm
}  // namespace flashinfer

void CutlassGemmGroupwiseScaledSM100(TensorView float_workspace_buffer, TensorView A, TensorView B,
                                     TensorView SFA, TensorView SFB, TensorView C,
                                     int64_t scale_granularity_m, int64_t scale_granularity_n,
                                     int64_t scale_granularity_k, std::string scale_major_mode,
                                     int64_t mma_sm) {
  hipSetDevice(float_workspace_buffer->device.device_id);
  const hipStream_t stream = get_stream(C->device);
  DISPATCH_SCALE_MAJOR_K(scale_major_mode, SCALE_MAJOR_K, [&] {
    return DISPATCH_MMA_SM(mma_sm, MMA_SM, [&] {
      return DISPATCH_DLPACK_INPUT_OUTPUT_DTYPE(A->dtype, C->dtype, c_type_in, c_type_out, [&] {
        return DISPATCH_SCALE_GRANULARITY(
            scale_granularity_m, scale_granularity_n, scale_granularity_k, SCALE_GRANULARITY_M,
            SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, [&] {
              using cutlass_t_in = cutlass_dtype_t<c_type_in>;
              using cutlass_t_out = cutlass_dtype_t<c_type_out>;
              auto status = flashinfer::gemm::CutlassGroupwiseScaledGEMMSM100<
                  SCALE_GRANULARITY_M, SCALE_GRANULARITY_N, SCALE_GRANULARITY_K, SCALE_MAJOR_K,
                  MMA_SM>(
                  static_cast<float*>(float_workspace_buffer->data),
                  get_element_size(float_workspace_buffer) * float_workspace_buffer->shape[0],
                  static_cast<cutlass_t_in*>(A->data), static_cast<cutlass_t_in*>(B->data),
                  static_cast<float*>(SFA->data), static_cast<float*>(SFB->data),
                  static_cast<cutlass_t_out*>(C->data), A->shape[0], B->shape[0], A->shape[1], 1,
                  stream);
              TVM_FFI_ICHECK_EQ(status, hipSuccess)
                  << "Failed to run cutlass gemm groupwise scaled sm100"
                  << hipGetErrorString(status);
              return true;
            });
      });
    });
  });
}
