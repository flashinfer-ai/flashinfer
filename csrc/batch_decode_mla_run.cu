#include <flashinfer/attention/decode.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <optional>

#include "mla_config.inc"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

void BatchDecodeWithPagedKVCacheRunMLA(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q_nope, at::Tensor q_pe,
    at::Tensor paged_ckv_cache, at::Tensor paged_kpe_cache, at::Tensor paged_kv_indptr,
    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len, at::Tensor o, float sm_scale,
    int window_left, float logits_soft_cap, float rope_scale, float rope_theta,
    std::optional<at::Tensor> maybe_lse, int64_t hip_stream) {
  DecodePlanInfo plan_info;
  plan_info.FromVector(plan_info_vec);

  auto device = q_nope.device();
  int64_t batch_size = q_nope.size(0);
  int64_t num_qo_heads = q_nope.size(1);
  int64_t page_size = paged_ckv_cache.size(1);

  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == batch_size, lse.size(0), q_nope.size(0));
    TORCH_CHECK(lse.size(1) == num_qo_heads, lse.size(1), q_nope.size(1));
  }

  TORCH_CHECK(logits_soft_cap >= 0.f, "logits_soft_cap must be non-negative");

  void* float_buffer = static_cast<void*>(float_workspace_buffer.data_ptr());
  void* int_buffer = static_cast<void*>(int_workspace_buffer.data_ptr());

  paged_kv_mla_t<DTypeKV, IdType> paged_kv(
      page_size, HEAD_DIM_CKV, HEAD_DIM_KPE, batch_size,
      static_cast<DTypeKV*>(paged_ckv_cache.data_ptr()), paged_ckv_cache.strides().data(),
      static_cast<DTypeKV*>(paged_kpe_cache.data_ptr()), paged_kpe_cache.strides().data(),
      static_cast<IdType*>(paged_kv_indices.data_ptr()),
      static_cast<IdType*>(paged_kv_indptr.data_ptr()),
      static_cast<IdType*>(paged_kv_last_page_len.data_ptr()));
  Params params(static_cast<DTypeQ*>(q_nope.data_ptr()), static_cast<DTypeQ*>(q_pe.data_ptr()),
                /*q_offset=*/nullptr, paged_kv, static_cast<DTypeO*>(o.data_ptr()),
                /*lse=*/(maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr),
                num_qo_heads, window_left, logits_soft_cap, sm_scale, rope_scale, rope_theta);

  DTypeO* tmp_v = nullptr;
  float* tmp_s = nullptr;
  params.request_indices =
      GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.request_indices_offset);
  params.kv_tile_indices =
      GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.kv_tile_indices_offset);
  params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.o_indptr_offset);
  params.kv_chunk_size_ptr =
      GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.kv_chunk_size_ptr_offset);
  if (plan_info.split_kv) {
    tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer, plan_info.v_offset);
    tmp_s = GetPtrFromBaseOffset<float>(float_buffer, plan_info.s_offset);
    if (plan_info.enable_cuda_graph) {
      params.block_valid_mask =
          GetPtrFromBaseOffset<bool>(int_buffer, plan_info.block_valid_mask_offset);
    }
  }
  params.padded_batch_size = plan_info.padded_batch_size;

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status =
      BatchDecodeWithPagedKVCacheDispatchedMLA<HEAD_DIM_CKV, HEAD_DIM_KPE, AttentionVariant,
                                               Params>(params, tmp_v, tmp_s, /*stream=*/stream);
  TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCache failed with error ",
              hipGetErrorString(status));
}
