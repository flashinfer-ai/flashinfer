#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/cuda/EmptyTensor.h>
#include <hip/hip_fp16.h>

#include <cstddef>
#include <cstdint>
#include <functional>
#include <type_traits>
#include <vector>

#include "flashinfer/gemm/cutlass_gemm_configs.h"
#include "flashinfer/gemm/fp4_gemm_cutlass.h"
#include "flashinfer/gemm/fp4_gemm_cutlass_template.h"
#include "pytorch_extension_utils.h"

using flashinfer::gemm::ClusterShape;
using flashinfer::gemm::CutlassFp4GemmRunner;
using flashinfer::gemm::CutlassFp4GemmRunnerInterface;
using flashinfer::gemm::CutlassGemmConfig;
using flashinfer::gemm::CutlassTileConfigSM100;
using flashinfer::gemm::EpilogueScheduleType;
using flashinfer::gemm::FP4GemmType;
using flashinfer::gemm::MainloopScheduleType;

namespace flashinfer {
namespace gemm {
template class CutlassFp4GemmRunner<__hip_bfloat16, FP4GemmType::W4A4_NVFP4_NVFP4>;
template class CutlassFp4GemmRunner<half, FP4GemmType::W4A4_NVFP4_NVFP4>;
}  // namespace gemm
}  // namespace flashinfer

namespace torch_ext {

namespace {

CutlassGemmConfig getFp4GemmConfig(int64_t m, int64_t n, int64_t k, int64_t tactic) {
  auto getCutlassFp4GemmConfigs = []() {
    CutlassFp4GemmRunner<__hip_bfloat16, FP4GemmType::W4A4_NVFP4_NVFP4> gemmRunner;
    return gemmRunner.getConfigs();
  };
  static std::vector<CutlassGemmConfig> globalConfigs = getCutlassFp4GemmConfigs();
  TORCH_CHECK(tactic >= 0 && tactic < globalConfigs.size(), "tactic must be between 0 and ",
              globalConfigs.size());
  return globalConfigs[tactic];
}

template <typename T>
void runGemm(at::Tensor& out, at::Tensor const& mat1, at::Tensor const& mat2,
             at::Tensor const& mat1Scale, at::Tensor const& mat2Scale,
             at::Tensor const& globalScale, int64_t m, int64_t n, int64_t k, int64_t batch_count,
             CutlassGemmConfig const& gemmConfig, at::Tensor workspace_buffer) {
  CutlassFp4GemmRunner<T, FP4GemmType::W4A4_NVFP4_NVFP4> gemmRunner;

  int64_t const required_workspace_size = gemmRunner.getWorkspaceSize(m, n, k, batch_count);
  int64_t const provided_workspace_size =
      workspace_buffer.numel() * workspace_buffer.element_size();

  auto runKernel = [&](void* workspace) {
    gemmRunner.gemm(out.data_ptr(), mat1.const_data_ptr(), mat2.const_data_ptr(),
                    mat1Scale.const_data_ptr(), mat2Scale.const_data_ptr(),
                    globalScale.data_ptr<float>(), m, n, k, batch_count, gemmConfig,
                    reinterpret_cast<char*>(workspace), required_workspace_size,
                    at::cuda::getCurrentCUDAStream(mat1.get_device()));
  };

  if (provided_workspace_size < required_workspace_size) {
    at::Tensor new_workspace = at::detail::empty_cuda(
        {required_workspace_size}, at::ScalarType::Char, mat1.device(), std::nullopt);

    runKernel(new_workspace.data_ptr());
  } else {
    runKernel(workspace_buffer.data_ptr());
  }
}

constexpr auto FLOAT4_E2M1X2 = at::ScalarType::Byte;  // uint8_t
constexpr auto SF_DTYPE = at::ScalarType::Byte;       // uint8_t

// mat1: [B, M, K / 2], FLOAT4_E2M1X2 or [B, M, K], FLOAT8_E4M3FN
// mat2: [B, N, K / 2], FLOAT4_E2M1X2
// out: [B, M, N], fp16/bf16/fp32
// mat1Scale: ceil(M / 128) * 128 * ceil(K / sfVecSize / 4) * 4, SF_DTYPE (UE4M3 or UE8M0)
// mat2Scale: ceil(N / 128) * 128 * ceil(K / sfVecSize / 4) * 4, SF_DTYPE (UE4M3 or UE8M0)
// globalScale: [1], 1 / (((448 * 6) / mat1.abs().max()) * ((448 * 6) / mat2.abs().max()))
// B = 1 for GEMM op as a special case
at::Tensor fp4_bmm_impl(at::Tensor const& mat1, at::Tensor const& mat2, at::Tensor const& mat1Scale,
                        at::Tensor const& mat2Scale, at::Tensor const& globalScale, at::Tensor out,
                        at::Tensor workspace_buffer, int64_t tactic) {
  CHECK_INPUT_AND_TYPE(mat1, FLOAT4_E2M1X2);
  CHECK_INPUT_AND_TYPE(mat2, FLOAT4_E2M1X2);

  int mat2_k_scale = 1;

  CHECK_INPUT_AND_TYPE(mat1Scale, SF_DTYPE);
  CHECK_INPUT_AND_TYPE(mat2Scale, SF_DTYPE);

  CHECK_INPUT_AND_TYPE(globalScale, at::ScalarType::Float);

  int64_t m, n, k, b;
  if (mat1.dim() == 2) {
    TORCH_CHECK(mat2.dim() == 2, "mat2 must be a matrix");
    TORCH_CHECK(mat1.sizes()[1] == mat2.sizes()[1] * mat2_k_scale,
                "mat1 and mat2 shapes cannot be multiplied (", mat1.sizes()[0], "x",
                mat1.sizes()[1], " and ", mat2.sizes()[0], "x", mat2.sizes()[1], ")");
    m = mat1.sizes()[0];
    n = mat2.sizes()[0];
    k = mat2.sizes()[1] * 2;
    b = 1;
  } else if (mat1.dim() == 3) {
    TORCH_CHECK(mat2.dim() == 3, "mat2 must be a batch of matrices");
    TORCH_CHECK(mat1.sizes()[0] == mat2.sizes()[0], "mat1 and mat2 must have the same batch size (",
                mat1.sizes()[0], " and ", mat2.sizes()[0], ")");
    TORCH_CHECK(mat1.sizes()[2] == mat2.sizes()[2] * mat2_k_scale,
                "mat1 and mat2 shapes cannot be multiplied (", mat1.sizes()[1], "x",
                mat1.sizes()[2], " and ", mat2.sizes()[1], "x", mat2.sizes()[2], ")");
    m = mat1.sizes()[1];
    n = mat2.sizes()[1];
    k = mat2.sizes()[2] * 2;
    b = mat1.sizes()[0];
  } else {
    C10_THROW_ERROR(NotImplementedError, "mat1 must be a matrix or a batch of matrices");
  }

  // No heuristic for now, we rely on the autotuner to select the best tactic.
  if (tactic == -1) {
    tactic = 0;
  }
  auto config = getFp4GemmConfig(m, n, k, tactic);

  constexpr int alignment = 32;
  TORCH_CHECK(k % alignment == 0, "Expected k to be divisible by ", alignment,
              ", but got mat1 shape: (", mat1.sizes()[0], "x", mat1.sizes()[1], "), k: ", k, ".");
  TORCH_CHECK(n % alignment == 0, "Expected n to be divisible by ", alignment,
              ", but got mat2 shape: (", mat2.sizes()[0], "x", mat2.sizes()[1], ").");

  // Validate out dimensions
  std::vector<int64_t> out_shape =
      mat1.dim() == 2 ? std::vector<int64_t>{m, n} : std::vector<int64_t>{b, m, n};
  TORCH_CHECK(out.dim() == out_shape.size(), "out must have ", out_shape.size(),
              " dimensions, but got ", out.dim());
  for (int i = 0; i < out_shape.size(); ++i) {
    TORCH_CHECK(out.sizes()[i] == out_shape[i], "out shape mismatch at dimension ", i,
                ": expected ", out_shape[i], ", got ", out.sizes()[i]);
  }

  c10::ScalarType out_dtype = out.scalar_type();

  switch (out_dtype) {
    case at::ScalarType::Half:
      runGemm<half>(out, mat1, mat2, mat1Scale, mat2Scale, globalScale, m, n, k, b, config,
                    workspace_buffer);
      break;
    case at::ScalarType::BFloat16:
      runGemm<__hip_bfloat16>(out, mat1, mat2, mat1Scale, mat2Scale, globalScale, m, n, k, b, config,
                             workspace_buffer);
      break;
    default:
      TORCH_CHECK(false, "out_dtype must be one of fp16/bf16.");
  }
  return out;
}

}  // namespace

at::Tensor fp4_gemm(at::Tensor const& mat1, at::Tensor const& mat2, at::Tensor const& mat1Scale,
                    at::Tensor const& mat2Scale, at::Tensor const& globalScale, at::Tensor out,
                    at::Tensor workspace_buffer, int64_t tactic) {
  return fp4_bmm_impl(mat1, mat2, mat1Scale, mat2Scale, globalScale, out, workspace_buffer, tactic);
}

int64_t fp4_gemm_tactic_num() {
  auto getCutlassConfigs = []() {
    CutlassFp4GemmRunner<__hip_bfloat16, FP4GemmType::W4A4_NVFP4_NVFP4> gemmRunner;
    return gemmRunner.getConfigs();
  };
  static int64_t totalTactics = getCutlassConfigs().size();
  return totalTactics;
}

}  // namespace torch_ext

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("fp4_gemm", &torch_ext::fp4_gemm);
  m.def("fp4_gemm_tactic_num", &torch_ext::fp4_gemm_tactic_num);
}
