/*
 * Copyright (c) 2023-2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "batch_prefill_sm90_config.inc"
#include "pytorch_extension_utils.h"

std::vector<int64_t> BatchPrefillWithKVCacheSM90Plan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor qo_indptr, at::Tensor kv_indptr,
    at::Tensor kv_len_arr, unsigned total_num_rows, unsigned int batch_size,
    unsigned int num_qo_heads, unsigned int num_kv_heads, unsigned int page_size,
    bool enable_cuda_graph, unsigned int head_dim, bool causal, int64_t hip_stream);

void BatchPrefillWithRaggedKVCacheSM90Run(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor k, at::Tensor v,
    at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor o, std::optional<at::Tensor> maybe_lse,
    unsigned int mask_mode_code, unsigned int layout, int32_t window_left ADDITIONAL_FUNC_PARAMS,
    int64_t hip_stream);

void BatchPrefillWithPagedKVCacheSM90Run(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, at::Tensor qo_indptr, at::Tensor paged_kv_indptr,
    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len, at::Tensor o,
    std::optional<at::Tensor> maybe_lse, unsigned int mask_mode_code, unsigned int layout,
    int32_t window_left ADDITIONAL_FUNC_PARAMS, int64_t hip_stream);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("plan", &BatchPrefillWithKVCacheSM90Plan,
        "Batch-request prefill attention with KV-Cache plan");
  m.def("ragged_run", &BatchPrefillWithRaggedKVCacheSM90Run,
        "Batch-request prefill attention with KV-Cache operator");
  m.def("paged_run", &BatchPrefillWithPagedKVCacheSM90Run,
        "Batch-request prefill attention with KV-Cache operator");
}
