/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/pos_enc.cuh>
#include <optional>

#include "batch_prefill_config.inc"
#include "pytorch_extension_utils.h"

namespace flashinfer {

template <uint32_t CTA_TILE_Q, uint32_t HEAD_DIM, PosEncodingMode POS_ENCODING_MODE,
          bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE, typename AttentionVariant,
          typename Params>
hipError_t BatchPrefillWithPagedKVCacheDispatched(Params params, typename Params::DTypeO* tmp_v,
                                                   float* tmp_s, hipStream_t stream);

template <uint32_t CTA_TILE_Q, uint32_t HEAD_DIM, PosEncodingMode POS_ENCODING_MODE,
          bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE, typename AttentionVariant,
          typename Params>
hipError_t BatchPrefillWithRaggedKVCacheDispatched(Params params, typename Params::DTypeO* tmp_v,
                                                    float* tmp_s, hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

std::vector<int64_t> BatchPrefillWithKVCachePlan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor qo_indptr, at::Tensor kv_indptr,
    at::Tensor kv_len_arr, unsigned total_num_rows, unsigned int batch_size,
    unsigned int num_qo_heads, unsigned int num_kv_heads, unsigned int page_size,
    bool enable_cuda_graph, unsigned int head_dim, bool causal, int64_t hip_stream) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();

  PrefillPlanInfo plan_info;

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status = PrefillPlan<IdType>(
      float_workspace_buffer.data_ptr(), float_workspace_size_in_bytes,
      int_workspace_buffer.data_ptr(), page_locked_int_workspace_buffer.data_ptr(),
      int_workspace_size_in_bytes, plan_info, qo_indptr.data_ptr<IdType>(),
      kv_indptr.data_ptr<IdType>(), total_num_rows, batch_size, num_qo_heads, num_kv_heads,
      head_dim, page_size, enable_cuda_graph, /*sizeof_dtype_o=*/2, stream);

  TORCH_CHECK(status == hipSuccess,
              "Failed to plan prefill with error: ", hipGetErrorString(status));

  return plan_info.ToVector();
}

void BatchPrefillWithRaggedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor k, at::Tensor v,
    at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor o, std::optional<at::Tensor> maybe_lse,
    unsigned int mask_mode_code, unsigned int layout, int32_t window_left ADDITIONAL_FUNC_PARAMS,
    int64_t hip_stream) {
  PrefillPlanInfo plan_info;
  plan_info.FromVector(plan_info_vec);
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);

  int64_t num_qo_heads = q.size(1);
  int64_t head_dim = q.size(2);
  int64_t num_kv_heads = (kv_layout == QKVLayout::kNHD) ? k.size(1) : k.size(0);
  uint32_t q_stride_n = q.stride(0), q_stride_h = q.stride(1), kv_stride_n, kv_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    kv_stride_n = k.stride(0);
    kv_stride_h = k.stride(1);
  } else {
    kv_stride_h = k.stride(0);
    kv_stride_n = k.stride(1);
  }

  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == q.size(0), lse.size(0), q.size(0));
    TORCH_CHECK(lse.size(1) == q.size(1), lse.size(1), q.size(1));
  }

  void* float_buffer_ptr = float_workspace_buffer.data_ptr();
  void* int_buffer_ptr = int_workspace_buffer.data_ptr();

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM, POS_ENCODING_MODE, USE_SLIDING_WINDOW,
      USE_LOGITS_SOFT_CAP, USE_FP16_QK_REDUCTION, AttentionVariant, RaggedParams, PagedParams, [&] {
        RaggedParams params;

        params.q = static_cast<DTypeQ*>(q.data_ptr());
        params.k = static_cast<DTypeKV*>(k.data_ptr());
        params.v = static_cast<DTypeKV*>(v.data_ptr());
        params.o = static_cast<DTypeO*>(o.data_ptr());
        params.lse = maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
        params.q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());
        params.kv_indptr = static_cast<IdType*>(kv_indptr.data_ptr());
        params.num_qo_heads = num_qo_heads;
        params.num_kv_heads = num_kv_heads;
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.kv_stride_n = kv_stride_n;
        params.kv_stride_h = kv_stride_h;
        params.window_left = window_left;

        params.request_indices = nullptr;
        params.qo_tile_indices = nullptr;
        params.kv_tile_indices = nullptr;
        params.merge_indptr = nullptr;
        params.o_indptr = nullptr;
        params.kv_chunk_size_ptr = nullptr;
        params.block_valid_mask = nullptr;
        params.total_num_rows = nullptr;
        params.max_total_num_rows = 0;
        params.padded_batch_size = 0;
        params.partition_kv = false;

        ADDITIONAL_PARAMS_SETTER

        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;

        params.request_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.kv_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
        params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
        params.kv_chunk_size_ptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset);
        if (plan_info.split_kv) {
          params.merge_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
          tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
          tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
          if (plan_info.enable_cuda_graph) {
            params.block_valid_mask =
                GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
          }
        }
        params.padded_batch_size = plan_info.padded_batch_size;
        params.max_total_num_rows = plan_info.total_num_rows;
        if (plan_info.enable_cuda_graph) {
          params.total_num_rows =
              GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
        }

        hipError_t status = hipSuccess;

        DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q, CTA_TILE_Q, {
          status = flashinfer::BatchPrefillWithRaggedKVCacheDispatched<
              CTA_TILE_Q, HEAD_DIM, POS_ENCODING_MODE,
              /*use_fp16_qk_reduction=*/USE_FP16_QK_REDUCTION, MASK_MODE, AttentionVariant,
              RaggedParams>(params, tmp_v, tmp_s, stream);
        });

        TORCH_CHECK(status == hipSuccess, "BatchPrefillWithRaggedKVCache failed with error ",
                    hipGetErrorString(status));
        return true;
      });
}

void BatchPrefillWithPagedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, at::Tensor qo_indptr, at::Tensor paged_kv_indptr,
    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len, at::Tensor o,
    std::optional<at::Tensor> maybe_lse, unsigned int mask_mode_code, unsigned int layout,
    int32_t window_left ADDITIONAL_FUNC_PARAMS, int64_t hip_stream) {
  PrefillPlanInfo plan_info;
  plan_info.FromVector(plan_info_vec);
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  auto device = q.device();
  int64_t batch_size = paged_kv_indptr.size(0) - 1;
  int64_t num_qo_heads = q.size(1);
  int64_t num_kv_heads, page_size;
  uint32_t head_dim = q.size(2);
  if (kv_layout == QKVLayout::kHND) {
    num_kv_heads = paged_k_cache.size(1);
    page_size = paged_k_cache.size(2);
  } else {
    page_size = paged_k_cache.size(1);
    num_kv_heads = paged_k_cache.size(2);
  }

  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == q.size(0), lse.size(0), q.size(0));
    TORCH_CHECK(lse.size(1) == q.size(1), lse.size(1), q.size(1));
  }

  void* float_buffer_ptr = static_cast<void*>(float_workspace_buffer.data_ptr());
  void* int_buffer_ptr = static_cast<void*>(int_workspace_buffer.data_ptr());

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = paged_k_cache.scalar_type();

  // get q_stride_n and q_stride_h
  const auto q_stride_n = q.stride(0);
  const auto q_stride_h = q.stride(1);

  // get kv_cache_strides
  const int64_t* kv_cache_strides = nullptr;
  auto k_strides = paged_k_cache.strides();
  auto v_strides = paged_v_cache.strides();
  TORCH_CHECK(k_strides == v_strides, "k/v strides must be identical");
  kv_cache_strides = k_strides.data();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM, POS_ENCODING_MODE, USE_SLIDING_WINDOW,
      USE_LOGITS_SOFT_CAP, USE_FP16_QK_REDUCTION, AttentionVariant, RaggedParams, PagedParams, [&] {
        PagedParams params;

        params.q = static_cast<DTypeQ*>(q.data_ptr());
        paged_kv_t<DTypeKV, IdType> paged_kv(
            num_kv_heads, page_size, HEAD_DIM, batch_size, kv_layout,
            static_cast<DTypeKV*>(paged_k_cache.data_ptr()),
            static_cast<DTypeKV*>(paged_v_cache.data_ptr()), kv_cache_strides,
            static_cast<IdType*>(paged_kv_indices.data_ptr()),
            static_cast<IdType*>(paged_kv_indptr.data_ptr()),
            static_cast<IdType*>(paged_kv_last_page_len.data_ptr()));
        params.paged_kv = paged_kv;
        params.q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());
        params.o = static_cast<DTypeO*>(o.data_ptr());

        params.lse = maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
        params.num_qo_heads = num_qo_heads;
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.window_left = window_left;

        params.request_indices = nullptr;
        params.qo_tile_indices = nullptr;
        params.kv_tile_indices = nullptr;
        params.merge_indptr = nullptr;
        params.o_indptr = nullptr;
        params.kv_chunk_size_ptr = nullptr;
        params.block_valid_mask = nullptr;
        params.total_num_rows = nullptr;
        params.max_total_num_rows = 0;
        params.padded_batch_size = 0;
        params.partition_kv = false;

        ADDITIONAL_PARAMS_SETTER

        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;

        params.request_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.kv_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
        params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
        params.kv_chunk_size_ptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset);
        if (plan_info.split_kv) {
          params.merge_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
          tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
          tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
          if (plan_info.enable_cuda_graph) {
            params.block_valid_mask =
                GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
          }
        }
        params.padded_batch_size = plan_info.padded_batch_size;
        params.max_total_num_rows = plan_info.total_num_rows;
        if (plan_info.enable_cuda_graph) {
          params.total_num_rows =
              GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
        }

        hipError_t status = hipSuccess;

        DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q, CTA_TILE_Q, {
          status = flashinfer::BatchPrefillWithPagedKVCacheDispatched<
              CTA_TILE_Q, HEAD_DIM, POS_ENCODING_MODE,
              /*use_fp16_qk_reduction=*/USE_FP16_QK_REDUCTION, MASK_MODE, AttentionVariant,
              PagedParams>(params, tmp_v, tmp_s, stream);
        });

        TORCH_CHECK(status == hipSuccess, "BatchPrefillWithPagedKVCache failed with error ",
                    hipGetErrorString(status));
        return true;
      });
}
