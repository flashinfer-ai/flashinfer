/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/pos_enc.cuh>
#include <optional>

#include "batch_prefill_config.inc"
#include "pytorch_extension_utils.h"
#include "pytorch_conversion_utils.h"

namespace flashinfer {

template <uint32_t CTA_TILE_Q, uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO,
          PosEncodingMode POS_ENCODING_MODE, bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE,
          typename AttentionVariant, typename Params>
hipError_t BatchPrefillWithPagedKVCacheDispatched(Params params, typename Params::DTypeO* tmp_v,
                                                   float* tmp_s, hipStream_t stream);

template <uint32_t CTA_TILE_Q, uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO,
          PosEncodingMode POS_ENCODING_MODE, bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE,
          typename AttentionVariant, typename Params>
hipError_t BatchPrefillWithRaggedKVCacheDispatched(Params params, typename Params::DTypeO* tmp_v,
                                                    float* tmp_s, hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

at::Tensor BatchPrefillWithKVCachePlan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor qo_indptr, at::Tensor kv_indptr,
    at::Tensor kv_len_arr, int64_t total_num_rows, int64_t batch_size,
    int64_t num_qo_heads, int64_t num_kv_heads, int64_t page_size,
    bool enable_cuda_graph, int64_t head_dim_qk, int64_t head_dim_vo, bool causal,
    int64_t hip_stream) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();

  PrefillPlanInfo plan_info;

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status = PrefillPlan<IdType>(
      float_workspace_buffer.data_ptr(), float_workspace_size_in_bytes,
      int_workspace_buffer.data_ptr(), page_locked_int_workspace_buffer.data_ptr(),
      int_workspace_size_in_bytes, plan_info, qo_indptr.data_ptr<IdType>(),
      kv_indptr.data_ptr<IdType>(), total_num_rows, batch_size, num_qo_heads, num_kv_heads,
      head_dim_qk, head_dim_vo, page_size, enable_cuda_graph, /*sizeof_dtype_o=*/2, stream);

  TORCH_CHECK(status == hipSuccess,
              "Failed to plan prefill with error: ", hipGetErrorString(status));

  return vec_to_tensor(plan_info.ToVector());
}

void BatchPrefillWithRaggedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor plan_info_vec, at::Tensor q, at::Tensor k, at::Tensor v,
    at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor o, std::optional<at::Tensor> maybe_lse,
    int64_t mask_mode_code, int64_t layout, int64_t window_left ADDITIONAL_FUNC_PARAMS,
    int64_t hip_stream) {
  PrefillPlanInfo plan_info;
  plan_info.FromVector(tensor_to_vec(plan_info_vec));
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);

  int64_t num_qo_heads = q.size(1);
  int64_t head_dim_qk = q.size(2);
  int64_t num_kv_heads = (kv_layout == QKVLayout::kNHD) ? k.size(1) : k.size(0);
  uint32_t q_stride_n = q.stride(0), q_stride_h = q.stride(1), k_stride_n, k_stride_h, v_stride_n,
           v_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    k_stride_n = k.stride(0);
    k_stride_h = k.stride(1);
    v_stride_n = v.stride(0);
    v_stride_h = v.stride(1);
  } else {
    k_stride_h = k.stride(0);
    k_stride_n = k.stride(1);
    v_stride_h = v.stride(0);
    v_stride_n = v.stride(1);
  }

  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == q.size(0), lse.size(0), q.size(0));
    TORCH_CHECK(lse.size(1) == q.size(1), lse.size(1), q.size(1));
  }

  void* float_buffer_ptr = float_workspace_buffer.data_ptr();
  void* int_buffer_ptr = int_workspace_buffer.data_ptr();

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
      USE_SLIDING_WINDOW, USE_LOGITS_SOFT_CAP, USE_FP16_QK_REDUCTION, AttentionVariant,
      RaggedParams, PagedParams, [&] {
        RaggedParams params;

        params.q = static_cast<DTypeQ*>(q.data_ptr());
        params.k = static_cast<DTypeKV*>(k.data_ptr());
        params.v = static_cast<DTypeKV*>(v.data_ptr());
        params.o = static_cast<DTypeO*>(o.data_ptr());
        params.lse = maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
        params.q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());
        params.kv_indptr = static_cast<IdType*>(kv_indptr.data_ptr());
        params.num_qo_heads = num_qo_heads;
        params.num_kv_heads = num_kv_heads;
        params.group_size = uint_fastdiv(num_qo_heads / num_kv_heads);
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.k_stride_n = k_stride_n;
        params.k_stride_h = k_stride_h;
        params.v_stride_n = v_stride_n;
        params.v_stride_h = v_stride_h;
        params.window_left = window_left;

        params.request_indices = nullptr;
        params.qo_tile_indices = nullptr;
        params.kv_tile_indices = nullptr;
        params.merge_indptr = nullptr;
        params.o_indptr = nullptr;
        params.kv_chunk_size_ptr = nullptr;
        params.block_valid_mask = nullptr;
        params.total_num_rows = nullptr;
        params.max_total_num_rows = 0;
        params.padded_batch_size = 0;
        params.partition_kv = false;

        ADDITIONAL_PARAMS_SETTER

        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;

        params.request_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.kv_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
        params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
        params.kv_chunk_size_ptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset);
        if (plan_info.split_kv) {
          params.merge_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
          tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
          tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
          if (plan_info.enable_cuda_graph) {
            params.block_valid_mask =
                GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
          }
        }
        params.padded_batch_size = plan_info.padded_batch_size;
        params.max_total_num_rows = plan_info.total_num_rows;
        if (plan_info.enable_cuda_graph) {
          params.total_num_rows =
              GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
        }

        hipError_t status = hipSuccess;

        DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q, CTA_TILE_Q, {
          status = flashinfer::BatchPrefillWithRaggedKVCacheDispatched<
              CTA_TILE_Q, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
              /*use_fp16_qk_reduction=*/USE_FP16_QK_REDUCTION, MASK_MODE, AttentionVariant,
              RaggedParams>(params, tmp_v, tmp_s, stream);
        });

        TORCH_CHECK(status == hipSuccess, "BatchPrefillWithRaggedKVCache failed with error ",
                    hipGetErrorString(status));
        return true;
      });
}

void BatchPrefillWithPagedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, at::Tensor qo_indptr, at::Tensor paged_kv_indptr,
    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len, at::Tensor o,
    std::optional<at::Tensor> maybe_lse, int64_t mask_mode_code, int64_t layout,
    int64_t window_left ADDITIONAL_FUNC_PARAMS, int64_t hip_stream) {
  PrefillPlanInfo plan_info;
  plan_info.FromVector(tensor_to_vec(plan_info_vec));
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  auto device = q.device();
  int64_t batch_size = paged_kv_indptr.size(0) - 1;
  int64_t num_qo_heads = q.size(1);
  int64_t num_kv_heads, page_size;
  uint32_t head_dim_qk = q.size(2);
  if (kv_layout == QKVLayout::kHND) {
    num_kv_heads = paged_k_cache.size(1);
    page_size = paged_k_cache.size(2);
  } else {
    page_size = paged_k_cache.size(1);
    num_kv_heads = paged_k_cache.size(2);
  }

  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == q.size(0), lse.size(0), q.size(0));
    TORCH_CHECK(lse.size(1) == q.size(1), lse.size(1), q.size(1));
  }

  void* float_buffer_ptr = static_cast<void*>(float_workspace_buffer.data_ptr());
  void* int_buffer_ptr = static_cast<void*>(int_workspace_buffer.data_ptr());

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = paged_k_cache.scalar_type();

  // get q_stride_n and q_stride_h
  const auto q_stride_n = q.stride(0);
  const auto q_stride_h = q.stride(1);

  // get kv_cache_strides
  const int64_t* kv_cache_strides = nullptr;
  auto k_strides = paged_k_cache.strides();
  auto v_strides = paged_v_cache.strides();
  TORCH_CHECK(k_strides == v_strides, "k/v strides must be identical");
  kv_cache_strides = k_strides.data();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
      USE_SLIDING_WINDOW, USE_LOGITS_SOFT_CAP, USE_FP16_QK_REDUCTION, AttentionVariant,
      RaggedParams, PagedParams, [&] {
        PagedParams params;

        params.q = static_cast<DTypeQ*>(q.data_ptr());
        paged_kv_t<DTypeKV, IdType> paged_kv(
            num_kv_heads, page_size, HEAD_DIM_VO, batch_size, kv_layout,
            static_cast<DTypeKV*>(paged_k_cache.data_ptr()),
            static_cast<DTypeKV*>(paged_v_cache.data_ptr()), kv_cache_strides,
            static_cast<IdType*>(paged_kv_indices.data_ptr()),
            static_cast<IdType*>(paged_kv_indptr.data_ptr()),
            static_cast<IdType*>(paged_kv_last_page_len.data_ptr()));
        params.paged_kv = paged_kv;
        params.q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());
        params.o = static_cast<DTypeO*>(o.data_ptr());

        params.lse = maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
        params.num_qo_heads = num_qo_heads;
        params.group_size = uint_fastdiv(num_qo_heads / paged_kv.num_heads);
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.window_left = window_left;

        params.request_indices = nullptr;
        params.qo_tile_indices = nullptr;
        params.kv_tile_indices = nullptr;
        params.merge_indptr = nullptr;
        params.o_indptr = nullptr;
        params.kv_chunk_size_ptr = nullptr;
        params.block_valid_mask = nullptr;
        params.total_num_rows = nullptr;
        params.max_total_num_rows = 0;
        params.padded_batch_size = 0;
        params.partition_kv = false;

        ADDITIONAL_PARAMS_SETTER

        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;

        params.request_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.kv_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
        params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
        params.kv_chunk_size_ptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset);
        if (plan_info.split_kv) {
          params.merge_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
          tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
          tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
          if (plan_info.enable_cuda_graph) {
            params.block_valid_mask =
                GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
          }
        }
        params.padded_batch_size = plan_info.padded_batch_size;
        params.max_total_num_rows = plan_info.total_num_rows;
        if (plan_info.enable_cuda_graph) {
          params.total_num_rows =
              GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
        }

        hipError_t status = hipSuccess;

        DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q, CTA_TILE_Q, {
          status = flashinfer::BatchPrefillWithPagedKVCacheDispatched<
              CTA_TILE_Q, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
              /*use_fp16_qk_reduction=*/USE_FP16_QK_REDUCTION, MASK_MODE, AttentionVariant,
              PagedParams>(params, tmp_v, tmp_s, stream);
        });

        TORCH_CHECK(status == hipSuccess, "BatchPrefillWithPagedKVCache failed with error ",
                    hipGetErrorString(status));
        return true;
      });
}
