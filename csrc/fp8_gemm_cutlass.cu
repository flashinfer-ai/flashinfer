#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hip/hip_fp16.h>

#include <cstddef>
#include <cstdint>
#include <functional>
#include <type_traits>
#include <vector>

#include "flashinfer/gemm/cutlass_gemm_configs.h"
#include "flashinfer/gemm/fp8_gemm_cutlass.h"
#include "flashinfer/gemm/fp8_gemm_cutlass_template.h"
#include "tvm_ffi_utils.h"

using flashinfer::gemm::ClusterShape;
using flashinfer::gemm::CutlassFp8GemmRunner;
using flashinfer::gemm::CutlassFp8GemmRunnerInterface;
using flashinfer::gemm::CutlassGemmConfig;
using flashinfer::gemm::CutlassTileConfigSM100;
using flashinfer::gemm::EpilogueScheduleType;
using flashinfer::gemm::MainloopScheduleType;

namespace flashinfer {
namespace gemm {
template class CutlassFp8GemmRunner<__hip_bfloat16>;
template class CutlassFp8GemmRunner<half>;
}  // namespace gemm
}  // namespace flashinfer

namespace torch_ext {

namespace {

CutlassGemmConfig getFp8GemmConfig(int64_t m, int64_t n, int64_t k, int64_t tactic) {
  auto getCutlassFp8GemmConfigs = []() {
    CutlassFp8GemmRunner<__hip_bfloat16> gemmRunner;
    return gemmRunner.getConfigs();
  };
  static std::vector<CutlassGemmConfig> globalConfigs = getCutlassFp8GemmConfigs();
  TVM_FFI_ICHECK(tactic >= 0 && tactic < globalConfigs.size())
      << "tactic must be between 0 and " << globalConfigs.size();
  return globalConfigs[tactic];
}

template <typename T>
void runGemm(TensorView out, TensorView mat1, TensorView mat2, TensorView scale_a,
             TensorView scale_b, int64_t m, int64_t n, int64_t k, int64_t b,
             CutlassGemmConfig const& gemmConfig, TensorView workspace_buffer) {
  CutlassFp8GemmRunner<T> gemmRunner;

  int64_t const required_workspace_size = gemmRunner.getWorkspaceSize(m, n, k);
  int64_t const provided_workspace_size =
      workspace_buffer.numel() * get_element_size(workspace_buffer);

  auto runKernel = [&](void* workspace) {
    gemmRunner.gemm(static_cast<__hip_fp8_e4m3_fnuz*>(mat1->data),
                    static_cast<__hip_fp8_e4m3_fnuz*>(mat2->data), static_cast<float*>(scale_a->data),
                    static_cast<float*>(scale_b->data), out->data, m, n, k, b, gemmConfig,
                    static_cast<char*>(workspace), required_workspace_size,
                    get_stream(mat1->device));
  };

  if (provided_workspace_size < required_workspace_size) {
    Tensor new_workspace =
        alloc_tensor({required_workspace_size}, DLDataType{kDLInt, 8, 1}, mat1->device);

    runKernel(new_workspace->data);
  } else {
    runKernel(workspace_buffer->data);
  }
}

void fp8_bmm_impl(TensorView mat1, TensorView mat2, TensorView scale_a, TensorView scale_b,
                  TensorView out, TensorView workspace_buffer, int64_t tactic) {
  CHECK_INPUT(mat1);
  CHECK_INPUT(mat2);
  CHECK_INPUT(scale_a);
  CHECK_INPUT(scale_b);

  int mat2_k_scale = 1;

  int64_t m, n, k, b;
  if (mat1->ndim == 2) {
    TVM_FFI_ICHECK_EQ(mat2->ndim, 2) << "mat2 must be a matrix";
    TVM_FFI_ICHECK_EQ(mat1->shape[1], mat2->shape[1] * mat2_k_scale)
        << "mat1 and mat2 shapes cannot be multiplied (" << mat1->shape[0] << "x" << mat1->shape[1]
        << " and " << mat2->shape[0] << "x" << mat2->shape[1] << ")";
    m = mat1->shape[0];
    n = mat2->shape[0];
    k = mat2->shape[1];
    b = 1;
  } else if (mat1->ndim == 3) {
    TVM_FFI_ICHECK_EQ(mat2->ndim, 3) << "mat2 must be a batch of matrices";
    TVM_FFI_ICHECK_EQ(mat1->shape[0], mat2->shape[0])
        << "mat1 and mat2 must have the same batch size (" << mat1->shape[0] << " and "
        << mat2->shape[0] << ")";
    TVM_FFI_ICHECK_EQ(mat1->shape[2], mat2->shape[2] * mat2_k_scale)
        << "mat1 and mat2 shapes cannot be multiplied (" << mat1->shape[1] << "x" << mat1->shape[2]
        << " and " << mat2->shape[1] << "x" << mat2->shape[2] << ")";
    m = mat1->shape[1];
    n = mat2->shape[1];
    k = mat2->shape[2];
    b = mat1->shape[0];
  } else {
    TVM_FFI_LOG_AND_THROW(NotImplementedError) << "mat1 must be a matrix or a batch of matrices";
  }

  // No heuristic for now, we rely on the autotuner to select the best tactic.
  if (tactic == -1) {
    tactic = 0;
  }
  auto config = getFp8GemmConfig(m, n, k, tactic);

  // Validate out dimensions
  std::vector<int64_t> out_shape =
      mat1->ndim == 2 ? std::vector<int64_t>{m, n} : std::vector<int64_t>{b, m, n};
  TVM_FFI_ICHECK_EQ(out->ndim, out_shape.size())
      << "out must have " << out_shape.size() << " dimensions, but got " << out->ndim;
  for (int i = 0; i < out_shape.size(); ++i) {
    TVM_FFI_ICHECK_EQ(out->shape[i], out_shape[i])
        << "out shape mismatch at dimension " << i << ": expected " << out_shape[i] << ", got "
        << out->shape[i];
  }

  switch (encode_dlpack_dtype(out->dtype)) {
    case float16_code:
      runGemm<half>(out, mat1, mat2, scale_a, scale_b, m, n, k, b, config, workspace_buffer);
      break;
    case bfloat16_code:
      runGemm<__hip_bfloat16>(out, mat1, mat2, scale_a, scale_b, m, n, k, b, config,
                             workspace_buffer);
      break;
    default:
      TVM_FFI_LOG_AND_THROW(NotImplementedError) << "out_dtype must be one of fp16/bf16.";
  }
}

}  // namespace

void fp8_gemm(TensorView mat1, TensorView mat2, TensorView scale_a, TensorView scale_b,
              TensorView out, TensorView workspace_buffer, int64_t tactic) {
  fp8_bmm_impl(mat1, mat2, scale_a, scale_b, out, workspace_buffer, tactic);
}

int64_t fp8_gemm_tactic_num() {
  auto getCutlassConfigs = []() {
    CutlassFp8GemmRunner<__hip_bfloat16> gemmRunner;
    return gemmRunner.getConfigs();
  };
  static int64_t totalTactics = getCutlassConfigs().size();
  return totalTactics;
}

}  // namespace torch_ext

TVM_FFI_DLL_EXPORT_TYPED_FUNC(fp8_gemm, torch_ext::fp8_gemm);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(fp8_gemm_tactic_num, torch_ext::fp8_gemm_tactic_num);
