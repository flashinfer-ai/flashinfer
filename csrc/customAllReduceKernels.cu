#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/customAllReduceKernels.h>
#include <hip/hip_cooperative_groups.h>
#include <tuple>
#include <type_traits>


bool pdlEnabled() {
    static int sm_version = []() -> int {
        int device = 0;
        hipError_t err = hipGetDevice(&device);
        TORCH_CHECK(err == hipSuccess, "hipGetDevice failed: ", hipGetErrorString(err));

        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, device);
        TORCH_CHECK(err == hipSuccess, "hipGetDeviceProperties failed: ", hipGetErrorString(err));
        return prop.major * 10 + prop.minor;
    }();
    return sm_version >= 90;
}

template <typename T1, typename T2>
inline size_t divUp(T1 const& a, T2 const& b)
{
    auto const tmp_a = static_cast<size_t>(a);
    auto const tmp_b = static_cast<size_t>(b);
    return (tmp_a + tmp_b - 1) / tmp_b;
}

inline int roundUp(int a, int b)
{
    return divUp(a, b) * b;
}

inline std::optional<bool> isCudaLaunchBlocking()
{
    thread_local bool firstCall = true;
    thread_local std::optional<bool> result = std::nullopt;
    if (!firstCall)
    {
        char const* env = std::getenv("CUDA_LAUNCH_BLOCKING");
        if (env != nullptr && std::string(env) == "1")
        {
            result = true;
        }
        else
        {
            result = false;
        }
        firstCall = false;
    }
    return result;
}

inline bool isCapturing(hipStream_t stream)
{
    hipStreamCaptureStatus status;
    hipError_t err = hipStreamIsCapturing(stream, &status);
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));
    return status == hipStreamCaptureStatus::hipStreamCaptureStatusActive;
}

inline bool doCheckError(hipStream_t stream)
{
    auto const cudaLaunchBlocking = isCudaLaunchBlocking();
    if (cudaLaunchBlocking.has_value() && cudaLaunchBlocking.value())
    {
        return !isCapturing(stream);
    }

#ifndef NDEBUG
    // Debug builds will sync when we're not capturing unless explicitly
    // disabled.
    bool const checkError = cudaLaunchBlocking.value_or(!isCapturing(stream));
#else
    bool const checkError = cudaLaunchBlocking.value_or(false);
#endif

    return checkError;
}

inline void syncAndCheck(hipStream_t stream, char const* const file, int const line)
{
    if (doCheckError(stream))
    {
        hipStreamSynchronize(stream);
        hipError_t err = hipGetLastError();
        TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err), file, line);
    }
}

#define sync_check_cuda_error(stream) syncAndCheck(stream, __FILE__, __LINE__)

namespace tensorrt_llm::kernels {

constexpr size_t NUM_POINTERS_PER_RANK = 7;

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void st_flag_release(uint32_t const& flag, uint32_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#else
    __threadfence_system();
    asm volatile("st.global.volatile.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t ld_flag_acquire(uint32_t* flag_addr)
{
    uint32_t flag;
#if __CUDA_ARCH__ >= 700
    asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
    asm volatile("ld.global.volatile.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#endif
    return flag;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Type Converter that packs data format to 128 bits data type
//
using PackedFloat = union
{
    int4 packed;
    float unpacked[4];
};

using PackedHalf = union
{
    int4 packed;
    half2 unpacked[4];
};

template <typename T>
struct PackedOn16Bytes
{
};

template <>
struct PackedOn16Bytes<float>
{
    using Type = PackedFloat;
};

template <>
struct PackedOn16Bytes<half>
{
    using Type = PackedHalf;
};

#ifdef ENABLE_BF16
using PackedBFloat16 = union
{
    int4 packed;
    __hip_bfloat162 unpacked[4];
};

template <>
struct PackedOn16Bytes<__hip_bfloat16>
{
    using Type = PackedBFloat16;
};

#endif

// add two 128b data
template <typename T>
inline __device__ int4 add128b(T& a, T& b)
{
    T c;
    c.unpacked[0] = a.unpacked[0] + b.unpacked[0];
    c.unpacked[1] = a.unpacked[1] + b.unpacked[1];
    c.unpacked[2] = a.unpacked[2] + b.unpacked[2];
    c.unpacked[3] = a.unpacked[3] + b.unpacked[3];
    return c.packed;
}

__inline__ __device__ void multi_gpu_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
    size_t const world_size, int const tidx, int const bidx)
{
    // After this function, at least one block in each GPU has reached the barrier
    if (tidx < world_size)
    {
        // we can think of signals having the shape [world_size, world_size]
        // Dimension 0 is the "listening" dimension, dimension 1 is "emitting" dimension

        // Block 0 broadcasts its flag (local_rank on emitting dimension) to all receivers
        size_t offset = (flag % 2) ? world_size : 0;

        if (bidx == 0)
        {
            st_flag_release(flag, signals[tidx] + offset + local_rank);
        }

        // All blocks check that corresponding block 0 on other GPUs have set the flag
        // No deadlock because block #0 is always the first block started
        uint32_t* peer_barrier_d = signals[local_rank] + offset + tidx;
        while (ld_flag_acquire(peer_barrier_d) != flag)
        {
        }
    }

    __syncthreads();
}

__inline__ __device__ void block_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
    size_t const world_size, int const tidx, int const bidx, int const grid_size)
{
    // After this function, the block of id == bidx of each GPU has reached the barrier
    if (tidx < world_size)
    {
        // we can think of signals having the shape [world_size, 2, num_blocks, world_size]
        // (+ an offset on dim 2 to account for flags used in multi_gpu_barrier)
        // Dimension 0 is the "listening" dimension, dimension 3 is "emitting" dimension

        // Block broadcast its flag (local_rank on emitting dimension) to all receivers
        uint32_t flag_block_offset = world_size + bidx * world_size;

        if (flag % 2 == 1)
        {
            flag_block_offset += (grid_size + 1) * world_size;
        }

        st_flag_release(flag, signals[tidx] + flag_block_offset + local_rank);

        // Blocks check that corresponding blocks on other GPUs have also set the flag
        uint32_t* peer_barrier_d = signals[local_rank] + flag_block_offset + tidx;

        while (ld_flag_acquire(peer_barrier_d) != flag)
        {
        }
    }

    __syncthreads();
}

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true, bool PUSH_MODE = false>
static __global__ void oneShotAllReduceKernel(AllReduceParams params)
{
    // Suppose that two GPUs participate in the AR exchange, and we start four blocks.
    // The message is partitioned into chunks as detailed below:
    //               message
    //       |-------------------|
    // GPU 0 | B0 | B1 | B2 | B3 |
    // GPU 1 | B0 | B1 | B2 | B3 |
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 copies the chunk it  is responsible for, from local_input to shareable buffer
    // 2. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier)
    // 3. B0 on GPU 0 pull and sum the chunk from GPU 1, writes the result to local_output
    //
    // With COPY_INPUT == false, skip step 1. and use gpu_barrier instead of block barrier during step 2.
    // We only to know if the other GPU as arrived at the AR kernel, that would mean that data is ready
    //
    // With PUSH_MODE, we consider that the shared buffer is of size:
    // params.peer_comm_buffer_ptrs: [world_size, world_size, message_size]
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 push the chunk is it responsible for into all other GPUs:
    //    params.peer_comm_buffer_ptrs[:, local_gpu, B0 slice]
    // 2. block sync so the block is shared by other GPUs
    // 3. Reduce along second dimension params.peer_comm_buffer_ptrs[local_gpu, :, B0 slice]

    int const bidx = blockIdx.x;
    int const tidx = threadIdx.x;
    int const grid_size = gridDim.x;

    // The number of elements packed into one for comms
    static constexpr int PACKED_ELTS = 16 / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);

    // Start and end offsets of the thread
    size_t const chunk_start = bidx * params.elts_per_block + tidx * PACKED_ELTS;
    size_t const chunk_end = std::min((bidx + 1) * params.elts_per_block, params.elts_total);

    T* buffers[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        // buffers[0] is always the local buffers. Helps load balancing reads.
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    if constexpr (PUSH_MODE || COPY_INPUT)
    {
        // Copy from local buffer to shareable buffer
        for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * PACKED_ELTS)
        {
            if constexpr (PUSH_MODE)
            {
#pragma unroll
                for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
                {
                    *reinterpret_cast<int4*>(&buffers[ii][params.local_rank * params.elts_total + iter_offset])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[iter_offset]);
                }
            }
            else
            {
                *reinterpret_cast<int4*>(&local_shared_buffer[iter_offset])
                    = *reinterpret_cast<int4 const*>(&local_input_buffer[iter_offset]);
            }
        }

        // wait for equivalent blocks of other GPUs to have copied data to their shareable buffer
        block_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);
    }
    else
    {
        // In the non-copy case, we assume that once the kernel has been started, data is ready to be consumed
        multi_gpu_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * PACKED_ELTS)
    {
        // Iterate over the different ranks/devices on the node to load the values.
        PackedStruct vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            if constexpr (PUSH_MODE)
            {
                vals[ii].packed
                    = *reinterpret_cast<int4 const*>(&buffers[params.local_rank][ii * params.elts_total + iter_offset]);
            }
            else
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][iter_offset]);
            }
        }

        // Sum the values from the different ranks.
        PackedStruct sums;
        sums.packed = {0, 0, 0, 0};
#pragma unroll
        for (int rank = 0; rank < RANKS_PER_NODE; ++rank)
        {
            // Always reduce from rank 0 to ensure stable reduce order.
            int ii = (rank + RANKS_PER_NODE - params.local_rank) % RANKS_PER_NODE;
            sums.packed = add128b(sums, vals[ii]);
        }
        // Store to the destination buffer.
        *reinterpret_cast<int4*>(&local_output_buffer[iter_offset]) = sums.packed;
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true, bool PUSH_MODE = false, bool Bias = false,
    bool Residual = false>
static __global__ void __launch_bounds__(512, 1) twoShotAllReduceKernel(AllReduceParams params)
{
    // Suppose that two GPUs participate in the AR exchange, and we start two blocks.
    // The message is partitioned into chunks as detailed below:
    //               message
    //       |-------------------|
    //       |--GPU 0--|--GPU 1--| (GPU responsibility parts)
    // GPU 0 | B0 | B1 | B0 | B1 |
    // GPU 1 | B0 | B1 | B0 | B1 |
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 copies all chunks is it responsible for, from local_input to shareable buffer
    // 2. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier #0)
    // 3. B0 on GPU 0 gather and sum the B0 chunks from GPU 1, that are in the GPU 0 responsibility
    //    part (the first half of the message, see GPU responsibility row above)
    // 3bis. Likewise, B0 on GPU 1 copies and sum the chunks for GPU 0,
    //       where GPU 1 is responsible: the second half of the message.
    // 4. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier #1)
    // 5. B0 writes result to local_output. It gathers each chunk from its responsible GPU.
    //    For example, here it reads the first chunk from GPU 0 and second chunk from GPU 1.
    //
    // With COPY_INPUT == false, skip step 1. and use gpu_barrier instead of block barrier during step 2.
    // We only to know if the other GPU as arrived at the AR kernel, that would mean that data is ready
    // to be read.
    //
    // Note that compared to one-shot, one block (CTA) writes multiple input chunks and write multiple output chunks.
    // However, it's only responsible for the summation of a single chunk.
    //
    // With PUSH_MODE, we consider that the shared buffer is of size:
    // params.peer_comm_buffer_ptrs: [world_size, world_size, message_size / world_size]
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 push the chunks is it responsible for into the corresponding GPUs:
    //    params.peer_comm_buffer_ptrs[target_gpu, local_gpu, current B0 slice]
    // 2. block sync so the blocks have been shared by other GPUs
    // 3. Reduce along second dimension params.peer_comm_buffer_ptrs[local_gpu, :, B0 slice]
    // 4. block barrier (corresponding blocks have finished reduction)
    // 5. pull and write on local buffer, by reading params.peer_comm_buffer_ptrs[:, 0, B0 slice] (reduction result is
    //    written at index 0 of 2nd dim)

    int const bidx = blockIdx.x;
    int const tidx = threadIdx.x;
    int const grid_size = gridDim.x;

    // The number of elements packed into one for comms
    static constexpr int PACKED_ELTS = 16 / sizeof(T);
    using PackedType = typename PackedOn16Bytes<T>::Type;

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);

    size_t const chunk_start = bidx * params.elts_per_block + tidx * PACKED_ELTS;
    size_t const chunk_end = min(chunk_start + params.elts_per_block, params.elts_per_rank);

    T* buffers[RANKS_PER_NODE];
    int ranks[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        // A mapping of the ranks to scatter reads as much as possible
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        ranks[ii] = rank;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    if constexpr (PUSH_MODE || COPY_INPUT)
    {
        // Copy all blocks from local buffer to shareable buffer
        for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
        {
#pragma unroll
            for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
            {
                size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
                if (offset_rank >= params.elts_total)
                {
                    continue;
                }

                if constexpr (PUSH_MODE)
                {
                    *reinterpret_cast<int4*>(&buffers[ii][params.local_rank * params.elts_per_rank + local_offset])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[offset_rank]);
                }
                else
                {
                    *reinterpret_cast<int4*>(&local_shared_buffer[offset_rank])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[offset_rank]);
                }
            }
        }
        block_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);
    }
    else
    {
        // In the non-copy case, we assume that once the kernel has been started, data is ready to be consumed
        multi_gpu_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
    {
        size_t const responsible_block_offset = local_offset + params.rank_offset;

        // Iterate over the different ranks/devices on the node to load the values.
        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            if constexpr (PUSH_MODE)
            {
                vals[ii].packed
                    = *reinterpret_cast<int4 const*>(&local_shared_buffer[ii * params.elts_per_rank + local_offset]);
            }
            else
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][responsible_block_offset]);
            }
        }

        // Sum the values from the different ranks.
        PackedType sums;
        sums.packed = {0, 0, 0, 0};
#pragma unroll
        for (int rank = 0; rank < RANKS_PER_NODE; ++rank)
        {
            // Always reduce from rank 0 to ensure stable reduce order.
            int ii = (rank + RANKS_PER_NODE - params.local_rank) % RANKS_PER_NODE;
            sums.packed = add128b(sums, vals[ii]);
        }

        // Store to the local buffer.
        if constexpr (PUSH_MODE)
        {
            *reinterpret_cast<int4*>(&local_shared_buffer[local_offset]) = sums.packed;
        }
        else
        {
            *reinterpret_cast<int4*>(&local_shared_buffer[responsible_block_offset]) = sums.packed;
        }
    }

    block_barrier(
        params.peer_barrier_ptrs_out, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);

    // Gather all needed elts from other intra-node ranks
    for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
    {
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            // use round-robin gathering from other ranks
            size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
            if (offset_rank >= params.elts_total)
            {
                continue;
            }
            PackedType sums, residual_vec, bias_vec;
            if constexpr (Bias)
            {
                bias_vec.packed
                    = *reinterpret_cast<int4 const*>(reinterpret_cast<T const*>(params.fusion_params.bias_buffer)
                        + offset_rank % params.fusion_params.hidden_size);
            }
            if constexpr (Residual)
            {
                residual_vec.packed = *reinterpret_cast<int4 const*>(
                    reinterpret_cast<T const*>(params.fusion_params.residual_buffer) + offset_rank);
            }
            if constexpr (PUSH_MODE)
            {
                *reinterpret_cast<int4*>(&local_output_buffer[offset_rank])
                    = *reinterpret_cast<int4*>(&buffers[ii][local_offset]);
                sums.packed = *reinterpret_cast<int4*>(&buffers[ii][local_offset]);
            }
            else
            {
                *reinterpret_cast<int4*>(&local_output_buffer[offset_rank])
                    = *reinterpret_cast<int4*>(&buffers[ii][offset_rank]);
                sums.packed = *reinterpret_cast<int4*>(&buffers[ii][offset_rank]);
            }
            if constexpr (Bias)
            {
                sums.packed = add128b(sums, bias_vec);
            }
            if constexpr (Residual)
            {
                sums.packed = add128b(sums, residual_vec);
            }
            *reinterpret_cast<int4*>(&local_output_buffer[offset_rank]) = sums.packed;
        }
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

bool configurationSupported(AllReduceStrategyType algo, size_t msg_size, size_t n_ranks, at::ScalarType type)
{
    size_t elts_per_thread = 16 / at::elementSize(type);
    int const msg_align = (algo == AllReduceStrategyType::TWOSHOT) ? n_ranks * elts_per_thread : elts_per_thread;
    bool supported_algo = (algo == AllReduceStrategyType::ONESHOT || algo == AllReduceStrategyType::TWOSHOT);
    return supported_algo && (msg_size % msg_align == 0);
}

std::tuple<int, int> kernelLaunchConfig(AllReduceStrategyType algo, AllReduceParams& params, size_t elts_per_thread, int num_ctas)
{
    int blocks_per_grid = 1, threads_per_block = DEFAULT_BLOCK_SIZE;

    switch (algo)
    {
    case AllReduceStrategyType::ONESHOT:
    {
        TORCH_CHECK(params.elts_total % elts_per_thread == 0);
        size_t const total_threads = roundUp(params.elts_total / elts_per_thread, WARP_SIZE);
        threads_per_block = std::min(DEFAULT_BLOCK_SIZE, total_threads);
        blocks_per_grid = std::min(static_cast<size_t>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
        params.elts_per_block = roundUp(divUp(params.elts_total, blocks_per_grid), elts_per_thread);
        break;
    }
    case AllReduceStrategyType::TWOSHOT:
    {
        TORCH_CHECK(params.elts_total % (elts_per_thread * params.ranks_per_node) == 0);
        size_t const total_threads = roundUp(params.elts_total / (elts_per_thread * params.ranks_per_node), WARP_SIZE);

        /*
        threads_per_block = std::min(DEFAULT_BLOCK_SIZE, total_threads);
        blocks_per_grid = std::min(static_cast<size_t>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
        */
        while (total_threads % blocks_per_grid != 0 || total_threads / blocks_per_grid > DEFAULT_BLOCK_SIZE)
        {
            blocks_per_grid += 1;
        }

        threads_per_block = total_threads / blocks_per_grid;

        // NOTE: need to adjust here
        if (blocks_per_grid > MAX_ALL_REDUCE_BLOCKS)
        {
            size_t iter_factor = 1;
            while (blocks_per_grid / iter_factor > MAX_ALL_REDUCE_BLOCKS || blocks_per_grid % iter_factor)
            {
                iter_factor += 1;
            }
            blocks_per_grid /= iter_factor;
        }
        params.elts_per_rank = params.elts_total / params.ranks_per_node;
        params.rank_offset = params.local_rank * params.elts_per_rank;
        params.elts_per_block = roundUp(divUp(params.elts_per_rank, blocks_per_grid), elts_per_thread);
        break;
    }
    default: throw std::runtime_error("Algorithm not supported here.");
    }

    blocks_per_grid = std::min(blocks_per_grid, num_ctas);
    return std::make_tuple(blocks_per_grid, threads_per_block);
}

// template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false, bool Bias = false,
//     bool Affine = false>
// void AllReduceNormKernelLaunch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
//     AllReduceParams& params, hipStream_t stream)
// {
//     TORCH_CHECK(
//         (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM || fusionOp == AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM),
//         "Unsupported AllReduceFusionOp: ", static_cast<int>(fusionOp));
//     if (algo == AllReduceStrategyType::ONESHOT)
//     {
//         reduce_fusion::one_shot_all_reduce_norm_kernel_launcher<T, RANKS_PER_NODE, Bias, Affine>(
//             params, stream, fusionOp);
//     }
//     else
//     {
//         TORCH_CHECK(!(USE_MEMCPY && PUSH_MODE), "Memcpy cannot be used with PUSH_MODE.");
//         size_t elts_per_thread = 16 / sizeof(T);
//         auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(algo, params, elts_per_thread);
//         if (USE_MEMCPY)
//         {
//             hipMemcpyAsync(params.peer_comm_buffer_ptrs[params.local_rank], params.local_input_buffer_ptr,
//                 params.elts_total * sizeof(T), hipMemcpyDeviceToDevice, stream);
//         }
//         auto output_ptr = params.local_output_buffer_ptr;
//         params.local_output_buffer_ptr = params.fusion_params.intermediate_buffer;

//         twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE, Bias, true>
//             <<<blocks_per_grid, threads_per_block, 0, stream>>>(params);

//         params.local_output_buffer_ptr = output_ptr;
//         reduce_fusion::rms_norm_kernel_launcher<T, false, false, Affine>(params, stream, fusionOp);
//     }
// }

// template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
// void AllReduceNormDispatch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
//     AllReduceParams& params, hipStream_t stream)
// {
//     if (params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
//     {
//         AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, true, true>(
//             algo, config, fusionOp, params, stream);
//     }
//     else if (params.fusion_params.bias_buffer && !params.fusion_params.weight_buffer)
//     {
//         AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, true, false>(
//             algo, config, fusionOp, params, stream);
//     }
//     else if (!params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
//     {
//         AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, false, true>(
//             algo, config, fusionOp, params, stream);
//     }
//     else
//     {
//         AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, false, false>(
//             algo, config, fusionOp, params, stream);
//     }
// }

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceDispatch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream, int num_ctas)
{
    TORCH_CHECK(fusionOp == AllReduceFusionOp::NONE);
    TORCH_CHECK(!(USE_MEMCPY && PUSH_MODE), "Memcpy cannot be used with PUSH_MODE.");
    size_t elts_per_thread = 16 / sizeof(T);
    auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(algo, params, elts_per_thread, num_ctas);
    if (USE_MEMCPY)
    {
        hipMemcpyAsync(params.peer_comm_buffer_ptrs[params.local_rank], params.local_input_buffer_ptr,
            params.elts_total * sizeof(T), hipMemcpyDeviceToDevice, stream);
    }
    if (algo == AllReduceStrategyType::ONESHOT)
    {
        auto* kernel_instance = &oneShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE>;
        hipLaunchConfig_t config;
        config.gridDim = blocks_per_grid;
        config.blockDim = threads_per_block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attribute[1];
        attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attribute[0].val.programmaticStreamSerializationAllowed = pdlEnabled();
        config.attrs = attribute;
        config.numAttrs = 1;
        cudaLaunchKernelEx(&config, kernel_instance, params);
    }
    else
    {
        auto* kernel_instance = &twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE>;
        hipLaunchConfig_t config;
        config.gridDim = blocks_per_grid;
        config.blockDim = threads_per_block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attribute[1];
        attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attribute[0].val.programmaticStreamSerializationAllowed = pdlEnabled();
        config.attrs = attribute;
        config.numAttrs = 1;
        cudaLaunchKernelEx(&config, kernel_instance, params);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceDispatchMemcpy(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream, int num_ctas)
{
    if (fusionOp == AllReduceFusionOp::NONE)
    {
        // TLLM_LOG_DEBUG("AllReduceDispatch enabled");
        AllReduceDispatch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY>(algo, config, fusionOp, params, stream, num_ctas);
    }
    else
    {
        // TLLM_LOG_DEBUG("AllReduceNormDispatch enabled");
        throw std::runtime_error("AllReduce with fusion op currently not supported.");
        // AllReduceNormDispatch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY>(algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false>
void AllReduceDispatchPushMode(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream, int num_ctas)
{
    if (static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(config)
        & static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(AllReduceStrategyConfig::USE_MEMCPY))
    {
        AllReduceDispatchMemcpy<T, RANKS_PER_NODE, PUSH_MODE, true>(algo, config, fusionOp, params, stream, num_ctas);
    }
    else
    {
        AllReduceDispatchMemcpy<T, RANKS_PER_NODE, PUSH_MODE, false>(algo, config, fusionOp, params, stream, num_ctas);
    }
}

template <typename T, int RANKS_PER_NODE> //, bool USE_MEMCPY = false, bool PUSH_MODE = false>
void AllReduceDispatchRanksPerNode(AllReduceStrategyType algo, AllReduceStrategyConfig config,
    AllReduceFusionOp fusionOp, AllReduceParams& params, hipStream_t stream, int num_ctas)
{
    if (static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(config)
        & static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(AllReduceStrategyConfig::PUSH_MODE))
    {
        AllReduceDispatchPushMode<T, RANKS_PER_NODE, true>(algo, config, fusionOp, params, stream, num_ctas);
    }
    else
    {
        AllReduceDispatchPushMode<T, RANKS_PER_NODE, false>(algo, config, fusionOp, params, stream, num_ctas);
    }
}

template <typename T>
void AllReduceDispatchType(AllReduceParams& params, AllReduceStrategyType strat, AllReduceStrategyConfig config,
    AllReduceFusionOp fusionOp, hipStream_t stream, int num_ctas)
{
    switch (params.ranks_per_node)
    {
    case 2: AllReduceDispatchRanksPerNode<T, 2>(strat, config, fusionOp, params, stream, num_ctas); break;
    case 4: AllReduceDispatchRanksPerNode<T, 4>(strat, config, fusionOp, params, stream, num_ctas); break;
    case 6: AllReduceDispatchRanksPerNode<T, 6>(strat, config, fusionOp, params, stream, num_ctas); break;
    case 8: AllReduceDispatchRanksPerNode<T, 8>(strat, config, fusionOp, params, stream, num_ctas); break;
    case 16: AllReduceDispatchRanksPerNode<T, 16>(strat, config, fusionOp, params, stream, num_ctas); break;
    default: throw std::runtime_error("Custom all reduce only supported on {2, 4, 6, 8, 16} GPUs per node.");
    }
}

AllReduceParams AllReduceParams::deserialize(int64_t* buffer, size_t tpSize, size_t tpRank, at::ScalarType dataType,
    int token_num, int hidden_size, AllReduceFusionOp op)
{
    void* const* buffer_ptrs = reinterpret_cast<void* const*>(buffer);
    int flag_offset = 1;
    auto const flag_ptr = &buffer[NUM_POINTERS_PER_RANK * tpSize + flag_offset];
    // cannot use 0 since 0 represents released state for barrier
    *flag_ptr += 1;
    // TLLM_LOG_TRACE("AllReduceParams's flag value is %d, flag offset %d", *flag_ptr, flag_offset);
    uint32_t flag_value = *flag_ptr;
    AllReduceParams params;
    // Even plugins use ping buffers, odd plugins use pong.
    // That way, we don't need to wait for other GPUs to be done
    // before copying input tensor to workspace.
    auto const buffer_offset = (flag_value % 2 == 0) ? 0 : tpSize;

    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_comm_buffer_ptrs[i] = buffer_ptrs[buffer_offset + i];
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_in[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[2 * tpSize + i]);
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_out[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[3 * tpSize + i]);
    }
    params.barrier_flag = flag_value;
    params.ranks_per_node = tpSize;
    params.local_rank = tpRank;

    return params;
}

void customAllReduce(kernels::AllReduceParams& params, at::ScalarType dataType, AllReduceStrategyType strat,
    AllReduceStrategyConfig config, AllReduceFusionOp fusionOp, hipStream_t stream, int num_ctas)
{
    TORCH_CHECK(configurationSupported(strat, params.elts_total, params.ranks_per_node, dataType),
        "Custom all-reduce configuration unsupported");

    sync_check_cuda_error(stream);

    switch (dataType)
    {
    case at::ScalarType::Float: AllReduceDispatchType<float>(params, strat, config, fusionOp, stream, num_ctas); break;
    case at::ScalarType::Half: AllReduceDispatchType<half>(params, strat, config, fusionOp, stream, num_ctas); break;
#ifdef ENABLE_BF16
    case at::ScalarType::BFloat16:
        AllReduceDispatchType<__hip_bfloat16>(params, strat, config, fusionOp, stream, num_ctas);
        break;
#endif
    default: throw std::runtime_error("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error(stream);
}

} // namespace tensorrt_llm::kernels
