/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "pytorch_extension_utils.h"

void merge_state(at::Tensor v_a, at::Tensor s_a, at::Tensor v_b, at::Tensor s_b,
                 at::Tensor v_merged, at::Tensor s_merged, int64_t hip_stream);

void merge_state_in_place(at::Tensor v, at::Tensor s, at::Tensor v_other, at::Tensor s_other,
                          std::optional<at::Tensor> mask, int64_t hip_stream);

void merge_states(at::Tensor v, at::Tensor s, at::Tensor v_merged, at::Tensor s_merged,
                  int64_t hip_stream);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("merge_state", &merge_state, "Merge two self-attention states");
  m.def("merge_state_in_place", &merge_state_in_place,
        "Merge another self-attention state in-place.");
  m.def("merge_states", &merge_states, "Merge multiple self-attention states");
}
