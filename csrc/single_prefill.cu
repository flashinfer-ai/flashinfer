/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/pos_enc.cuh>
#include <optional>

#include "pytorch_extension_utils.h"
#include "single_prefill_config.inc"

namespace flashinfer {

template <uint32_t HEAD_DIM, PosEncodingMode POS_ENCODING_MODE, bool USE_FP16_QK_REDUCTION,
          MaskMode MASK_MODE, typename AttentionVariant, typename Params>
hipError_t SinglePrefillWithKVCacheDispatched(Params params, typename Params::DTypeO* tmp,
                                               hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

void single_prefill_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                  at::Tensor o, std::optional<at::Tensor> maybe_lse,
                                  unsigned int mask_mode_code, unsigned int layout,
                                  int32_t window_left ADDITIONAL_FUNC_PARAMS, int64_t hip_stream) {
  auto device = q.device();
  unsigned int head_dim = q.size(2);
  unsigned int kv_len, qo_len, num_kv_heads, num_qo_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  qo_len = q.size(0);
  num_qo_heads = q.size(1);
  uint32_t q_stride_n = q.stride(0), q_stride_h = q.stride(1), kv_stride_n, kv_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    kv_len = k.size(0);
    num_kv_heads = k.size(1);
    kv_stride_n = k.stride(0);
    kv_stride_h = k.stride(1);
  } else {
    kv_len = k.size(1);
    num_kv_heads = k.size(0);
    kv_stride_h = k.stride(0);
    kv_stride_n = k.stride(1);
  }
  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == qo_len, lse.size(0), q.size(0));
    TORCH_CHECK(lse.size(1) == num_qo_heads, lse.size(1), q.size(1));
  }

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM, POS_ENCODING_MODE, USE_SLIDING_WINDOW,
      USE_LOGITS_SOFT_CAP, USE_FP16_QK_REDUCTION, AttentionVariant, Params, [&] {
        Params params;

        params.q = static_cast<DTypeQ*>(q.data_ptr());
        params.k = static_cast<DTypeKV*>(k.data_ptr());
        params.v = static_cast<DTypeKV*>(v.data_ptr());
        params.o = static_cast<DTypeO*>(o.data_ptr());
        params.lse = maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
        params.num_qo_heads = num_qo_heads;
        params.num_kv_heads = num_kv_heads;
        params.qo_len = qo_len;
        params.kv_len = kv_len;
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.kv_stride_n = kv_stride_n;
        params.kv_stride_h = kv_stride_h;
        params.head_dim = head_dim;
        params.window_left = window_left;
        params.partition_kv = false;

        ADDITIONAL_PARAMS_SETTER

        hipError_t status = flashinfer::SinglePrefillWithKVCacheDispatched<
            HEAD_DIM, POS_ENCODING_MODE,
            /*use_fp16_qk_reduction=*/USE_FP16_QK_REDUCTION, MASK_MODE, AttentionVariant>(
            params, static_cast<DTypeO*>(tmp.data_ptr()), stream);
        TORCH_CHECK(status == hipSuccess,
                    "SinglePrefillWithKVCache kernel launch failed, error: " +
                        std::string(hipGetErrorString(status)));
        return true;
      });
}
