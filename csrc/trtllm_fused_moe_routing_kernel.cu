#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flashinfer/trtllm/fused_moe/RoutingKernel.cuh"

namespace moe::dev::routing {

namespace routingDeepSeek {

////////////////////////////////////////////////////////////////////////////////////////////////////

static constexpr int NumThreads = 256;
static constexpr int NumWarps = NumThreads / WarpSize;
static constexpr int NumTopGroupScores = 2;
static constexpr int MaxNumTopExperts = 8;
static constexpr int MaxNumTopGroups = 4;

template <typename KernelParams>
__global__ void routingMainKernel(KernelParams params) {
  // declare types
  using OutputT = typename KernelParams::OutputT;
  using InputT = typename KernelParams::InputT;

  // declare shared memory structure
  // number of experts is bounded by number of threads
  __shared__ float __attribute((aligned(128))) smemScoreSigmoid[NumThreads];
  __shared__ float __attribute((aligned(128))) smemScoreBias[NumThreads];
  // number of expert groups is bounded by number of warps
  __shared__ float __attribute((aligned(128))) smemGroupScores[NumWarps];

  // needed for warp reduce
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WarpSize>(block);
  // for the final reduction of weight norm, only some lanes need to participate
  int32_t laneIdx = threadIdx.x % WarpSize;
  int32_t warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
  // warps outside the range of expert groups do not participate
  if constexpr (KernelParams::UseGroups) {
    if (warpIdx >= params.mNumExpertGroups) {
      return;
    }
  }

  // note that for invalid scores, we simply use a negative value:
  // they work well even with the compacted format used in topK, and
  // sigmoid / bias activated scores cannot be negative
  static constexpr float invalidScoreFloat = -1.F;
  const OutputT invalidScore = OutputT{invalidScoreFloat};

  // load bias already; each warp represents one expert group
  auto threadExpert = threadIdx.x;
  bool expertSelected = threadExpert < params.mNumExperts;
  if constexpr (KernelParams::UseGroups) {
    threadExpert = warpIdx * params.mNumExpertsPerGroup + laneIdx;
    expertSelected = laneIdx < params.mNumExpertsPerGroup;
  }
  auto scoreIdx = int64_t{blockIdx.x} * int64_t{params.mNumExperts} + threadExpert;
  auto biasVal = expertSelected ? params.mPtrRoutingBias[threadExpert] : invalidScore;

  // initialize the mPtrExpertCounts
  if (params.mPtrExpertCounts) {
    int32_t globalThreadIdx = blockIdx.x * NumThreads + threadIdx.x;
    int32_t globalThreadStride = gridDim.x * NumThreads;
    int32_t expertCountsNum = 2 * params.mNumExperts;
    initArr(globalThreadIdx, expertCountsNum, globalThreadStride, params.mPtrExpertCounts, 0);
  }

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
  // trigger the secondary kernel when using PDL, then wait on primary
  if constexpr (KernelParams::UsePdl) {
    cudaTriggerProgrammaticLaunchCompletion();
    cudaGridDependencySynchronize();
  }
#endif

  // get our assigned thread score; each warp represents one expert group
  float score =
      expertSelected ? static_cast<float>(params.mPtrScores[scoreIdx]) : invalidScoreFloat;
  // get the sigmoid score
  // note that for invalid values, we simply use a negative value:
  // sigmoig scores are always strictly positive
  auto scoreSigmoid = sigmoid_accurate(score);
  // write the sigmoid score to shared for later use
  if (expertSelected) {
    smemScoreSigmoid[threadExpert] = scoreSigmoid;
  }
  // get the score with bias
  // note that with invalid values, because sigmoid is < 1 and bias is -1,
  // we must get a negative value, which is smaller than any valid value
  auto scoreBias = float{scoreSigmoid + float{biasVal}};

  if (expertSelected) {
    smemScoreBias[threadExpert] = scoreBias;
  }

  // registers for top group score reduction
  float topExpGroupScores[NumTopGroupScores];
  [[maybe_unused]] int32_t topExpGroupIdx[NumTopGroupScores];
  float topGroups[MaxNumTopGroups];  // bound of params.mNumLimitedGroups
  int32_t topGroupIdx[MaxNumTopGroups];
  float expertScoreGroup[MaxNumTopGroups];
  int32_t expertIdxGroup[MaxNumTopGroups];
  float topScores[MaxNumTopExperts];  // bound of params.mTopK
  int32_t topExperts[MaxNumTopExperts];

  if constexpr (KernelParams::UseGroups) {
    topk::reduceTopK(warp, topExpGroupScores, topExpGroupIdx, scoreBias, threadExpert,
                     /* minValue */ invalidScoreFloat);

    // get the final group score and write it to shared
    if (cute::elect_one_sync()) {
      auto groupScore = topExpGroupScores[0] + topExpGroupScores[1];
      smemGroupScores[warpIdx] = groupScore;
    }
  }

  // make group scores available to all warps
  __syncthreads();

  auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
  if (warpIdx == 0) {
    // a single warp performs the selection of top groups, and goes on to select the final experts
    if constexpr (KernelParams::UseGroups) {
      float groupScore =
          laneIdx < params.mNumExpertGroups ? smemGroupScores[laneIdx] : invalidScoreFloat;

      topk::reduceTopK(warp, topGroups, topGroupIdx, groupScore, laneIdx,
                       /* minValue */ invalidScoreFloat);

      // final expert selection: get relevant indexes and scores from shared

#pragma unroll
      for (int ii = 0; ii < MaxNumTopGroups; ++ii) {  // bound of params.mNumLimitedGroups
        auto groupIdx = topGroupIdx[ii];
        expertIdxGroup[ii] = groupIdx * params.mNumExpertsPerGroup + laneIdx;
        // note: expertSelected implies laneIdx < params.mNumExpertsPerGroup.
        // we have params.mNumExpertsPerGroup == params.mNumExperts / params.mNumExpertGroups,
        // thus groupIdx <= params.mNumExpertGroups - 1 =>
        // groupIdx * params.mNumExpertsPerGroup <= params.mNumExperts - params.mNumExpertsPerGroup
        // => expertIdxGroup[ii] < params.mNumExperts <= NumThreads,
        // so the access is safe here
        expertScoreGroup[ii] = groupIdx < params.mNumExpertGroups && expertSelected
                                   ? smemScoreBias[expertIdxGroup[ii]]
                                   : invalidScoreFloat;
      }
    } else {
      // without groups, each thread just takes `MaxNumTopGroups` experts

#pragma unroll
      for (int ii = 0; ii < MaxNumTopGroups; ++ii) {
        auto expertIdx = ii * WarpSize + laneIdx;
        expertIdxGroup[ii] = expertIdx;
        expertScoreGroup[ii] =
            expertIdx < params.mNumExperts ? smemScoreBias[expertIdx] : invalidScoreFloat;
      }
    }

    topk::reduceTopK(warp, topScores, topExperts, expertScoreGroup, expertIdxGroup,
                     /* minValue */ invalidScoreFloat, params.mTopK);

    // determine our lane's expert index and write to output
    int32_t expertIdx = 0;
#pragma unroll
    for (int ii = 0; ii < params.mTopK; ++ii) {  // bound of params.mTopK
      expertIdx = laneIdx == ii ? topExperts[ii] : expertIdx;
    }
    // determine whether our expert is local to this GPU
    auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
    auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent &&
                         (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;

    float scoreNorm = laneIdx < params.mTopK ? smemScoreSigmoid[expertIdx] : 0.F;
    auto redNorm = cg::reduce(warp, scoreNorm, cg::plus<float>{});
    auto finalScore = OutputT{scoreNorm * params.mRouteScale / redNorm};

    // write expert idx out already
    auto idxTopK = blockIdx.x * params.mTopK + laneIdx;
    if (laneIdx < params.mTopK && params.mPtrExpertIdx != nullptr) {
      PackedScoreIdx<OutputT> packedScore{static_cast<OutputT>(finalScore),
                                          static_cast<int16_t>(expertIdx)};
      params.mPtrExpertIdx[idxTopK] = packedScore;
    }

    if (laneIdx < params.mTopK && params.mPtrExpertWeights != nullptr) {
      params.mPtrExpertWeights[idxTopK] = finalScore;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __cluster_dims__(NumBlocksPerCluster, 1, 1) __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams params) {
  using OutputT = typename KernelParams::OutputT;

  int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
  int32_t const clusterBlockRank = blockIdx.x;

  //@todo: try to move it into routingPermutation
  // then wait on primary grid
  if constexpr (KernelParams::UsePdl) {
    cudaGridDependencySynchronize();
  }

  routingPermutation<KernelParams, OutputT, NumThreads, NumWarps, MaxNumTopExperts,
                     /*LoadExpertIdxFromGlobal=*/true>(params, nullptr, warpIdx, clusterBlockRank);
}
#else
__global__ void routingIndicesClusterKernel(KernelParams params) {
  assert(false && "routingIndicesClusterKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __launch_bounds__(NumThreads) routingIndicesCoopKernel(KernelParams params) {
  // number of experts is bounded by number of threads
  __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];
  __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreads];
  // needed for the exclusive sum of token offsets
  using Scan = hipcub::BlockScan<int32_t, NumThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
  __shared__ typename Scan::TempStorage tempStorage;
  // 64 elements -> 128+ registers. Above that we may start to see spilling to local memory.
  static constexpr int MaxExpandedIdxPerThread = 64;

  // Initialize grid.
  cg::grid_group grid = cg::this_grid();
  // Note: the following is more efficient than grid.block_index() because we don't use y and z.
  int32_t const gridBlockIdx = blockIdx.x;
  int32_t const gridThreadIdx = NumThreads * gridBlockIdx + threadIdx.x;
  int32_t const numBlocks = gridDim.x;
  int32_t const numThreadsPerGrid = numBlocks * NumThreads;

  int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);

  auto expandedIdxSize = params.mNumTokens * params.mTopK;

  // pre-fill the counts with 0
  smemExpertCount[threadIdx.x] = 0;
  __syncthreads();

  // then wait on primary grid
  if constexpr (KernelParams::UsePdl) {
    cudaGridDependencySynchronize();
  }

  // each thread keeps has some number of "expanded indexes" assigned to it
  // for each of these, we keep the associated expert and offset within expert in registers
  int32_t expertIndexes[MaxExpandedIdxPerThread];
  int32_t expertOffsets[MaxExpandedIdxPerThread];
  auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
  // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
  // time, and branch between a fast path without bound checks and a slow path with bound checks.
  int constexpr IterStride = 4;
  static_assert(MaxExpandedIdxPerThread % IterStride == 0);

  // Define a lambda to avoid code duplication in both branches.
  auto loopBody = [&](int ii, int expandedIdx) {
    int32_t expertIdx = params.mPtrExpertIdx[expandedIdx].idx;
    expertIndexes[ii] = expertIdx;
    // check whether this expert is local to our GPU at all and ignore if not
    auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
    auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent &&
                         (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
    expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + expertIdx, 1) : 0;
  };

#pragma unroll
  for (int32_t ii0 = 0; ii0 < MaxExpandedIdxPerThread; ii0 += IterStride) {
    // Whether it's safe to do multiple iterations without bound checks.
    bool const takeFastPath = (ii0 + IterStride) * numThreadsPerGrid <= expandedIdxSize;
    if (takeFastPath) {
#pragma unroll
      for (int32_t jj = 0; jj < IterStride; jj++) {
        int const ii = ii0 + jj;
        auto expandedIdx = static_cast<int32_t>(gridThreadIdx) + ii * numThreadsPerGrid;
        loopBody(ii, expandedIdx);
      }
    } else {
      bool doBreak = false;
#pragma unroll
      for (int32_t jj = 0; jj < IterStride; jj++) {
        int const ii = ii0 + jj;
        auto expandedIdx = static_cast<int32_t>(gridThreadIdx) + ii * numThreadsPerGrid;
        if (expandedIdx >= expandedIdxSize) {
          doBreak = true;
          break;
        }
        loopBody(ii, expandedIdx);
      }
      if (doBreak) {
        break;
      }
    }
  }

  // Make histogram (token counts per expert) available to all threads in the block.
  __syncthreads();

  //
  // Each thread now represents one expert
  //

  // Add the local bin count to the common bin count and get a per-CTA offset.
  int32_t const localExpertCount = smemExpertCount[threadIdx.x];

  int32_t blockExpertOffset = 0;
  if (threadIdx.x < params.mNumExperts) {
    blockExpertOffset = atomicAdd(&params.mPtrExpertCounts[threadIdx.x], localExpertCount);
  }

  // Sync to wait for completion of the histogram reduction.
  grid.sync();

  // Get total count for this expert.
  int32_t count = (threadIdx.x < params.mNumExperts) ? params.mPtrExpertCounts[threadIdx.x] : 0;

  // Note: the scan is redundant in all CTAs, but doing it in only 1 CTA would be worse for latency.

  // Compute the runtime config for projections
  // Whether or not an expert is local is taken into account when smemExpertCount is computed
  // so we do not need to take it into account here.
  const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
  int32_t ctaOffset;
  int32_t numNonExitingCtas;
  Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

  for (int32_t cta = gridBlockIdx; cta < numCta; cta += numBlocks) {
    const int32_t localExpertIdx =
        (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
    params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
    params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta] =
        min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
            mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + count);
  }

  // get the padded offset associated with this expert
  const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
  const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);

  // write out padded count
  if (gridBlockIdx == 0 && warpIdx == NumWarps - 1 && cute::elect_one_sync()) {
    params.mPtrPermutedIdxSize[0] = permutedIdxSize;
    params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
  }

  // write expert offsets to shared
  smemExpertOffset[threadIdx.x] = offset + blockExpertOffset;

  // make expert offsets available to all threads
  __syncthreads();

  // trigger the secondary kernel when using PDL
  // We can't do it earlier because FC1 depends on the mPtrCtaIdxXyToBatchIdx,
  // mPtrCtaIdxXyToMnLimit, mPtrNumNonExitingCtas and mPtrTotalNumPaddedTokens
  // TODO: this is not sufficient to ensure visibility in the next kernel!
  if constexpr (KernelParams::UsePdl) {
    cudaTriggerProgrammaticLaunchCompletion();
  }

// each thread has the same "expanded indexes" assigned to it as above
// at this point, we know the final offsets of experts and the offsets within
// experts, which allows writing the final index values
#pragma unroll
  for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ++ii) {
    auto expandedIdx = static_cast<int32_t>(gridThreadIdx) + ii * numThreadsPerGrid;
    if (expandedIdx >= expandedIdxSize) {
      break;
    }
    auto expertIdx = expertIndexes[ii];
    // check whether this expert is local to our GPU at all
    auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
    auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent &&
                         (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
    auto tokenIdx = expandedIdx / params.mTopK;
    auto permutedIdx =
        isLocalExpert ? int32_t{smemExpertOffset[expertIdx]} + expertOffsets[ii] : int32_t{-1};
    if (params.mPtrExpandedIdxToPermutedIdx != nullptr) {
      params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
    }
    if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert) {
      params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
    }
  }
}
#else
__global__ void routingIndicesCoopKernel(KernelParams params) {
  assert(false && "routingIndicesCoopKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data& data, void* stream) {
  TORCH_CHECK(data.mPtrExpertIdx != nullptr || data.mPtrPermutedIdxSize != nullptr ||
                  data.mPtrExpertWeights != nullptr,
              "Routing kernel requires at least one output parameter");
  if (data.mPtrExpandedIdxToPermutedIdx != nullptr || data.mPtrPermutedIdxToTokenIdx != nullptr) {
    TORCH_CHECK(data.mPtrExpertIdx != nullptr && data.mPtrPermutedIdxSize,
                "If permuted index is required, `mPtrExpertIdx` is also required");
  }
  TORCH_CHECK(!data.mUseRoutingSoftmax, "Routing with softmax not implemented yet");
  TORCH_CHECK(data.mNumLimitedGroups <= MaxNumTopGroups,
              "Routing kernel expects <= %d top groups, got %d", MaxNumTopGroups,
              data.mNumLimitedGroups);
  TORCH_CHECK(data.mTopK <= MaxNumTopExperts, "Routing kernel expects topK experts <= %d, got %d",
              MaxNumTopExperts, data.mTopK);
  TORCH_CHECK(data.mTopK <= WarpSize, "Routing kernel expects top K <= warp size, got %d",
              data.mTopK);
  TORCH_CHECK(data.mTopK * data.mNumLimitedGroups <= WarpSize,
              "Routing kernel expects top K * top groups <= warp size (for now), got %d * %d",
              data.mTopK, data.mNumLimitedGroups);
  TORCH_CHECK(data.mNumExperts >= MaxNumTopExperts,
              "Routing kernel expects %d to be at most #experts %d", MaxNumTopExperts,
              data.mNumExperts);
  TORCH_CHECK(data.mNumExperts <= NumThreads, "Routing kernel expects #experts %d  <= #threads %d",
              data.mNumExperts, NumThreads);
  TORCH_CHECK(data.mNumExpertGroups >= data.mNumLimitedGroups,
              "Routing kernel expects top groups %d to be limited by #expert groups %d",
              data.mNumLimitedGroups, data.mNumExpertGroups);
  if (data.mNumExpertGroups > 1) {
    TORCH_CHECK(data.mNumExpertGroups <= NumWarps,
                "Routing kernel expects #experts groups %d to be <= #warps %d",
                data.mNumExpertGroups, NumWarps);
    TORCH_CHECK(data.mNumExperts % data.mNumExpertGroups == 0,
                "Routing kernel expects #experts %d to be a multiple of #expert groups %d",
                data.mNumExperts, data.mNumExpertGroups);
    TORCH_CHECK(data.mNumExperts / data.mNumExpertGroups <= WarpSize,
                "Routing kernel expects #experts per group <= warp size, got %d",
                data.mNumExperts / data.mNumExpertGroups);
  } else {
    TORCH_CHECK(data.mNumExperts <= WarpSize * MaxNumTopGroups,
                "Routing kernel expects #experts %d <= WarpSize * MaxNumTopGroups %d",
                data.mNumExperts, WarpSize * MaxNumTopGroups);
    TORCH_CHECK(data.mTopK <= NumWarps, "Routing kernel expects top K %d to be <= #warps %d",
                data.mTopK, NumWarps);
  }
  TORCH_CHECK(data.mNumExperts % 4 == 0,
              "Routing kernel expects #experts %d to be a multiple of 4.", data.mNumExperts);
  TORCH_CHECK(data.mPaddingLog2 < 8, "Routing kernel expects padding log2 < 8, got %d",
              data.mPaddingLog2);
  int const numBlocks = data.mNumTokens;

  bool const useSingleCluster = data.mNumTokens <= 1024;
  if (!useSingleCluster) {
    // Reset the global histograms (not used in single-cluster code path).
    // Cover both for the cooperative and two-kernel code paths.
    TORCH_CHECK(data.mPtrExpertCounts != nullptr,
                "When #tokens is large, `mPtrExpertCounts` is a required input.");
  } else {
    data.mPtrExpertCounts =
        nullptr;  // Set it to nullptr for single-cluster code path, as it won't be used
  }

  // Number of blocks we can use in the cooperative kernel
  // The number of blocks must be:
  //   >= ⌈(numTokens * topK) / (MaxExpandedIdxPerThread * NumThreads)⌉
  //   <= numSms, assuming an occupancy of 1 block/SM
  //
  // If too small for the given numTokens, fall back to the less performant two-step method.
  //
  // The upper bound is a strict requirement. The number of blocks should be determined by querying
  // the device properties, or conservatively low.
  // /!\ The following number is not portable!! (but works on H100 and B200)
  int const numBlocksCoop = 128;

  // Maximum number of tokens supported by the kernel using a cooperative launch.
  int const maxTokensCoop = (numBlocksCoop * NumThreads * 64) / data.mTopK;
  LAUNCH_ROUTING_WITH_EXTRA_FLAG(data,
                                 /*coopLaunch=*/false, routingMainKernel, numBlocks, NumThreads,
                                 /*smemSize=*/0,  // No dynamic smem
                                 stream, data.mNumExpertGroups > 1, /*forceFloatInput=*/true);

  if (data.mPtrPermutedIdxSize != nullptr) {
    if (useSingleCluster) {
      LAUNCH_ROUTING_WITH_EXTRA_FLAG(data,
                                     /*coopLaunch=*/false, routingIndicesClusterKernel,
                                     NumBlocksPerCluster, NumThreads,
                                     /*smemSize=*/0,  // No dynamic smem
                                     stream, data.mNumExpertGroups > 1, /*forceFloatInput=*/true);
    } else if (data.mNumTokens <= maxTokensCoop) {
      LAUNCH_ROUTING_WITH_EXTRA_FLAG(data,
                                     /*coopLaunch=*/true, routingIndicesCoopKernel, numBlocksCoop,
                                     NumThreads,
                                     /*smemSize=*/0,  // No dynamic smem
                                     stream, data.mNumExpertGroups > 1, /*forceFloatInput=*/true);
    } else {
      const int32_t expandedIdxSize = data.mNumTokens * data.mTopK;

      const int32_t histogramEltsPerBlock = 8 * NumThreads;
      const int32_t offsetEltsPerBlock = NumEltsPerOffsetTilePerThread * NumThreads;

      // Limit grid size (both kernels use a grid-stride loop).
      const int32_t maxNumBlocks = 1024;

      int const numBlocksHistogram = std::min(
          (expandedIdxSize + histogramEltsPerBlock - 1) / histogramEltsPerBlock, maxNumBlocks);
      int const numBlocksOffsets =
          std::min((expandedIdxSize + offsetEltsPerBlock - 1) / offsetEltsPerBlock, maxNumBlocks);

      LAUNCH_ROUTING_WITH_EXTRA_FLAG(data,
                                     /*coopLaunch=*/false, routingIndicesHistogramKernel,
                                     numBlocksHistogram, NumThreads,
                                     /*smemSize=*/0,  // No dynamic smem
                                     stream, data.mNumExpertGroups > 1, /*forceFloatInput=*/true);
      LAUNCH_ROUTING_WITH_EXTRA_FLAG(data,
                                     /*coopLaunch=*/false, routingIndicesOffsetsKernel,
                                     numBlocksOffsets, NumThreads,
                                     /*smemSize=*/0,  // No dynamic smem
                                     stream, data.mNumExpertGroups > 1, /*forceFloatInput=*/true);
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

}  // namespace routingDeepSeek

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace routingLlama4 {

////////////////////////////////////////////////////////////////////////////////////////////////////

static constexpr int NumThreads = 1024;
static constexpr int NumWarps = NumThreads / WarpSize;
static constexpr int MaxNumTopExperts = 1;
static constexpr int MaxNumExperts = 128;
static constexpr int MaxNumTokensSingleCluster = NumBlocksPerCluster * NumThreads;
static constexpr int MaxNumTokensSingleClusterScores = NumBlocksPerCluster * NumWarps;
static constexpr int WarpKernelSmemStride = 33;
// with further optimization to `routingIndicesWarpKernel`, this limit may
// increase. For now, it is a good cut-off point for when the block-wise
// operations are more efficient end-to-end.
static constexpr int WarpKernelMaxNumTokens = 4;

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename DataType, int VecSize>
__forceinline__ __device__ void routingTopKExperts(cg::thread_block_tile<WarpSize> const& warp,
                                                   DataType (&warpMaxScore)[MaxNumTopExperts],
                                                   int32_t (&warpMaxExpertIdx)[MaxNumTopExperts],
                                                   int32_t const laneIdx, int32_t const numExperts,
                                                   DataType const* ptrScores) {
  DataType minScore = DataType{-INFINITY};
  DataType maxScore = minScore;
  int32_t maxExpertIdx{-1};
  using DataTypeVec = std::conditional_t<sizeof(DataType) == 2, float2, float4>;

  // Non-vectorized loading: directly access ptrScores with expertIdx
  for (int i = 0; i < VecSize; ++i) {
    auto expertIdx = i * WarpSize + laneIdx;
    auto newScore = expertIdx < numExperts ? ptrScores[expertIdx] : minScore;
    // note: use `>=` s.t. highest index always wins, just like in `reduceTopK`
    if (newScore > maxScore) {
      maxScore = newScore;
      maxExpertIdx = expertIdx;
    }
  }

  topk::reduceTopK(warp, warpMaxScore, warpMaxExpertIdx, maxScore, maxExpertIdx, minScore);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void __launch_bounds__(WarpSize) routingIndicesWarpKernel(KernelParams params) {
  // types used in this kernel
  using OutputT = typename KernelParams::OutputT;
  using InputT = typename KernelParams::InputT;
  using TypePacked = PackedScoreIdx<OutputT>;
  // use the default cub warp-scan, with shfl
  using Scan = hipcub::WarpScan<int32_t>;
  __shared__ typename Scan::TempStorage tempStorage;

  // each thread encodes 4 experts in one `int32_t`. The assumption is that
  // we don't have more than 127 tokens, but `WarpKernelMaxNumTokens` must be
  // smaller than that because other approaches will be more efficient for
  // 127 tokens.
  static constexpr int ExpertsPerThread = sizeof(int32_t);
  static_assert(WarpKernelMaxNumTokens <= 127);
  // this is a full table of which token is routed to which expert.
  // the assumption here is that there are no more than 128 experts.
  // we use a stride of 33 instead of 32 to avoid shared memory bank conflicts.
  __shared__ int32_t __attribute((
      aligned(128))) smemExpertTokenCountFull[WarpKernelMaxNumTokens][WarpKernelSmemStride];
  static_assert(WarpKernelSmemStride == WarpSize + 1);
  static_assert(MaxNumExperts / sizeof(int32_t) <= WarpSize);

  // values needed for the top-1 reduction, if required
  InputT minScore = InputT{-INFINITY};
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WarpSize>(block);

#pragma unroll
  for (int tokenIdx = 0; tokenIdx < WarpKernelMaxNumTokens; ++tokenIdx) {
    // reset full shared memory field to 0
    smemExpertTokenCountFull[tokenIdx][threadIdx.x] = 0;
  }
  __syncwarp();

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
  // then wait on primary grid
  if constexpr (KernelParams::UsePdl) {
    cudaGridDependencySynchronize();
  }
#endif

  if (params.mPtrScores != nullptr) {
    // if we use `mPtrScores` as input, we need to perform the top-1 reduction
    // for each token, we load the scores then use `reduceTopK` for this.
    // each thread works on 4 experts, so a local reduction is done before
    for (int tokenIdx = 0; tokenIdx < params.mNumTokens; ++tokenIdx) {
      auto scoreOffset = tokenIdx * params.mNumExperts;
      int32_t warpMaxExpertIdx[MaxNumTopExperts];
      InputT warpMaxScore[MaxNumTopExperts];

      // Use routingTopKExperts function instead of inline logic
      routingTopKExperts<InputT, ExpertsPerThread>(warp, warpMaxScore, warpMaxExpertIdx,
                                                   threadIdx.x, params.mNumExperts,
                                                   params.mPtrScores + scoreOffset);

      if (cute::elect_one_sync()) {
        // one thread updates the count linking token to chosen expert
        auto expertTokenCount = 0;
        setBits</* IsZero= */ true>(expertTokenCount, 1, warpMaxExpertIdx[0] % ExpertsPerThread);
        smemExpertTokenCountFull[tokenIdx][warpMaxExpertIdx[0] / ExpertsPerThread] =
            expertTokenCount;
        // we also compute the final score here and write it out if required
        auto finalScore = OutputT{sigmoid_accurate(float{warpMaxScore[0]})};
        if (params.mPtrExpertWeights != nullptr) {
          params.mPtrExpertWeights[tokenIdx] = finalScore;
        }
      }
    }
  } else {
    // if we do not have `mPtrScores` as input, we expect that `mPtrExpertWeights`
    // contains the top-1 packed score and index already.
    // Each thread represents a token here, and we extract the relevant score
    // The assumption is that the #tokens is limited by warp-size
    static_assert(WarpKernelMaxNumTokens <= WarpSize);
    TypePacked scoreIdx =
        threadIdx.x < params.mNumTokens ? params.mPtrExpertIdx[threadIdx.x] : TypePacked{};
    int32_t expertTokenCount = 0;
    setBits</* IsZero= */ true>(expertTokenCount, 1, scoreIdx.idx % ExpertsPerThread);
    if (threadIdx.x < params.mNumTokens) {
      smemExpertTokenCountFull[threadIdx.x][scoreIdx.idx / ExpertsPerThread] = expertTokenCount;
    }
    // we also compute the final score here and write it out if required
    auto finalScore = OutputT{sigmoid_accurate(float{scoreIdx.score})};
    if (params.mPtrExpertWeights != nullptr && threadIdx.x < params.mNumTokens) {
      params.mPtrExpertWeights[threadIdx.x] = finalScore;
    }
  }

  // make the full table available to all threads
  __syncwarp();

  // at this point, each thread keeps a count of its 4 assigned experts in
  // `expertCount`, as well as the offsets for all tokens w.r.t. these 4 experts
  // in `expertOffset`.
  int32_t expertCount = 0;
  int32_t expertOffset[WarpKernelMaxNumTokens + 1];
#pragma unroll
  for (int tokenIdx = 0; tokenIdx < WarpKernelMaxNumTokens + 1; ++tokenIdx) {
    if (tokenIdx > params.mNumTokens) break;
    // simple reduction for `expertCount`, and scan for `expertOffset`
    auto expertTokenCount =
        tokenIdx < params.mNumTokens ? smemExpertTokenCountFull[tokenIdx][threadIdx.x] : 0;
    expertOffset[tokenIdx] = expertCount;
    expertCount += expertTokenCount;
  }

  // at this point, we are ready for the scan across all experts to get the
  // thread-wise offsets across experts
  // first, we need to reduce across our 4 experts into `numCta`
  int32_t numCta = 0;
#pragma unroll
  for (int ii = 0; ii < ExpertsPerThread; ++ii) {
    auto count = getBits(expertCount, ii);
    numCta += divUpLog2<int32_t>(count, params.mPaddingLog2);
  }
  // second, we perform the exclusive sum across the warp
  int32_t ctaOffset;
  int32_t numNonExitingCtas;
  Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

  // finally, we perform a scan across our local experts, starting with the
  // warp-wide scan result (`ctaOffset`)
  auto ctaOffsetExp = ctaOffset;
#pragma unroll
  for (int ii = 0; ii < ExpertsPerThread; ++ii) {
    auto count = getBits(expertCount, ii);
    auto finalNumCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    auto expertIdx = threadIdx.x * ExpertsPerThread + ii;
    // during the scan for expert offsets, we can already write out
    // both `mPtrCtaIdxXyToBatchIdx` and `mPtrCtaIdxXyToMnLimit`
    for (int cta = 0; cta < finalNumCta; ++cta) {
      params.mPtrCtaIdxXyToBatchIdx[ctaOffsetExp + cta] = expertIdx;
      params.mPtrCtaIdxXyToMnLimit[ctaOffsetExp + cta] =
          min(mulLog2<int32_t>(ctaOffsetExp + cta + 1, params.mPaddingLog2),
              mulLog2<int32_t>(ctaOffsetExp, params.mPaddingLog2) + count);
    }
    ctaOffsetExp += finalNumCta;
  }

  // at this point, we can write out padded count from the warp-aggregate
  if (cute::elect_one_sync()) {
    const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);
    params.mPtrPermutedIdxSize[0] = permutedIdxSize;
    params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
  }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
#if !defined(PDL_PROFILE) || PDL_PROFILE == 0
  // we can trigger the next kernel at this point
  if constexpr (KernelParams::UsePdl) {
    cudaTriggerProgrammaticLaunchCompletion();
  }
#endif
#endif

  // at this point, all values for offsets are ready, except the final offsets
  // within the padded index (`permutedIdx`)
  // for this, we perform a scan similar to the one directly after the warp-scan:
  // here, we keep the local offset for each of the thread's experts in a field
  // of registers
  auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
  int32_t finalExpertOffset[ExpertsPerThread];
  finalExpertOffset[0] = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
#pragma unroll
  for (int ii = 1; ii < ExpertsPerThread; ++ii) {
    finalExpertOffset[ii] =
        finalExpertOffset[ii - 1] +
        divUpMulLog2<int32_t>(getBits(expertCount, ii - 1), params.mPaddingLog2);
  }

#pragma unroll
  for (int tokenIdx = 0; tokenIdx < WarpKernelMaxNumTokens; ++tokenIdx) {
    // at this point, we can calculate the final index:
    // we simply loop over all tokens, and all experts assigned to this thread.
    // For each pair, we determine whether that token was routed to that expert
    // based on whether the offset for that token changed.
    // we can then easily compute the final `expertIdx` and `permutedIdx` relative
    // to this token and expert, and write them out.
    if (tokenIdx >= params.mNumTokens) break;

#pragma unroll
    for (int ii = 0; ii < ExpertsPerThread; ++ii) {
      // determine whether the offset for this expert and token changes
      auto localOffsetToken = getBits(expertOffset[tokenIdx], ii);
      auto isTokenRouted = getBits(expertOffset[tokenIdx + 1], ii) > localOffsetToken;
      // the expert index of this expert
      auto expertIdx = threadIdx.x * ExpertsPerThread + ii;
      auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
      auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent &&
                           (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
      // the permuted index: we add the local offset relative to this expert and token
      // to the global offset from the scan for this expert
      auto permutedIdx = isLocalExpert ? finalExpertOffset[ii] + localOffsetToken : int32_t{-1};
      // write out `mPtrExpandedIdxToPermutedIdx` if required
      if (params.mPtrExpandedIdxToPermutedIdx != nullptr && isTokenRouted) {
        params.mPtrExpandedIdxToPermutedIdx[tokenIdx] = permutedIdx;
      }
      // write out `mPtrPermutedIdxToTokenIdx` if required
      if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert && isTokenRouted) {
        params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
      }
    }
  }
}
////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __cluster_dims__(NumBlocksPerCluster, 1, 1) __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams params) {
  // number of tokens/expanded idx is bounded by total number of warps
  using OutputT = typename KernelParams::OutputT;
  using InputT = typename KernelParams::InputT;
  using TypePacked = PackedScoreIdx<OutputT>;
  __shared__ TypePacked __attribute((aligned(128))) smemPackedScoreIdx[NumWarps];

  uint32_t const clusterBlockRank = blockIdx.x;
  int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
  int32_t const laneIdx = cutlass::arch::LaneId();

  // TODO(mjoux): expand to more tokens (possibly)
  auto warpTokenIdx = clusterBlockRank * NumWarps + warpIdx;
  auto scoreOffset = warpTokenIdx * params.mNumExperts;
  bool validToken = warpTokenIdx < params.mNumTokens;
  InputT minScore = InputT{-INFINITY};

  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WarpSize>(block);

  // then wait on primary grid
  if constexpr (KernelParams::UsePdl) {
    cudaGridDependencySynchronize();
  }

  if (params.mPtrScores != nullptr) {
    // in this case, each warp represents a token
    // we then exchange all token max scores, s.t. afterwards, each thread
    // represents a token
    InputT warpMaxScore[MaxNumTopExperts];
    int32_t warpMaxExpertIdx[MaxNumTopExperts];

    if (validToken) {
      routingTopKExperts<InputT, MaxNumExperts / WarpSize>(warp, warpMaxScore, warpMaxExpertIdx,
                                                           laneIdx, params.mNumExperts,
                                                           params.mPtrScores + scoreOffset);
      if (cute::elect_one_sync()) {
        auto finalScore = OutputT{sigmoid_accurate(float{warpMaxScore[0]})};
        TypePacked packedScore{finalScore, static_cast<int16_t>(warpMaxExpertIdx[0])};
        smemPackedScoreIdx[warpIdx] = packedScore;
      }
    }
    // make packed scores available to all threads in cluster
    __cluster_barrier_arrive();
    __cluster_barrier_wait();
  }

  routingPermutation<KernelParams, OutputT, NumThreads, NumWarps, MaxNumTopExperts,
                     /*LoadExpertIdxFromGlobal=*/false>(params, smemPackedScoreIdx, warpIdx,
                                                        clusterBlockRank);
}
#else
__global__ void routingIndicesClusterKernel(KernelParams params) {
  assert(false && "routingIndicesClusterKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

// this kernel is needed in case we have scores as input for the histogram kernel
template <typename KernelParams>
__global__ void __launch_bounds__(NumThreadsHist)
    routingIndicesHistogramScoresKernel(KernelParams params) {
  using OutputT = typename KernelParams::OutputT;
  using InputT = typename KernelParams::InputT;
  using TypePacked = PackedScoreIdx<OutputT>;
  static constexpr int VecSize = MaxNumExperts / WarpSize;
  //  we assume that #experts is a multiple of 4, so VecSize must be 4.
  static_assert(VecSize == 4);

  int32_t const laneIdx = cutlass::arch::LaneId();
  int32_t const warpIdx = threadIdx.x / WarpSize;
  int32_t const globalWarpIdx = blockIdx.x * NumWarpsHist + warpIdx;
  int32_t const globalWarpStride = gridDim.x * NumWarpsHist;
  InputT minScore = InputT{-INFINITY};
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WarpSize>(block);

  // initialize the mPtrExpertCounts
  int32_t expertCountsNum = 2 * params.mNumExperts;
  int32_t globalThreadIdx = blockIdx.x * NumThreads + threadIdx.x;
  int32_t globalThreadStride = gridDim.x * NumThreads;
  initArr(globalThreadIdx, expertCountsNum, globalThreadStride, params.mPtrExpertCounts, 0);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
  // Wait on primary grid and trigger secondary kernel.
  if constexpr (KernelParams::UsePdl) {
    cudaGridDependencySynchronize();
    cudaTriggerProgrammaticLaunchCompletion();
  }
#endif

  // in this case, each warp represents a token, and we use a grid-stride loop
  // over all warps/tokens
  for (int tokenIdx = globalWarpIdx; tokenIdx < params.mNumTokens; tokenIdx += globalWarpStride) {
    auto scoreOffset = tokenIdx * params.mNumExperts;
    int32_t warpMaxExpertIdx[MaxNumTopExperts];
    InputT warpMaxScore[MaxNumTopExperts];

    routingTopKExperts<InputT, MaxNumExperts / WarpSize>(warp, warpMaxScore, warpMaxExpertIdx,
                                                         laneIdx, params.mNumExperts,
                                                         params.mPtrScores + scoreOffset);

    if (cute::elect_one_sync()) {
      auto finalScore = OutputT{sigmoid_accurate(float{warpMaxScore[0]})};
      TypePacked packedScore{finalScore, static_cast<int16_t>(warpMaxExpertIdx[0])};
      params.mPtrExpertIdx[tokenIdx] = packedScore;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream) {
  TORCH_CHECK(data.mPtrExpertIdx != nullptr || data.mPtrScores != nullptr,
              "Routing kernel requires at least one input parameter");
  TORCH_CHECK(data.mPtrPermutedIdxSize != nullptr && data.mPtrCtaIdxXyToBatchIdx != nullptr &&
                  data.mPtrCtaIdxXyToMnLimit != nullptr && data.mPtrNumNonExitingCtas != nullptr,
              "Llama4 routing kernel expects permuted idx and grouped Gemm launch config buffers");
  TORCH_CHECK(data.mTopK <= MaxNumTopExperts, "Routing kernel expects topK experts <= %d, got %d",
              MaxNumTopExperts, data.mTopK);
  TORCH_CHECK(data.mNumExperts <= MaxNumExperts,
              "Routing kernel expects #experts %d to be at most max #experts %d", data.mNumExperts,
              MaxNumExperts);
  static_assert(MaxNumExperts <= NumThreads, "#experts must be bounded by #threads");
  static_assert(MaxNumExperts <= NumThreadsHist, "#experts must be bounded by #threads");
  TORCH_CHECK(data.mNumExperts % 4 == 0,
              "Routing kernel expects #experts %d to be a multiple of 4.", data.mNumExperts);
  TORCH_CHECK(data.mPaddingLog2 < 8, "Routing kernel expects padding log2 < 8, got %d",
              data.mPaddingLog2);

  bool const useSingleWarp =
      (data.mPtrScores == nullptr && data.mNumTokens <= WarpKernelMaxNumTokens) ||
      data.mNumTokens < WarpKernelMaxNumTokens;
  bool const useSingleCluster =
      data.mNumTokens <=
      (data.mPtrScores != nullptr ? MaxNumTokensSingleClusterScores : MaxNumTokensSingleCluster);
  if (!useSingleCluster) {
    TORCH_CHECK(data.mPtrExpertIdx != nullptr,
                "When #tokens is large, `mPtrExpertIdx` is a required input.");
    TORCH_CHECK(data.mPtrExpertCounts != nullptr,
                "When #tokens is large, `mPtrExpertCounts` is a required input.");
  }

  if (useSingleWarp) {
    LAUNCH_ROUTING(data,
                   /*coopLaunch=*/false, routingIndicesWarpKernel, 1, WarpSize,
                   /*smemSize=*/0,  // No dynamic smem
                   stream);
  } else if (useSingleCluster) {
    LAUNCH_ROUTING(data,
                   /*coopLaunch=*/false, routingIndicesClusterKernel, NumBlocksPerCluster,
                   NumThreads,
                   /*smemSize=*/0,  // No dynamic smem
                   stream);
  } else {
    const uint32_t expandedIdxSize = data.mNumTokens * data.mTopK;

    const uint32_t histogramEltsPerBlock = 8 * NumThreadsHist;
    const uint32_t offsetEltsPerBlock = NumEltsPerOffsetTilePerThread * NumThreadsHist;

    // Limit grid size (all kernels use a grid-stride loop).
    const uint32_t maxNumBlocks = 1024;

    int const numBlocksHistogram = std::min(
        (expandedIdxSize + histogramEltsPerBlock - 1) / histogramEltsPerBlock, maxNumBlocks);
    int const numBlocksOffsets =
        std::min((expandedIdxSize + offsetEltsPerBlock - 1) / offsetEltsPerBlock, maxNumBlocks);

    if (data.mPtrScores != nullptr) {
      LAUNCH_ROUTING(data,
                     /*coopLaunch=*/false, routingIndicesHistogramScoresKernel, maxNumBlocks,
                     NumThreadsHist,
                     /*smemSize=*/0,  // No dynamic smem
                     stream);
    } else {
      // Reset the global histograms.
      TORCH_CHECK(hipMemsetAsync(data.mPtrExpertCounts, 0,
                                  static_cast<size_t>(2 * NumThreads) * sizeof(int32_t),
                                  (hipStream_t)stream));
    }
    LAUNCH_ROUTING(data,
                   /*coopLaunch=*/false, routingIndicesHistogramKernel, numBlocksHistogram,
                   NumThreadsHist,
                   /*smemSize=*/0,  // No dynamic smem
                   stream);
    LAUNCH_ROUTING(data,
                   /*coopLaunch=*/false, routingIndicesOffsetsKernel, numBlocksOffsets,
                   NumThreadsHist,
                   /*smemSize=*/0,  // No dynamic smem
                   stream);
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

}  // namespace routingLlama4

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace routingRenormalize {
////////////////////////////////////////////////////////////////////////////////////////////////////

static constexpr int NumThreads = 1024;
static constexpr int NumWarps = NumThreads / WarpSize;
static constexpr int MaxNumTopExperts = 8;
static constexpr int MaxNumExperts = 128;
static constexpr int MaxNumTokensSingleCluster = NumBlocksPerCluster * NumThreads;
static constexpr int MaxNumTokensSingleClusterScores = NumBlocksPerCluster * NumWarps;

template <typename DataType, typename InputType, int VecSize, bool DoSoftmaxBeforeTopK>
__forceinline__ __device__ void routingTopKExperts(
    cg::thread_block_tile<WarpSize> const& warp, DataType (&score)[VecSize],
    int32_t (&idx)[VecSize], DataType (&warpTopKScore)[MaxNumTopExperts],
    int32_t (&warpTopKExpertIdx)[MaxNumTopExperts], int32_t const laneIdx, int32_t const numExperts,
    int32_t topK, InputType const* ptrScores, bool const normTopkProb) {
  DataType minScore = DataType{-INFINITY};

  for (int i = 0; i < VecSize; i++) {
    auto expertIdx = i * WarpSize + laneIdx;
    auto newScore = expertIdx < numExperts ? static_cast<DataType>(ptrScores[expertIdx]) : minScore;
    score[i] = newScore;
    idx[i] = expertIdx;
  }
  if constexpr (DoSoftmaxBeforeTopK) {
    calcSoftmax(warp, score);
  }

  // Get the top-k scores and their corresponding expert indices
  topk::reduceTopK(warp, warpTopKScore, warpTopKExpertIdx, score, idx, minScore, topK);

  // Normalize the scores
  if constexpr (DoSoftmaxBeforeTopK) {
    float sum = float{1.f};
    if (normTopkProb) {
      sum = static_cast<float>(laneIdx < topK ? warpTopKScore[laneIdx] : 0);
      sum = cg::reduce(warp, sum, cg::plus<float>());
    }
    if (laneIdx < topK) {
      warpTopKScore[laneIdx] = warpTopKScore[laneIdx] / sum;
    }
  } else {
    auto softmaxScore =
        calcSoftmax(warp, laneIdx < topK ? warpTopKScore[laneIdx] : minScore, laneIdx, topK);
    if (laneIdx < topK) {
      warpTopKScore[laneIdx] = softmaxScore;
    }
  }
}

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __cluster_dims__(NumBlocksPerCluster, 1, 1) __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams params) {
  // number of tokens/expanded idx is bounded by total number of warps
  using OutputT = typename KernelParams::OutputT;
  using InputT = typename KernelParams::InputT;

  using BaseType = std::conditional_t<KernelParams::DoSoftmaxBeforeTopK, float, InputT>;
  using TypePacked = PackedScoreIdx<BaseType>;

  static constexpr int VecSize = MaxNumExperts / WarpSize;
  // we assume that #experts is a multiple of 4, so VecSize must be 4.
  static_assert(VecSize == 4);

  __shared__ TypePacked __attribute((aligned(128))) smemPackedScoreIdx[NumWarps * MaxNumTopExperts];

  uint32_t const clusterBlockRank = blockIdx.x;

  int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
  int32_t const laneIdx = cutlass::arch::LaneId();

  auto warpTokenIdx = clusterBlockRank * NumWarps + warpIdx;
  auto scoreOffset = warpTokenIdx * params.mNumExperts;
  bool validToken = warpTokenIdx < params.mNumTokens;

  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WarpSize>(block);

  // then wait on primary grid
  if constexpr (KernelParams::UsePdl) {
    cudaGridDependencySynchronize();
  }

  if (params.mPtrScores != nullptr) {
    // in this case, each warp represents a token
    BaseType score[VecSize];
    int32_t idx[VecSize];

    BaseType warpTopKScore[MaxNumTopExperts];
    int32_t warpTopKExpertIdx[MaxNumTopExperts];

    BaseType minScore = BaseType{-INFINITY};
    if (validToken) {
      routingTopKExperts<BaseType, InputT, VecSize, KernelParams::DoSoftmaxBeforeTopK>(
          warp, score, idx, warpTopKScore, warpTopKExpertIdx, laneIdx, params.mNumExperts,
          params.mTopK, params.mPtrScores + scoreOffset, params.mNormTopkProb);

      if (laneIdx < params.mTopK) {
        smemPackedScoreIdx[warpIdx * params.mTopK + laneIdx] =
            TypePacked{warpTopKScore[laneIdx], static_cast<int16_t>(warpTopKExpertIdx[laneIdx])};
      }
    }  // end if (validToken)

    // make packed scores available to all threads in cluster
    __cluster_barrier_arrive();
    __cluster_barrier_wait();
  }

  routingPermutation<KernelParams, BaseType, NumThreads, NumWarps, MaxNumTopExperts,
                     /*LoadExpertIdxFromGlobal=*/false>(params, smemPackedScoreIdx, warpIdx,
                                                        clusterBlockRank);
}
#else
__global__ void __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams /* params */) {
  assert(false && "routingIndicesClusterKernel is only supported on SM90+ architectures");
}
#endif  // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
////////////////////////////////////////////////////////////////////////////////////////////////////

// this kernel is needed in case we have scores as input for the histogram kernel
template <typename KernelParams>
__global__ void __launch_bounds__(NumThreadsHist)
    routingIndicesHistogramScoresKernel(KernelParams params) {
  using OutputT = typename KernelParams::OutputT;
  using InputT = typename KernelParams::InputT;
  using BaseType = std::conditional_t<KernelParams::DoSoftmaxBeforeTopK, float, InputT>;

  static constexpr int VecSize = MaxNumExperts / WarpSize;
  // we assume that #experts is a multiple of 4, so VecSize must be 4.
  static_assert(VecSize == 4);

  int32_t const laneIdx = cutlass::arch::LaneId();
  int32_t const warpIdx = threadIdx.x / WarpSize;
  int32_t const globalWarpIdx = blockIdx.x * NumWarpsHist + warpIdx;
  int32_t const globalWarpStride = gridDim.x * NumWarpsHist;
  BaseType minScore = BaseType{-INFINITY};
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WarpSize>(block);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
  // Wait on primary grid.
  if constexpr (KernelParams::UsePdl) {
    cudaGridDependencySynchronize();
  }
#endif  // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))

  // initialize the mPtrExpertCounts
  int32_t expertCountsNum = 2 * params.mNumExperts;
  int32_t globalThreadIdx = blockIdx.x * NumThreads + threadIdx.x;
  int32_t globalThreadStride = gridDim.x * NumThreads;
  initArr(globalThreadIdx, expertCountsNum, globalThreadStride, params.mPtrExpertCounts, 0);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
  // Trigger secondary kernel.
  if constexpr (KernelParams::UsePdl) {
    cudaTriggerProgrammaticLaunchCompletion();
  }
#endif  // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))

  // in this case, each warp represents a token, and we use a grid-stride loop
  // over all warps/tokens
  BaseType allScores[VecSize];
  int32_t allExpertIdx[VecSize];
  BaseType warpTopKScore[MaxNumTopExperts];
  int32_t warpTopKExpertIdx[MaxNumTopExperts];
  for (int tokenIdx = globalWarpIdx; tokenIdx < params.mNumTokens; tokenIdx += globalWarpStride) {
    auto scoreOffset = tokenIdx * params.mNumExperts;

    routingTopKExperts<BaseType, InputT, VecSize, KernelParams::DoSoftmaxBeforeTopK>(
        warp, allScores, allExpertIdx, warpTopKScore, warpTopKExpertIdx, laneIdx,
        params.mNumExperts, params.mTopK, params.mPtrScores + scoreOffset, params.mNormTopkProb);

    if (laneIdx < params.mTopK) {
      PackedScoreIdx<OutputT> packedScore{static_cast<OutputT>(warpTopKScore[laneIdx]),
                                          static_cast<int16_t>(warpTopKExpertIdx[laneIdx])};
      params.mPtrExpertIdx[tokenIdx * params.mTopK + laneIdx] = packedScore;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream) {
  TORCH_CHECK(data.mPtrExpertIdx != nullptr || data.mPtrScores != nullptr,
              "Routing kernel requires at least one input parameter");
  TORCH_CHECK(data.mPtrPermutedIdxSize != nullptr && data.mPtrCtaIdxXyToBatchIdx != nullptr &&
                  data.mPtrCtaIdxXyToMnLimit != nullptr && data.mPtrNumNonExitingCtas != nullptr,
              "Llama4 routing kernel expects permuted idx and grouped Gemm launch config buffers");
  TORCH_CHECK(data.mTopK <= MaxNumTopExperts, "Routing kernel expects topK experts <= %d, got %d",
              MaxNumTopExperts, data.mTopK);
  TORCH_CHECK(data.mNumExperts <= MaxNumExperts,
              "Routing kernel expects #experts %d to be at most max #experts %d", data.mNumExperts,
              MaxNumExperts);
  static_assert(MaxNumExperts <= NumThreads, "#experts must be bounded by #threads");
  static_assert(MaxNumExperts <= NumThreadsHist, "#experts must be bounded by #threads");
  TORCH_CHECK(data.mNumExperts % 4 == 0,
              "Routing kernel expects #experts %d to be a multiple of 4.", data.mNumExperts);
  TORCH_CHECK(data.mPaddingLog2 < 8, "Routing kernel expects padding log2 < 8, got %d",
              data.mPaddingLog2);

  bool const useSingleCluster =
      data.mNumTokens <=
      (data.mPtrScores != nullptr ? MaxNumTokensSingleClusterScores : MaxNumTokensSingleCluster);

  if (!useSingleCluster) {
    TORCH_CHECK(data.mPtrExpertIdx != nullptr,
                "When #tokens is large, `mPtrExpertIdx` is a required input.");
    TORCH_CHECK(data.mPtrExpertCounts != nullptr,
                "When #tokens is large, `mPtrExpertCounts` is a required input.");
  }

  if (useSingleCluster) {
    LAUNCH_ROUTING_WITH_EXTRA_FLAG(data, false, routingIndicesClusterKernel, NumBlocksPerCluster,
                                   NumThreads,
                                   /*smemSize=*/0,  // No dynamic smem
                                   stream, data.mDoSoftmaxBeforeTopK, /*forceFloatInput=*/false);
  } else {
    uint32_t const expandedIdxSize = data.mNumTokens * data.mTopK;

    uint32_t const histogramEltsPerBlock = 8 * NumThreadsHist;
    uint32_t const offsetEltsPerBlock = NumEltsPerOffsetTilePerThread * NumThreadsHist;

    // Limit grid size (all kernels use a grid-stride loop).
    uint32_t const maxNumBlocks = 1024;

    int const numBlocksHistogram = std::min(
        (expandedIdxSize + histogramEltsPerBlock - 1) / histogramEltsPerBlock, maxNumBlocks);
    int const numBlocksOffsets =
        std::min((expandedIdxSize + offsetEltsPerBlock - 1) / offsetEltsPerBlock, maxNumBlocks);

    if (data.mPtrScores != nullptr) {
      LAUNCH_ROUTING_WITH_EXTRA_FLAG(data, false, routingIndicesHistogramScoresKernel, maxNumBlocks,
                                     NumThreadsHist,
                                     /*smemSize=*/0,  // No dynamic smem
                                     stream, data.mDoSoftmaxBeforeTopK, /*forceFloatInput=*/false);
    } else {
      // Reset the global histograms.
      TORCH_CHECK(hipMemsetAsync(data.mPtrExpertCounts, 0,
                                  static_cast<size_t>(2 * NumThreads) * sizeof(int32_t),
                                  (hipStream_t)stream));
    }
    LAUNCH_ROUTING_WITH_EXTRA_FLAG(data, false, routingIndicesHistogramKernel, numBlocksHistogram,
                                   NumThreadsHist,
                                   /*smemSize=*/0,  // No dynamic smem
                                   stream, data.mDoSoftmaxBeforeTopK, /*forceFloatInput=*/false);
    LAUNCH_ROUTING_WITH_EXTRA_FLAG(data, false, routingIndicesOffsetsKernel, numBlocksOffsets,
                                   NumThreadsHist,
                                   /*smemSize=*/0,  // No dynamic smem
                                   stream, data.mDoSoftmaxBeforeTopK, /*forceFloatInput=*/false);
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

}  // namespace routingRenormalize
}  // namespace moe::dev::routing
