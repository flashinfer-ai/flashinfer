#include <optional>

#include "pytorch_extension_utils.h"

#include "mla_config.inc"

#include <flashinfer/attention/decode_mla_cute_sm80.cuh>
#include <flashinfer/attention/scheduler.cuh>

using namespace flashinfer;

std::vector<int64_t> BatchDecodeWithPagedKVCachePlanMLA(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor indptr, unsigned int batch_size,
    unsigned int num_qo_heads, unsigned int page_size, bool enable_cuda_graph,
    int64_t hip_stream) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();

  DecodePlanInfo plan_info;
  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);

  auto work_estimation_func =
      BatchDecodeWithPagedKVCacheWorkEstimationDispatchedMlaCuteSM80<HEAD_DIM_CKV, HEAD_DIM_KPE, QO_TILE_LEN,
                                                             AttentionVariant, Params>;
  hipError_t status =
      DecodePlan<HEAD_DIM_CKV, flashinfer::PosEncodingMode::kNone, AttentionVariant, Params>(
          static_cast<void*>(float_workspace_buffer.data_ptr()), float_workspace_size_in_bytes,
          static_cast<void*>(int_workspace_buffer.data_ptr()),
          static_cast<void*>(page_locked_int_workspace_buffer.data_ptr()),
          int_workspace_size_in_bytes, plan_info, static_cast<IdType*>(indptr.data_ptr()),
          batch_size, num_qo_heads, page_size, enable_cuda_graph, /*stream=*/stream,
          work_estimation_func);

  TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCachePlanMLA failed with error ",
              hipGetErrorString(status));

  return plan_info.ToVector();
}


void BatchDecodeWithPagedKVCacheRunMLA(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q_nope, at::Tensor q_pe,
    at::Tensor paged_ckv_cache, at::Tensor paged_kpe_cache, at::Tensor paged_kv_indptr,
    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len, at::Tensor o, float sm_scale,
    int window_left, float logits_soft_cap, float rope_scale, float rope_theta,
    std::optional<at::Tensor> maybe_lse, int64_t hip_stream) {
  DecodePlanInfo plan_info;
  plan_info.FromVector(plan_info_vec);

  auto device = q_nope.device();
  int64_t batch_size = q_nope.size(0);
  int64_t num_qo_heads = q_nope.size(1);
  int64_t page_size = paged_ckv_cache.size(1);

  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == batch_size, lse.size(0), q_nope.size(0));
    TORCH_CHECK(lse.size(1) == num_qo_heads, lse.size(1), q_nope.size(1));
  }

  TORCH_CHECK(logits_soft_cap >= 0.f, "logits_soft_cap must be non-negative");

  void* float_buffer = static_cast<void*>(float_workspace_buffer.data_ptr());
  void* int_buffer = static_cast<void*>(int_workspace_buffer.data_ptr());

  paged_kv_mla_t<DTypeKV, IdType> paged_kv(
      page_size, HEAD_DIM_CKV, HEAD_DIM_KPE, batch_size,
      static_cast<DTypeKV*>(paged_ckv_cache.data_ptr()), paged_ckv_cache.strides().data(),
      static_cast<DTypeKV*>(paged_kpe_cache.data_ptr()), paged_kpe_cache.strides().data(),
      static_cast<IdType*>(paged_kv_indices.data_ptr()),
      static_cast<IdType*>(paged_kv_indptr.data_ptr()),
      static_cast<IdType*>(paged_kv_last_page_len.data_ptr()));
  Params params(static_cast<DTypeQ*>(q_nope.data_ptr()), static_cast<DTypeQ*>(q_pe.data_ptr()),
                /*q_offset=*/nullptr, paged_kv, static_cast<DTypeO*>(o.data_ptr()),
                /*lse=*/(maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr),
                num_qo_heads, window_left, logits_soft_cap, sm_scale, rope_scale, rope_theta);

  DTypeO* tmp_v = nullptr;
  float* tmp_s = nullptr;
  params.request_indices =
      GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.request_indices_offset);
  params.kv_tile_indices =
      GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.kv_tile_indices_offset);
  params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.o_indptr_offset);
  params.kv_chunk_size_ptr =
      GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.kv_chunk_size_ptr_offset);
  if (plan_info.split_kv) {
    tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer, plan_info.v_offset);
    tmp_s = GetPtrFromBaseOffset<float>(float_buffer, plan_info.s_offset);
    if (plan_info.enable_cuda_graph) {
      params.block_valid_mask =
          GetPtrFromBaseOffset<bool>(int_buffer, plan_info.block_valid_mask_offset);
    }
  }
  params.padded_batch_size = plan_info.padded_batch_size;

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status =
      BatchDecodeWithPagedKVCacheDispatchedMlaCuteSM80<HEAD_DIM_CKV, HEAD_DIM_KPE, QO_TILE_LEN, 
                                               Params>(params, tmp_v, tmp_s, /*stream=*/stream);
  TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCache failed with error ",
              hipGetErrorString(status));
}
