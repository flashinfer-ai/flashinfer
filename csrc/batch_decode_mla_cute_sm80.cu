
#include <flashinfer/attention/decode_mla_cute_sm80.cuh>
#include <flashinfer/attention/scheduler.cuh>

#include "mla_config.inc"
#include "tvm/ffi/container/array.h"
#include "tvm_ffi_utils.h"

using namespace flashinfer;

using tvm::ffi::Array;
using tvm::ffi::Optional;

Array<int64_t> BatchDecodeWithPagedKVCachePlanMLA(ffi::TensorView float_workspace_buffer,
                                                  ffi::TensorView int_workspace_buffer,
                                                  ffi::TensorView page_locked_int_workspace_buffer,
                                                  ffi::TensorView indptr, int64_t batch_size,
                                                  int64_t num_qo_heads, int64_t page_size,
                                                  bool enable_cuda_graph) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer->shape[0] * get_element_size(float_workspace_buffer);
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer->shape[0] * get_element_size(int_workspace_buffer);

  DecodePlanInfo plan_info;
  hipSetDevice(float_workspace_buffer->device.device_id);
  const hipStream_t stream = get_stream(float_workspace_buffer->device);

  auto work_estimation_func = BatchDecodeWithPagedKVCacheWorkEstimationDispatchedMlaCuteSM80<
      HEAD_DIM_CKV, HEAD_DIM_KPE, QO_TILE_LEN, AttentionVariant, Params>;
  hipError_t status =
      DecodePlan<HEAD_DIM_CKV, flashinfer::PosEncodingMode::kNone, AttentionVariant, Params>(
          static_cast<void*>(float_workspace_buffer->data), float_workspace_size_in_bytes,
          static_cast<void*>(int_workspace_buffer->data),
          static_cast<void*>(page_locked_int_workspace_buffer->data), int_workspace_size_in_bytes,
          plan_info, static_cast<IdType*>(indptr->data), batch_size, num_qo_heads, page_size,
          enable_cuda_graph, /*stream=*/stream, work_estimation_func);

  TVM_FFI_ICHECK(status == hipSuccess)
      << "BatchDecodeWithPagedKVCachePlanMLA failed with error " << hipGetErrorString(status);

  return Array(plan_info.ToVector());
}

void BatchDecodeWithPagedKVCacheRunMLA(
    ffi::TensorView float_workspace_buffer, ffi::TensorView int_workspace_buffer,
    Array<int64_t> plan_info_vec, ffi::TensorView q_nope, ffi::TensorView q_pe,
    ffi::TensorView paged_ckv_cache, ffi::TensorView paged_kpe_cache,
    ffi::TensorView paged_kv_indptr, ffi::TensorView paged_kv_indices,
    ffi::TensorView paged_kv_last_page_len, ffi::TensorView o, double sm_scale, int64_t window_left,
    double logits_soft_cap, double rope_scale, double rope_theta,
    Optional<ffi::TensorView> maybe_lse,
    bool enable_pdl  // fake placeholder, sm80 does not support pdl
) {
  DecodePlanInfo plan_info;
  plan_info.FromVector(std::vector<int64_t>(plan_info_vec.begin(), plan_info_vec.end()));

  int64_t batch_size = q_nope->shape[0];
  int64_t num_qo_heads = q_nope->shape[1];
  int64_t page_size = paged_ckv_cache->shape[1];

  if (maybe_lse.has_value()) {
    const auto& lse = maybe_lse.value();
    TVM_FFI_ICHECK_EQ(lse->shape[0], batch_size);
    TVM_FFI_ICHECK_EQ(lse->shape[1], num_qo_heads);
  }

  TVM_FFI_ICHECK_GE(logits_soft_cap, 0.f) << "logits_soft_cap must be non-negative";

  void* float_buffer = static_cast<void*>(float_workspace_buffer->data);
  void* int_buffer = static_cast<void*>(int_workspace_buffer->data);

  paged_kv_mla_t<DTypeKV, IdType> paged_kv(
      page_size, HEAD_DIM_CKV, HEAD_DIM_KPE, batch_size,
      static_cast<DTypeKV*>(paged_ckv_cache->data), paged_ckv_cache.strides().data(),
      static_cast<DTypeKV*>(paged_kpe_cache->data), paged_kpe_cache.strides().data(),
      static_cast<IdType*>(paged_kv_indices->data), static_cast<IdType*>(paged_kv_indptr->data),
      static_cast<IdType*>(paged_kv_last_page_len->data));
  Params params(
      static_cast<DTypeQ*>(q_nope->data), static_cast<DTypeQ*>(q_pe->data),
      /*q_offset=*/nullptr, paged_kv, static_cast<DTypeO*>(o->data),
      /*lse=*/(maybe_lse.has_value() ? static_cast<float*>(maybe_lse.value()->data) : nullptr),
      num_qo_heads, window_left, logits_soft_cap, sm_scale, rope_scale, rope_theta);

  DTypeO* tmp_v = nullptr;
  float* tmp_s = nullptr;
  params.request_indices =
      GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.request_indices_offset);
  params.kv_tile_indices =
      GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.kv_tile_indices_offset);
  params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.o_indptr_offset);
  params.kv_chunk_size_ptr =
      GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.kv_chunk_size_ptr_offset);
  if (plan_info.split_kv) {
    tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer, plan_info.v_offset);
    tmp_s = GetPtrFromBaseOffset<float>(float_buffer, plan_info.s_offset);
    if (plan_info.enable_cuda_graph) {
      params.block_valid_mask =
          GetPtrFromBaseOffset<bool>(int_buffer, plan_info.block_valid_mask_offset);
    }
  }
  params.padded_batch_size = plan_info.padded_batch_size;

  hipSetDevice(paged_ckv_cache->device.device_id);
  const hipStream_t stream = get_stream(paged_ckv_cache->device);
  hipError_t status = BatchDecodeWithPagedKVCacheDispatchedMlaCuteSM80<HEAD_DIM_CKV, HEAD_DIM_KPE,
                                                                        QO_TILE_LEN, Params>(
      params, tmp_v, tmp_s, /*stream=*/stream);
  TVM_FFI_ICHECK(status == hipSuccess)
      << "BatchDecodeWithPagedKVCache failed with error " << hipGetErrorString(status);
}
