/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/cuda/EmptyTensor.h>
#include <hip/hip_fp16.h>

#include <cstddef>
#include <cstdint>
#include <functional>
#include <type_traits>
#include <vector>

#include "hip/hip_runtime.h"
#include "flashinfer/cutlass_utils.cuh"
#include "flashinfer/gemm/tgv_gemm.cuh"
#include "flashinfer/gemm/tgv_gemm_configs.h"
#include "pytorch_extension_utils.h"

// CUTLASS type includes
#include <cutlass/numeric_types.h>

#define SUPPORTED_TGV_GEMM_CONFIGS \
  TGV_GEMM_CONFIG(64, 8, 6)        \
  TGV_GEMM_CONFIG(64, 8, 8)        \
  TGV_GEMM_CONFIG(64, 8, 10)       \
  TGV_GEMM_CONFIG(64, 8, 12)       \
  TGV_GEMM_CONFIG(64, 16, 6)       \
  TGV_GEMM_CONFIG(64, 16, 8)       \
  TGV_GEMM_CONFIG(64, 16, 10)      \
  TGV_GEMM_CONFIG(64, 32, 6)       \
  TGV_GEMM_CONFIG(64, 32, 8)       \
  TGV_GEMM_CONFIG(64, 64, 6)       \
  TGV_GEMM_CONFIG(128, 16, 6)

#define TGV_GEMM_CONFIG(CTA_M, CTA_N, DMA_STAGE)                                                 \
  if (cta_m == CTA_M && cta_n == CTA_N && dma_stage == DMA_STAGE) {                              \
    *func_ptr = &flashinfer::gemm::tgv_gemm_host<TypeA, TypeB, TypeC, AccType, TypeBias, CTA_M,  \
                                                 CTA_N, 128, DMA_STAGE, UmmaMajorA, UmmaMajorB>; \
    return;                                                                                      \
  }

template <typename TypeA, typename TypeB, typename TypeC, typename AccType, typename TypeBias>
using GemmFuncPtr = void (*)(TypeA*, TypeB*, TypeC*, TypeBias*, int, int, int, int, int, int, int,
                             int, int, int, int, int, int, bool, int, hipStream_t);

template <typename TypeA, typename TypeB, typename TypeC, typename AccType, typename TypeBias,
          cute::UMMA::Major UmmaMajorA, cute::UMMA::Major UmmaMajorB>
void dispatch_kernel(int cta_m, int cta_n, int cta_k, int dma_stage,
                     GemmFuncPtr<TypeA, TypeB, TypeC, AccType, TypeBias>* func_ptr) {
  SUPPORTED_TGV_GEMM_CONFIGS

  TORCH_CHECK(false, "Unsupported tile configuration: cta_m=" + std::to_string(cta_m) +
                         ", cta_n=" + std::to_string(cta_n) + ", cta_k=" + std::to_string(cta_k));
}
#undef TGV_GEMM_CONFIG

namespace torch_ext {

namespace {
// Use the shared function from the header file

using flashinfer::gemm::getAllTgvConfigs;
using flashinfer::gemm::TGVGemmConfig;

TGVGemmConfig getTgvGemmConfig(int64_t tactic) {
  auto globalConfigs = getAllTgvConfigs();

  TORCH_CHECK(tactic >= 0 && tactic < globalConfigs.size(), "tactic must be between 0 and ",
              globalConfigs.size());
  return globalConfigs[tactic];
}

template <typename input_type, typename output_type>
void tgv_gemm_impl(input_type* mat1_ptr, input_type* mat2_ptr, output_type* output_ptr,
                   output_type* bias_ptr, int M, int N, int K, int stride_A_M, int stride_A_K,
                   int stride_A_L, int stride_B_N, int stride_B_K, int stride_B_L, int stride_C_M,
                   int stride_C_N, int stride_C_L, int cta_m, int cta_n, int dma_stage, bool pdl,
                   hipStream_t stream) {
  // Kernel config constants
  using TypeA = input_type;
  using TypeB = input_type;
  using TypeC = output_type;
  using AccType = float;
  using TypeBias = TypeC;
  // only supports K major now
  static constexpr cute::UMMA::Major UmmaMajorA = cute::UMMA::Major::K;
  static constexpr cute::UMMA::Major UmmaMajorB = cute::UMMA::Major::K;
  static constexpr int CTA_K = 128;  // Fixed for now

  // Function pointer for the selected template instantiation
  GemmFuncPtr<TypeA, TypeB, TypeC, AccType, TypeBias> func_ptr = nullptr;

  dispatch_kernel<TypeA, TypeB, TypeC, AccType, TypeBias, UmmaMajorA, UmmaMajorB>(
      cta_m, cta_n, CTA_K, dma_stage, &func_ptr);

  // Call the selected function
  func_ptr(mat1_ptr, mat2_ptr, output_ptr, bias_ptr, M, N, K, 1, stride_A_M, stride_A_K, stride_A_L,
           stride_B_N, stride_B_K, stride_B_L, stride_C_M, stride_C_N, stride_C_L, pdl, -1,
           stream);  // pdl_count=-1 for gemm
}

}  // namespace

at::Tensor tgv_gemm(at::Tensor const& mat1, at::Tensor const& mat2, std::optional<at::Tensor> bias,
                    int64_t tactic, bool pdl) {
  // Input validation
  TORCH_CHECK(mat1.is_cuda(), "mat1 tensor must be on CUDA");
  TORCH_CHECK(mat2.is_cuda(), "mat2 tensor must be on CUDA");
  TORCH_CHECK(mat1.dim() == 2, "mat1 tensor must be 2D (M, K)");
  TORCH_CHECK(mat2.dim() == 2, "mat2 tensor must be 2D (K, N)");
  TORCH_CHECK(mat1.size(1) == mat2.size(0), "mat1.K must match mat2.K");
  TORCH_CHECK(mat1.scalar_type() == mat2.scalar_type(), "mat1 and mat2 must have the same dtype");

  // No heuristic for now, we use 64x8 with 8 DMA stages as the default tactic.
  if (tactic == -1) {
    tactic = 1;
  }
  auto config = getTgvGemmConfig(tactic);

  // Get tile parameters from config
  int cta_m, cta_n, dma_stage;
  config.getTileParams(cta_m, cta_n, dma_stage);

  // Validate DMA_Stage
  TORCH_CHECK(dma_stage == 6 || dma_stage == 8 || dma_stage == 10 || dma_stage == 12,
              "dma_stage must be one of: 6, 8, 10, 12");

  // Validate tile sizes
  TORCH_CHECK(cta_m == 64 || cta_m == 128, "cta_m must be one of: 64, 128");

  // Get dimensions
  int M = mat1.size(0);
  int K = mat1.size(1);
  int N = mat2.size(1);

  // validity check for bias
  if (bias.has_value()) {
    TORCH_CHECK(bias.value().is_cuda(), "Bias tensor must be on CUDA");
    TORCH_CHECK(bias.value().dim() == 1, "Bias tensor must be 1D (M,)");
    TORCH_CHECK(bias.value().size(0) == M, "Bias tensor must have M elements");
    TORCH_CHECK(bias.value().scalar_type() == mat1.scalar_type(),
                "Bias tensor must have the same dtype as input matrices");
    TORCH_CHECK(bias.value().stride(0) == 1, "Bias tensor must be M contiguous");
  }

  // Create output tensor [N, M] row major
  at::Tensor C = at::detail::empty_cuda({N, M}, mat1.scalar_type(), mat1.device(), std::nullopt);

  // manually calculate the L stride
  // A [M, K] row major
  int stride_A_M = mat1.stride(0);
  int stride_A_K = mat1.stride(1);
  int stride_A_L = M * K;
  // B [K, N] column major
  int stride_B_N = mat2.stride(1);
  int stride_B_K = mat2.stride(0);
  int stride_B_L = N * K;
  // original C [N, M] row major
  int stride_C_M = C.stride(1);
  int stride_C_N = C.stride(0);
  int stride_C_L = M * N;

  // Get CUDA stream
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // Dispatch based on dtype
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(mat1.scalar_type(), c_type, [&] {
    using cutlass_input_type = flashinfer::cutlass_dtype_t<c_type>;
    using cutlass_output_type = flashinfer::cutlass_dtype_t<c_type>;

    cutlass_input_type* mat1_ptr = static_cast<cutlass_input_type*>(mat1.data_ptr());
    cutlass_input_type* mat2_ptr = static_cast<cutlass_input_type*>(mat2.data_ptr());
    cutlass_output_type* output_ptr = static_cast<cutlass_output_type*>(C.data_ptr());
    cutlass_output_type* bias_ptr =
        bias.has_value() ? static_cast<cutlass_output_type*>(bias->data_ptr()) : nullptr;

    tgv_gemm_impl<cutlass_input_type, cutlass_output_type>(
        mat1_ptr, mat2_ptr, output_ptr, bias_ptr, M, N, K, stride_A_M, stride_A_K, stride_A_L,
        stride_B_N, stride_B_K, stride_B_L, stride_C_M, stride_C_N, stride_C_L, cta_m, cta_n,
        dma_stage, pdl, stream);
    return true;
  });

  // original C is [N, M] row major
  // after transpose, it's [M, N] column major
  // the storage is unchanged, only the logical coordinates are changed
  return C.t();
}

// Keep backward compatibility functions
at::Tensor bf16_gemm(at::Tensor const& mat1, at::Tensor const& mat2, std::optional<at::Tensor> bias,
                     int64_t tactic, bool pdl) {
  // Check that inputs are bfloat16 for backward compatibility
  TORCH_CHECK(mat1.scalar_type() == at::ScalarType::BFloat16, "mat1 tensor must be bfloat16");
  TORCH_CHECK(mat2.scalar_type() == at::ScalarType::BFloat16, "mat2 tensor must be bfloat16");
  return tgv_gemm(mat1, mat2, bias, tactic, pdl);
}

int64_t tgv_gemm_tactic_num() {
  static int64_t totalTactics = getAllTgvConfigs().size();
  return totalTactics;
}

int64_t bf16_gemm_tactic_num() { return tgv_gemm_tactic_num(); }

}  // namespace torch_ext

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("tgv_gemm", &torch_ext::tgv_gemm);
  m.def("tgv_gemm_tactic_num", &torch_ext::tgv_gemm_tactic_num);
}
