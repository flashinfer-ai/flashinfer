/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/cutlass_mla.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;
using namespace flashinfer::attention;

void CutlassMLAPagedAttention(at::Tensor workspace, at::Tensor out, at::Tensor lse,
                              at::Tensor q_nope_pe, at::Tensor ckv_kpe_cache, at::Tensor kv_lens,
                              at::Tensor page_table) {
  const c10::cuda::OptionalCUDAGuard device_guard(q_nope_pe.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  int device_index = q_nope_pe.device().index();
  int batches = q_nope_pe.sizes()[0];
  int page_count_per_seq = page_table.sizes()[1];
  int page_count_total = ckv_kpe_cache.sizes()[0];
  int page_size = ckv_kpe_cache.sizes()[1];

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(q_nope_pe.scalar_type(), c_type, [&] {
    using cutlass_t = cutlass_dtype_t<c_type>;
    auto status = runMla<cutlass_t>(
        workspace.data_ptr(), out.data_ptr(), lse.data_ptr(), q_nope_pe.data_ptr(),
        ckv_kpe_cache.data_ptr(), kv_lens.data_ptr(), page_table.data_ptr(), batches,
        page_count_per_seq, page_count_total, page_size, device_index, stream);
    TORCH_CHECK(status == hipSuccess,
                "Failed to run CutlassMLAPagedAttention: ", hipGetErrorString(status));
    return true;
  });
}
