#include "hip/hip_runtime.h"
#include <string>

#include "flashinfer/comm/trtllm_moe_allreduce_fusion.cuh"
#include "pytorch_extension_utils.h"

using namespace flashinfer::trtllm_moe_allreduce_fusion;

#define DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE(scalar_type, c_type, ...)                           \
  [&] {                                                                                           \
    switch (scalar_type) {                                                                        \
      case at::ScalarType::Half: {                                                                \
        using c_type = half;                                                                      \
        return __VA_ARGS__();                                                                     \
      }                                                                                           \
      case at::ScalarType::BFloat16: {                                                            \
        using c_type = __hip_bfloat16;                                                             \
        return __VA_ARGS__();                                                                     \
      }                                                                                           \
      default:                                                                                    \
        TORCH_CHECK(false,                                                                        \
                    "Unsupported dtype in DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE: ", scalar_type); \
    }                                                                                             \
  }()

void trtllm_moe_allreduce_fusion(
    int64_t world_size, int64_t world_rank, int64_t token_num, int64_t hidden_size,
    at::Tensor& workspace_ptrs, bool launch_with_pdl, at::Tensor& residual_in,
    at::Tensor& rms_gamma, double rms_eps, double scale_factor,
    int64_t moe_reduction_device_num_experts, at::Tensor& moe_reduction_scale_input,
    at::Tensor& moe_reduction_active_experts_token_input, at::Tensor& moe_reduction_token_input,
    std::optional<int64_t> layout_code, std::optional<at::Tensor> moe_allreduce_out,
    std::optional<at::Tensor> residual_out, std::optional<at::Tensor> norm_out,
    std::optional<at::Tensor> quant_out, std::optional<at::Tensor> scale_out) {
  const c10::cuda::OptionalCUDAGuard device_guard(
      moe_reduction_active_experts_token_input.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE(
      moe_reduction_active_experts_token_input.scalar_type(), c_type, [&] {
        MoeReductionAllReduceFusionParams<c_type> params;
        params.nranks = world_size;
        params.rank = world_rank;
        params.size = token_num * hidden_size;
        params.hidden_dim = hidden_size;
        params.workspace = reinterpret_cast<void**>(workspace_ptrs.data_ptr());

        params.moe_allreduce_out =
            moe_allreduce_out.has_value()
                ? reinterpret_cast<void*>(moe_allreduce_out.value().data_ptr())
                : nullptr;
        params.residual_in = reinterpret_cast<void*>(residual_in.data_ptr());
        params.residual_out = residual_out.has_value()
                                  ? reinterpret_cast<void*>(residual_out.value().data_ptr())
                                  : nullptr;
        params.norm_out =
            norm_out.has_value() ? reinterpret_cast<void*>(norm_out.value().data_ptr()) : nullptr;
        params.quant_out =
            quant_out.has_value() ? reinterpret_cast<void*>(quant_out.value().data_ptr()) : nullptr;
        params.scale_out =
            scale_out.has_value() ? reinterpret_cast<void*>(scale_out.value().data_ptr()) : nullptr;
        params.rms_gamma = reinterpret_cast<void*>(rms_gamma.data_ptr());
        params.rms_eps = static_cast<float>(rms_eps);
        params.scale_factor = static_cast<float>(scale_factor);
        params.layout = layout_code.has_value()
                            ? static_cast<FP4QuantizationSFLayout>(layout_code.value())
                            : FP4QuantizationSFLayout::SWIZZLED;
        params.stream = stream;

        params.moe_reduction_device_num_experts = moe_reduction_device_num_experts;
        params.moe_reduction_scale_input =
            reinterpret_cast<float*>(moe_reduction_scale_input.data_ptr());
        params.moe_reduction_active_experts_token_input =
            reinterpret_cast<void*>(moe_reduction_active_experts_token_input.data_ptr());
        params.moe_reduction_token_input =
            reinterpret_cast<void*>(moe_reduction_token_input.data_ptr());

        auto status = moereduction_allreduce_fusion_op(params, launch_with_pdl);
        TORCH_CHECK(status == hipSuccess,
                    "moereduction_allreduce_fusion_op failed with error code ",
                    hipGetErrorString(status));
      });
}

void trtllm_moe_finalize_allreduce_fusion(
    at::Tensor const& allreduce_in, at::Tensor const& residual_in, at::Tensor const& norm_weight,
    at::Tensor const& expanded_idx_to_permuted_idx, at::Tensor& norm_out, at::Tensor& residual_out,
    bool launch_with_pdl, at::Tensor& workspace, int64_t const world_rank, int64_t const world_size,
    double const eps, std::optional<at::Tensor> const& shared_expert_output,
    std::optional<at::Tensor> const& expert_scale_factor) {
  DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE(residual_in.scalar_type(), c_type, [&] {
    MoeFinalizeAllReduceFusionParams<c_type> params;

    int hidden_dim = residual_in.size(-1);
    int top_k = expanded_idx_to_permuted_idx.size(-1);

    params.quant_out = nullptr;
    params.scale_out = nullptr;

    params.nranks = static_cast<int>(world_size);
    params.rank = static_cast<int>(world_rank);
    // size: num_token * hidden_dim
    params.size = residual_in.numel();
    params.hidden_dim = hidden_dim;

    // workspace: AR scratch space
    params.workspace = reinterpret_cast<void**>(workspace.mutable_data_ptr());
    params.rms_gamma = norm_weight.data_ptr();
    params.rms_eps = static_cast<float>(eps);
    params.residual_in = residual_in.data_ptr();
    params.stream = at::cuda::getCurrentCUDAStream(norm_weight.get_device());

    // MOE Reduction specific params
    params.top_k = top_k;
    params.allreduce_in = allreduce_in.data_ptr();
    params.expert_scale_factor =
        expert_scale_factor.has_value() ? expert_scale_factor.value().data_ptr() : nullptr;
    TORCH_CHECK(expanded_idx_to_permuted_idx.scalar_type() == at::ScalarType::Int,
                "expanded_idx_to_permuted_idx must be int32");
    params.expanded_idx_to_permuted_idx =
        static_cast<int32_t*>(expanded_idx_to_permuted_idx.data_ptr());
    params.shared_expert_output =
        shared_expert_output.has_value() ? shared_expert_output.value().data_ptr() : nullptr;

    // output tensors
    params.norm_out = norm_out.mutable_data_ptr();
    params.residual_out = residual_out.mutable_data_ptr();

    auto status = moefinalize_allreduce_fusion_op(params, launch_with_pdl);
    TORCH_CHECK(status == hipSuccess, "moefinalize_allreduce_fusion_op failed with error code ",
                hipGetErrorString(status));
  });
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_moe_allreduce_fusion", &trtllm_moe_allreduce_fusion);
  m.def("trtllm_moe_finalize_allreduce_fusion", &trtllm_moe_finalize_allreduce_fusion);
}
