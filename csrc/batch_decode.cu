/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/pos_enc.cuh>
#include <flashinfer/utils.cuh>

#include "batch_decode_config.inc"
#include "tvm/ffi/container/array.h"
#include "tvm_ffi_utils.h"

namespace flashinfer {

template <uint32_t HEAD_DIM, PosEncodingMode POS_ENCODING_MODE, typename AttentionVariant,
          typename Params>
hipError_t BatchDecodeWithPagedKVCacheDispatched(Params params, typename Params::DTypeO* tmp_v,
                                                  float* tmp_s, bool enable_pdl,
                                                  hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

using tvm::ffi::Array;
using tvm::ffi::Optional;

Array<int64_t> BatchDecodeWithPagedKVCachePlan(
    Tensor float_workspace_buffer, Tensor int_workspace_buffer,
    Tensor page_locked_int_workspace_buffer, Tensor indptr, int64_t batch_size,
    int64_t num_qo_heads, int64_t num_kv_heads, int64_t page_size, bool enable_cuda_graph,
    int64_t window_left, double logits_soft_cap, int64_t head_dim_qk, int64_t head_dim_vo,
    Tensor empty_q_data, Tensor empty_kv_data) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer->shape[0] * get_element_size(float_workspace_buffer);
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer->shape[0] * get_element_size(int_workspace_buffer);

  DecodePlanInfo plan_info;

  TVM_FFI_ICHECK_EQ(head_dim_qk, head_dim_vo)
      << "CUDA cores template only supports equal head dim for QK and VO, please use tensor "
         "cores template for different head dim";

  hipSetDevice(float_workspace_buffer->device.device_id);
  const hipStream_t stream = get_stream(float_workspace_buffer->device);
  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
      USE_SLIDING_WINDOW, USE_LOGITS_SOFT_CAP, AttentionVariant, Params, [&] {
        DISPATCH_GQA_GROUP_SIZE(num_qo_heads / num_kv_heads, GROUP_SIZE, {
          auto work_estimation_func = BatchDecodeWithPagedKVCacheWorkEstimationDispatched<
              GROUP_SIZE, HEAD_DIM_QK, POS_ENCODING_MODE, AttentionVariant, Params>;
          hipError_t status = DecodePlan<HEAD_DIM_QK, POS_ENCODING_MODE, AttentionVariant, Params>(
              static_cast<void*>(float_workspace_buffer->data), float_workspace_size_in_bytes,
              static_cast<void*>(int_workspace_buffer->data),
              static_cast<void*>(page_locked_int_workspace_buffer->data),
              int_workspace_size_in_bytes, plan_info, static_cast<IdType*>(indptr->data),
              batch_size, num_qo_heads, page_size, enable_cuda_graph,
              /*stream=*/stream, work_estimation_func);

          TVM_FFI_ICHECK(status == hipSuccess)
              << "BatchDecodeWithPagedKVCache failed with error " << hipGetErrorString(status);
          return true;
        });
      });

  return Array(plan_info.ToVector());
}

void BatchDecodeWithPagedKVCacheRun(Tensor float_workspace_buffer, Tensor int_workspace_buffer,
                                    Array<int64_t> plan_info_vec, Tensor q, Tensor paged_k_cache,
                                    Tensor paged_v_cache, Tensor paged_kv_indptr,
                                    Tensor paged_kv_indices, Tensor paged_kv_last_page_len,
                                    Tensor o, Optional<Tensor> maybe_lse, int64_t kv_layout_code,
                                    int64_t window_left, bool enable_pdl ADDITIONAL_FUNC_PARAMS) {
  DecodePlanInfo plan_info;
  plan_info.FromVector(std::vector<int64_t>(plan_info_vec.begin(), plan_info_vec.end()));
  QKVLayout kv_layout = static_cast<QKVLayout>(kv_layout_code);
  int64_t batch_size = q->shape[0];
  int64_t num_qo_heads = q->shape[1];
  int64_t num_kv_heads, page_size;

  if (kv_layout == QKVLayout::kHND) {
    num_kv_heads = paged_k_cache->shape[1];
    page_size = paged_k_cache->shape[2];
  } else {
    page_size = paged_k_cache->shape[1];
    num_kv_heads = paged_k_cache->shape[2];
  }
  uint32_t head_dim_qk = q->shape[2];
  uint32_t head_dim_vo = paged_v_cache->shape[3];

  TVM_FFI_ICHECK_EQ(head_dim_qk, head_dim_vo)
      << "CUDA cores template only supports equal head dim for QK and VO, please use tensor "
         "cores template for different head dim";

  if (maybe_lse.has_value()) {
    const auto& lse = maybe_lse.value();
    TVM_FFI_ICHECK_EQ(lse->shape[0], batch_size);
    TVM_FFI_ICHECK_EQ(lse->shape[1], num_qo_heads);
  }

  void* float_buffer = static_cast<void*>(float_workspace_buffer->data);
  void* int_buffer = static_cast<void*>(int_workspace_buffer->data);

  // get q_stride_n and q_stride_h
  const auto q_stride_n = q->strides[0];
  const auto q_stride_h = q->strides[1];

  // get kv_cache_strides
  const int64_t* kv_cache_strides = nullptr;
  auto k_strides = paged_k_cache.strides();
  auto v_strides = paged_v_cache.strides();
  TVM_FFI_ICHECK_EQ(k_strides.size(), v_strides.size());
  for (int i = 0; i < k_strides.size(); ++i) {
    TVM_FFI_ICHECK_EQ(k_strides[i], v_strides[i]);
  }
  kv_cache_strides = k_strides.data();

  hipSetDevice(q->device.device_id);
  const hipStream_t stream = get_stream(q->device);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
      USE_SLIDING_WINDOW, USE_LOGITS_SOFT_CAP, AttentionVariant, Params, [&] {
        paged_kv_t<DTypeKV, IdType> paged_kv(
            num_kv_heads, page_size, HEAD_DIM_QK, batch_size, kv_layout,
            static_cast<DTypeKV*>(paged_k_cache->data), static_cast<DTypeKV*>(paged_v_cache->data),
            kv_cache_strides, static_cast<IdType*>(paged_kv_indices->data),
            static_cast<IdType*>(paged_kv_indptr->data),
            static_cast<IdType*>(paged_kv_last_page_len->data));

        Params params;
        params.q = static_cast<DTypeQ*>(q->data);
        params.paged_kv = paged_kv;
        params.o = static_cast<DTypeO*>(o->data);
        params.lse = maybe_lse.has_value() ? static_cast<float*>(maybe_lse.value()->data) : nullptr;
        params.padded_batch_size = 0;
        params.num_qo_heads = num_qo_heads;
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.window_left = window_left;
        params.request_indices = nullptr;
        params.kv_tile_indices = nullptr;
        params.o_indptr = nullptr;
        params.kv_chunk_size_ptr = nullptr;
        params.block_valid_mask = nullptr;
        params.partition_kv = false;

        ADDITIONAL_PARAMS_SETTER

        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;
        params.request_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.request_indices_offset);
        params.kv_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.kv_tile_indices_offset);
        params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.o_indptr_offset);
        params.kv_chunk_size_ptr =
            GetPtrFromBaseOffset<IdType>(int_buffer, plan_info.kv_chunk_size_ptr_offset);
        if (plan_info.split_kv) {
          tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer, plan_info.v_offset);
          tmp_s = GetPtrFromBaseOffset<float>(float_buffer, plan_info.s_offset);
          if (plan_info.enable_cuda_graph) {
            params.block_valid_mask =
                GetPtrFromBaseOffset<bool>(int_buffer, plan_info.block_valid_mask_offset);
          }
        }
        params.padded_batch_size = plan_info.padded_batch_size;

        hipError_t status =
            flashinfer::BatchDecodeWithPagedKVCacheDispatched<HEAD_DIM_QK, POS_ENCODING_MODE,
                                                              AttentionVariant>(params, tmp_v,
                                                                                tmp_s, enable_pdl,
                                                                                /*stream=*/stream);
        TVM_FFI_ICHECK(status == hipSuccess)
            << "BatchDecodeWithPagedKVCache failed with error " << hipGetErrorString(status);
        return true;
      });
}
