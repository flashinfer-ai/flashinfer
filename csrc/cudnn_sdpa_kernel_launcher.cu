#include "hip/hip_runtime.h"
/*
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_fp16.h>
#include <flashinfer/exception.h>
#include <flashinfer/trtllm/common.h>
#include <hip/hiprtc.h>

#include <algorithm>
#include <array>
#include <cmath>
#include <iostream>
#include <map>

#include "cudnn_sdpa_utils.h"
#include "pytorch_extension_utils.h"

namespace flashinfer {

namespace cudnn_sdpa_kernel_launcher {

#include <flashinfer/cubin_loader.h>

inline __host__ int clz(int x) {
  for (int i = 31; i >= 0; --i) {
    if ((1 << i) & x) {
      return 31 - i;
    }
  }
  return 32;
}

inline __host__ int find_log_2(int x, bool round_up = false) {
  int a = 31 - clz(x);
  if (round_up) {
    a += (x & (x - 1)) ? 1 : 0;
  }
  return a;
}

inline __host__ void setFastDivisor(cudnn_sdpa::FastDivisor_t& d, uint32_t val) {
  uint32_t p = 31 + find_log_2(2 * val, true);
  uint32_t m = (uint32_t)(((1ull << p) + (uint32_t)(2 * val) - 1) / (uint32_t)(2 * val));

  d.val = val;
  d.mul = m;
  d.shr = p - 32;
}

static std::once_flag init_cudnn_cubin_flag;

constexpr size_t DIMS_QKV = 4;
constexpr int32_t BYTES_PER_ELEMENT = 2;

enum KernelType { PREFILL, PREFILL_DEEPSEEK, DECODE };

enum PrefillType {
  KERNEL_PREFILL,
  KERNEL_PREFILL_DEEPSEEK,
  KERNEL_PREFILL_CAUSAL,
  KERNEL_PREFILL_DEEPSEEK_CAUSAL,
  KERNEL_NUM_PREFILL_TYPES
};

void init_cudnn_cubin(std::map<KernelType, std::string>& cubin_map) {
  cubin_map[PREFILL] = getCubin(
      "4c623163877c8fef5751c9c7a59940cd2baae02e/fmha/cudnn/"
      "cudnn_sm100_fprop_sdpa_prefill_d128_bf16",
      "ff14e8dcfc04d9b3a912dd44056be37d9aa8a85976e0070494ca0cce0524f2a1");

  cubin_map[DECODE] = getCubin(
      "4c623163877c8fef5751c9c7a59940cd2baae02e/fmha/cudnn/cudnn_sm100_fprop_sdpa_decode_d128_bf16",
      "e7ce0408b4c3a36c42616498228534ee64cab785ef570af5741deaf9dd1b475c");

  cubin_map[PREFILL_DEEPSEEK] = getCubin(
      "4c623163877c8fef5751c9c7a59940cd2baae02e/fmha/cudnn/"
      "cudnn_sm100_fprop_sdpa_prefill_d192_bf16",
      "2190967b8733e193cdcecc054eeb7c2907080a158a33fe7ba2004523a4aff6f9");
}

auto get_cudnn_cubin(KernelType kernel_type) -> std::string {
  static std::map<KernelType, std::string> cubin_map;
  std::call_once(init_cudnn_cubin_flag, init_cudnn_cubin, std::ref(cubin_map));
  return cubin_map[kernel_type];
}

__global__ static void __launch_bounds__(128)
    qkv_tma_setup_decode(const unsigned int b, const unsigned int h_qo, const unsigned int h_kv,
                         const unsigned int d, const unsigned int total_num_pages,
                         const unsigned int page_size, const unsigned int split_factor,
                         const unsigned int tile_m_1, const unsigned int tile_n_1,
                         const unsigned int kv_strides_2, const unsigned int kv_strides_1,
                         const unsigned int kv_strides_0, void* q_ptr, const void* k_ptr,
                         const void* v_ptr, void* o_ptr, void* partial_o_ptr,
                         tma::cudaTmaDesc* tma_desc_q_array, tma::cudaTmaDesc* tma_desc_k,
                         tma::cudaTmaDesc* tma_desc_v, tma::cudaTmaDesc* tma_desc_o_array,
                         tma::cudaTmaDesc* tma_desc_partial_o_array, int64_t* batch_strides_dev) {
  const int tid = threadIdx.x;

  constexpr unsigned int DIMS_QKV = 4;
  constexpr unsigned int BYTES_PER_ELEMENT = 2;

  std::array<uint32_t, DIMS_QKV> tensor_traversal_stride_qkv = {1, 1, 1, 1};
  std::array<uint32_t, DIMS_QKV> tensor_box_size_qo = {64, 1, 1, 1};
  std::array<uint32_t, DIMS_QKV> tensor_box_size_kv = {64, std::min(tile_n_1, page_size), 1, 1};
  std::array<uint32_t, DIMS_QKV> tensor_box_size_partial_o = {32, 1, 1, 1};

  std::array<uint32_t, DIMS_QKV> tensor_size_qo = {d, 1 /* s_qo */, h_qo, b};
  std::array<uint32_t, DIMS_QKV> tensor_size_kv = {d, page_size, h_kv, total_num_pages};

  std::array<uint64_t, DIMS_QKV - 1> tensor_stride_qo = {h_qo * d * BYTES_PER_ELEMENT,
                                                         d * BYTES_PER_ELEMENT, 0};
  std::array<uint64_t, DIMS_QKV - 1> tensor_stride_kv = {kv_strides_2 * (BYTES_PER_ELEMENT),
                                                         kv_strides_1 * (BYTES_PER_ELEMENT),
                                                         kv_strides_0 * (BYTES_PER_ELEMENT)};

  std::array<uint32_t, DIMS_QKV> tensor_size_partial_o = {d, split_factor, h_qo, b};
  std::array<uint64_t, DIMS_QKV - 1> tensor_stride_partial_o = {
      h_qo * d * b * sizeof(float), d * b * sizeof(float), d * h_qo * sizeof(float)};

  tma::cudaSetTmaTileDescriptor(
      reinterpret_cast<tma::cudaTmaDesc*>(tma_desc_k), k_ptr, DIMS_QKV, tensor_size_kv.data(),
      tensor_stride_kv.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_kv.data(),
      tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

  tma::cudaSetTmaTileDescriptor(
      reinterpret_cast<tma::cudaTmaDesc*>(tma_desc_v), v_ptr, DIMS_QKV, tensor_size_kv.data(),
      tensor_stride_kv.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_kv.data(),
      tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

  int64_t batch_offset_qo = 0;
  int64_t batch_offset_partial_o = 0;
#pragma unroll 1
  for (int i = 0; i < b; ++i) {
    batch_strides_dev[i] = batch_offset_qo;
    uint16_t* per_batch_q_ptr =
        reinterpret_cast<uint16_t*>(static_cast<std::byte*>(q_ptr) + batch_offset_qo);
    uint16_t* per_batch_out_ptr =
        reinterpret_cast<uint16_t*>(static_cast<std::byte*>(o_ptr) + batch_offset_qo);
    // The two below comes from half
    float* per_batch_partial_o_ptr =
        reinterpret_cast<float*>(static_cast<std::byte*>(partial_o_ptr) + (batch_offset_partial_o));

    tma::cudaTmaDesc desc_q;
    tma::cudaTmaDesc desc_o;
    tma::cudaTmaDesc desc_partial_o;

    tma::cudaSetTmaTileDescriptor(&desc_q, (void*)per_batch_q_ptr, DIMS_QKV, tensor_size_qo.data(),
                                  tensor_stride_qo.data(), tensor_traversal_stride_qkv.data(),
                                  tensor_box_size_qo.data(), tma::cudaTmaDescFormat::BF16_RN,
                                  tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    tma::cudaSetTmaTileDescriptor(
        &desc_o, (void*)per_batch_out_ptr, DIMS_QKV, tensor_size_qo.data(), tensor_stride_qo.data(),
        tensor_traversal_stride_qkv.data(), tensor_box_size_qo.data(),
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    tma::cudaSetTmaTileDescriptor(&desc_partial_o, (void*)per_batch_partial_o_ptr, DIMS_QKV,
                                  tensor_size_partial_o.data(), tensor_stride_partial_o.data(),
                                  tensor_traversal_stride_qkv.data(),
                                  tensor_box_size_partial_o.data(), tma::cudaTmaDescFormat::F32_RN,
                                  tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    reinterpret_cast<tma::cudaTmaDesc*>(tma_desc_q_array)[i] = desc_q;
    reinterpret_cast<tma::cudaTmaDesc*>(tma_desc_o_array)[i] = desc_o;
    reinterpret_cast<tma::cudaTmaDesc*>(tma_desc_partial_o_array)[i] = desc_partial_o;

    batch_offset_qo += d * h_qo * BYTES_PER_ELEMENT;
    batch_offset_partial_o += d * h_qo * sizeof(float);
  }
}

__global__ static void __launch_bounds__(128)
    qkv_tma_setup_prefill(const unsigned int b, const unsigned int h_qo, const unsigned int h_kv,
                          const unsigned int d_qk, const unsigned int d_vo, const bool is_kv_ragged,
                          const unsigned int page_size, const unsigned int total_num_pages,

                          const int64_t k_strides_2, const int64_t k_strides_1,
                          const int64_t k_strides_0, const int64_t v_strides_2,
                          const int64_t v_strides_1, const int64_t v_strides_0,

                          int32_t* actual_seq_lens_q_data, int32_t* actual_seq_lens_kv_data,

                          void* q_ptr, void* k_ptr, void* v_ptr, void* o_ptr,

                          tma::cudaTmaDesc* tma_desc_q_array, tma::cudaTmaDesc* tma_desc_k,
                          tma::cudaTmaDesc* tma_desc_v, tma::cudaTmaDesc* tma_desc_o_array
                          /* const int64_t *batch_offset_array */) {
  const int tid = threadIdx.x;

  constexpr unsigned int DIMS_QKV = 4;
  constexpr unsigned int TILE_M_1 = 128;
  constexpr unsigned int TILE_N_1 = 128;
  constexpr unsigned int BYTES_PER_ELEMENT = 2;
  std::array<uint32_t, DIMS_QKV> tensor_traversal_stride_qkv = {1, 1, 1, 1};

  if (is_kv_ragged) {
    int64_t batch_offset_k = 0;
    int64_t batch_offset_v = 0;
    std::array<uint32_t, DIMS_QKV> tensor_box_size_kv = {64, TILE_N_1, 1, 1};

#pragma unroll 1
    for (int i = 0; i < b; ++i) {
      const uint32_t actual_s_kv = static_cast<uint32_t>(actual_seq_lens_kv_data[i]);
      std::array<uint32_t, DIMS_QKV> packed_tensor_size_k = {d_qk, actual_s_kv, h_kv, 1};
      std::array<uint64_t, DIMS_QKV - 1> packed_tensor_stride_k = {h_kv * d_qk * BYTES_PER_ELEMENT,
                                                                   d_qk * BYTES_PER_ELEMENT, 0};
      std::array<uint32_t, DIMS_QKV> packed_tensor_size_v = {d_vo, actual_s_kv, h_kv, 1};
      std::array<uint64_t, DIMS_QKV - 1> packed_tensor_stride_v = {h_kv * d_vo * BYTES_PER_ELEMENT,
                                                                   d_vo * BYTES_PER_ELEMENT, 0};

      uint16_t* k_batch_ptr =
          reinterpret_cast<uint16_t*>(reinterpret_cast<std::byte*>(k_ptr) + batch_offset_k);
      uint16_t* v_batch_ptr =
          reinterpret_cast<uint16_t*>(reinterpret_cast<std::byte*>(v_ptr) + batch_offset_v);

      tma::cudaSetTmaTileDescriptor(&tma_desc_k[i], (void*)k_batch_ptr, DIMS_QKV,
                                    packed_tensor_size_k.data(), packed_tensor_stride_k.data(),
                                    tensor_traversal_stride_qkv.data(), tensor_box_size_kv.data(),
                                    tma::cudaTmaDescFormat::BF16_RN,
                                    tma::cudaTmaDescSwizzle::SWIZZLE_128B);

      tma::cudaSetTmaTileDescriptor(&tma_desc_v[i], (void*)v_batch_ptr, DIMS_QKV,
                                    packed_tensor_size_v.data(), packed_tensor_stride_v.data(),
                                    tensor_traversal_stride_qkv.data(), tensor_box_size_kv.data(),
                                    tma::cudaTmaDescFormat::BF16_RN,
                                    tma::cudaTmaDescSwizzle::SWIZZLE_128B);

      batch_offset_k += static_cast<int64_t>(actual_s_kv) * d_qk * h_kv *
                        BYTES_PER_ELEMENT;  // Becomes a no-op if batch_offset_array is provided
      batch_offset_v += static_cast<int64_t>(actual_s_kv) * d_vo * h_kv *
                        BYTES_PER_ELEMENT;  // Becomes a no-op if batch_offset_array is provided
    }
  } else {
    bool kv_cache_enabled = d_qk == 192 ? false : true;

    std::array<uint32_t, DIMS_QKV> tensor_size_k = {d_qk, page_size, h_kv, total_num_pages};
    std::array<uint64_t, DIMS_QKV - 1> tensor_stride_k = {k_strides_2 * (BYTES_PER_ELEMENT),
                                                          k_strides_1 * (BYTES_PER_ELEMENT),
                                                          k_strides_0 * (BYTES_PER_ELEMENT)};
    std::array<uint32_t, DIMS_QKV> tensor_size_v = {d_vo, page_size, h_kv, total_num_pages};
    std::array<uint64_t, DIMS_QKV - 1> tensor_stride_v = {v_strides_2 * (BYTES_PER_ELEMENT),
                                                          v_strides_1 * (BYTES_PER_ELEMENT),
                                                          v_strides_0 * (BYTES_PER_ELEMENT)};
    std::array<uint32_t, DIMS_QKV> tensor_box_size_k = {
        64, kv_cache_enabled ? std::min(TILE_N_1, page_size) : TILE_N_1, 1, 1};

    tma::cudaSetTmaTileDescriptor(
        reinterpret_cast<tma::cudaTmaDesc*>(tma_desc_k), k_ptr, DIMS_QKV, tensor_size_k.data(),
        tensor_stride_k.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_k.data(),
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    tma::cudaSetTmaTileDescriptor(
        reinterpret_cast<tma::cudaTmaDesc*>(tma_desc_v), v_ptr, DIMS_QKV, tensor_size_v.data(),
        tensor_stride_v.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_k.data(),
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);
  }

  int64_t batch_offset_q = 0;
  int64_t batch_offset_k = 0;
  int64_t batch_offset_v = 0;
  int64_t batch_offset_o = 0;
  std::array<uint32_t, DIMS_QKV> tensor_box_size_q = {64, TILE_M_1, 1, 1};

#pragma unroll 1
  for (int i = 0; i < b; ++i) {
    const uint32_t actual_s_q = static_cast<uint32_t>(actual_seq_lens_q_data[i]);

    // batch_offset_qo = batch_offset_array ? batch_offset_array[i] : batch_offset_qo;
    std::array<uint32_t, DIMS_QKV> packed_tensor_size_q = {d_qk, actual_s_q, h_qo, 1};
    std::array<uint64_t, DIMS_QKV - 1> packed_tensor_stride_q = {h_qo * d_qk * BYTES_PER_ELEMENT,
                                                                 d_qk * BYTES_PER_ELEMENT, 0};
    std::array<uint32_t, DIMS_QKV> packed_tensor_size_o = {d_vo, actual_s_q, h_qo, 1};
    std::array<uint64_t, DIMS_QKV - 1> packed_tensor_stride_o = {h_qo * d_vo * BYTES_PER_ELEMENT,
                                                                 d_vo * BYTES_PER_ELEMENT, 0};

    uint16_t* per_batch_q_ptr =
        reinterpret_cast<uint16_t*>(reinterpret_cast<std::byte*>(q_ptr) + batch_offset_q);
    uint16_t* per_batch_out_ptr =
        reinterpret_cast<uint16_t*>(reinterpret_cast<std::byte*>(o_ptr) + batch_offset_o);

    tma::cudaTmaDesc desc_q;
    tma::cudaTmaDesc desc_o;

    tma::cudaSetTmaTileDescriptor(
        &desc_q, (void*)per_batch_q_ptr, DIMS_QKV, packed_tensor_size_q.data(),
        packed_tensor_stride_q.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_q.data(),
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    tma::cudaSetTmaTileDescriptor(
        &desc_o, (void*)per_batch_out_ptr, DIMS_QKV, packed_tensor_size_o.data(),
        packed_tensor_stride_o.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_q.data(),
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    reinterpret_cast<tma::cudaTmaDesc*>(tma_desc_q_array)[i] = desc_q;
    reinterpret_cast<tma::cudaTmaDesc*>(tma_desc_o_array)[i] = desc_o;

    batch_offset_q += static_cast<int64_t>(actual_s_q) * d_qk * h_qo *
                      BYTES_PER_ELEMENT;  // Becomes a no-op if batch_offset_array is provided
    batch_offset_o += static_cast<int64_t>(actual_s_q) * d_vo * h_qo *
                      BYTES_PER_ELEMENT;  // Becomes a no-op if batch_offset_array is provided
  }
}

static void create_packed_tma_desc_kv_prefill(int b, int32_t* actual_seq_lens_kv_data, int64_t d_qk,
                                              int64_t d_vo, int64_t h_kv,
                                              uint32_t* tensor_traversal_stride_qkv,
                                              uint32_t* tensor_box_size_kv,
                                              tma::cudaTmaDesc* packed_tma_desc_k,
                                              tma::cudaTmaDesc* packed_tma_desc_v, at::Tensor k,
                                              at::Tensor v) {
  int64_t batch_offset_k = 0;
  int64_t batch_offset_v = 0;
  // tma descriptors for packed q and o
  for (int i = 0; i < b; ++i) {
    const uint32_t actual_s_kv = static_cast<uint32_t>(actual_seq_lens_kv_data[i]);
    std::array<uint32_t, DIMS_QKV> packed_tensor_size_k = {d_qk, actual_s_kv, h_kv, 1};
    std::array<uint64_t, DIMS_QKV - 1> packed_tensor_stride_k = {h_kv * d_qk * BYTES_PER_ELEMENT,
                                                                 d_qk * BYTES_PER_ELEMENT, 0};
    std::array<uint32_t, DIMS_QKV> packed_tensor_size_v = {d_vo, actual_s_kv, h_kv, 1};
    std::array<uint64_t, DIMS_QKV - 1> packed_tensor_stride_v = {h_kv * d_vo * BYTES_PER_ELEMENT,
                                                                 d_vo * BYTES_PER_ELEMENT, 0};

    uint16_t* k_ptr = reinterpret_cast<uint16_t*>(k.data_ptr() + batch_offset_k);
    uint16_t* v_ptr = reinterpret_cast<uint16_t*>(v.data_ptr() + batch_offset_v);

    tma::cudaSetTmaTileDescriptor(
        &packed_tma_desc_k[i], (void*)k_ptr, DIMS_QKV, packed_tensor_size_k.data(),
        packed_tensor_stride_k.data(), tensor_traversal_stride_qkv, tensor_box_size_kv,
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    tma::cudaSetTmaTileDescriptor(
        &packed_tma_desc_v[i], (void*)v_ptr, DIMS_QKV, packed_tensor_size_v.data(),
        packed_tensor_stride_v.data(), tensor_traversal_stride_qkv, tensor_box_size_kv,
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    batch_offset_k += static_cast<int64_t>(actual_s_kv) * d_qk * h_kv *
                      BYTES_PER_ELEMENT;  // Becomes a no-op if batch_offset_array is provided
    batch_offset_v += static_cast<int64_t>(actual_s_kv) * d_vo * h_kv *
                      BYTES_PER_ELEMENT;  // Becomes a no-op if batch_offset_array is provided
  }
}

static void create_packed_tma_desc_qo_prefill(int b, int32_t* actual_seq_lens_q_data, int64_t d_qk,
                                              int64_t d_vo, int64_t h_qo,
                                              uint32_t* tensor_traversal_stride_qkv,
                                              uint32_t* tensor_box_size_q,
                                              tma::cudaTmaDesc* packed_tma_desc_q,
                                              tma::cudaTmaDesc* packed_tma_desc_o, at::Tensor q,
                                              at::Tensor out, int64_t* batch_offset_array) {
  int64_t batch_offset_q = 0;
  int64_t batch_offset_o = 0;
  // tma descriptors for packed q and o
  for (int i = 0; i < b; ++i) {
    const uint32_t actual_s_q = static_cast<uint32_t>(actual_seq_lens_q_data[i]);

    batch_offset_q = batch_offset_array ? batch_offset_array[i] : batch_offset_q;
    batch_offset_o = batch_offset_array ? batch_offset_array[i] : batch_offset_o;
    std::array<uint32_t, DIMS_QKV> packed_tensor_size_q = {d_qk, actual_s_q, h_qo, 1};
    std::array<uint64_t, DIMS_QKV - 1> packed_tensor_stride_q = {h_qo * d_qk * BYTES_PER_ELEMENT,
                                                                 d_qk * BYTES_PER_ELEMENT, 0};
    std::array<uint32_t, DIMS_QKV> packed_tensor_size_o = {d_vo, actual_s_q, h_qo, 1};
    std::array<uint64_t, DIMS_QKV - 1> packed_tensor_stride_o = {h_qo * d_vo * BYTES_PER_ELEMENT,
                                                                 d_vo * BYTES_PER_ELEMENT, 0};

    uint16_t* q_ptr = reinterpret_cast<uint16_t*>(q.data_ptr() + batch_offset_q);
    uint16_t* out_ptr = reinterpret_cast<uint16_t*>(out.data_ptr() + batch_offset_o);

    tma::cudaSetTmaTileDescriptor(
        &packed_tma_desc_q[i], (void*)q_ptr, DIMS_QKV, packed_tensor_size_q.data(),
        packed_tensor_stride_q.data(), tensor_traversal_stride_qkv, tensor_box_size_q,
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    tma::cudaSetTmaTileDescriptor(
        &packed_tma_desc_o[i], (void*)out_ptr, DIMS_QKV, packed_tensor_size_o.data(),
        packed_tensor_stride_o.data(), tensor_traversal_stride_qkv, tensor_box_size_q,
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

    batch_offset_q += static_cast<int64_t>(actual_s_q) * d_qk * h_qo *
                      BYTES_PER_ELEMENT;  // Becomes a no-op if batch_offset_array is provided
    batch_offset_o += static_cast<int64_t>(actual_s_q) * d_vo * h_qo *
                      BYTES_PER_ELEMENT;  // Becomes a no-op if batch_offset_array is provided
  }
}

void setup_prefill(hipFunction_t* prefill_func) {
  // Use cu++filt to get the kernel name
  std::string kernel_name_deepseek_causal =
      "_Z47cudnn_sm100_fprop_sdpa_prefill_bf16_"
      "128x128x192ILb1ELb0EEvN4fmha19AttentionDescriptorEPKN3tma11cudaTmaDescES5_fPfNS0_"
      "7stridesES5_S5_PKjS9_S9_jjNS0_11FastDivisorE";

  std::string kernel_name_causal =
      "_Z47cudnn_sm100_fprop_sdpa_prefill_bf16_"
      "128x128x128ILb1ELb1EEvN4fmha19AttentionDescriptorEPKN3tma11cudaTmaDescES5_fPfNS0_"
      "7stridesES5_S5_PKjS9_S9_jjNS0_11FastDivisorE";

  std::string kernel_name_deepseek =
      "_Z47cudnn_sm100_fprop_sdpa_prefill_bf16_"
      "128x128x192ILb0ELb0EEvN4fmha19AttentionDescriptorEPKN3tma11cudaTmaDescES5_fPfNS0_"
      "7stridesES5_S5_PKjS9_S9_jjNS0_11FastDivisorE";

  std::string kernel_name =
      "_Z47cudnn_sm100_fprop_sdpa_prefill_bf16_"
      "128x128x128ILb0ELb1EEvN4fmha19AttentionDescriptorEPKN3tma11cudaTmaDescES5_fPfNS0_"
      "7stridesES5_S5_PKjS9_S9_jjNS0_11FastDivisorE";

  std::string cubin = get_cudnn_cubin(PREFILL);
  std::string cubin_deepseek = get_cudnn_cubin(PREFILL_DEEPSEEK);

  if (cubin.empty()) {
    throw std::runtime_error("Failed to load cubin for prefill");
  }
  if (cubin_deepseek.empty()) {
    throw std::runtime_error("Failed to load cubin for prefill_deepseek");
  }

  hipModule_t hmod{0};
  hipModule_t hmod_deepseek{0};
  if (hipModuleLoadData(&hmod_deepseek, cubin_deepseek.data()) != hipSuccess) {
    throw std::runtime_error("Failed to hipModuleLoadData for prefill_deepseek");
  }

  if (hipModuleLoadData(&hmod, cubin.data()) != hipSuccess) {
    throw std::runtime_error("Failed to hipModuleLoadData for prefill");
  }

  if (hipModuleGetFunction(&prefill_func[KERNEL_PREFILL], hmod, kernel_name.c_str()) !=
      hipSuccess) {
    throw std::runtime_error("Failed to hipModuleGetFunction for prefill");
  }

  if (hipModuleGetFunction(&prefill_func[KERNEL_PREFILL_DEEPSEEK], hmod_deepseek,
                          kernel_name_deepseek.c_str()) != hipSuccess) {
    throw std::runtime_error("Failed to hipModuleGetFunction for prefill_deepseek");
  }

  if (hipModuleGetFunction(&prefill_func[KERNEL_PREFILL_CAUSAL], hmod, kernel_name_causal.c_str()) !=
      hipSuccess) {
    throw std::runtime_error("Failed to hipModuleGetFunction for prefill");
  }

  if (hipModuleGetFunction(&prefill_func[KERNEL_PREFILL_DEEPSEEK_CAUSAL], hmod_deepseek,
                          kernel_name_deepseek_causal.c_str()) != hipSuccess) {
    throw std::runtime_error("Failed to hipModuleGetFunction for prefill_deepseek");
  }
};

void setup_decode(hipFunction_t* hfunc_decode, hipFunction_t* lean_attn_reduction) {
  constexpr int NUM_DECODE_KERNELS = 5;

  std::string decode_kernel_name[NUM_DECODE_KERNELS] = {
      "_Z44cudnn_sm100_fprop_sdpa_decode_bf16_"
      "Mx128x128ILb1ELi1EEvN4fmha19AttentionDescriptorEPKN3tma11cudaTmaDescES5_ifPfNS0_7stridesES5_"
      "S5_PKjS9_S9_jjNS0_11FastDivisorE",
      "_Z44cudnn_sm100_fprop_sdpa_decode_bf16_"
      "Mx128x128ILb1ELi8EEvN4fmha19AttentionDescriptorEPKN3tma11cudaTmaDescES5_ifPfNS0_7stridesES5_"
      "S5_PKjS9_S9_jjNS0_11FastDivisorE",
      "_Z44cudnn_sm100_fprop_sdpa_decode_bf16_"
      "Mx128x128ILb1ELi16EEvN4fmha19AttentionDescriptorEPKN3tma11cudaTmaDescES5_ifPfNS0_"
      "7stridesES5_S5_PKjS9_S9_jjNS0_11FastDivisorE",
      "_Z44cudnn_sm100_fprop_sdpa_decode_bf16_"
      "Mx128x128ILb1ELi32EEvN4fmha19AttentionDescriptorEPKN3tma11cudaTmaDescES5_ifPfNS0_"
      "7stridesES5_S5_PKjS9_S9_jjNS0_11FastDivisorE",
      "_Z44cudnn_sm100_fprop_sdpa_decode_bf16_"
      "Mx128x128ILb1ELi64EEvN4fmha19AttentionDescriptorEPKN3tma11cudaTmaDescES5_ifPfNS0_"
      "7stridesES5_S5_PKjS9_S9_jjNS0_11FastDivisorE",
  };

  std::string lean_attn_reduction_kernel_name =
      "_Z19lean_attn_reductionN4fmha19AttentionDescriptorEiP13__nv_bfloat16PfS3_S3_NS_7stridesES4_"
      "S4_S4_Pl";

  std::string cubin = get_cudnn_cubin(DECODE);
  if (cubin.empty()) {
    throw std::runtime_error("Failed to load cubin for decode");
  }

  hipModule_t hmod{0};
  if (hipModuleLoadData(&hmod, cubin.data()) != hipSuccess) {
    throw std::runtime_error("Failed to hipModuleLoadData for decode");
  }

  for (int i = 0; i < NUM_DECODE_KERNELS; i++) {
    if (hipModuleGetFunction(&hfunc_decode[i], hmod, decode_kernel_name[i].c_str()) !=
        hipSuccess) {
      throw std::runtime_error("Failed to hipModuleGetFunction for decode at location " +
                               std::to_string(i) + " " + decode_kernel_name[i]);
    }
  }
  if (hipModuleGetFunction(lean_attn_reduction, hmod, lean_attn_reduction_kernel_name.c_str()) !=
      hipSuccess) {
    throw std::runtime_error("Failed to hipModuleGetFunction for lean_attn_reduction decode");
  }
};

void prefill(int64_t b, int64_t s_qo, int64_t max_s_kv, at::Tensor q, at::Tensor k_cache,
             at::Tensor v_cache, double scale, at::Tensor workspace_buffer,
             at::Tensor actual_seq_lens_q, at::Tensor actual_seq_lens_kv,
             at::Tensor actual_seq_lens_q_gpu, at::Tensor actual_seq_lens_kv_gpu,
             at::Tensor block_tables, bool causal, bool return_lse, at::Tensor out, at::Tensor lse,
             std::optional<at::Tensor> batch_offset_q_array,
             std::optional<at::Tensor> batch_offset_o_array,
             std::optional<at::Tensor> batch_offset_k_array,
             std::optional<at::Tensor> batch_offset_v_array, bool is_cuda_graph_compatible) {
  constexpr size_t SMEM_SIZE = 227 * 1024;  // All smem
  constexpr int64_t TILE_M_1 = 128;
  constexpr int64_t TILE_N_1 = 128;

  constexpr int32_t NUM_THREADS = 512;

  auto device = q.device();
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(device.index());

  int64_t* batch_offset_q_array_data = nullptr;
  int64_t* batch_offset_o_array_data = nullptr;
  int64_t* batch_offset_k_array_data = nullptr;
  int64_t* batch_offset_v_array_data = nullptr;
  int64_t* batch_offset_array_data = nullptr;
  if (batch_offset_q_array.has_value()) {
    batch_offset_array_data =
        batch_offset_q_array.value().data_ptr<int64_t>();  // Fix this to make it operational later
  }

  // Step 1: Setup the kernel pointer

  static hipFunction_t prefill_func[KERNEL_NUM_PREFILL_TYPES] = {nullptr, nullptr, nullptr, nullptr};

  int64_t d_qk = q.size(2);

  int64_t d_vo = v_cache.dim() == 3 ? v_cache.size(2) : v_cache.size(3);

  if (prefill_func[0] == nullptr) {
    setup_prefill(prefill_func);

    for (int i = 0; i < KERNEL_NUM_PREFILL_TYPES; i++) {
      if (prefill_func[i] != nullptr) {
        cuErrCheck(cuFuncSetAttribute(prefill_func[i],
                                      HIP_FUNC_ATTRIBUTE_MAX_DYNAMIC_SHARED_SIZE_BYTES, SMEM_SIZE));
        cuErrCheck(cuFuncSetAttribute(prefill_func[i],
                                      HIP_FUNC_ATTRIBUTE_PREFERRED_SHARED_MEMORY_CARVEOUT, 100));
        cuErrCheck(cuFuncSetAttribute(prefill_func[i],
                                      CU_FUNC_ATTRIBUTE_NON_PORTABLE_CLUSTER_SIZE_ALLOWED, 1));
      }
    }
  }

  // Step 2: Extract attention descriptor

  // TORCH_CHECK(k_cache.dim() >= 3, "Input tensor k_cache must have at least 3 dimensions");

  int64_t h_qo = q.size(1);

  int64_t h_kv = k_cache.size(1);

  int64_t page_size = k_cache.dim() == 4 ? k_cache.size(2) : 1;

  int64_t s_kv = max_s_kv;

  int64_t num_pages_per_seq = static_cast<int64_t>(std::ceil(1.0 * s_kv / page_size));

  int64_t total_num_pages = k_cache.dim() == 4 ? k_cache.size(0) : 1;

  bool kv_cache_enabled = d_qk == 192 ? false : true;

  // Step 3: Setup the launch configuration

  HIP_LAUNCH_CONFIG config;

  constexpr int NUM_ATTRS = 1;
  hipLaunchAttribute attrs[NUM_ATTRS];
  config.numAttrs = NUM_ATTRS;
  attrs[0].id = CU_LAUNCH_ATTRIBUTE_CLUSTER_DIMENSION;
  attrs[0].value.clusterDim.x = 1;
  attrs[0].value.clusterDim.y = 1;
  attrs[0].value.clusterDim.z = 1;
  config.attrs = attrs;

  config.sharedMemBytes = SMEM_SIZE;
  config.hStream = stream;

  if (is_cuda_graph_compatible == false) {
    TORCH_CHECK(actual_seq_lens_q.is_cuda() == false,
                "actual_seq_lens_q must be on the same device as q");
    TORCH_CHECK(actual_seq_lens_kv.is_cuda() == false,
                "actual_seq_lens_kv must be on the same device as q");
    auto actual_seq_lens_q_data = actual_seq_lens_q.data_ptr<int32_t>();
    auto actual_seq_lens_kv_data = actual_seq_lens_kv.data_ptr<int32_t>();

    uint32_t actual_num_tiles_per_head = std::transform_reduce(
        actual_seq_lens_q_data, actual_seq_lens_q_data + b, 0U, std::plus<>(), [](int32_t seq_len) {
          return static_cast<uint32_t>(std::ceil(seq_len / (TILE_M_1 * 2.0f)));
        });
    config.gridDimX = actual_num_tiles_per_head;

  } else {
    config.gridDimX = static_cast<int>(std::ceil(s_qo / (TILE_M_1 * 2.0f))) * b;
  }

  config.gridDimY = h_qo;
  config.gridDimZ = 1;

  config.blockDimX = NUM_THREADS;
  config.blockDimY = 1;
  config.blockDimZ = 1;

  // Step 4: Set up the launch arguments

  auto k_strides = k_cache.strides();
  auto v_strides = v_cache.strides();

  bool is_kv_ragged = k_cache.dim() == 3;

  std::array<uint32_t, DIMS_QKV> tensor_traversal_stride_qkv = {1, 1, 1, 1};
  std::array<uint32_t, DIMS_QKV> tensor_size_k = {d_qk, page_size, h_kv, total_num_pages};
  std::array<uint64_t, DIMS_QKV - 1> tensor_stride_k = {k_strides[2] * (BYTES_PER_ELEMENT),
                                                        k_strides[1] * (BYTES_PER_ELEMENT),
                                                        k_strides[0] * (BYTES_PER_ELEMENT)};
  std::array<uint32_t, DIMS_QKV> tensor_size_v = {d_vo, page_size, h_kv, total_num_pages};
  std::array<uint64_t, DIMS_QKV - 1> tensor_stride_v = {v_strides[2] * (BYTES_PER_ELEMENT),
                                                        v_strides[1] * (BYTES_PER_ELEMENT),
                                                        v_strides[0] * (BYTES_PER_ELEMENT)};

  std::array<uint32_t, DIMS_QKV> tensor_box_size_q = {64, TILE_M_1, 1, 1};
  std::array<uint32_t, DIMS_QKV> tensor_box_size_k = {
      64, kv_cache_enabled ? std::min(TILE_N_1, page_size) : TILE_N_1, 1, 1};
  std::array<uint32_t, DIMS_QKV> tensor_box_size_v = {
      64, kv_cache_enabled ? std::min(TILE_N_1, page_size) : TILE_N_1, 1, 1};

  uint64_t batch_offset_qo = 0;
  int8_t* workspace_start = workspace_buffer.data_ptr<int8_t>();

  // These tensors are allocated in the workspace buffer
  // Using 2 * b for q and o
  std::unique_ptr<tma::cudaTmaDesc[]> packed_tma_desc(new tma::cudaTmaDesc[(4 * b)]);
  auto packed_tma_desc_q = packed_tma_desc.get();
  auto packed_tma_desc_o = packed_tma_desc.get() + b;
  auto tma_desc_k_host = packed_tma_desc.get() + (2 * b);
  auto tma_desc_v_host = packed_tma_desc.get() + (3 * b);

  tma::cudaTmaDesc* packed_tma_desc_q_dev = reinterpret_cast<tma::cudaTmaDesc*>(workspace_start);
  tma::cudaTmaDesc* packed_tma_desc_o_dev =
      reinterpret_cast<tma::cudaTmaDesc*>(workspace_start + sizeof(tma::cudaTmaDesc) * b);

  // These TMA descriptors are allocated in the host and passed by value
  tma::cudaTmaDesc* tma_desc_k =
      reinterpret_cast<tma::cudaTmaDesc*>(workspace_start + sizeof(tma::cudaTmaDesc) * (2 * b));
  tma::cudaTmaDesc* tma_desc_v =
      reinterpret_cast<tma::cudaTmaDesc*>(workspace_start + sizeof(tma::cudaTmaDesc) * (3 * b));

  if (is_cuda_graph_compatible == false) {
    if (is_kv_ragged) {
      auto actual_seq_lens_kv_data = actual_seq_lens_kv.data_ptr<int32_t>();
      create_packed_tma_desc_kv_prefill(
          b, actual_seq_lens_kv_data, d_qk, d_vo, h_kv, tensor_traversal_stride_qkv.data(),
          tensor_box_size_k.data(), tma_desc_k_host, tma_desc_v_host, k_cache, v_cache);
    } else {
      // tma descriptors for k and v
      tma::cudaSetTmaTileDescriptor(
          tma_desc_k_host, k_cache.data_ptr(), DIMS_QKV, tensor_size_k.data(),
          tensor_stride_k.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_k.data(),
          tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);

      tma::cudaSetTmaTileDescriptor(
          tma_desc_v_host, v_cache.data_ptr(), DIMS_QKV, tensor_size_v.data(),
          tensor_stride_v.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_v.data(),
          tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);
    }
    auto actual_seq_lens_q_data = actual_seq_lens_q.data_ptr<int32_t>();
    create_packed_tma_desc_qo_prefill(b, actual_seq_lens_q_data, d_qk, d_vo, h_qo,
                                      tensor_traversal_stride_qkv.data(), tensor_box_size_q.data(),
                                      packed_tma_desc_q, packed_tma_desc_o, q, out,
                                      batch_offset_array_data);

    hipMemcpyAsync(workspace_start, packed_tma_desc.get(), sizeof(tma::cudaTmaDesc) * (4 * b),
                    hipMemcpyHostToDevice, stream);
  } else {
    dim3 grid(1, 1, 1);
    dim3 block(128, 1, 1);

    at::cuda::CUDAStream hip_stream = at::cuda::getCurrentCUDAStream(device.index());
    hipStream_t raw_stream = hip_stream.stream();

    hipError_t err = hipStreamQuery(raw_stream);
    if (!(err == hipSuccess || err == hipErrorNotReady)) {
      throw std::runtime_error("CUDA cudnn stream error" + std::string(hipGetErrorString(err)));
    }

    qkv_tma_setup_prefill<<<grid, block, 0, raw_stream>>>(
        b, h_qo, h_kv, d_qk, d_vo, is_kv_ragged, page_size, total_num_pages,
        k_cache.strides().data()[2], k_cache.strides().data()[1], k_cache.strides().data()[0],
        v_cache.strides().data()[2], v_cache.strides().data()[1], v_cache.strides().data()[0],
        actual_seq_lens_q_gpu.data_ptr<int32_t>(), actual_seq_lens_kv_gpu.data_ptr<int32_t>(),
        q.data_ptr(), k_cache.data_ptr(), v_cache.data_ptr(), out.data_ptr(), packed_tma_desc_q_dev,
        tma_desc_k, tma_desc_v, packed_tma_desc_o_dev);
  }

  cudnn_sdpa::AttentionDescriptor_t attn_desc{
      static_cast<uint32_t>(b),    static_cast<uint32_t>(h_qo),        static_cast<uint32_t>(h_kv),
      static_cast<uint32_t>(h_kv), static_cast<uint32_t>(s_qo),        static_cast<uint32_t>(s_kv),
      static_cast<uint32_t>(d_qk), static_cast<uint32_t>(h_qo / h_kv), is_kv_ragged};

  float attn_scale = scale;

  cudnn_sdpa::strides_t lse_strides = {h_qo * s_qo, 1, h_qo, 1};

  cudnn_sdpa::FastDivisor_t page_size_div;
  setFastDivisor(page_size_div, page_size);

  uint32_t page_size32 = static_cast<uint32_t>(page_size);
  uint32_t num_pages_per_seq32 = static_cast<uint32_t>(num_pages_per_seq);

  void* lse_tensor_pointer = return_lse ? lse.data_ptr() : NULL;

  void* actual_seq_lens_q_gpu_pointer = actual_seq_lens_q_gpu.data_ptr<int32_t>();
  void* actual_seq_lens_kv_gpu_pointer = actual_seq_lens_kv_gpu.data_ptr<int32_t>();
  void* block_tables_pointer = d_qk == 192 ? NULL : block_tables.data_ptr<int32_t>();

  auto print_cudaTmaDescTiled = [](tma::cudaTmaDescTiled* desc) {
    printf("addr %p", desc->tensor_common0);
    printf(" common1 %x", desc->tensor_common1);
    printf(" stride %x", (desc->tensor_stride_lower[0] << 4));
    printf(" stride %x", (desc->tensor_stride_lower[1] << 4));
    printf(" stride %x", (desc->tensor_stride_lower[2] << 4));
    printf(" stride %x", (desc->tensor_stride_lower[3] << 4));
    printf(" stride %x", desc->tensor_stride_upper);
    printf(" size0 %x", desc->tensor_size[0]);
    printf(" size1 %x", desc->tensor_size[1]);
    printf(" size2 %x", desc->tensor_size[2]);
    printf(" size3 %x", desc->tensor_size[3]);
    printf(" size4 %x", desc->tensor_size[4]);
    printf(" stride %x", desc->traversal_stride_box_0);
    printf(" box_size_end %d", desc->box_size_end);
    printf("\n");
  };

  // for (int i = 0; i < b; i++) {
  //   print_cudaTmaDescTiled(reinterpret_cast<tma::cudaTmaDescTiled*>(&packed_tma_desc_q[i]));
  //   print_cudaTmaDescTiled(reinterpret_cast<tma::cudaTmaDescTiled*>(&packed_tma_desc_o[i]));
  // }
  // print_cudaTmaDescTiled(reinterpret_cast<tma::cudaTmaDescTiled*>(tma_desc_v_host));

  void* args[14];
  args[0] = (void*)&attn_desc;
  args[1] = (void*)&packed_tma_desc_q_dev;
  args[2] = (void*)&tma_desc_k;
  args[3] = (void*)&attn_scale;
  args[4] = &lse_tensor_pointer;
  args[5] = (void*)&lse_strides;
  args[6] = (void*)&tma_desc_v;
  args[7] = (void*)&packed_tma_desc_o_dev;
  args[8] = &actual_seq_lens_q_gpu_pointer;
  args[9] = &actual_seq_lens_kv_gpu_pointer;
  args[10] = &block_tables_pointer;
  args[11] = &page_size32;
  args[12] = &num_pages_per_seq32;
  args[13] = &page_size_div;

  auto err_launch = hipSuccess;

  auto choice = KERNEL_PREFILL;
  if (causal) {
    choice = d_qk == 192 ? KERNEL_PREFILL_DEEPSEEK_CAUSAL : KERNEL_PREFILL_CAUSAL;
  } else {
    choice = d_qk == 192 ? KERNEL_PREFILL_DEEPSEEK : KERNEL_PREFILL;
  }

  err_launch = hipDrvLaunchKernelEx(&config, prefill_func[choice], (void**)args, nullptr);

  if (err_launch != hipSuccess) {
    const char* errstr = NULL;
    hipDrvGetErrorString(err_launch, &errstr);
    throw std::runtime_error("Failed to hipDrvLaunchKernelEx for prefill");
  }
}

static int32_t compute_split_factor(int32_t b, int32_t h_kv, int32_t h_qo, int32_t s_kv,
                                    uint32_t sm_count) {
  uint32_t split_factor = 1;
  if ((b * h_kv <= (sm_count / 2))) {
    split_factor = std::ceil(1.f * sm_count / (b * h_kv));
    int i = 2;
    for (; i < 128; i *= 2) {
      if (split_factor <= (i + (i / 2) + (i / 4))) {
        split_factor = i;
        break;
      }
    }
    if (i == 128) {
      split_factor = 64;
    }
    if ((h_qo / h_kv) <= 8) {
      while (std::ceil(1.f * s_kv / split_factor) < (h_qo / h_kv)) {
        split_factor /= 2;
      }
      if (s_kv <= 512) {
        split_factor = 1;
      }
    } else {
      if (s_kv <= 1024) {
        split_factor = 1;
      }
    }
    if (split_factor == 0) {
      split_factor = 1;
    }
  }
  return split_factor;
}

int32_t get_kernel_id(int32_t q_heads_per_kv) {
  auto kernel_id = 0;
  if (q_heads_per_kv == 1) {
    kernel_id = 0;
  } else if (q_heads_per_kv <= 8) {
    kernel_id = 1;
  } else if (q_heads_per_kv <= 16) {
    kernel_id = 2;
  } else if (q_heads_per_kv <= 32) {
    kernel_id = 3;
  } else {
    kernel_id = 4;
  }
  return kernel_id;
}

void setup_tma_desc_decode(int64_t b, int64_t s_kv, int64_t h_qo, int64_t h_kv, int64_t d,
                           int64_t total_num_pages, at::Tensor q, at::Tensor out,
                           at::Tensor k_cache, at::Tensor v_cache, int32_t split_factor,
                           int64_t page_size, int8_t* partial_o_dev, tma::cudaTmaDesc* tma_desc_q,
                           tma::cudaTmaDesc* tma_desc_o, tma::cudaTmaDesc* tma_desc_partial_o,
                           tma::cudaTmaDesc* tma_desc_k, tma::cudaTmaDesc* tma_desc_v) {
  auto kid = get_kernel_id(h_qo / h_kv);
  int64_t TILE_M_1 = 1;
  int64_t TILE_N_1 = 128;
  switch (kid) {
    case 0:
      TILE_M_1 = 1;
      break;
    case 1:
      TILE_M_1 = 8;
      break;
    case 2:
      TILE_M_1 = 16;
      break;
    case 3:
      TILE_M_1 = 32;
      break;
    case 4:
      TILE_M_1 = 64;
      break;
  }

  constexpr int64_t DIMS_QKV = 4;

  std::array<uint32_t, DIMS_QKV> tensor_traversal_stride_qkv = {1, 1, 1, 1};
  std::array<uint32_t, DIMS_QKV> tensor_box_size_qo = {64, 1, 1, 1};
  std::array<uint32_t, DIMS_QKV> tensor_box_size_kv = {64, std::min(TILE_N_1, page_size), 1, 1};
  std::array<uint32_t, DIMS_QKV> tensor_box_size_partial_o = {32, 1, 1, 1};

  std::array<uint32_t, DIMS_QKV> tensor_size_qo = {d, 1 /* s_qo */, h_qo, b};
  std::array<uint32_t, DIMS_QKV> tensor_size_kv = {d, page_size, h_kv, total_num_pages};

  auto kv_strides = k_cache.strides();

  std::array<uint64_t, DIMS_QKV - 1> tensor_stride_qo = {h_qo * d * BYTES_PER_ELEMENT,
                                                         d * BYTES_PER_ELEMENT, 0};
  std::array<uint64_t, DIMS_QKV - 1> tensor_stride_kv = {kv_strides[2] * (BYTES_PER_ELEMENT),
                                                         kv_strides[1] * (BYTES_PER_ELEMENT),
                                                         kv_strides[0] * (BYTES_PER_ELEMENT)};

  std::array<uint32_t, DIMS_QKV> tensor_size_partial_o = {d, split_factor, h_qo, b};
  std::array<uint64_t, DIMS_QKV - 1> tensor_stride_partial_o = {
      h_qo * d * b * sizeof(float), d * b * sizeof(float), d * h_qo * sizeof(float)};
  uint16_t* q_ptr = reinterpret_cast<uint16_t*>(q.data_ptr());
  uint16_t* out_ptr = reinterpret_cast<uint16_t*>(out.data_ptr());
  float* partial_o_ptr = reinterpret_cast<float*>(partial_o_dev);

  int64_t batch_offset_qo = 0;

  for (int64_t i = 0; i < b; i++) {
    tma::cudaSetTmaTileDescriptor(
        &tma_desc_q[i], q_ptr + batch_offset_qo, DIMS_QKV, tensor_size_qo.data(),
        tensor_stride_qo.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_qo.data(),
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);
    tma::cudaSetTmaTileDescriptor(
        &tma_desc_o[i], out_ptr + batch_offset_qo, DIMS_QKV, tensor_size_qo.data(),
        tensor_stride_qo.data(), tensor_traversal_stride_qkv.data(), tensor_box_size_qo.data(),
        tma::cudaTmaDescFormat::BF16_RN, tma::cudaTmaDescSwizzle::SWIZZLE_128B);
    tma::cudaSetTmaTileDescriptor(&tma_desc_partial_o[i], partial_o_ptr + batch_offset_qo, DIMS_QKV,
                                  tensor_size_partial_o.data(), tensor_stride_partial_o.data(),
                                  tensor_traversal_stride_qkv.data(),
                                  tensor_box_size_partial_o.data(), tma::cudaTmaDescFormat::F32_RN,
                                  tma::cudaTmaDescSwizzle::SWIZZLE_128B);
    batch_offset_qo += h_qo * d;
  }

  tma::cudaSetTmaTileDescriptor(tma_desc_k, k_cache.data_ptr(), DIMS_QKV, tensor_size_kv.data(),
                                tensor_stride_kv.data(), tensor_traversal_stride_qkv.data(),
                                tensor_box_size_kv.data(), tma::cudaTmaDescFormat::BF16_RN,
                                tma::cudaTmaDescSwizzle::SWIZZLE_128B);

  tma::cudaSetTmaTileDescriptor(tma_desc_v, v_cache.data_ptr(), DIMS_QKV, tensor_size_kv.data(),
                                tensor_stride_kv.data(), tensor_traversal_stride_qkv.data(),
                                tensor_box_size_kv.data(), tma::cudaTmaDescFormat::BF16_RN,
                                tma::cudaTmaDescSwizzle::SWIZZLE_128B);
}

void decode(int64_t max_s_kv, at::Tensor q, at::Tensor k_cache, at::Tensor v_cache, double scale,
            at::Tensor workspace_buffer, at::Tensor actual_seq_lens_kv,
            at::Tensor actual_seq_lens_kv_gpu, at::Tensor block_tables, at::Tensor out,
            std::optional<at::Tensor> batch_offset_q_array,
            std::optional<at::Tensor> batch_offset_o_array, bool is_cuda_graph_compatible) {
  constexpr size_t SMEM_SIZE = 227 * 1024;  // All smem
  constexpr size_t REDUCTION_MEM_SIZE = 128 * 1024;
  constexpr int64_t TILE_N_1 = 128;

  constexpr int32_t NUM_THREADS = 384;

  int64_t* batch_offset_q_array_data = nullptr;
  if (batch_offset_q_array.has_value()) {
    batch_offset_q_array_data = batch_offset_q_array.value().data_ptr<int64_t>();
  }

  auto device = q.device();

  const hipStream_t stream = at::cuda::getCurrentCUDAStream(device.index());

  constexpr int NUM_DECODE_KERNELS = 5;
  static hipFunction_t hfunc_decode[NUM_DECODE_KERNELS] = {nullptr, nullptr, nullptr, nullptr,
                                                        nullptr};
  static hipFunction_t lean_attn_reduction{nullptr};

  static uint32_t sm_count = 0;

  // Setup decode kernels
  if (hfunc_decode[0] == nullptr) {
    setup_decode(hfunc_decode, &lean_attn_reduction);

    for (int i = 0; i < NUM_DECODE_KERNELS; i++) {
      if (hfunc_decode[i] != nullptr) {
        cuErrCheck(cuFuncSetAttribute(hfunc_decode[i],
                                      HIP_FUNC_ATTRIBUTE_MAX_DYNAMIC_SHARED_SIZE_BYTES, SMEM_SIZE));
        cuErrCheck(cuFuncSetAttribute(hfunc_decode[i],
                                      HIP_FUNC_ATTRIBUTE_PREFERRED_SHARED_MEMORY_CARVEOUT, 100));
        cuErrCheck(cuFuncSetAttribute(hfunc_decode[i],
                                      CU_FUNC_ATTRIBUTE_NON_PORTABLE_CLUSTER_SIZE_ALLOWED, 1));
      }
    }
    if (lean_attn_reduction != nullptr) {
      cuErrCheck(cuFuncSetAttribute(lean_attn_reduction,
                                    HIP_FUNC_ATTRIBUTE_MAX_DYNAMIC_SHARED_SIZE_BYTES,
                                    REDUCTION_MEM_SIZE));
      cuErrCheck(cuFuncSetAttribute(lean_attn_reduction,
                                    HIP_FUNC_ATTRIBUTE_PREFERRED_SHARED_MEMORY_CARVEOUT, 100));
      cuErrCheck(cuFuncSetAttribute(lean_attn_reduction,
                                    CU_FUNC_ATTRIBUTE_NON_PORTABLE_CLUSTER_SIZE_ALLOWED, 1));
    }

    // Get number of SMs perf GPU
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    sm_count = prop.multiProcessorCount;
  }

  int64_t b = q.size(0);
  int64_t h_qo = q.size(1);
  int64_t d = q.size(2);

  int64_t h_kv = k_cache.size(1);

  int64_t page_size = k_cache.dim() == 4 ? k_cache.size(2) : 1;

  int64_t total_num_pages = k_cache.dim() == 4 ? k_cache.size(0) : 1;

  int64_t s_kv = max_s_kv;

  int64_t s_qo = 1;

  int32_t split_factor = compute_split_factor(b, h_kv, h_qo, s_kv, sm_count);

  split_factor = 1;  // Fix split factor. Setting it to 1 for now

  // Set up TMA descriptors for Q, K, V, O
  auto qo_strides = q.strides();
  auto kv_strides = v_cache.strides();

  // Launch config for main kernel
  HIP_LAUNCH_CONFIG config;
  hipLaunchAttribute attrs[1];
  attrs[0].id = CU_LAUNCH_ATTRIBUTE_CLUSTER_DIMENSION;
  attrs[0].value.clusterDim.x = 1;
  attrs[0].value.clusterDim.y = 1;
  attrs[0].value.clusterDim.z = 1;

  const unsigned int CTAs_y = h_kv * std::ceil(1.f * (h_qo / h_kv) / 64);

  config.gridDimX = split_factor;  // Number of CTAs per row
  config.gridDimY = CTAs_y;
  config.gridDimZ = b;

  config.blockDimX = NUM_THREADS;
  config.blockDimY = 1;
  config.blockDimZ = 1;

  config.attrs = attrs;
  config.sharedMemBytes = SMEM_SIZE;

  config.hStream = stream;
  config.numAttrs = 1;

  int8_t* workspace_start = workspace_buffer.data_ptr<int8_t>();
  int8_t* partial_o_dev = workspace_start;
  int8_t* tma_descriptor_start =
      partial_o_dev + (b * s_qo * h_qo * d * sizeof(float) * split_factor);

  int8_t* batch_strides_dev = tma_descriptor_start + ((5 * b) * sizeof(tma::cudaTmaDesc));

  tma::cudaTmaDesc* packed_tma_desc_q_dev =
      reinterpret_cast<tma::cudaTmaDesc*>(tma_descriptor_start);
  tma::cudaTmaDesc* packed_tma_desc_o_dev =
      reinterpret_cast<tma::cudaTmaDesc*>(tma_descriptor_start + b * sizeof(tma::cudaTmaDesc));
  tma::cudaTmaDesc* packed_tma_desc_partial_o_dev =
      reinterpret_cast<tma::cudaTmaDesc*>(tma_descriptor_start + b * sizeof(tma::cudaTmaDesc) * 2);
  tma::cudaTmaDesc* tma_desc_k_dev =
      reinterpret_cast<tma::cudaTmaDesc*>(tma_descriptor_start + b * sizeof(tma::cudaTmaDesc) * 3);
  tma::cudaTmaDesc* tma_desc_v_dev =
      reinterpret_cast<tma::cudaTmaDesc*>(tma_descriptor_start + b * sizeof(tma::cudaTmaDesc) * 4);

  int8_t* lse_dev = batch_strides_dev + (b * sizeof(int64_t));

  if (is_cuda_graph_compatible) {
    dim3 grid(1, 1, 1);
    dim3 block(128, 1, 1);
    auto kid = get_kernel_id(h_qo / h_kv);
    int64_t TILE_M_1 = 1;
    switch (kid) {
      case 0:
        TILE_M_1 = 1;
        break;
      case 1:
        TILE_M_1 = 8;
        break;
      case 2:
        TILE_M_1 = 16;
        break;
      case 3:
        TILE_M_1 = 32;
        break;
      case 4:
        TILE_M_1 = 64;
        break;
    }

    qkv_tma_setup_decode<<<grid, block, 0, stream>>>(
        b, h_qo, h_kv, d, total_num_pages, page_size, split_factor, TILE_M_1, TILE_N_1,
        kv_strides[2], kv_strides[1], kv_strides[0], q.data_ptr(), k_cache.data_ptr(),
        v_cache.data_ptr(), out.data_ptr(), partial_o_dev, packed_tma_desc_q_dev, tma_desc_k_dev,
        tma_desc_v_dev, packed_tma_desc_o_dev, packed_tma_desc_partial_o_dev,
        reinterpret_cast<int64_t*>(batch_strides_dev));
  } else {
    std::unique_ptr<tma::cudaTmaDesc[]> tma_desc_host(new tma::cudaTmaDesc[5 * b]);

    tma::cudaTmaDesc* tma_desc_q = tma_desc_host.get();
    tma::cudaTmaDesc* tma_desc_o = tma_desc_host.get() + b;
    tma::cudaTmaDesc* tma_desc_partial_o = tma_desc_host.get() + b * 2;
    tma::cudaTmaDesc* tma_desc_k = tma_desc_host.get() + b * 3;
    tma::cudaTmaDesc* tma_desc_v = tma_desc_host.get() + b * 4;

    setup_tma_desc_decode(b, max_s_kv, h_qo, h_kv, d, total_num_pages, q, out, k_cache, v_cache,
                          split_factor, page_size, partial_o_dev, tma_desc_q, tma_desc_o,
                          tma_desc_partial_o, tma_desc_k, tma_desc_v);

    std::unique_ptr<int64_t[]> batch_strides(new int64_t[b]);
    for (int i = 0; i < b; i++) {
      batch_strides[i] = (i)*d * h_qo;
    }
    hipMemcpyAsync(batch_strides_dev, batch_strides.get(), sizeof(int64_t) * b,
                    hipMemcpyHostToDevice, stream);

    hipMemcpyAsync(tma_descriptor_start, tma_desc_host.get(), sizeof(tma::cudaTmaDesc) * (5 * b),
                    hipMemcpyHostToDevice, stream);
  }

  cudnn_sdpa::AttentionDescriptor_t attnDesc{b,        h_qo, h_kv,        h_kv, s_qo,
                                             max_s_kv, d,    h_qo / h_kv, 0};

  cudnn_sdpa::FastDivisor_t page_size_div;
  setFastDivisor(page_size_div, page_size);

  uint32_t page_size32 = static_cast<uint32_t>(page_size);
  uint32_t num_pages_per_seq32 = static_cast<uint32_t>(max_s_kv / page_size);

  void* args[15];

  float attn_scale = scale;
  void* actual_seq_lens_q_gpu_pointer = nullptr;
  void* actual_seq_lens_kv_gpu_pointer = actual_seq_lens_kv_gpu.data_ptr<int32_t>();
  void* block_tables_pointer = block_tables.data_ptr<int32_t>();

  cudnn_sdpa::strides_t lse_strides = {h_qo, 1, h_qo, 1};
  cudnn_sdpa::strides_t partial_lse_strides = {h_qo, 1, h_qo * b, 1};

  cudnn_sdpa::strides_t partial_o_strides = {split_factor * h_qo * d, h_qo * d, d, 1};

  args[0] = (void*)&attnDesc;
  args[1] = (void*)&packed_tma_desc_q_dev;
  args[2] = (void*)&tma_desc_k_dev;
  args[3] = (void*)&split_factor;
  args[4] = (void*)&attn_scale;
  args[5] = (void*)&lse_dev;
  args[6] = split_factor == 1 ? (void*)&lse_strides : (void*)&partial_lse_strides;
  args[7] = (void*)&tma_desc_v_dev;
  args[8] =
      split_factor == 1 ? (void*)&packed_tma_desc_o_dev : (void*)&packed_tma_desc_partial_o_dev;
  args[9] = (void*)&actual_seq_lens_q_gpu_pointer;
  args[10] = (void*)&actual_seq_lens_kv_gpu_pointer;
  args[11] = (void*)&block_tables_pointer;
  args[12] = (void*)&page_size32;
  args[13] = (void*)&num_pages_per_seq32;
  args[14] = (void*)&page_size_div;

  auto kernel_id = get_kernel_id(attnDesc.q_heads_per_kv);

  auto err_launch = hipDrvLaunchKernelEx(&config, hfunc_decode[kernel_id], (void**)args, nullptr);
  if (err_launch != hipSuccess) {
    std::cerr << "hipDrvLaunchKernelEx failed with error code " << err_launch << std::endl;
    throw std::runtime_error("hipDrvLaunchKernelEx failed for decode");
  }

  // Now setting up the reduction kernel
  if (split_factor > 1) {
    // TODO: Add support for split_factor > 1
    void* args_lean_attn_reduction[11];
    void* o_dev = out.data_ptr();

    void* lse_final_dev = nullptr;

    cudnn_sdpa::strides_t o_strides = {h_qo * d, d, 1};

    args_lean_attn_reduction[0] = (void*)&attnDesc;
    args_lean_attn_reduction[1] = (void*)&split_factor;
    args_lean_attn_reduction[2] = (void*)&o_dev;
    args_lean_attn_reduction[3] = (void*)&partial_o_dev;
    args_lean_attn_reduction[4] = (void*)&lse_final_dev;
    args_lean_attn_reduction[5] = (void*)&lse_dev;
    args_lean_attn_reduction[6] = (void*)&o_strides;
    args_lean_attn_reduction[7] = (void*)&partial_o_strides;
    args_lean_attn_reduction[8] = (void*)&lse_strides;
    args_lean_attn_reduction[9] = (void*)&partial_lse_strides;
    args_lean_attn_reduction[10] = (void*)&batch_strides_dev;

    // Launch config for reduction kernel

    HIP_LAUNCH_CONFIG reduction_config;

    reduction_config.gridDimX = h_qo;
    reduction_config.gridDimY = b;  // Same as CTAs_z of main kernel
    reduction_config.gridDimZ = 1;

    reduction_config.blockDimX = 128;  // 128 threads per block
    reduction_config.blockDimY = 1;
    reduction_config.blockDimZ = 1;

    reduction_config.sharedMemBytes = REDUCTION_MEM_SIZE;

    hipLaunchAttribute reduction_attrs[1];
    reduction_attrs[0].id = CU_LAUNCH_ATTRIBUTE_CLUSTER_DIMENSION;
    reduction_attrs[0].value.clusterDim.x = 1;
    reduction_attrs[0].value.clusterDim.y = 1;
    reduction_attrs[0].value.clusterDim.z = 1;

    reduction_config.hStream = stream;
    reduction_config.numAttrs = 1;
    reduction_config.attrs = reduction_attrs;

    auto err_launch = hipDrvLaunchKernelEx(&reduction_config, lean_attn_reduction,
                                       (void**)args_lean_attn_reduction, nullptr);
    if (err_launch != hipSuccess) {
      std::cerr << "hipDrvLaunchKernelEx failed with error code " << err_launch << std::endl;
      throw std::runtime_error("hipDrvLaunchKernelEx failed for decode");
    }
  }
}

}  // namespace cudnn_sdpa_kernel_launcher

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("prefill", cudnn_sdpa_kernel_launcher::prefill);
  m.def("decode", cudnn_sdpa_kernel_launcher::decode);
}

}  // namespace flashinfer
