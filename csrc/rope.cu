#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/pos_enc.cuh>

#include "tvm_ffi_utils.h"

using namespace flashinfer;

using tvm::ffi::Tensor;

void apply_rope(TensorView q, TensorView k, TensorView q_rope, TensorView k_rope, TensorView indptr,
                TensorView offsets, int64_t rotary_dim, bool interleave, double rope_scale,
                double rope_theta) {
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(q);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(k);
  CHECK_INPUT(indptr);
  CHECK_INPUT(offsets);

  CHECK_DEVICE(q, k);
  CHECK_DIM(3, q);        // q: (nnz, H_Q, D)
  CHECK_DIM(3, k);        // k: (nnz, H_K, D)
  CHECK_DIM(1, indptr);   // indptr: (B + 1)
  CHECK_DIM(1, offsets);  // offsets: (B)
  TVM_FFI_ICHECK_EQ(q->shape[0], k->shape[0]);
  TVM_FFI_ICHECK_EQ(q->shape[2], k->shape[2]);
  unsigned int num_qo_heads = q->shape[1];
  unsigned int num_kv_heads = k->shape[1];
  unsigned int head_dim = q->shape[2];
  unsigned int batch_size = offsets->shape[0];
  TVM_FFI_ICHECK_EQ(indptr->shape[0], batch_size + 1);
  size_t q_stride_n = q->strides[0];
  size_t q_stride_h = q->strides[1];
  size_t k_stride_n = k->strides[0];
  size_t k_stride_h = k->strides[1];
  size_t q_rope_stride_n = q_rope->strides[0];
  size_t q_rope_stride_h = q_rope->strides[1];
  size_t k_rope_stride_n = k_rope->strides[0];
  size_t k_rope_stride_h = k_rope->strides[1];
  TVM_FFI_ICHECK_EQ(indptr->dtype, offsets->dtype);

  hipSetDevice(q->device.device_id);
  const hipStream_t stream = get_stream(q->device);
  DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(q->dtype, c_type, [&] {
    return DISPATCH_DLPACK_IDTYPE_TO_CTYPE(indptr->dtype, c_idtype, [&] {
      hipError_t status = BatchQKApplyRotary(
          static_cast<c_type*>(q->data), static_cast<c_type*>(k->data),
          static_cast<c_type*>(q_rope->data), static_cast<c_type*>(k_rope->data),
          static_cast<c_idtype*>(indptr->data), static_cast<c_idtype*>(offsets->data), batch_size,
          num_qo_heads, num_kv_heads, rotary_dim, head_dim, q_stride_n, q_stride_h, k_stride_n,
          k_stride_h, q_rope_stride_n, q_rope_stride_h, k_rope_stride_n, k_rope_stride_h,
          interleave, rope_scale, rope_theta, stream);
      TVM_FFI_ICHECK(status == hipSuccess)
          << "BatchQKApplyRotary failed with error code " << hipGetErrorString(status);
      return true;
    });
  });
}

void apply_rope_pos_ids(TensorView q, TensorView k, TensorView q_rope, TensorView k_rope,
                        TensorView pos_ids, int64_t rotary_dim, bool interleave, double rope_scale,
                        double rope_theta) {
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(q);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(k);
  CHECK_INPUT(pos_ids);

  CHECK_DEVICE(q, k);
  CHECK_DIM(3, q);  // q: (nnz, H_Q, D)
  CHECK_DIM(3, k);  // k: (nnz, H_K, D)
  TVM_FFI_ICHECK_EQ(q->shape[0], k->shape[0]);
  TVM_FFI_ICHECK_EQ(q->shape[2], k->shape[2]);
  unsigned int num_qo_heads = q->shape[1];
  unsigned int num_kv_heads = k->shape[1];
  unsigned int head_dim = q->shape[2];
  unsigned int nnz = q->shape[0];
  size_t q_stride_n = q->strides[0];
  size_t q_stride_h = q->strides[1];
  size_t k_stride_n = k->strides[0];
  size_t k_stride_h = k->strides[1];
  size_t q_rope_stride_n = q_rope->strides[0];
  size_t q_rope_stride_h = q_rope->strides[1];
  size_t k_rope_stride_n = k_rope->strides[0];
  size_t k_rope_stride_h = k_rope->strides[1];

  hipSetDevice(q->device.device_id);
  const hipStream_t stream = get_stream(q->device);
  DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(q->dtype, c_type, [&] {
    return DISPATCH_DLPACK_IDTYPE_TO_CTYPE(pos_ids->dtype, c_idtype, [&] {
      hipError_t status = BatchQKApplyRotaryPosIds(
          static_cast<c_type*>(q->data), static_cast<c_type*>(k->data),
          static_cast<c_type*>(q_rope->data), static_cast<c_type*>(k_rope->data),
          static_cast<c_idtype*>(pos_ids->data), nnz, num_qo_heads, num_kv_heads, rotary_dim,
          head_dim, q_stride_n, q_stride_h, k_stride_n, k_stride_h, q_rope_stride_n,
          q_rope_stride_h, k_rope_stride_n, k_rope_stride_h, interleave, rope_scale, rope_theta,
          stream);

      TVM_FFI_ICHECK(status == hipSuccess)
          << "BatchQKApplyRotaryPosIds failed with error code " << hipGetErrorString(status);
      return true;
    });
  });
}

void apply_rope_pos_ids_cos_sin_cache(TensorView q, TensorView k, TensorView q_rope,
                                      TensorView k_rope, TensorView cos_sin_cache,
                                      TensorView pos_ids, bool interleave) {
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(q);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(k);
  CHECK_INPUT(cos_sin_cache);
  CHECK_INPUT(pos_ids);
  CHECK_DEVICE(q, k);
  CHECK_DEVICE(q, cos_sin_cache);
  CHECK_DEVICE(q, pos_ids);
  CHECK_DIM(3, q);  // q: (nnz, H_Q, D)
  CHECK_DIM(3, k);  // k: (nnz, H_K, D)
  // cos_sin_cache: (max_seq_len, R)
  // First half of R is cos, second half is sin
  CHECK_DIM(2, cos_sin_cache);
  TVM_FFI_ICHECK_EQ(q->shape[0], k->shape[0]);
  TVM_FFI_ICHECK_EQ(q->shape[2], k->shape[2]);
  unsigned int rotary_dim = cos_sin_cache->shape[1];
  unsigned int num_qo_heads = q->shape[1];
  unsigned int num_kv_heads = k->shape[1];
  unsigned int head_dim = q->shape[2];
  unsigned int nnz = q->shape[0];
  size_t q_stride_n = q->strides[0];
  size_t q_stride_h = q->strides[1];
  size_t k_stride_n = k->strides[0];
  size_t k_stride_h = k->strides[1];
  size_t q_rope_stride_n = q_rope->strides[0];
  size_t q_rope_stride_h = q_rope->strides[1];
  size_t k_rope_stride_n = k_rope->strides[0];
  size_t k_rope_stride_h = k_rope->strides[1];

  hipSetDevice(q->device.device_id);
  const hipStream_t stream = get_stream(q->device);
  DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(q->dtype, c_type, [&] {
    return DISPATCH_DLPACK_IDTYPE_TO_CTYPE(pos_ids->dtype, c_idtype, [&] {
      hipError_t status = BatchQKApplyRotaryPosIdsCosSinCache(
          static_cast<c_type*>(q->data), static_cast<c_type*>(k->data),
          static_cast<c_type*>(q_rope->data), static_cast<c_type*>(k_rope->data),
          static_cast<float*>(cos_sin_cache->data), static_cast<c_idtype*>(pos_ids->data), nnz,
          num_qo_heads, num_kv_heads, rotary_dim, head_dim, q_stride_n, q_stride_h, k_stride_n,
          k_stride_h, q_rope_stride_n, q_rope_stride_h, k_rope_stride_n, k_rope_stride_h,
          interleave, stream);

      TVM_FFI_ICHECK(status == hipSuccess)
          << "BatchQKApplyRotaryPosIdsCosSinCache failed with error code "
          << hipGetErrorString(status);
      return true;
    });
  });
}

void apply_llama31_rope(TensorView q, TensorView k, TensorView q_rope, TensorView k_rope,
                        TensorView indptr, TensorView offsets, int64_t rotary_dim, bool interleave,
                        double rope_scale, double rope_theta, double low_freq_factor,
                        double high_freq_factor, double old_context_length) {
  CHECK_CUDA(q);  // not necessarily contiguous
  CHECK_CUDA(k);  // not necessarily contiguous
  CHECK_INPUT(indptr);
  CHECK_INPUT(offsets);

  CHECK_DEVICE(q, k);
  CHECK_DIM(3, q);        // q: (nnz, H_Q, D)
  CHECK_DIM(3, k);        // k: (nnz, H_K, D)
  CHECK_DIM(1, indptr);   // indptr: (B + 1)
  CHECK_DIM(1, offsets);  // offsets: (B)
  TVM_FFI_ICHECK_EQ(q->shape[0], k->shape[0]);
  TVM_FFI_ICHECK_EQ(q->shape[2], k->shape[2]);
  unsigned int num_qo_heads = q->shape[1];
  unsigned int num_kv_heads = k->shape[1];
  unsigned int head_dim = q->shape[2];
  unsigned int batch_size = offsets->shape[0];
  TVM_FFI_ICHECK_EQ(indptr->shape[0], batch_size + 1);
  TVM_FFI_ICHECK_EQ(indptr->dtype, offsets->dtype);
  size_t q_stride_n = q->strides[0];
  size_t q_stride_h = q->strides[1];
  size_t k_stride_n = k->strides[0];
  size_t k_stride_h = k->strides[1];
  size_t q_rope_stride_n = q_rope->strides[0];
  size_t q_rope_stride_h = q_rope->strides[1];
  size_t k_rope_stride_n = k_rope->strides[0];
  size_t k_rope_stride_h = k_rope->strides[1];
  TVM_FFI_ICHECK_EQ(indptr->dtype, offsets->dtype);

  hipSetDevice(q->device.device_id);
  const hipStream_t stream = get_stream(q->device);
  DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(q->dtype, c_type, [&] {
    return DISPATCH_DLPACK_IDTYPE_TO_CTYPE(indptr->dtype, c_idtype, [&] {
      hipError_t status = BatchQKApplyLlama31Rotary(
          static_cast<c_type*>(q->data), static_cast<c_type*>(k->data),
          static_cast<c_type*>(q_rope->data), static_cast<c_type*>(k_rope->data),
          static_cast<c_idtype*>(indptr->data), static_cast<c_idtype*>(offsets->data), batch_size,
          num_qo_heads, num_kv_heads, rotary_dim, head_dim, q_stride_n, q_stride_h, k_stride_n,
          k_stride_h, q_rope_stride_n, q_rope_stride_h, k_rope_stride_n, k_rope_stride_h,
          interleave, rope_scale, rope_theta, low_freq_factor, high_freq_factor, old_context_length,
          stream);

      TVM_FFI_ICHECK(status == hipSuccess)
          << "BatchQKApplyLlama31Rotary failed with error code " << hipGetErrorString(status);
      return true;
    });
  });
}

void apply_llama31_rope_pos_ids(TensorView q, TensorView k, TensorView q_rope, TensorView k_rope,
                                TensorView pos_ids, int64_t rotary_dim, bool interleave,
                                double rope_scale, double rope_theta, double low_freq_factor,
                                double high_freq_factor, double old_context_length) {
  CHECK_CUDA(q);  // not necessarily contiguous
  CHECK_CUDA(k);  // not necessarily contiguous
  CHECK_INPUT(pos_ids);

  CHECK_DEVICE(q, k);
  CHECK_DIM(3, q);  // q: (nnz, H_Q, D)
  CHECK_DIM(3, k);  // k: (nnz, H_K, D)
  TVM_FFI_ICHECK_EQ(q->shape[0], k->shape[0]);
  TVM_FFI_ICHECK_EQ(q->shape[2], k->shape[2]);
  unsigned int num_qo_heads = q->shape[1];
  unsigned int num_kv_heads = k->shape[1];
  unsigned int head_dim = q->shape[2];
  unsigned int nnz = q->shape[0];
  size_t q_stride_n = q->strides[0];
  size_t q_stride_h = q->strides[1];
  size_t k_stride_n = k->strides[0];
  size_t k_stride_h = k->strides[1];
  size_t q_rope_stride_n = q_rope->strides[0];
  size_t q_rope_stride_h = q_rope->strides[1];
  size_t k_rope_stride_n = k_rope->strides[0];
  size_t k_rope_stride_h = k_rope->strides[1];

  hipSetDevice(q->device.device_id);
  const hipStream_t stream = get_stream(q->device);
  DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(q->dtype, c_type, [&] {
    return DISPATCH_DLPACK_IDTYPE_TO_CTYPE(pos_ids->dtype, c_idtype, [&] {
      hipError_t status = BatchQKApplyLlama31RotaryPosIds(
          static_cast<c_type*>(q->data), static_cast<c_type*>(k->data),
          static_cast<c_type*>(q_rope->data), static_cast<c_type*>(k_rope->data),
          static_cast<c_idtype*>(pos_ids->data), nnz, num_qo_heads, num_kv_heads, rotary_dim,
          head_dim, q_stride_n, q_stride_h, k_stride_n, k_stride_h, q_rope_stride_n,
          q_rope_stride_h, k_rope_stride_n, k_rope_stride_h, interleave, rope_scale, rope_theta,
          low_freq_factor, high_freq_factor, old_context_length, stream);

      TVM_FFI_ICHECK(status == hipSuccess)
          << "BatchQKApplyLlama31RotaryPosIds failed with error code "
          << hipGetErrorString(status);
      return true;
    });
  });
}

void mla_rope_quantize(TensorView q_rope_in, TensorView k_rope_in, TensorView q_nope_in,
                       TensorView k_nope_in, TensorView q_rope_out, TensorView k_rope_out,
                       TensorView q_nope_out, TensorView k_nope_out, TensorView cos_sin_cache,
                       TensorView pos_ids, double quant_scale_q, double quant_scale_kv,
                       bool interleave) {
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(q_rope_in);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(k_rope_in);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(q_nope_in);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(k_nope_in);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(q_rope_out);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(k_rope_out);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(q_nope_out);
  CHECK_LAST_DIM_CONTIGUOUS_INPUT(k_nope_out);
  CHECK_INPUT(cos_sin_cache);
  CHECK_INPUT(pos_ids);

  TVM_FFI_ICHECK_EQ(q_rope_in->shape[q_rope_in->ndim - 1], 64);
  TVM_FFI_ICHECK_EQ(k_rope_in->shape[k_rope_in->ndim - 1], 64);
  TVM_FFI_ICHECK_EQ(q_nope_in->shape[q_nope_in->ndim - 1], 512);
  TVM_FFI_ICHECK_EQ(k_nope_in->shape[k_nope_in->ndim - 1], 512);
  TVM_FFI_ICHECK_EQ(q_rope_out->shape[q_rope_out->ndim - 1], 64);
  TVM_FFI_ICHECK_EQ(k_rope_out->shape[k_rope_out->ndim - 1], 64);
  TVM_FFI_ICHECK_EQ(q_nope_out->shape[q_nope_out->ndim - 1], 512);
  TVM_FFI_ICHECK_EQ(k_nope_out->shape[k_nope_out->ndim - 1], 512);
  TVM_FFI_ICHECK_EQ(q_rope_in->dtype, k_rope_in->dtype);
  TVM_FFI_ICHECK_EQ(q_rope_in->dtype, q_nope_in->dtype);
  TVM_FFI_ICHECK_EQ(q_rope_in->dtype, k_nope_in->dtype);
  TVM_FFI_ICHECK_EQ(q_rope_out->dtype, k_rope_out->dtype);
  TVM_FFI_ICHECK_EQ(q_rope_out->dtype, q_nope_out->dtype);
  TVM_FFI_ICHECK_EQ(q_rope_out->dtype, k_nope_out->dtype);

  CHECK_DIM(3, q_rope_in);   // q_rope_in: (nnz, H_Q, 64)
  CHECK_DIM(3, q_nope_in);   // q_nope_in: (nnz, H_Q, 512)
  CHECK_DIM(2, k_rope_in);   // k_rope_in: (nnz, 64)
  CHECK_DIM(2, k_nope_in);   // k_nope_in: (nnz, 512)
  CHECK_DIM(3, q_rope_out);  // q_rope_out: (nnz, H_Q, 64)
  CHECK_DIM(3, q_nope_out);  // q_nope_out: (nnz, H_Q, 512)
  CHECK_DIM(2, k_rope_out);  // k_rope_out: (nnz, 64)
  CHECK_DIM(2, k_nope_out);  // k_nope_out: (nnz, 512)
  uint32_t nnz = q_rope_in->shape[0];
  TVM_FFI_ICHECK_EQ(q_nope_in->shape[0], nnz);
  TVM_FFI_ICHECK_EQ(k_nope_in->shape[0], nnz);
  TVM_FFI_ICHECK_EQ(q_rope_out->shape[0], nnz);
  TVM_FFI_ICHECK_EQ(k_rope_out->shape[0], nnz);
  TVM_FFI_ICHECK_EQ(q_nope_out->shape[0], nnz);
  TVM_FFI_ICHECK_EQ(k_nope_out->shape[0], nnz);
  uint32_t num_heads = q_rope_in->shape[1];
  TVM_FFI_ICHECK_EQ(q_rope_in->shape[1], num_heads);
  TVM_FFI_ICHECK_EQ(q_nope_in->shape[1], num_heads);
  TVM_FFI_ICHECK_EQ(q_rope_out->shape[1], num_heads);
  TVM_FFI_ICHECK_EQ(q_nope_out->shape[1], num_heads);

  const uint32_t q_rope_in_stride_n = q_rope_in->strides[0];
  const uint32_t q_rope_in_stride_h = q_rope_in->strides[1];
  const uint32_t q_nope_in_stride_n = q_nope_in->strides[0];
  const uint32_t q_nope_in_stride_h = q_nope_in->strides[1];
  const uint32_t q_rope_out_stride_n = q_rope_out->strides[0];
  const uint32_t q_rope_out_stride_h = q_rope_out->strides[1];
  const uint32_t q_nope_out_stride_n = q_nope_out->strides[0];
  const uint32_t q_nope_out_stride_h = q_nope_out->strides[1];
  const uint32_t k_rope_in_stride = k_rope_in->strides[0];
  const uint32_t k_nope_in_stride = k_nope_in->strides[0];
  const uint32_t k_rope_out_stride = k_rope_out->strides[0];
  const uint32_t k_nope_out_stride = k_nope_out->strides[0];

  hipSetDevice(q_rope_in->device.device_id);
  const hipStream_t stream = get_stream(q_rope_in->device);
  DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(q_rope_in->dtype, c_type, [&] {
    return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP8(q_rope_out->dtype, c_quant_type, [&] {
      return DISPATCH_DLPACK_IDTYPE_TO_CTYPE(pos_ids->dtype, c_idtype, [&] {
        hipError_t status = MLARopeQuantize(
            static_cast<c_type*>(q_rope_in->data), static_cast<c_type*>(k_rope_in->data),
            static_cast<c_type*>(q_nope_in->data), static_cast<c_type*>(k_nope_in->data),
            static_cast<c_quant_type*>(q_rope_out->data),
            static_cast<c_quant_type*>(k_rope_out->data),
            static_cast<c_quant_type*>(q_nope_out->data),
            static_cast<c_quant_type*>(k_nope_out->data), static_cast<float*>(cos_sin_cache->data),
            static_cast<c_idtype*>(pos_ids->data), nnz, num_heads, q_rope_in_stride_n,
            q_rope_in_stride_h, q_nope_in_stride_n, q_nope_in_stride_h, q_rope_out_stride_n,
            q_rope_out_stride_h, q_nope_out_stride_n, q_nope_out_stride_h, k_rope_in_stride,
            k_nope_in_stride, k_rope_out_stride, k_nope_out_stride, quant_scale_q, quant_scale_kv,
            interleave, stream);

        TVM_FFI_ICHECK(status == hipSuccess)
            << "BatchQKApplyRotaryPosIdsCosSinCache failed with error code "
            << hipGetErrorString(status);
        return true;
      });
    });
  });
}
