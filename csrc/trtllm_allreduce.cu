#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <string>

#include "flashinfer/comm/trtllm_allreduce.cuh"
#include "pytorch_extension_utils.h"

using namespace flashinfer::trtllm_allreduce;

#define DISPATCH_ALLREDUCE_DTYPE(TENSOR_SCALAR_TYPE, CTYPE_ALIAS, CODE_BLOCK)               \
  [&]() {                                                                                   \
    if (TENSOR_SCALAR_TYPE == at::ScalarType::Float) {                                      \
      using CTYPE_ALIAS = float;                                                            \
      CODE_BLOCK;                                                                           \
    } else if (TENSOR_SCALAR_TYPE == at::ScalarType::Half) {                                \
      using CTYPE_ALIAS = half;                                                             \
      CODE_BLOCK;                                                                           \
    } else if (TENSOR_SCALAR_TYPE == at::ScalarType::BFloat16) {                            \
      using CTYPE_ALIAS = __hip_bfloat16;                                                    \
      CODE_BLOCK;                                                                           \
    } else {                                                                                \
      TORCH_CHECK(false, "Unsupported DType for custom op dispatch: ", TENSOR_SCALAR_TYPE); \
    }                                                                                       \
  }()

#define DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE(ctype, in, ...)                                     \
  [&] {                                                                                           \
    const auto& scalar_type = (in).scalar_type();                                                 \
    switch (scalar_type) {                                                                        \
      case at::ScalarType::Float: {                                                               \
        using ctype = float;                                                                      \
        return __VA_ARGS__();                                                                     \
      }                                                                                           \
      /* Requires nv_half to be defined somewhere */                                              \
      case at::ScalarType::Half: {                                                                \
        using ctype = half;                                                                       \
        return __VA_ARGS__();                                                                     \
      }                                                                                           \
      /* Requires hip_bfloat16 to be defined somewhere */                                          \
      case at::ScalarType::BFloat16: {                                                            \
        using ctype = __hip_bfloat16;                                                              \
        return __VA_ARGS__();                                                                     \
      }                                                                                           \
      default:                                                                                    \
        TORCH_CHECK(false,                                                                        \
                    "Unsupported dtype in DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE: ", scalar_type); \
    }                                                                                             \
  }()

void trtllm_lamport_initialize(int64_t buffer_ptr, int64_t size, at::ScalarType dtype) {
  DISPATCH_ALLREDUCE_DTYPE(dtype, c_type, {
    hipStream_t raw_stream = at::cuda::getCurrentCUDAStream().stream();
    auto status = lamportInitialize<c_type>(reinterpret_cast<void*>(buffer_ptr),
                                            static_cast<size_t>(size), raw_stream);
    TORCH_CHECK(status == hipSuccess, "lamportInitialize failed with error code " +
                                           std::string(hipGetErrorString(status)));
  });
}

void trtllm_lamport_initialize_all(int64_t buffer_0_ptr, int64_t buffer_1_ptr, int64_t buffer_2_ptr,
                                   int64_t size, at::ScalarType dtype) {
  DISPATCH_ALLREDUCE_DTYPE(dtype, c_type, {
    hipStream_t raw_stream = at::cuda::getCurrentCUDAStream().stream();
    auto status = lamportInitializeAll<c_type>(
        reinterpret_cast<void*>(buffer_0_ptr), reinterpret_cast<void*>(buffer_1_ptr),
        reinterpret_cast<void*>(buffer_2_ptr), static_cast<size_t>(size), raw_stream);
    TORCH_CHECK(status == hipSuccess, "lamportInitializeAll failed with error code " +
                                           std::string(hipGetErrorString(status)));
  });
}

// refer to cpp/tests/unit_tests/kernels/allReduce/allReduceFusionTest.cu:L268
void trtllm_custom_all_reduce(at::Tensor& in, at::Tensor& out, int64_t tp_size, int64_t tp_rank,
                              int64_t token_num, int64_t fusion_op_code, int64_t strategy_code,
                              int64_t config_code, bool launch_with_pdl, int64_t flag_value,
                              at::Tensor peer_comm_buffer_ptrs,  // std::vector<void*>
                              at::Tensor peer_barrier_ptrs_in,   // std::vector<void*>
                              at::Tensor peer_barrier_ptrs_out,  // std::vector<void*>
                              std::optional<at::Tensor> bias, std::optional<at::Tensor> residual,
                              std::optional<at::Tensor> weight,
                              std::optional<at::Tensor> weight_pre_residual_norm,
                              std::optional<double> eps,
                              std::optional<at::Tensor> intermediate_buffer,
                              std::optional<at::Tensor> lamport_peer_comm_buffer_ptrs_0,
                              std::optional<at::Tensor> lamport_peer_comm_buffer_ptrs_1,
                              std::optional<at::Tensor> lamport_peer_comm_buffer_ptrs_2) {
  AllReduceFusionOp fusion_op = static_cast<AllReduceFusionOp>(fusion_op_code);
  const c10::cuda::OptionalCUDAGuard device_guard(in.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  // TODO(zihao): review dispatch type - support fp16, bf16 only
  DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE(c_type, in, [&] {
    // TODO(yingyi): remove type template here (used to check if lamport is supported)
    int64_t message_size = in.numel();
    int64_t hidden_size = in.numel() / token_num;

    AllReduceParams<c_type> params;
    params.elts_total = message_size;
    params.local_rank = tp_rank;
    params.ranks_per_node = tp_size;
    params.local_input_buffer_ptr = in.data_ptr();
    params.local_output_buffer_ptr = out.data_ptr();

    // NOTE(yingyi): review the barrier flag
    // int flag_offset;
    // if (fusion_op == AllReduceFusionOp::RESIDUAL_RMS_NORM &&
    //     is_lamport_supported<c_type>(token_num, hidden_size)) {
    //   flag_offset = 0;
    // } else {
    //   flag_offset = 1;
    // }

    // auto const flag_ptr = reinterpret_cast<int64_t*>(flag_buffer_ptr) + NUM_POINTERS_PER_RANK *
    // tp_size + flag_offset; *flag_ptr += 1; uint32_t flag_value = *flag_ptr;
    params.barrier_flag = flag_value;

    // add fusion params
    params.fusion_params.bias_buffer = bias.has_value() ? bias.value().data_ptr() : nullptr;
    params.fusion_params.residual_buffer =
        residual.has_value() ? residual.value().data_ptr() : nullptr;
    params.fusion_params.hidden_size = hidden_size;
    params.fusion_params.weight_buffer = weight.has_value() ? weight.value().data_ptr() : nullptr;
    params.fusion_params.weight_buffer_pre_residual_norm =
        weight_pre_residual_norm.has_value() ? weight_pre_residual_norm.value().data_ptr()
                                             : nullptr;
    params.fusion_params.eps = eps.has_value() ? eps.value() : 1e-5f;
    params.fusion_params.intermediate_buffer =
        intermediate_buffer.has_value() ? intermediate_buffer.value().data_ptr() : nullptr;

    // add ipc buffer pointers
    for (int i = 0; i < tp_size; ++i) {
      params.peer_comm_buffer_ptrs[i] =
          reinterpret_cast<void*>(peer_comm_buffer_ptrs.data_ptr<int64_t>()[i]);
      params.peer_barrier_ptrs_in[i] =
          reinterpret_cast<uint32_t*>(peer_barrier_ptrs_in.data_ptr<int64_t>()[i]);
      params.peer_barrier_ptrs_out[i] =
          reinterpret_cast<uint32_t*>(peer_barrier_ptrs_out.data_ptr<int64_t>()[i]);
    }

    if (lamport_peer_comm_buffer_ptrs_0.has_value()) {
      TORCH_CHECK(lamport_peer_comm_buffer_ptrs_1.has_value(),
                  "lamport_peer_comm_buffer_ptrs_1 is required if lamport_peer_comm_buffer_ptrs_0 "
                  "is provided");
      TORCH_CHECK(lamport_peer_comm_buffer_ptrs_2.has_value(),
                  "lamport_peer_comm_buffer_ptrs_2 is required if lamport_peer_comm_buffer_ptrs_0 "
                  "is provided");
      for (int i = 0; i < tp_size; ++i) {
        params.fusion_params.lamport_peer_comm_buffer_ptrs[i] =
            reinterpret_cast<void*>(lamport_peer_comm_buffer_ptrs_0.value().data_ptr<int64_t>()[i]);
        params.fusion_params.lamport_peer_comm_buffer_ptrs[i + tp_size] =
            reinterpret_cast<void*>(lamport_peer_comm_buffer_ptrs_1.value().data_ptr<int64_t>()[i]);
        params.fusion_params.lamport_peer_comm_buffer_ptrs[i + tp_size * 2] =
            reinterpret_cast<void*>(lamport_peer_comm_buffer_ptrs_2.value().data_ptr<int64_t>()[i]);
      }
    }

    auto strategy = static_cast<AllReduceStrategyType>(strategy_code);
    auto config = static_cast<AllReduceStrategyConfig>(config_code);

    auto status = customAllReduce(params, strategy, config, fusion_op, launch_with_pdl, stream);
    TORCH_CHECK(status == hipSuccess, "customAllReduce failed with error code " +
                                           std::string(hipGetErrorString(status)));
  });
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_lamport_initialize", &trtllm_lamport_initialize);
  m.def("trtllm_lamport_initialize_all", &trtllm_lamport_initialize_all);
  m.def("trtllm_custom_all_reduce", &trtllm_custom_all_reduce);
}
