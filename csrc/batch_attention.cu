/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/layout.cuh>
#include <flashinfer/pos_enc.cuh>
#include <optional>

#include "batch_attention_config.inc"
#include "pytorch_conversion_utils.h"
#include "pytorch_extension_utils.h"

namespace flashinfer {

template <uint32_t CTA_TILE_Q_1, uint32_t CTA_TILE_Q_2, uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO,
          MaskMode MASK_MODE, typename AttentionVariant, typename Params>
hipError_t BatchPagedAttentionPersistent(const Params params_1, const Params params_2,
                                          const uint32_t num_blks_x, const uint32_t num_blks_y,
                                          const hipStream_t stream);
}  // namespace flashinfer

using namespace flashinfer;

at::Tensor BatchPagedAttentionPlan(at::Tensor float_workspace_buffer,
                                   at::Tensor int_workspace_buffer,
                                   at::Tensor page_locked_int_workspace_buffer,
                                   at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor kv_len,
                                   int64_t batch_size, int64_t num_qo_heads, int64_t num_kv_heads,
                                   int64_t head_dim_o, bool causal) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();

  HolisticPlanInfo<2> plan_info;

  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer.device());
  const hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  hipError_t status = TwoStageHolisticPlan<IdType>(
      float_workspace_buffer.data_ptr(), float_workspace_size_in_bytes,
      int_workspace_buffer.data_ptr(), page_locked_int_workspace_buffer.data_ptr(),
      int_workspace_size_in_bytes, plan_info, qo_indptr.data_ptr<IdType>(),
      kv_indptr.data_ptr<IdType>(), kv_len.data_ptr<IdType>(), batch_size, num_qo_heads,
      num_kv_heads, head_dim_o, causal, stream);

  TORCH_CHECK(status == hipSuccess,
              "Failed to plan persistent paged attention, error: ", hipGetErrorString(status));

  return vec_to_tensor(plan_info.ToVector());
}

void BatchPagedAttentionRun(at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
                            at::Tensor plan_info_vec, at::Tensor q, at::Tensor k_cache,
                            at::Tensor v_cache, at::Tensor kv_indices, at::Tensor o,
                            std::optional<at::Tensor> maybe_lse, int64_t mask_mode_code,
                            int64_t layout_code, int64_t num_qo_heads, int64_t num_kv_heads,
                            int64_t page_size, double sm_scale ADDITIONAL_FUNC_PARAMS) {
  HolisticPlanInfo<2> plan_info;
  plan_info.FromVector(tensor_to_vec(plan_info_vec));

  auto device = q.device();

  void* float_buffer_ptr = float_workspace_buffer.data_ptr();
  void* int_buffer_ptr = int_workspace_buffer.data_ptr();

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k_cache.scalar_type();

  // NOTE (Yilong): assume both q and o are NHD
  unsigned int q_stride_n = q.stride(0);
  unsigned int q_stride_h = q.stride(1);

  // layout only constraint paged KV
  const QKVLayout kv_layout = static_cast<QKVLayout>(layout_code);
  unsigned int k_stride_page = k_cache.stride(0);
  unsigned int v_stride_page = v_cache.stride(0);
  unsigned int k_stride_n, k_stride_h, v_stride_n, v_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    k_stride_h = k_cache.stride(2);
    k_stride_n = k_cache.stride(1);
    v_stride_h = v_cache.stride(2);
    v_stride_n = v_cache.stride(1);
  } else {
    k_stride_h = k_cache.stride(1);
    k_stride_n = k_cache.stride(2);
    v_stride_h = v_cache.stride(1);
    v_stride_n = v_cache.stride(2);
  }

  const c10::cuda::OptionalCUDAGuard device_guard(device);
  const hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
      AttentionVariant, PersistentParams, [&] {
        PersistentParams params[2];

        for (int i = 0; i < 2; i++) {
          params[i].q = static_cast<DTypeQ*>(q.data_ptr());
          params[i].k = static_cast<DTypeKV*>(k_cache.data_ptr());
          params[i].v = static_cast<DTypeKV*>(v_cache.data_ptr());

          params[i].q_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].q_indptr_offset);
          params[i].kv_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_indptr_offset);
          params[i].partial_indptr = GetPtrFromBaseOffset<IdType>(
              int_buffer_ptr, plan_info.tasks[i].partial_indptr_offset);
          params[i].kv_indices = static_cast<int*>(kv_indices.data_ptr());
          params[i].q_len =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].q_len_offset);
          params[i].kv_len =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_len_offset);
          params[i].q_start =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].q_start_offset);
          params[i].kv_start =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_start_offset);
          params[i].kv_end =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_end_offset);
          params[i].kv_head_idx_arr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_head_idx_offset);
          params[i].work_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].work_indptr_offset);
          params[i].len_kv_chunk =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].len_kv_chunk_offset);

          params[i].final_o = static_cast<DTypeO*>(o.data_ptr());
          params[i].final_lse =
              maybe_lse.has_value() ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
          params[i].partial_o =
              GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.partial_o_offset);
          params[i].partial_lse =
              GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.partial_lse_offset);

          // for state reduction
          params[i].merge_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
          params[i].merge_o_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_o_indices_offset);
          params[i].num_packed_qo_len =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.num_qo_len_offset);

          params[i].num_kv_heads = num_kv_heads;
          params[i].gqa_group_size = uint_fastdiv(num_qo_heads / num_kv_heads);
          params[i].page_size = uint_fastdiv(page_size);

          params[i].q_stride_n = q_stride_n;
          params[i].q_stride_h = q_stride_h;
          params[i].k_stride_page = k_stride_page;
          params[i].k_stride_h = k_stride_h;
          params[i].k_stride_n = k_stride_n;
          params[i].v_stride_page = v_stride_page;
          params[i].v_stride_h = v_stride_h;
          params[i].v_stride_n = v_stride_n;

          params[i].sm_scale = sm_scale;

          ADDITIONAL_PARAMS_SETTER
        }

        hipError_t status = BatchPagedAttentionPersistent<128, 16, HEAD_DIM_QK, HEAD_DIM_VO,
                                                           MASK_MODE, AttentionVariant>(
            params[0], params[1], plan_info.num_blks_x, plan_info.num_blks_y, stream);
        TORCH_CHECK(status == hipSuccess, "Failed to run persistent paged attention, error: ",
                    hipGetErrorString(status));
        return true;
      });
}
