/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/layout.cuh>
#include <flashinfer/pos_enc.cuh>

#include "batch_attention_config.inc"
#include "tvm_ffi_utils.h"

namespace flashinfer {

using tvm::ffi::Array;
using tvm::ffi::Optional;

template <uint32_t CTA_TILE_Q_1, uint32_t CTA_TILE_Q_2, uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO,
          MaskMode MASK_MODE, typename AttentionVariant, typename Params>
hipError_t BatchPagedAttentionPersistent(const Params params_1, const Params params_2,
                                          const uint32_t num_blks_x, const uint32_t num_blks_y,
                                          const hipStream_t stream);
}  // namespace flashinfer

using namespace flashinfer;

Array<int64_t> BatchPagedAttentionPlan(Tensor float_workspace_buffer, Tensor int_workspace_buffer,
                                       Tensor page_locked_int_workspace_buffer, Tensor qo_indptr,
                                       Tensor kv_indptr, Tensor kv_len, int64_t batch_size,
                                       int64_t num_qo_heads, int64_t num_kv_heads,
                                       int64_t head_dim_o, bool causal) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer->shape[0] * get_element_size(float_workspace_buffer);
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer->shape[0] * get_element_size(int_workspace_buffer);

  HolisticPlanInfo<2> plan_info;

  hipSetDevice(float_workspace_buffer->device.device_id);
  const hipStream_t stream = get_stream(float_workspace_buffer->device);

  hipError_t status = TwoStageHolisticPlan<IdType>(
      float_workspace_buffer->data, float_workspace_size_in_bytes, int_workspace_buffer->data,
      page_locked_int_workspace_buffer->data, int_workspace_size_in_bytes, plan_info,
      static_cast<IdType*>(qo_indptr->data), static_cast<IdType*>(kv_indptr->data),
      static_cast<IdType*>(kv_len->data), batch_size, num_qo_heads, num_kv_heads, head_dim_o,
      causal, stream);

  TVM_FFI_ICHECK(status == hipSuccess)
      << "Failed to plan persistent paged attention, error: " << hipGetErrorString(status);

  return Array(plan_info.ToVector());
}

void BatchPagedAttentionRun(Tensor float_workspace_buffer, Tensor int_workspace_buffer,
                            Array<int64_t> plan_info_vec, Tensor q, Tensor k_cache, Tensor v_cache,
                            Tensor kv_indices, Tensor o, Optional<Tensor> maybe_lse,
                            int64_t mask_mode_code, int64_t layout_code, int64_t num_qo_heads,
                            int64_t num_kv_heads, int64_t page_size,
                            double v_scale,  // must use double due to pytorch binding
                            double sm_scale,
                            double logits_soft_cap ADDITIONAL_FUNC_PARAMS PROFILER_FUNC_PARAMS) {
  HolisticPlanInfo<2> plan_info;
  plan_info.FromVector(std::vector<int64_t>(plan_info_vec.begin(), plan_info_vec.end()));

  void* float_buffer_ptr = float_workspace_buffer->data;
  void* int_buffer_ptr = int_workspace_buffer->data;

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  // NOTE (Yilong): assume both q and o are NHD
  unsigned int q_stride_n = q->strides[0];
  unsigned int q_stride_h = q->strides[1];

  // layout only constraint paged KV
  const QKVLayout kv_layout = static_cast<QKVLayout>(layout_code);
  unsigned int k_stride_page = k_cache->strides[0];
  unsigned int v_stride_page = v_cache->strides[0];
  unsigned int k_stride_n, k_stride_h, v_stride_n, v_stride_h;
  if (kv_layout == QKVLayout::kNHD) {
    k_stride_h = k_cache->strides[2];
    k_stride_n = k_cache->strides[1];
    v_stride_h = v_cache->strides[2];
    v_stride_n = v_cache->strides[1];
  } else {
    k_stride_h = k_cache->strides[1];
    k_stride_n = k_cache->strides[2];
    v_stride_h = v_cache->strides[1];
    v_stride_n = v_cache->strides[2];
  }

  hipSetDevice(q->device.device_id);
  const hipStream_t stream = get_stream(q->device);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE,
      AttentionVariant, PersistentParams, [&] {
        PersistentParams params[2];
        IdType* len_kv_chunk =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.len_kv_chunk_offset);
        for (int i = 0; i < 2; i++) {
          params[i].q = static_cast<DTypeQ*>(q->data);
          params[i].k = static_cast<DTypeKV*>(k_cache->data);
          params[i].v = static_cast<DTypeKV*>(v_cache->data);

          params[i].q_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].q_indptr_offset);
          params[i].kv_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_indptr_offset);
          params[i].partial_indptr = GetPtrFromBaseOffset<IdType>(
              int_buffer_ptr, plan_info.tasks[i].partial_indptr_offset);
          params[i].kv_indices = static_cast<int*>(kv_indices->data);
          params[i].q_len =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].q_len_offset);
          params[i].kv_len =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_len_offset);
          params[i].q_start =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].q_start_offset);
          params[i].kv_start =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_start_offset);
          params[i].kv_end =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_end_offset);
          params[i].kv_head_idx_arr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_head_idx_offset);
          params[i].work_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].work_indptr_offset);
          params[i].len_kv_chunk = len_kv_chunk + i;

          params[i].final_o = static_cast<DTypeO*>(o->data);
          params[i].final_lse =
              maybe_lse.has_value() ? static_cast<float*>(maybe_lse.value()->data) : nullptr;
          params[i].partial_o =
              GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.partial_o_offset);
          params[i].partial_lse =
              GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.partial_lse_offset);

          // for state reduction
          params[i].merge_indptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
          params[i].merge_o_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_o_indices_offset);
          params[i].num_packed_qo_len =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.num_qo_len_offset);

          params[i].num_kv_heads = num_kv_heads;
          params[i].gqa_group_size = uint_fastdiv(num_qo_heads / num_kv_heads);
          params[i].page_size = uint_fastdiv(page_size);

          params[i].q_stride_n = q_stride_n;
          params[i].q_stride_h = q_stride_h;
          params[i].k_stride_page = k_stride_page;
          params[i].k_stride_h = k_stride_h;
          params[i].k_stride_n = k_stride_n;
          params[i].v_stride_page = v_stride_page;
          params[i].v_stride_h = v_stride_h;
          params[i].v_stride_n = v_stride_n;

          params[i].sm_scale = sm_scale;
          params[i].v_scale = v_scale;
          params[i].logits_soft_cap = logits_soft_cap;
          // NOTE(Wenxuan) directly using the additional_params_decl from generate_additional_params
          // will be problematic because of the params[i]
          ADDITIONAL_PARAMS_SETTER
          PROFILER_PARAMS_SETTER
        }

        hipError_t status = BatchPagedAttentionPersistent<128, 16, HEAD_DIM_QK, HEAD_DIM_VO,
                                                           MASK_MODE, AttentionVariant>(
            params[0], params[1], plan_info.num_blks_x, plan_info.num_blks_y, stream);
        TVM_FFI_ICHECK(status == hipSuccess)
            << "Failed to run persistent paged attention, error: " << hipGetErrorString(status);
        return true;
      });
}
