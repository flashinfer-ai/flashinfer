/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/gemm/group_gemm.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;
using namespace flashinfer::group_gemm;

void CutlassSegmentGEMM(at::Tensor workspace_buffer, at::Tensor all_problems, at::Tensor x_ptr,
                        at::Tensor w_ptr, at::Tensor y_ptr, at::Tensor x_ld, at::Tensor w_ld,
                        at::Tensor y_ld, at::Tensor empty_x_data, bool weight_column_major,
                        int64_t hip_stream) {
  unsigned int batch_size = x_ptr.size(0);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(empty_x_data.scalar_type(), c_type, [&] {
    using cutlass_t = cutlass_dtype_t<c_type>;
    auto status = CutlassSegmentGEMMRun<cutlass_t>(
        workspace_buffer.data_ptr(), workspace_buffer.element_size() * workspace_buffer.size(0),
        all_problems.data_ptr(), batch_size, x_ptr.data_ptr(), w_ptr.data_ptr(), y_ptr.data_ptr(),
        x_ld.data_ptr(), w_ld.data_ptr(), y_ld.data_ptr(), weight_column_major, stream);
    TORCH_CHECK(status == hipSuccess,
                "Failed to run CutlassSegmentGEMM: ", hipGetErrorString(status));
    return true;
  });
}
