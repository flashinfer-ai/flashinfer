#include <flashinfer/attention/decode.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <optional>

#include "mla_config.inc"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

std::vector<int64_t> BatchDecodeWithPagedKVCachePlanMLA(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor indptr, unsigned int batch_size,
    unsigned int num_qo_heads, unsigned int page_size, bool enable_cuda_graph,
    int64_t hip_stream) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();

  DecodePlanInfo plan_info;
  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);

  auto work_estimation_func =
      BatchDecodeWithPagedKVCacheWorkEstimationDispatchedMLA<HEAD_DIM_CKV, HEAD_DIM_KPE,
                                                             AttentionVariant, Params>;
  hipError_t status =
      DecodePlan<HEAD_DIM_CKV, flashinfer::PosEncodingMode::kRoPELlama, AttentionVariant, Params>(
          static_cast<void*>(float_workspace_buffer.data_ptr()), float_workspace_size_in_bytes,
          static_cast<void*>(int_workspace_buffer.data_ptr()),
          static_cast<void*>(page_locked_int_workspace_buffer.data_ptr()),
          int_workspace_size_in_bytes, plan_info, static_cast<IdType*>(indptr.data_ptr()),
          batch_size, num_qo_heads, page_size, enable_cuda_graph, /*stream=*/stream,
          work_estimation_func);

  TORCH_CHECK(status == hipSuccess, "BatchDecodeWithPagedKVCachePlanMLA failed with error ",
              hipGetErrorString(status));

  return plan_info.ToVector();
}
