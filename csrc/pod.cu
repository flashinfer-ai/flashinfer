/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/variants.cuh>
#include <flashinfer/pos_enc.cuh>

#include "pod_config.inc"
#include "tvm_ffi_utils.h"

namespace flashinfer {
template <uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO, PosEncodingMode POS_ENCODING_MODE,
          bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE_P, uint32_t CTA_TILE_Q,
          MaskMode MASK_MODE_D, typename PrefillAttentionVariant, typename DecodeAttentionVariant,
          typename PrefillParams, typename DecodeParams>
hipError_t PODWithKVCacheTensorDispatched(PrefillParams prefill_params,
                                           typename PrefillParams::DTypeO* tmp,
                                           DecodeParams decode_params,
                                           typename DecodeParams::DTypeO* tmp_v, float* tmp_s,
                                           bool enable_pdl, hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

using tvm::ffi::Array;
using tvm::ffi::Optional;

void pod_with_kv_cache_tensor(
    // Prefill params
    Tensor q_p, Tensor k_p, Tensor v_p, Tensor tmp_p, Tensor o_p, Optional<Tensor> maybe_lse_p,
    int64_t mask_mode_code_p, int64_t layout_p, int64_t window_left_p,
    Optional<Tensor> maybe_custom_mask_p, Optional<Tensor> maybe_alibi_slopes_p,
    double logits_soft_cap_p, double sm_scale_p, double rope_rcp_scale_p, double rope_rcp_theta_p,
    // Decode params
    Tensor float_workspace_buffer_d, Tensor int_workspace_buffer_d, Array<int64_t> plan_info_vec,
    Tensor q_d, Tensor paged_k_cache_d, Tensor paged_v_cache_d, Tensor qo_indptr_d,
    Tensor paged_kv_indptr_d, Tensor paged_kv_indices_d, Tensor paged_kv_last_page_len_d,
    Tensor o_d, Optional<Tensor> maybe_lse_d, int64_t mask_mode_code_d, int64_t layout_d,
    int64_t window_left_d, Optional<Tensor> maybe_custom_mask_d,
    Optional<Tensor> maybe_mask_indptr_d, Optional<Tensor> maybe_alibi_slopes_d,
    double logits_soft_cap_d, double sm_scale_d, double rope_rcp_scale_d, double rope_rcp_theta_d,
    bool enable_pdl) {
  // Prefill setup
  unsigned int head_dim_qk = q_p->shape[2];
  unsigned int kv_len_p, qo_len_p, num_kv_heads, num_qo_heads;
  QKVLayout kv_layout_p = static_cast<QKVLayout>(layout_p);
  qo_len_p = q_p->shape[0];
  num_qo_heads = q_p->shape[1];
  uint32_t q_stride_n_p = q_p->strides[0], q_stride_h_p = q_p->strides[1], k_stride_n_p,
           k_stride_h_p, v_stride_n_p, v_stride_h_p;
  if (kv_layout_p == QKVLayout::kNHD) {
    kv_len_p = k_p->shape[0];
    num_kv_heads = k_p->shape[1];
    k_stride_n_p = k_p->strides[0];
    k_stride_h_p = k_p->strides[1];
    v_stride_n_p = v_p->strides[0];
    v_stride_h_p = v_p->strides[1];
  } else {
    kv_len_p = k_p->shape[1];
    num_kv_heads = k_p->shape[0];
    k_stride_h_p = k_p->strides[0];
    k_stride_n_p = k_p->strides[1];
    v_stride_h_p = v_p->strides[0];
    v_stride_n_p = v_p->strides[1];
  }
  if (maybe_lse_p.has_value()) {
    const auto& lse = maybe_lse_p.value();
    TVM_FFI_ICHECK_EQ(lse->shape[0], qo_len_p);
    TVM_FFI_ICHECK_EQ(lse->shape[1], num_qo_heads);
  }

  const MaskMode mask_mode_p = static_cast<MaskMode>(mask_mode_code_p);

  // Decode setup (Tensor decode = batched prefill)
  PrefillPlanInfo plan_info;
  plan_info.FromVector(std::vector<int64_t>(plan_info_vec.begin(), plan_info_vec.end()));
  QKVLayout kv_layout_d = static_cast<QKVLayout>(layout_d);
  int64_t batch_size = paged_kv_indptr_d->shape[0] - 1;
  int64_t num_qo_heads_d = q_d->shape[1];

  TVM_FFI_ICHECK_EQ(num_qo_heads, num_qo_heads_d)
      << "POD currently requires same # Query heads for prefill and decode";

  int64_t num_kv_heads_d, page_size_d;
  uint32_t head_dim_qk_d = q_d->shape[2];
  if (kv_layout_d == QKVLayout::kHND) {
    num_kv_heads_d = paged_k_cache_d->shape[1];
    page_size_d = paged_k_cache_d->shape[2];
  } else {
    page_size_d = paged_k_cache_d->shape[1];
    num_kv_heads_d = paged_k_cache_d->shape[2];
  }
  TVM_FFI_ICHECK_EQ(num_kv_heads, num_kv_heads_d)
      << "POD currently requires same # KV heads for prefill and decode; Prefill: " << num_kv_heads
      << ", Decode: " << num_kv_heads_d;

  if (maybe_lse_d.has_value()) {
    const auto& lse = maybe_lse_d.value();
    TVM_FFI_ICHECK_EQ(lse->shape[0], q_d->shape[0]);
    TVM_FFI_ICHECK_EQ(lse->shape[1], q_d->shape[1]);
  }

  void* float_buffer_ptr = static_cast<void*>(float_workspace_buffer_d->data);
  void* int_buffer_ptr = static_cast<void*>(int_workspace_buffer_d->data);

  const MaskMode mask_mode_d = static_cast<MaskMode>(mask_mode_code_d);

  // get q_stride_n and q_stride_h
  const auto q_stride_n_d = q_d->strides[0];
  const auto q_stride_h_d = q_d->strides[1];

  // get kv_cache_strides
  const int64_t* kv_cache_strides_d = nullptr;
  auto k_strides_d = paged_k_cache_d.strides();
  auto v_strides_d = paged_v_cache_d.strides();
  TVM_FFI_ICHECK_EQ(k_strides_d.size(), v_strides_d.size());
  for (int i = 0; i < k_strides_d.size(); ++i) {
    TVM_FFI_ICHECK_EQ(k_strides_d[i], v_strides_d[i]);
  }
  kv_cache_strides_d = k_strides_d.data();

  hipSetDevice(float_workspace_buffer_d->device.device_id);
  const hipStream_t stream = get_stream(float_workspace_buffer_d->device);

  DISPATCH_context(
      MASK_MODE_P, MASK_MODE_D, DTypeQ, DTypeKV, HEAD_DIM_QK, USE_SLIDING_WINDOW_P,
      USE_SLIDING_WINDOW_D, USE_LOGITS_SOFT_CAP, [&] {
        PrefillParams prefill_params;
        {
          // Make params a reference to prefill_params to set values
          PrefillParams& params = prefill_params;
          params.q = static_cast<DTypeQ*>(q_p->data);
          params.k = static_cast<DTypeKV*>(k_p->data);
          params.v = static_cast<DTypeKV*>(v_p->data);
          params.o = static_cast<DTypeO*>(o_p->data);
          params.lse =
              maybe_lse_p.has_value() ? static_cast<float*>(maybe_lse_p.value()->data) : nullptr;
          params.num_qo_heads = num_qo_heads;
          params.num_kv_heads = num_kv_heads;
          params.group_size = uint_fastdiv(num_qo_heads / num_kv_heads);
          params.qo_len = qo_len_p;
          params.kv_len = kv_len_p;
          params.q_stride_n = q_stride_n_p;
          params.q_stride_h = q_stride_h_p;
          params.k_stride_n = k_stride_n_p;
          params.k_stride_h = k_stride_h_p;
          params.v_stride_n = v_stride_n_p;
          params.v_stride_h = v_stride_h_p;

          params.window_left = window_left_p;
          params.partition_kv = false;

          params.maybe_custom_mask = maybe_custom_mask_p.has_value()
                                         ? static_cast<uint8_t*>(maybe_custom_mask_p.value()->data)
                                         : nullptr;
          params.maybe_alibi_slopes = maybe_alibi_slopes_p.has_value()
                                          ? static_cast<float*>(maybe_alibi_slopes_p.value()->data)
                                          : nullptr;
          params.logits_soft_cap = logits_soft_cap_p;
          params.sm_scale = sm_scale_p;
          params.rope_rcp_scale = rope_rcp_scale_p;
          params.rope_rcp_theta = rope_rcp_theta_p;
        }

        DecodeParams decode_params;
        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;
        {
          DecodeParams& params = decode_params;
          params.q = static_cast<DTypeQ*>(q_d->data);
          paged_kv_t<DTypeKV, IdType> paged_kv(
              num_kv_heads, page_size_d, HEAD_DIM_VO, batch_size, kv_layout_d,
              static_cast<DTypeKV*>(paged_k_cache_d->data),
              static_cast<DTypeKV*>(paged_v_cache_d->data), kv_cache_strides_d,
              static_cast<IdType*>(paged_kv_indices_d->data),
              static_cast<IdType*>(paged_kv_indptr_d->data),
              static_cast<IdType*>(paged_kv_last_page_len_d->data));
          params.paged_kv = paged_kv;
          params.q_indptr = static_cast<IdType*>(qo_indptr_d->data);
          params.o = static_cast<DTypeO*>(o_d->data);

          params.lse =
              maybe_lse_d.has_value() ? static_cast<float*>(maybe_lse_d.value()->data) : nullptr;
          params.num_qo_heads = num_qo_heads;
          params.group_size = uint_fastdiv(num_qo_heads / paged_kv.num_heads);
          params.q_stride_n = q_stride_n_d;
          params.q_stride_h = q_stride_h_d;
          params.window_left = window_left_d;

          params.request_indices = nullptr;
          params.qo_tile_indices = nullptr;
          params.kv_tile_indices = nullptr;
          params.merge_indptr = nullptr;
          params.o_indptr = nullptr;
          params.kv_chunk_size_ptr = nullptr;
          params.block_valid_mask = nullptr;
          params.total_num_rows = nullptr;
          params.max_total_num_rows = 0;
          params.padded_batch_size = 0;
          params.partition_kv = false;

          params.maybe_mask_indptr = maybe_mask_indptr_d.has_value()
                                         ? static_cast<int32_t*>(maybe_mask_indptr_d.value()->data)
                                         : nullptr;
          params.maybe_alibi_slopes = maybe_alibi_slopes_d.has_value()
                                          ? static_cast<float*>(maybe_alibi_slopes_d.value()->data)
                                          : nullptr;
          params.logits_soft_cap = logits_soft_cap_d;
          params.sm_scale = sm_scale_d;
          params.rope_rcp_scale = rope_rcp_scale_d;
          params.rope_rcp_theta = rope_rcp_theta_d;

          params.request_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
          params.qo_tile_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
          params.kv_tile_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
          params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
          params.kv_chunk_size_ptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset);
          if (plan_info.split_kv) {
            params.merge_indptr =
                GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
            tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
            tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
            if (plan_info.enable_cuda_graph) {
              params.block_valid_mask =
                  GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
            }
          }
          params.padded_batch_size = plan_info.padded_batch_size;
          params.max_total_num_rows = plan_info.total_num_rows;
          if (plan_info.enable_cuda_graph) {
            params.total_num_rows =
                GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
          }
        }

        constexpr bool use_custom_mask_p = MASK_MODE_P == MaskMode::kCustom;
        using PrefillAttentionVariant =
            DefaultAttention</*use_custom_mask=*/use_custom_mask_p, USE_SLIDING_WINDOW_P,
                             USE_LOGITS_SOFT_CAP, /*use_alibi_bias=*/false>;
        constexpr bool use_custom_mask_d = MASK_MODE_D == MaskMode::kCustom;
        using DecodeAttentionVariant =
            DefaultAttention</*use_custom_mask=*/use_custom_mask_d, USE_SLIDING_WINDOW_D,
                             USE_LOGITS_SOFT_CAP, /*use_alibi_bias=*/false>;
        // DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q, CTA_TILE_Q, {
        constexpr size_t CTA_TILE_Q = 16;
        hipError_t status = flashinfer::PODWithKVCacheTensorDispatched<
            HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE, USE_FP16_QK_REDUCTION, MASK_MODE_P,
            CTA_TILE_Q, MASK_MODE_D, PrefillAttentionVariant, DecodeAttentionVariant>(
            prefill_params, static_cast<DTypeO*>(tmp_p->data), decode_params, tmp_v, tmp_s,
            enable_pdl, stream);
        TVM_FFI_ICHECK(status == hipSuccess)
            << "PODWithKVCache kernel launch failed, error: " << hipGetErrorString(status);
        //});
      });
}
