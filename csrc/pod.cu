/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/variants.cuh>
#include <flashinfer/pos_enc.cuh>

#include "pod_config.inc"
#include "tvm_ffi_utils.h"

namespace flashinfer {
template <uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO, PosEncodingMode POS_ENCODING_MODE,
          bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE_P, uint32_t CTA_TILE_Q_P,
          uint32_t CTA_TILE_Q_D, MaskMode MASK_MODE_D, typename PrefillAttentionVariant,
          typename DecodeAttentionVariant, typename PrefillParams, typename DecodeParams>
hipError_t PODWithKVCacheTensorDispatched(PrefillParams prefill_params, DecodeParams decode_params,
                                           typename DecodeParams::DTypeO* tmp_v, float* tmp_s,
                                           bool enable_pdl, hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;
using tvm::ffi::Array;
using tvm::ffi::Optional;

Array<int64_t> PODWithKVCachePlan(
    TensorView float_workspace_buffer, TensorView int_workspace_buffer,
    TensorView page_locked_int_workspace_buffer, TensorView qo_indptr_p, TensorView kv_indptr_p,
    int64_t total_num_rows_p, int64_t batch_size_p, TensorView qo_indptr_d, TensorView kv_indptr_d,
    int64_t total_num_rows_d, int64_t batch_size_d, int64_t num_qo_heads, int64_t num_kv_heads,
    int64_t head_dim_qk, int64_t head_dim_vo, int64_t page_size, bool enable_cuda_graph) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * get_element_size(float_workspace_buffer);
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * get_element_size(int_workspace_buffer);

  PODPlanInfo plan_info;

  hipSetDevice(float_workspace_buffer.device().device_id);
  const hipStream_t stream = get_stream(float_workspace_buffer.device());
  hipError_t status = PODPlan<IdType>(
      float_workspace_buffer.data_ptr(), float_workspace_size_in_bytes,
      int_workspace_buffer.data_ptr(), page_locked_int_workspace_buffer.data_ptr(),
      int_workspace_size_in_bytes, plan_info, static_cast<IdType*>(qo_indptr_p.data_ptr()),
      static_cast<IdType*>(kv_indptr_p.data_ptr()), total_num_rows_p, batch_size_p,
      static_cast<IdType*>(qo_indptr_d.data_ptr()), static_cast<IdType*>(kv_indptr_d.data_ptr()),
      total_num_rows_d, batch_size_d, num_qo_heads, num_kv_heads, head_dim_qk, head_dim_vo,
      page_size, enable_cuda_graph, /*sizeof_dtype_o=*/2, stream);

  TVM_FFI_ICHECK(status == hipSuccess)
      << "Failed to plan prefill with error: " << hipGetErrorString(status);

  return Array(plan_info.ToVector());
}

void PODWithKVCacheTensorRun(
    // Shared params
    TensorView float_workspace_buffer_d, TensorView int_workspace_buffer_d,
    Array<int64_t> plan_info_vec, TensorView paged_k_cache, TensorView paged_v_cache,
    TensorView qo_indptr, TensorView paged_kv_indptr, TensorView paged_kv_indices,
    TensorView paged_kv_last_page_len, TensorView o, Optional<TensorView> maybe_lse, int64_t layout,
    // Prefill params
    TensorView q_p, int64_t mask_mode_code_p, int64_t window_left_p,
    Optional<TensorView> maybe_custom_mask_p, Optional<TensorView> maybe_alibi_slopes_p,
    double logits_soft_cap_p, double sm_scale_p, double rope_rcp_scale_p, double rope_rcp_theta_p,
    // Decode params
    TensorView q_d, int64_t mask_mode_code_d, int64_t window_left_d,
    Optional<TensorView> maybe_custom_mask_d, Optional<TensorView> maybe_mask_indptr_d,
    Optional<TensorView> maybe_alibi_slopes_d, double logits_soft_cap_d, double sm_scale_d,
    double rope_rcp_scale_d, double rope_rcp_theta_d, bool enable_pdl) {
  PODPlanInfo plan_info;
  plan_info.FromVector(std::vector<int64_t>(plan_info_vec.begin(), plan_info_vec.end()));
  uint32_t batch_size = paged_kv_indptr.size(0) - 1;
  void* float_buffer_ptr = static_cast<void*>(float_workspace_buffer_d.data_ptr());
  void* int_buffer_ptr = static_cast<void*>(int_workspace_buffer_d.data_ptr());

  // Prefill setup
  uint32_t head_dim_qk = q_p.size(2);
  uint32_t qo_len, num_qo_heads_p;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  qo_len = q_p.size(0) + q_d.size(0);
  num_qo_heads_p = q_p.size(1);
  uint32_t q_stride_n_p = q_p.stride(0), q_stride_h_p = q_p.stride(1);
  if (maybe_lse.has_value()) {
    const auto& lse = maybe_lse.value();
    TVM_FFI_ICHECK_EQ(lse.size(0), qo_len);
    TVM_FFI_ICHECK_EQ(lse.size(1), num_qo_heads_p);
  }

  const MaskMode mask_mode_p = static_cast<MaskMode>(mask_mode_code_p);

  // Decode setup (Tensor decode = batched prefill)
  uint32_t num_qo_heads = q_d.size(1);
  TVM_FFI_ICHECK_EQ(num_qo_heads_p, num_qo_heads)
      << "POD currently requires same # Query heads for prefill and decode";

  uint32_t num_kv_heads_d, num_kv_heads, page_size;
  if (kv_layout == QKVLayout::kHND) {
    num_kv_heads = paged_k_cache.size(1);
    num_kv_heads_d = paged_k_cache.size(1);
    page_size = paged_k_cache.size(2);
  } else {
    num_kv_heads = paged_k_cache.size(2);
    num_kv_heads_d = paged_k_cache.size(2);
    page_size = paged_k_cache.size(1);
  }
  TVM_FFI_ICHECK_EQ(num_kv_heads, num_kv_heads_d)
      << "POD currently requires same # KV heads for prefill and decode; Prefill: " << num_kv_heads
      << ", Decode: " << num_kv_heads_d;

  const MaskMode mask_mode_d = static_cast<MaskMode>(mask_mode_code_d);

  // get q_stride_n and q_stride_h
  const auto q_stride_n_d = q_d.stride(0);
  const auto q_stride_h_d = q_d.stride(1);

  hipSetDevice(float_workspace_buffer_d.device().device_id);
  const hipStream_t stream = get_stream(float_workspace_buffer_d.device());

  DISPATCH_context(
      MASK_MODE_P, MASK_MODE_D, DTypeQ, DTypeKV, HEAD_DIM_QK, USE_SLIDING_WINDOW_P,
      USE_SLIDING_WINDOW_D, USE_LOGITS_SOFT_CAP, [&] {
        // Compute kv_cache_strides from tensor strides
        // paged_kv_t expects [stride_page, stride_n, stride_h] where:
        // - stride_page is stride(0)
        // - stride_n and stride_h depend on layout
        int64_t kv_strides[3];
        kv_strides[0] = paged_k_cache.stride(0);  // stride_page
        if (kv_layout == QKVLayout::kHND) {
          kv_strides[1] = paged_k_cache.stride(1);  // stride_h
          kv_strides[2] = paged_k_cache.stride(2);  // stride_n
        } else {
          kv_strides[1] = paged_k_cache.stride(1);  // stride_n
          kv_strides[2] = paged_k_cache.stride(2);  // stride_h
        }
        TVM_FFI_ICHECK(paged_k_cache.stride(0) == paged_v_cache.stride(0) &&
                       paged_k_cache.stride(1) == paged_v_cache.stride(1) &&
                       paged_k_cache.stride(2) == paged_v_cache.stride(2))
            << "k/v strides must be identical";

        paged_kv_t<DTypeKV, IdType> paged_kv(
            num_kv_heads, page_size, HEAD_DIM_VO, batch_size, kv_layout,
            static_cast<DTypeKV*>(paged_k_cache.data_ptr()),
            static_cast<DTypeKV*>(paged_v_cache.data_ptr()), kv_strides,
            static_cast<IdType*>(paged_kv_indices.data_ptr()),
            static_cast<IdType*>(paged_kv_indptr.data_ptr()),
            static_cast<IdType*>(paged_kv_last_page_len.data_ptr()));
        IdType* q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());

        // debug indices
        PrefillParams prefill_params;
        {
          // Make params a reference to prefill_params to set values
          PrefillParams& params = prefill_params;
          params.q = static_cast<DTypeQ*>(q_p.data_ptr());
          params.paged_kv = paged_kv;
          params.q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());

          params.o = static_cast<DTypeO*>(o.data_ptr());
          params.lse =
              maybe_lse.has_value() ? static_cast<float*>(maybe_lse.value().data_ptr()) : nullptr;
          params.group_size = uint_fastdiv(num_qo_heads / paged_kv.num_heads);
          params.q_stride_n = q_stride_n_p;
          params.q_stride_h = q_stride_h_p;
          params.window_left = window_left_p;
          params.paged_kv.num_heads = num_kv_heads;
          params.num_qo_heads = num_qo_heads;

          params.request_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
          params.qo_tile_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
          params.kv_tile_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
          params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
          if (plan_info.split_kv) {
            params.merge_indptr =
                GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
            if (plan_info.enable_cuda_graph) {
              params.block_valid_mask =
                  GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
            }
          }
          params.kv_chunk_size_ptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset_p);
          params.padded_batch_size = plan_info.padded_batch_size_p;
          params.maybe_custom_mask =
              maybe_custom_mask_p.has_value()
                  ? static_cast<uint8_t*>(maybe_custom_mask_p.value().data_ptr())
                  : nullptr;
          params.maybe_alibi_slopes =
              maybe_alibi_slopes_p.has_value()
                  ? static_cast<float*>(maybe_alibi_slopes_p.value().data_ptr())
                  : nullptr;
          params.logits_soft_cap = logits_soft_cap_p;
          params.sm_scale = sm_scale_p;
          params.rope_rcp_scale = rope_rcp_scale_p;
          params.rope_rcp_theta = rope_rcp_theta_p;
          params.max_total_num_rows = plan_info.total_num_rows;
          if (plan_info.enable_cuda_graph) {
            params.total_num_rows =
                GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
          }
          params.partition_kv = plan_info.split_kv;
          if (plan_info.split_kv) {
            if (plan_info.enable_cuda_graph) {
              params.block_valid_mask =
                  GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
            }
          }
        }

        DecodeParams decode_params;
        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;
        {
          DecodeParams& params = decode_params;
          params.q = static_cast<DTypeQ*>(q_d.data_ptr());
          params.paged_kv = paged_kv;
          params.q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());
          params.o = static_cast<DTypeO*>(o.data_ptr());
          params.lse =
              maybe_lse.has_value() ? static_cast<float*>(maybe_lse.value().data_ptr()) : nullptr;
          params.group_size = uint_fastdiv(num_qo_heads / paged_kv.num_heads);
          params.q_stride_n = q_stride_n_d;
          params.q_stride_h = q_stride_h_d;
          params.window_left = window_left_d;
          params.paged_kv.num_heads = num_kv_heads;
          params.num_qo_heads = num_qo_heads;

          params.request_indices = prefill_params.request_indices;
          params.qo_tile_indices = prefill_params.qo_tile_indices;
          params.kv_tile_indices = prefill_params.kv_tile_indices;
          params.o_indptr = prefill_params.o_indptr;
          params.kv_chunk_size_ptr = prefill_params.kv_chunk_size_ptr;

          params.partition_kv = plan_info.split_kv;
          if (plan_info.split_kv) {
            params.merge_indptr = prefill_params.merge_indptr;
            // These should be assigned from plan info, not from prefill_params
            tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
            tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
            if (plan_info.enable_cuda_graph) {
              params.block_valid_mask = prefill_params.block_valid_mask;
            }
          }
          params.padded_batch_size = plan_info.padded_batch_size_d;
          params.max_total_num_rows = plan_info.total_num_rows;

          params.maybe_mask_indptr =
              maybe_mask_indptr_d.has_value()
                  ? static_cast<int32_t*>(maybe_mask_indptr_d.value().data_ptr())
                  : nullptr;
          params.maybe_alibi_slopes =
              maybe_alibi_slopes_d.has_value()
                  ? static_cast<float*>(maybe_alibi_slopes_d.value().data_ptr())
                  : nullptr;
          params.logits_soft_cap = logits_soft_cap_d;
          params.sm_scale = sm_scale_d;
          params.rope_rcp_scale = rope_rcp_scale_d;
          params.rope_rcp_theta = rope_rcp_theta_d;

          if (plan_info.enable_cuda_graph) {
            params.total_num_rows = prefill_params.total_num_rows;
          }
        }

        constexpr bool use_custom_mask_p = MASK_MODE_P == MaskMode::kCustom;
        using PrefillAttentionVariant =
            DefaultAttention</*use_custom_mask=*/use_custom_mask_p, USE_SLIDING_WINDOW_P,
                             USE_LOGITS_SOFT_CAP, /*use_alibi_bias=*/false>;
        constexpr bool use_custom_mask_d = MASK_MODE_D == MaskMode::kCustom;
        using DecodeAttentionVariant =
            DefaultAttention</*use_custom_mask=*/use_custom_mask_d, USE_SLIDING_WINDOW_D,
                             USE_LOGITS_SOFT_CAP, /*use_alibi_bias=*/false>;
        DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q_p, CTA_TILE_Q_P, {
          TVM_FFI_ICHECK(plan_info.cta_tile_q_d == 16)
              << "Decode tile size should be 16 for POD. Check planner.";
          constexpr size_t CTA_TILE_Q_D = 16;
          hipError_t status = flashinfer::PODWithKVCacheTensorDispatched<
              HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE, USE_FP16_QK_REDUCTION, MASK_MODE_P,
              CTA_TILE_Q_P, CTA_TILE_Q_D, MASK_MODE_D, PrefillAttentionVariant,
              DecodeAttentionVariant>(prefill_params, decode_params, tmp_v, tmp_s, enable_pdl,
                                      stream);
          TVM_FFI_ICHECK(status == hipSuccess)
              << "PODWithKVCache kernel launch failed, error: " << hipGetErrorString(status);
        });
      });
}
