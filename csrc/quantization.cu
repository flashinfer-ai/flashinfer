/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/quantization.cuh>

#include "tvm_ffi_utils.h"

using namespace flashinfer;

void packbits(Tensor x, const std::string& bitorder, Tensor y) {
  CHECK_INPUT(x);
  auto device = x->device;
  TVM_FFI_ICHECK(bitorder == "big" || bitorder == "little") << "bitorder must be 'big' or 'little'";

  int64_t num_elements = get_numel(x);
  auto stream = get_stream(x->device);
  hipError_t status = quantization::PackBits(
      static_cast<bool*>(x->data), static_cast<uint8_t*>(y->data), num_elements,
      bitorder == "big" ? quantization::BitOrder::kBig : quantization::BitOrder::kLittle, stream);

  TVM_FFI_ICHECK(status == hipSuccess)
      << "PackBits failed with error code " << hipGetErrorString(status);
}

void segment_packbits(Tensor x, Tensor input_indptr, Tensor output_indptr,
                      const std::string& bitorder, Tensor y) {
  CHECK_INPUT(x);
  CHECK_INPUT(input_indptr);
  CHECK_INPUT(output_indptr);
  CHECK_DEVICE(input_indptr, x);
  CHECK_DEVICE(output_indptr, x);
  TVM_FFI_ICHECK(bitorder == "big" || bitorder == "little") << "bitorder must be 'big' or 'little'";
  unsigned int batch_size = input_indptr->shape[0] - 1;
  TVM_FFI_ICHECK_EQ(output_indptr->shape[0], batch_size + 1)
      << "output_indptr must be on the same device as x";

  auto stream = get_stream(x->device);
  hipError_t status = quantization::SegmentPackBits(
      static_cast<bool*>(x->data), static_cast<uint8_t*>(y->data),
      static_cast<int32_t*>(input_indptr->data), static_cast<int32_t*>(output_indptr->data),
      batch_size,
      bitorder == "big" ? quantization::BitOrder::kBig : quantization::BitOrder::kLittle, stream);
}
