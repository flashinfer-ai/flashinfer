#include "hip/hip_runtime.h"
#include <string>

#include "flashinfer/comm/trtllm_allreduce_fusion.cuh"
#include "pytorch_extension_utils.h"

using namespace flashinfer::trtllm_allreduce_fusion;

#define DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE(scalar_type, c_type, ...)                           \
  [&] {                                                                                           \
    switch (scalar_type) {                                                                        \
      case at::ScalarType::Half: {                                                                \
        using c_type = half;                                                                      \
        return __VA_ARGS__();                                                                     \
      }                                                                                           \
      case at::ScalarType::BFloat16: {                                                            \
        using c_type = __hip_bfloat16;                                                             \
        return __VA_ARGS__();                                                                     \
      }                                                                                           \
      case at::ScalarType::Float: {                                                               \
        using c_type = float;                                                                     \
        return __VA_ARGS__();                                                                     \
      }                                                                                           \
      default:                                                                                    \
        TORCH_CHECK(false,                                                                        \
                    "Unsupported dtype in DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE: ", scalar_type); \
    }                                                                                             \
  }()

void trtllm_allreduce_fusion(
    at::Tensor& allreduce_in, int64_t world_size, int64_t world_rank, int64_t token_num,
    int64_t hidden_size, at::Tensor& workspace_ptrs, bool launch_with_pdl, bool use_oneshot,
    bool trigger_completion_at_end, bool fp32_acc, int64_t pattern_code,
    std::optional<at::Tensor> allreduce_out, std::optional<at::Tensor> residual_in,
    std::optional<at::Tensor> residual_out, std::optional<at::Tensor> norm_out,
    std::optional<at::Tensor> quant_out, std::optional<at::Tensor> scale_out,
    std::optional<at::Tensor> rms_gamma, std::optional<double> rms_eps,
    std::optional<at::Tensor> scale_factor, std::optional<int64_t> layout_code) {
  const c10::cuda::OptionalCUDAGuard device_guard(allreduce_in.device());
  // todo(Yingyi): add dispatch for float and bfloat16

  DISPATCH_FLOATING_TYPES_FOR_ALLREDUCE(allreduce_in.scalar_type(), c_type, [&] {
    AllReduceFusionParams<c_type> params;
    params.nranks = world_size;
    params.rank = world_rank;
    params.size = token_num * hidden_size;
    params.hidden_dim = hidden_size;
    params.workspace = reinterpret_cast<void**>(workspace_ptrs.data_ptr());

    // todo(Yingyi): update optional params
    // todo(Yingyi): add params check with pattern
    params.allreduce_in = reinterpret_cast<void*>(allreduce_in.data_ptr());
    params.allreduce_out = allreduce_out.has_value()
                               ? reinterpret_cast<void*>(allreduce_out.value().data_ptr())
                               : nullptr;
    params.residual_in =
        residual_in.has_value() ? reinterpret_cast<void*>(residual_in.value().data_ptr()) : nullptr;
    params.residual_out = residual_out.has_value()
                              ? reinterpret_cast<void*>(residual_out.value().data_ptr())
                              : nullptr;
    params.norm_out =
        norm_out.has_value() ? reinterpret_cast<void*>(norm_out.value().data_ptr()) : nullptr;
    params.quant_out =
        quant_out.has_value() ? reinterpret_cast<void*>(quant_out.value().data_ptr()) : nullptr;
    params.scale_out =
        scale_out.has_value() ? reinterpret_cast<void*>(scale_out.value().data_ptr()) : nullptr;
    params.rms_gamma =
        rms_gamma.has_value() ? reinterpret_cast<void*>(rms_gamma.value().data_ptr()) : nullptr;
    params.rms_eps = rms_eps.has_value() ? static_cast<float>(rms_eps.value()) : 0.0f;
    params.scale_factor = scale_factor.has_value()
                              ? reinterpret_cast<float*>(scale_factor.value().data_ptr())
                              : nullptr;
    params.use_oneshot = use_oneshot;
    params.layout = layout_code.has_value()
                        ? static_cast<FP4QuantizationSFLayout>(layout_code.value())
                        : FP4QuantizationSFLayout::SWIZZLED;
    params.pattern = static_cast<AllReduceFusionPattern>(pattern_code);
    params.trigger_completion_at_end = trigger_completion_at_end;
    params.stream = at::cuda::getCurrentCUDAStream();

    auto status = allreduce_fusion_op(params, launch_with_pdl, fp32_acc);
    TORCH_CHECK(status == hipSuccess, "allreduce_fusion_op failed with error code",
                hipGetErrorString(status));
  });
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_allreduce_fusion", &trtllm_allreduce_fusion);
}
