// flashinfer: adapted from sglang + vllm code
// refer to: https://github.com/vllm-project/vllm/blob/v0.8.2/csrc/common_extension.cc
/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "flashinfer/distributed/comm_ops.h"
#include "pytorch_extension_utils.h"

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("get_graph_buffer_ipc_meta", &get_graph_buffer_ipc_meta);
  m.def("register_graph_buffers", &register_graph_buffers);
  m.def("dispose", &dispose);
  m.def("meta_size", &meta_size);
  m.def("register_buffer", &register_buffer);

  m.def(
      "init_custom_ar(int[] ipc_tensors, Tensor rank_data, "
      "int rank, bool full_nvlink) -> int");
  m.impl("init_custom_ar", torch::kCUDA, &init_custom_ar);

  m.def(
      "all_reduce(int fa, Tensor inp, Tensor! out, int reg_buffer, "
      "int reg_buffer_sz_bytes, int num_ctas) -> ()");
  m.impl("all_reduce", torch::kCUDA, &all_reduce);
}
