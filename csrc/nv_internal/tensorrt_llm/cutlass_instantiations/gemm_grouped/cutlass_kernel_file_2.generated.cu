#include "hip/hip_runtime.h"
#include "tensorrt_llm/kernels/internal_cutlass_kernels/src/moe_gemm/launchers/moe_gemm_tma_ws_launcher.inl"
namespace tensorrt_llm {
namespace kernels {
namespace cutlass_kernels {

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128, 64,
                                          64, 2, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          128, 64, 2, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          128, 64, 2, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          256, 64, 2, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          256, 64, 2, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          512, 64, 2, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128, 32,
                                          64, 1, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128, 64,
                                          64, 1, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128, 64,
                                          64, 1, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          128, 64, 1, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          128, 64, 1, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          256, 64, 1, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          256, 64, 1, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, half, half, half, EpilogueOpDefault, NONE, 128,
                                          512, 64, 1, 2, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 64, 64, 2, 1, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 64, 2, 2, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 64, 2, 1, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 64, 2, 2, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 64, 2, 1, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 512, 64, 2, 2, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 32, 64, 1, 1, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 64, 64, 1, 2, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 64, 64, 1, 1, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 64, 1, 2, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 64, 1, 1, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 64, 1, 2, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 64, 1, 1, 1, false);

#endif

#if defined(ENABLE_BF16)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 512, 64, 1, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          64, 32, 2, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          128, 32, 2, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          128, 32, 2, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          256, 32, 2, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          256, 32, 2, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          512, 32, 2, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          32, 32, 1, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          64, 32, 1, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          64, 32, 1, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          128, 32, 1, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          128, 32, 1, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          256, 32, 1, 2, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          256, 32, 1, 1, 1, false);

#endif

#if 1

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, float, float, float, EpilogueOpDefault, NONE, 128,
                                          512, 32, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 64, 128, 2, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 64, 128, 2, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 2, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 2, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 2, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 2, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 2, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 2, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 2, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 2, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 512, 128, 2, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 512, 128, 2, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 16, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 16, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 32, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 32, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 64, 128, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 64, 128, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 64, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 64, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                                          EpilogueOpDefault, NONE, 128, 512, 128, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP8)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 512, 128, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, half, EpilogueOpDefault, NONE,
                                          128, 64, 256, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, __hip_bfloat16, EpilogueOpDefault,
                                          NONE, 128, 64, 256, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, half, EpilogueOpDefault, NONE,
                                          128, 128, 256, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, __hip_bfloat16, EpilogueOpDefault,
                                          NONE, 128, 128, 256, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, half, EpilogueOpDefault, NONE,
                                          128, 128, 256, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, __hip_bfloat16, EpilogueOpDefault,
                                          NONE, 128, 128, 256, 1, 1, 1, false);

#endif

#if defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, half, EpilogueOpDefault, NONE,
                                          128, 256, 256, 1, 2, 1, false);

#endif

#if defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, __hip_bfloat16, EpilogueOpDefault,
                                          NONE, 128, 256, 256, 1, 2, 1, false);

#endif

}  // namespace cutlass_kernels
}  // namespace kernels
}  // namespace tensorrt_llm
