#include "hip/hip_runtime.h"
#include "tensorrt_llm/kernels/cutlass_kernels/moe_gemm/launchers/moe_gemm_tma_ws_launcher.inl"
namespace tensorrt_llm {
namespace kernels {
namespace cutlass_kernels {

#if defined(ENABLE_FP4) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, half, EpilogueOpDefault, NONE,
                                          128, 64, 256, 1, 1, 1, false, false);

#endif

#if defined(ENABLE_FP4) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, __hip_bfloat16, EpilogueOpDefault,
                                          NONE, 128, 64, 256, 1, 1, 1, false, false);

#endif

#if defined(ENABLE_FP4) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, half, EpilogueOpDefault, NONE,
                                          128, 128, 256, 1, 2, 1, false, false);

#endif

#if defined(ENABLE_FP4) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, __hip_bfloat16, EpilogueOpDefault,
                                          NONE, 128, 128, 256, 1, 2, 1, false, false);

#endif

#if defined(ENABLE_FP4) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, half, EpilogueOpDefault, NONE,
                                          128, 128, 256, 1, 1, 1, false, false);

#endif

#if defined(ENABLE_FP4) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, __hip_bfloat16, EpilogueOpDefault,
                                          NONE, 128, 128, 256, 1, 1, 1, false, false);

#endif

#if defined(ENABLE_FP4) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, half, EpilogueOpDefault, NONE,
                                          128, 256, 256, 1, 2, 1, false, false);

#endif

#if defined(ENABLE_FP4) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, SafeFP4, SafeFP4, __hip_bfloat16, EpilogueOpDefault,
                                          NONE, 128, 256, 256, 1, 2, 1, false, false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, half, EpilogueOpDefault,
                                          NONE, 128, 64, 128, 1, 1, 1, true, false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 64, 128, 1, 1, 1, true,
                                          false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, half, EpilogueOpDefault,
                                          NONE, 128, 128, 128, 1, 2, 1, true, false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 1, 2, 1, true,
                                          false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, half, EpilogueOpDefault,
                                          NONE, 128, 128, 128, 1, 1, 1, true, false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 128, 128, 1, 1, 1, true,
                                          false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, half, EpilogueOpDefault,
                                          NONE, 128, 256, 128, 1, 2, 1, true, false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 1, 2, 1, true,
                                          false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, half, EpilogueOpDefault,
                                          NONE, 128, 256, 128, 1, 1, 1, true, false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, half, EpilogueOpDefault,
                                          NONE, 256, 256, 128, 2, 1, 1, true, false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 128, 256, 128, 1, 1, 1, true,
                                          false);

#endif

#if defined(ENABLE_FP8) && defined(ENABLE_FP4)

INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm100, __hip_fp8_e4m3_fnuz, SafeFP4, __hip_bfloat16,
                                          EpilogueOpDefault, NONE, 256, 256, 128, 2, 1, 1, true,
                                          false);

#endif

}  // namespace cutlass_kernels
}  // namespace kernels
}  // namespace tensorrt_llm
