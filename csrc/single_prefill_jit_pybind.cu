/*
 * Copyright (c) 2023-2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "pytorch_extension_utils.h"
#include "single_prefill_config.inc"

void single_prefill_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                  at::Tensor o, std::optional<at::Tensor> maybe_lse,
                                  int64_t mask_mode_code, int64_t layout,
                                  int64_t window_left ADDITIONAL_FUNC_PARAMS, int64_t hip_stream);

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  // Single-request prefill attention with KV-Cache operator
  m.def("run", single_prefill_with_kv_cache);
}
