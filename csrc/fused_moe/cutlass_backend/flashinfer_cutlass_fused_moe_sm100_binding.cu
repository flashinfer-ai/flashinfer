#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#if defined(USING_OSS_CUTLASS_MOE_GEMM)
#include "moe_kernels.h"
#else
#include "moe_gemm_kernels.h"
#include "moe_kernels.h"
#endif
// Always include the public header for moe_gemm_kernels.h

#include <tvm/ffi/extra/module.h>

#include "../../tvm_ffi_utils.h"
#include "cutlass_kernel_selector.h"
#include "moe_gemm_kernels.h"
#include "tensorrt_llm/common/workspace.h"
#include "tensorrt_llm/kernels/cutlass_kernels/fp8_blockscale_gemm/fp8_blockscale_gemm.h"

namespace common = tensorrt_llm::common;
namespace kernels = CUTLASS_MOE_GEMM_KERNELS_NAMESPACE;
using ActivationParams = CUTLASS_MOE_GEMM_NAMESPACE::ActivationParams;
using ActivationType = CUTLASS_MOE_GEMM_NAMESPACE::ActivationType;
// Always use public header as it is just utility functions and types
using TmaWarpSpecializedGroupedGemmInput =
    tensorrt_llm::kernels::cutlass_kernels::TmaWarpSpecializedGroupedGemmInput;
using profiler_backend = CUTLASS_MOE_GEMM_KERNELS_NAMESPACE::GemmProfilerBackend;

using tvm::ffi::Array;
using tvm::ffi::DLDataTypeToString;
using tvm::ffi::Function;
using tvm::ffi::Optional;
constexpr DLDataType dl_uint4x2 = DLDataType{kDLUInt, 4, 2};

class DtypeUtils {
 public:
  static nvinfer1::DataType dataType(DLDataType dtype) {
    switch (encode_dlpack_dtype(dtype)) {
      case float32_code:
        return nvinfer1::DataType::kFLOAT;
      case float16_code:
        return nvinfer1::DataType::kHALF;
      case encode_dlpack_dtype(dl_int8):
        return nvinfer1::DataType::kINT8;
      case encode_dlpack_dtype(dl_uint8):
        return nvinfer1::DataType::kUINT8;
      case int32_code:
        return nvinfer1::DataType::kINT32;
      case int64_code:
        return nvinfer1::DataType::kINT64;
      case encode_dlpack_dtype(dl_bool):
        return nvinfer1::DataType::kBOOL;
      case float8_e4m3fn_code:
        return nvinfer1::DataType::kFP8;
      case bfloat16_code:
        return nvinfer1::DataType::kBF16;
      case encode_dlpack_dtype(dl_uint4x2):
        return nvinfer1::DataType::kINT4;
      default:
        TVM_FFI_ICHECK(false) << "unsupported data type";
    }
  }

 private:
  DtypeUtils() = default;
};

class FusedMoeRunner : public tvm::ffi::ModuleObj {
 public:
  template <typename TypeAct, typename TypeWeight, bool NeedQuant = false>
  std::unique_ptr<kernels::CutlassMoeFCRunnerInterface> switch_output_type(DLDataType output_type) {
    switch (encode_dlpack_dtype(output_type)) {
      case int64_code:  // INT64 == FP4
      case float8_e4m3fn_code:
        // TODO We need an atomic FP8 reduction for the finalize fusions
        TVM_FFI_LOG_AND_THROW(NotImplementedError)
            << "Outputting " << DLDataTypeToString(output_type)
            << " directly is not currently supported";
        // return std::make_unique<kernels::CutlassMoeFCRunner<Type, Type>>();
      case float16_code:
        if constexpr (NeedQuant) {
          return std::make_unique<kernels::CutlassMoeFCRunner<TypeAct, TypeWeight, half, half>>();
        } else {
          return std::make_unique<
              kernels::CutlassMoeFCRunner<TypeAct, TypeWeight, half, TypeAct>>();
        }
#ifdef ENABLE_BF16
      case bfloat16_code:
        if constexpr (NeedQuant) {
          return std::make_unique<
              kernels::CutlassMoeFCRunner<TypeAct, TypeWeight, __hip_bfloat16, __hip_bfloat16>>();
        } else {
          return std::make_unique<
              kernels::CutlassMoeFCRunner<TypeAct, TypeWeight, __hip_bfloat16, TypeAct>>();
        }
#endif
      default:
        TVM_FFI_ICHECK(false) << "Invalid output type " << DLDataTypeToString(output_type)
                              << " specified for " << DLDataTypeToString(mActivationDtype);
    }
  };

  FusedMoeRunner(DLDataType activation_dtype, DLDataType weight_dtype, DLDataType output_dtype,
                 bool use_deepseek_fp8_block_scale, bool use_w4_group_scaling,
                 bool use_mxfp8_act_scaling) {
    mActivationDtype = activation_dtype;
    mWeightDtype = weight_dtype;
    mOutputDtype = output_dtype;
    mUseDeepSeekFP8BlockScaling = use_deepseek_fp8_block_scale;
    mUseW4GroupScaling = use_w4_group_scaling;
    mUseMxfp8ActScaling = use_mxfp8_act_scaling;
    mInnerDimMultiplier = 1;

    // keep consistent with cpp/tensorrt_llm/plugins/mixtureOfExperts/mixtureOfExpertsPlugin.cpp
    if (mActivationDtype == dl_float16 && mWeightDtype == dl_float16) {
      mKernelRunner = std::make_shared<kernels::CutlassMoeFCRunner<half, half>>();
    }
#ifdef ENABLE_BF16
    else if (mActivationDtype == dl_bfloat16 && mWeightDtype == dl_bfloat16) {
      mKernelRunner = std::make_shared<kernels::CutlassMoeFCRunner<__hip_bfloat16, __hip_bfloat16>>();
    }
#ifdef ENABLE_FP8
    else if (mActivationDtype == dl_bfloat16 && mWeightDtype == dl_float8_e4m3fn) {
      mKernelRunner = std::make_unique<kernels::CutlassMoeFCRunner<__hip_bfloat16, __hip_fp8_e4m3_fnuz>>();
    }
#endif
#endif

#ifdef ENABLE_FP8
    if (isFp8Quant()) {
      mKernelRunner = switch_output_type<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz>(mOutputDtype);
    }
#endif
#ifdef ENABLE_FP4
    if (isWMxfp4AMxfp8Quant() || isWMxfp4AFp8Quant()) {
      mInnerDimMultiplier = 16;  // 16 FP4 -> 1 LONG
      mKernelRunner = switch_output_type<__hip_fp8_e4m3_fnuz, __hip_fp4_e2m1>(mOutputDtype);
    }

    if (isNvfp4Quant()) {
      mInnerDimMultiplier = 16;
      switch (encode_dlpack_dtype(mActivationDtype)) {
        case float16_code:
#ifdef ENABLE_BF16
        case bfloat16_code:
#endif
          mKernelRunner = switch_output_type<__hip_fp4_e2m1, __hip_fp4_e2m1, true>(mOutputDtype);
          break;
        default:
          mKernelRunner = switch_output_type<__hip_fp4_e2m1, __hip_fp4_e2m1, false>(mOutputDtype);
      }
    }

    if (isWFP4A16Quant()) {
      mInnerDimMultiplier = 2;
      if (mActivationDtype == dl_float16) {
        mKernelRunner = std::make_shared<kernels::CutlassMoeFCRunner<half, __hip_fp4_e2m1>>();
      }
#ifdef ENABLE_BF16
      else if (mActivationDtype == dl_bfloat16) {
        mKernelRunner =
            std::make_shared<kernels::CutlassMoeFCRunner<__hip_bfloat16, __hip_fp4_e2m1>>();
      }
#endif
    }

#endif
    if (isInt4Quant()) {
      mInnerDimMultiplier = 2;
      if (mActivationDtype == dl_float16) {
#ifdef ENABLE_FP8
        if (mUseW4GroupScaling) {
          mKernelRunner = std::make_unique<
              kernels::CutlassMoeFCRunner<__hip_fp8_e4m3_fnuz, cutlass::uint4b_t, half, half>>();
        } else {
          mKernelRunner = std::make_shared<kernels::CutlassMoeFCRunner<half, cutlass::uint4b_t>>();
        }
#else
        mKernelRunner = std::make_shared<kernels::CutlassMoeFCRunner<half, cutlass::uint4b_t>>();
#endif
      }
#ifdef ENABLE_BF16
      else if (mActivationDtype == dl_bfloat16) {
#ifdef ENABLE_FP8
        if (mUseW4GroupScaling) {
          mKernelRunner =
              std::make_unique<kernels::CutlassMoeFCRunner<__hip_fp8_e4m3_fnuz, cutlass::uint4b_t,
                                                           __hip_bfloat16, __hip_bfloat16>>();
        } else {
          mKernelRunner =
              std::make_shared<kernels::CutlassMoeFCRunner<__hip_bfloat16, cutlass::uint4b_t>>();
        }
#else
        mKernelRunner =
            std::make_shared<kernels::CutlassMoeFCRunner<__hip_bfloat16, cutlass::uint4b_t>>();
#endif
      }
#endif
    }
    if (!mKernelRunner) {
      TVM_FFI_ICHECK(false)
          << "Could not construct fused moe op with the requested input combination Activation: "
          << DLDataTypeToString(mActivationDtype)
          << ", Weight: " << DLDataTypeToString(mWeightDtype)
          << ", Output: " << DLDataTypeToString(mOutputDtype);
    }

    mProfiler = std::make_shared<kernels::GemmProfilerBackend>();
    mAllProfiles = mKernelRunner->getTactics();
    TVM_FFI_ICHECK(!mAllProfiles.empty())
        << "No valid tactics available for fused moe op with the requested input combination "
           "Activation: "
        << DLDataTypeToString(mActivationDtype) << ", Weight: " << DLDataTypeToString(mWeightDtype)
        << ", Output: " << DLDataTypeToString(mOutputDtype);
  }

  void runMoe(Tensor output, Tensor input, Tensor token_selected_experts,
              Optional<Tensor> token_final_scales, Tensor fc1_expert_weights,
              Optional<Tensor> fc1_expert_biases, Tensor fc2_expert_weights,
              Optional<Tensor> fc2_expert_biases, Optional<Array<Tensor>> quant_scales,
              Optional<Tensor> input_sf, Optional<Tensor> swiglu_alpha,
              Optional<Tensor> swiglu_beta, Optional<Tensor> swiglu_limit, int64_t tp_size,
              int64_t tp_rank, int64_t ep_size, int64_t ep_rank, int64_t cluster_size,
              int64_t cluster_rank, bool enable_alltoall, bool min_latency_mode,
              Optional<Array<int64_t>> profile_ids, bool enable_pdl) {
    std::lock_guard<std::mutex> lock(mMutex);

    TVM_FFI_ICHECK(cluster_size == 1 && cluster_rank == 0)
        << "smart_router is supported in min_latency mode";

    CHECK_INPUT_TYPE(input, mActivationDtype)
    CHECK_INPUT_TYPE(token_selected_experts, dl_int32)
    if (token_final_scales) {
      CHECK_INPUT_TYPE(token_final_scales.value(), dl_float32)
    }
    if (mWeightDtype == dl_uint4x2) {
      // Since dlpack does not support uint4x2, here uses uint8 to bypass
      CHECK_INPUT_TYPE(fc1_expert_weights, dl_uint8);
      CHECK_INPUT_TYPE(fc2_expert_weights, dl_uint8)
    } else {
      CHECK_INPUT_TYPE(fc1_expert_weights, mWeightDtype);
      CHECK_INPUT_TYPE(fc2_expert_weights, mWeightDtype)
    }

    CHECK_DIM(2, input);
    CHECK_DIM(2, token_selected_experts);

    CHECK_DIM(3, fc1_expert_weights);
    CHECK_DIM(3, fc2_expert_weights);

    if (fc1_expert_biases.has_value() || fc2_expert_biases.has_value()) {
      CHECK_INPUT_TYPE(fc1_expert_biases.value(), mOutputDtype);
      CHECK_INPUT_TYPE(fc2_expert_biases.value(), mOutputDtype);

      CHECK_DIM(2, fc1_expert_biases.value());
      CHECK_DIM(2, fc2_expert_biases.value());
      TVM_FFI_ICHECK_EQ(fc1_expert_weights->shape[0], fc1_expert_biases.value()->shape[0])
          << "fc1_expert_weights and fc1_expert_biases must have the same number of experts.";
      TVM_FFI_ICHECK_EQ(fc2_expert_weights->shape[0], fc2_expert_biases.value()->shape[0])
          << "fc2_expert_weights and fc2_expert_biases must have the same number of experts.";
      TVM_FFI_ICHECK_EQ(fc1_expert_biases.value()->shape[1], fc1_expert_weights->shape[1])
          << "fc1_expert_biases should match fc1_expert_weights output shape.";
      TVM_FFI_ICHECK_EQ(fc2_expert_biases.value()->shape[1], fc2_expert_weights->shape[1])
          << "fc2_expert_biases should match fc2_expert_weights output shape.";
    }

    TVM_FFI_ICHECK_EQ(input->shape[0], token_selected_experts->shape[0])
        << "input and token_selected_experts must have the same num tokens.";
    if (token_final_scales.has_value()) {
      CHECK_DIM(2, token_final_scales.value());
      TVM_FFI_ICHECK_EQ(input->shape[0], token_final_scales.value()->shape[0])
          << "input and token_selected_experts_probs must have the same num tokens.";
      TVM_FFI_ICHECK_EQ(token_selected_experts->shape[1], token_final_scales.value()->shape[1])
          << "token_selected_experts and token_final_scales must have the same number of "
             "experts per token.";
    }
    TVM_FFI_ICHECK_EQ(fc1_expert_weights->shape[0], fc2_expert_weights->shape[0])
        << "fc1_expert_weights and fc2_expert_weights must have the same number of experts.";
    TVM_FFI_ICHECK_EQ(fc1_expert_weights->shape[1],
                      fc2_expert_weights->shape[2] * mInnerDimMultiplier * 2)
        << "fc1_expert_weights inter size must be 2 times fc2_expert_weights inter size.";

    int experts_per_token = token_selected_experts->shape[1];
    int64_t num_rows = input->shape[0];
    int64_t hidden_size = fc2_expert_weights->shape[1];
    int64_t inter_size = fc2_expert_weights->shape[2] * mInnerDimMultiplier;

    if (isWMxfp4AMxfp8Quant() || isWMxfp4AFp8Quant()) {
      // MXFP4 weights are required to bealigned to 128 bytes
      TVM_FFI_ICHECK_EQ(hidden_size % 128, 0)
          << "hidden_size must be divisible by 128 for MXFP4 weights";
      TVM_FFI_ICHECK_EQ(inter_size % 128, 0)
          << "inter_size must be divisible by 128 for MXFP4 weights";
    } else {
      // TMA requires at least 128 bit alignment
      auto min_alignment = 128 / (8 * std::min(mActivationDtype.bits * mActivationDtype.lanes / 8,
                                               mWeightDtype.bits * mWeightDtype.lanes / 8));
      TVM_FFI_ICHECK_EQ(hidden_size % min_alignment, 0)
          << "hidden_size " << hidden_size << " must be divisible by " << min_alignment
          << " for weights";
      TVM_FFI_ICHECK_EQ(inter_size % min_alignment, 0)
          << "inter_size " << inter_size << " must be divisible by " << min_alignment
          << " for weights";
    }

    int const num_experts_on_rank = fc2_expert_weights->shape[0];
    auto const num_experts_total = static_cast<int>(num_experts_on_rank * ep_size);
    auto parallelism_config = kernels::MOEParallelismConfig(tp_size, tp_rank, ep_size, ep_rank);
    ActivationType base_activation_type = ActivationType::Swiglu;
    if (swiglu_alpha.has_value()) {
      CHECK_INPUT_AND_TYPE(swiglu_alpha.value(), dl_float32);
      TVM_FFI_ICHECK_EQ(swiglu_alpha.value()->shape[0], num_experts_on_rank)
          << "swiglu_alpha must have num_experts_on_rank elements.";
      base_activation_type = ActivationType::SwigluBias;
    }
    if (swiglu_beta.has_value()) {
      CHECK_INPUT_AND_TYPE(swiglu_beta.value(), dl_float32);
      TVM_FFI_ICHECK_EQ(swiglu_beta.value()->shape[0], num_experts_on_rank)
          << "swiglu_beta must have num_experts_on_rank elements.";
      base_activation_type = ActivationType::SwigluBias;
    }
    if (swiglu_limit.has_value()) {
      CHECK_INPUT_AND_TYPE(swiglu_limit.value(), dl_float32);
      TVM_FFI_ICHECK_EQ(swiglu_limit.value()->shape[0], num_experts_on_rank)
          << "swiglu_limit must have num_experts_on_rank elements.";
      base_activation_type = ActivationType::SwigluBias;
    }
    auto activation_params = ActivationParams(
        base_activation_type,
        reinterpret_cast<float const*>(swiglu_alpha.has_value() ? swiglu_alpha.value()->data
                                                                : nullptr),
        reinterpret_cast<float const*>(swiglu_beta.has_value() ? swiglu_beta.value()->data
                                                               : nullptr),
        reinterpret_cast<float const*>(swiglu_limit.has_value() ? swiglu_limit.value()->data
                                                                : nullptr));

    setRunnerProfiles(profile_ids);

    auto stream = get_stream(input->device);

    WorkspaceInfo workspace_info = getWorkspaceInfo(
        num_rows, hidden_size, inter_size, num_experts_total, static_cast<int>(experts_per_token),
        base_activation_type, parallelism_config, min_latency_mode);

    auto const quant_params =
        getQuantParams(num_experts_on_rank, hidden_size, inter_size, quant_scales);
    kernels::MoeMinLatencyParams min_latency_params{};

    // TODO: support lora in the future
    ::tensorrt_llm::kernels::LoraParams lora_params{};
#ifdef USING_OSS_CUTLASS_MOE_GEMM
    mKernelRunner->runMoe(input->data, input_sf.has_value() ? input_sf.value()->data : nullptr,
                          reinterpret_cast<int const*>(token_selected_experts->data),
                          token_final_scales.has_value()
                              ? reinterpret_cast<float const*>(token_final_scales.value()->data)
                              : nullptr,
                          fc1_expert_weights->data,
                          fc1_expert_biases.has_value() ? fc1_expert_biases.value()->data : nullptr,
                          activation_params, fc2_expert_weights->data,
                          fc2_expert_biases.has_value() ? fc2_expert_biases.value()->data : nullptr,
                          quant_params, num_rows, hidden_size, inter_size, num_experts_total,
                          static_cast<int>(experts_per_token),
                          static_cast<char*>(workspace_info.workspace->data), output->data,
                          static_cast<int*>(workspace_info.src_to_dest_map), parallelism_config,
                          enable_alltoall, false, lora_params, mUseDeepSeekFP8BlockScaling,
                          min_latency_mode, min_latency_params, enable_pdl, stream);
#else
    mKernelRunner->runMoe(
        input->data, input_sf.has_value() ? input_sf.value()->data : nullptr,
        reinterpret_cast<int const*>(token_selected_experts->data),
        token_final_scales.has_value()
            ? reinterpret_cast<float const*>(token_final_scales.value()->data)
            : nullptr,
        fc1_expert_weights->data,
        fc1_expert_biases.has_value() ? fc1_expert_biases.value()->data : nullptr,
        activation_params, fc2_expert_weights->data,
        fc2_expert_biases.has_value() ? fc2_expert_biases.value()->data : nullptr, quant_params,
        num_rows, hidden_size, inter_size, num_experts_total, static_cast<int>(experts_per_token),
        static_cast<char*>(workspace_info.workspace), output->data,
        static_cast<int*>(workspace_info.src_to_dest_map), parallelism_config, false, lora_params,
        mUseDeepSeekFP8BlockScaling, min_latency_mode, min_latency_params, enable_pdl, stream);
#endif
  }

  void runMoeMinLantency(Tensor output, Tensor input, Tensor token_selected_experts,
                         Optional<Tensor> token_final_scales, Tensor fc1_expert_weights,
                         Optional<Tensor> fc1_expert_biases, Tensor fc2_expert_weights,
                         Optional<Tensor> fc2_expert_biases, Optional<Array<Tensor>> quant_scales,
                         Optional<Tensor> input_sf, Optional<Tensor> swiglu_alpha,
                         Optional<Tensor> swiglu_beta, Optional<Tensor> swiglu_limit,
                         Tensor num_active_experts_per_node, Tensor experts_to_token_score,
                         Tensor active_expert_global_ids, int64_t tp_size, int64_t tp_rank,
                         int64_t ep_size, int64_t ep_rank, int64_t cluster_size,
                         int64_t cluster_rank, bool enable_alltoall, bool min_latency_mode,
                         Optional<Array<int64_t>> profile_ids, bool enable_pdl) {
    std::lock_guard<std::mutex> lock(mMutex);

    CHECK_INPUT_TYPE(input, mActivationDtype)
    CHECK_INPUT_TYPE(token_selected_experts, dl_int32)
    if (token_final_scales) {
      CHECK_INPUT_TYPE(token_final_scales.value(), dl_float32)
    }
    if (mWeightDtype == dl_uint4x2) {
      // Since dlpack does not support uint4x2, here uses uint8 to bypass
      CHECK_INPUT_TYPE(fc1_expert_weights, dl_uint8);
      CHECK_INPUT_TYPE(fc2_expert_weights, dl_uint8)
    } else {
      CHECK_INPUT_TYPE(fc1_expert_weights, mWeightDtype);
      CHECK_INPUT_TYPE(fc2_expert_weights, mWeightDtype)
    }

    CHECK_DIM(2, input);
    CHECK_DIM(2, token_selected_experts);

    CHECK_DIM(3, fc1_expert_weights);
    CHECK_DIM(3, fc2_expert_weights);

    if (fc1_expert_biases.has_value() || fc2_expert_biases.has_value()) {
      CHECK_INPUT_TYPE(fc1_expert_biases.value(), mOutputDtype);
      CHECK_INPUT_TYPE(fc2_expert_biases.value(), mOutputDtype);
      CHECK_DIM(2, fc1_expert_biases.value());
      CHECK_DIM(2, fc2_expert_biases.value());
      TVM_FFI_ICHECK_EQ(fc1_expert_weights->shape[0], fc1_expert_biases.value()->shape[0])
          << "fc1_expert_weights and fc1_expert_biases must have the same number of experts.";
      TVM_FFI_ICHECK_EQ(fc2_expert_weights->shape[0], fc2_expert_biases.value()->shape[0])
          << "fc2_expert_weights and fc2_expert_biases must have the same number of experts.";
      TVM_FFI_ICHECK_EQ(fc1_expert_biases.value()->shape[1], fc1_expert_weights->shape[1])
          << "fc1_expert_biases should match fc1_expert_weights output shape.";
      TVM_FFI_ICHECK_EQ(fc2_expert_biases.value()->shape[1], fc2_expert_weights->shape[1])
          << "fc2_expert_biases should match fc2_expert_weights output shape.";
    }

    TVM_FFI_ICHECK_EQ(input->shape[0], token_selected_experts->shape[0])
        << "input and token_selected_experts must have the same num tokens.";
    if (token_final_scales) {
      CHECK_DIM(2, token_final_scales.value());
      TVM_FFI_ICHECK_EQ(input->shape[0], token_final_scales.value()->shape[0])
          << "input and token_selected_experts_probs must have the same num tokens.";
      TVM_FFI_ICHECK_EQ(token_selected_experts->shape[1], token_final_scales.value()->shape[1])
          << "token_selected_experts and token_final_scales must have the same number of "
             "experts per token.";
    }
    TVM_FFI_ICHECK_EQ(fc1_expert_weights->shape[0], fc2_expert_weights->shape[0])
        << "fc1_expert_weights and fc2_expert_weights must have the same number of experts.";
    TVM_FFI_ICHECK_EQ(fc1_expert_weights->shape[1],
                      fc2_expert_weights->shape[2] * mInnerDimMultiplier * 2)
        << "fc1_expert_weights inter size must be 2 times fc2_expert_weights inter size.";

    TVM_FFI_ICHECK(!input_sf.has_value() || isWMxfp4AMxfp8Quant() || isNvfp4Quant())
        << "Block-scaling factors provided for non block-scaling quantization";

    int experts_per_token = token_selected_experts->shape[1];
    int64_t num_rows = input->shape[0];
    int64_t hidden_size = fc2_expert_weights->shape[1];
    int64_t inter_size = fc2_expert_weights->shape[2] * mInnerDimMultiplier;

    int const num_experts_on_rank = fc2_expert_weights->shape[0];
    auto const num_experts_total = static_cast<int>(num_experts_on_rank * ep_size);
    auto parallelism_config = kernels::MOEParallelismConfig(tp_size, tp_rank, ep_size, ep_rank);
    ActivationType base_activation_type = ActivationType::Swiglu;
    if (swiglu_alpha.has_value()) {
      CHECK_INPUT_AND_TYPE(swiglu_alpha.value(), dl_float32);
      TVM_FFI_ICHECK_EQ(swiglu_alpha.value()->shape[0], num_experts_on_rank)
          << "swiglu_alpha must have num_experts_on_rank elements.";
      base_activation_type = ActivationType::SwigluBias;
    }
    if (swiglu_beta.has_value()) {
      CHECK_INPUT_AND_TYPE(swiglu_beta.value(), dl_float32);
      TVM_FFI_ICHECK_EQ(swiglu_beta.value()->shape[0], num_experts_on_rank)
      "swiglu_beta must have num_experts_on_rank elements.";
      base_activation_type = ActivationType::SwigluBias;
    }
    if (swiglu_limit.has_value()) {
      CHECK_INPUT_AND_TYPE(swiglu_limit.value(), dl_float32);
      TVM_FFI_ICHECK_EQ(swiglu_limit.value()->shape[0], num_experts_on_rank)
          << "swiglu_limit must have num_experts_on_rank elements.";
      base_activation_type = ActivationType::SwigluBias;
    }
    auto activation_params = ActivationParams(
        base_activation_type,
        reinterpret_cast<float const*>(swiglu_alpha.has_value() ? swiglu_alpha.value()->data
                                                                : nullptr),
        reinterpret_cast<float const*>(swiglu_beta.has_value() ? swiglu_beta.value()->data
                                                               : nullptr),
        reinterpret_cast<float const*>(swiglu_limit.has_value() ? swiglu_limit.value()->data
                                                                : nullptr));

    setRunnerProfiles(profile_ids);

    auto stream = get_stream(input->device);

    CHECK_DIM(1, num_active_experts_per_node);
    CHECK_INPUT_TYPE(num_active_experts_per_node, dl_int32);
    TVM_FFI_ICHECK_EQ(num_active_experts_per_node->shape[0], 1);

    CHECK_DIM(2, experts_to_token_score);
    CHECK_INPUT_TYPE(experts_to_token_score, dl_float32);
    TVM_FFI_ICHECK_EQ(experts_to_token_score->shape[0], num_experts_on_rank);
    TVM_FFI_ICHECK_EQ(experts_to_token_score->shape[1], num_rows);

    CHECK_DIM(1, active_expert_global_ids);
    CHECK_INPUT_TYPE(active_expert_global_ids, dl_int32);
    TVM_FFI_ICHECK_EQ(active_expert_global_ids->shape[0], num_experts_on_rank);

    kernels::MoeMinLatencyParams min_latency_params{};
    min_latency_params.num_active_experts_per_node =
        static_cast<int*>(num_active_experts_per_node->data);
    min_latency_params.experts_to_token_score = static_cast<float*>(experts_to_token_score->data);
    min_latency_params.active_expert_global_ids = static_cast<int*>(active_expert_global_ids->data);

    WorkspaceInfo workspace_info = getWorkspaceInfo(
        num_rows, hidden_size, inter_size, num_experts_total, static_cast<int>(experts_per_token),
        base_activation_type, parallelism_config, min_latency_mode);

    auto const quant_params =
        getQuantParams(num_experts_on_rank, hidden_size, inter_size, quant_scales);

    // TODO: support lora in the future
    ::tensorrt_llm::kernels::LoraParams lora_params{};
#ifdef USING_OSS_CUTLASS_MOE_GEMM
    mKernelRunner->runMoe(input->data, input_sf.has_value() ? input_sf.value()->data : nullptr,
                          reinterpret_cast<int const*>(token_selected_experts->data),
                          token_final_scales.has_value()
                              ? reinterpret_cast<float const*>(token_final_scales.value()->data)
                              : nullptr,
                          fc1_expert_weights->data,
                          fc1_expert_biases.has_value() ? fc1_expert_biases.value()->data : nullptr,
                          activation_params, fc2_expert_weights->data,
                          fc2_expert_biases.has_value() ? fc2_expert_biases.value()->data : nullptr,
                          quant_params, num_rows, hidden_size, inter_size, num_experts_total,
                          static_cast<int>(experts_per_token),
                          static_cast<char*>(workspace_info.workspace->data), output->data,
                          static_cast<int*>(workspace_info.src_to_dest_map), parallelism_config,
                          enable_alltoall, false, lora_params, mUseDeepSeekFP8BlockScaling,
                          min_latency_mode, min_latency_params, enable_pdl, stream);
#else
    mKernelRunner->runMoe(
        input->data, input_sf.has_value() ? input_sf.value()->data : nullptr,
        reinterpret_cast<int const*>(token_selected_experts->data),
        token_final_scales.has_value()
            ? reinterpret_cast<float const*>(token_final_scales.value()->data)
            : nullptr,
        fc1_expert_weights->data,
        fc1_expert_biases.has_value() ? fc1_expert_biases.value()->data : nullptr,
        activation_params, fc2_expert_weights->data,
        fc2_expert_biases.has_value() ? fc2_expert_biases.value()->data : nullptr, quant_params,
        num_rows, hidden_size, inter_size, num_experts_total, static_cast<int>(experts_per_token),
        static_cast<char*>(workspace_info.workspace), output->data,
        static_cast<int*>(workspace_info.src_to_dest_map), parallelism_config, false, lora_params,
        mUseDeepSeekFP8BlockScaling, min_latency_mode, min_latency_params, enable_pdl, stream);
#endif
  }

  int64_t getTacticNum() {
    std::lock_guard<std::mutex> lock(mMutex);
    return mAllProfiles.size();
  }

  void runGemmProfile(Tensor input, Tensor fc1_expert_weights, Optional<Tensor> fc1_expert_biases,
                      Tensor fc2_expert_weights, Optional<Tensor> fc2_expert_biases, int64_t top_k,
                      int64_t tp_size, int64_t tp_rank, int64_t ep_size, int64_t ep_rank,
                      int64_t cluster_size, int64_t cluster_rank, bool enable_alltoall,
                      bool min_latency_mode, int64_t gemm_idx, int64_t profile_id,
                      bool do_preparation, bool enable_pdl) {
    std::lock_guard<std::mutex> lock(mMutex);

    // TODO: support profiling under fp8 block scaling in the future
    if (mUseDeepSeekFP8BlockScaling) {
      return;
    }

    int64_t num_rows = input->shape[0];
    int64_t hidden_size = fc2_expert_weights->shape[1];
    int64_t inter_size = fc2_expert_weights->shape[2] * mInnerDimMultiplier;
    int64_t group_size_ =
        isInt4Quant() ? TmaWarpSpecializedGroupedGemmInput::INT4GroupwiseParams::int4_group_size
                      : -1;
    int64_t group_size =
        isWFP4A16Quant()
            ? TmaWarpSpecializedGroupedGemmInput::INT4GroupwiseParams::wfp4a16_group_size
            : group_size_;
    int const num_experts = static_cast<int>(fc2_expert_weights->shape[0] * ep_size);

    // Get specific profile configs according to the profile_id.
    // Fallback tactic is set to be 0
    // TODO: use the best tactic id found offline for a better default inference perf
    auto profile = profile_id == -1 ? mAllProfiles.front() : mAllProfiles[profile_id];

    auto stream = get_stream(input->device);

    auto const* expert_weights_ptr =
        (gemm_idx == 1) ? fc1_expert_weights->data : fc2_expert_weights->data;

    // Preparation phase, only enabled during autotuning warmup phase.
    if (do_preparation) {
      // Set profiled gemm idx
      mProfiler->mGemmToProfile = (gemm_idx == 1) ? profiler_backend::GemmToProfile::GEMM_1
                                                  : profiler_backend::GemmToProfile::GEMM_2;

      // mProfiler init
      auto parallelism_config = kernels::MOEParallelismConfig(
          static_cast<int>(tp_size), static_cast<int>(tp_rank), static_cast<int>(ep_size),
          static_cast<int>(ep_rank), static_cast<int>(cluster_size),
          static_cast<int>(cluster_rank));

      bool USE_BIAS = fc1_expert_biases.has_value() || fc2_expert_biases.has_value();
      bool USE_LORA = false;
      auto activation_dtype =
          (mUseW4GroupScaling && !isWFP4A16Quant()) ? dl_float8_e4m3fn : mActivationDtype;
      activation_dtype = isNvfp4Quant() ? dl_int64 : activation_dtype;
#ifdef USING_OSS_CUTLASS_MOE_GEMM
      mProfiler->init(*mKernelRunner.get(), mProfiler->mGemmToProfile,
                      DtypeUtils::dataType(activation_dtype), DtypeUtils::dataType(mWeightDtype),
                      DtypeUtils::dataType(mOutputDtype), num_experts, static_cast<int>(top_k),
                      hidden_size, inter_size, group_size, ActivationType::Swiglu, USE_BIAS,
                      USE_LORA, min_latency_mode,
                      /*need_weights*/ false, parallelism_config, enable_alltoall);
#else
      mProfiler->init(*mKernelRunner.get(), mProfiler->mGemmToProfile,
                      DtypeUtils::dataType(activation_dtype), DtypeUtils::dataType(mWeightDtype),
                      DtypeUtils::dataType(mOutputDtype), num_experts, static_cast<int>(top_k),
                      hidden_size, inter_size, group_size, ActivationType::Swiglu, USE_BIAS,
                      USE_LORA, min_latency_mode,
                      /*need_weights*/ false, parallelism_config);
#endif

      size_t profile_workspace_size = mProfiler->getWorkspaceSize(num_rows);
      int device_id;
      hipGetDevice(&device_id);
      mProfileWorkspace = alloc_tensor({static_cast<int64_t>(profile_workspace_size)}, dl_int8,
                                       DLDevice{kDLCUDA, device_id});

      mProfiler->prepare(num_rows, static_cast<char*>(mProfileWorkspace->data), expert_weights_ptr,
                         enable_pdl, stream);
    }

    // Profile specific tactic. Assuming at least one preparation phase has been executed already.
    mProfiler->runProfiler(num_rows, profile, static_cast<char*>(mProfileWorkspace->data),
                           expert_weights_ptr, enable_pdl, stream);
  }

  const char* kind() const final { return "fused_moe_runner"; }
  Optional<Function> GetFunction(const tvm::ffi::String& name) final {
    if (name == "run_gemm_profile") {
      return Function::FromTyped(
          [this](Tensor input, Tensor fc1_expert_weights, Optional<Tensor> fc1_expert_biases,
                 Tensor fc2_expert_weights, Optional<Tensor> fc2_expert_biases, int64_t top_k,
                 int64_t tp_size, int64_t tp_rank, int64_t ep_size, int64_t ep_rank,
                 int64_t cluster_size, int64_t cluster_rank, bool enable_alltoall,
                 bool min_latency_mode, int64_t gemm_idx, int64_t profile_id, bool do_preparation,
                 bool enable_pdl) {
            runGemmProfile(input, fc1_expert_weights, fc1_expert_biases, fc2_expert_weights,
                           fc2_expert_biases, top_k, tp_size, tp_rank, ep_size, ep_rank,
                           cluster_size, cluster_rank, enable_alltoall, min_latency_mode, gemm_idx,
                           profile_id, do_preparation, enable_pdl);
          });
    } else if (name == "get_tactic_num") {
      return Function::FromTyped([this]() -> int64_t { return getTacticNum(); });
    } else if (name == "run_moe") {
      return Function::FromTyped(
          [this](Tensor output, Tensor input, Tensor token_selected_experts,
                 Optional<Tensor> token_final_scales, Tensor fc1_expert_weights,
                 Optional<Tensor> fc1_expert_biases, Tensor fc2_expert_weights,
                 Optional<Tensor> fc2_expert_biases, Optional<Array<Tensor>> quant_scales,
                 Optional<Tensor> input_sf, Optional<Tensor> swiglu_alpha,
                 Optional<Tensor> swiglu_beta, Optional<Tensor> swiglu_limit, int64_t tp_size,
                 int64_t tp_rank, int64_t ep_size, int64_t ep_rank, int64_t cluster_size,
                 int64_t cluster_rank, bool enable_alltoall, bool min_latency_mode,
                 Optional<Array<int64_t>> profile_ids, bool enable_pdl) {
            runMoe(output, input, token_selected_experts, token_final_scales, fc1_expert_weights,
                   fc1_expert_biases, fc2_expert_weights, fc2_expert_biases, quant_scales, input_sf,
                   swiglu_alpha, swiglu_beta, swiglu_limit, tp_size, tp_rank, ep_size, ep_rank,
                   cluster_size, cluster_rank, enable_alltoall, min_latency_mode, profile_ids,
                   enable_pdl);
          });
    } else if (name == "run_moe_min_latency") {
      return Function::FromTyped(
          [this](Tensor output, Tensor input, Tensor token_selected_experts,
                 Optional<Tensor> token_final_scales, Tensor fc1_expert_weights,
                 Optional<Tensor> fc1_expert_biases, Tensor fc2_expert_weights,
                 Optional<Tensor> fc2_expert_biases, Optional<Array<Tensor>> quant_scales,
                 Optional<Tensor> input_sf, Optional<Tensor> swiglu_alpha,
                 Optional<Tensor> swiglu_beta, Optional<Tensor> swiglu_limit,
                 Tensor num_active_experts_per_node, Tensor experts_to_token_score,
                 Tensor active_expert_global_ids, int64_t tp_size, int64_t tp_rank, int64_t ep_size,
                 int64_t ep_rank, int64_t cluster_size, int64_t cluster_rank, bool enable_alltoall,
                 bool min_latency_mode, Optional<Array<int64_t>> profile_ids, bool enable_pdl) {
            runMoeMinLantency(output, input, token_selected_experts, token_final_scales,
                              fc1_expert_weights, fc1_expert_biases, fc2_expert_weights,
                              fc2_expert_biases, quant_scales, input_sf, swiglu_alpha, swiglu_beta,
                              swiglu_limit, num_active_experts_per_node, experts_to_token_score,
                              active_expert_global_ids, tp_size, tp_rank, ep_size, ep_rank,
                              cluster_size, cluster_rank, enable_alltoall, min_latency_mode,
                              profile_ids, enable_pdl);
          });
    } else {
      return Function(nullptr);
    }
  }

 private:
  struct WorkspaceInfo {
    Tensor workspace{};
    void* src_to_dest_map{};
  };

  std::mutex mMutex;
  std::shared_ptr<kernels::CutlassMoeFCRunnerInterface> mKernelRunner;
  std::shared_ptr<kernels::GemmProfilerBackend> mProfiler;
  DLDataType mActivationDtype;
  DLDataType mWeightDtype;
  DLDataType mOutputDtype;
  // number of elements packed into the inner dimension of a matrix
  // e.g. 16 nvfp4 elements are packed into a single int64 element
  int64_t mInnerDimMultiplier;
  Tensor mProfileWorkspace;

  bool mUseDeepSeekFP8BlockScaling = false;
  bool mUseW4GroupScaling = false;
  bool mUseMxfp8ActScaling = false;

  using Profile = tensorrt_llm::cutlass_extensions::CutlassGemmConfig;
  std::vector<Profile> mAllProfiles;

  void setRunnerProfiles(Optional<Array<int64_t>> profile_ids) {
    if (mUseDeepSeekFP8BlockScaling) {
      auto config = tensorrt_llm::cutlass_extensions::CutlassGemmConfig(
          tensorrt_llm::cutlass_extensions::CutlassTileConfigSM90::CtaShape128x16x128B,
          tensorrt_llm::cutlass_extensions::MainloopScheduleType::AUTO,
          tensorrt_llm::cutlass_extensions::EpilogueScheduleType::AUTO,
          tensorrt_llm::cutlass_extensions::ClusterShape::ClusterShape_1x1x1);
      mKernelRunner->setTactic(config, config);
      return;
    }

    auto best_gemm1_profile = mAllProfiles.front();
    auto best_gemm2_profile = mAllProfiles.front();
    if (profile_ids.has_value()) {
      TVM_FFI_ICHECK_EQ(profile_ids.value().size(), 2) << "Expecting 2 profile ids";
      best_gemm1_profile = profile_ids.value()[0] == -1 ? best_gemm1_profile
                                                        : mAllProfiles.at(profile_ids.value()[0]);
      best_gemm2_profile = profile_ids.value()[1] == -1 ? best_gemm2_profile
                                                        : mAllProfiles.at(profile_ids.value()[1]);
    }
    mKernelRunner->setTactic(best_gemm1_profile, best_gemm2_profile);
  }

  WorkspaceInfo getWorkspaceInfo(int64_t num_rows, int64_t hidden_size, int64_t inter_size,
                                 int num_experts, int experts_per_token,
                                 ActivationType activation_type,
                                 kernels::MOEParallelismConfig parallelismConfig,
                                 bool min_latency_mode) {
    size_t moe_workspace_size = mKernelRunner->getWorkspaceSize(
        num_rows, hidden_size, inter_size, num_experts, experts_per_token, activation_type,
        parallelismConfig, /* use_lora */ false, mUseDeepSeekFP8BlockScaling, min_latency_mode,
        mUseW4GroupScaling);
    size_t src_to_dest_map_size = experts_per_token * num_rows * sizeof(int);

    std::vector<size_t> workspaces{moe_workspace_size, src_to_dest_map_size};

    size_t total_workspace_size =
        common::calculateTotalWorkspaceSize(workspaces.data(), workspaces.size());

    WorkspaceInfo info{};
    int device_id;
    hipGetDevice(&device_id);
    info.workspace = alloc_tensor({static_cast<int64_t>(total_workspace_size)}, dl_int8,
                                  DLDevice{kDLCUDA, device_id});
    info.src_to_dest_map =
        common::nextWorkspacePtr(static_cast<int8_t*>(info.workspace->data), moe_workspace_size);

    return info;
  }

  kernels::QuantParams getQuantParams(int64_t num_experts_on_rank, int64_t hidden_size,
                                      int64_t inter_size,
                                      Optional<Array<Tensor>> quant_scales) const {
    if (isFp8Quant()) {
      TVM_FFI_ICHECK(quant_scales.has_value()) << "Expecting quant scales for fp8 quantization";
      TVM_FFI_ICHECK_EQ(quant_scales.value().size(), 4)
          << "Expecting 4 quant scales for fp8 quantization";

      auto const fc1_dequant = quant_scales.value()[0];
      auto const fc2_quant = quant_scales.value()[1];
      auto const fc2_dequant = quant_scales.value()[2];
      auto const fc1_input_dequant = quant_scales.value()[3];

      // Check types
      CHECK_INPUT_TYPE(fc1_dequant, dl_float32);
      CHECK_INPUT_TYPE(fc2_quant, dl_float32);
      CHECK_INPUT_TYPE(fc2_dequant, dl_float32);
      CHECK_INPUT_TYPE(fc1_input_dequant, dl_float32);
      // Check ranks
      CHECK_DIM(1, fc1_dequant);
      TVM_FFI_ICHECK_LE(fc2_quant->ndim, 1) << "fc2 quant must be a scalar or 1-D tensor";
      CHECK_DIM(1, fc2_dequant);
      CHECK_DIM(0, fc1_input_dequant);
      // Check shapes
      TVM_FFI_ICHECK_EQ(fc1_dequant->shape[0], num_experts_on_rank)
          << "fc1 dequant size must be (num_experts_on_rank,)";
      TVM_FFI_ICHECK(fc2_quant->ndim == 0 || fc2_quant->shape[0] == num_experts_on_rank)
          << "fc2 quant must be scalar or (num_experts_on_rank,)";
      TVM_FFI_ICHECK_EQ(fc2_dequant->shape[0], num_experts_on_rank)
          << "fc2 dequant size must be (num_experts_on_rank,)";

      return kernels::QuantParams::FP8(
          static_cast<float const*>(fc1_dequant->data), static_cast<float const*>(fc2_quant->data),
          static_cast<float const*>(fc2_dequant->data),
          /* fp8 output quant scale */ nullptr, static_cast<float const*>(fc1_input_dequant->data),
          fc2_quant->ndim == 1);
    } else if (isWMxfp4AFp8Quant()) {
      TVM_FFI_ICHECK(quant_scales.has_value())
          << "Expecting quant scales for W4A8_MXFP4_MXF8 quantization";
      TVM_FFI_ICHECK_EQ(quant_scales.value().size(), 5)
          << "Expecting 5 quant scales for W4A8_MXFP4_FP8 quantization";

      auto const fc1_weight_block = quant_scales.value()[0];
      auto const fc1_global = quant_scales.value()[1];
      auto const fc2_act_global = quant_scales.value()[2];
      auto const fc2_weight_block = quant_scales.value()[3];
      auto const fc2_global = quant_scales.value()[4];

      // The input for scale fc1_weight_block / fc2_weight_block is packed into INT32
      constexpr int FP8_PER_INT32 = 4;
      // Check types
      CHECK_INPUT_TYPE(fc1_weight_block, dl_int32);
      CHECK_INPUT_TYPE(fc1_global, dl_float32);
      CHECK_INPUT_TYPE(fc2_act_global, dl_float32);
      CHECK_INPUT_TYPE(fc2_weight_block, dl_int32);
      CHECK_INPUT_TYPE(fc2_global, dl_float32);
      // Check ranks
      CHECK_DIM(3, fc1_weight_block);
      CHECK_DIM(1, fc1_global);
      TVM_FFI_ICHECK_LE(fc2_act_global->ndim, 1) << "fc2 act global must be a scalar or 1-D tensor";
      CHECK_DIM(3, fc2_weight_block);
      CHECK_DIM(1, fc2_global);
      // Check shapes
      TVM_FFI_ICHECK(
          fc1_weight_block->shape[0] == num_experts_on_rank &&
          fc1_weight_block->shape[1] ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  inter_size, TmaWarpSpecializedGroupedGemmInput::MinNDimAlignmentMXFPX) *
                  2 &&
          fc1_weight_block->shape[2] * FP8_PER_INT32 *
                  TmaWarpSpecializedGroupedGemmInput::MXFPXBlockScaleVectorSize ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  hidden_size, TmaWarpSpecializedGroupedGemmInput::MinKDimAlignmentMXFPX))
          << "fc1 weight block size must be (num_experts_on_rank, inter_size * 2, hidden_size // 4 "
             "// block_scale_vector_size)";
      TVM_FFI_ICHECK_EQ(fc1_global->shape[0], num_experts_on_rank)
          << "fc1 global size must be (num_experts_on_rank,)";
      TVM_FFI_ICHECK(fc2_act_global->ndim == 0 || fc2_act_global->shape[0] == num_experts_on_rank)
          << "fc2 act global must be scalar or (num_experts_on_rank,)";
      TVM_FFI_ICHECK(
          fc2_weight_block->shape[0] == num_experts_on_rank &&
          fc2_weight_block->shape[1] ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  hidden_size, TmaWarpSpecializedGroupedGemmInput::MinNDimAlignmentMXFPX) &&
          fc2_weight_block->shape[2] * FP8_PER_INT32 *
                  TmaWarpSpecializedGroupedGemmInput::MXFPXBlockScaleVectorSize ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  inter_size, TmaWarpSpecializedGroupedGemmInput::MinKDimAlignmentMXFPX))
          << "fc2 weight block size must be (num_experts_on_rank, hidden_size, inter_size // 4 // "
             "block_scale_vector_size)";
      TVM_FFI_ICHECK_EQ(fc2_global->shape[0], num_experts_on_rank)
          << "fc2 global size must be (num_experts_on_rank,)";

      return kernels::QuantParams::FP8MXFP4(
          nullptr,
          static_cast<TmaWarpSpecializedGroupedGemmInput::ElementSF*>(fc1_weight_block->data),
          static_cast<float const*>(fc1_global->data),
          static_cast<float const*>(fc2_act_global->data),
          static_cast<TmaWarpSpecializedGroupedGemmInput::ElementSF*>(fc2_weight_block->data),
          static_cast<float const*>(fc2_global->data), false, fc2_act_global->ndim == 1);
    } else if (isWMxfp4AMxfp8Quant()) {
#ifdef USING_OSS_CUTLASS_MOE_GEMM
      TVM_FFI_ICHECK(quant_scales.has_value())
          << "Expecting quant scales for W4A8_MXFP4_MXFP8 quantization";
      TVM_FFI_ICHECK_EQ(quant_scales.value().size(), 4)
      "Expecting 4 quant scales for W4A8_MXFP4_MXFP8 quantization";

      Tensor fc1_weight_block = quant_scales.value()[0];
      Tensor fc1_global = quant_scales.value()[1];
      Tensor fc2_weight_block = quant_scales.value()[2];
      Tensor fc2_global = quant_scales.value()[3];

      // The input for scale fc1_weight_block / fc2_weight_block is packed into INT32
      constexpr int FP8_PER_INT32 = 4;
      CHECK_INPUT_TYPE(fc1_weight_block, dl_int32);
      CHECK_INPUT_TYPE(fc1_global, dl_float32);
      CHECK_INPUT_TYPE(fc2_weight_block, dl_int32);
      CHECK_INPUT_TYPE(fc2_global, dl_float32);
      CHECK_DIM(3, fc1_weight_block);
      CHECK_DIM(1, fc1_global);
      CHECK_DIM(3, fc2_weight_block);
      CHECK_DIM(1, fc2_global);
      TVM_FFI_ICHECK(
          fc1_weight_block->shape[0] == num_experts_on_rank &&
          fc1_weight_block->shape[1] ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  inter_size, TmaWarpSpecializedGroupedGemmInput::MinNDimAlignmentMXFPX) *
                  2 &&
          fc1_weight_block->shape[2] * FP8_PER_INT32 *
                  TmaWarpSpecializedGroupedGemmInput::MXFPXBlockScaleVectorSize ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  hidden_size, TmaWarpSpecializedGroupedGemmInput::MinKDimAlignmentMXFPX))
          << "fc1 weight block size must be (num_experts_on_rank, inter_size * 2, hidden_size // 4 "
             "// block_scale_vector_size)";
      TVM_FFI_ICHECK_EQ(fc1_global->shape[0], num_experts_on_rank)
          << "fc1 global size must be (num_experts_on_rank,)";
      TVM_FFI_ICHECK(
          fc2_weight_block->shape[0] == num_experts_on_rank &&
          fc2_weight_block->shape[1] ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  hidden_size, TmaWarpSpecializedGroupedGemmInput::MinNDimAlignmentMXFPX) &&
          fc2_weight_block->shape[2] * FP8_PER_INT32 *
                  TmaWarpSpecializedGroupedGemmInput::MXFPXBlockScaleVectorSize ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  inter_size, TmaWarpSpecializedGroupedGemmInput::MinKDimAlignmentMXFPX))
          << "fc2 weight block size must be (num_experts_on_rank, hidden_size, inter_size // 4 // "
             "block_scale_vector_size)";
      TVM_FFI_ICHECK_EQ(fc2_global->shape[0], num_experts_on_rank)
          << "fc2 global size must be (num_experts_on_rank,)";

      return kernels::QuantParams::MXFP8MXFP4(
          static_cast<TmaWarpSpecializedGroupedGemmInput::ElementSF*>(fc1_weight_block->data),
          static_cast<float const*>(fc1_global->data),
          static_cast<TmaWarpSpecializedGroupedGemmInput::ElementSF*>(fc2_weight_block->data),
          static_cast<float const*>(fc2_global->data));
#else
      TVM_FFI_ICHECK(false)
          << "MXFP8 x MXFP4 quantization is not supported in OSS Cutlass Moe Gemm";
#endif
    }

    else if (isNvfp4Quant()) {
      TVM_FFI_ICHECK(quant_scales.has_value()) << "Expecting quant scales for nvfp4 quantization";
      TVM_FFI_ICHECK_EQ(quant_scales.value().size(), 6)
          << "Expecting 6 quant scales for nvfp4 quantization";

      Tensor fc1_act_global = quant_scales.value()[0];
      Tensor fc1_weight_block = quant_scales.value()[1];
      Tensor fc1_global = quant_scales.value()[2];
      Tensor fc2_act_global = quant_scales.value()[3];
      Tensor fc2_weight_block = quant_scales.value()[4];
      Tensor fc2_global = quant_scales.value()[5];

      // The input for scale fc1_weight_block / fc2_weight_block is packed into INT32
      constexpr int FP8_PER_INT32 = 4;
      // Check types
      CHECK_INPUT_TYPE(fc1_act_global, dl_float32);
      CHECK_INPUT_TYPE(fc1_weight_block, dl_int32);
      CHECK_INPUT_TYPE(fc1_global, dl_float32);
      CHECK_INPUT_TYPE(fc2_act_global, dl_float32);
      CHECK_INPUT_TYPE(fc2_weight_block, dl_int32);
      CHECK_INPUT_TYPE(fc2_global, dl_float32);
      // Check ranks
      TVM_FFI_ICHECK_LE(fc1_act_global->ndim, 1) << "fc1 act global must be a scalar or 1-D tensor";
      CHECK_DIM(3, fc1_weight_block);
      CHECK_DIM(1, fc1_global);
      TVM_FFI_ICHECK_LE(fc2_act_global->ndim, 1) << "fc2 act global must be a scalar or 1-D tensor";
      CHECK_DIM(3, fc2_weight_block);
      CHECK_DIM(1, fc2_global);
      // Check shapes
      TVM_FFI_ICHECK(fc1_act_global->ndim == 0 || fc1_act_global->shape[0] == num_experts_on_rank)
          << "fc1 act global must be scalar or (num_experts_on_rank,)";
      TVM_FFI_ICHECK(
          fc1_weight_block->shape[0] == num_experts_on_rank &&
          fc1_weight_block->shape[1] ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  inter_size, TmaWarpSpecializedGroupedGemmInput::MinKDimAlignmentNVFP4) *
                  2 &&
          fc1_weight_block->shape[2] * FP8_PER_INT32 *
                  TmaWarpSpecializedGroupedGemmInput::NVFP4BlockScaleVectorSize ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  hidden_size, TmaWarpSpecializedGroupedGemmInput::MinKDimAlignmentNVFP4))
          << "fc1 weight block size must be (num_experts_on_rank, inter_size * 2, hidden_size // 4 "
             "// block_scale_vector_size)";
      TVM_FFI_ICHECK_EQ(fc1_global->shape[0], num_experts_on_rank)
          << "fc1 global size must be (num_experts_on_rank,)";
      TVM_FFI_ICHECK(fc2_act_global->ndim == 0 || fc2_act_global->shape[0] == num_experts_on_rank)
          << "fc2 act global must be scalar or (num_experts_on_rank,)";
      TVM_FFI_ICHECK(
          fc2_weight_block->shape[0] == num_experts_on_rank &&
          fc2_weight_block->shape[1] ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  hidden_size, TmaWarpSpecializedGroupedGemmInput::MinNDimAlignmentNVFP4) &&
          fc2_weight_block->shape[2] * FP8_PER_INT32 *
                  TmaWarpSpecializedGroupedGemmInput::NVFP4BlockScaleVectorSize ==
              TmaWarpSpecializedGroupedGemmInput::alignToSfDim(
                  inter_size, TmaWarpSpecializedGroupedGemmInput::MinKDimAlignmentNVFP4))
          << "fc2 weight block size must be (num_experts_on_rank, hidden_size, inter_size // 4 // "
             "block_scale_vector_size)";
      TVM_FFI_ICHECK_EQ(fc2_global->shape[0], num_experts_on_rank)
          << "fc2 global size must be (num_experts_on_rank,)";

      return kernels::QuantParams::FP4(
          static_cast<float const*>(fc1_act_global->data),
          static_cast<TmaWarpSpecializedGroupedGemmInput::ElementSF*>(fc1_weight_block->data),
          static_cast<float const*>(fc1_global->data),
          static_cast<float const*>(fc2_act_global->data),
          static_cast<TmaWarpSpecializedGroupedGemmInput::ElementSF*>(fc2_weight_block->data),
          static_cast<float const*>(fc2_global->data), fc1_act_global->ndim == 1,
          fc2_act_global->ndim == 1);
    } else if (mUseDeepSeekFP8BlockScaling) {
      Tensor fc1_scales = quant_scales.value()[0];
      Tensor fc2_scales = quant_scales.value()[1];
      return kernels::QuantParams::FP8BlockScaling(static_cast<float const*>(fc1_scales->data),
                                                   static_cast<float const*>(fc2_scales->data));
    } else if (isWFP4A16Quant()) {
      TVM_FFI_ICHECK(quant_scales.has_value()) << "Expecting quant scales for W4 quantization";
      TVM_FFI_ICHECK_EQ(quant_scales.value().size(), 2)
          << "Expecting 2 quant scales for W4A16 quantization";

      Tensor fc1_weight_scales = quant_scales.value()[0];
      Tensor fc2_weight_scales = quant_scales.value()[1];
      int group_size = TmaWarpSpecializedGroupedGemmInput::INT4GroupwiseParams::wfp4a16_group_size;
      return kernels::QuantParams::GroupWise(group_size,
                                             static_cast<void const*>(fc1_weight_scales->data),
                                             static_cast<void const*>(fc2_weight_scales->data),
                                             nullptr, nullptr, nullptr, nullptr, nullptr, nullptr);
    } else if (isInt4Quant()) {
      TVM_FFI_ICHECK(quant_scales.has_value()) << "Expecting quant scales for INT4 quantization";
      TVM_FFI_ICHECK_EQ(quant_scales.value().size(), 8)
          << "Expecting 8 quant scales for INT4 quantization";
      Tensor fc1_weight_scales = quant_scales.value()[0];
      Tensor fc2_weight_scales = quant_scales.value()[1];
      Tensor fc1_act_scales = quant_scales.value()[2];
      Tensor fc2_act_scales = quant_scales.value()[3];
      Tensor fc1_weight_zeros = quant_scales.value()[4];
      Tensor fc2_weight_zeros = quant_scales.value()[5];
      Tensor fc1_alpha = quant_scales.value()[6];
      Tensor fc2_alpha = quant_scales.value()[7];
      int group_size = TmaWarpSpecializedGroupedGemmInput::INT4GroupwiseParams::int4_group_size;
      return kernels::QuantParams::GroupWise(
          group_size, static_cast<void const*>(fc1_weight_scales->data),
          static_cast<void const*>(fc2_weight_scales->data),
          static_cast<void const*>(get_numel(fc1_act_scales) > 0 ? fc1_act_scales->data : nullptr),
          static_cast<void const*>(get_numel(fc2_act_scales) > 0 ? fc2_act_scales->data : nullptr),
          static_cast<void const*>(get_numel(fc1_weight_zeros) > 0 ? fc1_weight_zeros->data
                                                                   : nullptr),
          static_cast<void const*>(get_numel(fc2_weight_zeros) > 0 ? fc2_weight_zeros->data
                                                                   : nullptr),
          static_cast<float const*>(get_numel(fc1_alpha) > 0 ? fc1_alpha->data : nullptr),
          static_cast<float const*>(get_numel(fc2_alpha) > 0 ? fc2_alpha->data : nullptr));
    } else {
      return kernels::QuantParams{};
    }
  }

  bool isFp8Quant() const {
    return !mUseDeepSeekFP8BlockScaling && mActivationDtype == dl_float8_e4m3fn &&
           mWeightDtype == dl_float8_e4m3fn;
  }

  bool isNvfp4Quant() const {
    return mWeightDtype == dl_int64 &&
           mActivationDtype != dl_float8_e4m3fn;  // FP8 activation does not use FP4
  }

  bool isWFP4A16Quant() const { return mUseW4GroupScaling && mWeightDtype == dl_uint8; }

  bool isInt4Quant() const { return mWeightDtype == dl_uint4x2; }

  bool isW4AFp8Quant() const { return mActivationDtype == dl_float8_e4m3fn && isInt4Quant(); }

  bool isWMxfp4AFp8Quant() const {
    return mActivationDtype == dl_float8_e4m3fn && mWeightDtype == dl_int64 && !mUseMxfp8ActScaling;
  }

  bool isWMxfp4AMxfp8Quant() const {
    return mActivationDtype == dl_float8_e4m3fn && mWeightDtype == dl_int64 && mUseMxfp8ActScaling;
  }
};

tvm::ffi::Module init(DLDataType activation_dtype, DLDataType weight_dtype, DLDataType output_dtype,
                      bool use_deepseek_fp8_block_scale, bool use_w4_group_scaling,
                      bool use_mxfp8_act_scaling) {
  auto ptr = tvm::ffi::make_object<FusedMoeRunner>(activation_dtype, weight_dtype, output_dtype,
                                                   use_deepseek_fp8_block_scale,
                                                   use_w4_group_scaling, use_mxfp8_act_scaling);
  return tvm::ffi::Module(ptr);
}

TVM_FFI_DLL_EXPORT_TYPED_FUNC(init, init);
