#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cutlass_fused_moe_kernels.cuh"
#include "moe_kernels.h"

namespace tensorrt_llm::kernels {
// ==================== Variable batched GEMM specializations ==================================
template class CutlassMoeFCRunner<float, float>;

#ifdef ENABLE_BF16
template class CutlassMoeFCRunner<__hip_bfloat16, __hip_bfloat16>;
template class CutlassMoeFCRunner<__hip_bfloat16, uint8_t>;
template class CutlassMoeFCRunner<__hip_bfloat16, cutlass::uint4b_t>;
#endif

template class CutlassMoeFCRunner<half, half>;
template class CutlassMoeFCRunner<half, uint8_t>;
template class CutlassMoeFCRunner<half, cutlass::uint4b_t>;
#ifdef ENABLE_FP8
// template class CutlassMoeFCRunner<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz>;
template class CutlassMoeFCRunner<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half>;
template class CutlassMoeFCRunner<__hip_fp8_e4m3_fnuz, cutlass::uint4b_t, half, half>;
#ifdef ENABLE_BF16
template class CutlassMoeFCRunner<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16>;
template class CutlassMoeFCRunner<__hip_bfloat16, __hip_fp8_e4m3_fnuz, __hip_bfloat16>;
template class CutlassMoeFCRunner<__hip_fp8_e4m3_fnuz, cutlass::uint4b_t, __hip_bfloat16, __hip_bfloat16>;
#endif
#endif
#ifdef ENABLE_FP4
template class CutlassMoeFCRunner<__hip_fp4_e2m1, __hip_fp4_e2m1, half>;
template class CutlassMoeFCRunner<__hip_fp4_e2m1, __hip_fp4_e2m1, half, half>;
#ifdef ENABLE_BF16
template class CutlassMoeFCRunner<__hip_fp4_e2m1, __hip_fp4_e2m1, __hip_bfloat16>;
template class CutlassMoeFCRunner<__hip_fp4_e2m1, __hip_fp4_e2m1, __hip_bfloat16, __hip_bfloat16>;
#endif
#endif

};  // namespace tensorrt_llm::kernels
