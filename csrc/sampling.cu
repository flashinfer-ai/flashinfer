/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/sampling.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;

void sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                         bool deterministic, int64_t hip_stream) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(1, uniform_samples);  // uniform_samples: (batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(0));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status = sampling::SamplingFromProb(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), batch_size, vocab_size, deterministic, stream);
  TORCH_CHECK(status == hipSuccess, "SamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));
}

void top_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               at::Tensor success, std::optional<at::Tensor> maybe_top_p_arr,
                               double top_p_val, bool deterministic, int64_t hip_stream) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_p_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_p_rounds = uniform_samples.size(0);
  bool has_top_p_arr = maybe_top_p_arr.has_value();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status = sampling::TopPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()),
      has_top_p_arr ? static_cast<float*>(maybe_top_p_arr->data_ptr()) : nullptr, batch_size,
      top_p_val, vocab_size, max_top_p_rounds, deterministic, stream);
  TORCH_CHECK(status == hipSuccess, "TopPSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));
}

void top_k_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               at::Tensor success, std::optional<at::Tensor> maybe_top_k_arr,
                               unsigned int top_k_val, bool deterministic, int64_t hip_stream) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_k_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_k_rounds = uniform_samples.size(0);
  bool has_top_k_arr = maybe_top_k_arr.has_value();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status = sampling::TopKSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()),
      has_top_k_arr ? static_cast<float*>(maybe_top_k_arr->data_ptr()) : nullptr, batch_size,
      top_k_val, vocab_size, max_top_k_rounds, deterministic, stream);
  TORCH_CHECK(status == hipSuccess, "TopKSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));
}

void min_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               std::optional<at::Tensor> maybe_min_p_arr, double min_p_val,
                               bool deterministic, int64_t hip_stream) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(1, uniform_samples);  // uniform_samples: (batch_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  CHECK_EQ(uniform_samples.size(0), batch_size);
  bool has_min_p_arr = maybe_min_p_arr.has_value();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status = sampling::MinPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      has_min_p_arr ? static_cast<float*>(maybe_min_p_arr->data_ptr()) : nullptr,
      static_cast<int*>(samples.data_ptr()), batch_size, min_p_val, vocab_size, deterministic,
      stream);
  TORCH_CHECK(status == hipSuccess, "MinPSamplingFromProb failed with error code " +
                                         std::string(hipGetErrorString(status)));
}

void top_k_top_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples,
                                     at::Tensor samples, at::Tensor success,
                                     std::optional<at::Tensor> maybe_top_k_arr, double top_k_val,
                                     std::optional<at::Tensor> maybe_top_p_arr, double top_p_val,
                                     bool deterministic, int64_t hip_stream) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_rounds, batch_size)
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_rounds = uniform_samples.size(0);
  CHECK_EQ(uniform_samples.size(1), batch_size);
  bool has_top_k_arr = maybe_top_k_arr.has_value();
  bool has_top_p_arr = maybe_top_p_arr.has_value();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status = sampling::TopKTopPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      has_top_k_arr ? static_cast<int*>(maybe_top_k_arr->data_ptr()) : nullptr,
      has_top_p_arr ? static_cast<float*>(maybe_top_p_arr->data_ptr()) : nullptr,
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()), batch_size,
      top_k_val, top_p_val, vocab_size, max_rounds, deterministic, stream);
  TORCH_CHECK(status == hipSuccess, "TopKTopPSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));
}

void chain_speculative_sampling(at::Tensor draft_probs, at::Tensor draft_token_ids,
                                at::Tensor uniform_samples, at::Tensor target_probs,
                                at::Tensor output_token_ids, at::Tensor output_accepted_token_num,
                                at::Tensor output_emitted_token_num, bool deterministic,
                                int64_t hip_stream) {
  CHECK_INPUT(draft_probs);
  CHECK_INPUT(draft_token_ids);
  CHECK_INPUT(uniform_samples);
  CHECK_INPUT(target_probs);
  auto device = draft_probs.device();
  CHECK_EQ(draft_token_ids.device(), device);
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_EQ(target_probs.device(), device);
  CHECK_DIM(3, draft_probs);      // draft_probs: (batch_size, num_speculate_tokens, vocab_size)
  CHECK_DIM(2, draft_token_ids);  // draft_token_ids: (batch_size, num_speculate_tokens)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (batch_size, num_speculate_tokens + 1)
  CHECK_DIM(3, target_probs);  // target_probs: (batch_size, num_speculate_tokens + 1, vocab_size)
  unsigned int batch_size = draft_probs.size(0);
  unsigned int num_speculate_tokens = draft_probs.size(1);
  unsigned int vocab_size = draft_probs.size(2);
  CHECK_EQ(batch_size, draft_token_ids.size(0));
  CHECK_EQ(batch_size, uniform_samples.size(0));
  CHECK_EQ(batch_size, target_probs.size(0));
  CHECK_EQ(num_speculate_tokens + 1, uniform_samples.size(1));
  CHECK_EQ(num_speculate_tokens + 1, target_probs.size(1));
  CHECK_EQ(vocab_size, target_probs.size(2));
  CHECK_EQ(batch_size, output_accepted_token_num.size(0));
  CHECK_EQ(batch_size, output_emitted_token_num.size(0));

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status = sampling::ChainSpeculativeSampling<float, int>(
      static_cast<float*>(draft_probs.data_ptr()), static_cast<int*>(draft_token_ids.data_ptr()),
      static_cast<float*>(uniform_samples.data_ptr()), static_cast<float*>(target_probs.data_ptr()),
      static_cast<int*>(output_token_ids.data_ptr()),
      static_cast<int*>(output_accepted_token_num.data_ptr()),
      static_cast<int*>(output_emitted_token_num.data_ptr()), batch_size, num_speculate_tokens,
      vocab_size, deterministic, stream);

  TORCH_CHECK(status == hipSuccess, "ChainSpeculativeSampling failed with error code " +
                                         std::string(hipGetErrorString(status)));
}
