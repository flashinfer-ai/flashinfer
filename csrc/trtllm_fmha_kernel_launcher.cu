/*
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <flashinfer/exception.h>
#include <flashinfer/trtllm/common.h>
#include <flashinfer/trtllm/fmha/decoder_impl_common.h>
#include <flashinfer/trtllm/fmha/fmhaRunnerParams.h>
#include <hip/hiprtc.h>

#include <flashinfer/semaphore_utils.cuh>
#include <flashinfer/trtllm/fmha/fmhaRunner.cuh>
#include <flashinfer/trtllm/fmha/gen_kernel_launcher.cuh>
#include <flashinfer/utils.cuh>
#include <iostream>
#include <optional>
#include <sstream>
#include <unordered_map>

#include "pytorch_extension_utils.h"

namespace flashinfer {

enum class TllmPagedAttentionMode {
  Context,
  ForGen,
};

#include <memory>
#include <mutex>

class TllmGenFmhaRunnerCache {
 public:
  using Key = std::tuple<Data_type, Data_type, Data_type>;

  static std::shared_ptr<TllmGenFmhaRunner> get(Data_type q_data_type, Data_type kv_data_type,
                                                Data_type o_data_type) {
    static std::unordered_map<Key, std::shared_ptr<TllmGenFmhaRunner>, KeyHash> cache;
    static std::mutex cache_mutex;
    Key key = std::make_tuple(q_data_type, kv_data_type, o_data_type);

    std::lock_guard<std::mutex> lock(cache_mutex);
    auto it = cache.find(key);
    if (it != cache.end()) {
      return it->second;
    } else {
      auto runner = std::make_shared<TllmGenFmhaRunner>(q_data_type, kv_data_type, o_data_type);
      cache.emplace(key, runner);
      return runner;
    }
  }

 private:
  struct KeyHash {
    std::size_t operator()(const Key& k) const {
      return std::hash<int>()(static_cast<int>(std::get<0>(k))) ^
             (std::hash<int>()(static_cast<int>(std::get<1>(k))) << 1) ^
             (std::hash<int>()(static_cast<int>(std::get<2>(k))) << 2);
    }
  };
};

void trtllm_paged_attention_launcher(
    void* out, void* query, void* key_cache, void* value_cache, void* workspace_buffer,
    int* block_tables, int* seq_lens, int* cum_seq_lens_q, int* cum_seq_lens_kv,
    Data_type q_data_type, Data_type kv_data_type, Data_type o_data_type,
    TllmPagedAttentionMode mode, int64_t batch_size, int64_t max_q_len, int64_t max_kv_len,
    int64_t num_pages_in_mem_pool, int64_t num_qo_heads, int64_t num_kv_heads, int64_t head_dim_qk,
    int64_t head_dim_vo, int64_t page_size, int64_t kv_stride_keys_values, int64_t kv_stride_heads,
    int64_t kv_stride_batch, int64_t max_num_blocks_per_seq, double bmm1_scale, double bmm2_scale,
    int64_t window_left, int64_t sum_seq_q, int64_t sm_count, hipStream_t stream) {
  if (num_qo_heads % num_kv_heads != 0) {
    std::ostringstream err_msg;
    err_msg << "num_qo_heads must be a multiple of num_kv_heads, got num_kv_heads: " << num_kv_heads
            << " and num_qo_heads: " << num_qo_heads;
    FLASHINFER_ERROR(err_msg.str());
  }

  auto fmha_runner = TllmGenFmhaRunnerCache::get(q_data_type, kv_data_type, o_data_type);
  TllmGenFmhaRunnerParams runner_params;

  // Common params
  runner_params.qPtr = query;
  runner_params.kPtr = key_cache;
  runner_params.vPtr = value_cache;
  runner_params.kvPageIdxPtr = block_tables;
  runner_params.seqLensKvPtr = seq_lens;
  runner_params.oPtr = out;
  runner_params.mHeadDimQk = head_dim_qk;
  runner_params.mHeadDimV = head_dim_vo;
  runner_params.mNumHeadsQ = num_qo_heads;
  runner_params.mNumHeadsKv = num_kv_heads;
  runner_params.mNumHeadsQPerKv = num_qo_heads / num_kv_heads;
  runner_params.mBatchSize = batch_size;
  runner_params.mMaxSeqLenKv = max_kv_len;
  runner_params.mMaxNumPagesPerSeqKv = max_num_blocks_per_seq;
  runner_params.mNumTokensPerPage = page_size;
  runner_params.mQkvLayout = QkvLayout::PagedKv;
  runner_params.mMultiProcessorCount = sm_count;
  runner_params.kvStrideKeysValues = kv_stride_keys_values;
  runner_params.kvStrideHeads = kv_stride_heads;
  runner_params.kvStrideBatch = kv_stride_batch;
  runner_params.mNumPagesInMemPool = num_pages_in_mem_pool;
  runner_params.stream = stream;
  runner_params.outputScale = bmm2_scale;
  runner_params.scaleSoftmaxLog2 = bmm1_scale * M_LOG2E;
  runner_params.mChunkedAttentionSize = INT_MAX;  // disable chunked attention by INT_MAX
  runner_params.mAttentionWindowSize =
      window_left == -1 ? INT_MAX : window_left + 1;  // disable window attention by INT_MAX
  runner_params.mMaxSeqLenQ = max_q_len;
  runner_params.mSumOfSeqLensQ = sum_seq_q;

  if (mode == TllmPagedAttentionMode::Context) {
    runner_params.mMaskType = TrtllmGenAttentionMaskType::Causal;
    runner_params.mKernelType = FmhaKernelType::Context;
    runner_params.mTileScheduler = TileScheduler::Persistent;
    runner_params.mMultiCtasKvMode = false;

    runner_params.cumSeqLensQPtr = cum_seq_lens_q;
    runner_params.cumSeqLensKvPtr = cum_seq_lens_kv;
  } else {
    // ForGen
    runner_params.mMaskType = TrtllmGenAttentionMaskType::Dense;
    runner_params.mKernelType = FmhaKernelType::Generation;
    bool use_multi_block = true;
    runner_params.mTileScheduler =
        use_multi_block ? TileScheduler::Static : TileScheduler::Persistent;
    runner_params.mMultiCtasKvMode = use_multi_block;

    size_t num_semaphores =
        round_up(batch_size * num_qo_heads, 8);  // align multiCtasKvScratchPtr to 16 bytes
    runner_params.multiCtasKvScratchPtr = reinterpret_cast<void*>(
        static_cast<char*>(workspace_buffer) + num_semaphores * sizeof(uint32_t));
    runner_params.multiCtasKvCounterPtr = reinterpret_cast<int32_t*>(workspace_buffer);
    zero_gmem_semaphore_launcher(runner_params.multiCtasKvCounterPtr, num_semaphores,
                                 /*enable_pdl=*/true, stream);
  }

  auto [foundKernels, kinfo] = fmha_runner->isSupportedWithInfo(runner_params);
  if (!foundKernels) {
    std::ostringstream err_msg;
    err_msg << "Missing TRTLLM-GEN kernel ("
            << (mode == TllmPagedAttentionMode::Context ? "context" : "decode") << "): " << kinfo;
    FLASHINFER_ERROR(err_msg.str());
  }

  fmha_runner->run(runner_params);
}

inline Data_type torch_dtype_to_tllm_data_type(at::ScalarType dtype) {
  if (dtype == at::ScalarType::Float) {
    return Data_type::DATA_TYPE_FP32;
  } else if (dtype == at::ScalarType::Half) {
    return Data_type::DATA_TYPE_FP16;
  } else if (dtype == at::ScalarType::BFloat16) {
    return Data_type::DATA_TYPE_BF16;
  } else if (dtype == at::ScalarType::Float8_e4m3fn) {
    return Data_type::DATA_TYPE_E4M3;
  } else if (dtype == at::ScalarType::Float8_e5m2) {
    return Data_type::DATA_TYPE_E5M2;
  }
  return Data_type::DATA_TYPE_UNKNOWN;
}

void trtllm_paged_attention_decode(at::Tensor& out, at::Tensor& query, at::Tensor& key_value_cache,
                                   at::Tensor& workspace_buffer, at::Tensor& block_tables,
                                   at::Tensor& seq_lens, int64_t max_kv_len, double bmm1_scale,
                                   double bmm2_scale, int64_t window_left, int64_t sm_count) {
  auto q_data_type = torch_dtype_to_tllm_data_type(query.scalar_type());
  auto kv_data_type = torch_dtype_to_tllm_data_type(key_value_cache.scalar_type());
  auto o_data_type = torch_dtype_to_tllm_data_type(out.scalar_type());
  // NOTE(Zihao): query is [B, Q, H, D]
  // where Q is the number of query tokens per request, used in MTP
  // based on profiled results, always use decode mode for MTP (q_len is small)
  // example: when kv_len = 10000, q < 200, decode mode is faster
  int batch_size = query.size(0);
  int q_len_per_request = query.size(1);
  int sum_seq_q = batch_size * q_len_per_request;
  int num_qo_heads = query.size(2);
  int head_dim_qk = query.size(3);
  int head_dim_vo = out.size(-1);
  // NOTE(Zihao): key_value_cache is [num_pages, 1/2, num_kv_heads, page_size, head_dim]
  // For KV-Cache sharing (MLA), the second dimension is 1 (key/value cache are shared)
  // otherwise it is 2, one for key and one for value
  TORCH_CHECK(key_value_cache.size(1) == 1 || key_value_cache.size(1) == 2,
              "The second dimension of key_value_cache must be 1 or 2, got " +
                  std::to_string(key_value_cache.size(1)));
  bool share_kv_cache = key_value_cache.size(1) == 1;
  int page_size = key_value_cache.size(-2);
  int num_kv_heads = key_value_cache.size(-3);
  int max_num_blocks_per_seq = block_tables.size(-1);
  int num_pages_in_mem_pool = key_value_cache.size(0) * key_value_cache.size(1);

  int kv_stride_keys_values = key_value_cache.stride(-2);  // key/values
  int kv_stride_heads = key_value_cache.stride(-3);        // head
  int kv_stride_batch = key_value_cache.stride(0);         // batch

  auto device = query.device();
  const auto stream = at::cuda::getCurrentCUDAStream(device.index());

  trtllm_paged_attention_launcher(
      out.data_ptr(), query.data_ptr(), key_value_cache.data_ptr(),
      (char*)key_value_cache.data_ptr() +
          (share_kv_cache ? 0 : key_value_cache.stride(1) * key_value_cache.element_size()),
      workspace_buffer.data_ptr(), static_cast<int*>(block_tables.data_ptr()),
      static_cast<int*>(seq_lens.data_ptr()),
      /*cum_seq_lens_q=*/nullptr,
      /*cum_seq_lens_kv=*/nullptr, q_data_type, kv_data_type, o_data_type,
      TllmPagedAttentionMode::ForGen, batch_size, /*max_q_len=*/q_len_per_request, max_kv_len,
      num_pages_in_mem_pool, num_qo_heads, num_kv_heads, head_dim_qk, head_dim_vo, page_size,
      kv_stride_keys_values, kv_stride_heads, kv_stride_batch, max_num_blocks_per_seq, bmm1_scale,
      bmm2_scale, window_left, sum_seq_q, sm_count, stream);
}

void trtllm_paged_attention_context(at::Tensor& out, at::Tensor& query, at::Tensor& key_value_cache,
                                    at::Tensor& workspace_buffer, at::Tensor& block_tables,
                                    at::Tensor& seq_lens, int64_t max_q_len, int64_t max_kv_len,
                                    double bmm1_scale, double bmm2_scale, int64_t batch_size,
                                    int64_t window_left, at::Tensor& cum_seq_lens_q,
                                    at::Tensor& cum_seq_lens_kv, int64_t sm_count) {
  auto q_data_type = torch_dtype_to_tllm_data_type(query.scalar_type());
  auto kv_data_type = torch_dtype_to_tllm_data_type(key_value_cache.scalar_type());
  auto o_data_type = torch_dtype_to_tllm_data_type(out.scalar_type());
  int num_qo_heads = query.size(1);
  int sum_seq_q = query.size(0);
  int head_dim_qk = query.size(2);
  int head_dim_vo = out.size(-1);
  int max_num_blocks_per_seq = block_tables.size(-1);
  int num_pages_in_mem_pool = key_value_cache.size(0) * key_value_cache.size(1);
  // NOTE(Zihao): key_value_cache is [num_pages, 1/2, num_kv_heads, page_size, head_dim]
  // For KV-Cache sharing (MLA), the second dimension is 1 (key/value cache are shared)
  // otherwise it is 2, one for key and one for value
  TORCH_CHECK(key_value_cache.size(1) == 1 || key_value_cache.size(1) == 2,
              "The second dimension of key_value_cache must be 1 or 2, got " +
                  std::to_string(key_value_cache.size(1)));
  bool share_kv_cache = key_value_cache.size(1) == 1;
  int page_size = key_value_cache.size(-2);
  int num_kv_heads = key_value_cache.size(-3);

  int kv_stride_keys_values = key_value_cache.stride(-2);  // key/values
  int kv_stride_heads = key_value_cache.stride(-3);        // head
  int kv_stride_batch = key_value_cache.stride(0);         // batch

  auto device = query.device();
  const auto stream = at::cuda::getCurrentCUDAStream(device.index());

  trtllm_paged_attention_launcher(
      out.data_ptr(), query.data_ptr(), key_value_cache.data_ptr(),
      (char*)key_value_cache.data_ptr() +
          (share_kv_cache ? 0 : key_value_cache.stride(1) * key_value_cache.element_size()),
      workspace_buffer.data_ptr(), static_cast<int*>(block_tables.data_ptr()),
      static_cast<int*>(seq_lens.data_ptr()),
      /*cum_seq_lens_q=*/static_cast<int*>(cum_seq_lens_q.data_ptr()),
      /*cum_seq_lens_kv=*/static_cast<int*>(cum_seq_lens_kv.data_ptr()), q_data_type, kv_data_type,
      o_data_type, TllmPagedAttentionMode::Context, batch_size, max_q_len, max_kv_len,
      num_pages_in_mem_pool, num_qo_heads, num_kv_heads, head_dim_qk, head_dim_vo, page_size,
      kv_stride_keys_values, kv_stride_heads, kv_stride_batch, max_num_blocks_per_seq, bmm1_scale,
      bmm2_scale, window_left, sum_seq_q, sm_count, stream);
}

namespace trtllm_cubin_loader {
#include <flashinfer/cubin_loader.h>
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_paged_attention_decode", trtllm_paged_attention_decode);
  m.def("trtllm_paged_attention_context", trtllm_paged_attention_context);
}

}  // namespace flashinfer
