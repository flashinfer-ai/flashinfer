#include "hip/hip_runtime.h"
/*
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <flashinfer/exception.h>
#include <flashinfer/trtllm/common.h>
#include <flashinfer/trtllm/fmha/decoder_impl_common.h>
#include <flashinfer/trtllm/fmha/fmhaRunnerParams.h>
#include <hip/hiprtc.h>

#include <flashinfer/trtllm/fmha/fmhaRunner.cuh>
#include <flashinfer/utils.cuh>
#include <iostream>
#include <optional>
#include <sstream>
#include <unordered_map>

#include "pytorch_extension_utils.h"

namespace flashinfer {

enum class TllmPagedAttentionMode {
  Context,
  ForGen,
};

#include <memory>
#include <mutex>

class TllmGenFmhaRunnerCache {
 public:
  using Key = std::tuple<Data_type, Data_type, Data_type>;

  static std::shared_ptr<TllmGenFmhaRunner> get(Data_type q_data_type, Data_type kv_data_type,
                                                Data_type o_data_type) {
    static std::unordered_map<Key, std::shared_ptr<TllmGenFmhaRunner>, KeyHash> cache;
    static std::mutex cache_mutex;
    Key key = std::make_tuple(q_data_type, kv_data_type, o_data_type);

    std::lock_guard<std::mutex> lock(cache_mutex);
    auto it = cache.find(key);
    if (it != cache.end()) {
      return it->second;
    } else {
      auto runner = std::make_shared<TllmGenFmhaRunner>(q_data_type, kv_data_type, o_data_type);
      cache.emplace(key, runner);
      return runner;
    }
  }

 private:
  struct KeyHash {
    std::size_t operator()(const Key& k) const {
      return std::hash<int>()(static_cast<int>(std::get<0>(k))) ^
             (std::hash<int>()(static_cast<int>(std::get<1>(k))) << 1) ^
             (std::hash<int>()(static_cast<int>(std::get<2>(k))) << 2);
    }
  };
};

void trtllm_paged_attention_launcher(
    void* out, void* out_scale_factor, void* query, void* key_cache, void* value_cache,
    void* workspace_buffer, int* block_tables, int* seq_lens, int* cum_seq_lens_q,
    int* cum_seq_lens_kv, float* attention_sinks, Data_type q_data_type, Data_type kv_data_type,
    Data_type o_data_type, TllmPagedAttentionMode mode, int64_t batch_size, int64_t max_q_len,
    int64_t max_kv_len, int64_t num_pages_in_mem_pool, int64_t num_qo_heads, int64_t num_kv_heads,
    int64_t head_dim_qk, int64_t head_dim_vo, int64_t page_size, int64_t kv_stride_keys_values,
    int64_t kv_stride_heads, int64_t kv_stride_batch, int64_t max_num_blocks_per_seq,
    double bmm1_scale, double bmm2_scale, double o_sf_scale, int64_t o_sf_vec_size,
    int64_t o_sf_start_index, int64_t window_left, int64_t sum_seq_q, int64_t sm_count,
    bool enable_pdl, hipStream_t stream) {
  if (num_qo_heads % num_kv_heads != 0) {
    std::ostringstream err_msg;
    err_msg << "num_qo_heads must be a multiple of num_kv_heads, got num_kv_heads: " << num_kv_heads
            << " and num_qo_heads: " << num_qo_heads;
    FLASHINFER_ERROR(err_msg.str());
  }

  auto fmha_runner = TllmGenFmhaRunnerCache::get(q_data_type, kv_data_type, o_data_type);
  TllmGenFmhaRunnerParams runner_params;

  // Common params
  runner_params.qPtr = query;
  runner_params.kPtr = key_cache;
  runner_params.vPtr = value_cache;
  runner_params.kvPageIdxPtr = block_tables;
  runner_params.seqLensKvPtr = seq_lens;
  runner_params.oPtr = out;
  runner_params.mHeadDimQk = head_dim_qk;
  runner_params.mHeadDimV = head_dim_vo;
  runner_params.mNumHeadsQ = num_qo_heads;
  runner_params.mNumHeadsKv = num_kv_heads;
  runner_params.mNumHeadsQPerKv = num_qo_heads / num_kv_heads;
  runner_params.mBatchSize = batch_size;
  runner_params.mMaxSeqLenKv = max_kv_len;
  runner_params.mMaxNumPagesPerSeqKv = max_num_blocks_per_seq;
  runner_params.mNumTokensPerPage = page_size;
  runner_params.mQkvLayout = QkvLayout::PagedKv;
  runner_params.mMultiProcessorCount = sm_count;
  runner_params.kStrideKeysValues = kv_stride_keys_values;
  runner_params.kStrideHeads = kv_stride_heads;
  runner_params.kStrideBatch = kv_stride_batch;
  runner_params.vStrideKeysValues = kv_stride_keys_values;
  runner_params.vStrideHeads = kv_stride_heads;
  runner_params.vStrideBatch = kv_stride_batch;
  runner_params.mNumPagesInMemPool = num_pages_in_mem_pool;
  runner_params.stream = stream;
  runner_params.outputScale = bmm2_scale;
  runner_params.scaleSoftmaxLog2 = bmm1_scale * M_LOG2E;
  runner_params.oSfPtr = out_scale_factor;
  runner_params.mSfStartTokenIdx = o_sf_start_index;
  runner_params.mScaleSfO = o_sf_scale;
  TORCH_CHECK(o_sf_vec_size == 16 || o_sf_vec_size == -1,
              "Only support o_sf_vec_size == 16 or -1(not used)");
  runner_params.mChunkedAttentionSize = INT_MAX;  // disable chunked attention by INT_MAX
  runner_params.mAttentionWindowSize =
      window_left == -1 ? INT_MAX : window_left + 1;  // disable window attention by INT_MAX
  runner_params.mMaxSeqLenQ = max_q_len;
  runner_params.mSumOfSeqLensQ = sum_seq_q;
  runner_params.ptrAttentionSinks = attention_sinks;
  runner_params.enable_pdl = enable_pdl;
  if (mode == TllmPagedAttentionMode::Context) {
    runner_params.mMaskType = TrtllmGenAttentionMaskType::Causal;
    runner_params.mKernelType = FmhaKernelType::Context;
    runner_params.mTileScheduler = TileScheduler::Persistent;
    runner_params.mMultiCtasKvMode = false;

    runner_params.cumSeqLensQPtr = cum_seq_lens_q;
    runner_params.cumSeqLensKvPtr = cum_seq_lens_kv;
  } else {
    // ForGen
    runner_params.mMaskType = TrtllmGenAttentionMaskType::Dense;
    runner_params.mKernelType = FmhaKernelType::Generation;
    bool use_multi_block = true;
    runner_params.mTileScheduler =
        use_multi_block ? TileScheduler::Static : TileScheduler::Persistent;
    runner_params.mMultiCtasKvMode = use_multi_block;

    size_t max_batch_size = 8192;   // todo(Yingyi): get from dlfw
    size_t max_num_qo_heads = 256;  // todo(Yingyi): get from dlfw, in total 8MB
    size_t num_semaphores =
        round_up(max_batch_size * max_num_qo_heads, 8);  // max 8MB, should align to 16 bytes
    runner_params.multiCtasKvScratchPtr = reinterpret_cast<void*>(
        static_cast<char*>(workspace_buffer) + num_semaphores * sizeof(uint32_t));
    runner_params.multiCtasKvCounterPtr = reinterpret_cast<int32_t*>(workspace_buffer);
  }

  auto [foundKernels, kinfo] = fmha_runner->isSupportedWithInfo(runner_params);
  if (!foundKernels) {
    std::ostringstream err_msg;
    err_msg << "Missing TRTLLM-GEN kernel ("
            << (mode == TllmPagedAttentionMode::Context ? "context" : "decode") << "): " << kinfo;
    FLASHINFER_ERROR(err_msg.str());
  }

  fmha_runner->run(runner_params);
}

inline Data_type torch_dtype_to_tllm_data_type(at::ScalarType dtype) {
  if (dtype == at::ScalarType::Float) {
    return Data_type::DATA_TYPE_FP32;
  } else if (dtype == at::ScalarType::Half) {
    return Data_type::DATA_TYPE_FP16;
  } else if (dtype == at::ScalarType::BFloat16) {
    return Data_type::DATA_TYPE_BF16;
  } else if (dtype == at::ScalarType::Float8_e4m3fn) {
    return Data_type::DATA_TYPE_E4M3;
  } else if (dtype == at::ScalarType::Float8_e5m2) {
    return Data_type::DATA_TYPE_E5M2;
  } else if (dtype == at::ScalarType::Byte) {
    // fp4 tensor is not supported in torch and use uint8_t as container.
    return Data_type::DATA_TYPE_E2M1;
  }
  return Data_type::DATA_TYPE_UNKNOWN;
}

inline bool is_4bit(Data_type data_type) { return data_type == Data_type::DATA_TYPE_E2M1; }

void trtllm_paged_attention_decode(at::Tensor out, std::optional<at::Tensor> out_scale_factor,
                                   at::Tensor query, at::Tensor key_cache, at::Tensor value_cache,
                                   at::Tensor workspace_buffer, at::Tensor block_tables,
                                   at::Tensor seq_lens, int64_t max_kv_len, double bmm1_scale,
                                   double bmm2_scale, double o_sf_scale, int64_t o_sf_vec_size,
                                   int64_t o_sf_start_index, int64_t window_left, int64_t sm_count,
                                   bool enable_pdl, std::optional<at::Tensor> attention_sinks) {
  auto q_data_type = torch_dtype_to_tllm_data_type(query.scalar_type());
  auto kv_data_type = torch_dtype_to_tllm_data_type(key_cache.scalar_type());
  TORCH_CHECK_EQ(key_cache.dim(), value_cache.dim());
  for (int i = 0; i < key_cache.dim(); i++) {
    TORCH_CHECK_EQ(key_cache.size(i), value_cache.size(i));
  }
  auto o_data_type = torch_dtype_to_tllm_data_type(out.scalar_type());
  // NOTE(Zihao): query is [B, Q, H, D]
  // where Q is the number of query tokens per request, used in MTP
  // based on profiled results, always use decode mode for MTP (q_len is small)
  // example: when kv_len = 10000, q < 200, decode mode is faster
  int batch_size = query.size(0);
  int q_len_per_request = query.size(1);
  int sum_seq_q = batch_size * q_len_per_request;
  int num_qo_heads = query.size(2);
  // Multiply by two for FP4 tensor as it is stored as UINT8 dtype. Assume the dim is even.
  int head_dim_k = is_4bit(kv_data_type) ? key_cache.size(-1) * 2 : key_cache.size(-1);
  int head_dim_q = is_4bit(q_data_type) ? query.size(-1) * 2 : query.size(-1);
  int head_dim_v = is_4bit(kv_data_type) ? value_cache.size(-1) * 2 : value_cache.size(-1);
  int head_dim_o = is_4bit(o_data_type) ? out.size(-1) * 2 : out.size(-1);
  TORCH_CHECK(head_dim_k == head_dim_q, "head_dim_k and head_dim_q must be the same, got " +
                                            std::to_string(head_dim_k) + " and " +
                                            std::to_string(head_dim_q));
  TORCH_CHECK((head_dim_v == 576 && head_dim_o == 512) || head_dim_v == head_dim_o,
              "head_dim_v and head_dim_o must be the same for non-MLA attention, got " +
                  std::to_string(head_dim_v) + " and " + std::to_string(head_dim_o));
  int page_size = key_cache.size(-2);
  int num_kv_heads = key_cache.size(-3);
  int max_num_blocks_per_seq = block_tables.size(-1);
  bool is_shared_kv = key_cache.data_ptr() == value_cache.data_ptr();
  int num_pages_in_mem_pool = is_shared_kv ? key_cache.size(0) : key_cache.size(0) * 2;

  int kv_stride_keys_values = key_cache.stride(-2);  // key/values
  int kv_stride_heads = key_cache.stride(-3);        // head
  int kv_stride_batch = key_cache.stride(0);         // batch

  auto device = query.device();
  const auto stream = at::cuda::getCurrentCUDAStream(device.index());
  void* output_sf_ptr = out_scale_factor ? out_scale_factor.value().data_ptr() : nullptr;

  float* attention_sinks_ptr = nullptr;
  if (attention_sinks) {
    TORCH_CHECK(attention_sinks->scalar_type() == at::ScalarType::Float,
                "attention_sinks must be a float tensor");
    attention_sinks_ptr = attention_sinks->data_ptr<float>();
  }

  trtllm_paged_attention_launcher(
      out.data_ptr(), output_sf_ptr, query.data_ptr(), key_cache.data_ptr(), value_cache.data_ptr(),
      workspace_buffer.data_ptr(), static_cast<int*>(block_tables.data_ptr()),
      static_cast<int*>(seq_lens.data_ptr()),
      /*cum_seq_lens_q=*/nullptr,
      /*cum_seq_lens_kv=*/nullptr, attention_sinks_ptr, q_data_type, kv_data_type, o_data_type,
      TllmPagedAttentionMode::ForGen, batch_size, /*max_q_len=*/q_len_per_request, max_kv_len,
      num_pages_in_mem_pool, num_qo_heads, num_kv_heads, head_dim_q, head_dim_o, page_size,
      kv_stride_keys_values, kv_stride_heads, kv_stride_batch, max_num_blocks_per_seq, bmm1_scale,
      bmm2_scale, o_sf_scale, o_sf_vec_size, o_sf_start_index, window_left, sum_seq_q, sm_count,
      enable_pdl, stream);
}

void trtllm_paged_attention_context(at::Tensor out, std::optional<at::Tensor> out_scale_factor,
                                    at::Tensor query, at::Tensor key_cache, at::Tensor value_cache,
                                    at::Tensor workspace_buffer, at::Tensor block_tables,
                                    at::Tensor seq_lens, int64_t max_q_len, int64_t max_kv_len,
                                    double bmm1_scale, double bmm2_scale, double o_sf_scale,
                                    int64_t o_sf_vec_size, int64_t o_sf_start_index,
                                    int64_t batch_size, int64_t window_left,
                                    at::Tensor cum_seq_lens_q, at::Tensor cum_seq_lens_kv,
                                    int64_t sm_count, bool enable_pdl,
                                    std::optional<at::Tensor> attention_sinks) {
  auto q_data_type = torch_dtype_to_tllm_data_type(query.scalar_type());
  auto kv_data_type = torch_dtype_to_tllm_data_type(key_cache.scalar_type());
  auto o_data_type = torch_dtype_to_tllm_data_type(out.scalar_type());
  int num_qo_heads = query.size(1);
  int sum_seq_q = query.size(0);
  // Multiply by two for FP4 tensor as it is stored as UINT8 dtype. Assume the dim is even.
  int head_dim_k = is_4bit(kv_data_type) ? key_cache.size(-1) * 2 : key_cache.size(-1);
  int head_dim_q = is_4bit(q_data_type) ? query.size(-1) * 2 : query.size(-1);
  int head_dim_v = is_4bit(kv_data_type) ? value_cache.size(-1) * 2 : value_cache.size(-1);
  int head_dim_o = is_4bit(o_data_type) ? out.size(-1) * 2 : out.size(-1);
  TORCH_CHECK(head_dim_k == head_dim_q, "head_dim_k and head_dim_q must be the same, got " +
                                            std::to_string(head_dim_k) + " and " +
                                            std::to_string(head_dim_q));
  TORCH_CHECK(head_dim_v == head_dim_o, "head_dim_v and head_dim_o must be the same, got " +
                                            std::to_string(head_dim_v) + " and " +
                                            std::to_string(head_dim_o));
  int max_num_blocks_per_seq = block_tables.size(-1);
  bool is_shared_kv = key_cache.data_ptr() == value_cache.data_ptr();
  int num_pages_in_mem_pool = is_shared_kv ? key_cache.size(0) : key_cache.size(0) * 2;
  int page_size = key_cache.size(-2);
  int num_kv_heads = key_cache.size(-3);

  int kv_stride_keys_values = key_cache.stride(-2);  // key/values
  int kv_stride_heads = key_cache.stride(-3);        // head
  int kv_stride_batch = key_cache.stride(0);         // batch

  auto device = query.device();
  const auto stream = at::cuda::getCurrentCUDAStream(device.index());
  void* output_sf_ptr = out_scale_factor ? out_scale_factor.value().data_ptr() : nullptr;

  float* attention_sinks_ptr = nullptr;
  if (attention_sinks) {
    TORCH_CHECK(attention_sinks->scalar_type() == at::ScalarType::Float,
                "attention_sinks must be a float tensor");
    attention_sinks_ptr = attention_sinks->data_ptr<float>();
  }

  trtllm_paged_attention_launcher(
      out.data_ptr(), output_sf_ptr, query.data_ptr(), key_cache.data_ptr(), value_cache.data_ptr(),
      workspace_buffer.data_ptr(), static_cast<int*>(block_tables.data_ptr()),
      static_cast<int*>(seq_lens.data_ptr()),
      /*cum_seq_lens_q=*/static_cast<int*>(cum_seq_lens_q.data_ptr()),
      /*cum_seq_lens_kv=*/static_cast<int*>(cum_seq_lens_kv.data_ptr()), attention_sinks_ptr,
      q_data_type, kv_data_type, o_data_type, TllmPagedAttentionMode::Context, batch_size,
      max_q_len, max_kv_len, num_pages_in_mem_pool, num_qo_heads, num_kv_heads, head_dim_q,
      head_dim_o, page_size, kv_stride_keys_values, kv_stride_heads, kv_stride_batch,
      max_num_blocks_per_seq, bmm1_scale, bmm2_scale, o_sf_scale, o_sf_vec_size, o_sf_start_index,
      window_left, sum_seq_q, sm_count, enable_pdl, stream);
}

void trtllm_ragged_attention_launcher(
    void* out, void* query, void* key, void* value, void* workspace_buffer, int* seq_lens,
    int* cum_seq_lens_q, int* cum_seq_lens_kv, float* attention_sinks, float* lse,
    Data_type q_data_type, Data_type kv_data_type, Data_type o_data_type, int64_t max_q_len,
    int64_t max_kv_len, int64_t num_qo_heads, int64_t num_kv_heads, int64_t head_dim_qk,
    int64_t head_dim_v, int64_t sum_seq_q, int64_t sum_seq_kv, double bmm1_scale, double bmm2_scale,
    double o_sf_scale, int64_t batch_size, int64_t window_left, int64_t sm_count, bool enable_pdl,
    bool is_causal, int64_t k_stride_keys_values, int64_t k_stride_heads, int64_t k_stride_batch,
    int64_t v_stride_keys_values, int64_t v_stride_heads, int64_t v_stride_batch,
    hipStream_t stream) {
  if (num_qo_heads % num_kv_heads != 0) {
    std::ostringstream err_msg;
    err_msg << "num_qo_heads must be a multiple of num_kv_heads, got num_kv_heads: " << num_kv_heads
            << " and num_qo_heads: " << num_qo_heads;
    FLASHINFER_ERROR(err_msg.str());
  }
  auto fmha_runner = TllmGenFmhaRunnerCache::get(q_data_type, kv_data_type, o_data_type);
  TllmGenFmhaRunnerParams runner_params;

  runner_params.qPtr = query;
  runner_params.kPtr = key;
  runner_params.vPtr = value;
  runner_params.kvPageIdxPtr = nullptr;
  runner_params.seqLensKvPtr = seq_lens;
  runner_params.oPtr = out;
  runner_params.mHeadDimQk = head_dim_qk;
  runner_params.mHeadDimV = head_dim_v;
  runner_params.mNumHeadsQ = num_qo_heads;
  runner_params.mNumHeadsKv = num_kv_heads;
  runner_params.mNumHeadsQPerKv = num_qo_heads / num_kv_heads;
  runner_params.mBatchSize = batch_size;
  runner_params.mMaxSeqLenKv = max_kv_len;
  runner_params.mQkvLayout = QkvLayout::SeparateQkv;
  runner_params.mMultiProcessorCount = sm_count;
  runner_params.stream = stream;
  runner_params.outputScale = bmm2_scale;
  runner_params.scaleSoftmaxLog2 = bmm1_scale * M_LOG2E;
  runner_params.mScaleSfO = o_sf_scale;
  runner_params.mChunkedAttentionSize = INT_MAX;  // disable chunked attention by INT_MAX
  runner_params.mAttentionWindowSize =
      window_left == -1 ? INT_MAX : window_left + 1;  // disable window attention by INT_MAX
  runner_params.mMaxSeqLenQ = max_q_len;
  runner_params.mSumOfSeqLensQ = sum_seq_q;
  runner_params.mSumOfSeqLensKv = sum_seq_kv;
  runner_params.cumSeqLensKvPtr = cum_seq_lens_kv;
  runner_params.cumSeqLensQPtr = cum_seq_lens_q;
  runner_params.ptrAttentionSinks = attention_sinks;
  runner_params.enable_pdl = enable_pdl;

  runner_params.kStrideKeysValues = k_stride_keys_values;
  runner_params.kStrideHeads = k_stride_heads;
  runner_params.kStrideBatch = k_stride_batch;
  runner_params.vStrideKeysValues = v_stride_keys_values;
  runner_params.vStrideHeads = v_stride_heads;
  runner_params.vStrideBatch = v_stride_batch;

  runner_params.mKernelType = FmhaKernelType::Context;
  runner_params.mTileScheduler = TileScheduler::Persistent;
  runner_params.mMaskType =
      is_causal ? TrtllmGenAttentionMaskType::Causal : TrtllmGenAttentionMaskType::Dense;
  runner_params.lsePtr = lse;
  size_t max_batch_size = 8192;
  size_t max_num_qo_heads = 256;
  size_t num_semaphores =
      round_up(max_batch_size * max_num_qo_heads, 8);  // max 8MB, should align to 16 bytes
  runner_params.multiCtasKvScratchPtr = reinterpret_cast<void*>(
      static_cast<char*>(workspace_buffer) + num_semaphores * sizeof(uint32_t) +
      sizeof(float2) * num_qo_heads * runner_params.mSumOfSeqLensQ);
  runner_params.multiCtasKvCounterPtr =
      reinterpret_cast<int32_t*>(static_cast<char*>(workspace_buffer) +
                                 sizeof(float2) * num_qo_heads * runner_params.mSumOfSeqLensQ);
  runner_params.softmaxStatsPtr = reinterpret_cast<float2*>(workspace_buffer);

  auto [foundKernels, kinfo] = fmha_runner->isSupportedWithInfo(runner_params);
  if (!foundKernels) {
    std::ostringstream err_msg;
    err_msg << "Missing TRTLLM-GEN kernel ragged attention: " << kinfo;
    FLASHINFER_ERROR(err_msg.str());
  }

  fmha_runner->run(runner_params);
}

void trtllm_ragged_attention(at::Tensor out, at::Tensor query, at::Tensor key, at::Tensor value,
                             at::Tensor workspace_buffer, at::Tensor seq_lens, int64_t max_q_len,
                             int64_t max_kv_len, double bmm1_scale, double bmm2_scale,
                             double o_sf_scale, int64_t batch_size, int64_t window_left,
                             at::Tensor cum_seq_lens_q, at::Tensor cum_seq_lens_kv,
                             int64_t sm_count, bool enable_pdl, bool is_causal,
                             std::optional<at::Tensor> attention_sinks,
                             std::optional<at::Tensor> lse) {
  float* attention_sinks_ptr = nullptr;
  if (attention_sinks) {
    TORCH_CHECK(attention_sinks->scalar_type() == at::ScalarType::Float,
                "attention_sinks must be a float tensor");
    attention_sinks_ptr = attention_sinks->data_ptr<float>();
  }
  float* lse_ptr = nullptr;
  if (lse) {
    TORCH_CHECK(lse->scalar_type() == at::ScalarType::Float, "lse must be a float tensor");
    lse_ptr = lse->data_ptr<float>();
  }
  TORCH_CHECK(out.dim() == 3, "out must be a 3D tensor");
  TORCH_CHECK(query.dim() == 3, "query must be a 3D tensor");
  TORCH_CHECK(key.dim() == 3, "key must be a 3D tensor");
  TORCH_CHECK(value.dim() == 3, "value must be a 3D tensor");

  auto q_data_type = torch_dtype_to_tllm_data_type(query.scalar_type());
  auto kv_data_type = torch_dtype_to_tllm_data_type(key.scalar_type());
  auto o_data_type = torch_dtype_to_tllm_data_type(out.scalar_type());
  auto device = query.device();
  const auto stream = at::cuda::getCurrentCUDAStream(device.index());
  int num_qo_heads = query.size(1);
  int num_kv_heads = key.size(1);
  int sum_seq_q = query.size(0);
  int sum_seq_kv = key.size(0);
  int head_dim_qk = query.size(2);
  int head_dim_v = value.size(2);
  int k_stride_keys_values = key.stride(0);
  int k_stride_heads = key.stride(1);
  int k_stride_batch = key.numel();
  int v_stride_keys_values = value.stride(0);
  int v_stride_heads = value.stride(1);
  int v_stride_batch = value.numel();

  trtllm_ragged_attention_launcher(
      out.data_ptr(), query.data_ptr(), key.data_ptr(), value.data_ptr(),
      workspace_buffer.data_ptr(), static_cast<int*>(seq_lens.data_ptr()),
      static_cast<int*>(cum_seq_lens_q.data_ptr()), static_cast<int*>(cum_seq_lens_kv.data_ptr()),
      attention_sinks_ptr, lse_ptr, q_data_type, kv_data_type, o_data_type, max_q_len, max_kv_len,
      num_qo_heads, num_kv_heads, head_dim_qk, head_dim_v, sum_seq_q, sum_seq_kv, bmm1_scale,
      bmm2_scale, o_sf_scale, batch_size, window_left, sm_count, enable_pdl, is_causal,
      k_stride_keys_values, k_stride_heads, k_stride_batch, v_stride_keys_values, v_stride_heads,
      v_stride_batch, stream);
}

namespace trtllm_cubin_loader {
#include <flashinfer/cubin_loader.h>
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_paged_attention_decode", trtllm_paged_attention_decode);
  m.def("trtllm_paged_attention_context", trtllm_paged_attention_context);
  m.def("trtllm_ragged_attention", trtllm_ragged_attention);
}

}  // namespace flashinfer
