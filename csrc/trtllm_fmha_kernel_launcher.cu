#include "hip/hip_runtime.h"
/*
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <flashinfer/exception.h>
#include <hip/hiprtc.h>

#include <algorithm>
#include <cmath>
#include <flashinfer/semaphore_utils.cuh>
#include <flashinfer/trtllm/fmha/fmhaRunner.cuh>
#include <flashinfer/trtllm/fmha/gen_kernel_launcher.cuh>
#include <iostream>

namespace flashinfer {
template <typename T, Data_type CACHE_T>
void trtllm_paged_attention_launcher(at::Tensor& out, at::Tensor& query,
                                     at::Tensor& key_value_cache, at::Tensor& workspace_buffer,
                                     int64_t num_q_heads, int64_t num_kv_heads,
                                     at::Tensor& block_tables, at::Tensor& seq_lens,
                                     int64_t block_size, int64_t max_seq_len,
                                     const std::string kv_cache_dtype, double bmm1_scale,
                                     double bmm2_scale) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  TORCH_CHECK(num_heads == static_cast<int>(num_q_heads),
              "num_q_heads params and query shape does not match!");
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(-1);

  auto device = query.device();
  const auto stream = at::cuda::getCurrentCUDAStream(device.index());

  uint32_t tokens_per_page = block_size;

  uint32_t num_k_heads = num_kv_heads;
  uint32_t num_v_heads = num_k_heads;
  if (num_heads % num_k_heads != 0) {
    std::ostringstream err_msg;
    err_msg << "num_heads must be a multiple of num_k_heads, got num_k_heads: " << num_k_heads
            << "and num_heads: " << num_heads;
    FLASHINFER_ERROR(err_msg.str());
  }
  auto batch_size = num_seqs;

  int const beam_width = num_seqs / batch_size;  // always 1

  auto q_heads = reinterpret_cast<T*>(query.data_ptr());
  auto output_ptr = reinterpret_cast<T*>(out.data_ptr());

  auto cache_heads = reinterpret_cast<void*>(key_value_cache.data_ptr());

  auto io_type = TypeToDataType<T>::value;

  bool use_multi_block = true;
  auto q_data_type = (kv_cache_dtype == "fp8_e4m3") ? DATA_TYPE_E4M3 : io_type;
  auto output_dtype = io_type;
  static auto fmha_runner = TllmGenFmhaRunner(q_data_type, CACHE_T, io_type);

  TllmGenFmhaRunnerParams runner_params;
  memset(&runner_params, 0, sizeof(runner_params));

  runner_params.mMaskType = TrtllmGenAttentionMaskType::Dense;
  runner_params.mKernelType = FmhaKernelType::Generation;
  runner_params.mTileScheduler =
      use_multi_block ? TileScheduler::Static : TileScheduler::Persistent;
  runner_params.mMultiCtasKvMode = use_multi_block;

  runner_params.qPtr = q_heads;

  runner_params.mQkvLayout = QkvLayout::PagedKv;
  runner_params.kvPtr = cache_heads;
  runner_params.kvPageIdxPtr = block_tables.data_ptr<KVCachePageIndex>();
  runner_params.mMaxNumPagesPerSeqKv = max_num_blocks_per_seq;
  runner_params.mNumTokensPerPage = tokens_per_page;

  // num_kv_heads should be enough, but num_heads for safty at long seq len.
  size_t num_semaphores = batch_size * num_heads;

  runner_params.multiCtasKvScratchPtr = reinterpret_cast<void*>(
      static_cast<char*>(workspace_buffer.data_ptr()) + num_semaphores * sizeof(uint32_t));
  runner_params.multiCtasKvCounterPtr = reinterpret_cast<int32_t*>(workspace_buffer.data_ptr());

  zero_gmem_semaphore_launcher(runner_params.multiCtasKvCounterPtr, num_semaphores,
                               /*enable_pdl=*/true, stream);

  if (head_size != 64 && head_size != 128 && head_size != 192 && head_size != 256) {
    std::ostringstream err_msg;
    err_msg << "head_size " << head_size << " is not supported!";
    FLASHINFER_ERROR(err_msg.str());
  }

  runner_params.seqLensKvPtr = reinterpret_cast<int const*>(seq_lens.data_ptr<int>());

  runner_params.oPtr = output_ptr;
  runner_params.mHeadDimQk = head_size;
  runner_params.mHeadDimV = head_size;
  runner_params.mNumHeadsQ = num_heads;
  runner_params.mNumHeadsKv = num_k_heads;
  runner_params.mNumHeadsQPerKv = num_heads / num_k_heads;
  runner_params.mBatchSize = batch_size;
  runner_params.mMaxSeqLenQ = 1;
  runner_params.mMaxSeqLenKv = max_seq_len;
  runner_params.mSumOfSeqLensQ = int(batch_size * runner_params.mMaxSeqLenQ);
  runner_params.mScaleQ = 1.0;
  // Set the chunked attention size and sliding window size to INT_MAX to disable them when checking
  // if the kernel is supported.
  runner_params.mChunkedAttentionSize = INT_MAX;
  runner_params.mAttentionWindowSize = INT_MAX;
  auto [foundKernels, kinfo] = fmha_runner.isSupportedWithInfo(runner_params);
  if (!foundKernels) {
    std::ostringstream err_msg;
    err_msg << "Missing TRTLLM-GEN decode kernel:" << kinfo;
    FLASHINFER_ERROR(err_msg.str());
  }

  runner_params.mMultiProcessorCount = getMultiProcessorCount();
  auto const [free_memory, total_memory] = getDeviceMemoryInfo(false);
  int max_head_dim_kv = head_size;

  runner_params.mNumPagesInMemPool =
      total_memory / (runner_params.mNumHeadsKv * runner_params.mNumTokensPerPage *
                      max_head_dim_kv * get_size_in_bytes(CACHE_T));

  runner_params.stream = stream;

  runner_params.outputScale = bmm2_scale;
  runner_params.scaleSoftmaxLog2 = bmm1_scale * M_LOG2E;

  fmha_runner.run(runner_params);
}

#define CALL_GEN_LAUNCHER(T, CACHE_T_ENUM)                                                    \
  trtllm_paged_attention_launcher<T, CACHE_T_ENUM>(                                           \
      out, query, key_value_cache, workspace_buffer, num_q_heads, num_kv_heads, block_tables, \
      seq_lens, block_size, max_seq_len, kv_cache_dtype, bmm1_scale, bmm2_scale);

// The following macro is used to dispatch the conversion function based on
// the data type of the key and value cache. The FN is a macro that calls a
// function with template<typename scalar_t, typename cache_t>
#define DISPATCH_BY_KV_CACHE_ELEM_ENUM(SRC_DTYPE, KV_DTYPE, FN)                \
  if (KV_DTYPE == "auto") {                                                    \
    if (SRC_DTYPE == at::ScalarType::Half) {                                   \
      FN(half, Data_type::DATA_TYPE_FP16);                                     \
    } else if (SRC_DTYPE == at::ScalarType::BFloat16) {                        \
      FN(__hip_bfloat16, Data_type::DATA_TYPE_BF16);                            \
    } else {                                                                   \
      TORCH_CHECK(false, "Unsupported input type of kv cache: ", SRC_DTYPE);   \
    }                                                                          \
  } else {                                                                     \
    if (KV_DTYPE == "fp8" || KV_DTYPE == "fp8_e4m3") {                         \
      if (SRC_DTYPE == at::ScalarType::Half) {                                 \
        FN(half, Data_type::DATA_TYPE_E4M3);                                   \
      } else if (SRC_DTYPE == at::ScalarType::BFloat16) {                      \
        FN(__hip_bfloat16, Data_type::DATA_TYPE_E4M3);                          \
      } else {                                                                 \
        TORCH_CHECK(false, "Unsupported input type of kv cache: ", SRC_DTYPE); \
      }                                                                        \
    } else {                                                                   \
      TORCH_CHECK(false, "Unsupported data type of kv cache: ", KV_DTYPE);     \
    }                                                                          \
  }

void trtllm_paged_attention(at::Tensor& out, at::Tensor& query, at::Tensor& key_value_cache,
                            at::Tensor& workspace_buffer, int64_t num_q_heads, int64_t num_kv_heads,
                            at::Tensor& block_tables, at::Tensor& seq_lens, int64_t block_size,
                            int64_t max_seq_len, const std::string kv_cache_dtype,
                            double bmm1_scale, double bmm2_scale) {
  DISPATCH_BY_KV_CACHE_ELEM_ENUM(query.dtype(), kv_cache_dtype, CALL_GEN_LAUNCHER);
}

namespace trtllm_cubin_loader {
#include <flashinfer/cubin_loader.h>
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("trtllm_paged_attention", trtllm_paged_attention);
}

}  // namespace flashinfer
