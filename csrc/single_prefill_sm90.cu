/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/layout.cuh>
#include <flashinfer/math.cuh>
#include <optional>

#include "pytorch_extension_utils.h"
#include "single_prefill_sm90_config.inc"

namespace flashinfer {

template <uint32_t HEAD_DIM, MaskMode MASK_MODE, bool LEFT_SLINDING_WINDOW,
          typename AttentionVariant, typename Params>
hipError_t SinglePrefillWithKVCacheDispatched(Params& params, hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

void single_prefill_with_kv_cache_sm90(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                       at::Tensor o, std::optional<at::Tensor> maybe_lse,
                                       unsigned int mask_mode_code, unsigned int layout,
                                       int32_t window_left ADDITIONAL_FUNC_PARAMS,
                                       int64_t hip_stream) {
  unsigned int head_dim = q.size(2);
  unsigned int num_qo_heads = q.size(1);
  unsigned int qo_len = q.size(0);

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();

  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM, USE_SLIDING_WINDOW, USE_LOGITS_SOFT_CAP,
      AttentionVariant, Params, [&] {
        Params params;
        params.q_ptr = static_cast<DTypeQ*>(q.data_ptr());
        params.k_ptr = static_cast<DTypeKV*>(k.data_ptr());
        params.v_ptr = static_cast<DTypeKV*>(v.data_ptr());
        params.o_ptr = static_cast<DTypeO*>(o.data_ptr());
        params.lse_ptr = maybe_lse ? (static_cast<float*>(maybe_lse->data_ptr())) : nullptr;
        params.q_stride_n = q.stride(0);
        params.q_stride_h = q.stride(1);
        params.o_stride_n = o.stride(0);
        params.o_stride_h = o.stride(1);
        if (kv_layout == QKVLayout::kNHD) {
          params.k_stride_n = k.stride(0);
          params.k_stride_h = k.stride(1);
          params.v_stride_n = v.stride(0);
          params.v_stride_h = v.stride(1);
        } else {
          params.k_stride_h = k.stride(0);
          params.k_stride_n = k.stride(1);
          params.v_stride_h = v.stride(0);
          params.v_stride_n = v.stride(1);
        }
        params.qo_len = q.size(0);
        params.kv_len = k.size(0);
        params.head_dim = head_dim;
        params.num_qo_heads = q.size(1);
        params.num_kv_heads = k.size(1);
        params.causal = mask_mode == MaskMode::kCausal;
        params.group_size = params.num_qo_heads / params.num_kv_heads;
        params.window_left = window_left;

        ADDITIONAL_PARAMS_SETTER

        hipError_t status =
            SinglePrefillWithKVCacheDispatched<HEAD_DIM, MASK_MODE, USE_SLIDING_WINDOW,
                                               AttentionVariant>(params, stream);
        TORCH_CHECK(status == hipSuccess, "single_prefill_with_kv_cache_sm90 failed with error: " +
                                               std::string(hipGetErrorString(status)));
        return true;
      });
}
