/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/blackwell/fmha_cutlass_sm100.cuh>
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/cutlass_utils.cuh>

#include "pytorch_extension_utils.h"

#define DISPATCH_mask_mode(mask_mode, MASK_MODE, ...)   \
  [&]() -> bool {                                       \
    if (mask_mode == MaskMode::kNone) {                 \
      constexpr MaskMode MASK_MODE = MaskMode::kNone;   \
      return __VA_ARGS__();                             \
    } else if (mask_mode == MaskMode::kCausal) {        \
      constexpr MaskMode MASK_MODE = MaskMode::kCausal; \
      return __VA_ARGS__();                             \
    }                                                   \
    return false;                                       \
  }()

#define DISPATCH_head_dim(head_dim_qk, head_dim_vo, HEAD_DIM_QK, HEAD_DIM_VO, ...) \
  [&]() -> bool {                                                                  \
    if (head_dim_qk == 192 && head_dim_vo == 128) {                                \
      constexpr int HEAD_DIM_QK = 192;                                             \
      constexpr int HEAD_DIM_VO = 128;                                             \
      return __VA_ARGS__();                                                        \
    } else if (head_dim_qk == 128 && head_dim_vo == 128) {                         \
      constexpr int HEAD_DIM_QK = 128;                                             \
      constexpr int HEAD_DIM_VO = 128;                                             \
      return __VA_ARGS__();                                                        \
    }                                                                              \
    return false;                                                                  \
  }()

#define DISPATCH_DTYPE_IN_OUT(in_dtype, out_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                              \
    if (in_dtype == out_dtype) {                                               \
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(in_dtype, c_type_in, [&] {   \
        using c_type_out = c_type_in;                                          \
        return __VA_ARGS__();                                                  \
      });                                                                      \
    }                                                                          \
    return false;                                                              \
  }()

#define DISPATCH_context(DTypeIn, DTypeOut, HEAD_DIM_QK, HEAD_DIM_VO, MaskMode, ...)         \
  {                                                                                          \
    DISPATCH_mask_mode(mask_mode, MaskMode, [&] {                                            \
      return DISPATCH_DTYPE_IN_OUT(scalar_type_in, scalar_type_out, DTypeIn, DTypeOut, [&] { \
        return DISPATCH_head_dim(head_dim_qk, head_dim_vo, HEAD_DIM_QK, HEAD_DIM_VO,         \
                                 [&] { return __VA_ARGS__(); });                             \
      });                                                                                    \
    });                                                                                      \
  }

using namespace flashinfer;

void FMHACutlassSM100Run(at::Tensor workspace_buffer, at::Tensor q, at::Tensor k, at::Tensor v,
                         at::Tensor qo_segment_offsets, at::Tensor kv_segment_offsets,
                         at::Tensor work_indptr, at::Tensor qo_tile_indices,
                         at::Tensor qo_head_indices, at::Tensor batch_indices, at::Tensor o,
                         std::optional<at::Tensor> maybe_lse, int64_t mask_mode_code,
                         double sm_scale, int64_t num_qo_heads, int64_t num_kv_heads,
                         int64_t head_dim_qk, int64_t head_dim_vo, int64_t max_qo_len) {
  CHECK(q.scalar_type() == k.scalar_type());
  auto scalar_type_in = q.scalar_type();
  auto scalar_type_out = o.scalar_type();
  MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  int total_qo_len = q.size(0);
  int total_kv_len = k.size(0);
  int batch_size = qo_segment_offsets.size(0) - 1;
  int q_stride_n = q.stride(0);
  int q_stride_h = q.stride(1);
  int k_stride_n = k.stride(0);
  int k_stride_h = k.stride(1);
  int v_stride_n = v.stride(0);
  int v_stride_h = v.stride(1);

  const c10::cuda::OptionalCUDAGuard device_guard(qo_segment_offsets.device());
  const hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  DISPATCH_context(DTypeIn, DTypeOut, HEAD_DIM_QK, HEAD_DIM_VO, MASK_MODE, [&] {
    using cutlass_type_in = cutlass_dtype_t<DTypeIn>;
    using cutlass_type_out = cutlass_dtype_t<DTypeOut>;
    using TILE_Q = _256;
    using TILE_KV = _128;
    using D_QK = cute::Int<HEAD_DIM_QK>;
    using D_VO = cute::Int<HEAD_DIM_VO>;
    using TileShapeQK = Shape<TILE_Q, TILE_KV, D_QK>;
    using TileShapePV = Shape<TILE_Q, D_VO, TILE_KV>;
    using CutlassMaskMode =
        typename std::conditional<MASK_MODE == MaskMode::kCausal, CausalMask, ResidualMask>::type;
    auto status = run_fmha_fwd<cutlass_type_in, cutlass_type_out, int32_t, TileShapeQK, TileShapePV,
                               CutlassMaskMode>(
        workspace_buffer.data_ptr(), static_cast<cutlass_type_in*>(q.data_ptr()),
        static_cast<cutlass_type_in*>(k.data_ptr()), static_cast<cutlass_type_in*>(v.data_ptr()),
        static_cast<int*>(qo_segment_offsets.data_ptr()),
        static_cast<int*>(kv_segment_offsets.data_ptr()), static_cast<int*>(work_indptr.data_ptr()),
        static_cast<int*>(qo_tile_indices.data_ptr()),
        static_cast<int*>(qo_head_indices.data_ptr()), static_cast<int*>(batch_indices.data_ptr()),
        static_cast<cutlass_type_out*>(o.data_ptr()),
        maybe_lse.has_value() ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr,
        mask_mode_code, sm_scale, num_qo_heads, num_kv_heads, head_dim_qk, head_dim_vo, q_stride_n,
        q_stride_h, k_stride_n, k_stride_h, v_stride_n, v_stride_h, batch_size, total_qo_len,
        total_kv_len, max_qo_len, stream);
    TORCH_CHECK(status == hipSuccess, "Cutlass FMHA forward pass failed",
                hipGetErrorString(status));

    return true;
  });
}
