/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/blackwell/fmha_cutlass_sm100.cuh>
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/cutlass_utils.cuh>

#include "tvm_ffi_utils.h"

using tvm::ffi::Optional;

#define DISPATCH_mask_mode(mask_mode, MASK_MODE, ...)   \
  [&]() -> bool {                                       \
    if (mask_mode == MaskMode::kNone) {                 \
      constexpr MaskMode MASK_MODE = MaskMode::kNone;   \
      return __VA_ARGS__();                             \
    } else if (mask_mode == MaskMode::kCausal) {        \
      constexpr MaskMode MASK_MODE = MaskMode::kCausal; \
      return __VA_ARGS__();                             \
    }                                                   \
    return false;                                       \
  }()

#define DISPATCH_head_dim(head_dim_qk, head_dim_vo, HEAD_DIM_QK, HEAD_DIM_VO, ...) \
  [&]() -> bool {                                                                  \
    if (head_dim_qk == 192 && head_dim_vo == 128) {                                \
      constexpr int HEAD_DIM_QK = 192;                                             \
      constexpr int HEAD_DIM_VO = 128;                                             \
      return __VA_ARGS__();                                                        \
    } else if (head_dim_qk == 128 && head_dim_vo == 128) {                         \
      constexpr int HEAD_DIM_QK = 128;                                             \
      constexpr int HEAD_DIM_VO = 128;                                             \
      return __VA_ARGS__();                                                        \
    } else if (head_dim_qk == 64 && head_dim_vo == 64) {                           \
      constexpr int HEAD_DIM_QK = 64;                                              \
      constexpr int HEAD_DIM_VO = 64;                                              \
      return __VA_ARGS__();                                                        \
    }                                                                              \
    return false;                                                                  \
  }()

#define DISPATCH_DTYPE_IN_OUT(in_dtype, out_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                              \
    if (in_dtype == out_dtype) {                                               \
      return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(in_dtype, c_type_in, [&] {    \
        using c_type_out = c_type_in;                                          \
        return __VA_ARGS__();                                                  \
      });                                                                      \
    }                                                                          \
    return false;                                                              \
  }()

#define DISPATCH_context(DTypeIn, DTypeOut, HEAD_DIM_QK, HEAD_DIM_VO, MaskMode, ...)         \
  {                                                                                          \
    DISPATCH_mask_mode(mask_mode, MaskMode, [&] {                                            \
      return DISPATCH_DTYPE_IN_OUT(scalar_type_in, scalar_type_out, DTypeIn, DTypeOut, [&] { \
        return DISPATCH_head_dim(head_dim_qk, head_dim_vo, HEAD_DIM_QK, HEAD_DIM_VO,         \
                                 [&] { return __VA_ARGS__(); });                             \
      });                                                                                    \
    });                                                                                      \
  }

using namespace flashinfer;

void FMHACutlassSM100Run(ffi::Tensor workspace_buffer, ffi::Tensor q, ffi::Tensor k, ffi::Tensor v,
                         ffi::Tensor qo_segment_offsets, ffi::Tensor kv_segment_offsets,
                         ffi::Tensor work_indptr, ffi::Tensor qo_tile_indices,
                         ffi::Tensor qo_head_indices, ffi::Tensor batch_indices, ffi::Tensor o,
                         Optional<ffi::Tensor> maybe_lse, int64_t mask_mode_code, double sm_scale,
                         int64_t num_qo_heads, int64_t num_kv_heads, int64_t head_dim_qk,
                         int64_t head_dim_vo, int64_t max_qo_len) {
  TVM_FFI_ICHECK_EQ(q->dtype, k->dtype);
  auto scalar_type_in = q->dtype;
  auto scalar_type_out = o->dtype;
  MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  int total_qo_len = q->shape[0];
  int total_kv_len = k->shape[0];
  int batch_size = qo_segment_offsets->shape[0] - 1;
  int q_stride_n = q->strides[0];
  int q_stride_h = q->strides[1];
  int k_stride_n = k->strides[0];
  int k_stride_h = k->strides[1];
  int v_stride_n = v->strides[0];
  int v_stride_h = v->strides[1];

  hipSetDevice(qo_segment_offsets->device.device_id);
  const hipStream_t stream = get_stream(o->device);

  DISPATCH_context(DTypeIn, DTypeOut, HEAD_DIM_QK, HEAD_DIM_VO, MASK_MODE, [&] {
    using cutlass_type_in = cutlass_dtype_t<DTypeIn>;
    using cutlass_type_out = cutlass_dtype_t<DTypeOut>;
    using TILE_Q = _256;
    using TILE_KV = _128;
    using D_QK = cute::Int<HEAD_DIM_QK>;
    using D_VO = cute::Int<HEAD_DIM_VO>;
    using TileShapeQK = Shape<TILE_Q, TILE_KV, D_QK>;
    using TileShapePV = Shape<TILE_Q, D_VO, TILE_KV>;
    using CutlassMaskMode =
        typename std::conditional<MASK_MODE == MaskMode::kCausal, CausalMask, ResidualMask>::type;
    auto status = run_fmha_fwd<cutlass_type_in, cutlass_type_out, int32_t, TileShapeQK, TileShapePV,
                               CutlassMaskMode>(
        workspace_buffer->data, static_cast<cutlass_type_in*>(q->data),
        static_cast<cutlass_type_in*>(k->data), static_cast<cutlass_type_in*>(v->data),
        static_cast<int*>(qo_segment_offsets->data), static_cast<int*>(kv_segment_offsets->data),
        static_cast<int*>(work_indptr->data), static_cast<int*>(qo_tile_indices->data),
        static_cast<int*>(qo_head_indices->data), static_cast<int*>(batch_indices->data),
        static_cast<cutlass_type_out*>(o->data),
        maybe_lse.has_value() ? static_cast<float*>(maybe_lse.value()->data) : nullptr,
        mask_mode_code, sm_scale, num_qo_heads, num_kv_heads, head_dim_qk, head_dim_vo, q_stride_n,
        q_stride_h, k_stride_n, k_stride_h, v_stride_n, v_stride_h, batch_size, total_qo_len,
        total_kv_len, max_qo_len, stream);
    TVM_FFI_ICHECK_EQ(status, hipSuccess)
        << "Cutlass FMHA forward pass failed" << hipGetErrorString(status);

    return true;
  });
}
