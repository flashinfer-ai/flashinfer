/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/scheduler.cuh>
#include <optional>

#include "batch_mla_config.inc"
#include "pytorch_conversion_utils.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

at::Tensor BatchMLAPagedAttentionPlan(at::Tensor float_workspace_buffer,
                                      at::Tensor int_workspace_buffer,
                                      at::Tensor page_locked_int_workspace_buffer,
                                      at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor kv_len,
                                      int64_t num_heads, int64_t head_dim_o, bool causal,
                                      int64_t hip_stream) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();

  MLAPlanInfo plan_info;

  int batch_size = kv_len.size(0);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status =
      MLAPlan(float_workspace_buffer.data_ptr(), float_workspace_size_in_bytes,
              int_workspace_buffer.data_ptr(), page_locked_int_workspace_buffer.data_ptr(),
              int_workspace_size_in_bytes, plan_info, static_cast<IdType*>(qo_indptr.data_ptr()),
              static_cast<IdType*>(kv_indptr.data_ptr()), static_cast<IdType*>(kv_len.data_ptr()),
              batch_size, num_heads, head_dim_o, causal, stream);

  TORCH_CHECK(status == hipSuccess, "Failed to plan MLA, error: ", hipGetErrorString(status));

  return vec_to_tensor(plan_info.ToVector());
}
