/*
 * Copyright (c) 2023-2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/pos_enc.cuh>
#include <optional>

#include "pytorch_extension_utils.h"
#include "single_decode_config.inc"

namespace flashinfer {

template <uint32_t HEAD_DIM, PosEncodingMode POS_ENCODING_MODE, typename AttentionVariant,
          typename Params>
hipError_t SingleDecodeWithKVCacheDispatched(Params params, typename Params::DTypeO* tmp,
                                              hipStream_t stream);
}  // namespace flashinfer

using namespace flashinfer;

void single_decode_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                 at::Tensor o, unsigned int layout,
                                 int window_left ADDITIONAL_FUNC_PARAMS, int64_t hip_stream) {
  CHECK_INPUT(q);
  CHECK_INPUT(k);
  CHECK_INPUT(v);
  CHECK_INPUT(tmp);
  auto device = q.device();
  CHECK_EQ(k.device(), device);
  CHECK_EQ(v.device(), device);
  CHECK_EQ(tmp.device(), device);
  CHECK_DIM(2, q);
  CHECK_DIM(3, k);
  CHECK_DIM(3, v);
  CHECK_SHAPE(k, v);
  CHECK_EQ(q.size(1), k.size(2));
  CHECK_EQ(v.scalar_type(), k.scalar_type());
  unsigned int num_qo_heads = q.size(0);
  unsigned int head_dim = q.size(1);
  unsigned int kv_len, num_kv_heads;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  if (kv_layout == QKVLayout::kNHD) {
    kv_len = k.size(0);
    num_kv_heads = k.size(1);
  } else {
    num_kv_heads = k.size(0);
    kv_len = k.size(1);
  }
  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k.scalar_type();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, HEAD_DIM, POS_ENCODING_MODE, USE_SLIDING_WINDOW,
      USE_LOGITS_SOFT_CAP, AttentionVariant, Params, [&] {
        Params params;

        params.q = static_cast<DTypeQ*>(q.data_ptr());
        params.k = static_cast<DTypeKV*>(k.data_ptr());
        params.v = static_cast<DTypeKV*>(v.data_ptr());
        params.o = static_cast<DTypeO*>(o.data_ptr());
        params.lse = nullptr;
        params.kv_len = kv_len;
        params.num_qo_heads = num_qo_heads;
        params.num_kv_heads = num_kv_heads;
        params.q_stride_n = num_qo_heads * head_dim;
        params.q_stride_h = head_dim;
        params.kv_stride_n = (kv_layout == QKVLayout::kNHD) ? num_kv_heads * head_dim : head_dim;
        params.kv_stride_h = (kv_layout == QKVLayout::kNHD) ? head_dim : kv_len * head_dim;
        params.head_dim = head_dim;
        params.window_left = window_left;
        params.kv_chunk_size = 0;

        ADDITIONAL_PARAMS_SETTER

        hipError_t status =
            flashinfer::SingleDecodeWithKVCacheDispatched<HEAD_DIM, POS_ENCODING_MODE,
                                                          AttentionVariant>(
                params, static_cast<DTypeO*>(tmp.data_ptr()), stream);
        TORCH_CHECK(status == hipSuccess, "SingleDecodeWithKVCache kernel launch failed, error: " +
                                               std::string(hipGetErrorString(status)));
        return true;
      });
}
