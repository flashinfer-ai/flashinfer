/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/gemm/group_gemm_sm90.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;
using namespace flashinfer::group_gemm;

void CutlassSegmentGEMMSM90(at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
                            at::Tensor all_problems, at::Tensor x_ptr, at::Tensor w_ptr,
                            at::Tensor y_ptr, at::Tensor x_stride, at::Tensor weight_stride,
                            at::Tensor y_stride, at::Tensor empty_x_data, bool weight_column_major,
                            int64_t hip_stream) {
  unsigned int batch_size = x_ptr.size(0);
  auto device = float_workspace_buffer.device();

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE(empty_x_data.scalar_type(), c_type, [&] {
    using cutlass_t = cutlass_dtype_t<c_type>;
    auto status = CutlassSegmentGEMMSM90Run<cutlass_t, cutlass_t>(
        float_workspace_buffer.data_ptr(),
        float_workspace_buffer.element_size() * float_workspace_buffer.size(0),
        int_workspace_buffer.data_ptr(),
        int_workspace_buffer.element_size() * int_workspace_buffer.size(0), all_problems.data_ptr(),
        batch_size, x_ptr.data_ptr(), w_ptr.data_ptr(), y_ptr.data_ptr(), x_stride.data_ptr(),
        weight_stride.data_ptr(), y_stride.data_ptr(), weight_column_major, stream);
    TORCH_CHECK(status == hipSuccess,
                "Failed to run CutlassSegmentGEMM: ", hipGetErrorString(status));
    return true;
  });
}
