/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/cutlass_utils.cuh>

#include "tvm_ffi_utils.h"

using namespace flashinfer;

#define DISPATCH_DLPACK_INPUT_OUTPUT_DTYPE(input_dtype, output_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                                                 \
    if (input_dtype == output_dtype) {                                                            \
      return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(input_dtype, c_type_in, [&] {                    \
        using c_type_out = c_type_in;                                                             \
        return __VA_ARGS__();                                                                     \
      });                                                                                         \
    } else {                                                                                      \
      return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP16(output_dtype, c_type_out, [&] {                  \
        return DISPATCH_DLPACK_DTYPE_TO_CTYPE_FP8(input_dtype, c_type_in,                         \
                                                  [&] { return __VA_ARGS__(); });                 \
      });                                                                                         \
    }                                                                                             \
  }()

namespace flashinfer {
namespace group_gemm {

template <typename DTypeIn, typename DTypeOut>
hipError_t CutlassSegmentGEMMSM90Run(void* float_buffer, size_t float_buffer_size_in_bytes,
                                      void* int_buffer, size_t int_buffer_size_in_bytes,
                                      void* all_problems, int64_t batch_size, void* x, void* w,
                                      void* y, void* x_stride, void* w_stride, void* y_stride,
                                      bool weight_column_major, hipStream_t stream);

}  // namespace group_gemm
}  // namespace flashinfer

void CutlassSegmentGEMMSM90(TensorView float_workspace_buffer, TensorView int_workspace_buffer,
                            TensorView all_problems, TensorView x_ptr, TensorView w_ptr,
                            TensorView y_ptr, TensorView x_stride, TensorView weight_stride,
                            TensorView y_stride, TensorView empty_x_data, TensorView empty_y_data,
                            bool weight_column_major) {
  unsigned int batch_size = x_ptr->shape[0];
  hipSetDevice(float_workspace_buffer->device.device_id);
  const hipStream_t stream = get_stream(float_workspace_buffer->device);
  DISPATCH_DLPACK_INPUT_OUTPUT_DTYPE(
      empty_x_data->dtype, empty_y_data->dtype, c_type_in, c_type_out, [&] {
        using cutlass_t_in = cutlass_dtype_t<c_type_in>;
        using cutlass_t_out = cutlass_dtype_t<c_type_out>;
        auto status =
            flashinfer::group_gemm::CutlassSegmentGEMMSM90Run<cutlass_t_in, cutlass_t_out>(
                float_workspace_buffer->data,
                get_element_size(float_workspace_buffer) * float_workspace_buffer->shape[0],
                int_workspace_buffer->data,
                get_element_size(int_workspace_buffer) * int_workspace_buffer->shape[0],
                all_problems->data, batch_size, x_ptr->data, w_ptr->data, y_ptr->data,
                x_stride->data, weight_stride->data, y_stride->data, weight_column_major, stream);
        TVM_FFI_ICHECK(status == hipSuccess)
            << "Failed to run CutlassSegmentGEMM: " << hipGetErrorString(status);
        return true;
      });
}
