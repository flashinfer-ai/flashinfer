/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/cutlass_utils.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;

#define DISPATCH_PYTORCH_INPUT_OUTPUT_DTYPE(input_dtype, output_dtype, c_type_in, c_type_out, ...) \
  [&]() -> bool {                                                                                  \
    if (input_dtype == output_dtype) {                                                             \
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input_dtype, c_type_in, [&] {                    \
        using c_type_out = c_type_in;                                                              \
        return __VA_ARGS__();                                                                      \
      });                                                                                          \
    } else {                                                                                       \
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(output_dtype, c_type_out, [&] {                  \
        return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(input_dtype, c_type_in,                         \
                                                   [&] { return __VA_ARGS__(); });                 \
      });                                                                                          \
    }                                                                                              \
  }()

namespace flashinfer {
namespace group_gemm {

template <typename DTypeIn, typename DTypeOut>
hipError_t CutlassSegmentGEMMSM90Run(void* float_buffer, size_t float_buffer_size_in_bytes,
                                      void* int_buffer, size_t int_buffer_size_in_bytes,
                                      void* all_problems, int64_t batch_size, void* x, void* w,
                                      void* y, void* x_stride, void* w_stride, void* y_stride,
                                      bool weight_column_major, hipStream_t stream);

}  // namespace group_gemm
}  // namespace flashinfer

void CutlassSegmentGEMMSM90(at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
                            at::Tensor all_problems, at::Tensor x_ptr, at::Tensor w_ptr,
                            at::Tensor y_ptr, at::Tensor x_stride, at::Tensor weight_stride,
                            at::Tensor y_stride, at::Tensor empty_x_data, at::Tensor empty_y_data,
                            bool weight_column_major) {
  unsigned int batch_size = x_ptr.size(0);
  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer.device());
  auto stream = at::cuda::getCurrentCUDAStream();
  DISPATCH_PYTORCH_INPUT_OUTPUT_DTYPE(
      empty_x_data.scalar_type(), empty_y_data.scalar_type(), c_type_in, c_type_out, [&] {
        using cutlass_t_in = cutlass_dtype_t<c_type_in>;
        using cutlass_t_out = cutlass_dtype_t<c_type_out>;
        auto status =
            flashinfer::group_gemm::CutlassSegmentGEMMSM90Run<cutlass_t_in, cutlass_t_out>(
                float_workspace_buffer.data_ptr(),
                float_workspace_buffer.element_size() * float_workspace_buffer.size(0),
                int_workspace_buffer.data_ptr(),
                int_workspace_buffer.element_size() * int_workspace_buffer.size(0),
                all_problems.data_ptr(), batch_size, x_ptr.data_ptr(), w_ptr.data_ptr(),
                y_ptr.data_ptr(), x_stride.data_ptr(), weight_stride.data_ptr(),
                y_stride.data_ptr(), weight_column_major, stream);
        TORCH_CHECK(status == hipSuccess,
                    "Failed to run CutlassSegmentGEMM: ", hipGetErrorString(status));
        return true;
      });
}
