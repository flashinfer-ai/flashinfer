/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "aot_extension_utils.h"

void CutlassSegmentGEMMSM90(at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
                            at::Tensor all_problems, at::Tensor x_ptr, at::Tensor w_ptr,
                            at::Tensor y_ptr, at::Tensor x_stride, at::Tensor weight_stride,
                            at::Tensor y_stride, at::Tensor empty_x_data, bool weight_column_major,
                            int64_t hip_stream);

void single_prefill_with_kv_cache_sm90(unsigned int mask_mode_code, at::Tensor q, at::Tensor k,
                                       at::Tensor v,
                                       std::optional<at::Tensor> maybe_packed_custom_mask,
                                       std::optional<at::Tensor> maybe_alibi_slopes, at::Tensor o,
                                       unsigned int layout, int32_t window_left,
                                       float logits_soft_cap, float sm_scale, float rope_scale,
                                       float rope_theta, std::optional<at::Tensor> maybe_lse,
                                       int64_t hip_stream);

std::vector<int64_t> BatchPrefillWithKVCacheSM90Plan(
    unsigned int head_dim, bool causal, at::Tensor float_workspace_buffer,
    at::Tensor int_workspace_buffer, at::Tensor page_locked_int_workspace_buffer,
    at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor kv_len_arr, unsigned int total_num_rows,
    unsigned int batch_size, unsigned int num_qo_heads, unsigned int num_kv_heads,
    unsigned int page_size, bool enable_cuda_graph, int64_t hip_stream);

void BatchPrefillWithRaggedKVCacheSM90Run(
    unsigned int mask_mode_code, at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor k, at::Tensor v,
    std::optional<at::Tensor> maybe_custom_mask, std::optional<at::Tensor> maybe_alibi_slopes,
    at::Tensor qo_indptr, at::Tensor kv_indptr, std::optional<at::Tensor> maybe_qk_indptr,
    at::Tensor o, unsigned int layout, int32_t window_left, float logits_soft_cap, float sm_scale,
    float rope_scale, float rope_theta, std::optional<at::Tensor> maybe_lse, int64_t hip_stream);

void BatchPrefillWithPagedKVCacheSM90Run(
    unsigned int mask_mode_code, at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, std::optional<at::Tensor> maybe_custom_mask,
    std::optional<at::Tensor> maybe_alibi_slopes, at::Tensor qo_indptr, at::Tensor paged_kv_indptr,
    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len,
    std::optional<at::Tensor> maybe_qk_indptr, at::Tensor o, unsigned int layout,
    int32_t window_left, float logits_soft_cap, float sm_scale, float rope_scale, float rope_theta,
    std::optional<at::Tensor> maybe_lse, int64_t hip_stream);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("cutlass_segment_gemm_sm90", &CutlassSegmentGEMMSM90,
        "Cutlass Segment GEMM operator for SM90");
  m.def("single_prefill_with_kv_cache_sm90", &single_prefill_with_kv_cache_sm90);
  m.def("batch_prefill_with_kv_cache_sm90_plan", &BatchPrefillWithKVCacheSM90Plan);
  m.def("batch_prefill_with_ragged_kv_cache_sm90_run", &BatchPrefillWithRaggedKVCacheSM90Run);
  m.def("batch_prefill_with_paged_kv_cache_sm90_run", &BatchPrefillWithPagedKVCacheSM90Run);
}
