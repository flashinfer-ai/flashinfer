/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/mla.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/fastdiv.cuh>
#include <optional>

#include "batch_mla_config.inc"
#include "pytorch_conversion_utils.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

void BatchMLAPagedAttentionRun(at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
                               at::Tensor plan_info_vec, at::Tensor q_nope, at::Tensor q_pe,
                               at::Tensor ckv_cache, at::Tensor kpe_cache, at::Tensor kv_indices,
                               at::Tensor o, std::optional<at::Tensor> maybe_lse,
                               int64_t mask_mode_code, int64_t num_heads, int64_t page_size,
                               double sm_scale, int64_t hip_stream) {
  // q_nope: [n, num_heads, head_dim_ckv]
  // q_pe: [n, num_heads, head_dim_kpe]
  // ckv_cache: [num_pages, page_size, head_dim_ckv]
  // kpe_cache: [num_pages, page_size, head_dim_kpe]
  MLAPlanInfo plan_info;
  plan_info.FromVector(tensor_to_vec(plan_info_vec));

  auto device = q_nope.device();

  void* float_buffer_ptr = float_workspace_buffer.data_ptr();
  void* int_buffer_ptr = int_workspace_buffer.data_ptr();

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  auto q_scalar_type = q_nope.scalar_type();
  auto kv_scalar_type = ckv_cache.scalar_type();

  unsigned int q_nope_stride_n = q_nope.stride(0);
  unsigned int q_nope_stride_h = q_nope.stride(1);
  unsigned int q_pe_stride_n = q_pe.stride(0);
  unsigned int q_pe_stride_h = q_pe.stride(1);
  unsigned int ckv_stride_page = ckv_cache.stride(0);
  unsigned int ckv_stride_n = ckv_cache.stride(1);
  unsigned int kpe_stride_page = kpe_cache.stride(0);
  unsigned int kpe_stride_n = kpe_cache.stride(1);
  unsigned int o_stride_n = o.stride(0);
  unsigned int o_stride_h = o.stride(1);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_CKV, HEAD_DIM_KPE, Params, [&] {
        Params params;

        params.q_nope = static_cast<DTypeQ*>(q_nope.data_ptr());
        params.q_pe = static_cast<DTypeQ*>(q_pe.data_ptr());
        params.ckv = static_cast<DTypeKV*>(ckv_cache.data_ptr());
        params.kpe = static_cast<DTypeKV*>(kpe_cache.data_ptr());

        params.q_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.q_indptr_offset);
        params.kv_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_indptr_offset);
        params.partial_indptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.partial_indptr_offset);
        params.kv_indices = static_cast<IdType*>(kv_indices.data_ptr());
        params.q_len = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.q_len_offset);
        params.kv_len = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_len_offset);
        params.q_start = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.q_start_offset);
        params.kv_start = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_start_offset);
        params.kv_end = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_end_offset);
        params.work_indptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.work_indptr_offset);
        params.merge_packed_offset_start = GetPtrFromBaseOffset<IdType>(
            int_buffer_ptr, plan_info.merge_packed_offset_start_offset);
        params.merge_packed_offset_end =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_packed_offset_end_offset);
        params.merge_partial_packed_offset_start = GetPtrFromBaseOffset<IdType>(
            int_buffer_ptr, plan_info.merge_partial_packed_offset_start_offset);
        params.merge_partial_packed_offset_end = GetPtrFromBaseOffset<IdType>(
            int_buffer_ptr, plan_info.merge_partial_packed_offset_end_offset);
        params.merge_partial_stride =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_partial_stride_offset);
        params.final_o = static_cast<DTypeO*>(o.data_ptr());
        params.final_lse =
            maybe_lse.has_value() ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
        params.partial_o =
            GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.partial_o_offset);
        params.partial_lse =
            GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.partial_lse_offset);

        params.num_heads = uint_fastdiv(num_heads);
        params.block_size = uint_fastdiv(page_size);

        params.q_nope_stride_n = q_nope_stride_n;
        params.q_nope_stride_h = q_nope_stride_h;
        params.q_pe_stride_n = q_pe_stride_n;
        params.q_pe_stride_h = q_pe_stride_h;
        params.ckv_stride_page = ckv_stride_page;
        params.ckv_stride_n = ckv_stride_n;
        params.kpe_stride_page = kpe_stride_page;
        params.kpe_stride_n = kpe_stride_n;
        params.o_stride_n = o_stride_n;
        params.o_stride_h = o_stride_h;

        params.sm_scale = sm_scale;

        hipError_t status = mla::BatchMLAPagedAttention<MASK_MODE, HEAD_DIM_CKV, HEAD_DIM_KPE>(
            params, plan_info.num_blks_x, plan_info.num_blks_y, stream);

        TORCH_CHECK(status == hipSuccess,
                    "Failed to run MLA, error: ", hipGetErrorString(status));
      });
}
