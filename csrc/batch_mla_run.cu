/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cstdint>
#include <flashinfer/attention/mla.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/fastdiv.cuh>

#include "batch_mla_config.inc"
#include "tvm/ffi/container/array.h"
#include "tvm_ffi_utils.h"

using namespace flashinfer;

using tvm::ffi::Array;
using tvm::ffi::Optional;

void BatchMLAPagedAttentionRun(TensorView float_workspace_buffer, TensorView int_workspace_buffer,
                               Array<int64_t> plan_info_vec, TensorView q_nope, TensorView q_pe,
                               TensorView ckv_cache, TensorView kpe_cache, TensorView kv_indices,
                               TensorView o, Optional<TensorView> maybe_lse, int64_t mask_mode_code,
                               int64_t num_heads, int64_t page_size, double sm_scale) {
  // q_nope: [n, num_heads, head_dim_ckv]
  // q_pe: [n, num_heads, head_dim_kpe]
  // ckv_cache: [num_pages, page_size, head_dim_ckv]
  // kpe_cache: [num_pages, page_size, head_dim_kpe]
  MLAPlanInfo plan_info;
  plan_info.FromVector(std::vector<int64_t>(plan_info_vec.begin(), plan_info_vec.end()));

  void* float_buffer_ptr = float_workspace_buffer->data;
  void* int_buffer_ptr = int_workspace_buffer->data;

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  unsigned int q_nope_stride_n = q_nope->strides[0];
  unsigned int q_nope_stride_h = q_nope->strides[1];
  unsigned int q_pe_stride_n = q_pe->strides[0];
  unsigned int q_pe_stride_h = q_pe->strides[1];
  unsigned int ckv_stride_page = ckv_cache->strides[0];
  unsigned int ckv_stride_n = ckv_cache->strides[1];
  unsigned int kpe_stride_page = kpe_cache->strides[0];
  unsigned int kpe_stride_n = kpe_cache->strides[1];
  unsigned int o_stride_n = o->strides[0];
  unsigned int o_stride_h = o->strides[1];

  hipSetDevice(q_nope->device.device_id);
  const hipStream_t stream = get_stream(q_nope->device);

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_CKV, HEAD_DIM_KPE, Params, [&] {
        Params params;

        params.q_nope = static_cast<DTypeQ*>(q_nope->data);
        params.q_pe = static_cast<DTypeQ*>(q_pe->data);
        params.ckv = static_cast<DTypeKV*>(ckv_cache->data);
        params.kpe = static_cast<DTypeKV*>(kpe_cache->data);

        params.q_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.q_indptr_offset);
        params.kv_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_indptr_offset);
        params.partial_indptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.partial_indptr_offset);
        params.kv_indices = static_cast<IdType*>(kv_indices->data);
        params.q_len = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.q_len_offset);
        params.kv_len = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_len_offset);
        params.q_start = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.q_start_offset);
        params.kv_start = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_start_offset);
        params.kv_end = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_end_offset);
        params.work_indptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.work_indptr_offset);
        params.merge_packed_offset_start = GetPtrFromBaseOffset<IdType>(
            int_buffer_ptr, plan_info.merge_packed_offset_start_offset);
        params.merge_packed_offset_end =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_packed_offset_end_offset);
        params.merge_partial_packed_offset_start = GetPtrFromBaseOffset<IdType>(
            int_buffer_ptr, plan_info.merge_partial_packed_offset_start_offset);
        params.merge_partial_packed_offset_end = GetPtrFromBaseOffset<IdType>(
            int_buffer_ptr, plan_info.merge_partial_packed_offset_end_offset);
        params.merge_partial_stride =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_partial_stride_offset);
        params.final_o = static_cast<DTypeO*>(o->data);
        params.final_lse =
            maybe_lse.has_value() ? static_cast<float*>(maybe_lse.value()->data) : nullptr;
        params.partial_o =
            GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.partial_o_offset);
        params.partial_lse =
            GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.partial_lse_offset);

        params.num_heads = uint_fastdiv(num_heads);
        params.block_size = uint_fastdiv(page_size);

        params.q_nope_stride_n = q_nope_stride_n;
        params.q_nope_stride_h = q_nope_stride_h;
        params.q_pe_stride_n = q_pe_stride_n;
        params.q_pe_stride_h = q_pe_stride_h;
        params.ckv_stride_page = ckv_stride_page;
        params.ckv_stride_n = ckv_stride_n;
        params.kpe_stride_page = kpe_stride_page;
        params.kpe_stride_n = kpe_stride_n;
        params.o_stride_n = o_stride_n;
        params.o_stride_h = o_stride_h;

        params.sm_scale = sm_scale;

        hipError_t status = mla::BatchMLAPagedAttention<MASK_MODE, HEAD_DIM_CKV, HEAD_DIM_KPE>(
            params, plan_info.num_blks_x, plan_info.num_blks_y, stream);

        TVM_FFI_ICHECK(status == hipSuccess)
            << "Failed to run MLA, error: " << hipGetErrorString(status);
      });
}
