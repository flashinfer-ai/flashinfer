#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2025 Perplexity AI
 */
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>

#include <cstdint>
#include <cstdlib>
#include <string>
#include <vector>

#include "tvm_ffi_utils.h"

#define NVSHMEMCHECK(stmt)                                                                    \
  do {                                                                                        \
    int result = (stmt);                                                                      \
    if (NVSHMEMX_SUCCESS != result) {                                                         \
      fprintf(stderr, "[%s:%d] nvshmem failed with error %d \n", __FILE__, __LINE__, result); \
      exit(-1);                                                                               \
    }                                                                                         \
  } while (0)

namespace {

constexpr int nvshmemx_uniqueid_t_size = sizeof(nvshmemx_uniqueid_t);

using tvm::ffi::Array;
using tvm::ffi::Shape;

void get_unique_id(Tensor uid) {
  CHECK_CONTIGUOUS(uid);
  TVM_FFI_ICHECK_EQ(get_numel(uid) * get_element_size(uid), nvshmemx_uniqueid_t_size);
  TVM_FFI_ICHECK_EQ(uid->device.device_type, kDLCPU);
  nvshmemx_uniqueid_t* uid_ptr = reinterpret_cast<nvshmemx_uniqueid_t*>(uid->data);
  *uid_ptr = NVSHMEMX_UNIQUEID_INITIALIZER;
  nvshmemx_get_uniqueid(uid_ptr);
}

int64_t unique_id_size() { return nvshmemx_uniqueid_t_size; }

int64_t init(Tensor uid, int64_t rank, int64_t world_size) {
  CHECK_CONTIGUOUS(uid);
  TVM_FFI_ICHECK_EQ(get_numel(uid) * get_element_size(uid), nvshmemx_uniqueid_t_size);
  TVM_FFI_ICHECK_EQ(uid->device.device_type, kDLCPU);
  nvshmemx_uniqueid_t* uid_ptr = reinterpret_cast<nvshmemx_uniqueid_t*>(uid->data);
  nvshmemx_init_attr_t attr = NVSHMEMX_INIT_ATTR_INITIALIZER;
  nvshmemx_set_attr_uniqueid_args(rank, world_size, uid_ptr, &attr);
  return nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr);
}

void finalize() { nvshmem_finalize(); }

int64_t my_pe() { return nvshmem_my_pe(); }

int64_t n_pes() { return nvshmem_n_pes(); }

struct NVSHMEMNDAlloc {
  void AllocData(DLTensor* tensor) {
    size_t size = tvm::ffi::GetDataSize(*tensor);
    tensor->data = nvshmem_malloc(size);
    TVM_FFI_ICHECK_NE(tensor->data, nullptr) << "nvshmem_malloc failed. size: " << size;
  }
  void FreeData(DLTensor* tensor) { nvshmem_free(tensor->data); }
};

Tensor malloc_tensor(Shape shape, DLDataType dtype, int device_id) {
  return Tensor::FromNDAlloc(NVSHMEMNDAlloc(), tvm::ffi::Shape(shape), dtype,
                             DLDevice{kDLCUDA, device_id});
}

void barrier_all() { nvshmem_barrier_all(); }

void barrier_all_on_current_stream() {
  hipStream_t stream = get_current_stream();
  nvshmemx_barrier_all_on_stream(stream);
}

void alltoall(Tensor dest, Tensor source) {
  CHECK_CONTIGUOUS(dest);
  CHECK_CONTIGUOUS(source);
  TVM_FFI_ICHECK_EQ(dest->dtype, source->dtype) << "dest and source must have the same dtype";

  size_t nbytes = get_numel(dest) * get_element_size(dest) / dest->shape[0];
  hipStream_t stream = get_stream(dest->device);
  NVSHMEMCHECK(nvshmemx_alltoallmem_on_stream(NVSHMEM_TEAM_WORLD, static_cast<uint8_t*>(dest->data),
                                              static_cast<uint8_t*>(source->data), nbytes, stream));
}

void fake_alltoall(Tensor dest, Tensor source) {}

void sum_reduce(Tensor dest, Tensor source, int64_t nelems) {
  CHECK_CONTIGUOUS(dest);
  CHECK_CONTIGUOUS(source);
  TVM_FFI_ICHECK_EQ(dest->dtype, source->dtype) << "dest and source must have the same dtype";

  // Add validation and conversion
  TVM_FFI_ICHECK_GE(nelems, 0) << "nelems must be non-negative, got " << nelems;
  TVM_FFI_ICHECK_LE(nelems, SIZE_MAX) << "nelems too large: " << nelems << " > " << SIZE_MAX;
  size_t nelems_size_t = static_cast<size_t>(nelems);

  hipStream_t stream = get_stream(dest->device);

  switch (encode_dlpack_dtype(dest->dtype)) {
    case float16_code:  // float16
      NVSHMEMCHECK(nvshmemx_half_sum_reduce_on_stream(
          NVSHMEM_TEAM_WORLD, static_cast<nv_half*>(dest->data),
          static_cast<nv_half*>(source->data), nelems_size_t, stream));
      break;
    case float32_code:  // float32
      NVSHMEMCHECK(nvshmemx_float_sum_reduce_on_stream(
          NVSHMEM_TEAM_WORLD, static_cast<float*>(dest->data), static_cast<float*>(source->data),
          nelems_size_t, stream));
      break;
    case bfloat16_code:  // bfloat16
      NVSHMEMCHECK(nvshmemx_bfloat16_sum_reduce_on_stream(
          NVSHMEM_TEAM_WORLD, static_cast<hip_bfloat16*>(dest->data),
          static_cast<hip_bfloat16*>(source->data), nelems_size_t, stream));
      break;

    default:
      TVM_FFI_LOG_AND_THROW(NotImplementedError)
          << "Unsupported dtype for nvshmem_sum_reduce: " << dest->dtype;
  }
}

void fake_sum_reduce(Tensor dest, Tensor source, int64_t nelems) {}

void allreduce_on_stream_with_copy(Tensor dest_symm, Tensor source_symm, Tensor dest_local,
                                   Tensor source_local, int64_t nelems) {
  CHECK_CONTIGUOUS(dest_symm);
  CHECK_CONTIGUOUS(source_symm);
  CHECK_CONTIGUOUS(dest_local);
  CHECK_CONTIGUOUS(source_local);
  TVM_FFI_ICHECK_EQ(dest_symm->dtype, source_symm->dtype)
      << "dest_symm and source_symm must have the same dtype";
  TVM_FFI_ICHECK_EQ(dest_symm->dtype, source_local->dtype)
      << "dest_symm and source_local must have the same dtype";
  TVM_FFI_ICHECK_EQ(dest_local->dtype, source_local->dtype)
      << "dest_local and source_local must have the same dtype";

  hipStream_t stream = get_stream(source_symm->device);

  hipMemcpyAsync(source_symm->data, source_local->data, nelems * get_element_size(source_local),
                  hipMemcpyDefault, stream);
  nvshmemx_barrier_on_stream(NVSHMEM_TEAM_WORLD, stream);
  sum_reduce(dest_symm, source_symm, nelems);
  hipMemcpyAsync(dest_local->data, dest_symm->data, nelems * get_element_size(dest_local),
                  hipMemcpyDefault, stream);
  hipStreamSynchronize(stream);
}

void fake_allreduce_on_stream_with_copy(Tensor dest_symm, Tensor source_symm, Tensor dest_local,
                                        Tensor source_local, int64_t nelems) {}

TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_get_unique_id, get_unique_id);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_unique_id_size, unique_id_size);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_init, init);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_finalize, finalize);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_my_pe, my_pe);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_n_pes, n_pes);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_malloc, malloc_tensor);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_barrier_all, barrier_all);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_barrier_all_on_current_stream, barrier_all_on_current_stream);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_alltoall, alltoall);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_fake_alltoall, fake_alltoall);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_sum_reduce, sum_reduce);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_fake_sum_reduce, fake_sum_reduce);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_allreduce_on_stream_with_copy, allreduce_on_stream_with_copy);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(nvshmem_fake_allreduce_on_stream_with_copy,
                              fake_allreduce_on_stream_with_copy);

}  // namespace
