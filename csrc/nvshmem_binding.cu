#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2025 Perplexity AI
 */
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Exception.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <torch/library.h>

#include <cstdint>
#include <cstdlib>
#include <string>
#include <vector>

#define NVSHMEMCHECK(stmt)                                                                    \
  do {                                                                                        \
    int result = (stmt);                                                                      \
    if (NVSHMEMX_SUCCESS != result) {                                                         \
      fprintf(stderr, "[%s:%d] nvshmem failed with error %d \n", __FILE__, __LINE__, result); \
      exit(-1);                                                                               \
    }                                                                                         \
  } while (0)

namespace {

at::Tensor get_unique_id() {
  nvshmemx_uniqueid_t uid = NVSHMEMX_UNIQUEID_INITIALIZER;
  nvshmemx_get_uniqueid(&uid);
  return at::from_blob(&uid, sizeof(uid), at::kByte).clone();
}

int64_t unique_id_size() { return sizeof(nvshmemx_uniqueid_t); }

int64_t init(at::Tensor uid, int64_t rank, int64_t world_size) {
  TORCH_CHECK(uid.device().is_cpu(), "uid must be a CPU tensor");
  TORCH_CHECK(uid.scalar_type() == at::kByte, "uid must be a byte tensor");
  TORCH_CHECK(uid.numel() == sizeof(nvshmemx_uniqueid_t),
              "Invalid unique id size. Expected: ", sizeof(nvshmemx_uniqueid_t),
              ", Got: ", uid.numel(), ")");
  nvshmemx_uniqueid_t id;
  std::memcpy(&id, uid.data_ptr(), sizeof(id));
  nvshmemx_init_attr_t attr = NVSHMEMX_INIT_ATTR_INITIALIZER;
  nvshmemx_set_attr_uniqueid_args(rank, world_size, &id, &attr);
  return nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr);
}

void finalize() { nvshmem_finalize(); }

int64_t my_pe() { return nvshmem_my_pe(); }

int64_t n_pes() { return nvshmem_n_pes(); }

at::Tensor malloc_tensor(const std::vector<int64_t>& shape, c10::ScalarType dtype,
                         const c10::Device& device) {
  size_t size = c10::elementSize(dtype) * c10::multiply_integers(shape);
  void* ptr = nvshmem_malloc(size);
  if (ptr == nullptr) {
    AT_ERROR("nvshmem_malloc failed. size: ", size);
  }
  return at::from_blob(
      ptr, shape, [](void* ptr) { nvshmem_free(ptr); },
      at::TensorOptions().dtype(dtype).device(device));
}

void barrier_all() { nvshmem_barrier_all(); }

void barrier_all_on_current_stream() {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_barrier_all_on_stream(stream);
}

void alltoall(at::Tensor dest, at::Tensor source) {
  TORCH_CHECK(dest.is_contiguous(), "dest must be contiguous");
  TORCH_CHECK(source.is_contiguous(), "source must be contiguous");

  size_t nbytes = dest.numel() * dest.itemsize() / dest.size(0);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  NVSHMEMCHECK(nvshmemx_alltoallmem_on_stream(NVSHMEM_TEAM_WORLD, (uint8_t*)dest.data_ptr(),
                                              (uint8_t*)source.data_ptr(), nbytes, stream));
}

void fake_alltoall(at::Tensor dest, at::Tensor source) {}

void sum_reduce(at::Tensor dest, at::Tensor source, int64_t nelems) {
  TORCH_CHECK(dest.is_contiguous(), "dest must be contiguous");
  TORCH_CHECK(source.is_contiguous(), "source must be contiguous");
  TORCH_CHECK(dest.scalar_type() == source.scalar_type(),
              "dest and source must have the same dtype");

  // Add validation and conversion
  TORCH_CHECK(nelems >= 0, "nelems must be non-negative, got ", nelems);
  TORCH_CHECK(nelems <= SIZE_MAX, "nelems too large: ", nelems, " > ", SIZE_MAX);
  size_t nelems_size_t = static_cast<size_t>(nelems);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (dest.scalar_type()) {
    case at::kHalf:  // float16
      NVSHMEMCHECK(nvshmemx_half_sum_reduce_on_stream(NVSHMEM_TEAM_WORLD, (__half*)dest.data_ptr(),
                                                      (__half*)source.data_ptr(), nelems_size_t,
                                                      stream));
      break;
    case at::kFloat:  // float32
      NVSHMEMCHECK(nvshmemx_float_sum_reduce_on_stream(NVSHMEM_TEAM_WORLD, (float*)dest.data_ptr(),
                                                       (float*)source.data_ptr(), nelems_size_t,
                                                       stream));
      break;
    case at::kBFloat16:  // bfloat16
      NVSHMEMCHECK(nvshmemx_bfloat16_sum_reduce_on_stream(
          NVSHMEM_TEAM_WORLD, (__hip_bfloat16*)dest.data_ptr(), (__hip_bfloat16*)source.data_ptr(),
          nelems_size_t, stream));
      break;

    default:
      TORCH_CHECK(false, "Unsupported dtype for nvshmem_sum_reduce: ", dest.scalar_type());
  }
}

void fake_sum_reduce(at::Tensor dest, at::Tensor source, int64_t nelems) {}

void allreduce_on_stream_with_copy(at::Tensor dest_symm, at::Tensor source_symm,
                                   at::Tensor dest_local, at::Tensor source_local, int64_t nelems) {
  TORCH_CHECK(dest_symm.is_contiguous(), "dest_symm must be contiguous");
  TORCH_CHECK(source_symm.is_contiguous(), "source_symm must be contiguous");
  TORCH_CHECK(dest_local.is_contiguous(), "dest_local must be contiguous");
  TORCH_CHECK(source_local.is_contiguous(), "source_local must be contiguous");
  TORCH_CHECK(dest_symm.scalar_type() == source_symm.scalar_type(),
              "dest_symm and source_symm must have the same dtype");
  TORCH_CHECK(dest_symm.scalar_type() == source_local.scalar_type(),
              "dest_symm and source_local must have the same dtype");
  TORCH_CHECK(dest_local.scalar_type() == source_local.scalar_type(),
              "dest_local and source_local must have the same dtype");

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  hipMemcpyAsync(source_symm.data_ptr(), source_local.data_ptr(),
                  nelems * source_local.element_size(), hipMemcpyDefault, stream);
  nvshmemx_barrier_on_stream(NVSHMEM_TEAM_WORLD, stream);
  sum_reduce(dest_symm, source_symm, nelems);
  hipMemcpyAsync(dest_local.data_ptr(), dest_symm.data_ptr(), nelems * dest_local.element_size(),
                  hipMemcpyDefault, stream);
  hipStreamSynchronize(stream);
}

void fake_allreduce_on_stream_with_copy(at::Tensor dest_symm, at::Tensor source_symm,
                                        at::Tensor dest_local, at::Tensor source_local,
                                        int64_t nelems) {}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("nvshmem_get_unique_id", &get_unique_id);
  m.def("nvshmem_unique_id_size", &unique_id_size);
  m.def("nvshmem_init", &init);
  m.def("nvshmem_finalize", &finalize);
  m.def("nvshmem_my_pe", &my_pe);
  m.def("nvshmem_n_pes", &n_pes);
  m.def("nvshmem_malloc", &malloc_tensor);
  m.def("nvshmem_barrier_all", &barrier_all);
  m.def("nvshmem_barrier_all_on_current_stream", &barrier_all_on_current_stream);
  m.def("nvshmem_alltoall(Tensor! dest, Tensor src) -> ()");
  m.impl("nvshmem_alltoall", c10::kCUDA, &alltoall);
  m.impl("nvshmem_alltoall", c10::kMeta, &fake_alltoall);
  m.def("nvshmem_sum_reduce(Tensor! dest, Tensor src, int nelems) -> ()");
  m.impl("nvshmem_sum_reduce", c10::kCUDA, &sum_reduce);
  m.impl("nvshmem_sum_reduce", c10::kMeta, &fake_sum_reduce);
  m.def(
      "nvshmem_allreduce_on_stream_with_copy(Tensor! dest_symm, Tensor source_symm, Tensor "
      "dest_local, Tensor source_local, int nelems) -> ()");
  m.impl("nvshmem_allreduce_on_stream_with_copy", c10::kCUDA, &allreduce_on_stream_with_copy);
  m.impl("nvshmem_allreduce_on_stream_with_copy", c10::kMeta, &fake_allreduce_on_stream_with_copy);
};

}  // namespace
