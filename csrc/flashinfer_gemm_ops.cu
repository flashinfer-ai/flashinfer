/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "pytorch_extension_utils.h"

void bmm_fp8(at::Tensor A, at::Tensor B, at::Tensor D, at::Tensor A_scale, at::Tensor B_scale,
             at::Tensor workspace_buffer, int64_t cublas_handle, int64_t hip_stream);

void CutlassSegmentGEMM(at::Tensor workspace_buffer, at::Tensor all_problems, at::Tensor x_ptr,
                        at::Tensor w_ptr, at::Tensor y_ptr, at::Tensor x_ld, at::Tensor w_ld,
                        at::Tensor y_ld, at::Tensor empty_x_data, bool weight_column_major,
                        int64_t hip_stream);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("cutlass_segment_gemm", &CutlassSegmentGEMM, "Cutlass Segment GEMM");
  m.def("bmm_fp8", &bmm_fp8, "BMM FP8");
}
