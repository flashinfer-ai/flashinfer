#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/cuda/EmptyTensor.h>
#include <hip/hip_fp16.h>

#include <cstddef>
#include <cstdint>
#include <functional>
#include <type_traits>
#include <vector>

#include "flashinfer/gemm/cutlass_gemm_configs.h"
// Use SM120-specific dispatch template (includes fp4_gemm_cutlass.h)
#include "flashinfer/gemm/fp4_gemm_cutlass_template_sm120.h"
#include "pytorch_extension_utils.h"

using flashinfer::gemm::ClusterShape;
using flashinfer::gemm::CutlassFp4GemmRunner;
using flashinfer::gemm::CutlassGemmConfig;
using flashinfer::gemm::CutlassTileConfigSM120;
using flashinfer::gemm::EpilogueScheduleType;
using flashinfer::gemm::FP4GemmType;
using flashinfer::gemm::MainloopScheduleType;

namespace torch_ext {

namespace {

CutlassGemmConfig getFp4GemmConfig(int64_t m, int64_t n, int64_t k, int64_t tactic) {
  auto getCutlassFp4GemmConfigs = []() {
    CutlassFp4GemmRunner<__hip_bfloat16, FP4GemmType::W4A4_NVFP4_NVFP4> gemmRunner;
    return gemmRunner.getConfigs();
  };
  static std::vector<CutlassGemmConfig> globalConfigs = getCutlassFp4GemmConfigs();
  TORCH_CHECK(tactic >= 0 && tactic < globalConfigs.size(), "tactic must be between 0 and ",
              globalConfigs.size());
  return globalConfigs[tactic];
}

template <typename T>
void runGemm(at::Tensor& out, at::Tensor const& mat1, at::Tensor const& mat2,
             at::Tensor const& mat1Scale, at::Tensor const& mat2Scale,
             at::Tensor const& globalScale, int64_t m, int64_t n, int64_t k, int64_t batch_count,
             CutlassGemmConfig const& gemmConfig, at::Tensor workspace_buffer) {
  CutlassFp4GemmRunner<T, FP4GemmType::W4A4_NVFP4_NVFP4> gemmRunner;

  int64_t const required_workspace_size = gemmRunner.getWorkspaceSize(m, n, k, batch_count);
  int64_t const provided_workspace_size =
      workspace_buffer.numel() * workspace_buffer.element_size();

  auto runKernel = [&](void* workspace) {
    gemmRunner.gemm(out.data_ptr(), mat1.const_data_ptr(), mat2.const_data_ptr(),
                    mat1Scale.const_data_ptr(), mat2Scale.const_data_ptr(),
                    globalScale.data_ptr<float>(), m, n, k, batch_count, gemmConfig,
                    reinterpret_cast<char*>(workspace), required_workspace_size,
                    at::cuda::getCurrentCUDAStream(mat1.get_device()));
  };

  if (provided_workspace_size < required_workspace_size) {
    at::Tensor new_workspace = at::detail::empty_cuda(
        {required_workspace_size}, at::ScalarType::Char, mat1.device(), std::nullopt);

    runKernel(new_workspace.data_ptr());
  } else {
    runKernel(workspace_buffer.data_ptr());
  }
}

constexpr auto FLOAT4_E2M1X2 = at::ScalarType::Byte;  // uint8_t
constexpr auto SF_DTYPE = at::ScalarType::Byte;       // uint8_t

at::Tensor fp4_bmm_impl(at::Tensor const& mat1, at::Tensor const& mat2, at::Tensor const& mat1Scale,
                        at::Tensor const& mat2Scale, at::Tensor const& globalScale, at::Tensor out,
                        at::Tensor workspace_buffer, int64_t tactic) {
  // Validate inputs
  TORCH_CHECK(mat1.dtype() == FLOAT4_E2M1X2, "mat1 must be FLOAT4_E2M1X2 (uint8)");
  TORCH_CHECK(mat2.dtype() == FLOAT4_E2M1X2, "mat2 must be FLOAT4_E2M1X2 (uint8)");
  TORCH_CHECK(mat1Scale.dtype() == SF_DTYPE, "mat1Scale must be SF_DTYPE (uint8)");
  TORCH_CHECK(mat2Scale.dtype() == SF_DTYPE, "mat2Scale must be SF_DTYPE (uint8)");
  TORCH_CHECK(globalScale.dtype() == at::ScalarType::Float, "globalScale must be float");
  TORCH_CHECK(mat1.is_cuda(), "mat1 must be on CUDA device");
  TORCH_CHECK(mat2.is_cuda(), "mat2 must be on CUDA device");
  TORCH_CHECK(mat1Scale.is_cuda(), "mat1Scale must be on CUDA device");
  TORCH_CHECK(mat2Scale.is_cuda(), "mat2Scale must be on CUDA device");
  TORCH_CHECK(globalScale.is_cuda(), "globalScale must be on CUDA device");
  TORCH_CHECK(out.is_cuda(), "out must be on CUDA device");
  TORCH_CHECK(workspace_buffer.is_cuda(), "workspace_buffer must be on CUDA device");

  // Check device consistency
  TORCH_CHECK(mat1.device() == mat2.device() && mat1.device() == mat1Scale.device() &&
                  mat1.device() == mat2Scale.device() && mat1.device() == globalScale.device() &&
                  mat1.device() == out.device() && mat1.device() == workspace_buffer.device(),
              "All tensors must be on the same device");

  // Get dimensions
  int64_t b = 1;
  int64_t m, k_packed, n;

  if (mat1.dim() == 2) {
    m = mat1.size(0);
    k_packed = mat1.size(1);
  } else if (mat1.dim() == 3) {
    b = mat1.size(0);
    m = mat1.size(1);
    k_packed = mat1.size(2);
  } else {
    TORCH_CHECK(false, "mat1 must be 2D or 3D tensor");
  }

  if (mat2.dim() == 2) {
    n = mat2.size(0);
    TORCH_CHECK(mat2.size(1) == k_packed, "mat2.size(1) must match mat1.size(-1)");
  } else if (mat2.dim() == 3) {
    TORCH_CHECK(mat2.size(0) == b, "Batch dimensions must match");
    n = mat2.size(1);
    TORCH_CHECK(mat2.size(2) == k_packed, "mat2.size(2) must match mat1.size(-1)");
  } else {
    TORCH_CHECK(false, "mat2 must be 2D or 3D tensor");
  }

  // k_packed stores 2 FP4 values per byte
  int64_t k = k_packed * 2;

  TORCH_CHECK(globalScale.numel() == 1, "globalScale must be a scalar tensor");

  // Configure the kernel
  CutlassGemmConfig config =
      (tactic >= 0) ? getFp4GemmConfig(m, n, k, tactic)
                    : CutlassGemmConfig(CutlassTileConfigSM120::CtaShape128x128x128B,
                                        MainloopScheduleType::AUTO, EpilogueScheduleType::AUTO,
                                        ClusterShape::ClusterShape_1x1x1);

  // Validate output dimensions
  std::vector<int64_t> out_shape =
      (b > 1) ? std::vector<int64_t>{b, m, n} : std::vector<int64_t>{m, n};
  TORCH_CHECK(out.dim() == out_shape.size(), "out must have ", out_shape.size(), " dimensions");
  for (size_t i = 0; i < out_shape.size(); ++i) {
    TORCH_CHECK(out.sizes()[i] == out_shape[i], "out.size(", i, "): expected ", out_shape[i],
                ", got ", out.sizes()[i]);
  }

  c10::ScalarType out_dtype = out.scalar_type();

  switch (out_dtype) {
    case at::ScalarType::Half:
      runGemm<half>(out, mat1, mat2, mat1Scale, mat2Scale, globalScale, m, n, k, b, config,
                    workspace_buffer);
      break;
    case at::ScalarType::BFloat16:
      runGemm<__hip_bfloat16>(out, mat1, mat2, mat1Scale, mat2Scale, globalScale, m, n, k, b, config,
                             workspace_buffer);
      break;
    default:
      TORCH_CHECK(false, "out_dtype must be one of fp16/bf16.");
  }
  return out;
}

}  // namespace

at::Tensor fp4_gemm(at::Tensor const& mat1, at::Tensor const& mat2, at::Tensor const& mat1Scale,
                    at::Tensor const& mat2Scale, at::Tensor const& globalScale, at::Tensor out,
                    at::Tensor workspace_buffer, int64_t tactic) {
  return fp4_bmm_impl(mat1, mat2, mat1Scale, mat2Scale, globalScale, out, workspace_buffer, tactic);
}

int64_t fp4_gemm_tactic_num() {
  static const int64_t totalTactics =
      CutlassFp4GemmRunner<__hip_bfloat16, FP4GemmType::W4A4_NVFP4_NVFP4>{}.getConfigs().size();
  return totalTactics;
}

}  // namespace torch_ext

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("fp4_gemm", &torch_ext::fp4_gemm);
  m.def("fp4_gemm_tactic_num", &torch_ext::fp4_gemm_tactic_num);
}
