#include "hip/hip_runtime.h"
// flashinfer: adapted from sglang + vllm code
// refer to: https://github.com/vllm-project/vllm/blob/v0.8.2/csrc/custom_all_reduce.cu
#include <tvm/ffi/container/array.h>
#include <tvm/ffi/container/tuple.h>

#include <cstdint>
#include <vector>

#include "flashinfer/comm/vllm_custom_all_reduce.cuh"
#include "tvm_ffi_utils.h"

// Fake pointer type, must match fptr_t type in ops.h.
// We use this type alias to indicate when pointers are passed in as int64_t.
using fptr_t = int64_t;
static_assert(sizeof(void*) == sizeof(fptr_t));

using tvm::ffi::Array;
using tvm::ffi::Tuple;

fptr_t init_custom_ar(Array<fptr_t> fake_ipc_ptrs, Tensor rank_data, int64_t rank,
                      bool full_nvlink) {
  int world_size = fake_ipc_ptrs.size();
  if (world_size > 8) throw std::invalid_argument("world size > 8 is not supported");
  if (world_size % 2 != 0) throw std::invalid_argument("Odd num gpus is not supported for now");
  if (rank < 0 || rank >= world_size) throw std::invalid_argument("invalid rank passed in");

  vllm::Signal* ipc_ptrs[8];
  for (int i = 0; i < world_size; i++) {
    ipc_ptrs[i] = reinterpret_cast<vllm::Signal*>(fake_ipc_ptrs[i]);
  }
  return (fptr_t) new vllm::CustomAllreduce(ipc_ptrs, rank_data->data, get_numel(rank_data), rank,
                                            world_size, full_nvlink);
}

/**
 * Make sure tensor t's data lies completely within ((char)t->data) +
 * t.numel() * t.element_size(). This is slightly weaker than t.is_contiguous()
 * because it allows transpose of contiguous slice (i.e. slicing the first
 * dimension). Currently, we require this because stride information is not
 * passed into the kernels and we treat input tensors as flat.
 *
 * Examples
 * A = torch.zeros(3, 3, 3)
 * 1. A: OK
 * 2. A[1:]: OK
 * 3. A.permute(2, 0, 1): OK
 * 4. A[1:].permute(2, 0, 1): OK
 * 5. A[None].expand(2, -1, -1, -1): Not OK
 * 6. A[:, 1:, 1:]: Not OK
 */
bool _is_weak_contiguous(Tensor t) {
  auto numel = get_numel(t);
  auto element_size = get_element_size(t);
  return t.IsContiguous() ||
         (tvm::ffi::GetDataSize(numel, t->dtype) - t->byte_offset * element_size ==
          numel * element_size);
}

/**
 * Performs an out-of-place allreduce and stores result in out.
 *
 * If _reg_buffer is null, assumes inp->data is already IPC-registered.
 * Otherwise, _reg_buffer is assumed to be IPC-registered and inp is first
 * copied into _reg_buffer.
 */
void all_reduce(fptr_t _fa, Tensor inp, Tensor out, fptr_t _reg_buffer, int64_t reg_buffer_sz_bytes,
                int64_t num_ctas) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce*>(_fa);
  hipSetDevice(inp->device.device_id);
  auto stream = get_stream(inp->device);

  TVM_FFI_ICHECK_EQ(inp->dtype, out->dtype);
  TVM_FFI_ICHECK_EQ(get_numel(inp), get_numel(out));
  TVM_FFI_ICHECK(_is_weak_contiguous(out));
  TVM_FFI_ICHECK(_is_weak_contiguous(inp));
  auto input_size = get_numel(inp) * get_element_size(inp);
  auto reg_buffer = reinterpret_cast<void*>(_reg_buffer);
  if (reg_buffer) {
    TVM_FFI_ICHECK_LE(input_size, reg_buffer_sz_bytes);
    auto status =
        hipMemcpyAsync(reg_buffer, inp->data, input_size, hipMemcpyDeviceToDevice, stream);
    TVM_FFI_ICHECK(status == hipSuccess);
  } else {
    reg_buffer = inp->data;
  }
  switch (encode_dlpack_dtype(out->dtype)) {
    case float32_code: {
      fa->allreduce<float>(stream, reinterpret_cast<float*>(reg_buffer),
                           reinterpret_cast<float*>(out->data), get_numel(out), num_ctas);
      break;
    }
    case float16_code: {
      fa->allreduce<half>(stream, reinterpret_cast<half*>(reg_buffer),
                          reinterpret_cast<half*>(out->data), get_numel(out), num_ctas);
      break;
    }
#if (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
    case bfloat16_code: {
      fa->allreduce<hip_bfloat16>(stream, reinterpret_cast<hip_bfloat16*>(reg_buffer),
                                 reinterpret_cast<hip_bfloat16*>(out->data), get_numel(out),
                                 num_ctas);
      break;
    }
#endif
    default:
      throw std::runtime_error("custom allreduce only supports float32, float16 and bfloat16");
  }
}

void dispose(fptr_t _fa) { delete reinterpret_cast<vllm::CustomAllreduce*>(_fa); }

int64_t meta_size() { return sizeof(vllm::Signal); }

void register_buffer(fptr_t _fa, Array<fptr_t> fake_ipc_ptrs) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce*>(_fa);
  TVM_FFI_ICHECK_EQ(fake_ipc_ptrs.size(), fa->world_size_);
  void* ipc_ptrs[8];
  for (int i = 0; i < fake_ipc_ptrs.size(); i++) {
    ipc_ptrs[i] = reinterpret_cast<void*>(fake_ipc_ptrs[i]);
  }
  fa->register_buffer(ipc_ptrs);
}

// Use vector<int64_t> to represent byte data for python binding compatibility.
Tuple<Array<int64_t>, Array<int64_t>> get_graph_buffer_ipc_meta(fptr_t _fa) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce*>(_fa);
  auto [handle, offsets] = fa->get_graph_buffer_ipc_meta();
  std::vector<int64_t> bytes(handle.begin(), handle.end());
  return Tuple<Array<int64_t>, Array<int64_t>>(Array<int64_t>(bytes), Array<int64_t>(offsets));
}

// Use vector<int64_t> to represent byte data for python binding compatibility.
void register_graph_buffers(fptr_t _fa, Array<Array<int64_t>> handles,
                            Array<Array<int64_t>> offsets) {
  auto fa = reinterpret_cast<vllm::CustomAllreduce*>(_fa);
  std::vector<std::string> bytes;
  bytes.reserve(handles.size());
  for (int i = 0; i < handles.size(); i++) {
    bytes.emplace_back(handles[i].begin(), handles[i].end());
  }
  bytes.reserve(handles.size());
  std::vector<std::vector<int64_t>> off(offsets.size());
  for (int i = 0; i < offsets.size(); ++i) {
    off[i] = std::vector<int64_t>(offsets[i].begin(), offsets[i].end());
  }
  fa->register_graph_buffers(bytes, off);
}

TVM_FFI_DLL_EXPORT_TYPED_FUNC(get_graph_buffer_ipc_meta, get_graph_buffer_ipc_meta);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(register_graph_buffers, register_graph_buffers);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(dispose, dispose);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(meta_size, meta_size);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(register_buffer, register_buffer);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(init_custom_ar, init_custom_ar);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(all_reduce, all_reduce);
