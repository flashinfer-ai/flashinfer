#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/persistent.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/attention/variant_helper.cuh>
#include <flashinfer/pos_enc.cuh>
#include <optional>

#include "batch_persistent_config.inc"
#include "pytorch_conversion_utils.h"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

at::Tensor BatchPagedAttentionPlan(at::Tensor float_workspace_buffer,
                                   at::Tensor int_workspace_buffer,
                                   at::Tensor page_locked_int_workspace_buffer,
                                   at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor kv_len,
                                   int64_t batch_size, int64_t num_qo_heads, int64_t num_kv_heads,
                                   int64_t head_dim_o, bool causal) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();

  HolisticPlanInfo<2> plan_info;

  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer.device());
  const hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  hipError_t status = TwoStageHolisticPlan<IdType>(
      float_workspace_buffer.data_ptr(), float_workspace_size_in_bytes,
      int_workspace_buffer.data_ptr(), page_locked_int_workspace_buffer.data_ptr(),
      int_workspace_size_in_bytes, plan_info, qo_indptr.data_ptr<IdType>(),
      kv_indptr.data_ptr<IdType>(), kv_len.data_ptr<IdType>(), batch_size, num_qo_heads,
      num_kv_heads, head_dim_o, causal, stream);

  TORCH_CHECK(status == hipSuccess,
              "Failed to plan persistent paged attention, error: ", hipGetErrorString(status));

  return vec_to_tensor(plan_info.ToVector());
}

struct StandardAttention : AttentionVariantBase {
  float sm_scale_log2;

  PROFILER_CLOSURE_PARAMS_DECL

  template <typename Params>
  __device__ __host__ StandardAttention(const Params& params, uint32_t batch_idx,
                                        uint8_t* smem_ptr) {
    sm_scale_log2 = params.sm_scale * math::log2e;
  }
};

at::Tensor BatchPagedAttentionRun(at::Tensor float_workspace_buffer,
                                  at::Tensor int_workspace_buffer, at::Tensor plan_info_vec,
                                  at::Tensor q, at::Tensor k_cache, at::Tensor v_cache,
                                  at::Tensor kv_indices, at::Tensor o,
                                  std::optional<at::Tensor> maybe_lse, int64_t mask_mode_code,
                                  int64_t num_qo_heads, int64_t num_kv_heads, int64_t page_size,
                                  double sm_scale ADDITIONAL_FUNC_PARAMS) {
  HolisticPlanInfo<2> plan_info;
  plan_info.FromVector(tensor_to_vec(plan_info_vec));

  auto device = q.device();

  void* float_buffer_ptr = float_workspace_buffer.data_ptr();
  void* int_buffer_ptr = int_workspace_buffer.data_ptr();

  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = k_cache.scalar_type();

  unsigned int q_stride_n = q.stride(0);
  unsigned int q_stride_h = q.stride(1);
  unsigned int k_stride_page = k_cache.stride(0);
  unsigned int k_stride_h = k_cache.stride(2);
  unsigned int k_stride_n = k_cache.stride(1);
  unsigned int v_stride_page = v_cache.stride(0);
  unsigned int v_stride_h = v_cache.stride(2);
  unsigned int v_stride_n = v_cache.stride(1);
  unsigned int o_stride_n = o.stride(0);
  unsigned int o_stride_h = o.stride(1);

  const c10::cuda::OptionalCUDAGuard device_guard(device);
  const hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  PersistentParams<hip_bfloat16, hip_bfloat16, hip_bfloat16, int> params[2];

  for (int i = 0; i < 2; i++) {
    params[i].q = static_cast<hip_bfloat16*>(q.data_ptr());
    params[i].k = static_cast<hip_bfloat16*>(k_cache.data_ptr());
    params[i].v = static_cast<hip_bfloat16*>(v_cache.data_ptr());

    params[i].batch_indices =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].batch_indices_offset);
    params[i].q_indptr =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].q_indptr_offset);
    params[i].kv_indptr =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_indptr_offset);
    params[i].partial_indptr =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].partial_indptr_offset);
    params[i].kv_indices = static_cast<int*>(kv_indices.data_ptr());
    params[i].q_len = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].q_len_offset);
    params[i].kv_len =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_len_offset);
    params[i].q_start =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].q_start_offset);
    params[i].kv_start =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_start_offset);
    params[i].kv_end =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_end_offset);
    params[i].kv_head_idx_arr =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].kv_head_idx_offset);
    params[i].work_indptr =
        GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.tasks[i].work_indptr_offset);

    params[i].final_o = static_cast<hip_bfloat16*>(o.data_ptr());
    params[i].final_lse =
        maybe_lse.has_value() ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
    params[i].partial_o =
        GetPtrFromBaseOffset<hip_bfloat16>(float_buffer_ptr, plan_info.partial_o_offset);
    params[i].partial_lse =
        GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.partial_lse_offset);

    params[i].gqa_group_size = uint_fastdiv(num_qo_heads / num_kv_heads);
    params[i].page_size = uint_fastdiv(page_size);

    params[i].q_stride_n = q_stride_n;
    params[i].q_stride_h = q_stride_h;
    params[i].k_stride_page = k_stride_page;
    params[i].k_stride_h = k_stride_h;
    params[i].k_stride_n = k_stride_n;
    params[i].v_stride_page = v_stride_page;
    params[i].v_stride_h = v_stride_h;
    params[i].v_stride_n = v_stride_n;
    params[i].o_stride_n = o_stride_n;
    params[i].o_stride_h = o_stride_h;

    params[i].sm_scale = sm_scale;
  }

  ADDITIONAL_PARAMS_SETTER

  using AttentionVariant = StandardAttention;

  DISPATCH_MASK_MODE(mask_mode, MASK_MODE, {
    hipError_t status =
        BatchPagedAttentionPersistent<128, 16, 128, 128, MASK_MODE, AttentionVariant>(
            params[0], params[1], plan_info.num_blks_x, plan_info.num_blks_y, stream);
    TORCH_CHECK(status == hipSuccess,
                "Failed to run persistent paged attention, error: ", hipGetErrorString(status));
  });
  return o;
}
