#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "aot_default_additional_params.h"
#include "pytorch_extension_utils.h"

//========== activation ==========

void silu_and_mul(at::Tensor& out, at::Tensor& input, bool enable_pdl);
void gelu_tanh_and_mul(at::Tensor& out, at::Tensor& input, bool enable_pdl);
void gelu_and_mul(at::Tensor& out, at::Tensor& input, bool enable_pdl);

//========== cascade ==========

void merge_state(at::Tensor v_a, at::Tensor s_a, at::Tensor v_b, at::Tensor s_b,
                 at::Tensor v_merged, at::Tensor s_merged);

void merge_state_in_place(at::Tensor v, at::Tensor s, at::Tensor v_other, at::Tensor s_other,
                          std::optional<at::Tensor> mask);

void merge_states(at::Tensor v, at::Tensor s, at::Tensor v_merged, at::Tensor s_merged);

//========== decode ==========

void single_decode_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                 at::Tensor o, int64_t layout,
                                 int64_t window_left SINGLE_DECODE_ADDITIONAL_FUNC_PARAMS);

at::Tensor BatchDecodeWithPagedKVCachePlan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor indptr, int64_t batch_size,
    int64_t num_qo_heads, int64_t num_kv_heads, int64_t page_size, bool enable_cuda_graph,
    int64_t window_left, double logits_soft_cap, int64_t head_dim_qk, int64_t head_dim_vo,
    at::Tensor empty_q_data, at::Tensor empty_kv_data);

void BatchDecodeWithPagedKVCacheRun(at::Tensor float_workspace_buffer,
                                    at::Tensor int_workspace_buffer, at::Tensor plan_info_vec,
                                    at::Tensor q, at::Tensor paged_k_cache,
                                    at::Tensor paged_v_cache, at::Tensor paged_kv_indptr,
                                    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len,
                                    at::Tensor o, std::optional<at::Tensor> maybe_lse,
                                    int64_t kv_layout_code,
                                    int64_t window_left BATCH_DECODE_ADDITIONAL_FUNC_PARAMS);

//========== gemm ==========

void bmm_fp8(at::Tensor A, at::Tensor B, at::Tensor D, at::Tensor A_scale, at::Tensor B_scale,
             at::Tensor workspace_buffer, int64_t cublas_handle);

void CutlassSegmentGEMM(at::Tensor workspace_buffer, at::Tensor all_problems, at::Tensor x_ptr,
                        at::Tensor w_ptr, at::Tensor y_ptr, at::Tensor x_ld, at::Tensor w_ld,
                        at::Tensor y_ld, at::Tensor empty_x_data, bool weight_column_major);

//========== norm ==========

void rmsnorm(at::Tensor& out, at::Tensor& input, at::Tensor& weight, double eps, bool enable_pdl);

void fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight, double eps,
                       bool enable_pdl);

void gemma_rmsnorm(at::Tensor& out, at::Tensor& input, at::Tensor& weight, double eps,
                   bool enable_pdl);

void gemma_fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight,
                             double eps, bool enable_pdl);

//========== page ==========

void append_paged_kv_cache(at::Tensor append_key, at::Tensor append_value, at::Tensor batch_indices,
                           at::Tensor positions, at::Tensor paged_k_cache, at::Tensor paged_v_cache,
                           at::Tensor kv_indices, at::Tensor kv_indptr, at::Tensor kv_last_page_len,
                           int64_t layout);

void append_paged_mla_kv_cache(at::Tensor append_ckv, at::Tensor append_kpe,
                               at::Tensor batch_indices, at::Tensor positions, at::Tensor ckv_cache,
                               at::Tensor kpe_cache, at::Tensor kv_indices, at::Tensor kv_indptr,
                               at::Tensor kv_last_page_len);

void block_sparse_indices_to_vector_sparse_offsets(
    at::Tensor block_sparse_indices, at::Tensor block_sparse_indptr,
    at::Tensor vector_sparse_offsets, at::Tensor vector_sparse_indptr, at::Tensor kv_len_arr,
    int64_t stride_block, int64_t stride_n, int64_t batch_size, int64_t block_size);

//========== prefill ==========

void single_prefill_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                  at::Tensor o, std::optional<at::Tensor> maybe_lse,
                                  int64_t mask_mode_code, int64_t layout,
                                  int64_t window_left SINGLE_PREFILL_ADDITIONAL_FUNC_PARAMS);

at::Tensor BatchPrefillWithKVCachePlan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor qo_indptr, at::Tensor kv_indptr,
    at::Tensor kv_len_arr, int64_t total_num_rows, int64_t batch_size, int64_t num_qo_heads,
    int64_t num_kv_heads, int64_t page_size, bool enable_cuda_graph, int64_t head_dim_qk,
    int64_t head_dim_vo, bool causal);

void BatchPrefillWithRaggedKVCacheRun(at::Tensor float_workspace_buffer,
                                      at::Tensor int_workspace_buffer, at::Tensor plan_info_vec,
                                      at::Tensor q, at::Tensor k, at::Tensor v,
                                      at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor o,
                                      std::optional<at::Tensor> maybe_lse, int64_t mask_mode_code,
                                      int64_t layout,
                                      int64_t window_left BATCH_PREFILL_ADDITIONAL_FUNC_PARAMS);

void BatchPrefillWithPagedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer, at::Tensor plan_info_vec,
    at::Tensor q, at::Tensor paged_k_cache, at::Tensor paged_v_cache, at::Tensor qo_indptr,
    at::Tensor paged_kv_indptr, at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len,
    at::Tensor o, std::optional<at::Tensor> maybe_lse, int64_t mask_mode_code, int64_t layout,
    int64_t window_left BATCH_PREFILL_ADDITIONAL_FUNC_PARAMS);

//========== pod-attention =========
void pod_with_kv_cache_tensor(
    // Prefill params
    at::Tensor q_p, at::Tensor k_p, at::Tensor v_p, at::Tensor tmp_p, at::Tensor o_p,
    std::optional<at::Tensor> maybe_lse_p, int64_t mask_mode_code_p, int64_t layout_p,
    int64_t window_left_p, std::optional<at::Tensor> maybe_custom_mask_p,
    std::optional<at::Tensor> maybe_alibi_slopes_p, double logits_soft_cap_p, double sm_scale_p,
    double rope_rcp_scale_p, double rope_rcp_theta_p,
    // Decode params
    at::Tensor float_workspace_buffer_d, at::Tensor int_workspace_buffer_d,
    at::Tensor plan_info_vec, at::Tensor q_d, at::Tensor paged_k_cache_d,
    at::Tensor paged_v_cache_d, at::Tensor qo_indptr_d, at::Tensor paged_kv_indptr_d,
    at::Tensor paged_kv_indices_d, at::Tensor paged_kv_last_page_len_d, at::Tensor o_d,
    std::optional<at::Tensor> maybe_lse_d, int64_t mask_mode_code_d, int64_t layout_d,
    int64_t window_left, std::optional<at::Tensor> maybe_custom_mask_d,
    std::optional<at::Tensor> maybe_mask_indptr_d, std::optional<at::Tensor> maybe_alibi_slopes_d,
    double logits_soft_cap_d, double sm_scale_d, double rope_rcp_scale_d, double rope_rcp_theta_d);
//========== quantization ==========

void packbits(at::Tensor x, const std::string& bitorder, at::Tensor y);

void segment_packbits(at::Tensor x, at::Tensor input_indptr, at::Tensor output_indptr,
                      const std::string& bitorder, at::Tensor y);

//========== rope ==========

void apply_rope(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope, at::Tensor indptr,
                at::Tensor offsets, int64_t rotary_dim, bool interleave, double rope_scale,
                double rope_theta);

void apply_llama31_rope(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope,
                        at::Tensor indptr, at::Tensor offsets, int64_t rotary_dim, bool interleave,
                        double rope_scale, double rope_theta, double low_freq_factor,
                        double high_freq_factor, double old_context_length);

void apply_rope_pos_ids(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope,
                        at::Tensor pos_ids, int64_t rotary_dim, bool interleave, double rope_scale,
                        double rope_theta);

void apply_llama31_rope_pos_ids(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope,
                                at::Tensor pos_ids, int64_t rotary_dim, bool interleave,
                                double rope_scale, double rope_theta, double low_freq_factor,
                                double high_freq_factor, double old_context_length);

void apply_rope_pos_ids_cos_sin_cache(at::Tensor q, at::Tensor k, at::Tensor q_rope,
                                      at::Tensor k_rope, at::Tensor cos_sin_cache,
                                      at::Tensor pos_ids, bool interleave);

//========== sampling ==========

void sampling_from_probs(at::Tensor probs, at::Tensor output,
                         std::optional<at::Tensor> maybe_indices, bool deterministic,
                         std::optional<at::Generator> gen);

void top_p_sampling_from_probs(at::Tensor probs, at::Tensor output,
                               std::optional<at::Tensor> maybe_indices,
                               std::optional<at::Tensor> maybe_top_p_arr, double top_p_val,
                               bool deterministic, std::optional<at::Generator> gen);

void top_k_sampling_from_probs(at::Tensor probs, at::Tensor output,
                               std::optional<at::Tensor> maybe_indices,
                               std::optional<at::Tensor> maybe_top_k_arr, int64_t top_k_val,
                               bool deterministic, std::optional<at::Generator> gen);

void min_p_sampling_from_probs(at::Tensor probs, at::Tensor output,
                               std::optional<at::Tensor> maybe_indices,
                               std::optional<at::Tensor> maybe_min_p_arr, double min_p_val,
                               bool deterministic, std::optional<at::Generator> gen);

void top_k_top_p_sampling_from_probs(at::Tensor probs, at::Tensor output,
                                     std::optional<at::Tensor> maybe_indices,
                                     std::optional<at::Tensor> maybe_top_k_arr, double top_k_val,
                                     std::optional<at::Tensor> maybe_top_p_arr, double top_p_val,
                                     bool deterministic, std::optional<at::Generator> gen);

void top_p_renorm_probs(at::Tensor probs, at::Tensor renorm_probs,
                        std::optional<at::Tensor> maybe_top_p_arr, double top_p_val);

void top_k_renorm_probs(at::Tensor probs, at::Tensor renorm_probs,
                        std::optional<at::Tensor> maybe_top_k_arr, int64_t top_k_val);

void top_k_mask_logits(at::Tensor logits, at::Tensor mask_logits,
                       std::optional<at::Tensor> maybe_top_k_arr, int64_t top_k_val);

void chain_speculative_sampling(at::Tensor draft_probs, at::Tensor draft_token_ids,
                                at::Tensor target_probs, at::Tensor output_token_ids,
                                at::Tensor output_accepted_token_num,
                                at::Tensor output_emitted_token_num, bool deterministic,
                                std::optional<at::Generator> gen);

//========== Torch Library ==========

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  // activation
  // Fused SiLU and Mul
  m.def("silu_and_mul", silu_and_mul);
  // Fused GeLU Tanh and Mul
  m.def("gelu_tanh_and_mul", gelu_tanh_and_mul);
  // Fused GeLU and Mul
  m.def("gelu_and_mul", gelu_and_mul);

  // cascade
  // Merge two self-attention states
  m.def("merge_state", merge_state);
  // Merge another self-attention state in-place.
  m.def("merge_state_in_place", merge_state_in_place);
  // "Merge multiple self-attention states"
  m.def("merge_states", merge_states);

  // decode
  // "Single-request decode with KV-Cache operator"
  m.def("single_decode_with_kv_cache", single_decode_with_kv_cache);
  m.def("batch_decode_with_paged_kv_cache_plan", BatchDecodeWithPagedKVCachePlan);
  m.def("batch_decode_with_paged_kv_cache_run", BatchDecodeWithPagedKVCacheRun);

  // gemm
  // BMM FP8
  m.def("bmm_fp8", bmm_fp8);
  // Cutlass Segment GEMM operator
  m.def("cutlass_segment_gemm", CutlassSegmentGEMM);

  // norm
  // Root mean square normalization
  m.def("rmsnorm", rmsnorm);
  // Fused add root mean square normalization
  m.def("fused_add_rmsnorm", fused_add_rmsnorm);
  // Gemma Root mean square normalization
  m.def("gemma_rmsnorm", gemma_rmsnorm);
  // Gemma Fused add root mean square normalization
  m.def("gemma_fused_add_rmsnorm", gemma_fused_add_rmsnorm);

  // page
  // Append paged KV-Cache operator
  m.def("append_paged_kv_cache", append_paged_kv_cache);
  // Append paged MLA KV-Cache operator
  m.def("append_paged_mla_kv_cache", append_paged_mla_kv_cache);
  // Precompute block sparse offsets
  m.def("block_sparse_indices_to_vector_sparse_offsets",
        block_sparse_indices_to_vector_sparse_offsets);

  // prefill
  // Single-request prefill attention with KV-Cache operator
  m.def("single_prefill_with_kv_cache", single_prefill_with_kv_cache);
  m.def("batch_prefill_with_kv_cache_plan", BatchPrefillWithKVCachePlan);
  m.def("batch_prefill_with_ragged_kv_cache_run", BatchPrefillWithRaggedKVCacheRun);
  m.def("batch_prefill_with_paged_kv_cache_run", BatchPrefillWithPagedKVCacheRun);

  // pod-attention
  // Temporarily disabled because we don't generate the implementation yet.
  // m.def("pod_with_kv_cache_tensor", pod_with_kv_cache_tensor);

  // quantization
  // GPU packbits operator
  m.def("packbits", packbits);
  // GPU segment packbits operator
  m.def("segment_packbits", segment_packbits);

  // rope
  // "Apply RoPE"
  m.def("apply_rope", apply_rope);
  // "Apply Llama 3.1 style RoPE"
  m.def("apply_llama31_rope", apply_llama31_rope);
  // "Apply RoPE with positional ids"
  m.def("apply_rope_pos_ids", apply_rope_pos_ids);
  // "Apply Llama 3.1 style RoPE with positional ids"
  m.def("apply_llama31_rope_pos_ids", apply_llama31_rope_pos_ids);
  // "Apply RoPE with positional ids and cosine/sine cache"
  m.def("apply_rope_pos_ids_cos_sin_cache", apply_rope_pos_ids_cos_sin_cache);

  // sampling
  // Sample from probabilities
  m.def("sampling_from_probs", sampling_from_probs);
  // Top-k sampling from probabilities
  m.def("top_k_sampling_from_probs", top_k_sampling_from_probs);
  // Min-p sampling from probabilities
  m.def("min_p_sampling_from_probs", min_p_sampling_from_probs);
  // Top-p sampling from probabilities
  m.def("top_p_sampling_from_probs", top_p_sampling_from_probs);
  // Top-k and top-p sampling from probabilities
  m.def("top_k_top_p_sampling_from_probs", top_k_top_p_sampling_from_probs);
  // Renormalize probabilities by top-k mask
  m.def("top_k_renorm_probs", top_k_renorm_probs);
  // Renormalize probabilities by top-p mask
  m.def("top_p_renorm_probs", top_p_renorm_probs);
  // Mask logits by top-k mask
  m.def("top_k_mask_logits", top_k_mask_logits);
  // Speculative sampling from sequence of probabilities
  m.def("chain_speculative_sampling", chain_speculative_sampling);
}
