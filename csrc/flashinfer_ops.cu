#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "aot_default_additional_params.h"
#include "pytorch_extension_utils.h"

//========== activation ==========

void silu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);
void gelu_tanh_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);
void gelu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);

//========== cascade ==========

void merge_state(at::Tensor v_a, at::Tensor s_a, at::Tensor v_b, at::Tensor s_b,
                 at::Tensor v_merged, at::Tensor s_merged, int64_t hip_stream);

void merge_state_in_place(at::Tensor v, at::Tensor s, at::Tensor v_other, at::Tensor s_other,
                          std::optional<at::Tensor> mask, int64_t hip_stream);

void merge_states(at::Tensor v, at::Tensor s, at::Tensor v_merged, at::Tensor s_merged,
                  int64_t hip_stream);

//========== decode ==========

void single_decode_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                 at::Tensor o, int64_t layout,
                                 int64_t window_left SINGLE_DECODE_ADDITIONAL_FUNC_PARAMS,
                                 int64_t hip_stream);

at::Tensor BatchDecodeWithPagedKVCachePlan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor indptr, int64_t batch_size,
    int64_t num_qo_heads, int64_t num_kv_heads, int64_t page_size,
    bool enable_cuda_graph, int64_t window_left, double logits_soft_cap, int64_t head_dim_qk,
    int64_t head_dim_vo, at::Tensor empty_q_data, at::Tensor empty_kv_data, int64_t hip_stream);

void BatchDecodeWithPagedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, at::Tensor paged_kv_indptr, at::Tensor paged_kv_indices,
    at::Tensor paged_kv_last_page_len, at::Tensor o, std::optional<at::Tensor> maybe_lse,
    int64_t kv_layout_code, int64_t window_left BATCH_DECODE_ADDITIONAL_FUNC_PARAMS,
    int64_t hip_stream);

//========== gemm ==========

void bmm_fp8(at::Tensor A, at::Tensor B, at::Tensor D, at::Tensor A_scale, at::Tensor B_scale,
             at::Tensor workspace_buffer, int64_t cublas_handle, int64_t hip_stream);

void CutlassSegmentGEMM(at::Tensor workspace_buffer, at::Tensor all_problems, at::Tensor x_ptr,
                        at::Tensor w_ptr, at::Tensor y_ptr, at::Tensor x_ld, at::Tensor w_ld,
                        at::Tensor y_ld, at::Tensor empty_x_data, bool weight_column_major,
                        int64_t hip_stream);

//========== norm ==========

void rmsnorm(at::Tensor& out, at::Tensor& input, at::Tensor& weight, double eps,
             int64_t hip_stream);

void fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight, double eps,
                       int64_t hip_stream);

void gemma_rmsnorm(at::Tensor& out, at::Tensor& input, at::Tensor& weight, double eps,
                   int64_t hip_stream);

void gemma_fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight,
                             double eps, int64_t hip_stream);

//========== page ==========

void append_paged_kv_cache(at::Tensor append_key, at::Tensor append_value, at::Tensor batch_indices,
                           at::Tensor positions, at::Tensor paged_k_cache, at::Tensor paged_v_cache,
                           at::Tensor kv_indices, at::Tensor kv_indptr, at::Tensor kv_last_page_len,
                           int64_t layout, int64_t hip_stream);

void block_sparse_indices_to_vector_sparse_offsets(at::Tensor block_sparse_indices,
                                                   at::Tensor block_sparse_indptr,
                                                   at::Tensor vector_sparse_offsets,
                                                   at::Tensor vector_sparse_indptr,
                                                   at::Tensor kv_len_arr, int64_t stride_block,
                                                   int64_t stride_n, int64_t batch_size,
                                                   int64_t block_size, int64_t hip_stream);

//========== prefill ==========

void single_prefill_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                  at::Tensor o, std::optional<at::Tensor> maybe_lse,
                                  int64_t mask_mode_code, int64_t layout,
                                  int64_t window_left SINGLE_PREFILL_ADDITIONAL_FUNC_PARAMS,
                                  int64_t hip_stream);

at::Tensor BatchPrefillWithKVCachePlan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor qo_indptr, at::Tensor kv_indptr,
    at::Tensor kv_len_arr, int64_t total_num_rows, int64_t batch_size,
    int64_t num_qo_heads, int64_t num_kv_heads, int64_t page_size,
    bool enable_cuda_graph, int64_t head_dim_qk, int64_t head_dim_vo, bool causal,
    int64_t hip_stream);

void BatchPrefillWithRaggedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor plan_info_vec, at::Tensor q, at::Tensor k, at::Tensor v,
    at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor o, std::optional<at::Tensor> maybe_lse,
    int64_t mask_mode_code, int64_t layout,
    int64_t window_left BATCH_PREFILL_ADDITIONAL_FUNC_PARAMS, int64_t hip_stream);

void BatchPrefillWithPagedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, at::Tensor qo_indptr, at::Tensor paged_kv_indptr,
    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len, at::Tensor o,
    std::optional<at::Tensor> maybe_lse, int64_t mask_mode_code, int64_t layout,
    int64_t window_left BATCH_PREFILL_ADDITIONAL_FUNC_PARAMS, int64_t hip_stream);

//========== quantization ==========

void packbits(at::Tensor x, const std::string& bitorder, at::Tensor y, int64_t hip_stream);

void segment_packbits(at::Tensor x, at::Tensor input_indptr, at::Tensor output_indptr,
                      const std::string& bitorder, at::Tensor y, int64_t hip_stream);

//========== rope ==========

void apply_rope(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope, at::Tensor indptr,
                at::Tensor offsets, int64_t rotary_dim, bool interleave, double rope_scale,
                double rope_theta, int64_t hip_stream);

void apply_llama31_rope(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope,
                        at::Tensor indptr, at::Tensor offsets, int64_t rotary_dim,
                        bool interleave, double rope_scale, double rope_theta, double low_freq_factor,
                        double high_freq_factor, double old_context_length, int64_t hip_stream);

void apply_rope_pos_ids(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope,
                        at::Tensor pos_ids, int64_t rotary_dim, bool interleave,
                        double rope_scale, double rope_theta, int64_t hip_stream);

void apply_llama31_rope_pos_ids(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope,
                                at::Tensor pos_ids, int64_t rotary_dim, bool interleave,
                                double rope_scale, double rope_theta, double low_freq_factor,
                                double high_freq_factor, double old_context_length,
                                int64_t hip_stream);

void apply_rope_pos_ids_cos_sin_cache(at::Tensor q, at::Tensor k, at::Tensor q_rope,
                                      at::Tensor k_rope, at::Tensor cos_sin_cache,
                                      at::Tensor pos_ids, bool interleave, int64_t hip_stream);

//========== sampling ==========

void sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                         bool deterministic, int64_t hip_stream);

void top_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               at::Tensor success, std::optional<at::Tensor> maybe_top_p_arr,
                               double top_p_val, bool deterministic, int64_t hip_stream);

void top_k_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               at::Tensor success, std::optional<at::Tensor> maybe_top_k_arr,
                               int64_t top_k_val, bool deterministic, int64_t hip_stream);

void min_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               std::optional<at::Tensor> maybe_min_p_arr, double min_p_val,
                               bool deterministic, int64_t hip_stream);

void top_k_top_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples,
                                     at::Tensor samples, at::Tensor success,
                                     std::optional<at::Tensor> maybe_top_k_arr, double top_k_val,
                                     std::optional<at::Tensor> maybe_top_p_arr, double top_p_val,
                                     bool deterministic, int64_t hip_stream);

void top_p_renorm_probs(at::Tensor probs, at::Tensor renorm_probs,
                        std::optional<at::Tensor> maybe_top_p_arr, double top_p_val,
                        int64_t hip_stream);

void top_k_renorm_probs(at::Tensor probs, at::Tensor renorm_probs,
                        std::optional<at::Tensor> maybe_top_k_arr, int64_t top_k_val,
                        int64_t hip_stream);

void top_k_mask_logits(at::Tensor logits, at::Tensor mask_logits,
                       std::optional<at::Tensor> maybe_top_k_arr, int64_t top_k_val,
                       int64_t hip_stream);

void chain_speculative_sampling(at::Tensor draft_probs, at::Tensor draft_token_ids,
                                at::Tensor uniform_samples, at::Tensor target_probs,
                                at::Tensor output_token_ids, at::Tensor output_accepted_token_num,
                                at::Tensor output_emitted_token_num, bool deterministic,
                                int64_t hip_stream);

//========== Torch Library ==========

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  // activation
  // Fused SiLU and Mul
  m.def("silu_and_mul", silu_and_mul);
  // Fused GeLU Tanh and Mul
  m.def("gelu_tanh_and_mul", gelu_tanh_and_mul);
  // Fused GeLU and Mul
  m.def("gelu_and_mul", gelu_and_mul);

  // cascade
  // Merge two self-attention states
  m.def("merge_state", merge_state);
  // Merge another self-attention state in-place.
  m.def("merge_state_in_place", merge_state_in_place);
  // "Merge multiple self-attention states"
  m.def("merge_states", merge_states);

  // decode
  // "Single-request decode with KV-Cache operator"
  m.def("single_decode_with_kv_cache", single_decode_with_kv_cache);
  m.def("batch_decode_with_paged_kv_cache_plan", BatchDecodeWithPagedKVCachePlan);
  m.def("batch_decode_with_paged_kv_cache_run", BatchDecodeWithPagedKVCacheRun);

  // gemm
  // BMM FP8
  m.def("bmm_fp8", bmm_fp8);
  // Cutlass Segment GEMM operator
  m.def("cutlass_segment_gemm", CutlassSegmentGEMM);

  // norm
  // Root mean square normalization
  m.def("rmsnorm", rmsnorm);
  // Fused add root mean square normalization
  m.def("fused_add_rmsnorm", fused_add_rmsnorm);
  // Gemma Root mean square normalization
  m.def("gemma_rmsnorm", gemma_rmsnorm);
  // Gemma Fused add root mean square normalization
  m.def("gemma_fused_add_rmsnorm", gemma_fused_add_rmsnorm);

  // page
  // Append paged KV-Cache operator
  m.def("append_paged_kv_cache", append_paged_kv_cache);
  // Precompute block sparse offsets
  m.def("block_sparse_indices_to_vector_sparse_offsets",
        block_sparse_indices_to_vector_sparse_offsets);

  // prefill
  // Single-request prefill attention with KV-Cache operator
  m.def("single_prefill_with_kv_cache", single_prefill_with_kv_cache);
  m.def("batch_prefill_with_kv_cache_plan", BatchPrefillWithKVCachePlan);
  m.def("batch_prefill_with_ragged_kv_cache_run", BatchPrefillWithRaggedKVCacheRun);
  m.def("batch_prefill_with_paged_kv_cache_run", BatchPrefillWithPagedKVCacheRun);

  // quantization
  // GPU packbits operator
  m.def("packbits", packbits);
  // GPU segment packbits operator
  m.def("segment_packbits", segment_packbits);

  // rope
  // "Apply RoPE"
  m.def("apply_rope", apply_rope);
  // "Apply Llama 3.1 style RoPE"
  m.def("apply_llama31_rope", apply_llama31_rope);
  // "Apply RoPE with positional ids"
  m.def("apply_rope_pos_ids", apply_rope_pos_ids);
  // "Apply Llama 3.1 style RoPE with positional ids"
  m.def("apply_llama31_rope_pos_ids", apply_llama31_rope_pos_ids);
  // "Apply RoPE with positional ids and cosine/sine cache"
  m.def("apply_rope_pos_ids_cos_sin_cache", apply_rope_pos_ids_cos_sin_cache);

  // sampling
  // Sample from probabilities
  m.def("sampling_from_probs", sampling_from_probs);
  // Top-k sampling from probabilities
  m.def("top_k_sampling_from_probs", top_k_sampling_from_probs);
  // Min-p sampling from probabilities
  m.def("min_p_sampling_from_probs", min_p_sampling_from_probs);
  // Top-p sampling from probabilities
  m.def("top_p_sampling_from_probs", top_p_sampling_from_probs);
  // Top-k and top-p sampling from probabilities
  m.def("top_k_top_p_sampling_from_probs", top_k_top_p_sampling_from_probs);
  // Renormalize probabilities by top-k mask
  m.def("top_k_renorm_probs", top_k_renorm_probs);
  // Renormalize probabilities by top-p mask
  m.def("top_p_renorm_probs", top_p_renorm_probs);
  // Mask logits by top-k mask
  m.def("top_k_mask_logits", top_k_mask_logits);
  // Speculative sampling from sequence of probabilities
  m.def("chain_speculative_sampling", chain_speculative_sampling);
}
