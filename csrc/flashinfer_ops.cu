/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "aot_extension_utils.h"

//========== decode ==========

void single_decode_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                 std::optional<at::Tensor> alibi_slopes, at::Tensor o,
                                 unsigned int layout, int window_left, float logits_soft_cap,
                                 float sm_scale, float rope_scale, float rope_theta,
                                 int64_t hip_stream);

std::vector<int64_t> BatchDecodeWithPagedKVCachePlan(
    bool use_logits_soft_cap, unsigned int head_dim, at::Tensor empty_q_data,
    at::Tensor empty_kv_data, at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor indptr, unsigned int batch_size,
    unsigned int num_qo_heads, unsigned int num_kv_heads, unsigned int page_size,
    bool enable_cuda_graph, int64_t hip_stream);

void BatchDecodeWithPagedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, at::Tensor paged_kv_indptr, at::Tensor paged_kv_indices,
    at::Tensor paged_kv_last_page_len, std::optional<at::Tensor> alibi_slopes, at::Tensor o,
    unsigned int kv_layout_code, int window_left, float logits_soft_cap, float sm_scale,
    float rope_scale, float rope_theta, std::optional<at::Tensor> maybe_lse, int64_t hip_stream);

//========== prefill ==========

void single_prefill_with_kv_cache(unsigned int mask_mode_code, at::Tensor q, at::Tensor k,
                                  at::Tensor v, std::optional<at::Tensor> maybe_packed_custom_mask,
                                  at::Tensor tmp, std::optional<at::Tensor> maybe_alibi_slopes,
                                  at::Tensor o, unsigned int layout, int32_t window_left,
                                  float logits_soft_cap, float sm_scale, float rope_scale,
                                  float rope_theta, std::optional<at::Tensor> maybe_lse,
                                  int64_t hip_stream);

std::vector<int64_t> BatchPrefillWithKVCachePlan(
    unsigned int head_dim, at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor qo_indptr, at::Tensor kv_indptr,
    unsigned total_num_rows, unsigned int batch_size, unsigned int num_qo_heads,
    unsigned int num_kv_heads, unsigned int page_size, bool enable_cuda_graph, int64_t hip_stream);

void BatchPrefillWithRaggedKVCacheRun(
    unsigned int mask_mode_code, at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor k, at::Tensor v,
    std::optional<at::Tensor> maybe_custom_mask, std::optional<at::Tensor> maybe_alibi_slopes,
    at::Tensor qo_indptr, at::Tensor kv_indptr, std::optional<at::Tensor> maybe_qk_indptr,
    at::Tensor o, unsigned int layout, int32_t window_left, float logits_soft_cap, float sm_scale,
    float rope_scale, float rope_theta, std::optional<at::Tensor> maybe_lse, int64_t hip_stream);

void BatchPrefillWithPagedKVCacheRun(
    unsigned int mask_mode_code, at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, std::optional<at::Tensor> maybe_custom_mask,
    std::optional<at::Tensor> maybe_alibi_slopes, at::Tensor qo_indptr, at::Tensor paged_kv_indptr,
    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len,
    std::optional<at::Tensor> maybe_qk_indptr, at::Tensor o, unsigned int layout,
    int32_t window_left, float logits_soft_cap, float sm_scale, float rope_scale, float rope_theta,
    std::optional<at::Tensor> maybe_lse, int64_t hip_stream);

//========== pybind11 ==========

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  // decode
  m.def("single_decode_with_kv_cache", &single_decode_with_kv_cache,
        "Single-request decode with KV-Cache operator");
  m.def("batch_decode_with_paged_kv_cache_plan", &BatchDecodeWithPagedKVCachePlan);
  m.def("batch_decode_with_paged_kv_cache_run", &BatchDecodeWithPagedKVCacheRun);

  // prefill
  m.def("single_prefill_with_kv_cache", &single_prefill_with_kv_cache,
        "Single-request prefill attention with KV-Cache operator");
  m.def("batch_prefill_with_kv_cache_plan", &BatchPrefillWithKVCachePlan);
  m.def("batch_prefill_with_ragged_kv_cache_run", &BatchPrefillWithRaggedKVCacheRun);
  m.def("batch_prefill_with_paged_kv_cache_run", &BatchPrefillWithPagedKVCacheRun);
}
