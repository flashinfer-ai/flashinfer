#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "aot_default_additional_params.h"
#include "pytorch_extension_utils.h"

//========== activation ==========

void silu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);
void gelu_tanh_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);
void gelu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);

//========== cascade ==========

void merge_state(at::Tensor v_a, at::Tensor s_a, at::Tensor v_b, at::Tensor s_b,
                 at::Tensor v_merged, at::Tensor s_merged, int64_t hip_stream);

void merge_state_in_place(at::Tensor v, at::Tensor s, at::Tensor v_other, at::Tensor s_other,
                          std::optional<at::Tensor> mask, int64_t hip_stream);

void merge_states(at::Tensor v, at::Tensor s, at::Tensor v_merged, at::Tensor s_merged,
                  int64_t hip_stream);

//========== decode ==========

void single_decode_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                 at::Tensor o, unsigned int layout,
                                 int window_left SINGLE_DECODE_ADDITIONAL_FUNC_PARAMS,
                                 int64_t hip_stream);

std::vector<int64_t> BatchDecodeWithPagedKVCachePlan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor indptr, unsigned int batch_size,
    unsigned int num_qo_heads, unsigned int num_kv_heads, unsigned int page_size,
    bool enable_cuda_graph, int window_left, float logits_soft_cap, unsigned int head_dim,
    at::Tensor empty_q_data, at::Tensor empty_kv_data, int64_t hip_stream);

void BatchDecodeWithPagedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, at::Tensor paged_kv_indptr, at::Tensor paged_kv_indices,
    at::Tensor paged_kv_last_page_len, at::Tensor o, std::optional<at::Tensor> maybe_lse,
    unsigned int kv_layout_code, int window_left BATCH_DECODE_ADDITIONAL_FUNC_PARAMS,
    int64_t hip_stream);

//========== gemm ==========

void bmm_fp8(at::Tensor A, at::Tensor B, at::Tensor D, at::Tensor A_scale, at::Tensor B_scale,
             at::Tensor workspace_buffer, int64_t cublas_handle, int64_t hip_stream);

void CutlassSegmentGEMM(at::Tensor workspace_buffer, at::Tensor all_problems, at::Tensor x_ptr,
                        at::Tensor w_ptr, at::Tensor y_ptr, at::Tensor x_ld, at::Tensor w_ld,
                        at::Tensor y_ld, at::Tensor empty_x_data, bool weight_column_major,
                        int64_t hip_stream);

//========== norm ==========

void rmsnorm(at::Tensor& out, at::Tensor& input, at::Tensor& weight, double eps,
             int64_t hip_stream);

void fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight, double eps,
                       int64_t hip_stream);

void gemma_rmsnorm(at::Tensor& out, at::Tensor& input, at::Tensor& weight, double eps,
                   int64_t hip_stream);

void gemma_fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight,
                             double eps, int64_t hip_stream);

//========== page ==========

void append_paged_kv_cache(at::Tensor append_key, at::Tensor append_value, at::Tensor batch_indices,
                           at::Tensor positions, at::Tensor paged_k_cache, at::Tensor paged_v_cache,
                           at::Tensor kv_indices, at::Tensor kv_indptr, at::Tensor kv_last_page_len,
                           unsigned int layout, int64_t hip_stream);

void block_sparse_indices_to_vector_sparse_offsets(at::Tensor block_sparse_indices,
                                                   at::Tensor block_sparse_indptr,
                                                   at::Tensor vector_sparse_offsets,
                                                   at::Tensor vector_sparse_indptr,
                                                   at::Tensor kv_len_arr, unsigned int stride_block,
                                                   unsigned int stride_n, unsigned int batch_size,
                                                   unsigned int block_size, int64_t hip_stream);

//========== prefill ==========

void single_prefill_with_kv_cache(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor tmp,
                                  at::Tensor o, std::optional<at::Tensor> maybe_lse,
                                  unsigned int mask_mode_code, unsigned int layout,
                                  int32_t window_left SINGLE_PREFILL_ADDITIONAL_FUNC_PARAMS,
                                  int64_t hip_stream);

std::vector<int64_t> BatchPrefillWithKVCachePlan(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    at::Tensor page_locked_int_workspace_buffer, at::Tensor qo_indptr, at::Tensor kv_indptr,
    at::Tensor kv_len_arr, unsigned total_num_rows, unsigned int batch_size,
    unsigned int num_qo_heads, unsigned int num_kv_heads, unsigned int page_size,
    bool enable_cuda_graph, unsigned int head_dim, bool causal, int64_t hip_stream);

void BatchPrefillWithRaggedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor k, at::Tensor v,
    at::Tensor qo_indptr, at::Tensor kv_indptr, at::Tensor o, std::optional<at::Tensor> maybe_lse,
    unsigned int mask_mode_code, unsigned int layout,
    int32_t window_left BATCH_PREFILL_ADDITIONAL_FUNC_PARAMS, int64_t hip_stream);

void BatchPrefillWithPagedKVCacheRun(
    at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
    std::vector<int64_t> plan_info_vec, at::Tensor q, at::Tensor paged_k_cache,
    at::Tensor paged_v_cache, at::Tensor qo_indptr, at::Tensor paged_kv_indptr,
    at::Tensor paged_kv_indices, at::Tensor paged_kv_last_page_len, at::Tensor o,
    std::optional<at::Tensor> maybe_lse, unsigned int mask_mode_code, unsigned int layout,
    int32_t window_left BATCH_PREFILL_ADDITIONAL_FUNC_PARAMS, int64_t hip_stream);

//========== quantization ==========

void packbits(at::Tensor x, const std::string& bitorder, at::Tensor y, int64_t hip_stream);

void segment_packbits(at::Tensor x, at::Tensor input_indptr, at::Tensor output_indptr,
                      const std::string& bitorder, at::Tensor y, int64_t hip_stream);

//========== rope ==========

void apply_rope(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope, at::Tensor indptr,
                at::Tensor offsets, unsigned int rotary_dim, bool interleave, float rope_scale,
                float rope_theta, int64_t hip_stream);

void apply_llama31_rope(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope,
                        at::Tensor indptr, at::Tensor offsets, unsigned int rotary_dim,
                        bool interleave, float rope_scale, float rope_theta, float low_freq_factor,
                        float high_freq_factor, float old_context_length, int64_t hip_stream);

void apply_rope_pos_ids(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope,
                        at::Tensor pos_ids, unsigned int rotary_dim, bool interleave,
                        float rope_scale, float rope_theta, int64_t hip_stream);

void apply_llama31_rope_pos_ids(at::Tensor q, at::Tensor k, at::Tensor q_rope, at::Tensor k_rope,
                                at::Tensor pos_ids, unsigned int rotary_dim, bool interleave,
                                float rope_scale, float rope_theta, float low_freq_factor,
                                float high_freq_factor, float old_context_length,
                                int64_t hip_stream);

void apply_rope_pos_ids_cos_sin_cache(at::Tensor q, at::Tensor k, at::Tensor q_rope,
                                      at::Tensor k_rope, at::Tensor cos_sin_cache,
                                      at::Tensor pos_ids, bool interleave, int64_t hip_stream);

//========== sampling ==========

void sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                         bool deterministic, int64_t hip_stream);

void top_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               at::Tensor success, std::optional<at::Tensor> maybe_top_p_arr,
                               double top_p_val, bool deterministic, int64_t hip_stream);

void top_k_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               at::Tensor success, std::optional<at::Tensor> maybe_top_k_arr,
                               unsigned int top_k_val, bool deterministic, int64_t hip_stream);

void min_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               std::optional<at::Tensor> maybe_min_p_arr, double min_p_val,
                               bool deterministic, int64_t hip_stream);

void top_k_top_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples,
                                     at::Tensor samples, at::Tensor success,
                                     std::optional<at::Tensor> maybe_top_k_arr, double top_k_val,
                                     std::optional<at::Tensor> maybe_top_p_arr, double top_p_val,
                                     bool deterministic, int64_t hip_stream);

void top_p_renorm_probs(at::Tensor probs, at::Tensor renorm_probs,
                        std::optional<at::Tensor> maybe_top_p_arr, double top_p_val,
                        int64_t hip_stream);

void top_k_renorm_probs(at::Tensor probs, at::Tensor renorm_probs,
                        std::optional<at::Tensor> maybe_top_k_arr, unsigned int top_k_val,
                        int64_t hip_stream);

void top_k_mask_logits(at::Tensor logits, at::Tensor mask_logits,
                       std::optional<at::Tensor> maybe_top_k_arr, unsigned int top_k_val,
                       int64_t hip_stream);

void chain_speculative_sampling(at::Tensor draft_probs, at::Tensor draft_token_ids,
                                at::Tensor uniform_samples, at::Tensor target_probs,
                                at::Tensor output_token_ids, at::Tensor output_accepted_token_num,
                                at::Tensor output_emitted_token_num, bool deterministic,
                                int64_t hip_stream);

//========== pybind11 ==========

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  // activation
  m.def("silu_and_mul", &silu_and_mul, "Fused SiLU and Mul");
  m.def("gelu_tanh_and_mul", &gelu_tanh_and_mul, "Fused GeLU Tanh and Mul");
  m.def("gelu_and_mul", &gelu_and_mul, "Fused GeLU and Mul");

  // cascade
  m.def("merge_state", &merge_state, "Merge two self-attention states");
  m.def("merge_state_in_place", &merge_state_in_place,
        "Merge another self-attention state in-place.");
  m.def("merge_states", &merge_states, "Merge multiple self-attention states");

  // decode
  m.def("single_decode_with_kv_cache", &single_decode_with_kv_cache,
        "Single-request decode with KV-Cache operator");
  m.def("batch_decode_with_paged_kv_cache_plan", &BatchDecodeWithPagedKVCachePlan);
  m.def("batch_decode_with_paged_kv_cache_run", &BatchDecodeWithPagedKVCacheRun);

  // gemm
  m.def("bmm_fp8", &bmm_fp8, "BMM FP8");
  m.def("cutlass_segment_gemm", &CutlassSegmentGEMM, "Cutlass Segment GEMM operator");

  // norm
  m.def("rmsnorm", &rmsnorm, "Root mean square normalization");
  m.def("fused_add_rmsnorm", &fused_add_rmsnorm, "Fused add root mean square normalization");
  m.def("gemma_rmsnorm", &gemma_rmsnorm, "Gemma Root mean square normalization");
  m.def("gemma_fused_add_rmsnorm", &gemma_fused_add_rmsnorm,
        "Gemma Fused add root mean square normalization");

  // page
  m.def("append_paged_kv_cache", &append_paged_kv_cache, "Append paged KV-Cache operator");
  m.def("block_sparse_indices_to_vector_sparse_offsets",
        &block_sparse_indices_to_vector_sparse_offsets, "Precompute block sparse offsets");

  // prefill
  m.def("single_prefill_with_kv_cache", &single_prefill_with_kv_cache,
        "Single-request prefill attention with KV-Cache operator");
  m.def("batch_prefill_with_kv_cache_plan", &BatchPrefillWithKVCachePlan);
  m.def("batch_prefill_with_ragged_kv_cache_run", &BatchPrefillWithRaggedKVCacheRun);
  m.def("batch_prefill_with_paged_kv_cache_run", &BatchPrefillWithPagedKVCacheRun);

  // quantization
  m.def("packbits", &packbits, "GPU packbits operator");
  m.def("segment_packbits", &segment_packbits, "GPU segment packbits operator");

  // rope
  m.def("apply_rope", &apply_rope, "Apply RoPE");
  m.def("apply_llama31_rope", &apply_llama31_rope, "Apply Llama 3.1 style RoPE");
  m.def("apply_rope_pos_ids", &apply_rope_pos_ids, "Apply RoPE with positional ids");
  m.def("apply_llama31_rope_pos_ids", &apply_llama31_rope_pos_ids,
        "Apply Llama 3.1 style RoPE with positional ids");
  m.def("apply_rope_pos_ids_cos_sin_cache", &apply_rope_pos_ids_cos_sin_cache,
        "Apply RoPE with positional ids and cosine/sine cache");

  // sampling
  m.def("sampling_from_probs", &sampling_from_probs, "Sample from probabilities");
  m.def("top_k_sampling_from_probs", &top_k_sampling_from_probs,
        "Top-k sampling from probabilities");
  m.def("min_p_sampling_from_probs", &min_p_sampling_from_probs,
        "Min-p sampling from probabilities");
  m.def("top_p_sampling_from_probs", &top_p_sampling_from_probs,
        "Top-p sampling from probabilities");
  m.def("top_k_top_p_sampling_from_probs", &top_k_top_p_sampling_from_probs,
        "Top-k and top-p sampling from probabilities");
  m.def("top_k_renorm_probs", &top_k_renorm_probs, "Renormalize probabilities by top-k mask");
  m.def("top_p_renorm_probs", &top_p_renorm_probs, "Renormalize probabilities by top-p mask");
  m.def("top_k_mask_logits", &top_k_mask_logits, "Mask logits by top-k mask");
  m.def("chain_speculative_sampling", &chain_speculative_sampling,
        "Speculative sampling from sequence of probabilities");
}
