/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/scheduler.cuh>
#include <flashinfer/layout.cuh>
#include <flashinfer/math.cuh>

#include "batch_prefill_sm90_config.inc"
#include "tvm_ffi_utils.h"

namespace flashinfer {

template <uint32_t HEAD_DIM, MaskMode MASK_MODE, bool LEFT_SLIDING_WINDOW,
          bool SAME_SCHEDULE_FOR_ALL_HEADS, typename AttentionVariant, typename Params>
hipError_t BatchFP8PrefillWithPagedKVCacheDispatched(Params& params, bool enable_pdl,
                                                      hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

using tvm::ffi::Array;
using tvm::ffi::Optional;

Array<int64_t> BatchPrefillWithKVCacheSM90Plan(
    ffi::Tensor float_workspace_buffer, ffi::Tensor int_workspace_buffer,
    ffi::Tensor page_locked_int_workspace_buffer, ffi::Tensor qo_indptr, ffi::Tensor kv_indptr,
    ffi::Tensor kv_len_arr, int64_t total_num_rows, int64_t batch_size, int64_t num_qo_heads,
    int64_t num_kv_heads, int64_t page_size, bool enable_cuda_graph, int64_t head_dim_qk,
    int64_t head_dim_vo, bool causal, int64_t window_left) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer->shape[0] * get_element_size(float_workspace_buffer);
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer->shape[0] * get_element_size(int_workspace_buffer);

  flashinfer::PrefillPlanSM90Info plan_info;

  hipSetDevice(float_workspace_buffer->device.device_id);
  const hipStream_t stream = get_stream(float_workspace_buffer->device);

  hipError_t status = PrefillSM90Plan(
      float_workspace_buffer->data, float_workspace_size_in_bytes, int_workspace_buffer->data,
      page_locked_int_workspace_buffer->data, int_workspace_size_in_bytes, plan_info,
      static_cast<IdType*>(qo_indptr->data), static_cast<IdType*>(kv_indptr->data),
      static_cast<IdType*>(kv_len_arr->data), total_num_rows, batch_size, num_qo_heads,
      num_kv_heads, head_dim_qk, head_dim_vo, page_size, causal, enable_cuda_graph,
      /*sizeof_dtype_o=*/2, stream);

  TVM_FFI_ICHECK(status == hipSuccess)
      << "PrefillSM90Plan failed with error: " << hipGetErrorString(status);

  return Array(plan_info.ToVector());
}

void BatchPrefillWithRaggedKVCacheSM90Run(ffi::Tensor float_workspace_buffer,
                                          ffi::Tensor int_workspace_buffer,
                                          Array<int64_t> plan_info_vec, ffi::Tensor q,
                                          ffi::Tensor k, ffi::Tensor v, ffi::Tensor qo_indptr,
                                          ffi::Tensor kv_indptr, ffi::Tensor o,
                                          Optional<ffi::Tensor> maybe_lse, int64_t mask_mode_code,
                                          int64_t layout, int64_t window_left,
                                          bool enable_pdl  // placeholder
                                              ADDITIONAL_FUNC_PARAMS) {
  return;  // TODO: Implement this function
}

void BatchPrefillWithPagedKVCacheSM90Run(
    ffi::Tensor float_workspace_buffer, ffi::Tensor int_workspace_buffer,
    Array<int64_t> plan_info_vec, ffi::Tensor q, ffi::Tensor paged_k_cache,
    ffi::Tensor paged_v_cache, ffi::Tensor qo_indptr, ffi::Tensor paged_kv_indptr,
    ffi::Tensor paged_kv_indices, ffi::Tensor paged_kv_last_page_len, ffi::Tensor o,
    Optional<ffi::Tensor> maybe_lse, int64_t mask_mode_code, int64_t layout, int64_t window_left,
    bool enable_pdl ADDITIONAL_FUNC_PARAMS) {
  PrefillPlanSM90Info plan_info;
  plan_info.FromVector(std::vector<int64_t>(plan_info_vec.begin(), plan_info_vec.end()));

  if (maybe_lse.has_value()) {
    const auto& lse = maybe_lse.value();
    TVM_FFI_ICHECK_EQ(lse->shape[0], q->shape[0]);
    TVM_FFI_ICHECK_EQ(lse->shape[1], q->shape[1]);
  }
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  int64_t num_kv_heads, page_size;
  int64_t head_dim_qk = q->shape[2];
  int64_t head_dim_vo = paged_v_cache->shape[3];
  if (kv_layout == QKVLayout::kHND) {
    num_kv_heads = paged_k_cache->shape[1];
    page_size = paged_k_cache->shape[2];
  } else {
    page_size = paged_k_cache->shape[1];
    num_kv_heads = paged_k_cache->shape[2];
  }

  void* float_buffer_ptr = float_workspace_buffer->data;
  void* int_buffer_ptr = int_workspace_buffer->data;

  hipSetDevice(float_workspace_buffer->device.device_id);
  const hipStream_t stream = get_stream(float_workspace_buffer->device);
  const MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  bool use_swa = window_left != -1;

  DISPATCH_context(
      DTypeQ, DTypeKV, DTypeO, IdType, MASK_MODE, HEAD_DIM_QK, HEAD_DIM_VO, USE_SLIDING_WINDOW,
      USE_LOGITS_SOFT_CAP, AttentionVariant, RaggedParams, PagedParams, [&] {
        PagedParams params;

        params.q_ptr = static_cast<DTypeQ*>(q->data);
        params.k_ptr = static_cast<DTypeKV*>(paged_k_cache->data);
        params.v_ptr = static_cast<DTypeKV*>(paged_v_cache->data);
        params.o_ptr = static_cast<DTypeO*>(o->data);
        params.lse_ptr = maybe_lse ? static_cast<float*>(maybe_lse.value()->data) : nullptr;
        params.q_stride_n = q->strides[0];
        params.q_stride_h = q->strides[1];
        params.o_stride_n = o->strides[0];
        params.o_stride_h = o->strides[1];
        if (kv_layout == QKVLayout::kNHD) {
          // (num_pages, page_size, num_heads, head_dim)
          params.k_stride_n = paged_k_cache->strides[1];
          params.k_stride_h = paged_k_cache->strides[2];
          params.v_stride_n = paged_v_cache->strides[1];
          params.v_stride_h = paged_v_cache->strides[2];
        } else {
          // (num_pages, num_heads, page_size, head_dim)
          params.k_stride_h = paged_k_cache->strides[1];
          params.k_stride_n = paged_k_cache->strides[2];
          params.v_stride_h = paged_v_cache->strides[1];
          params.v_stride_n = paged_v_cache->strides[2];
        }
        params.nnz_qo = q->shape[0];
        params.num_qo_heads = q->shape[1];
        params.num_kv_heads = num_kv_heads;
        params.group_size = params.num_qo_heads / num_kv_heads;
        params.page_size = page_size;
        params.window_left = window_left;
        params.causal = mask_mode_code == 1;
        params.qo_tile_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
        params.qo_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_indptr_offset);
        params.kv_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_indptr_offset);
        params.qo_lens = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_len_offset);
        params.kv_lens = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_len_offset);
        params.batch_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.batch_indices_offset);
        params.head_indices =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.head_indices_offset);
        params.work_indptr =
            GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.work_indptr_offset);
        params.kv_indices = static_cast<IdType*>(paged_kv_indices->data);

        ADDITIONAL_PARAMS_SETTER

        // Not support various head_dim for now
        static_assert(HEAD_DIM_QK == HEAD_DIM_VO, "head_dim_qk and head_dim_vo should be the same");
        // Currently only support same quantization precision
        static_assert(std::is_same_v<DTypeQ, DTypeKV>);

        bool same_schedule_for_all_heads = plan_info.same_schedule_for_all_heads;
        DISPATCH_BOOL(same_schedule_for_all_heads, SAME_SCHEDULER_FOR_ALL_HEADS, [&] {
          hipError_t status =
              BatchFP8PrefillWithPagedKVCacheDispatched<HEAD_DIM_QK, MASK_MODE, USE_SLIDING_WINDOW,
                                                        SAME_SCHEDULER_FOR_ALL_HEADS,
                                                        AttentionVariant>(params, enable_pdl,
                                                                          stream);

          TVM_FFI_ICHECK(status == hipSuccess)
              << "BatchPrefillWithPagedKVCacheSM90Run failed with error: "
              << hipGetErrorString(status);
          return true;
        });
      });
}
