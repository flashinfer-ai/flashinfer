/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "pytorch_extension_utils.h"

void rmsnorm(at::Tensor& out, at::Tensor& input, at::Tensor& weight, double eps,
             int64_t hip_stream);

void fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight, double eps,
                       int64_t hip_stream);

void gemma_rmsnorm(at::Tensor& out, at::Tensor& input, at::Tensor& weight, double eps,
                   int64_t hip_stream);

void gemma_fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight,
                             double eps, int64_t hip_stream);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("rmsnorm", &rmsnorm, "Root mean square normalization");
  m.def("fused_add_rmsnorm", &fused_add_rmsnorm, "Fused add root mean square normalization");
  m.def("gemma_rmsnorm", &gemma_rmsnorm, "Gemma Root mean square normalization");
  m.def("gemma_fused_add_rmsnorm", &gemma_fused_add_rmsnorm,
        "Gemma Fused add root mean square normalization");
}
