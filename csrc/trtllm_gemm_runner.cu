/*
 * Copyright (c) 2020-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <string>

#include "flashinfer/exception.h"
#include "flashinfer/trtllm/common.h"
#include "flashinfer/trtllm/gemm/trtllmGen_gemm_export/Enums.h"
#include "flashinfer/trtllm/gemm/trtllmGen_gemm_export/GemmInterface.h"
#include "flashinfer/trtllm/gemm/trtllmGen_gemm_export/trtllm/gen/DtypeDecl.h"
#include "flashinfer/trtllm/gemm/trtllmGen_gemm_export/trtllm/gen/SfLayoutDecl.h"
#include "tvm/ffi/container/array.h"
#include "tvm_ffi_utils.h"

namespace {
static thread_local gemm::gemm::GemmInterface::ModuleCache globalTrtllmGenGemmModuleCache;
}  // namespace

namespace flashinfer {

struct TrtllmGenGemmRunnerOptions {
  gemm::trtllm::gen::Dtype eltType;
  gemm::trtllm::gen::Dtype outputType;
  bool transposeMmaOutput{false};
  gemm::trtllm::gen::SfLayout sfLayoutB;
};

int64_t select_kernel_fp8(int32_t M, int32_t N, int32_t K,
                          const gemm::gemm::GemmInterface& interface) {
  static constexpr const char* KERNEL_NAME_HIGH_N_K_RATIO =
      "gemm_Bfloat16_E4m3E4m3_Fp32_t128x8x128u2_s6_et64x8_m64x8x32_cga1x1x1_16dp256b_rM_TN_"
      "transOut_"
      "noShflA_dsFp8_schedP2x2x1x3_sm100f";

  static constexpr const char* KERNEL_NAME_LOW_N_K_RATIO =
      "gemm_Bfloat16_E4m3E4m3_Fp32_t128x32x128u2_s6_et64x32_m64x32x32_cga1x1x1_16dp256b_rM_TN_"
      "transOut_noShflA_dsFp8_schedS_sm100f";

  static constexpr const char* KERNEL_NAME_LARGE_N =
      "gemm_Bfloat16_E4m3E4m3_Fp32_t128x32x128u2_s6_et64x32_m64x32x32_cga1x1x1_16dp256b_rM_TN_"
      "transOut_noShflA_dsFp8_schedP2x2x1x3_sm100f";

  static constexpr const char* KERNEL_NAME_DEFAULT =
      "gemm_Bfloat16_E4m3E4m3_Fp32_t128x16x128u2_s6_et64x16_m64x16x32_cga1x1x1_16dp256b_rM_TN_"
      "transOut_noShflA_dsFp8_schedS_sm100f";

  double const n_k_ratio = static_cast<double>(N) / static_cast<double>(K);

  std::string kernel_name;
  if (n_k_ratio >= 32) {
    kernel_name = KERNEL_NAME_HIGH_N_K_RATIO;
  } else if (n_k_ratio <= 2.0) {
    kernel_name = KERNEL_NAME_LOW_N_K_RATIO;
  } else if (N >= 20000) {
    kernel_name = KERNEL_NAME_LARGE_N;
  } else {
    kernel_name = KERNEL_NAME_DEFAULT;
  }

  auto const& configs = interface.getGemmConfigs();
  size_t const num_configs = interface.getNumGemmConfigs();

  for (size_t i = 0; i < num_configs; ++i) {
    if (std::string(configs[i].mFunctionName) == kernel_name) {
      return static_cast<int64_t>(i);
    }
  }

  TVM_FFI_ICHECK(false) << "Kernel not found";
}

class TrtllmGenGemmRunner {
 public:
  explicit TrtllmGenGemmRunner(TrtllmGenGemmRunnerOptions const& options) : mOptions(options) {
    // Select a GEMM kernel config to use
    auto const gemm = gemm::gemm::GemmInterface();
    auto const configs = gemm.getGemmConfigs();

    mPassingConfigIndices.clear();

    for (size_t i = 0; i < gemm.getNumGemmConfigs(); ++i) {
      auto const options = configs[i].mOptions;

      if (options.mDtypeA == mOptions.eltType && options.mDtypeC == mOptions.outputType &&
          options.mTransposeMmaOutput == mOptions.transposeMmaOutput &&
          options.mSfLayoutB == mOptions.sfLayoutB) {
        mPassingConfigIndices.push_back(i);
      }
    }

    FLASHINFER_CHECK(
        mPassingConfigIndices.size() > 0,
        "No valid tactic found for the given options (precision, transpose, sf layout)");
  }

  int64_t getWorkspaceSizeInBytes(int64_t m, int64_t n, int64_t k, int64_t tactic) {
    auto gemm = gemm::gemm::GemmInterface();
    auto const configs = gemm.getGemmConfigs();
    FLASHINFER_CHECK(tactic >= 0 && tactic < gemm.getNumGemmConfigs(),
                     "Invalid tactic in getWorkspaceSizeInBytes");
    auto const config = configs[tactic];

    gemm::gemm::GemmData gemmData;
    gemmData.mProblemDimensions.mM = mOptions.transposeMmaOutput ? n : m;
    gemmData.mProblemDimensions.mN = mOptions.transposeMmaOutput ? m : n;
    gemmData.mProblemDimensions.mK = k;
    gemmData.mProblemDimensions.mRank = 0;
    gemmData.mProblemDimensions.mWorldSize = 1;

    return gemm.getWorkspaceSizeInBytes(config, gemmData);
  }

  void run(int64_t m, int64_t n, int64_t k, void const* a, void const* aScale, void const* b,
           void const* bScale, void* c, void* cScale, void* cScalePtr, void* workspace,
           hipStream_t stream, int32_t device_index, int64_t tactic) {
    auto gemm = gemm::gemm::GemmInterface();
    auto const configs = gemm.getGemmConfigs();
    TVM_FFI_ICHECK(tactic >= 0 && tactic < gemm.getNumGemmConfigs()) << "Invalid tactic id in run";
    auto const& config = configs[tactic];
    TVM_FFI_ICHECK(config.mOptions.mSfLayoutB == mOptions.sfLayoutB) << "Invalid sf layout in run";

    gemm::gemm::GemmData gemmData;
    // Dims
    gemmData.mProblemDimensions.mM = mOptions.transposeMmaOutput ? n : m;
    gemmData.mProblemDimensions.mN = mOptions.transposeMmaOutput ? m : n;
    gemmData.mProblemDimensions.mK = k;
    gemmData.mProblemDimensions.mRank = 0;
    gemmData.mProblemDimensions.mWorldSize = 1;

    // Inputs
    gemmData.mInputBuffers.mPtrA = mOptions.transposeMmaOutput ? b : a;
    gemmData.mInputBuffers.mPtrSfA = mOptions.transposeMmaOutput ? bScale : aScale;
    gemmData.mInputBuffers.mPtrB = mOptions.transposeMmaOutput ? a : b;
    gemmData.mInputBuffers.mPtrSfB = mOptions.transposeMmaOutput ? aScale : bScale;
    gemmData.mInputBuffers.mPtrScaleC = cScale;

    // Outputs
    gemmData.mOutputBuffers.mPtrC = c;
    gemmData.mOutputBuffers.mPtrSfC = cScalePtr;

    TVM_FFI_ICHECK(gemm.isValidConfig(config, gemmData)) << "unsupported tactic id in run";

    const int32_t multiProcessorCount = [device_index]() {
      static thread_local int32_t cached_multi_processor_count = -1;
      static thread_local int cached_device_index = -1;

      if (device_index == cached_device_index && cached_multi_processor_count != -1) {
        return cached_multi_processor_count;
      } else {
        int32_t count;
        hipError_t cudaStatus =
            hipDeviceGetAttribute(&count, hipDeviceAttributeMultiprocessorCount, device_index);
        TVM_FFI_ICHECK(cudaStatus == hipSuccess)
            << "Failed to get device attribute: " << hipGetErrorString(cudaStatus);
        cached_multi_processor_count = count;
        cached_device_index = device_index;
        return count;
      }
    }();

    TVM_FFI_ICHECK(gemm.run(config, workspace, gemmData, static_cast<void*>(stream),
                            multiProcessorCount, true, globalTrtllmGenGemmModuleCache) == 0)
        << "Error occurred when running GEMM!";
  }

  std::vector<int64_t> getValidTactics(int64_t m, int64_t n, int64_t k) const {
    auto const gemm = gemm::gemm::GemmInterface();
    auto const configs = gemm.getGemmConfigs();

    gemm::gemm::GemmData gemmData;
    // Dims
    gemmData.mProblemDimensions.mM = mOptions.transposeMmaOutput ? n : m;
    gemmData.mProblemDimensions.mN = mOptions.transposeMmaOutput ? m : n;
    gemmData.mProblemDimensions.mK = k;
    gemmData.mProblemDimensions.mRank = 0;
    gemmData.mProblemDimensions.mWorldSize = 1;

    std::vector<int64_t> sortedIndices = mPassingConfigIndices;
    std::sort(sortedIndices.begin(), sortedIndices.end(), [&configs](int64_t idx0, int64_t idx1) {
      auto const& optionsA = configs[idx0].mOptions;
      auto const& optionsB = configs[idx1].mOptions;

      // Sort by tileK sizes first
      if (optionsA.mTileK != optionsB.mTileK) {
        return optionsA.mTileK > optionsB.mTileK;
      }

      // Then by splitK sizes
      if (optionsA.mNumSlicesForSplitK != optionsB.mNumSlicesForSplitK) {
        return optionsA.mNumSlicesForSplitK > optionsB.mNumSlicesForSplitK;
      }

      // Then by unroll loop 2x for mma
      if (optionsA.mUseUnrollLoop2xForMma != optionsB.mUseUnrollLoop2xForMma) {
        return optionsA.mUseUnrollLoop2xForMma;
      }

      return false;
    });

    bool findLoop2xMma = false;
    std::vector<int64_t> validTactics;
    for (auto const& configIndex : sortedIndices) {
      auto const& config = configs[configIndex];
      if (gemm.isValidConfig(config, gemmData)) {
        validTactics.push_back(configIndex);

        // when loop2x mma is found, only add the tactic that has loop2x mma
        if (!findLoop2xMma) {
          if (config.mOptions.mUseUnrollLoop2xForMma) {
            findLoop2xMma = true;
          }
        } else {
          if (!config.mOptions.mUseUnrollLoop2xForMma) {
            break;
          }
        }
      }
    }
    return validTactics;
  }

  int64_t selectHeuristic(int64_t m, int64_t n, int64_t k) const {
    if (mOptions.eltType == gemm::trtllm::gen::Dtype::E4m3) {
      return select_kernel_fp8(m, n, k, gemm::gemm::GemmInterface());
    } else if (mOptions.eltType == gemm::trtllm::gen::Dtype::E2m1) {
      auto sortedIndices = getValidTactics(m, n, k);
      TVM_FFI_ICHECK(!sortedIndices.empty()) << "No valid tactic found";

      // the getValidTactics is sorted by priority, so the first one is the best one
      return sortedIndices[0];
    } else {
      TVM_FFI_ICHECK(false) << "Unsupported eltType";
    }
  }

 private:
  TrtllmGenGemmRunnerOptions mOptions;
  std::vector<int64_t> mPassingConfigIndices;
};

using tvm::ffi::Array;
using tvm::ffi::Optional;

void trtllm_gemm(TensorView workspace_buffer, TensorView a, TensorView b, TensorView a_scale,
                 TensorView b_scale, Optional<TensorView> globalScale, TensorView out,
                 bool use_8x4_sf_layout, int64_t tactic) {
  CHECK_DEVICE(a, b);
  CHECK_DEVICE(a, out);
  CHECK_INPUT(a);
  CHECK_INPUT(b);
  CHECK_INPUT(out);
  CHECK_INPUT(workspace_buffer);
  TVM_FFI_ICHECK_EQ(workspace_buffer->ndim, 1);
  CHECK_DIM(2, a);
  CHECK_DIM(2, b);
  TVM_FFI_ICHECK_EQ(a->dtype, b->dtype);
  TVM_FFI_ICHECK(a->dtype == dl_float8_e4m3fn || a->dtype == dl_uint8)
      << "a must be a Float8 or Byte(e2m1) tensor";
  bool is_fp8 = a->dtype == dl_float8_e4m3fn;
  if (is_fp8) {
    TVM_FFI_ICHECK(!globalScale.has_value()) << "globalScale must be a none tensor";
  } else {
    CHECK_INPUT(a_scale);
    CHECK_INPUT(b_scale);
    if (globalScale.has_value()) {
      CHECK_INPUT(globalScale.value());
    }
  }

  int32_t m = a->shape[0];
  int32_t k = is_fp8 ? a->shape[1] : a->shape[1] * 2;
  int32_t n = b->shape[0];
  TVM_FFI_ICHECK_EQ(b->shape[1], a->shape[1]) << "Matrix dimensions don't match for multiplication";
  TVM_FFI_ICHECK(out->shape[0] == m && out->shape[1] == n) << "Output tensor has wrong dimensions";

  auto runner = flashinfer::TrtllmGenGemmRunner(flashinfer::TrtllmGenGemmRunnerOptions{
      .eltType = is_fp8 ? gemm::trtllm::gen::Dtype::E4m3 : gemm::trtllm::gen::Dtype::E2m1,
      .outputType = gemm::trtllm::gen::Dtype::Bfloat16,
      .transposeMmaOutput = true,
      .sfLayoutB = use_8x4_sf_layout ? gemm::trtllm::gen::SfLayout::R8c4
                                     : gemm::trtllm::gen::SfLayout::R128c4,
  });

  if (tactic == -1) {
    tactic = runner.selectHeuristic(m, n, k);
  }

  auto stream = get_stream(a->device);

  auto runKernel = [&](void* workspace) {
    runner.run(m, n, k, a->data, a_scale->data, b->data, b_scale->data, out->data,
               globalScale.has_value() ? globalScale.value()->data : nullptr, nullptr, workspace,
               stream, a->device.device_id, tactic);
  };

  int64_t const required_workspace_size = runner.getWorkspaceSizeInBytes(m, n, k, tactic);
  int64_t const provided_workspace_size =
      workspace_buffer.numel() * get_element_size(workspace_buffer);
  if (provided_workspace_size < required_workspace_size) {
    Tensor new_workspace = alloc_tensor({required_workspace_size}, dl_int8, a->device);
    runKernel(new_workspace->data);
  } else {
    runKernel(workspace_buffer->data);
  }
}

enum class Dtype : int64_t {
  E2m1 = 0,
  E4m3 = 1,
  Bfloat16 = 2,
};

Array<int64_t> trtllm_gemm_tactics(int64_t m, int64_t n, int64_t k, int64_t input_dtype,
                                   int64_t output_dtype, bool use_8x4_sf_layout) {
  TVM_FFI_ICHECK(input_dtype == static_cast<int64_t>(Dtype::E4m3) ||
                 input_dtype == static_cast<int64_t>(Dtype::E2m1))
      << "Unsupported input dtype";
  TVM_FFI_ICHECK_EQ(output_dtype, static_cast<int64_t>(Dtype::Bfloat16))
      << "Unsupported output dtype";

  auto runner = flashinfer::TrtllmGenGemmRunner(flashinfer::TrtllmGenGemmRunnerOptions{
      .eltType = input_dtype == static_cast<int64_t>(Dtype::E4m3) ? gemm::trtllm::gen::Dtype::E4m3
                                                                  : gemm::trtllm::gen::Dtype::E2m1,
      .outputType = gemm::trtllm::gen::Dtype::Bfloat16,
      .transposeMmaOutput = true,
      .sfLayoutB = use_8x4_sf_layout ? gemm::trtllm::gen::SfLayout::R8c4
                                     : gemm::trtllm::gen::SfLayout::R128c4,
  });

  return runner.getValidTactics(m, n, k);
}

namespace trtllm_cubin_loader {
#include <flashinfer/cubin_loader.h>
}

}  // namespace flashinfer

TVM_FFI_DLL_EXPORT_TYPED_FUNC(trtllm_gemm, flashinfer::trtllm_gemm);
TVM_FFI_DLL_EXPORT_TYPED_FUNC(trtllm_gemm_tactics, flashinfer::trtllm_gemm_tactics);
